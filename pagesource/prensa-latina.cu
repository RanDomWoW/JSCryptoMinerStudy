#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html class="no-js css-menubar" xmlns="http://www.w3.org/1999/xhtml" xml:lang="en-gb" lang="en-gb" dir="ltr">
    <head>
        <meta charset="utf-8">
        <meta http-equiv="X-UA-Compatible" content="IE=edge">
        <meta name="viewport" content="width=device-width, initial-scale=1.0, user-scalable=0, minimal-ui">
        <title>Prensa Latina - Agencia Latinoamericana de Noticias</title>        <link rel="apple-touch-icon" href="/templates/template_pl_2016/assets/images/apple-touch-icon.png">
        <link rel="shortcut icon" href="/templates/template_pl_2016/assets/images/images/favicon.ico">

        <!-- Stylesheets -->
        <link rel="stylesheet" href="/templates/template_pl_2016/global/css/bootstrap.min.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/css/bootstrap-extend.min.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/assets/css/site.min.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/css/template.css">

        <!-- Plugins -->

<!--
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/animsition/animsition.css">
-->
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/asscrollable/asScrollable.css">

<!--

        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/switchery/switchery.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/intro-js/introjs.css">
-->
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/slidepanel/slidePanel.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/flag-icon-css/flag-icon.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/waves/waves.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/assets/examples/css/dashboard/v1.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/assets/examples/css/structure/breadcrumbs.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/assets/examples/css/structure/ribbon.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/vendor/magnific-popup/magnific-popup.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/assets/examples/css/advanced/lightbox.css">

        <!-- Fonts -->
        <link rel="stylesheet" href="/templates/template_pl_2016/global/fonts/material-design/material-design.min.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/fonts/brand-icons/brand-icons.min.css">
        <link rel="stylesheet" href="/templates/template_pl_2016/global/fonts/themify/themify.css">

        <!--[if lt IE 9]>
        <script src="/templates/template_pl_2016/global/vendor/html5shiv/html5shiv.min.js"></script>
        <![endif]-->
        <!--[if lt IE 10]>
        <script src="/templates/template_pl_2016/global/vendor/media-match/media.match.min.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/respond/respond.min.js"></script>
        <![endif]-->
        <!-- Scripts -->
        <script src="/templates/template_pl_2016/global/vendor/modernizr/modernizr.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/breakpoints/breakpoints.js"></script>
        <script>
            Breakpoints();
        </script>
        <script>
            (function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
                (i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
                m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
            })(window,document,'script','https://www.google-analytics.com/analytics.js','ga');
            ga('create', 'UA-68617633-1', 'auto');
            ga('send', 'pageview');
        </script>
        <meta name="google-site-verification" content="Qdkc5NWM-rGqAs3AgO_pgjuytRmN75uLf0TVjdlXqrg" />
    </head>
    <body class="site-navbar-small dashboard" id="inicio">
    <!--[if lt IE 8]>
        <p class="browserupgrade">You are using an <strong>outdated</strong> browser. Please <a href="http://browsehappy.com/">upgrade your browser</a> to improve your experience.</p>
        <![endif]-->
        <nav class="site-navbar navbar navbar-default navbar-fixed-top navbar-mega navbar-inverse pl-nav" role="navigation">
            <div class="navbar-header pl-nav-header">
                <button type="button" class="navbar-toggle hamburger hamburger-close navbar-toggle-left hided" data-toggle="menubar">
                    <span class="sr-only">Toggle navigation</span>
                    <span class="hamburger-bar"></span>
                </button>
                <button type="button" class="navbar-toggle collapsed" data-target="#site-navbar-collapse" data-toggle="collapse">
                    <i class="icon md-more" aria-hidden="true"></i>
                </button>
                <a class="navbar-brand navbar-brand-center" href="/">
                    <img class="navbar-brand-logo navbar-brand-logo-normal" src="/templates/template_pl_2016/images/logo01.png" title="Prensa Latina">
                    <img class="navbar-brand-logo navbar-brand-logo-special" src="/templates/template_pl_2016/images/logo01.png" title="Prensa Latina">
                </a>
                <button type="button" class="navbar-toggle collapsed" data-target="#site-navbar-search" data-toggle="collapse">
                    <span class="sr-only">Toggle Search</span>
                    <i class="icon md-search" aria-hidden="true"></i>
                </button>
            </div>
            <img class="hidden-xs hidden-sm hidden-md" src="/templates/template_pl_2016/images/header_bg03.png" style="float: left;">
            <img class="hidden-xs hidden-lg hidden-xlg" src="/templates/template_pl_2016/images/header_bg04.png" style="float: left;">
            <div class="navbar-container container-fluid" >
                <!-- Navbar Collapse -->
                <div class="collapse navbar-collapse navbar-collapse-toolbar" id="site-navbar-collapse"  style="height: 120px;">
                    <!-- Navbar Toolbar -->
                    <ul class="nav navbar-toolbar">
                        <li class="hidden-float" id="toggleMenubar">
                            <a data-toggle="menubar" href="#" role="button">
                                <i class="icon hamburger hamburger-arrow-left">
                                    <span class="sr-only">Toggle menubar</span>
                                    <span class="hamburger-bar"></span>
                                </i>
                            </a>
                        </li>
                    </ul> <!-- End Navbar Toolbar -->
                    <!-- Navbar Toolbar Right -->
                    <ul class="nav navbar-toolbar navbar-right navbar-toolbar-right">
                        <li class="presentation" title="English">
                            <a href="http://www.plenglish.com" role="menuitem"><span class="flag-icon flag-icon-gb"></span></a>
                        </li>
                        <li role="presentation" title="Portuguese">
                            <a href="http://www.prensalatina.com.br" role="menuitem"><span class="flag-icon flag-icon-pt"></span></a>
                        </li>
                        <li role="presentation" title="Français">
                            <a href="http://frances.prensa-latina.cu" role="menuitem"><span class="flag-icon flag-icon-fr"></span></a>
                        </li>
                        <li role="presentation" title="Italiano">
                            <a href="http://italiano.prensa-latina.cu" role="menuitem"><span class="flag-icon flag-icon-it"></span></a>
                        </li>
                        <li role="presentation" title="Pусский">
                            <a href="http://ruso.prensa-latina.cu" role="menuitem"><span class="flag-icon flag-icon-ru"></span></a>
                        </li>
                        <li class="dropdown hidden-float">
                            <a class="icon md-search" data-toggle="collapse" href="#" data-target="#site-navbar-search" role="button">
                                <span class="sr-only">Toggle Search</span>
                            </a>
                        </li>
                        <li class="dropdown hidden-xs" id="toggleFullscreen">
                            <a class="icon icon-fullscreen" data-toggle="fullscreen" href="#" role="button">
                                <span class="sr-only">Toggle fullscreen</span>
                            </a>
                        </li>
                    </ul> <!-- End Navbar Toolbar Right -->
                    <div class="nav navbar-toolbar navbar-right navbar-toolbar-right hidden-md hidden-sm hidden-xs" style="margin-right:-315px;margin-top: 68px;font-size: 13pt;font-weight: bold;color:#455a64;">
                        S&aacute;bado 17 de marzo de 2018                    </div>
                    <div class="nav navbar-toolbar navbar-right navbar-toolbar-right hidden-lg hidden-xs" style="margin-right:-315px;margin-top: 65px;font-size: 10pt;font-weight: bold;color:#455a64;">
                        S&aacute;bado 17 de marzo de 2018                    </div>
                </div> <!-- End Navbar Collapse -->
                <!-- Site Navbar Seach -->
                <div class="collapse navbar-search-overlap" id="site-navbar-search">
                    <form role="search">
                        <div class="form-group">
                            <div class="input-search">
                                <i class="input-search-icon md-search" aria-hidden="true"></i>
                                <input id="searchBtn" type="text" class="form-control" name="site-search" placeholder="Buscar...">
                                <button type="button" class="input-search-close icon md-close" data-target="#site-navbar-search" data-toggle="collapse" aria-label="Close"></button>
                            </div>
                        </div>
                    </form>
                </div> <!-- End Site Navbar Seach -->
            </div>
        </nav>
        <br/><br/><br/>
        <div class="site-menubar" style="margin-top:60px;">
            <div class="site-menubar-body">
                <div>
                    <div>
                        <ul class="site-menu">
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Noticias</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class="is-list">
                                        <div>
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <li class="site-menu-item"><a class="animsition-link" href="/"><span class="site-menu-title">Portada</span></a></li>
                                                    <li class="divider"></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=cuba&amp;SEO=noticias-cuba"><span class="site-menu-title">Cuba</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=norteamerica&amp;SEO=noticias-cuba"><span class="site-menu-title">Norteamérica</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=centroamerica-caribe&amp;SEO=noticias-cuba"><span class="site-menu-title">Centroamérica / Caribe</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=salvador&amp;SEO=noticias-salvador"><span class="site-menu-title">El Salvador</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=sudamerica&amp;SEO=noticias-cuba"><span class="site-menu-title">Sudamérica</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=bolivia&amp;SEO=noticias-cuba"><span class="site-menu-title">Bolivia</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=europa&amp;SEO=noticias-cuba"><span class="site-menu-title">Europa</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=asia-oceania&amp;SEO=noticias-cuba"><span class="site-menu-title">Asia / Oceanía</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=africa-medio-oriente&amp;SEO=noticias-cuba"><span class="site-menu-title">África / Medio Oriente</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=deportes&amp;SEO=noticias-cuba"><span class="site-menu-title">Deportes</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=cultura&amp;SEO=noticias-cuba"><span class="site-menu-title">Cultura</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=ciencia&amp;SEO=noticias-cuba"><span class="site-menu-title">Ciencia</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=economia&amp;SEO=noticias-cuba"><span class="site-menu-title">Economía</span></a></li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Opinión</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class="site-menu-scroll-wrap is-list">
                                        <div>
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=opinion&amp;SEO=noticias-opinion"><span class="site-menu-title">Opinión</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="http://firmas.prensa-latina.cu"><span class="site-menu-title">Firmas Selectas</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="http://visiones.prensa-latina.cu"><span class="site-menu-title">Visiones Alternativas</span></a></li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="/index.php?o=vf&amp;id=fotos&amp;SEO=fotos" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Fotos</span>
                                </a>
                            </li>
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="http://radio.prensa-latina.cu" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Radio</span>
                                </a>
                            </li>
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="/index.php?o=vtv&amp;id=television&amp;SEO=television" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Televisión</span>
                                </a>
                            </li>
                            <li class="dropdown site-menu-item has-sub">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Publicaciones</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class="site-menu-scroll-wrap is-list">
                                        <div>
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <!-- inicio -->
                                                    <li class="site-menu-item has-sub">
                                                        <a href="javascript:void(0)">
                                                            <span class="site-menu-title">Orbe</span>
                                                            <span class="site-menu-arrow"></span>
                                                        </a>
                                                        <ul class="site-menu-sub">
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe&amp;SEO=publicacion-orbe"><span class="site-menu-title">&nbsp;&nbsp;&nbsp;&nbsp;Orbe</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe-mexico&amp;SEO=publicacion-orbe-edicion-mexico"><span class="site-menu-title">&nbsp;&nbsp;&nbsp;&nbsp;Edici&oacute;n M&eacute;xico</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe-bolivia&amp;SEO=publicacion-orbe-edicion-bolivia"><span class="site-menu-title">&nbsp;&nbsp;&nbsp;&nbsp;Edici&oacute;n Bolivia</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe-venezuela&amp;SEO=publicacion-orbe-edicion-venezuela"><span class="site-menu-title">&nbsp;&nbsp;&nbsp;&nbsp;Edici&oacute;n Venezuela</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe-el-salvador&amp;SEO=publicacion-orbe-edicion-el-salvador"><span class="site-menu-title">&nbsp;&nbsp;&nbsp;&nbsp;Edici&oacute;n El Salvador</span></a></li>
                                                        </ul>
                                                    </li>
                                                    <!-- fin -->



<!--                                                     <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=orbe&amp;SEO=publicacion-orbe"><span class="site-menu-title">Orbe</span></a></li>-->
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=negocios-en-cuba&amp;SEO=publicacion-negocios-en-cuba"><span class="site-menu-title">&nbsp;&nbsp;Negocios en Cuba</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=the-havana-reporter&amp;SEO=publicacion-the-havana-reporter"><span class="site-menu-title">&nbsp;&nbsp;The Havana Reporter</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=cuba-internacional&amp;SEO=publicacion-cuba-internacional"><span class="site-menu-title">&nbsp;&nbsp;Cuba Internacional</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vt&amp;id=avances-medicos&amp;SEO=publicacion-avances-medicos"><span class="site-menu-title">&nbsp;&nbsp;Avances Médicos</span></a></li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <!-- Ocultar en XS y LG -->
                            <li class="dropdown site-menu-item has-sub hidden-xs hidden-lg">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Más</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class="is-list">
                                        <div>
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <li class="site-menu-item has-sub">
                                                        <a href="javascript:void(0)">
                                                            <span class="site-menu-title">Especiales</span>
                                                            <span class="site-menu-arrow"></span>
                                                        </a>
                                                        <ul class="site-menu-sub">
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-cuba&amp;SEO=exclusivos-cuba">
                                                                    <span class="site-menu-title">Cuba</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-politica&amp;SEO=exclusivos-politica">
                                                                    <span class="site-menu-title">Política</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-economia&amp;SEO=exclusivos-economia">
                                                                    <span class="site-menu-title">Economía</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level2">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-mujeres-iii-milenio&amp;SEO=exclusivos-sociedad-mujeres-del-iii-milenio">
                                                                    <span class="site-menu-title">Mujeres del III milenio</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level2">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-infancia&amp;SEO=exclusivos-sociedad-infancia">
                                                                    <span class="site-menu-title">Infancia</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level2">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-por-la-ruta-del-esclavo&amp;SEO=exclusivos-sociedad-por-la-ruta-del-esclavo">
                                                                    <span class="site-menu-title">Por la Ruta del esclavo</span>
                                                                </a>
                                                            </li>

                                                            <li class="site-menu-item level2">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-pueblos-indigenas&amp;SEO=pueblos-indigenas">
                                                                    <span class="site-menu-title">Pueblos Indígenas</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-deportes&amp;SEO=exclusivos-deportes">
                                                                    <span class="site-menu-title">Deportes</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-cultura&amp;SEO=exclusivos-cultura">
                                                                    <span class="site-menu-title">Cultura</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item level1">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-ciencia&amp;SEO=exclusivos-ciencia">
                                                                    <span class="site-menu-title">Ciencia</span>
                                                                </a>
                                                            </li>
                                                            <li class="site-menu-item">
                                                                <a class="animsition-link" href="/index.php?o=vt&amp;id=especiales-cronicas&amp;SEO=especiales-cronicas">
                                                                    <span class="site-menu-title">Crónicas</span>
                                                                </a>
                                                            </li>
                                                        </ul>
                                                    </li>
                                                    <li class="site-menu-item has-sub">
                                                        <a href="/index.php?o=vo&amp;id=ofertas&amp;SEO=ofertas">
                                                            <span class="site-menu-title">Ofertas</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item has-sub">
                                                        <a href="javascript:void(0)">
                                                            <span class="site-menu-title">Contacto</span>
                                                            <span class="site-menu-arrow"></span>
                                                        </a>
                                                        <ul class="site-menu-sub">
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=qs&amp;id=quienessomos&amp;SEO=quienes-somos"><span class="site-menu-title">Quiénes Somos</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=ee&amp;id=emailaleditor&amp;SEO=email-al-editor"><span class="site-menu-title">Correo al Editor</span></a></li>
                                                        </ul>
                                                    </li>
                                                    <li class="site-menu-item has-sub">
                                                        <a href="javascript:void(0)">
                                                            <span class="site-menu-title">Red Social</span>
                                                            <span class="site-menu-arrow"></span>
                                                        </a>
                                                        <ul class="site-menu-sub">
                                                            <li class="site-menu-item"><a class="animsition-link" href="http://twitter.com/prensalatina_cu"><span class="site-menu-title"><i class="icon bd-twitter" aria-hidden="true"></i> Twitter</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="https://www.facebook.com/AgenciaPrensaLatina"><span class="site-menu-title"><i class="icon bd-facebook" aria-hidden="true"></i> Facebook</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="https://plus.google.com/+PrensaLatinaTVreal/about/p/pub?fd=1"><span class="site-menu-title"><i class="icon bd-google-plus" aria-hidden="true"></i> Google +</span></a></li>
                                                            <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=vr&amp;SEO=canales-rss&amp;page=1"><span class="site-menu-title"><i class="icon ti-rss-alt" aria-hidden="true"></i> Canal RSS</span></a></li>
                                                        </ul>
                                                    </li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Especiales</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class=" is-list">
                                        <div >
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-cuba&amp;SEO=exclusivos-cuba">
                                                            <span class="site-menu-title">Cuba</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-politica&amp;SEO=exclusivos-politica">
                                                            <span class="site-menu-title">Política</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-economia&amp;SEO=exclusivos-economia">
                                                            <span class="site-menu-title">Economía</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-mujeres-iii-milenio&amp;SEO=exclusivos-sociedad-mujeres-del-iii-milenio">
                                                            <span class="site-menu-title">Mujeres del III milenio</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-infancia&amp;SEO=exclusivos-sociedad-infancia">
                                                            <span class="site-menu-title">Infancia</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-por-la-ruta-del-esclavo&amp;SEO=exclusivos-sociedad-por-la-ruta-del-esclavo">
                                                            <span class="site-menu-title">Por la Ruta del esclavo</span>
                                                        </a>
                                                    </li>

                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-sociedad-pueblos-indigenas&amp;SEO=pueblos-indigenas">
                                                            <span class="site-menu-title">Pueblos Indígenas</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-deportes&amp;SEO=exclusivos-deportes">
                                                            <span class="site-menu-title">Deportes</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-cultura&amp;SEO=exclusivos-cultura">
                                                            <span class="site-menu-title">Cultura</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=exclusivos-ciencia&amp;SEO=exclusivos-ciencia">
                                                            <span class="site-menu-title">Ciencia</span>
                                                        </a>
                                                    </li>
                                                    <li class="site-menu-item">
                                                        <a class="animsition-link" href="/index.php?o=vt&amp;id=cronicas&amp;SEO=especiales-cronicas">
                                                            <span class="site-menu-title">Crónicas</span>
                                                        </a>
                                                    </li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="/index.php?o=vo&amp;id=ofertas&amp;SEO=ofertas" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Ofertas</span>
                                </a>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="javascript:void(0)" data-dropdown-toggle="false">
                                    <span class="site-menu-title">Contacto</span>
                                    <span class="site-menu-arrow"></span>
                                </a>
                                <div class="dropdown-menu">
                                    <div class="site-menu-scroll-wrap is-list">
                                        <div>
                                            <div>
                                                <ul class="site-menu-sub site-menu-normal-list">
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=qs&amp;id=quienessomos&amp;SEO=quienes-somos"><span class="site-menu-title">Quiénes Somos</span></a></li>
                                                    <li class="site-menu-item"><a class="animsition-link" href="/index.php?o=ee&amp;id=emailaleditor&amp;SEO=email-al-editor"><span class="site-menu-title">Correo al Editor</span></a></li>
                                                </ul>
                                            </div>
                                        </div>
                                    </div>
                                </div>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="http://twitter.com/prensalatina_cu" data-dropdown-toggle="false">
                                    <i class="icon bd-twitter" aria-hidden="true"></i>
                                </a>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="https://www.facebook.com/AgenciaPrensaLatina" data-dropdown-toggle="false">
                                    <i class="icon bd-facebook" aria-hidden="true"></i>
                                </a>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="https://plus.google.com/+PrensaLatinaTVreal/about/p/pub?fd=1" data-dropdown-toggle="false">
                                    <i class="icon bd-google-plus" aria-hidden="true"></i>
                                </a>
                            </li>
                            <!-- Ocultar en XS, SM, MD -->
                            <li class="dropdown site-menu-item has-sub hidden-sm hidden-md">
                                <a class="dropdown-toggle" href="/index.php?o=vr&amp;SEO=canales-rss&amp;page=1" data-dropdown-toggle="false">
                                    <i class="icon ti-rss-alt" aria-hidden="true" ></i>
                                </a>
                            </li>
                        </ul>

                    </div>
                </div>
            </div>
        </div>
        <!-- Page -->
        <div class="page animsition">
            <div class="page-content padding-20 container-fluid" style="background-color: #fff;">
                <div class="row" data-plugin="matchHeight" data-by-row="true">
                    <div class="col-xlg-7 col-md-7">
    <!-- inicio principal -->
    <div>
                    <div class="carousel slide" id="exampleCarouselCaptions" data-ride="carousel"><ol class="carousel-indicators carousel-indicators-fillin"><li class="active" data-slide-to="0" data-target="#exampleCarouselCaptions"></li><li class="" data-slide-to="1" data-target="#exampleCarouselCaptions"></li><li class="" data-slide-to="2" data-target="#exampleCarouselCaptions"></li><li class="" data-slide-to="3" data-target="#exampleCarouselCaptions"></li></ol><div class="carousel-inner" role="listbox" style="width: 100%"><div class="item active">
                                <img class='img-rounded img-responsive width-full' title='' class="width-full" src="/images/0-RAULKENYA1.jpg" alt="visita-de-presidente-kenyano-a-cuba-fortalece-nexos-bilaterales"/>
                                <div class="carousel-caption" >
                                    <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:10px; padding:0px;font-size: 20pt;"><a href="/index.php?o=rn&amp;id=161016&amp;SEO=visita-de-presidente-kenyano-a-cuba-fortalece-nexos-bilaterales">Visita de presidente kenyano a Cuba fortalece nexos bilaterales</a></h1>
                                    <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:10px;font-size: 24pt;"><a href="/index.php?o=rn&amp;id=161016&amp;SEO=visita-de-presidente-kenyano-a-cuba-fortalece-nexos-bilaterales">Visita de presidente kenyano a Cuba fortalece nexos bilaterales</a></h1>
                                    <h1 class="visible-xs" style="font-weight: bold;margin:10px; padding:0px;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=161016&amp;SEO=visita-de-presidente-kenyano-a-cuba-fortalece-nexos-bilaterales">Visita de presidente kenyano a Cuba fortalece nexos bilaterales</a></h1>
                                </div>
                            </div><div class="item">
                                <img class='img-rounded img-responsive width-full' title='' class="width-full" src="/images/2018/marzo/17/elecciones-2018.jpg" alt="rusia-el-electorado-de-las-sanciones"/>
                                <div class="carousel-caption" >
                                    <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:10px; padding:0px;font-size: 20pt;"><a href="/index.php?o=rn&amp;id=160965&amp;SEO=rusia-el-electorado-de-las-sanciones">Rusia: el electorado de las sanciones</a></h1>
                                    <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:10px;font-size: 24pt;"><a href="/index.php?o=rn&amp;id=160965&amp;SEO=rusia-el-electorado-de-las-sanciones">Rusia: el electorado de las sanciones</a></h1>
                                    <h1 class="visible-xs" style="font-weight: bold;margin:10px; padding:0px;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160965&amp;SEO=rusia-el-electorado-de-las-sanciones">Rusia: el electorado de las sanciones</a></h1>
                                </div>
                            </div><div class="item">
                                <img class='img-rounded img-responsive width-full' title='' class="width-full" src="/images/2018/marzo/17/rajoy-recortes.jpg" alt="alertan-de-estallido-social-en-espana-ante-recortes-de-rajoy"/>
                                <div class="carousel-caption" >
                                    <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:10px; padding:0px;font-size: 20pt;"><a href="/index.php?o=rn&amp;id=160950&amp;SEO=alertan-de-estallido-social-en-espana-ante-recortes-de-rajoy">Alertan de estallido social en España ante recortes de Rajoy</a></h1>
                                    <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:10px;font-size: 24pt;"><a href="/index.php?o=rn&amp;id=160950&amp;SEO=alertan-de-estallido-social-en-espana-ante-recortes-de-rajoy">Alertan de estallido social en España ante recortes de Rajoy</a></h1>
                                    <h1 class="visible-xs" style="font-weight: bold;margin:10px; padding:0px;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160950&amp;SEO=alertan-de-estallido-social-en-espana-ante-recortes-de-rajoy">Alertan de estallido social en España ante recortes de Rajoy</a></h1>
                                </div>
                            </div><div class="item">
                                <img class='img-rounded img-responsive width-full' title='' class="width-full" src="/images/2018/marzo/17/filosofo-mexicano-vene.jpg" alt="advierten-sobre-impacto-de-lo-que-pase-en-venezuela-en-la-region"/>
                                <div class="carousel-caption" >
                                    <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:10px; padding:0px;font-size: 20pt;"><a href="/index.php?o=rn&amp;id=160945&amp;SEO=advierten-sobre-impacto-de-lo-que-pase-en-venezuela-en-la-region">Advierten sobre impacto de lo que pase en Venezuela en la región</a></h1>
                                    <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:10px;font-size: 24pt;"><a href="/index.php?o=rn&amp;id=160945&amp;SEO=advierten-sobre-impacto-de-lo-que-pase-en-venezuela-en-la-region">Advierten sobre impacto de lo que pase en Venezuela en la región</a></h1>
                                    <h1 class="visible-xs" style="font-weight: bold;margin:10px; padding:0px;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160945&amp;SEO=advierten-sobre-impacto-de-lo-que-pase-en-venezuela-en-la-region">Advierten sobre impacto de lo que pase en Venezuela en la región</a></h1>
                                </div>
                            </div>        <a class="left carousel-control" href="#exampleCarouselCaptions" role="button" data-slide="prev">
                            <span class="icon md-chevron-left" aria-hidden="true"></span>
                            <span class="sr-only">Previous</span>
                        </a>
                        <a class="right carousel-control" href="#exampleCarouselCaptions" role="button" data-slide="next">
                            <span class="icon md-chevron-right" aria-hidden="true"></span>
                            <span class="sr-only">Next</span>
                        </a>
                    </div>
                </div>
            </div>
    <!-- fin principal -->



</div>
<div class="col-xlg-5 col-md-5">
    <div>
        <div class="widget " id="widgetLineareaDestacadas">
            <div class="widget-content">
                <div class="padding-0 padding-top-0">
                    <!-- inicio minuto a minuto -->
                    <table id="minutoMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td id="minuto1Title">Minuto a minuto</td>
                            <td id="minuto2Title">Más leídas</td>
                        </tr>
                    </table>
                    <br/>
                    <table id="minutoMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div id="minutoContainer1" class="list-group bg-grey-300 bg-inherit" >
                                    <a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161029&amp;SEO=documentales-compiten-en-festival-internacional-en-santiago-de-cuba">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        8:38&nbsp;&nbsp;Documentales compiten en Festival Internacional en Santiago de Cuba
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161028&amp;SEO=river-plate-unico-ganador-en-fecha-siete-del-futbol-uruguayo">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:49&nbsp;&nbsp;River Plate, único ganador en fecha siete del fútbol uruguayo
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161027&amp;SEO=preside-ecuador-comite-de-comercio-y-desarrollo-de-omc">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:19&nbsp;&nbsp;Preside Ecuador Comité de Comercio y Desarrollo de OMC
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161026&amp;SEO=marchan-en-angola-por-cambiar-proyecto-ley-repatriacion-de-capitales">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:18&nbsp;&nbsp;Marchan en Angola por cambiar proyecto ley repatriación de capitales
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161025&amp;SEO=resultados-y-posiciones-de-la-liga-italiana-de-futbol">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:17&nbsp;&nbsp;Resultados y posiciones de la liga italiana de fútbol
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161024&amp;SEO=juventus-empato-ante-el-spal-en-liga-italiana-de-futbol">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:17&nbsp;&nbsp;Juventus empató ante el SPAL en Liga italiana de fútbol
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161023&amp;SEO=resultados-y-posiciones-de-la-liga-espanola-de-futbol">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        6:16&nbsp;&nbsp;Resultados y posiciones de la liga española de fútbol
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161022&amp;SEO=valencia-gana-y-es-tercero-en-liga-espanola-de-futbol">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        5:44&nbsp;&nbsp;Valencia gana y es tercero en Liga española de fútbol
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=161021&amp;SEO=rennes-aprovecho-fecha-30-de-futbol-frances-para-ascender-en-la-tabla">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        5:37&nbsp;&nbsp;Rennes aprovechó fecha 30 de fútbol francés para ascender en la tabla
                    </a>                                    <div style="height: 50px;text-align: right;">
                                        <a href="/index.php?o=vmam&amp;id=minuto-a-minuto&amp;SEO=minuto-a-minuto"><button type="button" class="btn btn-outline btn-primary" style="margin:7px 7px 0 0;">...</button></a>
                                    </div>
                                </div>
                                <div id="minutoContainer2" class="list-group bg-grey-300 bg-inherit" >
                                    <a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160549&amp;SEO=convocan-docentes-a-paro-nacional-en-puerto-rico">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Convocan docentes a paro nacional en Puerto Rico
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160820&amp;SEO=gremio-alerta-sobre-corrupcion-con-fondos-de-educacion-en-puerto-rico">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Gremio alerta sobre corrupción con fondos de educación en Puerto Rico
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160713&amp;SEO=evento-de-negocios-dara-mayor-visibilidad-a-jamaica">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Evento de negocios dará mayor visibilidad a Jamaica
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160544&amp;SEO=todos-somos-hoy-un-poco-marielle-dice-lula-en-foro-social-mundial">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Todos somos hoy un poco Marielle, dice Lula en Foro Social Mundial
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160543&amp;SEO=gobierno-del-fmln-suma-otro-hito-en-salud-para-salvadorenos">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Gobierno del FMLN suma otro hito en salud para salvadoreños
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160812&amp;SEO=seis-proyectos-en-cuba-con-ayuda-china-por-36-millones-de-dolares">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Seis proyectos en Cuba con ayuda china por 36 millones de dólares
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160557&amp;SEO=concluye-congreso-internacional-de-bioetica-en-dominicana">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Concluye Congreso Internacional de Bioética en Dominicana
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160851&amp;SEO=cuba-expone-sus-novedades-turisticas-en-salon-mundial-de-paris-fotos">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Cuba expone sus novedades turísticas en Salón Mundial de París (+Fotos)
                    </a><a class="list-group-item grey-800 waves-effect waves-block waves-classic" href="/index.php?o=rn&amp;id=160551&amp;SEO=diputado-frente-amplio-tico-gana-recurso-amparo-contra-transgenicos">
                        <i class="icon md-chevron-right" aria-hidden="true"></i>
                        Diputado Frente Amplio tico gana recurso amparo contra transgénicos
                    </a>                                    <div style="height: 50px;text-align: right;">
                                        <a href="/index.php?o=vmreaded&amp;id=mas-leidas&amp;SEO=mas-leidas"><button type="button" class="btn btn-outline btn-primary" style="margin:7px 7px 0 0;">...</button></a>
                                    </div>
                                </div>
                            </td>
                        </tr>
                    </table>
                    <!-- fin minuto a minuto -->
                </div>
            </div>
        </div>
    </div>
    <br/>
</div>
<div class="clearfix"></div>
<!-- *****************
        Contenedor

-->
<div class="col-lg-4 col-md-4">
    <div >
        <div class="widget " id="widgetLineareaCuba">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- Inicio cuba -->
                    <table id="cubaMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr> <!-- alex -->
                            <td id="cuba1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=cuba&SEO=noticias-cuba">Cuba</a></td>
                            <td id="cuba2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="cubaMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="cubaBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=161015&amp;SEO=abren-en-cuba-primer-mercado-mayorista-de-alimentos">Abren en Cuba primer mercado mayorista de alimentos</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=161015&amp;SEO=abren-en-cuba-primer-mercado-mayorista-de-alimentos">Abren en Cuba primer mercado mayorista de alimentos</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=161015&amp;SEO=abren-en-cuba-primer-mercado-mayorista-de-alimentos">Abren en Cuba primer mercado mayorista de alimentos</a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/0-cuba-mercado.jpg" alt="" />La Habana, 17 mar (PL) El primer mercado mayorista de alimentos (Mercabal) entró en operaciones hoy aquí, como parte del proceso de reordenamiento del comercio, informó una fuente oficial.</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- fin cuba -->
                </div>
            </div>
        </div>
    </div>
    <div >
        <div class="widget " id="widgetLineareaDeportes">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- inicio Deportes -->
                    <table id="deportesMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr> <!-- alex -->
                            <td id="deportes1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=deportes&SEO=noticias-deportes">Deportes</a></td>
                            <td id="deportes2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="deportesMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="deportesBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160993&amp;SEO=presidente-de-fifa-elogia-seguridad-de-rusia-para-mundial-de-futbol">Presidente de FIFA elogia seguridad de Rusia para Mundial de Fútbol</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160993&amp;SEO=presidente-de-fifa-elogia-seguridad-de-rusia-para-mundial-de-futbol">Presidente de FIFA elogia seguridad de Rusia para Mundial de Fútbol</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160993&amp;SEO=presidente-de-fifa-elogia-seguridad-de-rusia-para-mundial-de-futbol">Presidente de FIFA elogia seguridad de Rusia para Mundial de Fútbol</a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/2018/marzo/17/pdte-fifa.jpg" alt="" />Panamá, 17 mar (PL) El presidente de la Federación Internacional de Fútbol Asociado (FIFA), Gianni Infantino, elogió hoy el nivel de seguridad que presenta Rusia con vistas al Mundial de Fútbol que tendrá lugar en ese país europeo.</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- fin Deportes -->
                </div>
            </div>
        </div>
    </div>
    <div >
        <div class="widget " id="widgetLineareaCiencia">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- inicio Ciencia -->
                    <table id="cienciaMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr> <!-- alex -->
                            <td id="ciencia1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=ciencia&SEO=noticias-ciencia">Ciencia</a></td>
                            <td id="ciencia2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="cienciaMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="cienciaBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160973&amp;SEO=facebook-suspende-a-firma-vinculada-a-trump-por-filtrar-datos">Facebook suspende a firma vinculada a Trump por filtrar datos </a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160973&amp;SEO=facebook-suspende-a-firma-vinculada-a-trump-por-filtrar-datos">Facebook suspende a firma vinculada a Trump por filtrar datos </a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160973&amp;SEO=facebook-suspende-a-firma-vinculada-a-trump-por-filtrar-datos">Facebook suspende a firma vinculada a Trump por filtrar datos </a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/2018/marzo/17/norte-cambridge.jpg" alt="" />Washington, 17 mar (PL) Facebook suspendió la cuenta de la firma Cambridge Analytica, vinculada a la campaña presidencial de Donald Trump, por recolectar información de más de 50 millones de usuarios sin su permiso, destaca hoy la prensa.</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- fin Ciencia -->
                </div>
            </div>
        </div>
    </div>
    <div >
        <div class="widget " id="widgetLineareaCiencia">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- inicio Television -->
                    <table id="televisionMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td id="television1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vtv&id=television&SEO=television">Televisión</a></td>
                            <td id="television2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="televisionMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div >
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a class="popup-youtube" href="https://www.youtube.com/watch?v=z8Y9UPJVQIY">Más de 70 por ciento de los electores apuestan por reelección de Putin</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a class="popup-youtube" href="https://www.youtube.com/watch?v=z8Y9UPJVQIY">Más de 70 por ciento de los electores apuestan por reelección de Putin</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a class="popup-youtube" href="https://www.youtube.com/watch?v=z8Y9UPJVQIY">Más de 70 por ciento de los electores apuestan por reelección de Putin</a></h1>
                        </div>
                        <div >
                            <a class="popup-youtube" href="https://www.youtube.com/watch?v=z8Y9UPJVQIY"><img class='img-rounded img-responsive' class="img-responsive img-rounded" src="/images/2018/marzo/17/TV-CANDIDATOS-RUSIA.jpg" alt="mas-de-70-por-ciento-de-los-electores-apuestan-por-reeleccion-de-putin"/></a>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- fin Television -->
                </div>
            </div>
        </div>
    </div>
</div>
<div class="col-lg-4 col-md-4">
    <div >
        <div class="widget " id="widgetLineareaOpinion">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- Inicio Opinion -->
                    <table id="opinionMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr> <!-- alex -->
                            <td id="opinion1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=opinion&SEO=noticias-opinion">Opinión</a></td>
                            <td id="opinion2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="opinionMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="opinionBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160953&amp;SEO=acusacion-de-londres-a-moscu-con-resultado-tres-en-uno">Acusación de Londres a Moscú con resultado tres en uno</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160953&amp;SEO=acusacion-de-londres-a-moscu-con-resultado-tres-en-uno">Acusación de Londres a Moscú con resultado tres en uno</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160953&amp;SEO=acusacion-de-londres-a-moscu-con-resultado-tres-en-uno">Acusación de Londres a Moscú con resultado tres en uno</a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/2018/marzo/17/londres-moscu.jpg" alt="" /><span class="autor">Por Antonio Rondón *</span><br /><br />Moscú (PL) La acusación de Londres a Moscú por uso de armas químicas contra el exagente doble Serguei Skripal pudiera llevar a una crisis diplomática, el refuerzo de la OTAN en la frontera rusa y ataques en Siria.</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- Fin Opinion -->
                </div>
            </div>
        </div>
    </div>
    <div >
        <div class="widget " id="widgetLineareaCultura">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- Inicio Cultura -->
                    <table id="culturaMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td id="cultura1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=cultura&SEO=noticias-cultura">Cultura</a></td>
                            <td id="cultura2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="culturaMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="culturaBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160988&amp;SEO=compania-salvadorena-irreal-teatro-actuara-en-cuba">Compañía salvadoreña Irreal Teatro actuará en Cuba</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160988&amp;SEO=compania-salvadorena-irreal-teatro-actuara-en-cuba">Compañía salvadoreña Irreal Teatro actuará en Cuba</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160988&amp;SEO=compania-salvadorena-irreal-teatro-actuara-en-cuba">Compañía salvadoreña Irreal Teatro actuará en Cuba</a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/2018/marzo/17/irreal-teatro.jpg" alt="" />San Salvador, 17 mar (PL) La compañía salvadoreña Irreal Teatro presentará en Cuba sus obras ''Tuber in Love'' y ''Clowntivante'', durante el IV Taller Internacional de Payasos, del 20 al 25 de marzo próximos.</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- Fin Cultura -->
                </div>
            </div>
        </div>
    </div>
    <div >
        <div class="widget " id="widgetLineareaEconomia">
            <div class="widget-content">
                <div class="padding-0 padding-top-10">
                    <!-- Inicio Economia -->
                    <table id="economiaMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td id="economia1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=economia&SEO=noticias-economia">Economía</a></td>
                            <td id="economia2Title" class="formato2Seccion"></td>
                        </tr>
                    </table>
                    <br/>
                    <table id="economiaMainContainerTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                        <tr>
                            <td valign="top" colspan="2" >
                                <div class="economiaBlock">
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=161020&amp;SEO=nicaragua-organiza-feria-internacional-de-turismo-2018">Nicaragua organiza Feria Internacional de Turismo 2018</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=161020&amp;SEO=nicaragua-organiza-feria-internacional-de-turismo-2018">Nicaragua organiza Feria Internacional de Turismo 2018</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=161020&amp;SEO=nicaragua-organiza-feria-internacional-de-turismo-2018">Nicaragua organiza Feria Internacional de Turismo 2018</a></h1>
                        </div>
                        <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <p><img class='img-rounded img-responsive' title='' src="/images/0-nicaraguaturismo.jpg" alt="" />Managua, 17 mar (PL) Nicaragua organiza la VII Feria Internacional de Turismo, en la que participarán al menos 50 empresas mayoristas entre el 8 y 9 de junio, informó hoy el Instituto nacional de la rama (Intur).</p>

                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                            </td>
                        </tr>
                    </table>
                    <!-- Fin Economia -->
                </div>
            </div>
        </div>
    </div>
</div>
<div class="col-lg-4 col-md-4">
    <div class="widget" id="widgetLineareaMinuto">
        <div class="widget-content">
            <div class="padding-0 padding-top-10">
                <!-- ?php
                    // Cosa especial para el aniversario 50 del che
                    echo $plApp->getMainNewsFromTagFidel90('che50',0,'principal');
                ?> -->
                <!-- Dossier de Che -->
                <!-- div style="text-align: center;width:100%;">
                    <a target="_blank" href="/../media/dossier/Dossier_Che/IndexChe.html"><img class="banners img-rounded" src="/templates/template_pl_2016/images/Banner-che.jpg"/></a></div>
                <br/><br/> -->
                <!-- Dossier de Fidel -->
                <div style="text-align: center;width:100%;">
                    <a target="_blank" href="/../media/dossier/fidel1.html"><img class="banners img-rounded" src="/templates/template_pl_2016/images/Banner-llamado-fidel.jpg"/></a>                </div>
                <br/><br/>
                <!-- inicio destacadas -->
                <table id="destacadasMainTable" class="pl-minuto-table" cellspacing="0" cellpadding="0" border="0">
                    <tr>
                        <td id="destacadas1Title" class="formato1Seccion"><a class="linkblanco" href="/index.php?o=vt&id=destacadas&SEO=noticias-destacadas">Destacadas</a></td>
                        <td id="destacadas2Title" class="formato2Seccion"></td>
                    </tr>
                </table>
                <br/>
                <div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/17/Kuczynski.jpg" alt="decepcion-por-explicaciones-agrava-situacion-de-presidente-peruano" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160980&amp;SEO=decepcion-por-explicaciones-agrava-situacion-de-presidente-peruano">Decepción por explicaciones agrava situación de presidente peruano</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160980&amp;SEO=decepcion-por-explicaciones-agrava-situacion-de-presidente-peruano">Decepción por explicaciones agrava situación de presidente peruano</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160980&amp;SEO=decepcion-por-explicaciones-agrava-situacion-de-presidente-peruano">Decepción por explicaciones agrava situación de presidente peruano</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/><div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/17/dominicana-Mitchell.jpg" alt="primer-ministro-keith-mitchell-juramenta-nuevo-periodo-de-gobierno" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160916&amp;SEO=primer-ministro-keith-mitchell-juramenta-nuevo-periodo-de-gobierno">Primer Ministro Keith Mitchell juramenta nuevo período de gobierno</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160916&amp;SEO=primer-ministro-keith-mitchell-juramenta-nuevo-periodo-de-gobierno">Primer Ministro Keith Mitchell juramenta nuevo período de gobierno</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160916&amp;SEO=primer-ministro-keith-mitchell-juramenta-nuevo-periodo-de-gobierno">Primer Ministro Keith Mitchell juramenta nuevo período de gobierno</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/><div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/16/fragata.jpg" alt="zarpa-de-buenos-aires-fragata-ara-libertad-rumbo-a-12-naciones" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160915&amp;SEO=zarpa-de-buenos-aires-fragata-ara-libertad-rumbo-a-12-naciones">Zarpa de Buenos Aires fragata ARA Libertad rumbo a 12 naciones</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160915&amp;SEO=zarpa-de-buenos-aires-fragata-ara-libertad-rumbo-a-12-naciones">Zarpa de Buenos Aires fragata ARA Libertad rumbo a 12 naciones</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160915&amp;SEO=zarpa-de-buenos-aires-fragata-ara-libertad-rumbo-a-12-naciones">Zarpa de Buenos Aires fragata ARA Libertad rumbo a 12 naciones</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/><div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/17/haiti1.jpg" alt="en-haiti-todos-los-ninos-cuentan-fotos" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160884&amp;SEO=en-haiti-todos-los-ninos-cuentan-fotos">En Haití todos los niños cuentan (+Fotos)</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160884&amp;SEO=en-haiti-todos-los-ninos-cuentan-fotos">En Haití todos los niños cuentan (+Fotos)</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160884&amp;SEO=en-haiti-todos-los-ninos-cuentan-fotos">En Haití todos los niños cuentan (+Fotos)</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/><div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/17/brasilena-muerte.jpg" alt="frente-amplio-uruguayo-lamenta-muerte-de-brasilena-marielle-franco" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160882&amp;SEO=frente-amplio-uruguayo-lamenta-muerte-de-brasilena-marielle-franco">Frente Amplio uruguayo lamenta muerte de brasileña Marielle Franco</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160882&amp;SEO=frente-amplio-uruguayo-lamenta-muerte-de-brasilena-marielle-franco">Frente Amplio uruguayo lamenta muerte de brasileña Marielle Franco</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160882&amp;SEO=frente-amplio-uruguayo-lamenta-muerte-de-brasilena-marielle-franco">Frente Amplio uruguayo lamenta muerte de brasileña Marielle Franco</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/><div class="col-xlg-12 col-md-12 col-sm-12 col-xs-12">
                        <div class="col-xlg-5 col-md-5 col-sm-5 col-xs-5">
                            <img class="img-responsive img-rounded" src="/images/2018/marzo/17/zuma-anc.jpg" alt="asociacion-de-veteranos-del-anc-duda-que-zuma-reciba-un-juicio-justo" title="" />
                        </div>
                        <div class="col-xlg-7 col-md-7 col-sm-7 col-xs-7">
                            <h1 class="hidden-xs hidden-sm hidden-md" style="font-weight:bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160881&amp;SEO=asociacion-de-veteranos-del-anc-duda-que-zuma-reciba-un-juicio-justo">Asociación de veteranos del ANC duda que Zuma reciba un juicio justo</a></h1>
                            <h1 class="visible-sm visible-md" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 14pt;"><a href="/index.php?o=rn&amp;id=160881&amp;SEO=asociacion-de-veteranos-del-anc-duda-que-zuma-reciba-un-juicio-justo">Asociación de veteranos del ANC duda que Zuma reciba un juicio justo</a></h1>
                            <h1 class="visible-xs" style="font-weight: bold;margin:0px; padding:0px;padding-bottom: 20pt;font-size: 12pt;"><a href="/index.php?o=rn&amp;id=160881&amp;SEO=asociacion-de-veteranos-del-anc-duda-que-zuma-reciba-un-juicio-justo">Asociación de veteranos del ANC duda que Zuma reciba un juicio justo</a></h1>
                        </div>
                    </div>
                    <div class="clearfix"></div>
                    <br/>                <hr style="border: solid 1px;"/>
                <br/>
                <!-- fin destacadas -->
                <!-- Inicio Banners -->
                <div style="text-align: center;">
                    <a href="https://lahoradcuba.wordpress.com/" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/bannerLaHoraDeCuba.jpg" title="La Hora de Cuba"/></a>
                    <br/><br/>
                    <a href="/publicaciones/anuncios/Suplemento_granma.pdf" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/BannerCubaVsBloqueo2016.jpg" title="Prensa Latina"/></a>
                    <br/><br/>
                    <a href="http://firmas.prensa-latina.cu" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/BannerFirmas.jpg" title="Prensa Latina"/></a>
                    <br/><br/>
                    <a href="http://plbolivia.bo/" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/bannerPlBolivia.jpg" title="Prensa Latina Bolivia"/></a>
                    <br/><br/>
                    <a href="/index.php?o=sms&amp;SEO=servicio-SMS-de-Prensa-Latina" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/BannerSMSmovil.jpg" title="Servicio SMS de Prensa Latina"/></a>
                    <br/><br/>
                    <img class="banners img-rounded" src="/templates/template_pl_2016/images/BazarVirtual.jpg" title="Prensa Latina"/>
                    <br/><br/>
                    <a href="http://www.mintrabajo.gob.bo" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/BannerBolivia.jpg" title="Ministerio de Trabajo, Empleo y Previsi&oacute;n Social de Bolivia"/></a>
                    <br/><br/>
                    <a href="http://promociondeeventos.sld.cu/saludparatodos2018/" target="_blank"><img class="banners img-rounded" src="/templates/template_pl_2016/images/BannerSaludTodos2018.jpg" title="XIV Feria Internacional Salud para Todos 2018"/></a>
                </div>
                <!-- End Banners -->
            </div>
        </div>
    </div>
</div>
<div class="clearfix"></div>
<div class="footer col-lg-12 col-md-12" >
    <div class="footerImg">
        <a href="http://espanol.almayadeen.net/" target="_blank"><img src="/templates/template_pl_2016/images/al-mayadeen-logo.jpg" alt="Al Mayadeen"/></a>
        <a href="http://www.hispantv.com/" target="_blank"><img src="/templates/template_pl_2016/images/hispantv.png" alt="HISPANTV"/></a>
        <a href="http://www.avn.info.ve/" target="_blank"><img src="/templates/template_pl_2016/images/avn-logo.png" alt="AVN"/></a>
	    <a href="http://www.abi.bo/" target="_blank"><img src="/templates/template_pl_2016/images/abi-logo.png" alt="ABI"/></a>
        <a href="http://spanish.xinhuanet.com/" target="_blank"><img src="/templates/template_pl_2016/images/xinhua-logo.png" alt="Xinhua"/></a>
        <a href="http://www.fidelcastro.cu/" target="_blank"><img src="/templates/template_pl_2016/images/fidel-logo.png" alt="Fidel, soldado de las ideas"/></a>
        <a href="http://news.vnanet.vn/?LangID=4" target="_blank"><img src="/templates/template_pl_2016/images/vna-logo.png" alt="VNA"/></a>
        <a href="http://www.lavozdelsandinismo.com/" target="_blank"><img src="/templates/template_pl_2016/images/voz-sandinismo-logo.png" alt="La voz del sandinismo"/></a>
    </div>
</div>
                </div>
            </div>
        </div><!-- End Page -->


        <!-- Footer -->
        <footer class="site-footer" style="text-align: center;">
            <div>Copyright &copy; 2016 <a href="http://www.prensa-latina.cu">Agencia Informativa Latinoamericana Prensa Latina</a></div>
            <div>
                Calle E # 454 esq 19, Vedado,  La Habana, Cuba.<br/>
                Tel&eacute;f: (+53) 7 838 3496, (+53) 7 838 3497, (+53) 7 838 3498, (+53) 7 838 3499
            </div>
        </footer>

        <!-- Modal escuchar audio -->
        <div class="modal modal-info fade" id="audioListening" aria-hidden="true" aria-labelledby="audioListening" role="dialog" tabindex="-1">
            <div class="modal-dialog modal-center">
                <div class="modal-content">
                    <div class="modal-header">
                        <button id="audioListening-close-btn" type="button" class="close" data-dismiss="modal" aria-label="Close">
                            <span aria-hidden="true">×</span>
                        </button>
                        <h4 class="modal-title">Escuchar Audio</h4>
                    </div>
                    <div class="modal-body">
                        <div id="cc-rated-video" name="cc-rated-video" style="height: 70px;">
                            <div class="bs-example" data-example-id="responsive-embed-16by9-iframe-youtube">
                                <div class="embed-responsive embed-responsive-16by9">
                                    <iframe id="iframeAudio" name="iframeAudio" class="embed-responsive-item" src="/" width="100%" allowfullscreen style="border:none;height:50px;"></iframe>
                                </div>
                            </div>
                        </div>
                    </div>
                </div>
            </div>
        </div>
        <!-- /Modal escuchar audio -->

        <!-- Core  -->
        <script src="/templates/template_pl_2016/global/vendor/jquery/jquery.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/bootstrap/bootstrap.js"></script>

<!--        <script src="/templates/template_pl_2016/global/vendor/animsition/animsition.js"></script>
-->
        <script src="/templates/template_pl_2016/global/vendor/asscroll/jquery-asScroll.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/mousewheel/jquery.mousewheel.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/asscrollable/jquery.asScrollable.all.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/ashoverscroll/jquery-asHoverScroll.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/waves/waves.js"></script>

    <!--

        <!-- Plugins
        <script src="/templates/template_pl_2016/global/vendor/switchery/switchery.min.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/intro-js/intro.js"></script>
-->
        <script src="/templates/template_pl_2016/global/vendor/screenfull/screenfull.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/slidepanel/jquery-slidePanel.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/chartist-plugin-tooltip/chartist-plugin-tooltip.min.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/jvectormap/maps/jquery-jvectormap-world-mill-en.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/matchheight/jquery.matchHeight-min.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/asbreadcrumbs/jquery-asBreadcrumbs.min.js"></script>
        <script src="/templates/template_pl_2016/global/vendor/magnific-popup/jquery.magnific-popup.js"></script>

        <!-- Scripts -->
        <script src="/templates/template_pl_2016/global/js/core.js"></script>
        <script src="/templates/template_pl_2016/assets/js/site.js"></script>
        <script src="/templates/template_pl_2016/assets/js/sections/menu.js"></script>
        <script src="/templates/template_pl_2016/assets/js/sections/menubar.js"></script>
        <script src="/templates/template_pl_2016/assets/js/sections/sidebar.js"></script>
        <script src="/templates/template_pl_2016/global/js/configs/config-colors.js"></script>
        <script src="/templates/template_pl_2016/assets/js/configs/config-tour.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/asscrollable.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/animsition.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/slidepanel.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/switchery.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/tabs.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/matchheight.js"></script>
        <script src="/templates/template_pl_2016/assets/examples/js/dashboard/v1.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/asbreadcrumbs.js"></script>
        <script src="/templates/template_pl_2016/global/js/components/magnific-popup.js"></script>
        <script src="/templates/template_pl_2016/assets/examples/js/advanced/lightbox.js"></script>

        <script>

            $( "#audioListening-close-btn").click(function() {  $('#iframeAudio').attr("src",'');  });
            $( '#audioListening').on('hidden.bs.modal', function () {
                $('#iframeAudio').attr("src",'');
            })

            // Defaults
            $( "#minuto1Title").addClass("minutoTitleActive");
            $( "#minuto1Title").removeClass("minuto1TitleInactive");
            $( "#minuto2Title").removeClass("minutoTitleActive");
            $( "#minuto2Title").addClass("minuto2TitleInactive");
            $( "#minutoContainer1").show();
            $( "#minutoContainer2").hide();

            // click en Minuto a minuto
            $( "#minuto1Title").click(function() {
                $( "#minuto1Title").addClass("minutoTitleActive");
                $( "#minuto1Title").removeClass("minuto1TitleInactive");
                $( "#minuto2Title").removeClass("minutoTitleActive");
                $( "#minuto2Title").addClass("minuto2TitleInactive");

                $( "#minutoContainer1").show();
                $( "#minutoContainer2").hide();

            });

            // click en Más leídas
            $( "#minuto2Title").click(function() {
                $( "#minuto2Title").addClass("minutoTitleActive");
                $( "#minuto2Title").removeClass("minuto2TitleInactive");
                $( "#minuto1Title").removeClass("minutoTitleActive");
                $( "#minuto1Title").addClass("minuto1TitleInactive");

                $( "#minutoContainer2").show();
                $( "#minutoContainer1").hide();

            });
            //boton busqueda
            $( "#searchBtn" ).keypress(function(e) {if(e.which == 13) {
                $.ajax({type: "POST", async: false, url: '/templates/template_pl_2016/includes/pl_actions.php', data: {
                    'o':    'setSearchResult',
                    'id': $.trim($( "#searchBtn" ).val())
                }
                }).done(function( valor ) {
                    window.location.href = "index.php?o=sr&id="+valor;
                })
                return false;
            }});

            // email al Editor
            $( "#sendEmailToEditor").click(function() {
                $.ajax({type: "POST", async: false, url: '/templates/template_pl_2016/includes/pl_actions.php', data: {
                    'o'         : 'setSendEmailToEditor',
                    'sitio'     : ' Español ',
                    'nombre'    : $.trim($( "#emailToEditor-name" ).val()),
                    'email'     : $.trim($( "#emailToEditor-email" ).val()),
                    'mensaje'   : $.trim($( "#emailToEditor-subject" ).val())
                }
                }).done(function( valor ) {
                    alert('Mensaje enviado al Editor...');
                })
            });

            $( "#print01" ).click(function() { PrintElem('#newsContainer'); });

            function PrintElem(elem){Popup($(elem).html());}

            function Popup(data){
                var mywindow = window.open('', 'my div', 'height=600,width=800,left=200');

                mywindow.document.write("<html><head><title>Imprimir Noticia</title>");
                mywindow.document.write('</head><body ><h1 style="font-size:20pt;">Prensa Latina</h1><br/>');
                mywindow.document.write(data);
                mywindow.document.write('<hr/><i><label style="font-size:10pt;font-weight:normal;">Fecha de impresión: 17-03-2018</label></i><br/></body></html>');

                mywindow.document.close(); // necessary for IE >= 10
                mywindow.focus(); // necessary for IE >= 10

                mywindow.print();
                mywindow.close();

                return true;
            }

            //escuchar audio
            function audioPlay(id){
                //$('#iframeAudio').attr("src",'NOTA-6_30.mp3');
                //$('#iframeAudio').attr("src",'http://radiopl.prensa-latina.cu/media/k2/attachments/NOTA-6_30.mp3');
                //http://radiopl.prensa-latina.cu/media/k2/attachments/NOTA-6_30.mp3
            }

        </script>

    <div class="modalLoader"></div>

    </body>
</html>