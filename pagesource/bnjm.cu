#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html xmlns="http://www.w3.org/1999/xhtml" xmlns:og="http://opengraphprotocol.org/schema/" >
    <head >
        <meta name="HandheldFriendly" content="True">
        <meta name="MobileOptimized" content="320">
        <meta name="autor" value ="Reinier Torres Fleites">
        <meta name="soporte" value ="Dayaris Flores, Angelo Ulacia Rey">
        <meta name="viewport" content="width=device-width, initial-scale=1.0">
        		<title>Biblioteca Nacional de Cuba José Martí</title>
		<meta name="keywords" content="bnjm, portal, snbp, ascubi, biblioteca, bibliotecarios" />
<meta name="description" content="Portal de la Biblioteca Nacional de Cuba Jos&eacute; Mart&iacute; donde encontrar&aacute;s informaci&oacute;n acerca de los servicios que ofrece, las colecciones que atesora asi como del Sistema Nacional de Bibliotecas p&uacute;blicas del pa&iacute;s" />
<meta name="robots" content="index, follow" />
<meta property="og:site_name" content="Portal de la Biblioteca Nacional de Cuba Jos&eacute; Mart&iacute;" />
<meta property="og:description" content="Portal de la Biblioteca Nacional de Cuba Jos&eacute; Mart&iacute; donde encontrar&aacute;s informaci&oacute;n acerca de los servicios que ofrece, las colecciones que atesora asi como del Sistema Nacional de Bibliotecas p&uacute;blicas del pa&iacute;s" />
<meta http-equiv="Content-Type" content="text/html; charset=utf-8" />

		
		

        <link rel="shortcut icon" href=" http://localhost/sfbnjm2.0/web/favicon.ico">
                            <!-- Bootstrap --><link href="/media/css/bootstrap.css" rel="stylesheet"><link href="/media/css/font-awesome.css" rel="stylesheet"><!-- css internos de la web que sobreescribe algunas clasess del bootstrap --><link href="/media/css/style.css" rel="stylesheet">            </head>
    <body>
                    <div class="container">
                <div id="header" class="row">
            <div class="col-md-3 col-sm-3" style="margin-left: 0px !important; padding-left: 0px !important;">
    <a href="/"><img src="/media/images/logo.png" class="img-responsive" style="max-width: 210px;"/></a>
</div>
<div class="col-md-7 pull-right col-xs-12 col-sm-9">
    <div class="row">
        <div class="col-md-5 pull-right">
            		</div>
    </div>
    <div class="row" style="margin-top: 20px;">
        <div class="col-md-12">
			<form name="bnjm_search" method="get" action="/search-post" class="form-inline">

	<div class="form-group col-md-1  ">	</div>
	<div class="form-group col-md-5 col-xs-12 col-sm-5">
		<input type="text" id="bnjm_search_search" name="bnjm_search[search]" required="required" class="form-control head_search" placeholder="Busque libros, ebooks, publicaciones ..." />
	</div>
	<div class="form-group col-md-4 col-xs-12 col-sm-4">
		<select id="bnjm_search_where" name="bnjm_search[where]" class="form-control">
        <option value="portal">Portal</option><option value="catalogo general">Catálogo General</option><option value="catalogo_digitalizado">Catálogo Digitalizado</option><option value="indice publicaciones cubanas">Índice Publicaciones Cubanas</option><option value="analiticas publicaciones extranjeras">Analíticas Publicaciones Extranjeras</option></select>
	</div>
	<div class="form-group col-md-2 col-xs-12 col-sm-2">
		<button type="submit" class="btn btn-danger pull-right"> <span class="glyphicon glyphicon-search"></span>Buscar</button>
	</div>
<input type="hidden" id="bnjm_search__token" name="bnjm_search[_token]" value="GHHVOHoyz8t7oR2__NV7IXhgr4xTZXGQleC4mVi2Kac" /></form>


        </div>
    </div>
    <div class="row">
        <div class="col-md-5 pull-right">
                    </div>
    </div>

</div>



        </div>
                <div id="menu" class="row" style="margin-top: 10px;">
            <nav class="navbar navbar-default navbar-static">
	<div class="navbar-header">
		<button class="navbar-toggle" type="button" data-toggle="collapse" data-target=".js-navbar-collapse">
			<span class="sr-only">Toggle navigation</span>
			<span class="icon-bar"></span>
			<span class="icon-bar"></span>
			<span class="icon-bar"></span>
		</button>
		<a class="navbar-brand" href="#"></a>
	</div>
	<div class="collapse navbar-collapse js-navbar-collapse">
		<ul class="nav navbar-nav">
			<li class="dropdown dropdown-large">
				<a href="/" class="dropdown-toggle" data-toggle="dropdown">INICIO</a>
				<div class="active_menu"></div>
								
<ul class="dropdown-menu dropdown-menu-large row">
	<li class="col-sm-8 col-md-7">
		<div class="row">
			<div class="col-md-3">
				<img src="/media/images/marti.jpg" alt="marti" class="img-responsive" style="max-width: 100px;">
			</div>
			<div class="col-lg-8">
				<blockquote>
					<p>Los Hombres son como los astros, que unos dan luz de sí y otros brillan con la que reciben.</p>
					<small>Nuestro apóstol <cite title="Source Title">José Martí</cite></small>
				</blockquote>
			</div>
		</div>
		<div class="row">
			<div class="col-md-12" style="color: #fed899;">
				<p><strong>Contactenos a través de: </strong> informacion@bnjm.cu, telefono: 7855-54-42 al 49</p>
				<address>
					Ave. Independencia #257 e/ Aranguren y 20 de Mayo, Plaza de la Revolución  C. P.  6670, La Habana, Cuba
				</address>
			</div>
		</div>
	</li>
	<li class="col-sm-4 col-md-5" style="color: #fed899;">
		<p style="text-align:justify"><span style="font-size:11px"><span style="font-family:verdana,geneva,sans-serif"><small><strong>Horarios de atenci&oacute;n al p&uacute;blico:</strong><br />
Lunes a Viernes: 8:15 am - 5:15 pm<br />
S&aacute;bados laborables: 8:15 am - 4:15 pm (Salas Especializadas y Colecci&oacute;n Cubana, 3er. piso)<br />
S&aacute;bados no laborables: 8:15 am - 4:15 pm (Sala General, 1er. piso)<br />
<br />
<strong>Sala Circulante Mar&iacute;a Teresa Freyre de Andrade</strong><br />
Lunes a Viernes: 9:15 am - 5:15 pm<br />
S&aacute;bados: 8:15 am - 4:15 pm<br />
<br />
<strong>Galer&iacute;a El Reino de este mundo</strong><br />
Lunes a s&aacute;bado: 9:00 am - 5:00 pm </small></span></span></p>
	</li>
</ul>


			</li>
			<li class="dropdown dropdown-large">
				<a href="#" class="dropdown-toggle" data-toggle="dropdown">LA BIBLIOTECA</a>
				<div class="active_menu"></div>
				<ul class="dropdown-menu dropdown-menu-large row">
	<li class="col-sm-3">
		<ul>
			<li class="submenuBar-header"><a href="/la-biblioteca/conocenos/">Conócenos</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/servicios/">Servicios</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/consulta-en-salas/">Consulta en salas</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/actividades/">Actividades</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/publicaciones/">Publicaciones</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/bibliotecas-digitales/">Bibliotecas Digitales</a></li>
			<li class="submenuBar-header"><a href="http://portal.bnjm.cu/la-biblioteca/servicios/otros-servicios/s/146">Bibliografía Cubana</a></li>
		</ul>
	</li>
	<li class="col-sm-3">
		<ul>
			<li class="submenuBar-header"><a href="/la-biblioteca/recursos-para-bibliotecarios/">Recursos para bibliotecarios</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/preguntas-al-bibliotecario/">Preguntas al bibliotecario</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/suscripcion/">Suscripción</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/noticias/">Noticias</a></li>
			<li class="submenuBar-header"><a href="/la-biblioteca/contacto/">Contacto</a></li>
		</ul>
	</li>

	<li class="col-sm-6">
		<ul>
			<li class="dropdown-header">Use la biblioteca</li>
		</ul>
		<div class="col-md-6">
			<p><strong>Investigadores</strong><br />
				<a href="/la-biblioteca/recursos-para-bibliotecarios/investigaciones-cientificas/investigadores"><i>Servicios científicos a investigadores y especialistas</i></a>
			</p>
			<p><strong>Editoriales y autores</strong><br />
				<a href="/colecciones/desarrollo-de-colecciones/deposito-legal"><i>Depósito Legal</i></a>
			</p>
			<p><strong>Información de interés</strong><br />
				<i><a href="https://bncjmblog.blogspot.com/">El Blog de la Biblioteca Nacional</a></i>
				<i><a href="/la-biblioteca/recursos-para-bibliotecarios/superacion-y-especializacion-bibliotecaria/cursos">Conferencias y clases en la Biblioteca Nacional</a></i>
				<i><a href="/la-biblioteca/la-biblioteca-por-dentro">La Biblioteca por dentro</a></i>
				<i><a href="/la-biblioteca/amigos-de-la-biblioteca/">Amigos de la Biblioteca</a></i>
			</p>
		</div>
		<div class="col-md-6">
			<p><strong>Escuelas y Universidades</strong><br />
				<a href="/la-biblioteca/soporte-escuela-universidades"><i>Soporte a escuelas y universidades</i></a>
			</p>
			<p><strong>Soporte a instituciones</strong><br />
				<a href="/la-biblioteca/servicios/otros-servicios/servicio-especializado"><i>Servicios de información especializada</i></a>
			</p>
			<p><strong>Préstamos interbibliotecario</strong><br />
				<a href="/la-biblioteca/servicios/prestamos/interbibliotecarios">
					<i>Préstamos a otras instituciones de libros y revistas editadas a partir de 1900</i></a>
				<a href="/la-biblioteca/servicios/prestamos/para-exposiciones"><i>Préstamos para exposiciones</i></a>
			</p>
		</div>
	</li>
</ul>			</li>
			<li class="dropdown dropdown-large">
				<a href="#" class="dropdown-toggle" data-toggle="dropdown">COLECCIONES</a>
				<div class="active_menu"></div>
				<ul class="dropdown-menu dropdown-menu-large row">
	<li class="col-sm-3">
		<ul>
			<li class="dropdown-header">Cómo usar las colecciones</li>
		</ul>
		<ul class="col-md-6">
			<li><a href="/colecciones/como-usar-las-colecciones/pistas-para-descubrir-colecciones">Pistas para descubrir las colecciones</a></li>
			
		</ul>
		<ul class="col-md-6">
			<li><a href="/colecciones/como-usar-las-colecciones/salas-y-colecciones">Salas y Colecciones</a></li>
			<li><a href="/colecciones/como-usar-las-colecciones/descubra-colecciones">Descubra colecciones</a></li>
		</ul>
	</li>
	<li class="col-sm-2">
		<ul>
			<li class="dropdown-header">Colecciones generales</li>
							<li><a href="/colecciones/generales/coleccion-de-documentos-en-braille-y-libros-hablados">Colección de documentos en Braille y libros hablados</a></li>
	<li><a href="/colecciones/generales/mediateca">Mediateca</a></li>
	<li><a href="/colecciones/generales/coleccion-de-documentos-siglos-xx-y-xxi">Colección de documentos Siglos XX y XXI</a></li>
	<li><a href="/colecciones/generales/referencias">Referencias</a></li>
	<li><a href="/colecciones/generales/fondo-de-naciones-unidas">Fondo de Naciones Unidas</a></li>
	<li><a href="/colecciones/generales/archivo-de-la-biblioteca-nacional">Archivo de la Biblioteca Nacional</a></li>

		</ul>
	</li>
	<li class="col-sm-5">
		<ul class="col-md-12">
			<li class="dropdown-header">Colecciones especiales</li>
				</ul>
			<ul class="col-md-6">
		<li><a href="/colecciones/especiales/manuscritos">Manuscritos</a></li>
			<li><a href="/colecciones/especiales/mapoteca">Mapoteca</a></li>
			<li><a href="/colecciones/especiales/fototeca">Fototeca</a></li>
			<li><a href="/colecciones/especiales/etnologia-y-folklore">Etnología y Folklore</a></li>
			<li><a href="/colecciones/especiales/documentos-en-idioma-ruso">Documentos en idioma ruso</a></li>
			<li><a href="/colecciones/especiales/fondos-raros-y-valiosos">Fondos Raros y Valiosos</a></li>
			<li><a href="/colecciones/especiales/coleccion-martiana">Colección Martiana</a></li>
			<li><a href="/colecciones/especiales/coleccion-cubana">Colección Cubana</a></li>
			<li><a href="/colecciones/especiales/coleccin-de-grabados-cubanos-y-extranjeros">Colección de Grabados Cubanos y Extranjeros</a></li>
	</ul>
<ul class="col-md-6">
			<li><a href="/colecciones/especiales/programas-de-mano">Programas de mano</a></li>
			<li><a href="/colecciones/especiales/coleccin-ravents">Colección Raventós</a></li>
			<li><a href="/colecciones/especiales/arte">Arte</a></li>
			<li><a href="/colecciones/especiales/publicaciones-peridicas-siglo-xix">Publicaciones Periódicas Siglo XIX</a></li>
			<li><a href="/colecciones/especiales/coleccin-facsimilar-raros-y-valiosos">Colección Facsimilar Raros y Valiosos</a></li>
			<li><a href="/colecciones/especiales/colecciones-digitales">Colecciones digitales</a></li>
			<li><a href="/colecciones/especiales/msica">Música</a></li>
		</ul>

			
	</li>
	<li class="col-sm-2">
		<ul>
			<li class="dropdown-header">Desarrollo de colecciones</li>
			<li><a href="/colecciones/desarrollo-de-colecciones/deposito-legal">Depósito legal</a></li>
			<li><a href="/colecciones/desarrollo-de-colecciones/canje">Canje</a></li>
			<li><a href="/colecciones/desarrollo-de-colecciones/donaciones">Donaciones</a></li>
			<li><a href="/colecciones/desarrollo-de-colecciones/compra">Compra</a></li>
					</ul>
	</li>
</ul>			</li>
			<li class="dropdown dropdown-large">
				<a href="#" class="dropdown-toggle" data-toggle="dropdown">CATÁLOGOS</a>
				<div class="active_menu"></div>
				<ul class="dropdown-menu dropdown-menu-large row">
	<li class="col-sm-5">
		<ul>
			<li class="dropdown-header">Como usar los catálogos</li>
			<li><a href="/catalogos/como-usar-los-catalogos/estructura-y-origen-de-los-catalogos">Estructura y origen de Catálogos</a></li>
		</ul>

							
		<ul class="col-md-6">
			<li><a href="/catalogos/catalogo/catalogo-general">Catalogo General</a></li>
					<li><a href="/catalogos/catalogo/catlogos-de-autoridades">Catálogos de Autoridades</a></li>
					<li><a href="/catalogos/catalogo/catlogos-digitalizados">Catálogos Digitalizados</a></li>
					<li><a href="/catalogos/catalogo/ndice-de-publicaciones-peridicas-nacionales">Índice de Publicaciones Periódicas Nacionales</a></li>
					<li><a href="/catalogos/catalogo/analticas-publicaciones-peridicas-extranjeras">Analíticas Publicaciones Periódicas Extranjeras</a></li>
					</ul>
			<ul class="col-md-6">
					<li><a href="/catalogos/catalogo/catlogos-colectivos">Catálogos Colectivos</a></li>
			</ul>


																											</li>
	<li class="col-sm-4">
		<ul>
			<li class="dropdown-header">Catálogos de la Biblioteca</li>
			<li><a href="http://catalogo.bnjm.cu/">Catálogo General</a></li>
			<li><a href="http://catalogo.bnjm.cu/cgi-bin/koha/opac-authorities-home.pl">Catálogo de Autoridades</a></li>
			<li><a href="http://cipac.bnjm.cu/cgi-bin/item.py?carousel=True">Catálogos Digitalizados</a></li>
			<li><a href="http://indice.bnjm.cu/analiticas/c">Índice Publicaciones Periódicas Cubanas</a></li>
			<li><a href="http://indice.bnjm.cu/analiticas/e">Analíticas Publicaciones Periódicas Extranjeras </a></li>
			<li><a href="http://bdigital.bnjm.cu/?secc=catalogo&tipo=colectivo&colece=mapas">Catálogo Colectivo del Patrimonio Cartográfico</a></li>
			<li><a href="http://bdigital.bnjm.cu/?secc=catalogo&tipo=colectivo&colece=ccpc">Catálogo Colectivo de la Prensa Cubana</a></li>
		</ul>
					</li>
																						<li class="col-sm-3">
		<ul>
			<li class="dropdown-header">Información de interés</li>
			<li><a href="/catalogos/informacion-de-interes/signatura-topografica-de-la-bncjm">Signatura topográfica de la BNCJM</a></li>
			<li><a href="/catalogos/informacion-de-interes/epigrafiario-de-la-bncjm">Epigrafiario de la BNCJM</a></li>
		</ul>
	</li>
</ul>			</li>
			<li class="dropdown dropdown-large">
				<a href="/la-red/">LA RED</a>
			</li>
			<li class="dropdown dropdown-large">
				<a href="/ascubi/">ASCUBI</a>
			</li>
						<li class="dropdown dropdown-large border-none">
				<a class="hidden-sm" href="/programa-por-la-lectura/">PROGRAMA NACIONAL POR LA LECTURA</a>
				<a class="visible-sm" href="/programa-por-la-lectura/">PROG. POR LA LECTURA</a>
			</li>
		</ul>
	</div><!-- /.nav-collapse -->
</nav>
        </div>
                <div id="container" class="row">
            

	
	<div id="carousel-example-generic" class="carousel slide" data-ride="carousel">
		<!-- Indicators -->
		<ol class="carousel-indicators">
							<li data-target="#carousel-example-generic" data-slide-to="0" class=" active "></li>
							<li data-target="#carousel-example-generic" data-slide-to="1" class=""></li>
							<li data-target="#carousel-example-generic" data-slide-to="2" class=""></li>
							<li data-target="#carousel-example-generic" data-slide-to="3" class=""></li>
							<li data-target="#carousel-example-generic" data-slide-to="4" class=""></li>
					</ol>

		<!-- Wrapper for slides -->
		<div class="carousel-inner">
															
								
													
				<div class="item  active ">
					<img src="/media/images/home/fondo-1.jpg" alt="fondo1">

					<div class="carousel-caption hidden-xs">
						<a href="/la-biblioteca/noticias/bnjm/biblioteca-en-concierto-con-alina-orraca-y-schola-cantorum-coralina"><h3>Biblioteca en Concierto con Alina Orraca y Schola Cantorum Coralina</h3></a>
					</div>
				</div>
															
								
													
				<div class="item ">
					<img src="/media/images/home/fondo-2.jpg" alt="fondo1">

					<div class="carousel-caption hidden-xs">
						<a href="/la-biblioteca/noticias/bnjm/inaugurada-la-expo-azul-de-composicin-geomtrica-del-artista-david-beltrn"><h3>Inaugurada la Expo, Azul de Composición Geométrica , del artista David Beltrán</h3></a>
					</div>
				</div>
															
								
													
				<div class="item ">
					<img src="/media/images/home/fondo-3.jpg" alt="fondo1">

					<div class="carousel-caption hidden-xs">
						<a href="/la-biblioteca/noticias/bnjm/invitacin-muestra-personal-azul-de-composicin-geomtrica"><h3>Invitación .Muestra Personal, Azul de Composición Geométrica</h3></a>
					</div>
				</div>
															
								
													
				<div class="item ">
					<img src="/media/images/home/fondo-4.jpg" alt="fondo1">

					<div class="carousel-caption hidden-xs">
						<a href="/la-biblioteca/noticias/bnjm/homenaje-al-actor-luis-alberto-garca-en-la-biblioteca-nacional"><h3>Homenaje al actor Luis Alberto García en la Biblioteca Nacional</h3></a>
					</div>
				</div>
															
								
													
				<div class="item ">
					<img src="/media/images/home/fondo-5.jpg" alt="fondo1">

					<div class="carousel-caption hidden-xs">
						<a href="/la-biblioteca/noticias/bnjm/visitan-estudiantes-universitarios-norteamericanos-la-biblioteca-nacional"><h3>Visitan estudiantes universitarios norteamericanos la Biblioteca Nacional</h3></a>
					</div>
				</div>
					</div>

		<!-- Controls -->
		<a class="left carousel-control" href="#carousel-example-generic" data-slide="prev">
			<span class="glyphicon glyphicon-chevron-left"></span>
		</a>
		<a class="right carousel-control" href="#carousel-example-generic" data-slide="next">
			<span class="glyphicon glyphicon-chevron-right"></span>
		</a>
	</div>
                    </div>
                <div  id="footer" class="row" style=" background-color: #4f4e4e; color: #FFF;">
            <div class="col-md-3 hidden-sm hidden-xs">
     <img src="/media/images/logo_neg.png" class="img-responsive" style="max-width: 200px; margin-top: 30px;"/>
 </div>
 <div class="col-md-3 col-sm-4">
     <address style="margin-top:10px; ">
         <strong>Biblioteca Nacional de Cuba</strong><br>
         <i class="glyphicon glyphicon-map-marker"></i> Ave. Independencia y 20 de Mayo.<br />
         Plaza de la Revoluci&oacute;n.<br />
         Apartado Postal 6670.<br />
         La Habana. Cuba.<br />                                            
         <i class="glyphicon glyphicon-phone-alt"></i> (537) 855 5442 - 49 <br />
         <i class="glyphicon glyphicon-envelope"></i> contacto@bnjm.cu<br />
          2016 &copy; BNCJM<br />
     </address>
 </div>
 <div class="col-md-3 hidden-sm hidden-xs">
     <!--iframe style="margin-top: 20px;" class="img-responsive" frameborder="0" scrolling="no" marginheight="0" marginwidth="0" src="https://maps.google.com/maps?f=q&amp;source=s_q&amp;hl=es&amp;q=Biblioteca+Nacional+%22Jos%C3%A9+Mart%C3%AD%22,+Avenida+20+de+Mayo,+Plaza+de+la+Revolucion,+La+Habana,+Plaza+de+la+Revoluci%C3%B3n,+Provincia+de+La+Habana,+Cuba&amp;aq=&amp;sll=23.123864,-82.383342&amp;sspn=0.012235,0.026157&amp;ie=UTF8&amp;geocode=FU_XYAEdIu4W-w&amp;split=0&amp;hq=&amp;hnear=Biblioteca+Nacional+%22Jos%C3%A9+Mart%C3%AD%22,+Avenida+20+de+Mayo,+Plaza+de+la+Revolucion,+La+Habana,+Provincia+de+La+Habana,+Cuba&amp;ll=23.123864,-82.383342&amp;spn=0.012235,0.026157&amp;t=m&amp;z=14&amp;output=embed"></iframe>
     <small><a href="https://maps.google.com/maps?f=q&amp;source=embed&amp;hl=es&amp;q=Biblioteca+Nacional+%22Jos%C3%A9+Mart%C3%AD%22,+Avenida+20+de+Mayo,+Plaza+de+la+Revolucion,+La+Habana,+Plaza+de+la+Revoluci%C3%B3n,+Provincia+de+La+Habana,+Cuba&amp;aq=&amp;sll=23.123864,-82.383342&amp;sspn=0.012235,0.026157&amp;ie=UTF8&amp;geocode=FU_XYAEdIu4W-w&amp;split=0&amp;hq=&amp;hnear=Biblioteca+Nacional+%22Jos%C3%A9+Mart%C3%AD%22,+Avenida+20+de+Mayo,+Plaza+de+la+Revolucion,+La+Habana,+Provincia+de+La+Habana,+Cuba&amp;ll=23.123864,-82.383342&amp;spn=0.012235,0.026157&amp;t=m&amp;z=14" style="color:#FFF;text-align:center">Ver mapa más grande</a></small-->
 </div>
 <div class="col-md-3 col-sm-4" style="margin-top: 10px;">
     <ul class="list-unstyled">
         <li><a href="/mapa-sitio"><span class="fa fa-sitemap"></span> Mapa del sitio</a></li>
         <li><a href="/accesibilidad"><span class="fa fa-wheelchair"></span> Accesibilidad</a></li>
         <li><a href="/aviso-legal"><span class="fa fa-legal"></span> Aviso legal</a></li>
                  <li><a href="/la-biblioteca/amigos-de-la-biblioteca/"><span class="fa fa-group"></span> Amigos de la Biblioteca</a></li>
                           <li>
             <a class="btn btn-primary social-facebook img-circle" href="#"><i class="fa fa-facebook"></i></a>
             <a class="btn btn-primary social-login-btn social-twitter" href="#"><i class="fa fa-twitter"></i></a>
             <a class="btn btn-primary social-login-btn social-linkedin" href="#"><i class="fa fa-linkedin"></i></a>
             <a class="btn btn-primary social-login-btn social-google" href="#"><i class="fa fa-google-plus"></i></a>
         </li>
     </ul>
 </div>
        </div>               
    </div>

                                <!-- jQuery (necessary for Bootstrap's JavaScript plugins) --><script src="/media/js/jquery-1.10.2.min.js"></script><!--Include all compiled plugins (below), or include individual files as needed --><script src="/media/js/bootstrap.min.js"></script><script>
		$(document).ready(function(){
			$(window).scroll(function () {
				if ($(this).scrollTop() > 50) {
					$('#back-to-top').fadeIn();
				} else {
					$('#back-to-top').fadeOut();
				}
			});
			// scroll body to 0px on click
			$('#back-to-top').click(function () {
				$('#back-to-top').tooltip('hide');
				$('body,html').animate({
					scrollTop: 0
				}, 800);
				return false;
			});

			$('#back-to-top').tooltip('show');

		});

		$(function () {
			$('[data-toggle="popover"]').popover();
			$('[data-toggle="tooltip"]').tooltip();
		})
	</script>        
	    <script>
        $('.carousel').carousel({
            interval: 5000
        });
    </script>
        		<a id="back-to-top" href="#" class="btn btn-primary btn-lg back-to-top" role="button" title="Click to return on the top page" data-toggle="tooltip" data-placement="left"><span class="glyphicon glyphicon-chevron-up"></span></a>
    </body>
</html>