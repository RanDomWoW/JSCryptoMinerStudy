#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es" dir="ltr"
  xmlns:content="http://purl.org/rss/1.0/modules/content/"
  xmlns:dc="http://purl.org/dc/terms/"
  xmlns:foaf="http://xmlns.com/foaf/0.1/"
  xmlns:og="http://ogp.me/ns#"
  xmlns:rdfs="http://www.w3.org/2000/01/rdf-schema#"
  xmlns:sioc="http://rdfs.org/sioc/ns#"
  xmlns:sioct="http://rdfs.org/sioc/types#"
  xmlns:skos="http://www.w3.org/2004/02/skos/core#"
  xmlns:xsd="http://www.w3.org/2001/XMLSchema#">
<head>
<meta charset="utf-8" />
<meta name="generator" content="Drupal 7 (http://drupal.org)" />
<link rel="shortcut icon" href="http://noticias.minag.gob.cu/sites/default/files/favicon_0.png" type="image/png" />
<link rel="shortlink" href="http://noticias.minag.gob.cu/" />
<link rel="canonical" href="http://noticias.minag.gob.cu/" />
<link rel="alternate" type="application/rss+xml" title="La Agricultura hoy &lt;/br&gt; República de Cuba RSS" href="http://noticias.minag.gob.cu/rss.xml" />
<meta name="viewport" content="width=device-width, initial-scale=1, maximum-scale=1" />
<title>Agricultura en la Prensa República de Cuba</title>
<style type="text/css" media="all">
@import url("http://noticias.minag.gob.cu/modules/system/system.base.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/system/system.menus.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/system/system.messages.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/system/system.theme.css?oldqig");
</style>
<style type="text/css" media="all">
@import url("http://noticias.minag.gob.cu/modules/aggregator/aggregator.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/comment/comment.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/counter/counter.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/date/date_api/date.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/date/date_popup/themes/datepicker.1.7.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/field/theme/field.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/node/node.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/search/search.css?oldqig");
@import url("http://noticias.minag.gob.cu/modules/user/user.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/views-7.x-3.5/views/css/views.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/back_to_top/css/back_to_top.css?oldqig");
</style>
<style type="text/css" media="all">
@import url("http://noticias.minag.gob.cu/sites/all/modules/ckeditor-7.x-1.12/ckeditor/ckeditor.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/colorbox/styles/default/colorbox_style.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/ctools-7.x-1.2/ctools/css/ctools.css?oldqig");
</style>
<style type="text/css" media="screen">
<!--/*--><![CDATA[/*><!--*/
body{font-size:76%  !important;}body.textsize-52{font-size:52% !important;}body.textsize-58{font-size:58% !important;}body.textsize-64{font-size:64% !important;}body.textsize-70{font-size:70% !important;}body.textsize-76{font-size:76% !important;}body.textsize-82{font-size:82% !important;}body.textsize-88{font-size:88% !important;}body.textsize-94{font-size:94% !important;}body.textsize-100{font-size:100% !important;}body.textsize-106{font-size:106% !important;}body.textsize-112{font-size:112% !important;}body.textsize-118{font-size:118% !important;}body.textsize-124{font-size:124% !important;}body.textsize-130{font-size:130% !important;}body.textsize-136{font-size:136% !important;}body.textsize-142{font-size:142% !important;}body.textsize-148{font-size:148% !important;}

/*]]>*/-->
</style>
<style type="text/css" media="all">
@import url("http://noticias.minag.gob.cu/sites/all/modules/textsize/textsize.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/flexslider/assets/css/flexslider_img.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/contact_form_blocks/contact_form_blocks.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/modules/better_search/css/expand_on_hover.css?oldqig");
</style>
<style type="text/css" media="all">
@import url("http://noticias.minag.gob.cu/sites/all/themes/nexus/css/bootstrap.min.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/themes/nexus/css/flexslider.css?oldqig");
@import url("http://noticias.minag.gob.cu/sites/all/themes/nexus/style.css?oldqig");
</style>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/jquery.js?v=1.4.4"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/jquery.once.js?v=1.2"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/drupal.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/ui/jquery.effects.core.min.js?v=1.8.7"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/jquery.cookie.js?v=1.0"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/jquery.form.js?v=2.52"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/ajax.js?v=7.44"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/back_to_top/js/back_to_top.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/default/files/languages/es_1jH8y3eM2L_MtWKTwu9fntWzrKICxSuzAfd5bIIQkAc.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/jquery.flexslider.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/slide.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/libraries/colorbox/jquery.colorbox-min.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/colorbox/js/colorbox.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/colorbox/styles/default/colorbox_style.js?oldqig"></script>
<script type="text/javascript">
<!--//--><![CDATA[//><!--
  var textsizeCookieExpires = 365;
  var textsizeCookieDomain = "/";
  var textsizeElement = "body";
  var textsizeElementClass = "";
  var textsizeIncrement = 6;
  var textsizeNormal = 76;
  var textsizeMinimum = 52;
  var textsizeMaximum = 148;
  var textsizeIncrementDisplay = 8;
  var textsizeNormalDisplay = 100;
  var textsizeDisplay = 1;
  var textsizeMinT = "<abbr title=\"Mínimo\" class=\"textsize\">Min.</abbr> ";
  var textsizeMaxT = "<abbr title=\"Máximo\" class=\"textsize\">Max.</abbr> ";
  var textsizeCurrentText = "Current Size";
  var textsizeReset = 0;
//--><!]]>
</script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/textsize/jquery.textsize.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/libraries/flexslider/jquery.flexslider-min.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/views-7.x-3.5/views/js/base.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/progress.js?v=7.44"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/misc/textarea.js?v=7.44"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/views-7.x-3.5/views/js/ajax_view.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/bootstrap.min.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/superfish.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/mobilemenu.js?oldqig"></script>
<script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/themes/nexus/js/custom.js?oldqig"></script>
<script type="text/javascript">
<!--//--><![CDATA[//><!--
jQuery.extend(Drupal.settings, {"basePath":"\/","pathPrefix":"","ajaxPageState":{"theme":"nexus","theme_token":"j8N63SPMv09PZz3YcZgyKUWLQmSTI_dKnDxG7BYTY3g","js":{"sites\/all\/modules\/flexslider\/assets\/js\/flexslider.load.js":1,"misc\/jquery.js":1,"misc\/jquery.once.js":1,"misc\/drupal.js":1,"misc\/ui\/jquery.effects.core.min.js":1,"misc\/jquery.cookie.js":1,"misc\/jquery.form.js":1,"misc\/ajax.js":1,"sites\/all\/modules\/back_to_top\/js\/back_to_top.js":1,"public:\/\/languages\/es_1jH8y3eM2L_MtWKTwu9fntWzrKICxSuzAfd5bIIQkAc.js":1,"sites\/all\/themes\/nexus\/js\/jquery.flexslider.js":1,"sites\/all\/themes\/nexus\/js\/slide.js":1,"sites\/all\/libraries\/colorbox\/jquery.colorbox-min.js":1,"sites\/all\/modules\/colorbox\/js\/colorbox.js":1,"sites\/all\/modules\/colorbox\/styles\/default\/colorbox_style.js":1,"0":1,"sites\/all\/modules\/textsize\/jquery.textsize.js":1,"sites\/all\/libraries\/flexslider\/jquery.flexslider-min.js":1,"sites\/all\/modules\/views-7.x-3.5\/views\/js\/base.js":1,"misc\/progress.js":1,"misc\/textarea.js":1,"sites\/all\/modules\/views-7.x-3.5\/views\/js\/ajax_view.js":1,"sites\/all\/themes\/nexus\/js\/bootstrap.min.js":1,"sites\/all\/themes\/nexus\/js\/superfish.js":1,"sites\/all\/themes\/nexus\/js\/mobilemenu.js":1,"sites\/all\/themes\/nexus\/js\/custom.js":1},"css":{"modules\/system\/system.base.css":1,"modules\/system\/system.menus.css":1,"modules\/system\/system.messages.css":1,"modules\/system\/system.theme.css":1,"modules\/aggregator\/aggregator.css":1,"modules\/comment\/comment.css":1,"sites\/all\/modules\/counter\/counter.css":1,"sites\/all\/modules\/date\/date_api\/date.css":1,"sites\/all\/modules\/date\/date_popup\/themes\/datepicker.1.7.css":1,"modules\/field\/theme\/field.css":1,"modules\/node\/node.css":1,"modules\/search\/search.css":1,"modules\/user\/user.css":1,"sites\/all\/modules\/views-7.x-3.5\/views\/css\/views.css":1,"sites\/all\/modules\/back_to_top\/css\/back_to_top.css":1,"sites\/all\/modules\/ckeditor-7.x-1.12\/ckeditor\/ckeditor.css":1,"sites\/all\/modules\/colorbox\/styles\/default\/colorbox_style.css":1,"sites\/all\/modules\/ctools-7.x-1.2\/ctools\/css\/ctools.css":1,"0":1,"sites\/all\/modules\/textsize\/textsize.css":1,"sites\/all\/modules\/flexslider\/assets\/css\/flexslider_img.css":1,"sites\/all\/libraries\/flexslider\/flexslider.css":1,"sites\/all\/modules\/contact_form_blocks\/contact_form_blocks.css":1,"sites\/all\/modules\/better_search\/css\/expand_on_hover.css":1,"sites\/all\/themes\/nexus\/css\/bootstrap.min.css":1,"sites\/all\/themes\/nexus\/css\/flexslider.css":1,"sites\/all\/themes\/nexus\/style.css":1}},"colorbox":{"opacity":"0.85","current":"{current} of {total}","previous":"\u00ab Prev","next":"Next \u00bb","close":"Close","maxWidth":"98%","maxHeight":"98%","fixed":true,"mobiledetect":true,"mobiledevicewidth":"480px"},"jcarousel":{"ajaxPath":"\/jcarousel\/ajax\/views"},"flexslider":{"optionsets":{"default":{"namespace":"flex-","selector":".slides \u003E li","easing":"swing","direction":"horizontal","reverse":false,"smoothHeight":false,"startAt":0,"animationSpeed":600,"initDelay":0,"useCSS":true,"touch":true,"video":false,"keyboard":true,"multipleKeyboard":false,"mousewheel":false,"controlsContainer":".flex-control-nav-container","sync":"","asNavFor":"","itemWidth":0,"itemMargin":0,"minItems":0,"maxItems":0,"move":0,"animation":"fade","slideshow":true,"slideshowSpeed":7000,"directionNav":true,"controlNav":"1","prevText":"Previous","nextText":"Siguiente","pausePlay":false,"pauseText":"Pause","playText":"Play","randomize":false,"animationLoop":true,"pauseOnAction":true,"pauseOnHover":false,"manualControls":""}},"instances":{"flexslider-1":"default"}},"views":{"ajax_path":"\/views\/ajax","ajaxViews":{"views_dom_id:79f64cdec590eb1f6b72127f65016749":{"view_name":"noticias_portada","view_display_id":"block_3","view_args":"","view_path":"node","view_base_path":null,"view_dom_id":"79f64cdec590eb1f6b72127f65016749","pager_element":0},"views_dom_id:56ba99eb615650cbde4bf3b4a1caa3e6":{"view_name":"noticias_portada","view_display_id":"block_2","view_args":"","view_path":"node","view_base_path":null,"view_dom_id":"56ba99eb615650cbde4bf3b4a1caa3e6","pager_element":0},"views_dom_id:f20342b81e36fa832b3a3de2bd6e3ab2":{"view_name":"noticias_portada","view_display_id":"block","view_args":"","view_path":"node","view_base_path":null,"view_dom_id":"f20342b81e36fa832b3a3de2bd6e3ab2","pager_element":0}}},"urlIsAjaxTrusted":{"\/":true},"back_to_top":{"back_to_top_button_trigger":"100","back_to_top_prevent_on_mobile":1,"back_to_top_prevent_in_admin":1,"back_to_top_button_type":"image","back_to_top_button_text":"Back to top","#attached":{"library":[["system","ui"]]}}});
//--><!]]>
</script>
<!--[if lt IE 9]><script src="/sites/all/themes/nexus/js/html5.js"></script><![endif]-->
</head>
<body class="html front not-logged-in one-sidebar sidebar-first page-node">
    <div id="page">
<header id="masthead" class="site-header container" role="banner">
    <div class="row">
        <div id="logo" class="site-branding col-sm-5">
                            <div id="site-logo"><a href="/" title="Inicio">
                    <img src="http://noticias.minag.gob.cu/sites/default/files/intranet_minag-05.png" alt="Inicio"/>
                </a></div>            <h1 id="site-title">
                <a href="/" title="Inicio">La Agricultura hoy </br> República de Cuba</a>
            </h1>
        </div>
        <div class="col-sm-7 mainmenu">
            <div class="mobilenavi"></div>
            <nav id="navigation" role="navigation">
                <div id="main-menu">
                    <ul class="menu"><li class="first leaf"><a href="/" title="" class="active">Inicio</a></li>
<li class="leaf"><a href="/node/1">¿Quiénes somos?</a></li>
<li class="leaf"><a href="/nacionales" title="">Nacionales</a></li>
<li class="last leaf"><a href="/internacionales" title="">Internacionales</a></li>
</ul><form action="/" method="post" id="search-block-form--2" accept-charset="UTF-8"><div><div class="container-inline">
      <h2 class="element-invisible">Formulario de búsqueda</h2>
    <div class="icon"><i class="better_search"></i></div><div class="form-item form-type-textfield form-item-search-block-form">
  <label class="element-invisible" for="edit-search-block-form--4">Buscar </label>
 <input title="Escriba lo que quiere buscar." placeholder="buscar" type="text" id="edit-search-block-form--4" name="search_block_form" value="" size="20" maxlength="128" class="form-text" />
</div>
<div class="element-invisible form-actions form-wrapper" id="edit-actions--4"><input type="submit" id="edit-submit--4" name="op" value="Buscar" class="form-submit" /></div><input type="hidden" name="form_build_id" value="form-wS6Q0W1uX6Q9BQ6JiLUzGg3ILG3TBXH9qHxOtH36DtE" />
<input type="hidden" name="form_id" value="search_block_form" />
</div>
</div></form>                </div>
            </nav>
        </div>
    </div>
</header>


    
    <div class="fondo" style="padding: 60px 0;">
                    <div class="container">
            <div class="region region-slider">
  <div id="block-views-slider-block" class="block block-views">

      
  <div class="content">
    <div class="view view-slider view-id-slider view-display-id-block view-dom-id-74bd128add1252959f80e9fd47958f3c">
        
  
  
      <div class="view-content">
      <div  id="flexslider-1" class="flex-flexslider flexslider">
  <ul class="slides"><li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1721"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/dsc_0190.jpg?itok=DjSCpePB" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1721">Por superar retos sector agrícola en Granma</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Juan Farrell Villa</p>
<p>El  2018 tiene, de nuevo, grandes retos para la agricultura no cañera, con peso determinante en el sector primario de la economía y en la calidad de vida de la población.</p>
<p>Granma, eminentemente agropecuaria, cuenta con el potencial suficiente para producir mucho más alimentos por la vía de incrementar la eficiencia, la productividad y los rendimientos agrícolas, lo cual  contribuiría a la exportación de más productos y en la sustitución de importaciones.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1721" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1641"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/canel-5.jpg?itok=g7lszxaA" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1641">Aprecia Díaz-Canel recuperación citrícola en Isla de la Juventud</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Ana Esther Zulueta</p>
<p>Miguel Díaz-Canel, integrante del Buró Político del Partido Comunista de Cuba y Primer Vicepresidente de los Consejos de Estado y de Ministros, dijo hoy que ya se puede hablar del inicio de la recuperación citrícola en Isla de la Juventud.<br />
El dirigente cubano constató in situ los avances que experimenta el vivero de cítricos concebido para nueve casas de cultivo protegido, las cuales aseguran la siembra de 50 hectáreas de ese rubro exportable, cuya terminación se prevé para el mes de enero próximo.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1641" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1555"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/indice_29.jpg?itok=leSub0b8" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1555">El cactus, el alimento del futuro, según la FAO </a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>La organización de Naciones Unidas para la Agricultura y la Alimentación (FAO) hizo un llamado este jueves en favor del cactus, en especial el nopal, planta de origen mexicano, considerado alimento y pienso clave para el futuro en numerosas zonas del mundo.</p>
<p>El nopal, higuera de pala, tuna o chumbera, entre otras denominaciones, debe ser considerado "valioso, especialmente como alimento y pienso para el ganado en áreas de tierras secas", explicó la FAO en un comunicado divulgado en Roma.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1555" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1538"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/encargo-de-fidel.jpg?itok=2Yl7FYDE" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1538">Ganaderos fieles al encargo del Líder histórico</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Juan Farrell Villa</p>
<p>El fomento de pastos y forrajes y plantas proteicas tiene una creciente expresión en la provincia de Granma, en particular entre los ganaderos del Grupo agropecuario de la empresa azucarera, quienes prosiguen la tarea para cumplir con el encargo de nuestro Fidel.</p>
<p>El artífice del programa de desarrollo ganadero cubano dedicó sus últimos años a demostrar que se podía disponer de suficiente alimento animal, sin necesidad de importarlo e incrementar las producciones de carne y leche vacuna y de otras especies.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1538" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1348"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/medium_portad_spa.png?itok=Ew7yAOCg" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1348">Advierte la FAO sobre aumento del hambre en América Latina y el Caribe </a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por: Ana Ivis Galán García</p>
<p>Según dio a conocer éste martes 10 de octubre, el número total de personas que sufre hambre en América Latina y el Caribe ha aumentado en seis países y alcanza ya la cifra de 2,4 millones de personas en la región, lo cual está revirtiendo décadas de progresos que había conquistado la región en la batalla contra ese flagelo.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1348" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1242"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/f0092023.jpg?itok=CM3DZHwU" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1242">El compromiso de producir más</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Recorre Machado Ventura obras de interés económico de Cienfuegos y Matanzas, donde exhortó a aumentar las producciones agrícolas</p>
<p>Por Julio Martínez Molina</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1242" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1239"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/0_021_fora_y_fauna_ganaderia_1.jpg?itok=6remqBQc" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1239">Potencia Empresa Flora y Fauna cría de ganado racial en Santiago de Cuba</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Israel Hernández Planas</p>
<p>La protección e incremento de las especies animales es uno de los objetos sociales de la Empresa Flora y Fauna. En Santiago de Cuba dicha institución desarrolla un trabajo sostenido para el crecimiento del ganado vacuno racial en las montañas.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1239" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1205"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/20170916_084023.jpg?itok=CBJO2Qa9" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1205">Recorre Machado Ventura áreas agrícolas de Artemisa</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Acompañado por las principales autoridades del territorio, Machado Ventura dialogó con productores de San Cristóbal, Candelaria, el municipio capital, Alquízar y Güira de Melena, y recibió información detallada sobre las principales inversiones que se ejecutan en el Instituto de Investigaciones de Granos y la recuperación de la vaquería gigante La Coronela</p>
<p>Por Rommell González Cabrera</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1205" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/1123"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/f0089819.jpg?itok=qdAm9Ufm" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/1123">Primeros frutos de un cambio de mentalidad</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Miguel Febles Hernández</p>
<p>En amplio recorrido por zonas ganaderas de esta provincia, José Ramón Machado Ventura, segundo secretario del Comité Central del Partido, constató cómo se abre paso entre los vaqueros agramontinos un nuevo modo de pensar y de actuar, que se sustenta en la necesidad de diversificar el espectro productivo e incrementar el aporte de alimentos a la sociedad.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/1123" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/999"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/ministro-2-600x424.jpg?itok=ZaYlJCAF" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/999">Examinan producción de alimentos en Isla de la Juventud</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Ana Esther Zulueta</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/999" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/985"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/0731-diaz_canel-2.jpg?itok=zZozZff7" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/985">Recorre Díaz-Canel polo productivo de referencia de Las Tunas</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Roger Aguilera</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/985" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/931"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/yogur_paraiso.jpg?itok=-YwC_a2N" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/931">Proyectan nueva planta de yogur probiótico en Matanzas</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Yunielis Moliner Isasi</p>
<p>La Empresa Productora y Comercializadora de Productos Biofarmacéuticos de Matanzas, LABIOFAM, proyecta la construcción de una nueva planta de yogur probiótico para aumentar capacidades en la producción.</p>
<p>“El Grupo aprobó este proyecto y actualmente se realiza un estudio de factibilidad. Además, con la puesta en marcha de la nueva planta se duplicarán los resultados económicos de la empresa”, declaró Roger Jiménez Ramírez, director de la entidad.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/931" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/863"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/f0087553.jpg?itok=roq9orgV" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/863">Constatan empeño por desarrollar rama agropecuaria</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Recorre José Ramón Machado Ventura importantes centros productivos en la provincia espirituana.</p>
<p>Por José L. Camellón Y Juan Antonio Borrego</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/863" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/828"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/pinar-del-rio-machado-ventura.jpg?itok=iJ0PU11m" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/828">Recorre Machado Ventura estructuras productoras de tabaco en Pinar del Río</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Rosmery Iznaga Carmona</p>
<p>Más calidad y mayor rendimiento caracterizan a la campaña tabacalera 2016-2017 en Pinar del Río, la mejor del territorio en los últimos diez años.</p>
<p>Así lo constató el Segundo Secretario del Comité Central del Partido Comunista de Cuba (PCC) y Vicepresidente de los Consejos de Estado y de Ministros, José Ramón Machado Ventura, durante su recorrido por estructuras productoras de tabaco, las dedicadas principalmente al acopio y beneficio.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/828" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/762"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/1-plan-porcino-en-cuba-final.jpg?itok=r9WOP-eH" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/762">Para que nunca falte: Producción porcina sobre la mesa</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p><strong>Por: </strong>Oscar Figueredo Reinaldo, Junior Hernández Castro</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/762" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/491"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/0509-sergio-rodriguez.jpg?itok=4UETtsvS" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/491">Experto cubano recuerda aportes de Fidel a la agricultura</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Marta Hernández</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/491" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/486"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/campamento-ejt-cienfuegos-ministro-agricultura-foto-mireya-ojeda.jpg?itok=U0ay1Nfv" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/486">Inaugura Ministro de la Agricultura Campamento EJT en Jefatura Escambray</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Mireya Ojeda Cabrera</p>
<p>El Ministro de la Agricultura, Gustavo Rodríguez Rollero inauguró un campamento, en la granja estatal Aguacate, del Ejército Juvenil del Trabajo (EJT) en medio de las montañas del Grupo Guamuhaya, como parte del programa nacional para recuperar el desarrollo cafetalero.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/486" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/483"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/minsitro-de-la-agricultura-en-cienfuegos-foto-mireya-ojeda.jpg?itok=mSq8o-YA" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/483">Aprecia ministro de la agricultura avance de programa cafetalero en Cienfuegos </a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Mireya Ojeda Cabrera</p>
<p>En esta provincia de Cienfuegos avanza el Programa cafetalero, que tiene dos años de atraso en el país, afirma el ministro de la agricultura, Gustavo Rodríguez Rollero durante un recorrido por las montañas del Grupo Guamuhaya.</p>
<p>“Cumplen el programa y estamos exigiendo que haya mucha disciplina en la parte agro técnica del café, no perder tiempo en aquellas plantaciones que el programa amerita la pena hay que hacerle una poda y volverlo a sembrar nuevo” especifica Rollero.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/483" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/467"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/1ca98d9c3a4a8177480757f1164fc28f_l.jpg?itok=BjOp6hwo" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/467">Ministro de la Agricultura visita el polo productivo en Jobabo </a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Yaidel M. Rodríguez Castro</p>
<p>La visita del Ministro de la Agricultura al polo productivo Melanio Ortiz fue una de las pruebas de fuego de esta área agrícola jobabense calificada de antemano como la más grande de su tipo en la provincia de Las Tunas.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/467" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/339"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/ibp-platano.jpg?itok=iKdj7pk9" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/339">Instituto de Biotecnología de las Plantas insiste en el mejoramiento genético </a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>La institución labora en ocho proyectos que abarcan el plátano, el banano, y café, los frijoles, la papa, árboles maderables, y la producción de plantas ornamentales.</p>
<p>Dos proyectos internacionales para el logro de variedades resistentes y tolerantes a las altas temperaturas, la sequía y a enfermedades comunes figuran en la cartera priorizada del Instituto de Biotecnología de las Plantas (IBP) para celebrar sus 25 años de creado.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/339" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/302"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/3428a4d45eefd1bdcb76832876a2651c_l.jpg?itok=n1J0DKRw" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/302">El otro cambio del cambio climático</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por José Armando Fernández Salazar</p>
<p>La sequía se ensaña con los campos de Las Tunas y cambia el paisaje. Ante las inclemencias del clima urge cambiar también mentalidades. En reciente visita a la provincia, Ulises Rosales del Toro conoció de las iniciativas que se aplican para mitigar los efectos del desastre natural e incrementar la producción de alimentos.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/302" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/267"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/mesa-redonda-arroz-y-granos-580x385.jpg?itok=xpCBuc9t" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/267">De moros, cristianos y la agricultura en Cuba</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por <a href="http://www.cubadebate.cu/autor/oscar-figueredo-reinaldo/" title="Ver todos los artículos de Oscar Figueredo Reinaldo">Oscar Figueredo Reinaldo</a>, <a href="http://www.cubadebate.cu/autor/susana-tesoro/" title="Ver todos los artículos de Susana Tesoro">Susana Tesoro</a></p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/267" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/262"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/indice.jpg?itok=J--ifgXq" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/262">La agricultura necesita estudio y ciencia aplicada</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Leidys María Labrador Herrera</p>
<p><strong>-</strong>A optimizar el agua en función de producir alimentos y a capacitar a los campesinos para que desarrollen un adecuado manejo animal llamó en este territorio Ulises Rosales del Toro, vicepresidente del Consejo de Ministros.</p>
<p>Durante un recorrido que comprendió las localidades de Jesús Menéndez y Majibacoa, el dirigente dialogó con directivos de recursos hidráulicos para conocer las medidas de enfrentamiento a la sequía y la situación real que al respecto enfrenta hoy la provincia.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/262" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/212"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/medico-veterinario-natan-vega.jpg?itok=VCyjDHxC" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/212">Mejores en la medicina veterinaria</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p><strong>Por Orestes Eugellés Mena</strong></p>
<p><strong>El Sindicato Nacional de Trabajadores Agropecuarios, Forestales y Tabacaleros y el Instituto Nacional de Medicina Veterinaria  otorgaron  a la provincia de Guantánamo la sede del Acto Central por los festejos del 24 de marzo, día del trabajador de esa especialidad</strong>.</p>
<p>Ese territorio se hizo acreedor del estímulo por obtener los mejores resultados en la reducción de la mortalidad en los ganados bobinos totales, terneros, búfalos, equinos, ovinos-caprinos.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/212" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/137"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/entrevista3.jpg?itok=Cb_wK3jm" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/137">Por eso Yo Soy Cubano.</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Por Yanel Blanco Miranda</p>
<p>Sergio respira profundo y a su nariz llegan los olores de la tierra cubierta por el rocío.</p>
<p>Aún no amanece, pero como si fuera la primera vez recorre la distancia que hay desde su casa a la oficina. Llegar temprano es muy importante para este hombre que apenas nota el paso del tiempo, y para quien la ciencia agrícola no es trabajo sino vida.</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/137" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/47"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/holstein_01.jpg?itok=OG6E9NVo" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/47">Preservan fondo genético de la raza Holstein</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Un proyecto encaminado a preservar el fondo genético (genofondo) de la raza bovina Holstein es impulsado en la granja genética El Abra, centro insigne de la empresa pecuaria El Tablón, de Cumanayagua.</p>
<p>Por Julio Martínez Molina-</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/47" class="readmore">Leer más</a></span>  </div></li>
<li>  
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/33"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/slider/public/noticias/carbon-vegetal-foto-acn-sosa.jpg?itok=USnbltYM" width="467" height="315" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/33">Carbón vegetal, el primero en llegar</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Un mercado en expansión con posibilidades para la inserción de mayor cantidad del producto cubano a precios ventajosos...</p>
<p>Por Juan Carlo Imbert Mayola-</p></div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a href="/node/33" class="readmore">Leer más</a></span>  </div></li>
</ul></div>
    </div>
  
  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
            </div>    </div>

        <div id="preface-area" style=" background-color: #D7D3D4;">
        <div class="container">
            <div class="row">
                                    <div class="preface-block col-sm-2" style="padding-top: 40px">
                    <div class="region region-preface-first">
  <div id="block-views-noticias-portada-block-3" class="block block-views">

        <h2 >Más Comentados</h2>
    
  <div class="content">
    <div class="view view-noticias-portada view-id-noticias_portada view-display-id-block_3 view-dom-id-79f64cdec590eb1f6b72127f65016749">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first">
      
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2022"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/thumbnail/public/noticias/sideromecanica.jpg?itok=xSAg_2-a" width="100" height="66" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2022">Sideromecánica avileña repara remolques arroceros</a></span>  </div>  
  <div class="views-field views-field-comment-count">    <span class="views-label views-label-comment-count">Comentarios: </span>    <span class="field-content">0</span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even">
      
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2021"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/thumbnail/public/noticias/ganado-ovino_1.jpg?itok=7dVgG3lN" width="100" height="67" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2021">Progresa la crianza de ovinos y caprinos en Ciego de Ávila</a></span>  </div>  
  <div class="views-field views-field-comment-count">    <span class="views-label views-label-comment-count">Comentarios: </span>    <span class="field-content">0</span>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd views-row-last">
      
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2020"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/thumbnail/public/noticias/1_1.jpg?itok=EAj2tCmf" width="100" height="51" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2020">Llaman a incorporar fuerza técnica calificada en las cooperativas  </a></span>  </div>  
  <div class="views-field views-field-comment-count">    <span class="views-label views-label-comment-count">Comentarios: </span>    <span class="field-content">0</span>  </div>  </div>
    </div>
  
  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                    </div>
                
                                <div id="main-content" class="content-area col-sm-8"
                     style=" background-color: #ffffff;">
                    <div class="container">
                        <div class="row">
                            <div id="primary">
                                <section id="content" role="main" class="clearfix" style="padding: 50px 15px">
                                                                                                                                                <div id="content-wrap">
                                                                                                                                                                                                                                                                                        <div class="region region-content">
  <div id="block-views-noticias-portada-block" class="block block-views">

      
  <div class="content">
    <div class="view view-noticias-portada view-id-noticias_portada view-display-id-block view-dom-id-f20342b81e36fa832b3a3de2bd6e3ab2">
        
  
  
      <div class="view-content">
      <table class="views-view-grid cols-2">
  <tbody>
          <tr class="row-1 row-first">
                  <td class="col-1 col-first">
              
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2022"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/noticias/public/noticias/sideromecanica.jpg?itok=P11KthvU" width="205" height="140" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2022">Sideromecánica avileña repara remolques arroceros</a></span>  </div>          </td>
                  <td class="col-2 col-last">
              
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2021"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/noticias/public/noticias/ganado-ovino_1.jpg?itok=5Tq00UGN" width="205" height="140" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2021">Progresa la crianza de ovinos y caprinos en Ciego de Ávila</a></span>  </div>          </td>
              </tr>
          <tr class="row-2 row-last">
                  <td class="col-1 col-first">
              
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2020"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/noticias/public/noticias/1_1.jpg?itok=EcOAneHi" width="205" height="140" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2020">Llaman a incorporar fuerza técnica calificada en las cooperativas  </a></span>  </div>          </td>
                  <td class="col-2 col-last">
              
  <div class="views-field views-field-field-image">        <div class="field-content"><a href="/node/2019"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/noticias/public/noticias/f0103397.jpg?itok=WFpkMUPO" width="205" height="140" alt="" /></a></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/2019">Kilowatts para los sembrados </a></span>  </div>          </td>
              </tr>
      </tbody>
</table>
    </div>
  
      <h2 class="element-invisible">Páginas</h2><div class="item-list"><ul class="pager"><li class="pager-current first">1</li>
<li class="pager-item"><a title="Go to page 2" href="/node?page=1">2</a></li>
<li class="pager-item"><a title="Go to page 3" href="/node?page=2">3</a></li>
<li class="pager-item"><a title="Go to page 4" href="/node?page=3">4</a></li>
<li class="pager-item"><a title="Go to page 5" href="/node?page=4">5</a></li>
<li class="pager-item"><a title="Go to page 6" href="/node?page=5">6</a></li>
<li class="pager-item"><a title="Go to page 7" href="/node?page=6">7</a></li>
<li class="pager-item"><a title="Go to page 8" href="/node?page=7">8</a></li>
<li class="pager-item"><a title="Go to page 9" href="/node?page=8">9</a></li>
<li class="pager-ellipsis">…</li>
<li class="pager-next"><a title="Ir a la página siguiente" href="/node?page=1">siguiente › </a></li>
<li class="pager-last last"><a title="Go to last page" href="/node?page=499">última »</a></li>
</ul></div>  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
<div id="block-system-main" class="block block-system">

      
  <div class="content">
      <article id="node-1721" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1721" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1721">Por superar retos sector agrícola en Granma</a></h2>
            <span property="dc:title" content="Por superar retos sector agrícola en Granma" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Juan Farrell Villa</p>
<p>El  2018 tiene, de nuevo, grandes retos para la agricultura no cañera, con peso determinante en el sector primario de la economía y en la calidad de vida de la población.</p>
<p>Granma, eminentemente agropecuaria, cuenta con el potencial suficiente para producir mucho más alimentos por la vía de incrementar la eficiencia, la productividad y los rendimientos agrícolas, lo cual  contribuiría a la exportación de más productos y en la sustitución de importaciones.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1721" rel="tag" title="Por superar retos sector agrícola en Granma">Leer más<span class="element-invisible"> sobre Por superar retos sector agrícola en Granma</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1641" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1641" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1641">Aprecia Díaz-Canel recuperación citrícola en Isla de la Juventud</a></h2>
            <span property="dc:title" content="Aprecia Díaz-Canel recuperación citrícola en Isla de la Juventud" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Ana Esther Zulueta</p>
<p>Miguel Díaz-Canel, integrante del Buró Político del Partido Comunista de Cuba y Primer Vicepresidente de los Consejos de Estado y de Ministros, dijo hoy que ya se puede hablar del inicio de la recuperación citrícola en Isla de la Juventud.<br />
El dirigente cubano constató in situ los avances que experimenta el vivero de cítricos concebido para nueve casas de cultivo protegido, las cuales aseguran la siembra de 50 hectáreas de ese rubro exportable, cuya terminación se prevé para el mes de enero próximo.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1641" rel="tag" title="Aprecia Díaz-Canel recuperación citrícola en Isla de la Juventud">Leer más<span class="element-invisible"> sobre Aprecia Díaz-Canel recuperación citrícola en Isla de la Juventud</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1555" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1555" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1555">El cactus, el alimento del futuro, según la FAO </a></h2>
            <span property="dc:title" content="El cactus, el alimento del futuro, según la FAO " class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>La organización de Naciones Unidas para la Agricultura y la Alimentación (FAO) hizo un llamado este jueves en favor del cactus, en especial el nopal, planta de origen mexicano, considerado alimento y pienso clave para el futuro en numerosas zonas del mundo.</p>
<p>El nopal, higuera de pala, tuna o chumbera, entre otras denominaciones, debe ser considerado "valioso, especialmente como alimento y pienso para el ganado en áreas de tierras secas", explicó la FAO en un comunicado divulgado en Roma.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1555" rel="tag" title="El cactus, el alimento del futuro, según la FAO ">Leer más<span class="element-invisible"> sobre El cactus, el alimento del futuro, según la FAO </span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1538" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1538" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1538">Ganaderos fieles al encargo del Líder histórico</a></h2>
            <span property="dc:title" content="Ganaderos fieles al encargo del Líder histórico" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Juan Farrell Villa</p>
<p>El fomento de pastos y forrajes y plantas proteicas tiene una creciente expresión en la provincia de Granma, en particular entre los ganaderos del Grupo agropecuario de la empresa azucarera, quienes prosiguen la tarea para cumplir con el encargo de nuestro Fidel.</p>
<p>El artífice del programa de desarrollo ganadero cubano dedicó sus últimos años a demostrar que se podía disponer de suficiente alimento animal, sin necesidad de importarlo e incrementar las producciones de carne y leche vacuna y de otras especies.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1538" rel="tag" title="Ganaderos fieles al encargo del Líder histórico">Leer más<span class="element-invisible"> sobre Ganaderos fieles al encargo del Líder histórico</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1348" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1348" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1348">Advierte la FAO sobre aumento del hambre en América Latina y el Caribe </a></h2>
            <span property="dc:title" content="Advierte la FAO sobre aumento del hambre en América Latina y el Caribe " class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por: Ana Ivis Galán García</p>
<p>Según dio a conocer éste martes 10 de octubre, el número total de personas que sufre hambre en América Latina y el Caribe ha aumentado en seis países y alcanza ya la cifra de 2,4 millones de personas en la región, lo cual está revirtiendo décadas de progresos que había conquistado la región en la batalla contra ese flagelo.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1348" rel="tag" title="Advierte la FAO sobre aumento del hambre en América Latina y el Caribe ">Leer más<span class="element-invisible"> sobre Advierte la FAO sobre aumento del hambre en América Latina y el Caribe </span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1242" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1242" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1242">El compromiso de producir más</a></h2>
            <span property="dc:title" content="El compromiso de producir más" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Recorre Machado Ventura obras de interés económico de Cienfuegos y Matanzas, donde exhortó a aumentar las producciones agrícolas</p>
<p>Por Julio Martínez Molina</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1242" rel="tag" title="El compromiso de producir más">Leer más<span class="element-invisible"> sobre El compromiso de producir más</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1239" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1239" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1239">Potencia Empresa Flora y Fauna cría de ganado racial en Santiago de Cuba</a></h2>
            <span property="dc:title" content="Potencia Empresa Flora y Fauna cría de ganado racial en Santiago de Cuba" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Israel Hernández Planas</p>
<p>La protección e incremento de las especies animales es uno de los objetos sociales de la Empresa Flora y Fauna. En Santiago de Cuba dicha institución desarrolla un trabajo sostenido para el crecimiento del ganado vacuno racial en las montañas.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1239" rel="tag" title="Potencia Empresa Flora y Fauna cría de ganado racial en Santiago de Cuba">Leer más<span class="element-invisible"> sobre Potencia Empresa Flora y Fauna cría de ganado racial en Santiago de Cuba</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1205" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1205" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1205">Recorre Machado Ventura áreas agrícolas de Artemisa</a></h2>
            <span property="dc:title" content="Recorre Machado Ventura áreas agrícolas de Artemisa" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Acompañado por las principales autoridades del territorio, Machado Ventura dialogó con productores de San Cristóbal, Candelaria, el municipio capital, Alquízar y Güira de Melena, y recibió información detallada sobre las principales inversiones que se ejecutan en el Instituto de Investigaciones de Granos y la recuperación de la vaquería gigante La Coronela</p>
<p>Por Rommell González Cabrera</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1205" rel="tag" title="Recorre Machado Ventura áreas agrícolas de Artemisa">Leer más<span class="element-invisible"> sobre Recorre Machado Ventura áreas agrícolas de Artemisa</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-1123" class="node node-noticias node-promoted node-teaser clearfix" about="/node/1123" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/1123">Primeros frutos de un cambio de mentalidad</a></h2>
            <span property="dc:title" content="Primeros frutos de un cambio de mentalidad" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Miguel Febles Hernández</p>
<p>En amplio recorrido por zonas ganaderas de esta provincia, José Ramón Machado Ventura, segundo secretario del Comité Central del Partido, constató cómo se abre paso entre los vaqueros agramontinos un nuevo modo de pensar y de actuar, que se sustenta en la necesidad de diversificar el espectro productivo e incrementar el aporte de alimentos a la sociedad.</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/1123" rel="tag" title="Primeros frutos de un cambio de mentalidad">Leer más<span class="element-invisible"> sobre Primeros frutos de un cambio de mentalidad</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
  <article id="node-999" class="node node-noticias node-promoted node-teaser clearfix" about="/node/999" typeof="sioc:Item foaf:Document">
        <header>
                    <h2 class="title" ><a href="/node/999">Examinan producción de alimentos en Isla de la Juventud</a></h2>
            <span property="dc:title" content="Examinan producción de alimentos en Isla de la Juventud" class="rdf-meta element-hidden"></span><span property="sioc:num_replies" content="0" datatype="xsd:integer" class="rdf-meta element-hidden"></span>  
      
          </header>
  
  <div class="content">
    <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><p>Por Ana Esther Zulueta</p></div></div></div>  </div>

      <footer>
      <ul class="links inline"><li class="node-readmore first"><a href="/node/999" rel="tag" title="Examinan producción de alimentos en Isla de la Juventud">Leer más<span class="element-invisible"> sobre Examinan producción de alimentos en Isla de la Juventud</span></a></li>
<li class="comment_forbidden last"></li>
</ul>    </footer>
  
    </article> <!-- /.node -->
<h2 class="element-invisible">Páginas</h2><div class="item-list"><ul class="pager"><li class="pager-current first">1</li>
<li class="pager-item"><a title="Go to page 2" href="/node?page=1">2</a></li>
<li class="pager-item"><a title="Go to page 3" href="/node?page=2">3</a></li>
<li class="pager-item"><a title="Go to page 4" href="/node?page=3">4</a></li>
<li class="pager-item"><a title="Go to page 5" href="/node?page=4">5</a></li>
<li class="pager-item"><a title="Go to page 6" href="/node?page=5">6</a></li>
<li class="pager-item"><a title="Go to page 7" href="/node?page=6">7</a></li>
<li class="pager-item"><a title="Go to page 8" href="/node?page=7">8</a></li>
<li class="pager-item"><a title="Go to page 9" href="/node?page=8">9</a></li>
<li class="pager-ellipsis">…</li>
<li class="pager-next"><a title="Ir a la página siguiente" href="/node?page=1">siguiente › </a></li>
<li class="pager-last last"><a title="Go to last page" href="/node?page=499">última »</a></li>
</ul></div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                                    </div>

                                </section>
                            </div>
                        </div>
                    </div>
                </div>

                                    <div class="preface-block col-sm-2" style="padding-top: 40px">
                    <div class="region region-preface-last">
  <div id="block-views-noticias-portada-block-2" class="block block-views">

        <h2 >Lo Más Leído</h2>
    
  <div class="content">
    <div class="view view-noticias-portada view-id-noticias_portada view-display-id-block_2 view-dom-id-56ba99eb615650cbde4bf3b4a1caa3e6">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first">
      
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/5">Cooperativas agropecuaria de Matanzas incrementaron sus ventas al Turismo</a></span>  </div>  
  <div class="views-field views-field-totalcount">    <span class="views-label views-label-totalcount">Total de vistas: </span>    <span class="field-content">85</span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even">
      
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/7">Érase una vez un zoológico</a></span>  </div>  
  <div class="views-field views-field-totalcount">    <span class="views-label views-label-totalcount">Total de vistas: </span>    <span class="field-content">75</span>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd">
      
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/108">Recibe Ministro cubano de la Agricultura a Delegación del Congreso de los Estados Unidos</a></span>  </div>  
  <div class="views-field views-field-totalcount">    <span class="views-label views-label-totalcount">Total de vistas: </span>    <span class="field-content">58</span>  </div>  </div>
  <div class="views-row views-row-4 views-row-even">
      
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/4">Prevé la Isla superar récord histórico en producción de miel este año</a></span>  </div>  
  <div class="views-field views-field-totalcount">    <span class="views-label views-label-totalcount">Total de vistas: </span>    <span class="field-content">46</span>  </div>  </div>
  <div class="views-row views-row-5 views-row-odd views-row-last">
      
  <div class="views-field views-field-title">        <span class="field-content"><a href="/node/419">Jagüey Grande: Crece producción de cítricos y frutales</a></span>  </div>  
  <div class="views-field views-field-totalcount">    <span class="views-label views-label-totalcount">Total de vistas: </span>    <span class="field-content">44</span>  </div>  </div>
    </div>
  
  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                    </div>            </div>
        </div>
    </div>

<div class="fondo">
    <div class="container">
                    <div class="region region-bottom1">
  <div id="block-views-frases-block" class="block block-views">

      
  <div class="content">
    <div class="view view-frases view-id-frases view-display-id-block view-dom-id-e340f4d41acda5d9e61b4c9a870209e3">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first views-row-last">
      
  <span class="views-field views-field-field-image">        <span class="field-content"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/medium/public/frases/fidel-castro.jpg?itok=2yXIw8Mg" width="220" height="124" alt="" /></span>  </span>  
  <span class="views-field views-field-body">        <span class="field-content"><h4><p>"El futuro de nuestra patria tiene que ser necesariamente un futuro de hombres de ciencia, tiene que ser un futuro de hombres de pensamiento, porque precisamente es lo que estamos sembrando; o que más estamos sembrando son oportunidades a la inteligencia (...)"</p>
<p class="rteright">Fidel Castro Ruz</p>
</h4></span>  </span>  </div>
    </div>
  
  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
            </div>
</div>


        <div id="bottom">
        <div class="container">
            <div class="row">
                                    <div class="footer-block col-sm-4">
                    <div class="region region-footer-first">
  <div id="block-menu-menu-sitios-de-inter-s" class="block block-menu">

        <h2 >Sitios de Interés</h2>
    
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="http://www.ausc.co.cu/" title="">Agricultura Urbana y Suburbana</a></li>
<li class="leaf"><a href="http://www.actaf.co.cu/" title="">Asociación Cubana de Técnicos Agrícolas y Forestales</a></li>
<li class="leaf"><a href="http://www.campesinocubano.anap.cu/" title="">Asociación Nac. Agricultores Pequeños</a></li>
<li class="leaf"><a href="http://www.ciapi.minag.cu" title="">Centro de Investigaciones Apícolas</a></li>
<li class="leaf"><a href="http://web.dlg.grm.minag.gob.cu/" title="">Delegación Provincial Granma</a></li>
<li class="leaf"><a href="http://www.dlg.iju.minag.gob.cu/" title="">Delegación Provincial IJ</a></li>
<li class="leaf"><a href="http://www.one.cu/" title="">Oficina Nac. de Estadísticas</a></li>
<li class="last leaf"><a href="http://www.fao.org/home/es/" title="">Organización de las Naciones Unidas para la Alimentación y la Agricultura</a></li>
</ul>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                    </div>                                    <div class="footer-block col-sm-4">
                    <div class="region region-footer-second">
  <div id="block-contact-form-blocks-0" class="block block-contact-form-blocks">

        <h2 >Envíanos un Mensaje</h2>
    
  <div class="content">
    <form class="user-info-from-cookie contact-form" action="/" method="post" id="contact-site-form" accept-charset="UTF-8"><div><div class="form-item form-type-textfield form-item-name">
  <label class="element-invisible" for="edit-name--2">Su nombre <span class="form-required" title="Este campo es obligatorio.">*</span></label>
 <input placeholder="Nombre..." type="text" id="edit-name--2" name="name" value="" size="60" maxlength="255" class="form-text required" />
</div>
<div class="form-item form-type-textfield form-item-mail">
  <label class="element-invisible" for="edit-mail">Su dirección de correo electrónico <span class="form-required" title="Este campo es obligatorio.">*</span></label>
 <input placeholder="Correo..." type="text" id="edit-mail" name="mail" value="" size="60" maxlength="255" class="form-text required" />
</div>
<div class="form-item form-type-textfield form-item-subject">
  <label class="element-invisible" for="edit-subject">Asunto <span class="form-required" title="Este campo es obligatorio.">*</span></label>
 <input placeholder="Asunto..." type="text" id="edit-subject" name="subject" value="" size="60" maxlength="255" class="form-text required" />
</div>
<div class="form-item form-type-textarea form-item-message">
  <label class="element-invisible" for="edit-message">Message <span class="form-required" title="Este campo es obligatorio.">*</span></label>
 <div class="form-textarea-wrapper resizable"><textarea placeholder="Mensaje..." id="edit-message" name="message" cols="60" rows="5" class="form-textarea required"></textarea></div>
</div>
<input type="hidden" name="form_build_id" value="form-tJ2ZpW-pWjYGZvoLATLdm630AB24GlcuhI4XYU352PU" />
<input type="hidden" name="form_id" value="contact_site_form" />
<div class="form-actions form-wrapper" id="edit-actions--3"><input type="submit" id="edit-submit--3" name="op" value="Enviar" class="form-submit" /></div></div></form>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                    </div>                                    <div class="footer-block col-sm-4">
                    <div class="region region-footer-third">
  <div id="block-views-datos-de-contacto-block" class="block block-views">

        <h2 >Datos de Contacto</h2>
    
  <div class="content">
    <div class="view view-datos-de-contacto view-id-datos_de_contacto view-display-id-block view-dom-id-5c1b5f3ae8331c9b8863380c269e93d8">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first views-row-last">
      
  <div class="views-field views-field-field-image">        <div class="field-content"><img typeof="foaf:Image" src="http://noticias.minag.gob.cu/sites/default/files/styles/medium/public/todas/1_0.png?itok=6PchZUT4" width="220" height="168" alt="" /></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><div>Dirección: Conill, Ave. Independencia, Plaza de la Revolución, La Habana, Cuba</div>
<div>Teléfono: (+53) 78847100</div>
<div style="float:left; margin-right: 10px;">Correo:</div>
<div><a href="mailto:web2@hab.eicma.cu">web2@hab.eicma.cu</a></div>
</div>  </div>  </div>
    </div>
  
  
  
  
  
  
</div>  </div>
  
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                    </div>                            </div>
        </div>
    </div>

<footer id="colophon" class="site-footer" role="contentinfo">
    <div class="container">
        <div class="row">
            <div class="fcred col-sm-12">
                Copyright &copy; 2018, <a
                    href="/">La Agricultura hoy </br> República de Cuba</a>, Desarrollado por EICMA.
            </div>
        </div>
    </div>
</div>
</div>  <script type="text/javascript" src="http://noticias.minag.gob.cu/sites/all/modules/flexslider/assets/js/flexslider.load.js?oldqig"></script>
</body>
</html>