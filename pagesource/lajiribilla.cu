#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">
  <head>
    <meta charset="utf-8">
    <meta http-equiv="X-UA-Compatible" content="IE=edge">
    <meta name="viewport" content="width=device-width, initial-scale=1">
    		<meta name="description" content="La Jiribilla es una revista sobre cultura cubana. Inserta en el espectro de los medios en Internet un punto de vista objetivo sobre la realidad cultural cubana, muchas veces manipulada o desconocida fuera de la Isla." >
        		<meta name="keywords" content="La Jiribilla, cultura cubana, José Lezama Lima, periodismo digital" >
        		<meta name="robots" content="INDEX, FOLLOW" >
    	<meta property="fb:app_id" content="108622606834"/>
	<!--Open Graph -->
	<meta property="og:site_name" content="La Jiribilla" />
        <meta property="og:locale" content="es_ES" />
		                <meta property="og:image" content="http://lajiribilla.cu/themes/jiribilla/assets/img/LaJiribilla.jpg" />
	        <meta property="og:image:width" content="478" />
        <meta property="og:image:height" content="600" />
					<meta property="og:type" content="website" />
											<meta property="article:tag" content="La Jiribilla" />
							<meta property="article:tag" content="cultura cubana" />
							<meta property="article:tag" content="José Lezama Lima" />
							<meta property="article:tag" content="periodismo digital" />
							
	
	
    <title>La Jiribilla - revista de cultura cubana</title>

    <link href="/themes/jiribilla/assets/img/favicon.ico" rel="shortcut icon">
    <link href="/themes/jiribilla/assets/img/favicon.ico" type="image/x-icon" rel="icon">
    <!-- Bootstrap -->
    <link href="/themes/jiribilla/assets/css/font-awesome.min.css" rel="stylesheet">
    <link href="/themes/jiribilla/assets/css/style.css" rel="stylesheet">

    <!-- HTML5 Shim and Respond.js IE8 support of HTML5 elements and media queries -->
    <!-- WARNING: Respond.js doesn't work if you view the page via file:// -->
    <!--[if lt IE 9]>
      <script src="https://oss.maxcdn.com/libs/html5shiv/3.7.0/html5shiv.js"></script>
      <script src="https://oss.maxcdn.com/libs/respond.js/1.4.2/respond.min.js"></script>
    <![endif]-->
			<script>
			(function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
			(i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
			m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
			})(window,document,'script','//www.google-analytics.com/analytics.js','ga');

			ga('create', 'UA-74154009-1', 'auto');
			ga('send', 'pageview');
		</script>
	
  </head>
  <body>
      <div class="container">
          <header>
                              <div class="col-sm-7 title">
                    <div class="col-sm-6">
                        <a title="La Jiribilla" href="/"><img class="img-responsive pull-left" src="/themes/jiribilla/assets/img/la-jiribilla.png" alt="La Jiribilla" /></a>                    </div>
                    <div class="year_number_date pull-left">
													<div class="numb_place">
								Año XII. La Habana, Cuba							</div>
                        
                            <div class="title_number">
                                843                            </div>
                            <div class="numb_date">
                                14 de Marzo al 04 de Abril del 2018                            </div>
                                                
                    </div>
                </div>
                <div class="col-sm-2 jiri-comunidad">
                    <p class="text-center">JIRI-COMUNIDAD</p>
                                            <div class="col-sm-12">
																														<a title="La Jiribilla en Twitter" target="_blank" href="http://twitter.com/lajiribilla"><img src="/uploads/appearance/54585289d6a68icono_twitter.jpg" alt="La Jiribilla en Twitter" /></a>                            																<a title="La Jiribilla en Facebook" target="_blank" href="http://www.facebook.com/www.lajiribilla.cu"><img src="/uploads/appearance/546239d62120ficono_facebook.jpg" alt="La Jiribilla en Facebook" /></a>                            																<a title="La Jiribilla en Youtube" target="_blank" href="http://www.youtube.com/jiribillacubana"><img src="/uploads/appearance/54623a6edf5b7icono_youtube.jpg" alt="La Jiribilla en Youtube" /></a>                            																<a title="La Jiribilla en Flickr" target="_blank" href="http://www.flickr.com/photos/lajiribilla/sets/"><img src="/uploads/appearance/54623ac915b29icono_flirck.jpg" alt="La Jiribilla en Flickr" /></a>                            								</div><div class='clearfix'></div><div class='col-sm-12'>								<a title="Suscribirse por correo" target="_self" href="/suscribirse-por-correo"><img src="/uploads/appearance/54623b8e36089icono_correo.jpg" alt="Suscribirse por correo" /></a>                            																<a title="RSS" target="_blank" href="http://lajiribilla.cu/rss"><img src="/uploads/appearance/54623ce198530icono_rss.jpg" alt="RSS" /></a>                            																<a title="Sitios Web recomendados" target="_self" href="/sitios-web-recomendados"><img src="/uploads/appearance/54675a925a095icono_enlaces.jpg" alt="Sitios Web recomendados" /></a>                            																<a title="Que es la Jiribilla" target="_self" href="/que-es-la-jiribilla"><img src="/uploads/appearance/54675ae647904icono_fav.jpg" alt="Que es la Jiribilla" /></a>                                                    </div>
                </div>
                                                        <div class="col-sm-3 anterior" style="background-size: cover; background-image: url('/uploads/number/5a8716f54aeddanterior 842.jpg'); background-position: right; background-repeat: no-repeat;">
                        <p class="lasted_title">ANTERIORES</p>
						                        <a href="/numeros/842" class="old_edition_number">n.842</a>
						<div class="old_edition_description">
															</div>
                        <a class="hemeroteca_link" href="/numeros">HEMEROTECA >>></a>
                    </div>
                          </header>
                     <div class="main-menu">
              <div class="col-md-8 main-content-home">
                  <ul class="main-menu-ul">
                                            <li><a title="Noticias" target="_self" href="/diario-que-diario"><img src="/uploads/appearance/549ee59cb621abt_noticias_on.gif" alt="Noticias" /></a></li>
                                            <li><a title="Noticias" target="_self" href="/cartelera"><img src="/uploads/appearance/549ee5d6943d6bt_cartelera_on.gif" alt="Noticias" /></a></li>
                                            <li><a title="Musica" target="_self" href="/discoteca-mp3-de-la-jiribilla"><img src="/uploads/appearance/549ee6341f10dbt_mp3_on.gif" alt="Musica" /></a></li>
                                            <li><a title="Libros digitales" target="_self" href="http://www.lajiribilla.co.cu/sumario/anteriores_c.asp?Seccion=3"><img src="/uploads/appearance/567870f0029de549ee66d7ff3fbt_digitales_on.gif" alt="Libros digitales" /></a></li>
                                            <li><a title="La Jiribilla de Papel" target="_self" href=" http://www.epoca2.lajiribilla.cu/jiribilla-de-papel"><img src="/uploads/appearance/567870ce951c2549ee6e7cbe59bt_papel_on.gif" alt="La Jiribilla de Papel" /></a></li>
                                            <li><a title="Videoteca" target="_self" href="/videoteca-de-la-jiribilla"><img src="/uploads/appearance/549ee755dfb3bbt_video_on.gif" alt="Videoteca" /></a></li>
                                            <li><a title="Especiales" target="_self" href="/especiales"><img src="/uploads/appearance/549ee7a7d45c8bt_especiales_on.gif" alt="Especiales" /></a></li>
                                      </ul>
              </div>
              <div id="searching" class="col-md-4 sidebar-home">
                  <form action="/buscar" method="GET">
                      <div class="input-group input-group-sm">
							<input type="text" class="form-control" name="q" value="">
							<span class="input-group-btn">
								<button class="btn btn-default" type="submit"><i class="fa fa-search"></i></button>
							</span>
                      </div>
                  </form>
              </div>
              <div class="clearfix"></div>
          </div>
          <div class="home-main-content">
					<br/>
        <div class="col-sm-8 main-content-home padding-left-0 padding-right-0">
                    <div id="articulos-especiales">
                        <div class="home_articles" style="background-image: url('/uploads/article/5ab116e4ca253E_Bejarano2.jpg'); background-repeat: no-repeat;height: 300px;">
            
                    
        </div>
                        <div class="home_articles" style="background-color: #e7e1db;">
            
                                
                                                                    <div class="home_subtitle"><p><br />
&nbsp;</p>
</div>
                                        
                					                                            <div class="home_title"><a href="/articulo/bejarano-eco-y-renacimiento"><p><span style="color:#552606">Bejarano: eco y renacimiento</span></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">Maikel Jos&eacute; Rodr&iacute;guez Calvi&ntilde;o</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">Bejarano construye una sobrecogedora met&aacute;fora sobre el devenir de la cultura Occidental, inmersa en una profunda crisis de creatividad y espiritualidad.</span></p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="background-color: #e7e1db;">
            
                                
                                                                    <div class="home_subtitle"><p><br />
&nbsp;</p>
</div>
                                        
                					                                            <div class="home_title"><a href="/articulo/reencuentros"><p><span style="color:#552606">Reencuentros</span></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">David Mateo</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">Agust&iacute;n Bejarano fue uno de los principales artistas con los que mantuvimos una estrecha relaci&oacute;n profesional; por eso me satisface mucho que &eacute;l haya decidido dedicar esta &uacute;ltima exposici&oacute;n suya, de cierto car&aacute;cter retrospectivo, decantador, a la memoria del excepcional cr&iacute;tico de arte, y que me haya invitado a escribir unas palabras para el cat&aacute;logo.</span></p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="background-color: #e7e1db;">
            
                                
                                                                    <div class="home_subtitle"><p><br />
&nbsp;</p>
</div>
                                        
                					                                            <div class="home_title"><a href="/articulo/le-acompano-en-su-sentimiento"><p><span style="color:#552606">Le acompa&ntilde;o en su sentimiento</span></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">Rufo Caballero</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">La historia del arte cubano est&aacute; hecha, en buena medida, de circunstancias y coyunturas. Si por alg&uacute;n raro episodio de su balbuciente talento, el artista logra instituirse como vaca sagrada, lo ser&aacute; por mucho tiempo, gracias a la vaporosa levedad de la eternidad insular.</span></p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="height: 195px;background-color: #e7e1db;">
            
            
                                
                                                                    <div class="home_subtitle" ><p><br />
&nbsp;</p>
</div>
                                        
                                    							<div class="home_title"><a href="/galeria/galeria-bejarano"><p style="text-align:center"><span style="color:#552606">En virtud del fuego</span></p>
</a></div>
                                    
                                         <div class="home_author"></div>    
                                                                    							                                    
                
                  
                    <div class="row gallery-container">
																							<div class="col-sm-3 gallery-dossier"> 
											<a href="/galeria/galeria-bejarano">
												<img class="preview" src="/uploads/gallery/5ab12e701a5a2prev_003.jpg" alt="Galería Bejarano" />											</a>
										</div>
																	<div class="col-sm-3 gallery-dossier"> 
											<a href="/galeria/galeria-bejarano">
												<img class="preview" src="/uploads/gallery/5ab12e7021a8cprev_004.jpg" alt="Galería Bejarano" />											</a>
										</div>
																	<div class="col-sm-3 gallery-dossier"> 
											<a href="/galeria/galeria-bejarano">
												<img class="preview" src="/uploads/gallery/5ab2a4f48aa9bprev_001.jpg" alt="Galería Bejarano" />											</a>
										</div>
																	<div class="col-sm-3 gallery-dossier"> 
											<a href="/galeria/galeria-bejarano">
												<img class="preview" src="/uploads/gallery/5ab2a4f4aa676prev_002.jpg" alt="Galería Bejarano" />											</a>
										</div>
													                    </div>
                    
                                    
                
                    
        </div>
            					<div class="dashed_separator2"></div>
                    <div class="clearfix"></div>
                                </div>
           
        <!--BEGIN DOSSIER -->
        <div class="begin_dossier" >
            <div class="col-sm-12 padding-left-0 padding-right-0">
                                        <div class="imagen_number_portada">
                            <img width="100%" src="/uploads/page/5aafb8935c175d_periodismo.jpg" alt="843" />                        </div>
                                <!--BEGIN BACKGROUND IMAGE -->
                                <!--BEGIN STYLE DOSSIER -->
                <div class="background-dossier-home" style="background-image: url('/uploads/number/5aa82b31461dbback.jpg');background-position: center; background-size: 100%;">
                                                            						<div id="dossier-articulos">
							
        <div class="home_articles" >
            
            
                                
                                                                    <div class="home_subtitle" style="color: #000000"><p><br />
Entrevista a Joel del R&iacute;o, periodista y cr&iacute;tico de cine</p>
</div>
                                        
                					                                            <div class="home_title"><a style="color: #ffffff" href="/articulo/es-preciso-abandonar-la-soberbia-de-hablar-con-el-espectador-mirandolo-por-encima-del-hombro"><p>Es preciso abandonar la soberbia de hablar con el espectador mir&aacute;ndolo por encima del hombro</p>
</a></div>
                                    
                                                            <div class="home_author" style="color: #000000"><p><span style="font-family:open sans,sans-serif; font-size:12px">La Jiribilla</span></p>
</div>    
                                    
                                    <div class="home_sumary" style="color: #000000"><p>Lo universal y lo cubano estuvieron combinados siempre en lo mejor del cine cubano y si vamos a buscar las razones de ciertos momentos de declive, estancamiento o decadencia, estos coincidir&aacute;n con la falta de informaci&oacute;n de los creadores. Se necesita ver mucho cine, m&aacute;s all&aacute; de Hollywood, el Oscar y el Paquete Semanal.</p>
</div>
                                    </div>
                                			<div class="dashed_separator" style="color:#000000">.......................................................................</div>
		                    <div class="clearfix"></div>
                    
        <div class="home_articles" >
            
            
                                
                                                                    <div class="home_subtitle" style="color: #000000"><p>Entrevista a Luis Sexto</p>
</div>
                                        
                					                                            <div class="home_title"><a style="color: #ffffff" href="/articulo/el-periodista-es-un-servidor-publico">“El periodista es un servidor público”</a></div>
                                    
                                                                                        <div class="home_author" style="color: #000000"> Flor de Paz</div>  
                                                                                              
                                    <div class="home_sumary" style="color: #000000"><p>Luis Sexto, es el primer protagonista de una serie audiovisual que comenz&oacute; a grabarse recientemente, para dignificar la labor de los periodistas cubanos en el a&ntilde;o de su X Congreso. Se trata, al mismo tiempo, de una alianza creativa entre la Uni&oacute;n de Periodistas de Cuba (UPEC), la Asociaci&oacute;n Hermanos Sa&iacute;z (AHS), la revista digital La Jiribilla y un grupo de talentosos j&oacute;venes egresados de la Facultad de medios Audiovisuales del ISA (FAMCA).</p>
</div>
                                    </div>
                                			<div class="dashed_separator" style="color:#000000">.......................................................................</div>
		                    <div class="clearfix"></div>
                    
        <div class="home_articles" >
            
            
                                
                                                        
                					                                            <div class="home_title"><a style="color: #ffffff" href="/articulo/para-que-perfeccionar-la-prensa-1">¿Para qué perfeccionar la prensa?</a></div>
                                    
                                                                                                                    <div class="home_author"><a style="color: #000000" href="/autor/luis-toledo-sande">Luis Toledo Sande</a></div>
                         							                                    
                                    <div class="home_sumary" style="color: #000000"><p>Atenta a los hechos, y sin ahogarse en las enga&ntilde;osas aguas del positivismo, la prensa ha de ser creativa, coherente y audaz, como el discurso pol&iacute;tico con el que, llamada a iluminarlo, se vincula.</p>
</div>
                                    </div>
                                			<div class="dashed_separator" style="color:#000000">.......................................................................</div>
		                    <div class="clearfix"></div>
                    
        <div class="home_articles" >
            
            
                                
                                                                    <div class="home_subtitle" style="color: #000000"><p><strong>&nbsp;</strong>Entrevista a Andr&eacute;s Machado Conte</p>
</div>
                                        
                					                                            <div class="home_title"><a style="color: #ffffff" href="/articulo/con-amenidad-revolucionaria-ese-es-el-periodismo-que-yo-siempre-quiero-hacer">Con amenidad revolucionaria…ese es el periodismo que yo siempre quiero hacer</a></div>
                                    
                                                                                                                    <div class="home_author"><a style="color: #000000" href="/autor/ana-maria-dominguez-cruz">Ana María Domínguez Cruz</a></div>
                         							                                    
                                    <div class="home_sumary" style="color: #000000"><p>Reconoce &eacute;l, Machado Conte, que ahora existen m&aacute;s espacios en los medios de comunicaci&oacute;n para ejercer el periodismo cultural. &ldquo;Y me alegra. Nos hac&iacute;a falta&rdquo;.&nbsp;Crear. Es otra pol&eacute;mica.</p>
</div>
                                    </div>
                                			<div class="dashed_separator" style="color:#000000">.......................................................................</div>
		                    <div class="clearfix"></div>
                    						</div>
					                </div>
                <!--END DOSSIER -->
                                </div>
        </div>
							<div class="dashed_separator2">&nbsp;</div>
				<div id="fuera-dossier-home-central" class="background-dossier-home">
					        <div class="home_articles" style="background-image: url('/uploads/article/5aa7ecebe739bE_fiesta_del_tambor.jpg'); background-repeat: no-repeat;height: 300px;">
            
                    
        </div>
                        <div class="home_articles" style="background-color: #009b3a;">
            
                                
                                                                    <div class="home_subtitle"><p><br />
&nbsp;</p>
</div>
                                        
                					                                            <div class="home_title"><a href="/articulo/maestria-e-innovacion-en-la-percusion-cubana-1"><p><a href="http://www.lajiribilla.cu/articulo/maestria-e-innovacion-en-la-percusion-cubana"><span style="color:#ffffff">Maestr&iacute;a e innovaci&oacute;n en la percusi&oacute;n cubana</span></a></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">Pedro de la Hoz</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">En los inicios, y por mucho tiempo, los percusionistas, aun siendo imprescindibles, eran valorados por el cumplimiento de su funci&oacute;n m&aacute;s que por el virtuosismo o la innovaci&oacute;n.</span><br />
&nbsp;</p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="background-color: #009b3a;">
            
                                
                                                        
                					                                            <div class="home_title"><a href="/articulo/la-rumba-llego-1"><p><a href="http://www.lajiribilla.cu/articulo/la-rumba-llego"><span style="color:#ffffff">La Rumba lleg&oacute;</span></a></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">Rafael Lam</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">Cuba es una potencia r&iacute;tmica, posee muchos g&eacute;neros musicales. La rumba es una de las m&uacute;sicas m&aacute;s cercanas a la ra&iacute;ces africana. Quiz&aacute;s la rumba no fue creada en los barracones de esclavos, necesitaba un clima m&aacute;s apropiado; pero la m&uacute;sica se va gestando, se va cocinando en el alma y la mente de los hombres donde la sangre bulle.</span><br />
&nbsp;</p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="background-color: #009b3a;">
            
                                
                                                                    <div class="home_subtitle"><p><span style="color:#000000">Entrevista a Giraldo Piloto, presidente de La Fiesta del Tambor</span></p>
</div>
                                        
                					                                            <div class="home_title"><a href="/articulo/la-percusion-es-la-columna-vertebral-de-la-musica-cubana-1"><p><a href="http://www.lajiribilla.cu/articulo/la-percusion-es-la-columna-vertebral-de-la-musica-cubana"><span style="color:#ffffff">La percusi&oacute;n es la columna vertebral de la m&uacute;sica cubana</span></a></p>
</a></div>
                                    
                                                            <div class="home_author"><p><span style="color:#000000">Ana Mar&iacute;a Dom&iacute;nguez Cruz</span></p>
</div>    
                                    
                                    <div class="home_sumary"><p><span style="color:#000000">El percusionista, arreglista, productor musical y l&iacute;der de Kl&iacute;max lamenta que existan prejuicios en el seno de la academia.</span><br />
&nbsp;</p>
</div>
                                    
                    
        </div>
                        <div class="home_articles" style="background-color: #009b3a;">
            
                                
                                                        
                
                                                                                    							                                    
                                    <div class="home_sumary"><table align="center" border="0" cellpadding="1" cellspacing="10" style="height:80px; width:600px">
	<tbody>
		<tr>
			<th style="height:0px; text-align:center; width:50px">
			<p><a href="http://www.lajiribilla.cu/articulo/la-percusion-es-mi-pasion"><img alt="" src="/uploads/article/2018/842/pv_vox populi-brenda-navarrete.jpg" style="height:120px; width:120px" /></a></p>
			</th>
			<td style="height:30px; text-align:center; width:50px">
			<p><a href="http://www.lajiribilla.cu/articulo/hey-degnis-me-gustan-tus-golpes"><img alt="" src="/uploads/article/2018/842/pv_vox populi-degnis.jpg" style="height:120px; width:120px" /></a></p>
			</td>
			<td style="height:30px; text-align:center; width:50px">
			<p><a href="http://www.lajiribilla.cu/articulo/rodney-barreto-canasta"><img alt="" src="/uploads/article/2018/842/pv_vox populi-Rodney-Barreto.jpg" style="height:120px; width:120px" /></a></p>
			</td>
			<td style="height:30px; text-align:center; width:50px">
			<p><a href="http://www.lajiribilla.cu/articulo/dedicacion-y-perseverancia-claves-en-el-estudio-musical"><img alt="" src="/uploads/article/2018/842/pv_vox populi-ruy.jpg" style="height:120px; width:120px" /></a></p>
			</td>
		</tr>
		<tr>
			<td style="height:50px; text-align:center; width:50px"><a href="http://www.lajiribilla.cu/articulo/la-percusion-es-mi-pasion"><span style="color:#000000">Brenda Navarrete:<br />
			La percusi&oacute;n es mi pasi&oacute;n</span></a></td>
			<td style="height:50px; text-align:center; width:50px"><a href="http://www.lajiribilla.cu/articulo/hey-degnis-me-gustan-tus-golpes" style="text-align: center;"><span style="color:rgb(0, 0, 0)">&iexcl;Hey, Degnis, me gustan tus &ldquo;golpes&rdquo;!</span></a></td>
			<td style="height:50px; text-align:center; width:50px"><a href="http://www.lajiribilla.cu/articulo/rodney-barreto-canasta" style="text-align: center;"><span style="color:rgb(0, 0, 0)">Rodney Barreto, &iexcl;canasta!</span></a></td>
			<td style="height:50px; text-align:center; width:50px"><a href="http://www.lajiribilla.cu/articulo/dedicacion-y-perseverancia-claves-en-el-estudio-musical"><span style="color:#000000">Ruy L&oacute;pez-Nussa:</span><br />
			&nbsp;<span style="color:rgb(0, 0, 0)">Dedicaci&oacute;n y perseverancia</span></a></td>
		</tr>
	</tbody>
</table>

<p>&nbsp;</p>
</div>
                                    
                    
        </div>
            					
                    <div class="dashed_separator2"></div>
                    <div class="clearfix"></div>
                            <div class="home_articles" style="background-image: url('/uploads/article/5ab2a8ac241eeLa tertulia.jpg'); background-repeat: no-repeat;height: 236px;">
            
                    
        </div>
                        <div class="home_articles" style="background-color: #040619;">
            
                                
                                                        
                
                                                                                    							                                    
                                    <div class="home_sumary"><p><br />
<a href="http://www.lajiribilla.cu/articulo/a-continuacion"><span style="color:#ffffff">Con frecuencia escuchamos decir que la poes&iacute;a no se lee, afirmaci&oacute;n que afecta la circulaci&oacute;n de textos formidables, escasamente promovidos. Nuestra revista ha escogido el D&iacute;a Mundial de la Poes&iacute;a para inaugurar esta nueva secci&oacute;n y compartir cada mi&eacute;rcoles un poema con nuestros lectores. Comenzamos con una pieza formidable del colombiano Fernando Rend&oacute;n, director del Festival Internacional de Poes&iacute;a de Medell&iacute;n.</span></a><br />
&nbsp;</p>
</div>
                                    
                    
        </div>
            					
                    <div class="dashed_separator2"></div>
                    <div class="clearfix"></div>
                            <div class="home_articles" >
            
                    
        </div>
                				</div>
			            
            <div id="secciones-izq-der">
				<div class="dashed_separator2">&nbsp;</div>
                <div class="col-sm-6 background-dossier-home padding-left-0 padding-section-izq">
                                <div class="home_section padding-seccion-top-5">
                                <div style='height: 70px'>
                                            <div class="col-sm-4">
                            <a href="/articulo/grau-politica-y-chivadera-cubiche">
                                <img class="section-images" src="/uploads/section/55c39d05324e6bt_cronica_on.gif" alt="Grau: Política y ‘chivadera cubiche’" />                            </a>
                        </div>
                                        							<div class="home_section_body col-sm-8 padding-left-0 ">
                                                                                                    
                                                                                                            <div class="title-left"><a href="/articulo/grau-politica-y-chivadera-cubiche">Grau: Política y ‘chivadera cubiche’</a></div>
                                                                    
                                                                    																						                                                                                                                    <div class="author-left"><a href="/autor/argelio-santiesteban">Argelio Santiesteban</a></div>
                                                                                                                												 											                                                                                                            
                                                            </div>
                        
                </div>
                                    				<div class="clearfix"></div>
             </div>
                    <div class="home_section padding-seccion-top-5">
                                <div style='height: 70px'>
                                            <div class="col-sm-4">
                            <a href="/articulo/de-todo-coro-son">
                                <img class="section-images" src="/uploads/section/55c39ae48546dbt_compactera_on.gif" alt="De todo coro son" />                            </a>
                        </div>
                                        							<div class="home_section_body col-sm-8 padding-left-0 ">
                                                                                                            <div class="subtitle-left"><p>Coro Nacional de Cuba</p>
</div>
                                                                    
                                                                                                            <div class="title-left"><a href="/articulo/de-todo-coro-son">De todo coro son</a></div>
                                                                    
                                                                    																						                                                                                                                    <div class="author-left"><a href="/autor/joaquin-borges-triana">Joaquín Borges-Triana</a></div>
                                                                                                                												 											                                                                                                            
                                                            </div>
                        
                </div>
                                    				<div class="clearfix"></div>
             </div>
                    <div class="home_section padding-seccion-top-5">
                                <div style='height: 85px' class="col-sm-12">
                                        <div>
                        <a href="/articulo/la-dulce-balada-de-la-calle">
                            <img class="section_cuento_poesia" src="/uploads/section/550dbc71255a8bt_poesia1_on.gif" alt="La dulce balada de la calle" />                        </a>
                    </div>
                                                            <div class="home_section_body">
                                                                            
                                                                                    <div class="title"><a href="/articulo/la-dulce-balada-de-la-calle">La dulce balada de la calle</a></div>
                                                    
                                                                                                                                <div class="author">Gustavo Pereira</div>  
                                                                                  								
                                                                                                                             
                                            </div>
                                </div>
                <div class="clearfix"></div>
                            </div>
                        </div>
                <div class="col-sm-6 background-dossier-home padding-right-0 padding-section-der">
                                <div class="home_section padding-seccion-top-5">
                                <div style='height: 70px'>
                                            <div class="col-sm-4">
                            <a href="/articulo/esto-es-teatro-o-titeres">
                                <img class="section-images" src="/uploads/section/550dbe5514e81bt_retablo2_on.gif" alt="¿Esto es teatro o títeres?" />                            </a>
                        </div>
                                        							<div class="home_section_body col-sm-8 padding-left-0 ">
                                                                                                    
                                                                                                            <div class="title-left"><a href="/articulo/esto-es-teatro-o-titeres">¿Esto es teatro o títeres?</a></div>
                                                                    
                                                                    																						                                                                                                                    <div class="author-left"><a href="/autor/ruben-dario-salazar">Rubén Darío Salazar</a></div>
                                                                                                                												 											                                                                                                            
                                                            </div>
                        
                </div>
                                    				<div class="clearfix"></div>
             </div>
                    <div class="home_section padding-seccion-top-5">
                                <div style='height: 70px'>
                                            <div class="col-sm-4">
                            <a href="/articulo/elogio-de-la-lectura">
                                <img class="section-images" src="/uploads/section/55c3a036e3fc1bt_opinion.gif" alt="Elogio de la lectura" />                            </a>
                        </div>
                                        							<div class="home_section_body col-sm-8 padding-left-0 ">
                                                                                                    
                                                                                                            <div class="title-left"><a href="/articulo/elogio-de-la-lectura">Elogio de la lectura</a></div>
                                                                    
                                                                    																						                                                                                                                    <div class="author-left"><a href="/autor/antonio-rodriguez-salvador">Antonio Rodríguez Salvador</a></div>
                                                                                                                												 											                                                                                                            
                                                            </div>
                        
                </div>
                                    				<div class="clearfix"></div>
             </div>
                    <div class="home_section padding-seccion-top-5">
                                <div style='height: 85px' class="col-sm-12">
                                        <div>
                        <a href="/articulo/el-mejor-amigo-de-un-muchacho">
                            <img class="section_cuento_poesia" src="/uploads/section/55c39b8b0d0b1bt_cuento_on.gif" alt="El mejor amigo de un muchacho" />                        </a>
                    </div>
                                                            <div class="home_section_body">
                                                                            
                                                                                    <div class="title"><a href="/articulo/el-mejor-amigo-de-un-muchacho">El mejor amigo de un muchacho</a></div>
                                                    
                                                                                                                                <div class="author">Isaac Asimov</div>  
                                                                                  								
                                                                                                                             
                                            </div>
                                </div>
                <div class="clearfix"></div>
                            </div>
                        </div>
            </div>
			    </div>
    <div class="col-sm-4 sidebar-home padding-left-0 padding-right-0">
        
                
        <div id="fuera-dossier-superior">
                        
                        			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/un-insulto-a-marti-concierne-a-toda-nuestra-sociedad" title="Banner Marti">
					<img width="100%" src="/uploads/banner/5ab4280e30536b_Martí.jpg" alt="Banner Marti" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles padding-top-0" >
                                <div style="background-image: url(/uploads/article/5ab4580d6af1eBanner un insulto a Martí.jpg);background-size: cover; background-repeat: no-repeat;">&nbsp;</div>
                                       <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/velasco-donde-una-maravilla-arquitectonica-peligra" title="Banner Velasco">
					<img width="100%" src="/uploads/banner/5ab3d99530556b_Velazco 005.jpg" alt="Banner Velasco" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/orishas-en-la-habana-nuevamente-cuando-lo-urbano-alguna-vez-fue-serio" title="Banner Orishas">
					<img width="100%" src="/uploads/banner/5ab426d97241fb_Orishas.jpg" alt="Banner Orishas" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " style="background-image: url('/uploads/article/5ab3b03991fa6b_Diego Gutiérrez.jpg'); background-size: cover; background-repeat: no-repeat;height: 250px;">
                                <div class="col-sm-12 padding-left-right">
                                
                                                        
                
                                                            															                                     
                            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " style="background-color: #57767f;">
                                <div class="col-sm-12 padding-left-right">
                                
                                                                    <div class="home_subtitle_sidebar"><p><br />
&nbsp;</p>
</div>
                                        
                					                                            <div class="home_title_sidebar"><a href="/articulo/mucho-diego-y-mas"><p style="text-align:center"><span style="color:#d8e1e2">Mucho Diego&hellip; y m&aacute;s</span></p>
</a></div>
                                    
                                                            <div class="home_author_sidebar"><p style="text-align:center"><span style="color:#b6c83d"><span style="font-family:open sans,sans-serif; font-size:12px">Onaisys Fonticoba Gener</span></span></p>
</div>    
                                    
                            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles sidebar-home-articles" style="height: 85px;background-color: #57767f;">
				<div class="col-sm-12 padding-left-right">
				
				
																	
																			<div class="home_title_sidebar"><a href="/galeria/galeria-diego-gutierrez"><p style="text-align:center"><span style="color:#d8e1e2">Sabor salado</span></p>
</a></div>
											
																																	 											
					
					  
						<div class="gallery-sidebar">
																<div> 
                                                                            <a href="/galeria/galeria-diego-gutierrez">
                                                                                <img class="preview-sidebar" src="/uploads/gallery/5ab3c05063972pv_1.jpg" alt="Galeria Diego Gutierrez" />									   </a>
									</div>
																<div> 
                                                                            <a href="/galeria/galeria-diego-gutierrez">
                                                                                <img class="preview-sidebar" src="/uploads/gallery/5ab3c0507e72fpv_2.jpg" alt="Galeria Diego Gutierrez" />									   </a>
									</div>
																<div> 
                                                                            <a href="/galeria/galeria-diego-gutierrez">
                                                                                <img class="preview-sidebar" src="/uploads/gallery/5ab3c0508aa5fpv_4.jpg" alt="Galeria Diego Gutierrez" />									   </a>
									</div>
																<div> 
                                                                            <a href="/galeria/galeria-diego-gutierrez">
                                                                                <img class="preview-sidebar" src="/uploads/gallery/5ab3c05096dc1pv-3.jpg" alt="Galeria Diego Gutierrez" />									   </a>
									</div>
													</div>
					

									
			</div>
				<div class="clearfix"></div>
        </div>
                    <div class="dashed_separator2"></div>    
                
        			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                        </div>
		
                    <div id="diario-que-diario">
                <div class="sidebar-head"><a href="/diario-que-diario"><img class="img-responsive" src="/themes/jiribilla/assets/img/diarioq2.jpg" alt="" /></a></div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/las-profecias-de-trend-kill-para-el-pinar-rock">
																					Las profecías de Trend Kill para el Pinar Rock																				</a>
									</div>
																		                                                                                                                                                                    <div class="home_author_sidebar">Miriela Fernández</div>
                                                                                																										<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/regresa-afrodita-oh-espejo-al-gran-teatro-de-la-habana">
																					<p>Regresa&nbsp;<em>Afrodita, &iexcl;Oh, Espejo!&nbsp;</em>al Gran Teatro de La Habana</p>
																				</a>
									</div>
																		                                                                                                                                                                    <div class="home_author_sidebar">La Jiribilla</div>
                                                                                																										<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/otro-marzo-de-havana-world-music-black-tears">
																					Otro marzo de Havana World Music & Black Tears 																				</a>
									</div>
																		                                                                                                                                                                    <div class="home_author_sidebar">Ailén Rivero </div>
                                                                                																										<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/xvii-muestra-cine-joven-icaic">
																					XVII Muestra Cine Joven ICAIC																				</a>
									</div>
																			<div class="home_author_sidebar">Declaración de la presidencia del ICAIC</div>
									 									<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/collage-habana-promociona-arte-joven-cubano-en-artmadrid-18">
																					<p><em>Collage Habana&nbsp;</em>promociona arte joven cubano en ArtMadrid&acute;18</p>
																				</a>
									</div>
																			<div class="home_author_sidebar">FCBC</div>
									 									<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p><br />
&nbsp;</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/declaracion-final-del-ii-foro-de-actores-sociales-y-de-la-sociedad-civil-cubana-pensando-americas">
																					Declaración Final del II Foro de Actores Sociales y de la Sociedad Civil Cubana “Pensando Américas”																				</a>
									</div>
																		                                                                                                                                                                    <div class="home_author_sidebar">La Jiribilla</div>
                                                                                																										<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                                                <div class="sidebar-home-articles">
                                <div class="col-sm-12 padding-left-right">
																			<div class="home_subtitle_sidebar">
											<p>&nbsp;</p>

<p>&nbsp;</p>

<p>Luego de tres a&ntilde;os de ausencia</p>
										</div>
									                                                                        <div class="home_title_sidebar"><a href="/noticias/silvio-rodriguez-vuelve-a-chile-con-dos-conciertos">
																					Silvio Rodríguez vuelve a Chile con dos conciertos																				</a>
									</div>
																		                                                                                                                                                                    <div class="home_author_sidebar">La Jiribilla</div>
                                                                                																										<div class="home_sumary_sidebar">
																			</div>    
                                </div>
								<div class="clearfix"></div>
                            </div>
                    					<div class="footer_more">
						<a class="pull-right" href="/diario-que-diario">:: VER TODAS...</a>
						<div class="clearfix"></div>
					</div>
            </div>
            
       		
        <div id="fuera-dossier-normal">
                        
                        			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/amo-la-vocacion-ecumenica-de-la-poesia" title="Banner Pilar Bonnett">
					<img width="100%" src="/uploads/banner/5ab1705d04638b_Piedad_Bonnett.jpg" alt="Banner Pilar Bonnett" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/aire-frio-la-maldita-circunstancia-del-calor-por-todas-partes" title="Banner aire frio">
					<img width="100%" src="/uploads/banner/5ab0158eaa63fb_Aire frío.jpg" alt="Banner aire frio" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/la-soledad-del-editor-de-fondo" title="Banner Macana 843">
					<img width="100%" src="/uploads/banner/5aa7ef6e65d8ab_macana.jpg" alt="Banner Macana 843" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/beisbol-cubano-sin-fama-y-sin-historia" title="Banner beisbol">
					<img width="100%" src="/uploads/banner/5ab00d05464a2b_Béisbol.jpg" alt="Banner beisbol" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			<div class="home_articles">
				<a href="http://www.lajiribilla.cu/articulo/la-gente-asi-no-puede-convocar-otra-cosa-que-afecto" title="Banner Palabras Despedida">
					<img width="100%" src="/uploads/banner/5ab3e4bf80e152.jpg" alt="Banner Palabras Despedida" />				</a>
				<div class="dashed_separator2"></div>    
			</div>
                    			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
            					
        </div>
			<div class="clearfix"></div>
        </div>
                        </div>
        <div id="fuera-dossier-inferior">
            <!--<div class="fidel-articles">-->
<!--    --><!--            <div class="subtitle_reflection">--><!--</div>-->
<!--    --><!--    --><!--            <div class="title_reflection"><a href="/reflexiones/--><!--">--><!--</a></div>-->
<!--    --><!--    --><!--            <div class="sumary_reflection">--><!--</div>-->
<!--    --><!--    --><!--    <div class="fecha_reflection">-->
<!--            --><!--    </div>-->
<!--</div>-->                        			
		<div class="home_articles sidebar-home-articles " >
                                <div class="col-sm-12 padding-left-right">
                                
                                                        
                					                                            <div class="home_title_sidebar"><a href="/articulo/ver-mas-fuera-de-dossier-843-1"><p style="text-align:right"><a href="http://www.lajiribilla.cu/articulo/ver-mas-fuera-de-dossier-843-1">:: ver m&aacute;s</a></p>
</a></div>
                                    
                                                            															                                     
                            					
        </div>
			<div class="clearfix"></div>
        </div>
                								           
        </div>
    </div>
</div>
<!--<div class="image-15b">-->
<!--    <img class="img-responsive" src="--><!--/uploads/banner/banner_15/--><!--"/>-->
<!--</div>-->
<div class="row">
    <div id="cierre" class="col-sm-12 background-dossier-home">
            </div>
</div>
          <footer class="text-center">
              
                  <div class="col-sm-3 most-popular">
                        <h4>Art&iacute;culos m&aacute;s visitados</h4>
                                                                            <ul class="list-unstyled">
                                <li><a href="/articulo/elena-siempre-fue-por-mas">Elena siempre fue por más</a></li>
                            </ul>
                                                    <ul class="list-unstyled">
                                <li><a href="/articulo/el-periodismo-tiene-que-ser-critico-su-ejercicio-implica-asumir-una-postura-politica">El periodismo tiene que ser crítico. Su ejercicio implica asumir una postura política</a></li>
                            </ul>
                                                    <ul class="list-unstyled">
                                <li><a href="/articulo/el-monstruo-y-sus-entranas">La causa profunda de todas las guerras</a></li>
                            </ul>
                                                    <ul class="list-unstyled">
                                <li><a href="/articulo/es-preciso-abandonar-la-soberbia-de-hablar-con-el-espectador-mirandolo-por-encima-del-hombro">Es preciso abandonar la soberbia de hablar con el espectador mirándolo por encima del hombro</a></li>
                            </ul>
                                                    <ul class="list-unstyled">
                                <li><a href="/articulo/pandora-su-caja-y-la-critica-musical-en-cuba">Pandora, su Caja y la crítica musical en Cuba</a></li>
                            </ul>
                                          </div>
                  <div class="col-sm-3 most-popular">
                        <h4>Galer&iacute;as m&aacute;s visitadas</h4>
                                                                            <ul class="list-unstyled">
                                <li><a href="/galeria/galeria-bejarano">Galería Bejarano</a></li>
                            </ul>
                                                    <ul class="list-unstyled">
                                <li><a href="/galeria/galeria-diego-gutierrez">Galeria Diego Gutierrez</a></li>
                            </ul>
                                          </div>
                  <div class="col-sm-3 most-popular">
                      <h4>Videos m&aacute;s visitados</h4>
                                                                    <ul class="list-unstyled">
                            <li><a href="/video/programa-23-y-m-television-cubana">Programa 23 y M, Televisión Cubana</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/video/vuelven-los-conciertos-a-el-patio-de-baldovina">Vuelven los conciertos a El Patio de Baldovina</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/video/sumertime-sala-jose-white-matanzas">Sumertime. Sala José White, Matanzas</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/video/amazing-grace-sala-jose-white-matanzas">Amazing grace, Sala José White, Matanzas</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/video/chan-chan-teatro-terry-cienfuegos">Chan Chan. Teatro Terry, Cienfuegos</a></li>
                        </ul>
                                        </div>
                  <div class="col-sm-3 most-popular">
                      <h4>M&uacute;sica m&aacute;s visitada</h4>
                                                                    <ul class="list-unstyled">
                            <li><a href="/musica/desolvidando-ese-pariente-llamado-sindo-garay">Desolvidando: Ese pariente llamado Sindo Garay</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/musica/desolvidando-pineiro-nacional">Desolvidando: Piñeiro, Nacional</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/musica/enrique-bryon">Desolvidando: El misterioso Enrique Bryon</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/musica/desolvidando-palabras-corona-y-trovuntivitis">DESOLVIDANDO: Palabras: Corona y Trovuntivitis</a></li>
                        </ul>
                                              <ul class="list-unstyled">
                            <li><a href="/musica/desolvidando-antonio-maria-romeu">Desolvidando: Antonio María Romeu</a></li>
                        </ul>
                                            
                  </div>
              <div class="clearfix"></div>
              <br/>
              <p><a href="mailto:lajiribilla@lajiribilla.cu">La Jiribilla</a>. Revista de Cultura Cubana<br/>
                  ISSN 2218-0869. La Habana, Cuba. 2001-2018.
              </p>
          </footer>
      </div>
  </body>
</html>