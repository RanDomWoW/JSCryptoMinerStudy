#include "hip/hip_runtime.h"
  
  

<!DOCTYPE html PUBLIC "-//W3C//DTD XHTML 1.0 Transitional//EN" "http://www.w3.org/TR/xhtml1/DTD/xhtml1-transitional.dtd">
<html xmlns="http://www.w3.org/1999/xhtml" xml:lang="es-es" lang="es-es" dir="ltr">
<head>
   <base href="http://www.acn.cu/" />
  <meta http-equiv="content-type" content="text/html; charset=utf-8" />
  <meta name="keywords" content="Cuba, agencias de noticias, ACN, informaciones, reportajes, Fidel Castro, Raúl Castro, deportes, cultura, ciencias, medio- ambiente, curiosidades" />
  <meta name="rights" content="Agencia Cubana de Noticias ACN" />
  <meta name="description" content="Sitio oficial de la Agencia Cubana de Noticias ACN, Genera, procesa y ofrece informaciones, comentarios y reportajes  en soportes como texto, imagen, audio y vídeo.
" />
  <meta name="generator" content="Joomla! - Open Source Content Management" />
  <title>Cuba, Agencia Cubana de Noticias - Inicio - ACN</title>
  <link href="/?format=feed&amp;type=rss" rel="alternate" type="application/rss+xml" title="RSS 2.0" />
  <link href="/?format=feed&amp;type=atom" rel="alternate" type="application/atom+xml" title="Atom 1.0" />
  <link href="/templates/acnfull3/favicon.ico" rel="shortcut icon" type="image/vnd.microsoft.icon" />
  <link rel="stylesheet" href="/modules/mod_newscalendar/css/blue-arrows.css" type="text/css" />
  <link rel="stylesheet" href="http://www.acn.cu/modules/mod_yj_newsflash2/css/stylesheet.css" type="text/css" />
  <link rel="stylesheet" href="http://www.acn.cu/modules/mod_yj_nf_ultimate/css/stylesheet.css" type="text/css" />
  <link rel="stylesheet" href="http://www.acn.cu/modules/mod_rssearch/assets/css/rssearch.css" type="text/css" media="screen"  />
  <style type="text/css">
.random_image_extended {text-align: center; margin: 10px 0 !important;}
.random_image_extended_small {text-align: right; font-size: 0.85em !important; margin-top: 15px !important;}

  </style>
  <script src="/administrator/components/com_remoteimage/includes/js/remoteimage-admin.js" type="text/javascript"></script>
  <script src="/media/system/js/mootools-core.js" type="text/javascript"></script>
  <script src="/media/system/js/core.js" type="text/javascript"></script>
  <script src="/media/system/js/caption.js" type="text/javascript"></script>
  <script src="/modules/mod_newscalendar/assets/newscalendar.js" type="text/javascript"></script>
  <script src="/media/system/js/mootools-more.js" type="text/javascript"></script>
  <script src="http://www.acn.cu/modules/mod_rssearch/assets/js/ajax.js" type="text/javascript"></script>
  <script type="text/javascript">
window.addEvent('load', function() {
				new JCaption('img.caption');
			});
var sfolder = '';
  </script>
  <link rel="stylesheet" href="/modules/mod_random_image_extended/slimbox/css/slimbox.css" type="text/css" media="screen" />
<script type="text/javascript" src="/modules/mod_random_image_extended/slimbox/js/slimbox.js"></script>

<meta property="fb:pages" content="111896838909504" /> 
 <link rel="stylesheet" href="/templates/system/css/system.css" type="text/css" />
 <link rel="stylesheet" href="/templates/system/css/general.css" type="text/css" />
 <link rel="stylesheet" type="text/css" href="/templates/acnfull3/css/template.css" media="screen" />

 <!--[if IE 6]><link rel="stylesheet" href="/templates/acnfull3/css/template.ie6.css" type="text/css" media="screen" /><![endif]-->
 <!--[if IE 7]><link rel="stylesheet" href="/templates/acnfull3/css/template.ie7.css" type="text/css" media="screen" /><![endif]-->
 <!-- Google Analytics -->
<script>
  (function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
  (i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
  m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
  })(window,document,'script','https://www.google-analytics.com/analytics.js','ga');

  ga('create', 'UA-86398795-1', 'auto');
  ga('send', 'pageview');

</script>
 <!-- Piwik --> 
<script type="text/javascript"> 
   var _paq = _paq || []; 
   _paq.push(["setDocumentTitle", document.domain + "/" + document.title]); 
   _paq.push(["setCookieDomain", "*.www.ain.cu"]); 
   _paq.push(["setDomains", ["*.www.ain.cu"]]); 
   _paq.push(['trackPageView']); 
   _paq.push(['enableLinkTracking']); 
   (function() { 
     var u=(("https:" == document.location.protocol) ? "https" : "http") 
+ "://piwikstats.cip.cu/"; 
     _paq.push(['setTrackerUrl', u+'piwik.php']); 
     _paq.push(['setSiteId', 115]); 
     var d=document, g=d.createElement('script'), 
s=d.getElementsByTagName('script')[0]; g.type='text/javascript'; 
     g.defer=true; g.async=true; g.src=u+'piwik.js'; 
s.parentNode.insertBefore(g,s); 
   })(); 
</script> 
<noscript><p><img src="/ http://piwikstats.cip.cu/piwik.php?idsite=115" 
style="border:0;" alt="" /></p></noscript> 
<!-- End Piwik Code -->  
 <script type="text/javascript">if ('undefined' != typeof jQuery) document._artxJQueryBackup = jQuery;</script>
 <script type="text/javascript" src="/templates/acnfull3/jquery.js"></script>
 <script type="text/javascript">jQuery.noConflict();</script>
 <script type="text/javascript" src="/templates/acnfull3/script.js"></script>
 <script type="text/javascript">if (document._artxJQueryBackup) jQuery = document._artxJQueryBackup;</script>
<script>
  window.fbAsyncInit = function() {
    FB.init({
      appId      : '125088631286755',
      xfbml      : true,
      version    : 'v2.7'
    });
  };

  (function(d, s, id){
     var js, fjs = d.getElementsByTagName(s)[0];
     if (d.getElementById(id)) {return;}
     js = d.createElement(s); js.id = id;
     js.src = "//connect.facebook.net/en_US/sdk.js";
     fjs.parentNode.insertBefore(js, fjs);
   }(document, 'script', 'facebook-jssdk'));
</script> 
<script type="text/javascript">
        // SeeVolution Script 
        svluStyle="color:#000000";
        (function () {
            var lc = document.createElement('script'); lc.type = 'text/javascript'; lc.async = true;
            lc.src = ('https:' == document.location.protocol ? 'https://' : 'http://') + 'cdn.seevolution.com/collector.js';
            var s = document.getElementsByTagName('script')[0]; s.parentNode.insertBefore(lc, s);
        })();
    </script>
</head>
<body>
  <div id="main">
  
    <div class="cleared reset-box"></div>
    <div class="header">
      <div class="header-position">
        <div class="header-wrapper">
          <div class="cleared reset-box"></div>
          <div class="header-inner">
            <a href="/"><div class="headerobject"></div></a>
            <!--div class="logo">
              <h1 class="logo-name" style="font-weight: bold;">AGENCIA CUBANA DE NOTICIAS</h1>
            </div-->
                      
            <div class="hmenu-extra4">

<div class="custom"  >
	<p style="text-decoration: none;"><a class="flag" href="#" title="English"><img class="nothumbnail" src="/images/banners/flags/inglaterra.png" border="0" /> </a><a class="flag" href="http://www.cubanews.acn.cu" title="ACN English">English</a> <a class="flag" href="http://www.cubainfo.acn.cu" title="ACN Français"><img class="nothumbnail" src="/images/banners/flags/francia.png" border="0" /> Français</a> <a class="flag" href="#" title="Ruso"><img class="nothumbnail" src="/images/banners/flags/rusia.png" border="0" /> </a><a class="flag" href="http://www.novosti.acn.cu" title="ACN Русский ">Русский</a></p></div>
</div>
            <div  class="hmenu-extra2" style="padding-top: 11px;position: absolute;left: 796px;" ><script type="text/javascript" charset="utf-8">
	var moduleId = 92;
	var RSFSearch92;

var results_box = document.getElementById('search_suggest92');
var searchReq = getXmlHttpRequestObject();

function searchSuggest92(){
	if(searchReq.readyState == 4 || searchReq.readyState == 0){
		var str = escape(document.getElementById('rsf_inp92').value);
		searchReq.open("GET", '/index.php?option=com_rssearch&view=results&layout=ajax&module_id=92&search='+str, true);
		searchReq.onreadystatechange = function(){
			if(searchReq.readyState == 4){
				var ss = document.getElementById('search_suggest92')

				ss.innerHTML = '';
				if (searchReq.responseText.indexOf('---RSSEARCH_SEPARATOR---') > -1){
					var str = searchReq.responseText.split("---RSSEARCH_SEPARATOR---");
					var suggest = '<p style="position:relative; float: right; cursor:pointer;padding: 5px 10px 2px 1px;margin:0px auto;" onclick="javascript:closeSearch92();"><img src="http://www.acn.cu/modules/mod_rssearch/assets/images/close.png" alt="" /></p>';
					suggest += '<ul style=" text-decoration:none;  text-style:none; width: 305px;  margin: 0.3em 0;  padding: 0; ">';
					
					var results = str.length;
					var limit = 5;
				
					if(results < limit){
						var max = results;
					}else {
						var max = limit;
					}

					for(i=0; i<max; i++){
						suggest += '<li onmouseover="javascript:suggestOver92(this);" ';
						suggest += 'onmouseout="javascript:suggestOut92(this);"';
						suggest += ' style=" background-image: url(\'../ACN%20Noticias/templates/acnfull3/images/postbullets_2.png\');background-repeat: no-repeat; background-position: top left; width: 305px; text-decoration:none; text-align:left; padding: 2px 6px 2px 6px; cursor:pointer;">' + str[i] + '</li>';
					}
					if(results > limit){
						suggest += '<br/><p style="text-align:left;padding: 2px 6px 2px 6px; cursor:pointer;" onclick="javascript:setSearch92(this.innerHTML);">RSF_MODULE_VIEW_ALL_RESULTS</a></p>';
					    suggest += '<p style="position: relative;float: right;cursor: pointer;padding: 0 0 0 0;margin: 0px auto;top: -15px;left: 25px;" onclick="javascript:closeSearch92();"><img src="http://www.acn.cu/modules/mod_rssearch/assets/images/close.png" alt="" /></p>';					
					}
					suggest += '</ul>';									
					ss.set('html',suggest);					RSFSearch92.slideIn();
					
				}
			}
		}; 
		searchReq.send(null);
	}
}

function suggestOver92(div_value){
	div_value.className = 'suggest_link_over';
}

function suggestOut92(div_value){
	div_value.className = 'suggest_link';
}

function setSearch92(value){
	document.getElementById('search_suggest92').innerHTML = '';
	document.getElementById('frmSearch'+92).submit();
}
function closeSearch92(){
	RSFSearch92.slideOut();
}

window.addEvent('domready', function(){
	$('search_suggest92').setStyle('height','auto');
	RSFSearch92 = new Fx.Slide('search_suggest92').hide();
	var parent = $('search_suggest92').getParent('div').setStyle('width','0px');
});
</script>

<div id="rssearch" class="rssearch_box">
	<form id="frmSearch92" action="/" method="GET" accept-charset="utf-8" class="rsf_form">

		<input type="hidden" name="option" value="com_rssearch" />
		<input type="hidden" name="view" value="results" />
		<input type="hidden" name="layout" value="default" />
		<input type="hidden" name="module_id" value="92" />
		<input type="text" id="rsf_inp92" name="search" alt="Enter keywords" onkeyup="searchSuggest92();" autocomplete="on" style="width:127px" />

		 <input  type="submit" class= "button" id="cmdSearch" name="cmdSearch" value="Buscar" />		<input type="hidden" name="Itemid" value="117" />
		<br />
		
		<div style="max-height: 400px; height=auto;  width:357px; ><!--?php style="height: 100px; overflow: auto; width:378px;"?-->
			<div id="search_suggest92" class="rsfsuggestions" style="width:0px;  ">

			</div>
		</div>
	</form>
</div>
</div>            
            <div style=" border-bottom: 1px solid #EBFBFE;height: 20px;position: relative;top: 8px;bottom: 0px;left: 400px;width: 585px;" ></div>                        
            <div class="hmenu-extra2"style="margin-top:3px; margin-left:50px;" ></div>
            <div class="hmenu-extra2" style=" position:relative;float:right; margin-top:13px; right: 170px; border-right: 1px solid #EBFBFE;height: 15px; padding-right: 17px;padding-left: 20px; top:8px;">

<div class="custom"  >
	<div style="bottom: 9px; position: relative; width: 100px;"><a href="/tiempo-en-cuba" title="El Tiempo"><img src="/images/banners/tiempo.png" border="0" alt="" /></a> <a href="/tiempo-en-cuba" title="El Tiempo" style="left: 4px; position: relative; bottom: 9px;">El Tiempo</a></div></div>
</div> 
            <div id="yjdate" style="padding-right: 15px;border-right: 1px solid #EBFBFE;color:black; float:right; font-size:11px; margin-top:22px; margin-right:166px;" >La Habana, Lunes 19 de Marzo de 2018   05:08 am </div>
                    </div>
        </div>
      </div>
    </div>
    <div class="cleared reset-box"></div>
        <div class="bar nav">
      <div class="nav-outer">
        <div class="nav-wrapper">
          <div class="nav-inner">
            <ul class="hmenu"><li class="item-101 current active"><a class=" active" href="/">Inicio</a></li><li class="item-102"><a href="/cuba">Cuba</a></li><li class="item-103"><a href="/economia">Economía</a></li><li class="item-105"><a href="/mundo">Mundo</a></li><li class="item-107"><a href="/cultura">Cultura</a></li><li class="item-106"><a href="/deportes">Deportes</a></li><li class="item-108"><a href="/salud">Salud</a></li><li class="item-109"><a href="/ciencia-y-tecnologia">Ciencia y Tecnología</a></li><li class="item-115"><a href="/medio-ambiente">Medio Ambiente</a></li><li class="item-156"><a href="/galerias">Galerías</a></li><li class="item-256"><a href="/quienes-somos">Conózcanos</a></li></ul>          </div>
        </div>
      </div>
    </div>
    <div class="cleared reset-box"></div>
        <div class="box sheet">
      <div class="box-body sheet-body">  
        <div class="content-layout">
        <div class="content-layout-row" style="">
                    <div style = "background:#D2FDFF;height: 26px;">
            <div class="layout-cell" style="width: 80%; ">
              <div class="nostyle">


<div class="custom"  >
	<p> <strong style="color: #ff0000;">ACN Hoy</strong>  <a href="http://www.acn.cu/alba-tcp">ALBA-TCP</a>    <a href="/elecciones" target="_blank">Elecciones</a> <strong style="color: #ff0000;">                                                                 Otros sitios de ACN   </strong> <a href="http://fotos.acn.cu"><img src="/images/banners/vineta-azul.png" border="0" width="9" height="9" style="margin-right: 5px;" />Fotografia </a>         <a href="http://radio.acn.cu"><img src="/images/banners/vineta-azul.png" border="0" width="9" height="9" style="margin-right: 5px;" />Radio</a><br /><a href="/obama-en-cuba" title="Radio"></a></p>
<div class="moviltext">
<div style="float: left; height: 15px; padding: 0 0px 0 10px;"> </div>
</div></div>
</div>
 
                                                        
            </div>
            <div class="layout-cell" style="width: 20%; ">
                                          <div class="nostyle2">


<div class="custom"  >
	<div>		<div class="moduletable">
					<a href="/?format=feed&amp;type=rss" class="rss-tag-icon syndicate-module"></a>		</div>
	
<div style="position: absolute; right: 11px; top: 5px;"><a href="http://www.facebook.com/ACNCuba" target="_blank" title="Página de Facebook de la Agencia"><img src="/images/banners/facebook-16x16.gif" border="0" alt="" /></a> <a href="https://twitter.com/@ACN_Cuba/" target="_blank" title="Página de Twitter de la Agencia"><img src="/images/banners/twitter-16x16 2.png" border="0" alt="" /></a> <a href="http://www.youtube.com/user/agenciacubana?feature=mhum" target="_blank" title="Canal de Youtube de la Agencia"><img src="/images/banners/youtube4.png" border="0" alt="" /></a></div>
</div></div>
</div>
              
            </div>
          </div>
                  </div></div>
        
        
            
        
                <div class="content-layout-row" style="padding-bottom:10px;">              
          <div class="content-layout">                  
              <div class="layout-cell" style="width: 33%;padding-bottom:5px;">
                              </div>
              <div class="layout-cell" style="width: 33%;">
                                
              </div>                
          </div>
        
          <div class="layout-cell" style="width: 34%;">
                          
          </div>              
        </div>
        
        
                                                
        <div class="layout-wrapper">
          <div class="content-layout">
            <div class="content-layout-row">
              <div class="layout-cell content">


                                
                <div style="float: left;width: 60%;">
                
<div class="blog-featured"><div class="items-leading"><div class="leading-0"><div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<h2 class="postheader"><a href="/salud/32678-convocan-a-foro-de-inversion-extranjera-en-salud" class="PostHeader">Convocan a Foro de inversión extranjera en salud</a></h2>
<div class="postcontent">
<div class="article"><a href="/salud/32678-convocan-a-foro-de-inversion-extranjera-en-salud"><img src="/images/2018/marzo/0318-pabexpo.png" border="0" alt="0318-pabexpo.png" /class="imagefeatured" width="400" height="240" style="float:left;" ></a><p>Por primera vez tendrá lugar en Cuba el Foro de inversiones extranjeras en salud, los días 24 y 25 de abril en el recinto de Pabexpo, como parte de la III Convención Internacional de Salud, que sesionará en la capital en abril venidero.</p>

</div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
</div><span class="rowfeatured-separator"></span></div><div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<div class="postcontent">
<div class="items-more">
		<div class="featured-div">
	
	<h4 style="padding-top:5px;">
		<a class="featured-title" href="/deportes/32677-ciclismo-en-vueltabajo-un-homenaje-a-los-protagonistas-pinarenos">
			Ciclismo en Vueltabajo, un homenaje a los protagonistas pinareños</a>   
	</h4>
	

	<div class="featured-intro">
				<a href="/deportes/32677-ciclismo-en-vueltabajo-un-homenaje-a-los-protagonistas-pinarenos"><img src="/images/2018/marzo/0318-ciclismo.jpg" border="0" alt="0318-ciclismo.jpg" /class="imagefeatured" width="160" height="100" style="float:left;" ></a><p>Un homenaje a atletas, entrenadores y promotores es Ciclismo en Vueltabajo, texto de la autoría de José Antonio Serantes Díaz, corresponsal de Radio Reloj en Pinar del Río, y que constituye el primer material publicado sobre el tema en la provincia.</p>

			</div>
	
			
    </div>	
		<div class="featured-div">
	
	<h4 style="padding-top:5px;">
		<a class="featured-title" href="/deportes/32680-medalla-de-plata-para-cubanos-martinez-y-oropesa-en-el-giraldilla-fotos">
			Medalla de plata para cubanos Martínez y Oropesa en el Giraldilla (+Fotos)</a>   
	</h4>
	

	<div class="featured-intro">
				<a href="/deportes/32680-medalla-de-plata-para-cubanos-martinez-y-oropesa-en-el-giraldilla-fotos"><img src="/images/2018/marzo/0308-leodannis.jpg" border="0" alt="0308-leodannis.jpg" /class="imagefeatured" width="160" height="100" style="float:left;" ></a><p>Los cubanos Leodannis Martínez y Taymara Oropesa debieron conformarse hoy con las medallas de plata en la prueba individual del torneo internacional de bádminton Giraldilla de La Habana, concluido este domingo.</p>

			</div>
	
			
    </div>	
		<div class="featured-div">
	
	<h4 style="padding-top:5px;">
		<a class="featured-title" href="/ciencia-y-tecnologia/32675-datazucar-expondra-en-feria-internacional-de-informatica">
			Datazucar expondrá en Feria Internacional de Informática</a>   
	</h4>
	

	<div class="featured-intro">
				<a href="/ciencia-y-tecnologia/32675-datazucar-expondra-en-feria-internacional-de-informatica"><img src="/images/2018/marzo/0318-informatica-2016.jpg" border="0" alt="0318-informatica-2016.jpg" /class="imagefeatured" width="160" height="100" style="float:left;" ></a><p>Una de las entidades del Grupo Azucarero AZCUBA, ZETI y su División Datazucar, expondrá su cartera de negocios en la XVII edición de la Convención y Feria Internacional sobre Informática.</p>

			</div>
	
			
    </div>	
		<div class="featured-div last">
	
	<h4 style="padding-top:5px;">
		<a class="featured-title" href="/cultura/32673-movimiento-de-instructores-de-arte-entrega-altos-reconocimientos">
			Movimiento de Instructores de Arte entrega altos reconocimientos (+Fotos)</a>   
	</h4>
	

	<div class="featured-intro">
				<a href="/cultura/32673-movimiento-de-instructores-de-arte-entrega-altos-reconocimientos"><img src="/images/2018/marzo/0318-gala-instructores-arte2.JPG" border="0" alt="Yamile García Zurita (C) y Nuria García Reyes (D), integrantes del Proyecto Cultural Comunitario &quot;Vocal Clave de Sol&quot;, del municipio La Lisa, reciben de la Dirección Nacional de la Brigada de Instructores de Arte José Martí, la Distinción de Miembro de Honor, de manos de Liliam Mendoza Estrada (I), presidenta nacional de la brigada, en el Memorial dedicado al Héroe Nacional de Cuba, en La Habana, el 17 de marzo de 2018. ACN FOTO/ Modesto GUTIÉRREZ CABO" title="Movimiento de Instructores de Arte entrega altos reconocimientos" /class="imagefeatured" width="160" height="100" style="float:left;" ></a><p>A la Unión de Jóvenes Comunistas (UJC) y al proyecto Vocal Clave de Sol se les confirió la condición de Miembro de Honor de la Brigada de Instructores de Arte José Martí, por su trabajo y aportes a la formación, superación y desempeño de los instructores de arte.</p>

			</div>
	
			
    </div>	


</div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
</div>                </div   >
                <div style="float: right; width: 40%;" >
                <div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<div class="postcontent">
<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash Ultimate Module for Joomla 1.5 starts here -->
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32681">
						<a class="itemtitle" style="font-size:16px; " href="/cuba/32681-los-santiagueros-en-debate-camino-al-xxi-congreso-de-la-ctc">
				Los santiagueros en debate camino al XXI Congreso de la CTC			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/cuba/32681-los-santiagueros-en-debate-camino-al-xxi-congreso-de-la-ctc" >
					<img src="/images/2018/marzo/0803-congresoctc-cuba.jpg" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32683">
						<a class="itemtitle" style="font-size:16px; " href="/cultura/32683-abogan-en-granma-por-enriquecer-practicas-culturales">
				Abogan en Granma por enriquecer prácticas culturales			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/cultura/32683-abogan-en-granma-por-enriquecer-practicas-culturales" >
					<img src="/images/2018/marzo/0318-taller-cultura.JPG" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32670">
						<a class="itemtitle" style="font-size:16px; " href="/cuba/32670-el-triunfo-simbolo-de-una-ciudad-cubana">
				El Triunfo, símbolo de una ciudad cubana			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/cuba/32670-el-triunfo-simbolo-de-una-ciudad-cubana" >
					<img src="/images/2018/marzo/0308-puente-triunfo.JPG" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32674">
						<a class="itemtitle" style="font-size:16px; " href="/deportes/32674-estrellas-del-beisbol-iluminan-noche-en-granma">
				Estrellas del béisbol iluminan noche en Granma (+Fotos)			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/deportes/32674-estrellas-del-beisbol-iluminan-noche-en-granma" >
					<img src="/images/2018/marzo/0318-beisbol-premiacion-granma6.JPG" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32671">
						<a class="itemtitle" style="font-size:16px; " href="/cultura/32671-documentalistas-rinden-tributo-a-fidel-y-a-proceres-de-la-patria">
				Documentalistas rinden tributo a Fidel y a próceres de la Patria...			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/cultura/32671-documentalistas-rinden-tributo-a-fidel-y-a-proceres-de-la-patria" >
					<img src="/images/2018/marzo/0318-documentalistas-santiago-alvarez.jpg" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
	<div class="yjnewsflashu">
		<div class="yjnewsflashu_in yjnfitem32682 last">
						<a class="itemtitle" style="font-size:16px; " href="/cuba/32682-destacan-en-onu-derechos-de-las-mujeres-rurales-en-cuba">
				Destacan en ONU derechos de las mujeres rurales en Cuba			</a>
									<div class="imageholder" style="width:254px;height:150px;float:left;">
				<a class="itemimage"  style="width:254px;height:150px;" href="/cuba/32682-destacan-en-onu-derechos-de-las-mujeres-rurales-en-cuba" >
					<img src="/images/articulos/Sociedad/campesina.jpg" alt="" style="width:254px;height:150px;" />
				</a>
			</div>
																	</div>
	</div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
                </div>                
                                
                <div class="content-layout"><div class="content-layout-row"><div class="layout-cell" style="width: 100%;">        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Especiales				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/especiales-acn/32616-baragua-sigue-latiendo-en-el-alma-de-cuba-audio">
                        Baraguá sigue latiendo en el alma de Cuba (+Audio)</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/especiales-acn/32616-baragua-sigue-latiendo-en-el-alma-de-cuba-audio">
							<img src="/images/2018/marzo/113-/mangos%20de%20baragua.JPG" style="width:330px  !important;height:240px!important;;" alt="" />
							</a>
						</div>
					
													<div class="yj_news_flash_text">La voz de Antonio Maceo, el insigne hijo de Santiago de Cuba que el 15 de marzo de 1878 habló por todos los cubanos dignos, se ha multiplicado de siglo en siglo para mantener la intransigencia revolucionaria enarbolada como bandera aquel día en que nació su viril protesta.
</div>
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/especiales-acn/32642-para-el-hockey-sobre-cesped-la-mayor-fidelidad-de-eliberto"> 
					Para el hockey sobre césped, la mayor fidelidad de Eliberto				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/especiales-acn/32641-terry-fox-esperanza-que-se-multiplica-cada-ano"> 
					Terry Fox, esperanza que se multiplica cada año				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
</div></div></div>
                <div class="cleared"></div>
              </div>
                              <div class="layout-cell sidebar1 sidebar1-fix">
                          <div class="box block">
            <div class="box-body block-body">
        
                
		
				
							<div class="box blockcontent1">
					<div class="box-body blockcontent-body">        
						

<div class="customblock"  >
	<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/GWVWqNQOlCg"><param name="movie" value="http://www.youtube.com/v/GWVWqNQOlCg" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
<p> </p></div>
						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Productos y Servicios				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						

<div class="custom"  >
	<div class="prod1"><a href="/index.php/productos-y-servicios" title="Productos y Servicios">Productos y servicios</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/188-fotos" title="Fotos" style="font-size: 18px;">Fotos</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/189-radio" title="Radio" style="font-size: 18px;">Radio</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/187-texto" title="Texto" style="font-size: 18px;">Texto</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/193-publicidad" title="Publicidad" style="font-size: 18px;">Publicidad</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/191-senal-acn" title="Señal ACN" style="font-size: 18px;">Señal ACN</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/productos-y-servicios/867-audiovisuales" title="Audiovisuales" style="font-size: 18px;">Audiovisuales</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/192-acreditaciones" title="Acreditaciones" style="font-size: 18px;">Acreditaciones</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; height: 26px; border-bottom-style: dashed; border-bottom-width: 1px; border-color: #BCBCBC; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/199-servicios-especiales" title="Servicios Especiales" style="font-size: 18px;">Servicios Especiales</a></div>
<div style="margin-left: 5px; margin-bottom: 3px; text-transform: none !important;"><img src="/images/banners/vineta.png" border="0" style="border: none; margin-bottom: 2px;" /> <a href="/index.php/productos-y-servicios/190-centro-de-documentacion" title="Centro de Documentación" style="font-size: 18px;">Centro de Documentación</a></div></div>
						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Puntos de Vista				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                       
                    
               
                    <div style = "min-height: 87px; ">
					<div class="yj_news_flash_title1">	
					<div class="yj_news_flash_photo">
							<a style="font-size:18px;" href="/punto-de-vista/17792-inocencia-infantil-contaminada-alerta">
							<img src="/images/articulos/maria-elena-balan-fb.jpg" style="width:80px  !important;height:80px!important;;" alt="" />
							</a>
						</div>
					
													<div class="yj_news_flash_text">María Elena Balán Sainz 
</div>
									
					</div>	
					 <a style="font-size:18px;" href="/punto-de-vista/17792-inocencia-infantil-contaminada-alerta">
                        Inocencia infantil contaminada, ¡alerta!</a>
                     </div>
			
		

													<div class="yj_news_flash_title_only"> 
				<div class="yj_news_flash_photo">
							<a style="font-size:18px;" href="/punto-de-vista/11292-sin-excusas-ni-pretextos-use-el-casco-de-proteccion">
							<img src="/images/articulos/Deportes/lino-luben.jpg" style="width:80px  !important;height:80px!important;;" alt="" />
							</a>
						</div>
									<div class="yj_news_flash_text">Lino Lubén Pérez
</div>
					
				
					<a style="font-size:18px;" href="/punto-de-vista/11292-sin-excusas-ni-pretextos-use-el-casco-de-proteccion"> 
						Sin excusas ni pretextos: use el casco de protección					</a> 
				</div>
															<div class="yj_news_flash_title_only_last"> 
				<div class="yj_news_flash_photo">
							<a style="font-size:18px;" href="/punto-de-vista/32575-hablemos-correctamente-significados-de-cancerigeno-y-canceroso">
							<img src="/images/2016/Elio Delgado Legón.jpg" style="width:80px  !important;height:80px!important;;" alt="" />
							</a>
						</div>
									<div class="yj_news_flash_text"> 
</div>
								
				
					<a style="font-size:18px;" href="/punto-de-vista/32575-hablemos-correctamente-significados-de-cancerigeno-y-canceroso"> 
						Hablemos correctamente: significados de cancerígeno y canceroso					</a> 
				</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Imágenes Destacadas				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- RIE - Random Image Extended - Kubik-Rubik Joomla! Extensions --><div class="random_image_extended block">
                                        <a href="/images/galeria/0127-marti-marcha-antorchas5.jpg" title="0127-marti-marcha-antorchas5" rel="lightbox.random">
                                                <img src="/images/galeria/0127-marti-marcha-antorchas5.jpg" alt="0127-marti-marcha-antorchas5" width="245" />                            </a>
                                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cgo.jpg" title="0127-marti-marcha-antorchas-cgo"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cfg5.jpg" title="0127-marti-marcha-antorchas-cfg5"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-ij2.jpg" title="0127-marti-marcha-antorchas-ij2"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas4.jpg" title="0127-marti-marcha-antorchas4"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas.jpg" title="0127-marti-marcha-antorchas"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cfg2.jpg" title="0127-marti-marcha-antorchas-cfg2"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-hol2.JPG" title="0127-marti-marcha-hol2"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cgo2.jpg" title="0127-marti-marcha-antorchas-cgo2"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-hol.JPG" title="0127-marti-marcha-hol"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-ij.jpg" title="0127-marti-marcha-antorchas-ij"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cfg.jpg" title="0127-marti-marcha-antorchas-cfg"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-ij3.jpg" title="0127-marti-marcha-antorchas-ij3"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cfg3.jpg" title="0127-marti-marcha-antorchas-cfg3"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cfg4.jpg" title="0127-marti-marcha-antorchas-cfg4"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cgo3.jpg" title="0127-marti-marcha-antorchas-cgo3"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-cgo5.jpg" title="0127-marti-marcha-antorchas-cgo5"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas-ij4.jpg" title="0127-marti-marcha-antorchas-ij4"></a>
                                                                <a rel="lightbox.random" href="/images/galeria/0127-marti-marcha-antorchas3.jpg" title="0127-marti-marcha-antorchas3"></a>
                                            </div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        

                  <div class="cleared"></div>
                </div>
                            
              
            </div>
          </div>
        </div>
        <div class="cleared"></div>                                          


        <div class="content-layout"><div class="content-layout-row"><div class="layout-cell" style="width: 100%;"><div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<div class="postcontent">
<div class="customarticle"  >
	<div style="border-top-style: solid; border-top-width: 3px; border-bottom-style: solid; border-bottom-width: 3px; border-color: #c1cfd1; right: 10px; position: relative; width: 998px; padding-top: 10px; padding-bottom: 10px;">
<div style="float: left; margin-right: 10px; margin-left: 10px;"> </div>
<div style="float: none;"><a href="/reflexiones-de-fidel"><img src="/images/banners/importantes/reflexiones-fidel.png" border="0" alt="reflexiones-fidel.png" width="213" height="63" />   </a><a href="/discursos-de-raul"><img src="/images/banners/importantes/discursos-raul.png" border="0" alt="discursos-raul.png" width="213" height="63" /> </a><a href="/index.php/curiosidades">  <img class="nothumbnail" src="/images/banners/curiosidades.jpg" border="0" width="237" />   </a><a href="/index.php/sexualidad"><img class="nothumbnail" src="/images/banners/sexualidad.jpg" border="0" width="237" /></a></div>
</div></div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
</div></div></div>        <div class="content-layout"><div class="content-layout-row"><div class="layout-cell layout-sidebar-bg" style="width: 32%;">        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Mundo				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/mundo/32650-crean-aplicacion-movil-en-belgica-para-combatir-el-abuso-sexual">
                        Crean aplicación móvil en Bélgica para combatir el abuso sexual</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/mundo/32650-crean-aplicacion-movil-en-belgica-para-combatir-el-abuso-sexual">
							<img src="/images/2018/marzo/app-movil.jpg" style="width:310px  !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/mundo/32649-exhortan-a-paises-caribenos-a-fortalecer-unidad-ante-desafios"> 
					Exhortan a países caribeños a fortalecer unidad ante desafíos				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/mundo/32648-lavrov-denuncia-campana-difamatoria-sin-precedentes-contra-rusia"> 
					Lavrov denuncia campaña difamatoria sin precedentes contra Rusia				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Deportes				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/deportes/32684-gano-cuba-el-torneo-centroamericano-de-boxeo-en-tijuana">
                        Ganó Cuba el Torneo Centroamericano de boxeo, en Tijuana</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/deportes/32684-gano-cuba-el-torneo-centroamericano-de-boxeo-en-tijuana">
							<img src="/images/2017/noviembre/1113-boxeo.jpg" style="width:310px !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/deportes/32679-conquistan-tres-titulos-duplas-cubanas-en-giraldilla-de-badminton-fotos"> 
					Conquistan tres títulos duplas cubanas en Giraldilla de bádminton (+Fotos)				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/deportes/32676-cierra-cuba-con-dos-oros-y-tres-bronces-en-abierto-de-judo"> 
					Cierra Cuba con dos oros y tres bronces en Abierto de judo				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Economía				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/economia/32665-inaugurado-en-la-habana-mercado-mayorista-de-alimentos-fotos">
                        Inaugurado en La Habana Mercado Mayorista de Alimentos (+Fotos)</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/economia/32665-inaugurado-en-la-habana-mercado-mayorista-de-alimentos-fotos">
							<img src="/images/2018/marzo/0317/0317-Mercado-mayorista-calle26-nuevo-vedado-ACN-.jpg" style="width:310px !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/economia/32608-preven-extender-produccion-y-venta-de-materiales-de-construccion-fotos-y-audio"> 
					Prevén extender producción y venta de materiales de construcción (+Fotos y Audio)				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/economia/32644-premiado-stand-cubano-en-feria-de-turismo-de-moscu-fotos"> 
					Premiado stand cubano en Feria de Turismo de Moscú (+Fotos)				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
</div><div class="layout-cell layout-sidebar-bg" style="width: 32%;">        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Salud				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/salud/32647-inversiones-benefician-instalaciones-de-salud-en-holguin">
                        Inversiones benefician instalaciones de salud en Holguín</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/salud/32647-inversiones-benefician-instalaciones-de-salud-en-holguin">
							<img src="/images/2018/febrero/hospital-mayari.jpg" style="width:310px !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/salud/32638-surfacen-disminuye-mortalidad-infantil"> 
					Surfacén disminuye mortalidad infantil				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/salud/32624-en-fase-de-terminacion-nueva-maternidad-provincial-en-matanzas-fotos"> 
					En fase de terminación nueva Maternidad provincial en Matanzas (+Fotos)				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Cultura				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/cultura/32685-miriam-talavera-una-enamorada-del-septimo-arte">
                        Miriam Talavera, una enamorada del séptimo arte  </a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/cultura/32685-miriam-talavera-una-enamorada-del-septimo-arte">
							<img src="/images/2018/marzo/0318-miriam-talavera.jpg" style="width:310px !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/cultura/32683-abogan-en-granma-por-enriquecer-practicas-culturales"> 
					Abogan en Granma por enriquecer prácticas culturales				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/cultura/32652-nominada-pelicula-cubana-a-los-premios-platino-2018"> 
					Nominada película cubana a los Premios Platino 2018				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Ciencia y Tecnología				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<!-- http://www.Youjoomla.com  Youjoomla YJ Newsflash 2 for Joomla 1.6 starts here -->
<div class="yj_news_flash_content">
				
		
		
				
			
			
                	
                        <div class="yj_news_flash_title"><a href="/ciencia-y-tecnologia/32645-premiaran-resultados-cientificos-de-la-universidad-de-la-habana">
                        Premiaran resultados científicos de la Universidad de La Habana</a>
                        </div>
                    
               
                    <div style = "min-height: 95px; ">
						<div class="yj_news_flash_photo">
							<a href="/ciencia-y-tecnologia/32645-premiaran-resultados-cientificos-de-la-universidad-de-la-habana">
							<img src="/images/2018/febrero/Alma-Mater-UH.jpg" style="width:310px !important;height:178px!important;;" alt="" />
							</a>
						</div>
					
									
					</div>	
			
		

												  
			<div class="yj_news_flash_title_only"> 
				<a href="/ciencia-y-tecnologia/32491-promueven-cuba-y-vietnam-colaboracion-en-aplicaciones-nucleares"> 
					Promueven Cuba y Vietnam colaboración en aplicaciones nucleares				</a> 
			</div>
													
			<div class="yj_news_flash_title_only_last"> 
				<a href="/ciencia-y-tecnologia/32479-concluye-hoy-convencion-internacional-cientifica"> 
					Concluye hoy convención internacional científica				</a> 
			</div>
						
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
</div><div class="layout-cell layout-sidebar-bg" style="width: 27.5%;">        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Publicidad				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						

<div class="custom"  >
	<p><a href="http://ofertas.cu" target="_blank"><img src="/images/publicidad/ofertas.cu.jpg" border="0" alt="ofertas.cu.jpg" width="253" height="109" /></a></p></div>
						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Efemérides				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<div class="newscalendar" id="newscalendar319">	<table class="nc-header" cellspacing="0">
    	<thead>
            <tr>
                <td width="20"><a href="javascript:void(0)" onClick="updateNewsCalendar(3,2017,319)" class="back-yr"></a></td>
                <td width="20"><a href="javascript:void(0)" onClick="updateNewsCalendar(2,2018,319)" class="back-mth"></a></td>
                <td align="center"><div id="monthyear_319"><span class="monthname">March</span> <span class="yearname">2018</span></div></td>
                <td width="20" align="right"><a href="javascript:void(0)" onClick="updateNewsCalendar(4,2018,319)" class="fwd-mth"></a></td>
                <td width="20" align="right"><a href="javascript:void(0)" onClick="updateNewsCalendar(3,2019,319)" class="fwd-yr"></a></td>
            </tr>
        </thead>
    </table>
    <table class="nc-body" cellspacing="0">
    	<thead>
            <tr> 
                <td class="dayname dayweek1" align="center">Mo</td>		
                <td class="dayname dayweek2" align="center">Tu</td>		
                <td class="dayname dayweek3" align="center">We</td>		
                <td class="dayname dayweek4" align="center">Th</td>		
                <td class="dayname dayweek5" align="center">Fr</td>		
                <td class="dayname dayweek6" align="center">Sa</td>		
                <td class="dayname dayweek7" align="center">Su</td>		
            </tr>
        </thead>
        <tbody>
		<tr class="week1">
				<td><span class="nc-day outofmonth">26</span></td>
                    <td><span class="nc-day outofmonth">27</span></td>
                    <td><span class="nc-day outofmonth">28</span></td>
                    <td><span class="nc-day weekday4">1</span></td>
                    <td><span class="nc-day weekday5">2</span></td>
                    <td><span class="nc-day weekday6">3</span></td>
                    <td><span class="nc-day weekday7">4</span></td>
                                    </tr>
		<tr class="week2">
				<td><span class="nc-day weekday1">5</span></td>
                    <td><span class="nc-day weekday2">6</span></td>
                    <td><span class="nc-day weekday3">7</span></td>
                    <td><span class="nc-day weekday4">8</span></td>
                    <td><span class="nc-day weekday5">9</span></td>
                    <td><span class="nc-day weekday6">10</span></td>
                    <td><span class="nc-day weekday7">11</span></td>
                                    </tr>
		<tr class="week3">
				<td><span class="nc-day weekday1">12</span></td>
                    <td><span class="nc-day weekday2">13</span></td>
                    <td><span class="nc-day weekday3">14</span></td>
                    <td><span class="nc-day weekday4">15</span></td>
                    <td><span class="nc-day weekday5">16</span></td>
                    <td><span class="nc-day weekday6 hasitems">17                            
                            <ul class="nc-items"> 
                                <li><a href="/efemerides/17009-17-de-marzo">17 de marzo</a></li></ul>
                        </span></td>
                    <td><span class="nc-day weekday7">18</span></td>
                                    </tr>
		<tr class="week4">
				<td><span class="nc-day weekday1 today">19</span></td>
                    <td><span class="nc-day weekday2">20</span></td>
                    <td><span class="nc-day weekday3">21</span></td>
                    <td><span class="nc-day weekday4">22</span></td>
                    <td><span class="nc-day weekday5">23</span></td>
                    <td><span class="nc-day weekday6">24</span></td>
                    <td><span class="nc-day weekday7">25</span></td>
                                    </tr>
		<tr class="week5">
				<td><span class="nc-day weekday1">26</span></td>
                    <td><span class="nc-day weekday2">27</span></td>
                    <td><span class="nc-day weekday3">28</span></td>
                    <td><span class="nc-day weekday4">29</span></td>
                    <td><span class="nc-day weekday5">30</span></td>
                    <td><span class="nc-day weekday6">31</span></td>
                    <td><span class="nc-day outofmonth">1</span></td>
                                    </tr>
		            
        </tbody>
    </table>
	    <div class="bgimgcall"><img src="/modules/mod_newscalendar/assets/opacity.png" /></div>
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Encuentranos en:				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<div id="itp-fblike-box">



<div id="fb-root"></div>

<script>(function(d, s, id) {
  var js, fjs = d.getElementsByTagName(s)[0];
  if (d.getElementById(id)) return;
  js = d.createElement(s); js.id = id;
  js.src = "//connect.facebook.net/es_ES/all.js#xfbml=1";
  fjs.parentNode.insertBefore(js, fjs);
}(document, 'script', 'facebook-jssdk'));</script>

	<div class="fb-like-box" 
	data-href="http://www.facebook.com/pages/Agencia-Cubana-de-Noticias/111896838909504" 
	data-width="270" 
	data-height="255" 
	data-show-faces="1" 
	data-border-color="#FFFFFF" 
	data-stream="0" 
	data-header="0"></div>
</div>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
        <div class="box block">
            <div class="box-body block-body">
        
                
		
					<div class="bar blockheader">
				<h3 class="t">
					Desde Twitter				</h3>
			</div>
							<div class="box blockcontent">
					<div class="box-body blockcontent-body">        
						<a class="twitter-timeline" data-theme="light" data-link-color="##4aed05" data-border-color="##cccccc"  data-chrome="nofooter "  3  href="https://twitter.com/ACN_Cuba" data-widget-id="408086734144950272" width="340" height="208">Tweets by @ACN_Cuba</a>

<script type="text/javascript">!function(d,s,id){var js,fjs=d.getElementsByTagName(s)[0],p=/^http:/.test(d.location)?'http':'https';if(!d.getElementById(id)){js=d.createElement(s);js.id=id;js.src=p+"://platform.twitter.com/widgets.js";fjs.parentNode.insertBefore(js,fjs);}}(document,"script","twitter-wjs");</script>						<div class="cleared"></div>
					</div>
				</div>        
			                
    

        		<div class="cleared"></div>
            </div>
        </div>
        
</div></div></div>        <div class = "bottom_6">
          <div class="content-layout"><div class="content-layout-row"><div class="layout-cell" style="width: 100%;"><div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<h2 class="postheader">Cuba en Videos</h2>
<div class="postcontent">
<div class="customarticle"  >
	<div class="ytp-html5-clipboard">
<div class="ytp-html5-clipboard">
<div class="ytp-html5-clipboard">
<div class="ytp-html5-clipboard">
<div class="ytp-html5-clipboard">
<table style="width: 686px; height: 131px;" border="0" align="center">
<tbody>
<tr>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/5V0vfTpmJj0&amp;t=41s"><param name="movie" value="http://www.youtube.com/v/5V0vfTpmJj0&amp;t=41s" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td>
<p>   </p>
</td>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/HdIcm1Q9oCw"><param name="movie" value="http://www.youtube.com/v/HdIcm1Q9oCw" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td>
<p>   </p>
</td>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/I3w_oEObE-s"><param name="movie" value="http://www.youtube.com/v/I3w_oEObE-s" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
</tr>
<tr>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/ZIByzNYuoWY"><param name="movie" value="http://www.youtube.com/v/ZIByzNYuoWY" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td> </td>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/o0DAemo5O94"><param name="movie" value="http://www.youtube.com/v/o0DAemo5O94" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td> </td>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/fmDsfMWJ17o&amp;t=3s"><param name="movie" value="http://www.youtube.com/v/fmDsfMWJ17o&amp;t=3s" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
</tr>
<tr>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/2D0J40en7LM"><param name="movie" value="http://www.youtube.com/v/2D0J40en7LM" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td> </td>
<td>
<p><object class="embed" width="249" height="210" type="application/x-shockwave-flash" data="http://www.youtube.com/v/wgrJKGa_JHU"><param name="movie" value="http://www.youtube.com/v/wgrJKGa_JHU" /><param name="wmode" value="transparent"><em>You need to a flashplayer enabled browser to view this YouTube video</em></object></p>
</td>
<td> </td>
<td> </td>
</tr>
</tbody>
</table>
</div>
</div>
</div>
</div>
</div>
<p> </p></div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
<div class="box post">
    <div class="box-body post-body">
<div class="post-inner">
<div class="postcontent">
<div class="custom"  >
	<hr width="1000" noshade="noshade" style="margin: 0; right: 10px; position: relative; width: 998px; border-width: 1px; border-style: solid; border-color: #5CEEFE;" />
<div style="background-color: #d2fdff; right: 10px; position: relative; width: 1000px;">
<div style="padding-left: 10px; padding-right: 10px; padding-bottom: 10px; padding-top: 5px;">
<div><span style="color: #666666; text-transform: uppercase; font-size: 12px;">Enlaces DE CUBA</span></div>
<div style="border: 1px solid #5CEEFE; border-radius: 5px; padding: 5px; background-color: white; height: 40px;">
<div class="list">
<ul>
<li class="cubadebate"><a href="http://www.cubadebate.cu/" title="Cubadebate">a</a></li>
<li class="juventudrebelde"><a href="http://www.juventudrebelde.cu/" title="Juventud Rebelde">a</a></li>
<li class="cubasi"><a href="http://www.cubasi.cu/" title="Cubasi.cu">a</a></li>
<li class="granma"><a href="http://www.granma.cubaweb.cu/" title="Periódico Granma">a</a></li>
<li class="trabajadores"><a href="http://www.trabajadores.cu/" title="Periódico Trabajadores">a</a></li>
<li class="bohemia"><a href="http://www.bohemia.cu" title="Revista Bohemia">a</a></li>
<li class="tvcubana"><a href="http://www.tvcubana.icrt.cu" title="Televisión Cubana">a</a></li>
</ul>
</div>
</div>
<div style="margin-top: 5px;"><span style="color: #666666; text-transform: uppercase; font-size: 12px;">Agencias</span></div>
<div style="border: 1px solid #5CEEFE; border-radius: 5px; padding: 5px; background-color: white; height: 40px;">
<div class="list">
<ul>
<li class="prensalatina"><a href="http://www.prensa-latina.cu" title="Agencia Prensa Latina">a</a></li>
<li class="telam"><a href="http://www.telam.com.ar/homepage.html" title="Agencia de Noticias TELAM">a</a></li>
<li class="avn"><a href="http://www.avn.info.ve/" title="Agencia Venezolana de Noticias">a</a></li>
<li class="bolpress"><a href="http://www.bolpress.com/" title="Agencia de Noticias BOLPRESS">a</a></li>
<li class="argenpress"><a href="http://www.argenpress.info/" title="Agencia de Noticias ARGENPRESS">a</a></li>
<li class="vietnam"><a href="http://www.vietnamplus.vn/" title="Agencia Vietnamita de Noticias">a</a></li>
<li class="ecured"><a href="http://www.ecured.cu" title="Enciclopedia Cubana">a</a></li>
</ul>
</div>
</div>
</div>
</div></div>
</div>
<div class="cleared"></div>
</div>

		<div class="cleared"></div>
    </div>
</div>
</div></div></div>        
        
        <div class="cleared"></div>
      </div>
    </div>
    <div class="footer">
      <div class="footer-body">
        <div class="footer-center">
          <div class="footer-wrapper">
            <div class="footer-text">
                                           <a href="/?format=feed&amp;type=rss"><div class="rss-tag-icon1"></div></a>              
                            <p><a href="/quienes-somos">Quiénes Somos</a> | <a href="/contacte-con-nosotros">Contáctenos</a> | <a href="/archivo">Archivo</a></p>

    <p>Todos los derechos reservados © 2014. Agencia Cubana de Noticias.</p>
    <p style="float: right;position: absolute;right: 23px;bottom: 15px;"><strong>Directora General:</strong> Edda Diz Garcés</p>
    <p style="float: right;position: absolute;right: 153px;bottom: 0px;"><strong>Teléfono:</strong> 8325542</p>
    <p style="float: right;position: absolute;right: 23px;bottom: 0px;"><strong>E-mail:</strong> <a href="mailto:web@ain.cu" >web@acn.cu</a></p>
    <div class="cleared"></div>
    <p class="page-footer"></p>

                                                        
            </div>
          </div>
        </div>
        <div class="cleared"></div>
      </div>
    </div>

      <div class="cleared"></div>
  </div>  
   

  </body>
<script>
  (function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
  (i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
  m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
  })(window,document,'script','https://www.google-analytics.com/analytics.js','ga');

  ga('create', 'UA-86398795-1', 'auto');
  ga('send', 'pageview');

</script></html>