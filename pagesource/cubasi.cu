#include "hip/hip_runtime.h"
<!DOCTYPE html PUBLIC "-//W3C//DTD XHTML 1.0 Transitional//EN" "http://www.w3.org/TR/xhtml1/DTD/xhtml1-transitional.dtd">
<html prefix="og: http://ogp.me/ns#" xmlns="http://www.w3.org/1999/xhtml>
<head>
<link rel="shortcut icon" href="http://test.cubasi.cu/favicon.ico" />
  <base href="http://cubasi.cu/" />
  <meta http-equiv="content-type" content="text/html; charset=utf-8" />
  <meta name="robots" content="index, follow" />
  <meta name="keywords" content="Cuba, Política, Cultura, Deportes, Exclusiva" />
  <meta name="description" content="Cubasi - Noticias de Cuba y el mundo." />
  <meta name="generator" content="Joomla! 1.5 - Open Source Content Management" />
  <title>Portal Cuba Sí, A un clic de distancia</title>
  <link href="/index.php?format=feed&amp;type=rss" rel="alternate" type="application/rss+xml" title="RSS 2.0" />
  <link href="/index.php?format=feed&amp;type=atom" rel="alternate" type="application/atom+xml" title="Atom 1.0" />
  <link href="/templates/gk_cuttingedge/favicon.ico" rel="shortcut icon" type="image/x-icon" />
  <link rel="stylesheet" href="/components/com_k2/css/k2.css" type="text/css" />
  <link rel="stylesheet" href="http://www.cubasi.cu/plugins/system/videoflow/style.css" type="text/css" />
  <link rel="stylesheet" href="http://www.cubasi.cu/components/com_videoflow/views/videoflow/tmpl/multibox/multibox.css" type="text/css" />
  <link rel="stylesheet" href="http://www.cubasi.cu/components/com_videoflow/extra/votitaly/css/votitaly.css" type="text/css" />
  <link rel="stylesheet" href="/plugins/system/jcemediabox/css/jcemediabox.css?1d12bb5a40100bbd1841bfc0e498ce7b" type="text/css" />
  <link rel="stylesheet" href="/plugins/system/jcemediabox/themes/shadow/css/style.css?587b87a7390648bab4f4f6d04a8c420d" type="text/css" />
  <link rel="stylesheet" href="/modules/mod_ddslide/js/galleryview.css" type="text/css" />
  <script type="text/javascript" src="/media/system/js/mootools.js"></script>
  <script type="text/javascript" src="//ajax.googleapis.com/ajax/libs/jquery/1.11.0/jquery.min.js"></script>
  <script type="text/javascript" src="/components/com_k2/js/k2.js?v2.6.9&amp;sitepath=/"></script>
  <script type="text/javascript" src="http://www.cubasi.cu/components/com_videoflow/jscript/swfobject.js"></script>
  <script type="text/javascript" src="http://www.cubasi.cu/components/com_videoflow/views/videoflow/tmpl/multibox/overlay_legacy.js"></script>
  <script type="text/javascript" src="http://www.cubasi.cu/components/com_videoflow/views/videoflow/tmpl/multibox/multibox_legacy.js"></script>
  <script type="text/javascript" src="http://www.cubasi.cu/components/com_videoflow/extra/votitaly/js/votitaly_legacy.js"></script>
  <script type="text/javascript" src="/media/system/js/caption.js"></script>
  <script type="text/javascript" src="/plugins/system/jcemediabox/js/jcemediabox.js?0c56fee23edfcb9fbdfe257623c5280e"></script>
  <script type="text/javascript" src="/plugins/content/flowplayer/flowplayer-3.1.4.min.js"></script>
  <script type="text/javascript" src="/modules/mod_ddslide/js/jquery-1.4.1.min.js"></script>
  <script type="text/javascript" src="/modules/mod_ddslide/js/jquery.timers.1.2.js"></script>
  <script type="text/javascript" src="/modules/mod_ddslide/js/jquery.easing.1.3_New1.js"></script>
  <script type="text/javascript" src="/modules/mod_ddslide/js/jquery.galleryview-3.0.js"></script>
  <script type="text/javascript">

    
	   window.addEvent('domready', function(){
	     var vf_rate = new VotitalyPlugin({
	  	    submiturl: 'http://www.cubasi.cu/index.php?option=com_videoflow&task=vote&format=raw',
		      loadingimg: 'http://www.cubasi.cu/components/com_videoflow/extra/votitaly/images/loading.gif',
			    show_stars: true,
			    star_description: '({num_votes} #VF_VOTES, #VF_AVERAGE {num_average} #VF_OUTOF)',		
			    language: {
				  updating: 'Updating...',
				  thanks: 'Thanks for voting!',
				  already_vote: 'You have already voted for this item!',
				  votes: 'Votes',
				  vote: 'Vote',
				  average: 'average',
				  outof: 'out of 5',
				  error1: 'An error has occured!',
				  error2: 'You have already voted for this item!',
				  error3: 'Rating out of range!',
				  error4: 'You must login to vote!',
				  error5: 'User rating already submitted for this item!'
			    }
	       });
	     });
      
    
JCEMediaBox.init({popup:{width:"",height:"",legacy:0,lightbox:0,shadowbox:0,resize:1,icons:1,overlay:1,overlayopacity:0.8,overlaycolor:"#000000",fadespeed:500,scalespeed:500,hideobjects:0,scrolling:"fixed",close:2,labels:{'close':'Cerrar','next':'Siguiente','previous':'Anterior','cancel':'Cancelar','numbers':'{$current} de {$total}'},cookie_expiry:"",google_viewer:0},tooltip:{className:"tooltip",opacity:0.8,speed:150,position:"br",offsets:{x: 16, y: 16}},base:"/",imgpath:"plugins/system/jcemediabox/img",theme:"shadow",themecustom:"",themepath:"plugins/system/jcemediabox/themes",mediafallback:0,mediaselector:"audio,video"});
  </script>
  


<link rel="shortcut icon" href="/images/favicon.ico" />
<link href="/templates/gk_cuttingedge/css/template_css.css" rel="stylesheet"  type="text/css" />
<link href="/templates/gk_cuttingedge/css/suckerfish.css" rel="stylesheet"  type="text/css" />
<link href="/templates/gk_cuttingedge/css/joomla_classes.css" rel="stylesheet"  type="text/css" />
<link href="/templates/gk_cuttingedge/css/gk_stuff.css" rel="stylesheet"  type="text/css" />
<!--[if IE 6]>
<link href="/templates/gk_cuttingedge/css/ie6_css.css" rel="stylesheet" type="text/css" />
<style type="text/css">
img { behavior: url(/templates/gk_cuttingedge/css/iepngfix.htc); }
a#logo {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/logo.png", sizingMethod="crop");}
div#topmenu ul li {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/arrow.png", sizingMethod="crop");}
a.readon {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/b_readon.png", sizingMethod="crop");}
a.readon:hover {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/b_readon-h.png", sizingMethod="crop");}
div#wrap-2 {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/p_middle.png", sizingMethod="scale");}
div#bottom {
background: none;
filter: progid:DXImageTransform.Microsoft.AlphaImageLoader(src="/templates/gk_cuttingedge/images/p_bottom.png", sizingMethod="image");}

</style>
<![endif]-->
<!--[if IE 7]>
<link href="/templates/gk_cuttingedge/css/ie7_css.css" rel="stylesheet" type="text/css" />
<![endif]-->
<script type="text/javascript" src="/templates/gk_cuttingedge/lib/scripts/mootools.js"></script>
<script type="text/javascript" src="/templates/gk_cuttingedge/lib/scripts/gk_blocksfix.js"></script>
<script type="text/javascript" src="/templates/gk_cuttingedge/lib/scripts/ie.js"></script>



<script language=JavaScript id=onDate ></script>
<script language=JavaScript src=/media/system/js/stat320.php ></script>
</head>
<body id="bg">
<div id="page">
        <div id="wrap-1"> 
          
        <div id="gk-top" class="gk-top">
        <table width="100%" border="0" cellpadding="0" cellspacing="1"><tr><td nowrap="nowrap"><img src="/images/stories/Englishflag.jpg" border="0" alt="Cubasi.com"/> <a href="http://news.cubasi.cu" class="mainlevel-nav" >Cubasi.com</a><span class="mainlevel-nav"> | </span><a href="/reflexiones-de-fidel" class="mainlevel-nav" >Reflexiones de Fidel</a></td></tr></table>
        </div>
                   
                <div id="top">
                        <a href="" class="nounder" id="logo"></a>
                                                                 <div id="gk-search"  class="search">
                                                        <form action="index.php" method="post">
	<div class="search">
		<input name="searchword" id="mod_search_searchword" maxlength="20" alt="Buscar" class="inputbox" type="text" size="20" value="buscar..."  onblur="if(this.value=='') this.value='buscar...';" onfocus="if(this.value=='buscar...') this.value='';" /><input type="submit" value="Buscar" class="button" onclick="this.form.searchword.focus();"/>	</div>
	<input type="hidden" name="task"   value="search" />
	<input type="hidden" name="option" value="com_search" />
	<input type="hidden" name="Itemid" value="17" />
</form>
                                                </div>
                                                     
                                                                  <div id="gk-social"  class="social">
                                                        <table width="100%" border="0" cellpadding="0" cellspacing="1"><tr><td nowrap="nowrap"><img src="/images/stories/icono-Facebook.jpg" border="0" alt="Facebook"/> <a href="https://www.facebook.com/Cubasi.cu" class="mainlevel-social" >Facebook</a><img src="/images/stories/icono-Twitter.jpg" border="0" alt="Twitter"/> <a href="http://twitter.com/#!/PortalCubasi" class="mainlevel-social" >Twitter</a><img src="/images/stories/icono-RSS.jpg" border="0" alt="Rss"/> <a href="/cubasi-noticias-cuba-mundo-ultima-hora/itemlist?format=feed&amp;type=rss" class="mainlevel-social" >Rss</a></td></tr></table>
                                                </div>
                                                         </div>
                 <div id="date">
lunes, 19 marzo 2018, 21:50</div>


               
                <div id="wrap-2" class="clearfix">
                        <div id="wrap-3">
                                <div id="wrap-4">
                                        <div id="ctop">
                                                                                                                                        </div>
                                        <div id="menu" >
                                                <div id="horiz-menu" class="nav">
                                                        <ul class="menu"><li id="current" class="active item1"><a href="http://www.cubasi.cu/"><span>Portada</span></a></li><li class="item27"><a href="/cuba"><span>Cuba</span></a></li><li class="item53"><a href="/internacionales"><span>Internacionales</span></a></li><li class="item123"><a href="/deportes"><span>Deportes</span></a></li><li class="item162"><a href="/culturales"><span>Culturales</span></a></li><li class="item185"><a href="/opiniones"><span>Opinion</span></a></li><li class="item126"><a href="/especiales"><span>Especiales</span></a></li><li class="item128"><a href="/multimedias"><span>Multimedias</span></a></li><li class="item219"><a href="/panoramicas"><span>Panorámicas</span></a></li><li class="item166"><a href="/videoscubasi"><span>Videos</span></a></li></ul>                                                </div>
                                                      
											        
                                              
                                                <!--menu -->
                                        </div>
                                                                                              <div id="cont-1">
                                                <div id="cont-2" class="clearfix">
                                                        <div id="patmenu-wrap1" class="clearfix">
                                                                                                                        </div>
                                                    
                                                <!-- MAINBODY -->
                                                       
                                                         <div id="mainbody">
						<table class="blog" cellpadding="0" cellspacing="0">
<tr>
	<td valign="top">
				</td>
</tr>


</table>

                                                                </div>
                                                       
                                                                                                                <!-- headers TOP COBERTURA ESPECIAL -->
  
                                <!-- fin las coberturas especiales -->
                                             <!-- headers column -->
                                             
<div id="gk-header" class="two-modules">		
		<div id="gk-header0" style="width:66%;">	
		<div id="gk-header00" style="width:100%;">
		
				<div class="module">
			<div>
				<div>
					<div>
											<style>#feature_slide_6983{
				display:block;
				margin:8px;
				padding:0px;
				overflow:hidden;
				font-family:Georgia, "Times New Roman", Times, serif;
				}
			
			#feature_slide_6983 a{
				color:#F00;
				background-color:transparent;
				}
			
			#feature_slide_6983 a, #feature_slide_6983 ul, #feature_slide_6983 img{
				margin:0;
				padding:0;
				border:none;
				outline:none;
				list-style:none;
				text-decoration:none;
				}
			
			#feature_slide_6983 ul{margin:15px 0 0 0;}
			
			#feature_slide_6983 h1, #feature_slide_6983 h2, #feature_slide_6983 h3, #feature_slide_6983 h4, #feature_slide_6983 h5, #feature_slide_6983 h6, #feature_slide_6983 p{
				margin:0 0 4px 0;
				padding:0;
				line-height:normal;
				font-size:20px;
				font-weight:normal;
				color:#FFF;
				font-family:Georgia, "Times New Roman", Times, serif;
				}
			
			#feature_slide_6983 a{color:#059AD8; background-color:transparent; font-weight:normal;}
			
			#feature_slide_6983 p{margin:0; font-size:12px;}
			
			#feature_slide_6983 ul li {background-image:none;}</style><script type="text/javascript">jQuery.noConflict();
				jQuery(document).ready(function () {
				var mywidth=jQuery('#gk-header00').width();
				jQuery('#feature_slide_6983').parents().each(function(){
					mywidth-=parseInt(jQuery(this).css('padding-left'));
					mywidth-=parseInt(jQuery(this).css('padding-right'));
					if (jQuery(this).attr('id')==jQuery('#zt-user1').attr('id')) {	
						return false;
					}
				});
				var p_width=640;
                                if (mywidth<p_width){
					p_width=mywidth;
				}
				
			    jQuery('#id_gallery_4721').galleryView({panel_height:350,show_panels:1,show_filmstrip:1,nav_theme:'dark',filmstrip_position:'bottom',pause_on_hover:1,overlay_position:'bottom',panel_scale:'crop',frame_scale:'crop',show_panel_nav:0,show_filmstrip_nav:1,panel_width:p_width});
			});
			</script><div id="feature_slide_6983">
	<ul id="id_gallery_4721"><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/13fa7618ea71e15319db0d62e4c8e284_XL.jpg" title="Informática 2018: una puerta al desarrollo sostenible" alt="Informática 2018: una puerta al desarrollo sostenible" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75214-informatica-2018-una-puerta-al-desarrollo-sostenible"><h2>Informática 2018: una puerta al desarrollo sostenible</h2></a>
			    </div>
			</li><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/d45a549b89742dc9a5cc56cfd5402760_XL.jpg" title="Cultura del miedo y control de masas: cómo Disney está programando a nuestros hijos" alt="Cultura del miedo y control de masas: cómo Disney está programando a nuestros hijos" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75187-cultura-del-miedo-y-control-de-masas-como-disney-esta-programando-a-nuestros-hijos"><h2>Cultura del miedo y control de masas: cómo Disney está programando a nuestros hijos</h2></a>
			    </div>
			</li><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/90254cefcffc16ee893fcae5f10f09c4_XL.jpg" title="Resultados finales: más del 85 por ciento de los cubanos acudió a las urnas" alt="Resultados finales: más del 85 por ciento de los cubanos acudió a las urnas" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75175-resultados-finales-mas-del-85-por-ciento-de-los-cubanos-acudio-a-las-urnas"><h2>Resultados finales: más del 85 por ciento de los cubanos acudió a las urnas</h2></a>
			    </div>
			</li><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/7d96cf0f3140d402b0adb22279a34074_XL.jpg" title="Nace bebé en pleno vuelo entre La Habana y Bogotá" alt="Nace bebé en pleno vuelo entre La Habana y Bogotá" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75173-nace-bebe-en-pleno-vuelo-entre-la-habana-y-bogota"><h2>Nace bebé en pleno vuelo entre La Habana y Bogotá</h2></a>
			    </div>
			</li><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/4006944c3a5edabd1eb45376d38b8454_XL.jpg" title="Elecciones rusas: Putin en su cuarto mandato presidencial" alt="Elecciones rusas: Putin en su cuarto mandato presidencial" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75169-elecciones-rusas-putin-en-su-cuarto-mandato-presidencial"><h2>Elecciones rusas: Putin en su cuarto mandato presidencial</h2></a>
			    </div>
			</li><li><img src="http://www.cubasi.cu/images/ddthumbs/640x350/media/k2/items/cache/8c03dcbfe3bb14baee9c5eb821632269_XL.jpg" title="Cuba termina segunda en Abierto de judo de Lima" alt="Cuba termina segunda en Abierto de judo de Lima" />
			    <div class="panel-overlay">
				    <a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75166-cuba-termina-segunda-en-abierto-de-judo-de-lima"><h2>Cuba termina segunda en Abierto de judo de Lima</h2></a>
			    </div>
			</li></ul>
    </div>
					</div>
				</div>
			</div>
		</div>
	
	</div>
		
			
    </div>
			<div id="gk-header3" style="width:33%;">
				<div class="two-modules">
							<div class="module-exclusiva">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-exclusiva"><div style="clear: both; background:#FBDBDB;">
<div> <span style="color:#304C84;
font-size: 2.1em;
line-height: 1.2em;
font-family:Georgia;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75053-la-felicidad-en-el-trabajo-y-el-cambio-de-paradigmas" title="La felicidad en el trabajo y el cambio de paradigmas">La felicidad en el trabajo y el cambio de paradigmas</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 19 Marzo 2018, 05.14</span>  </div><div>  </div><div>  </div><span style="float:left; padding-top: 2px; 
margin-left:5px; border: 0px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75053-la-felicidad-en-el-trabajo-y-el-cambio-de-paradigmas"  ><img src="http://www.cubasi.cu/media/k2/items/cache/8dd1bcfc13f46582cb1cad04586632e3_S.jpg" alt="La felicidad en el trabajo y el cambio de paradigmas" title="La felicidad en el trabajo y el cambio de paradigmas" border="0" width="300px" height="180px"/></a></span><div style="padding-top: 5px;
    color: #444;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    font-style:italic;
    line-height: 16px;
    margin: 0 0 6px;">¿Es usted feliz en el trabajo? ¿Le cuesta levantarse temprano para cumplir con la jornada laboral? ¿Siente satisfacción y placer por lo que realiza a diario fuera de su hogar?
</div><div> <div class='list_rel'><ul><li><span style=""><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/52445-disciplina-laboral-el-dinero-no-lo-es-todo" title="">Disciplina laboral: «El dinero no lo es todo»</a></span> </li><li><span style=""><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/52041-disciplina-laboral-¿que-hace-falta-para-alcanzarla?" title="">Disciplina laboral: ¿Qué hace falta para alcanzarla?</a></span> </li></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em;
color:#444;"> 8 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
			<div class="module-masleidas">
			<div>
				<div>
					<div>
											
<div id="k2ModuleBox367" class="k2ItemsBlock -masleidas">

	
	  <ul>
        <li class="even">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75118-60-anos-despues-kim-novak-reflexiona-sobre-“vertigo”">60 años después, Kim Novak reflexiona sobre “Vértigo”</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="odd">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75017-fabula-de-la-otra-palomera">Fábula de la otra palomera</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="even">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75112-zona-critica-luces-y-sombras-de-lado-a-lado">ZONA CRÍTICA: Luces y sombras de Lado a lado</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="odd">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74307-cuevas-de-bellamar-un-regalo-de-la-naturaleza- -video">Cuevas de Bellamar: un regalo de la naturaleza (+ Video)</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="even lastItem">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75137-abren-en-cuba-primer-mercado-mayorista-de-alimentos">Abren en Cuba primer mercado mayorista de alimentos</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="clearList"></li>
  </ul>
  
	
	
</div>
					</div>
				</div>
			</div>
		</div>
			<div class="module-separador">
			<div>
				<div>
					<div>
											
<div id="k2ModuleBox1032" class="k2ItemsBlock -separador">

	
	  <ul>
        <li class="even lastItem">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75070-en-video-laura-pausini-nadie-ha-dicho-feat-gente-de-zona">EN VIDEO: Laura Pausini - Nadie ha dicho feat. Gente de Zona</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

            <div class="moduleItemVideo">
      	<iframe width="305" height="215" src="https://www.youtube.com/embed/LgtUIqMoApc" frameborder="0" allow="autoplay; encrypted-media" allowfullscreen></iframe>      	<span class="moduleItemVideoCaption"></span>
      	<span class="moduleItemVideoCredits"></span>
      </div>
      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="clearList"></li>
  </ul>
  
		<a class="moduleCustomLink" href="http://" title=""></a>
	
	
</div>
					</div>
				</div>
			</div>
		</div>
			<div class="module-opiniones">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-opiniones"><div style="clear: both; ">
<div> <span style="font-size: 1.2em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75144-rusofobia-infames-y-carroneros-nutren-a-occidente" title="Rusofobia: Infames y carroñeros nutren a Occidente">Rusofobia: Infames y carroñeros nutren a Occidente</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 18 Marzo 2018, 07.35</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75144-rusofobia-infames-y-carroneros-nutren-a-occidente"  ><img src="http://www.cubasi.cu/media/k2/items/cache/d7ead09016602ece485070b82d861c9d_S.jpg" alt="Rusofobia: Infames y carroñeros nutren a Occidente" title="Rusofobia: Infames y carroñeros nutren a Occidente" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
color:#444;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Lo más reciente fue protagonizado por la primera ministra inglesa, The,,,</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
font-size: 0.8em; 
color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="font-size: 1.2em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75142-estados-unidos-¿el-¡basta-de-sus-jovenes?" title="Estados Unidos: ¿El ¡basta! de sus jóvenes?">Estados Unidos: ¿El ¡basta! de sus jóvenes?</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 18 Marzo 2018, 07.18</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75142-estados-unidos-¿el-¡basta-de-sus-jovenes?"  ><img src="http://www.cubasi.cu/media/k2/items/cache/f521227a528a4ad631ea03ddaaf1769f_S.jpg" alt="Estados Unidos: ¿El ¡basta! de sus jóvenes?" title="Estados Unidos: ¿El ¡basta! de sus jóvenes?" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
color:#444;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Días atrás tuvieron lugar hechos en Estados Unidos que expertos llega,,,</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
font-size: 0.8em; 
color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="font-size: 1.2em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75049-colombia-entre-votos-e-incumplimientos" title="Colombia: Entre votos e incumplimientos">Colombia: Entre votos e incumplimientos</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 17 Marzo 2018, 06.26</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75049-colombia-entre-votos-e-incumplimientos"  ><img src="http://www.cubasi.cu/media/k2/items/cache/364cda7626d8888df95417c553685cdb_S.jpg" alt="Colombia: Entre votos e incumplimientos" title="Colombia: Entre votos e incumplimientos" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
color:#444;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Aunque disminuida, la derecha seguirá dominando en el legislativo colo,,,</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
font-size: 0.8em; 
color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="font-size: 1.2em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75085-al-escenario-a-y-el-escenario-b-de-la-crisis" title="Al escenario A y el escenario B de la crisis">Al escenario A y el escenario B de la crisis</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 16 Marzo 2018, 12.52</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75085-al-escenario-a-y-el-escenario-b-de-la-crisis"  ><img src="http://www.cubasi.cu/media/k2/items/cache/d69e32b2c43a08662d7619de523f8a17_S.jpg" alt="Al escenario A y el escenario B de la crisis" title="Al escenario A y el escenario B de la crisis" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
color:#444;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Todos los países tienen dos escenarios, el escenario A que es la vida ,,,</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
font-size: 0.8em; 
color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
			<div class="module-recomendamos">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-recomendamos"><div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75112-zona-critica-luces-y-sombras-de-lado-a-lado" title="ZONA CRÍTICA: Luces y sombras de Lado a lado">ZONA CRÍTICA: Luces y sombras de Lado a lado</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 17 Marzo 2018, 06.33</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75112-zona-critica-luces-y-sombras-de-lado-a-lado"  ><img src="http://www.cubasi.cu/media/k2/items/cache/a7262e14fd6e3d665b88f983b46b80cf_S.jpg" alt="ZONA CRÍTICA: Luces y sombras de Lado a lado" title="ZONA CRÍTICA: Luces y sombras de Lado a lado" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">La telenovela brasileña que concluyó por Cubavisión ofreció una puesta deslumbrante... pero no pudo escapar de cierto didactismo.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 6 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75017-fabula-de-la-otra-palomera" title="Fábula de la otra palomera">Fábula de la otra palomera</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 17 Marzo 2018, 05.09</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75017-fabula-de-la-otra-palomera"  ><img src="http://www.cubasi.cu/media/k2/items/cache/3db92ac048e5d1709c6f831fa046cf26_S.jpg" alt="Fábula de la otra palomera" title="Fábula de la otra palomera" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">La entrevista que no fue con la palomera que seguirá siendo.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 4 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75077-mirarnos-gente-por-la-mitad" title="MIRAR(NOS): Gente por la mitad">MIRAR(NOS): Gente por la mitad</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 16 Marzo 2018, 09.40</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75077-mirarnos-gente-por-la-mitad"  ><img src="http://www.cubasi.cu/media/k2/items/cache/fa176086e3905a7aae487916932a736e_S.jpg" alt="MIRAR(NOS): Gente por la mitad" title="MIRAR(NOS): Gente por la mitad" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">La amiga de una amiga me lo contó. Al principio pensé en todas las personas que emprenden ese tipo de relación.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 4 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74931-acoso-sexual-entre-la-demanda-justa-y-la-caceria-de-brujas" title="Acoso Sexual: Entre la demanda justa y la «cacería de brujas»">Acoso Sexual: Entre la demanda justa y la «cacería de brujas»</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 16 Marzo 2018, 05.19</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74931-acoso-sexual-entre-la-demanda-justa-y-la-caceria-de-brujas"  ><img src="http://www.cubasi.cu/media/k2/items/cache/828d9fa44bb3ed1b3585da9b7ecc2adc_S.jpg" alt="Acoso Sexual: Entre la demanda justa y la «cacería de brujas»" title="Acoso Sexual: Entre la demanda justa y la «cacería de brujas»" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">Nada justifica el abuso sexual, el acoso siempre será condenable. Pero algunas campañas justas pueden ser sombrillas para ciertos fundamentalismos…
</div><div> <div class='list_rel'><ul><li><span style=""><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/72457-deneuve-defienden-la-libertad-de-importunar-de-los-hombres" title="MIRAR(NOS): Gente por la mitad">Deneuve defiende «la libertad de importunar» de los hombres</a></span> </li></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 4 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75040-archivos-parlanchines-mago-y-senor-de-las-abejas" title="ARCHIVOS PARLANCHINES: Mago y señor de las abejas">ARCHIVOS PARLANCHINES: Mago y señor de las abejas</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 16 Marzo 2018, 05.10</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75040-archivos-parlanchines-mago-y-senor-de-las-abejas"  ><img src="http://www.cubasi.cu/media/k2/items/cache/39b51cc2b2ca1a8fffaf17f72d61b414_S.jpg" alt="ARCHIVOS PARLANCHINES: Mago y señor de las abejas" title="ARCHIVOS PARLANCHINES: Mago y señor de las abejas" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">Fisco Varela pertenece a esa muchedumbre de bolsillos rotos que ya no espera la llegada de los tesoros de la reina; se lo ha entregado todo a la naturaleza sin esperar la fugaz monedita del desabrigo.
</div><div> <div class='list_rel'><ul><li><span style=""><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74777-archivos-parlanchines-el-hombre-de-la-dentadura-de-acero" title="Acoso Sexual: Entre la demanda justa y la «cacería de brujas»">ARCHIVOS PARLANCHINES: El hombre de la dentadura de acero</a></span> </li></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 1 Comentario</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#fbc88d;">
<div> <span style="font-size: 1.3em;
line-height: 1.2em;
color:#000000;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75027-lo-que-significa-ver-al-rayo-mcqueen" title="Lo que significa ver al Rayo McQueen">Lo que significa ver al Rayo McQueen</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;
color:#444;">| 15 Marzo 2018, 05.50</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75027-lo-que-significa-ver-al-rayo-mcqueen"  ><img src="http://www.cubasi.cu/media/k2/items/cache/0d744d1ab3366ed92dfadd1f181f532b_S.jpg" alt="Lo que significa ver al Rayo McQueen" title="Lo que significa ver al Rayo McQueen" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #000;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;
">Al alcance de una memoria flash y el módico precio de cinco pesos, el llamado «paquete semanal» no es algo que se controle fácilmente.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em; color:#000;"> 31 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
	
				</div>
			</div>		
			<div id="gk-header1" style="width:36%;">
				<div class="module-separador">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-separador"><table style="border: 0px solid #C3C3C3;"><tr><td style="vertical-align: top;"><div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.7em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75206-relaciones-cuba-eeuu-retrocedieron-en-2017" target="_blank" title="Relaciones Cuba-EE.UU. retrocedieron en 2017">Relaciones Cuba-EE.UU. retrocedieron en 2017</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.9em;">| 19 Marzo 2018, 15.30</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">Las relaciones Cuba-Estados Unidos experimentaron el año pasado una regresión tras la llegada a la Casa Blanca del presidente Donald Trump, señala hoy la Cancillería de la isla en un resumen sobre política exterior.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</td></td></tr><tr><td style="vertical-align: top;"><div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.7em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75196-lamentable-accidente-de-transito-en-sancti-spiritus" target="_blank" title="Lamentable accidente de tránsito en Sancti Spíritus">Lamentable accidente de tránsito en Sancti Spíritus</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.9em;">| 19 Marzo 2018, 13.34</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">El hecho tuvo lugar al ocurrir una colisión entre un ómnibus, perteneciente al Grupo Empresarial Transtur S.A., y un camión de transporte obrero de la Empresa de Construcción y Montaje de Sancti Spíritus.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</td></td></tr></tr></table></div>					</div>
				</div>
			</div>
		</div>
			<div class="module-especial">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-especial"><div style="clear: both; background:#e7e7e7;">
<div> <span style="float:none; padding-top: 2px; 
margin-left:40px; border: 0px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74307-cuevas-de-bellamar-un-regalo-de-la-naturaleza- -video"  ><img src="http://www.cubasi.cu/media/k2/items/cache/6b90b91f4265eadcab1e922039e8155d_S.jpg" alt="Cuevas de Bellamar: un regalo de la naturaleza (+ Video)" title="Cuevas de Bellamar: un regalo de la naturaleza (+ Video)" border="0" width="300px" height="200px"/></a></span> </div><div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/74307-cuevas-de-bellamar-un-regalo-de-la-naturaleza- -video" title="Cuevas de Bellamar: un regalo de la naturaleza (+ Video)">Cuevas de Bellamar: un regalo de la naturaleza (+ Video)</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 18 Marzo 2018, 06.15</span>  </div><div>  </div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;"> 2 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
			<div class="module-actualvideo">
			<div>
				<div>
					<div>
											<iframe width="360" height="215" src="https://www.youtube.com/embed/3yhmEdGPR9A" frameborder="0" allow="autoplay; encrypted-media" allowfullscreen></iframe>					</div>
				</div>
			</div>
		</div>
			<div class="module-cultura">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-cultura"><div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75209-mujer-dice-tenia-17-cuando-seagal-la-agredio-sexualmente" title="Mujer dice tenía 17 cuando Seagal la agredió sexualmente">Mujer dice tenía 17 cuando Seagal la agredió sexualmente</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 17.17</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75209-mujer-dice-tenia-17-cuando-seagal-la-agredio-sexualmente"  ><img src="http://www.cubasi.cu/media/k2/items/cache/93d0e003c736aec963319e0b1458f030_S.jpg" alt="Mujer dice tenía 17 cuando Seagal la agredió sexualmente" title="Mujer dice tenía 17 cuando Seagal la agredió sexualmente" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Una mujer dijo el lunes que era una modelo y aspirante a actriz de 17 años cuando Steven Seagal la agredió sexualmente en una supuesta audición en un cuarto de hotel en el 2002.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75198-facebook-censura-el-cuadro-la-libertad-guiando-al-pueblo-por-un-pecho" title="Facebook censura el cuadro 'La Libertad guiando al pueblo' por un pecho">Facebook censura el cuadro 'La Libertad guiando al pueblo' por un pecho</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 14.03</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75198-facebook-censura-el-cuadro-la-libertad-guiando-al-pueblo-por-un-pecho"  ><img src="http://www.cubasi.cu/media/k2/items/cache/afdd9a9d89388f43f9dd4a5e97f6115f_S.jpg" alt="Facebook censura el cuadro 'La Libertad guiando al pueblo' por un pecho" title="Facebook censura el cuadro 'La Libertad guiando al pueblo' por un pecho" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Facebook continúa censurando obras de arte históricas de su muro por mostrar desnudos, ya sean parciales o completos.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75186-jim-carrey-hace-un-monstruoso-retrato-de-la-cara-de-la-casa-blanca" title="Jim Carrey hace un "monstruoso" retrato de la 'cara' de la Casa Blanca">Jim Carrey hace un "monstruoso" retrato de la 'cara' de la Casa Blanca</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 10.10</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75186-jim-carrey-hace-un-monstruoso-retrato-de-la-cara-de-la-casa-blanca"  ><img src="http://www.cubasi.cu/media/k2/items/cache/e725db9c8dd1d6991fcd1f8dda853d9c_S.jpg" alt="Jim Carrey hace un "monstruoso" retrato de la 'cara' de la Casa Blanca" title="Jim Carrey hace un "monstruoso" retrato de la 'cara' de la Casa Blanca" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">La noche del pasado sábado el actor estadounidense Jim Carrey hizo público en su cuenta de Twitter un retrato de la secretaria de prensa de la Casa Blanca Sarah Huckabee Sanders.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75185-descubren-un-secreto-escondido-en-uno-de-los-dibujos-mas-conocidos-de-miguel-angel" title="Descubren un secreto escondido en uno de los dibujos más conocidos de Miguel Ángel">Descubren un secreto escondido en uno de los dibujos más conocidos de Miguel Ángel</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 09.33</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75185-descubren-un-secreto-escondido-en-uno-de-los-dibujos-mas-conocidos-de-miguel-angel"  ><img src="http://www.cubasi.cu/media/k2/items/cache/bff061663123a5ed5ede18a480ed23c7_S.jpg" alt="Descubren un secreto escondido en uno de los dibujos más conocidos de Miguel Ángel" title="Descubren un secreto escondido en uno de los dibujos más conocidos de Miguel Ángel" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Los expertos han descubierto un autorretrato en una obra que se encuentra en el Museo Británico de Londres.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75184-el-tosco-llama-a-los-medios-a-promover-la-musica-cubana" title="«El Tosco» llama a los medios a promover la música cubana">«El Tosco» llama a los medios a promover la música cubana</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 09.27</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75184-el-tosco-llama-a-los-medios-a-promover-la-musica-cubana"  ><img src="http://www.cubasi.cu/media/k2/items/cache/4a4e310655c2a5928cadb166a4f28637_S.jpg" alt="«El Tosco» llama a los medios a promover la música cubana" title="«El Tosco» llama a los medios a promover la música cubana" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">“El Tosco” fue uno de los precursores de la timba cubana durante los años 90 y con NG La Banda ha defendido una sonoridad llena de complejidades rítmicas que expandió los límites de la música cubana.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 4 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75172-graban-en-haiti-video-oficial-de-copa-mundial-de-futbol" title="Graban en Haití video oficial de Copa Mundial de Fútbol">Graban en Haití video oficial de Copa Mundial de Fútbol</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 19 Marzo 2018, 06.07</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75172-graban-en-haiti-video-oficial-de-copa-mundial-de-futbol"  ><img src="http://www.cubasi.cu/media/k2/items/cache/4b633ad48beef32f1f903d4854f2c002_S.jpg" alt="Graban en Haití video oficial de Copa Mundial de Fútbol" title="Graban en Haití video oficial de Copa Mundial de Fútbol" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">El cantante haitiano-americano Jason Derulo permanece en Haití para grabar Colors, himno oficial de la Copa Mundial de Fútbol Rusia 2018.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75154-desarrollara-las-tunas-iv-taller-internacional-de-payasos" title="Desarrollará Las Tunas IV Taller Internacional de Payasos">Desarrollará Las Tunas IV Taller Internacional de Payasos</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 18 Marzo 2018, 11.04</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75154-desarrollara-las-tunas-iv-taller-internacional-de-payasos"  ><img src="http://www.cubasi.cu/media/k2/items/cache/c9cf543e0e74dc7215ab2c620a2a2d53_S.jpg" alt="Desarrollará Las Tunas IV Taller Internacional de Payasos" title="Desarrollará Las Tunas IV Taller Internacional de Payasos" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">El Taller Internacional de Payasos 2018 se prestigia con la asistencia de teatrólogos, críticos de artes y estudiosos de las artes escénicas.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75149-willy-toledo-asegura-que-no-comparecera-ante-el-juez-por-insultar-a-dios" title="Willy Toledo asegura que no comparecerá ante el juez por insultar a Dios">Willy Toledo asegura que no comparecerá ante el juez por insultar a Dios</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 18 Marzo 2018, 08.12</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75149-willy-toledo-asegura-que-no-comparecera-ante-el-juez-por-insultar-a-dios"  ><img src="http://www.cubasi.cu/media/k2/items/cache/cb6d43911eaa59e619f2c983922e5af7_S.jpg" alt="Willy Toledo asegura que no comparecerá ante el juez por insultar a Dios" title="Willy Toledo asegura que no comparecerá ante el juez por insultar a Dios" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">El actor Willy Toledo ha manifestado a través de su muro de Facebook su intención de no comparecer ante el juez en la vista fijada para el próximo 18 de abril por haber insultado a Dios y a la Virgen María en unos comen</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 3 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75148-¿esta-matt-damon-harto-de-trump?" title="¿Está Matt Damon harto de Trump?">¿Está Matt Damon harto de Trump?</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size:10px;
">| 18 Marzo 2018, 08.03</span>  </div><div>  </div><div>  </div><span style="float:left; padding: 2px; margin: 3px; border: 1px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75148-¿esta-matt-damon-harto-de-trump?"  ><img src="http://www.cubasi.cu/media/k2/items/cache/de3bb493d14d458d2ee6256d186d4c13_S.jpg" alt="¿Está Matt Damon harto de Trump?" title="¿Está Matt Damon harto de Trump?" border="0" width="90px" height="70px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Ante los rumores publicados en varios medios de prensa que afirman que Matt Damon estaba decidido a mudarse a Australia junto a su familia, harto de las políticas del presidente Donald Trump, la portavoz del actor desminti</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size:11px;color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
	
	</div>		
			<div id="gk-header2" style="width:28%;">
				<div class="two-modules">
			
					<div class="module-separador">
			<div>
				<div>
					<div>
											
<div id="k2ModuleBox1026" class="k2ItemsBlock -separador">

	
	  <ul>
        <li class="even lastItem">

      <!-- Plugins: BeforeDisplay -->
      
      <!-- K2 Plugins: K2BeforeDisplay -->
      
      
            <a class="moduleItemTitle" href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75204-firma-afiliada-a-trump-uso-facebook-para-propagar-noticias-falsas">Firma afiliada a Trump usó Facebook para propagar noticias falsas</a>
      
      
      <!-- Plugins: AfterDisplayTitle -->
      
      <!-- K2 Plugins: K2AfterDisplayTitle -->
      
      <!-- Plugins: BeforeDisplayContent -->
      
      <!-- K2 Plugins: K2BeforeDisplayContent -->
      
      
      
      <div class="clr"></div>

            <div class="moduleItemVideo">
      	<iframe width="255" height="215" src="https://www.youtube.com/embed/GQo9sWk0kOs" frameborder="0" allow="autoplay; encrypted-media" allowfullscreen></iframe>      	<span class="moduleItemVideoCaption"></span>
      	<span class="moduleItemVideoCredits"></span>
      </div>
      
      <div class="clr"></div>

      <!-- Plugins: AfterDisplayContent -->
      
      <!-- K2 Plugins: K2AfterDisplayContent -->
      
      
      
      
      
			
			
			
      <!-- Plugins: AfterDisplay -->
      
      <!-- K2 Plugins: K2AfterDisplay -->
      
      <div class="clr"></div>
    </li>
        <li class="clearList"></li>
  </ul>
  
	
	
</div>
					</div>
				</div>
			</div>
		</div>
			<div class="module-separador">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-separador"><div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75213-la-cumbre-del-g20-en-buenos-aires-abordara-el-proteccionismo-de-trump" title="La cumbre del G20 en Buenos Aires abordará el proteccionismo de Trump">La cumbre del G20 en Buenos Aires abordará el proteccionismo de Trump</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 21.19</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">Ministros de Economía y presidentes de bancos centrales discutirán la arquitectura financiera internacional en la capital de Argentina.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;   font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em; color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75212-venezuela-rechaza-ataque-economico-de-eeuu-contra-el-petro" title="Venezuela rechaza ataque económico de EE.UU. contra el Petro">Venezuela rechaza ataque económico de EE.UU. contra el Petro</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 21.13</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">El presidente venezolano emitió un comunicado para rechazar la prohibición de EE.UU. a que sus ciudadanos realicen transacciones a través de la criptomoneda venezolana.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;   font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em; color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75207-trump-propone-pena-de-muerte-para-traficantes-de-opiaceos" title="Trump propone pena de muerte para traficantes de opiáceos">Trump propone pena de muerte para traficantes de opiáceos</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 17.07</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">El plan del presidente Donald Trump para combatir el tráfico de opiáceos incluye la pena de muerte para quienes vendan ese tipo de drogas, reveló una fuente oficial.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;   font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em; color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75205-el-papa-francisco-dice-que-la-prostitucion-es-tortura" title="El papa Francisco dice que la prostitución es tortura">El papa Francisco dice que la prostitución es tortura</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 15.16</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">El papa Francisco pidió perdón el lunes por todos los cristianos que pagan para tener relaciones sexuales con mujeres, diciendo que los hombres que frecuentan prostitutas son criminales con una “mentalidad enfermiza” </div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;   font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em; color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; ">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;
font-family:Georgia,Times New Roman,Times,serif;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75197-nino-de-9-anos-mata-de-un-tiro-a-la-cabeza-a-su-hermana-de-13" title="Niño de 9 años mata de un tiro a la cabeza a su hermana de 13">Niño de 9 años mata de un tiro a la cabeza a su hermana de 13</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 13.55</span>  </div><div>  </div><div>  </div><div style="padding-top: 5px;
    color:#4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 17px;
    margin: 0 0 6px;">La pelea entre dos hermanos que reñían por el control de un videojuego terminó en tragedia, cuando el menor de ellos, de 9 años, acabó con la vida de la mayor, de 13, de un tiro en la cabeza.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;   font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em; color:#444;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
			<div class="module-deportes">
			<div>
				<div>
					<div>
											﻿<div id="aidanews-deportes"><div style="clear: both; background:#e7e7e7;">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75211-sancti-spiritus-nuevo-campeon-de-liga-superior-de-basquet-f" title="Sancti Spíritus, nuevo campeón de Liga Superior de básquet (f)">Sancti Spíritus, nuevo campeón de Liga Superior de básquet (f)</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 21.06</span>  </div><div>  </div><div>  </div><span style="float:none; padding-top: 2px; 
margin-left:5px; border: 0px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75211-sancti-spiritus-nuevo-campeon-de-liga-superior-de-basquet-f"  ><img src="http://www.cubasi.cu/media/k2/items/cache/dd5f7fa61f8b8c5b599a997deceeaaa5_S.jpg" alt="Sancti Spíritus, nuevo campeón de Liga Superior de básquet (f)" title="Sancti Spíritus, nuevo campeón de Liga Superior de básquet (f)" border="0" width="250px" height="160px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">El equipo de Sancti Spíritus se proclamó hoy por primera vez campeón de la Liga Superior de Baloncesto (LSB), rama femenina, al vencer 74-63 a Capitalinas en el tercer duelo de la final.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#e7e7e7;">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75192-roel-santos-he-recibido-varias-propuestas-de-contrato" title="Roel Santos: he recibido varias propuestas de contrato">Roel Santos: he recibido varias propuestas de contrato</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 11.16</span>  </div><div>  </div><div>  </div><span style="float:none; padding-top: 2px; 
margin-left:5px; border: 0px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75192-roel-santos-he-recibido-varias-propuestas-de-contrato"  ><img src="http://www.cubasi.cu/media/k2/items/cache/fb0a423d09c82ec74f606a3a1177a5e8_S.jpg" alt="Roel Santos: he recibido varias propuestas de contrato" title="Roel Santos: he recibido varias propuestas de contrato" border="0" width="250px" height="160px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">Roel Santos Martínez, considerado el mejor jardinero central y primer bateador del béisbol cubano en la actualidad, resaltó en Bayamo que ha recibido varias propuestas de contrato en ligas extranjeras.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em;"> 0 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
<div style="clear: both; background:#e7e7e7;">
<div> <span style="color:#005577;
font-size: 1.5em;
line-height: 1.2em;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75190-grand-prix-de-arqueria-cierre-dorado-con-nombre-de-franco" title="Grand Prix de arquería: Cierre dorado con nombre de Franco">Grand Prix de arquería: Cierre dorado con nombre de Franco</a></span>  </div><div> <span style="    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.7em;">| 19 Marzo 2018, 10.32</span>  </div><div>  </div><div>  </div><span style="float:none; padding-top: 2px; 
margin-left:5px; border: 0px solid #C3C3C3;"><a href="/cubasi-noticias-cuba-mundo-ultima-hora/item/75190-grand-prix-de-arqueria-cierre-dorado-con-nombre-de-franco"  ><img src="http://www.cubasi.cu/media/k2/items/cache/ac63c0ae9f8a936fb023c9ebc0fa96b0_S.jpg" alt="Grand Prix de arquería: Cierre dorado con nombre de Franco" title="Grand Prix de arquería: Cierre dorado con nombre de Franco" border="0" width="250px" height="160px"/></a></span><div style="padding-top: 5px;
    color: #4D4D4D;
    font-family: georgia,times new roman,times,serif;
    font-size: 16px;
    line-height: 16px;
    margin: 0 0 6px;">El pinareño, campeón nacional hace apenas una semana, derrotó este domingo 6-4 a Carlos Javier Rojas, segundo hombre del ranking en la cita. Se ganaron otras dos platas y un bronce.
</div><div> <div class='list_rel'><ul></ul></div><span style="padding: 0; margin: 0;    font-family: Arial,Helvetica,sans-serif;
    font-size: 0.8em;"> 1 Comentarios</span>  </div><div style="clear:both; height: 2px; width: 100%; border-bottom: 1px solid #c3c3c3"></div></div>
</div>					</div>
				</div>
			</div>
		</div>
	
				</div>
			</div>		
	</div>
                                            
                                           
                                          <!-- fin de los headers column -->
                                           
                   
                                             <!-- ***************************  -->                                     
                                                        <div id="l_w-2">
                                                                <div id="l_w-wrap" class="clearfix">
                                                                                                                                                <div id="conwrap">
                                                                                <div id="r_w1-2">
                                                                                
                                                                                                                                                                                <div class="us-stand">
                                                                                                		<div class="module-separador">
			<div>
				<div>
					<div>
											<div class="bannergroup-separador">


</div>					</div>
				</div>
			</div>
		</div>
	
                                                                                        </div>
                                                                                                          <!-- columnas header 456 -->
                  
                                                               <!-- headers column -->
                                             





                                                                                                                                                                           
                                                                                                                                                                                                                                                                </div>
                                                                                                                                                                
                                                                        </div>
                                                                </div>
                                                        </div>
                                                      
                                                                                                                                                                        <div id="user_position-5">
                                                                <div>
                                                                        <div>
                                                                                <div class="clearfix">
                                                                                                                                                                                                                                                                        <div class="us_5-center  ">
                                                                                                		<div class="moduletable-extra">
					<center>
<p><span style="color: #ffffff;"><strong>Prensa Cubana</strong></span></p>
<p> </p>
<center><center>
<table style="width: 708px;" border="0">
<tbody>
<tr>
<td style="text-align: center;"><a href="http://www.sierramaestra.cu"><span style="color: #ffffff;"><strong>Sierra Maestra</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.cubahora.cu"><span style="color: #ffffff;"><strong>CubAhora</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.trabajadores.cu"><span style="color: #ffffff;"><strong>Trabajadores</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.tribuna.cu"><span style="color: #ffffff;"><strong>Tribuna</strong></span></a></td>
</tr>
<tr>
<td style="text-align: center;"><a href="http://www.venceremos.cu"><span style="color: #ffffff;"><strong>Venceremos</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.juventudrebelde.cu/"><span style="color: #ffffff;"><strong>Juventud Rebelde</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.vanguardia.cu"><span style="color: #ffffff;"><strong>Vanguardia</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.cubadebate.cu"><span style="color: #ffffff;"><strong>Cubadebate</strong></span></a></td>
</tr>
<tr>
<td style="text-align: center;"><a href="http://www.met.inf.cu/asp/genesis.asp?TB0=PLANTILLAS&amp;TB1=INICIAL"><span style="color: #ffffff;"><strong> Meteorología</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.granma.cu/"><span style="color: #ffffff;"><strong>Granma</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.radioreloj.cu"><span style="color: #ffffff;"><strong>Radio Reloj</strong></span></a></td>
<td style="text-align: center;"><a href="http://www.cubatv.cu"><span style="color: #ffffff;"><strong> Informativo de la TVC</strong></span></a></td>
</tr>
</tbody>
</table>
<center></center></center></center></center>		</div>
	
                                                                                        </div>
                                                                                                                                                                                                                                                                </div>
                                                                        </div>
                                                                </div>
                                                        </div>
                                                                                                        </div>
                                        </div>
                                </div>
                        </div>
                                                <div>
                                       <div id="footer">
                 		<div class="module-seprojo">
			<div>
				<div>
					<div>
											<p>                                                                                                         </p>
<p>                                                                                  Copyright © 2014 Cuba Si. Todos los derechos reservados.<br />                                                                                       Es un portal de Noticias basado en Software libre          <br />                                                                                                                    ETECSA</p>
<p>                                                                                                                                                          </p>
<p>                                                                       </p>
<p>                                                                                                                   <span style="font-size: 14pt;"> <a href="index.php?option=com_contact&view=contact&catid=12&id=3-cubasi"><span style="background-color: #ffffff; color: #ff0000;">Quiénes Somos</span></a></span></p>					</div>
				</div>
			</div>
		</div>
			<div class="module">
			<div>
				<div>
					<div>
																</div>
				</div>
			</div>
		</div>
	
                 </div>
                              </div>
                        <div id="footer_wrap">
                                               
                                                                            
                                </div>
                        </div>
                </div>
                                               

                                <div id="footer">
                                        
                
                <div id="bottom">
                </div>
        </div>
</div>
<!-- Piwik -->
<script type="text/javascript">
  var _paq = _paq || [];
  _paq.push(['trackPageView']);
  _paq.push(['enableLinkTracking']);
  (function() {
    var u="//estadisticas.cubasi.cu/";
    _paq.push(['setTrackerUrl', u+'piwik.php']);
    _paq.push(['setSiteId', 1]);
    var d=document, g=d.createElement('script'), s=d.getElementsByTagName('script')[0];
    g.type='text/javascript'; g.async=true; g.defer=true; g.src=u+'piwik.js'; s.parentNode.insertBefore(g,s);
  })();
</script>
<noscript><p><img src="//estadisticas.cubasi.cu/piwik.php?idsite=1" style="border:0;" alt="" /></p></noscript>
<!-- End Piwik Code -->
</body>
</html>