#include "hip/hip_runtime.h"
﻿<!DOCTYPE html>
<html lang="es">
    <head>        
		<meta http-equiv="X-UA-Compatible" content="IE=edge">
		<meta charset="UTF-8">
		<link rel="alternate" type="application/rss+xml" title="RSS 2.0" href="http://www.uci.cu/rss.xml">
		<meta name="twitter:card" content="summary">
		<meta name="twitter:creator" content="@universidad_uci">
		<meta name="twitter:site" content="@universidad_uci">
		<meta name="twitter:title" content="Universidad UCI">
		<meta name="twitter:description" content="Cuenta Oficial de la Universidad de las Ciencias Informáticas (UCI). Institución educacional y productiva que forma profesionales de excelencia.">
		<meta name="viewport" content="width=device-width, initial-scale=1">
        <meta http-equiv="Content-Type" content="text/html; charset=utf-8" />
<link rel="shortcut icon" href="http://www.uci.cu/sites/default/files/favicon_0_0.png" type="image/png" />
<link rel="image_src" href="http://www.uci.cu/sites/default/files/imagenes/portal/imagen-uci.jpg" />
<meta property="rc:admins" content="9500dc9297105c315509b5737b4c16d12143ee315531aa3286b3b7e54ab57c16" />
<meta name="description" content="La Universidad de las Ciencias Informáticas es un centro de estudios universitarios radicado en La Habana, Cuba, en el municipio de Boyeros" />
<meta name="abstract" content="La Universidad de las Ciencias Informáticas es un centro de estudios universitarios radicado en La Habana, Cuba, en el municipio de Boyeros" />
<meta name="keywords" content="universidad, ciencias, informática, uci, cuba" />
<link rel="canonical" href="http://www.uci.cu/inicio" />
<link rel="shortlink" href="http://www.uci.cu/inicio" />
<meta name="geo.placename" content="Universidad de las Ciencias Informáticas" />
<meta name="geo.position" content="22.9873463;-82.4657985" />
<meta name="geo.region" content="CU-LA" />
<meta name="icbm" content="22.9873463,-82.4657985" />
<meta property="og:site_name" content="Universidad de las Ciencias Informáticas" />
<meta property="og:type" content="website" />
<meta property="og:url" content="http://www.uci.cu" />
<meta property="og:title" content="Universidad de las Ciencias Informáticas" />
<meta http-equiv="cache-control" content="today" />
<meta property="og:determiner" content="auto" />
<meta property="og:description" content="La Universidad de las Ciencias Informáticas es un centro de estudios universitarios radicado en La Habana, Cuba, en el municipio de Boyeros, en la carretera con dirección a San Antonio de los Baños" />
<meta property="og:image:url" content="http://www.uci.cu/sites/default/files/imagenes/portal/imagen-uci.jpg" />
<meta property="og:image:type" content="image/jpeg" />
<meta property="og:image:width" content="4608" />
<meta property="og:image:height" content="1632" />
<meta property="og:email" content="uci@uci.cu" />
<meta property="og:phone_number" content="+5378372548" />
<meta property="og:locale" content="es_ES" />
<meta property="og:locale:alternate" content="en_US" />
        <title>Universidad de las Ciencias Informáticas</title>			
    </head>
    <body class="html front not-logged-in no-sidebars page-inicio i18n-es">
                <div class="site-preloader" style="visibility: visible;	position: fixed;	left: 0;	right: 0;	top: 0;	bottom: 0;	width: 100%;	height: 100%;	z-index: 9999;	background: #202a62;	text-align: center;">
	<div class="site-preloader-inner" style="width: 190px;	height: 140px;	top: 50%;	position: absolute;	left: 50%;		margin-left: -95px;	margin-top: -70px;	text-align: center;">
	<img src="/sites/all/themes/uci/images/logo-vertical.png" style="margin-bottom: 40px;" width="190px" alt="UCI" title="UCI"/>
	<div class="thecube" style="width: 32px;	height: 32px;	-webkit-transform: rotateZ(45deg);	transform: rotateZ(45deg);	-webkit-transition: all 2.6s ease-in-out;	transition: all 2.6s ease-in-out; display: inline-block;">
		<div class="cube c1" style=" 	position: relative;	float: left;	width: 16px;	height: 16px;"></div>
		<div class="cube c2" style=" 	position: relative;	float: left;	width: 16px;	height: 16px;  -webkit-transform: rotateZ(90deg); transform: rotateZ(90deg);"></div>
		<div class="cube c4" style=" 	position: relative;	float: left;	width: 16px;	height: 16px; -webkit-transform: rotateZ(270deg); transform: rotateZ(270deg); "></div>
		<div class="cube c3" style=" 	position: relative;	float: left;	width: 16px;	height: 16px;  -webkit-transform: rotateZ(180deg); transform: rotateZ(180deg);"></div>
	</div>
	</div>
</div>

    <section id="slide">
        <div class="col-md-5 col-sm-6 hidden-xs header-cold-left">
            <div class="site-logo">
                <a href="/" title="Inicio" rel="home" class="link-logo">
                    <img class="logo" src="http://www.uci.cu/sites/all/themes/uci/logo.png" alt="Inicio" width="190px" height="73px" />
                </a>
            </div>
            <div class="boton-bienvenido"><a href="#">Bienvenidos</a></div>
            <div class="header-relleno"></div>
        </div>
        <div class="col-xs-12 visible-xs header-cold-mobile-left">
            <div class="site-logo-mobile text-center">
                <a href="/" title="Inicio" rel="home" class="link-logo">
                    <img src="http://www.uci.cu/sites/all/themes/uci/logo.png" alt="Inicio"  width="190px" height="73px"  />
                </a>
            </div>
            <div class="boton-bienvenido-movil text-center"><a href="#">Bienvenidos</a></div>
        </div>		
        <div class="region region-slide">
  <div id="block-slider-slider-block" class="block block-slider" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div  style="background-image:url(http://www.uci.cu/sites/default/files/imagenes_promocional/estudiantes-5_1.jpg)" class="flexslider-container"><div class="flexslider"><ul class="slides"><li data-image="http://www.uci.cu/sites/default/files/imagenes_promocional/estudiantes-5_1.jpg"><div class="slide-trama-bg"><div class="slide-color-bg"><div data-wow-delay="1.8s" class="wow fadeInRight flex-caption col-md-7 col-md-offset-5 col-sm-6 col-sm-offset-6">
   <div class="flex-caption-content contextual-links-region"><div class="flex-caption-content-content">

   <div class="field field-name-body field-type-text-with-summary field-label-hidden">
<div class="field-items">
<div class="field-item even" property="content:encoded">
<p>CONECTADOS AL FUTURO</p>

<p>CONECTADOS A LA REVOLUCIÓN</p>
</div>
</div>
</div>
</div></div></div></div></div></li><li data-image="http://www.uci.cu/sites/default/files/imagenes_promocional/estudiantes-4_1.jpg"><div class="slide-trama-bg"><div class="slide-color-bg"><div data-wow-delay="1.8s" class="wow fadeInRight flex-caption col-md-7 col-md-offset-5 col-sm-6 col-sm-offset-6">
   <div class="flex-caption-content contextual-links-region"><div class="flex-caption-content-content">

   <p>CONECTADOS AL FUTURO</p>

<p>CONECTADOS A LA REVOLUCIÓN</p>
</div></div></div></div></div></li><li data-image="http://www.uci.cu/sites/default/files/imagenes_promocional/estudiantes-2_1.jpg"><div class="slide-trama-bg"><div class="slide-color-bg"><div data-wow-delay="1.8s" class="wow fadeInRight flex-caption col-md-7 col-md-offset-5 col-sm-6 col-sm-offset-6">
   <div class="flex-caption-content contextual-links-region"><div class="flex-caption-content-content">

   <p>CONECTADOS AL FUTURO</p>

<p>CONECTADOS A LA REVOLUCIÓN</p>
</div></div></div></div></div></li></ul></div></div>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
    </section>


<div class='dt-mobile-header'>
           <div class="dt-close-mobile-menu-icon"><span class="glyphicon glyphicon-remove"></span></div>
			<div class="text-center">
                <a href="/" title="Inicio" rel="home" class="link-logo">
                    <img src="http://www.uci.cu/sites/all/themes/uci/logo.png" alt="Inicio"  width="190px" height="73px"  />
                </a>
            </div>
			<div class="menu-principal-telefono"> <ul class="menu"><li class="first leaf"><a href="/#header" title="Inicio" class="scrolltop active">Inicio</a></li>
<li class="expanded"><span title="Sobre la Universidad" class="nolink">Universidad</span><ul class="menu"><li class="first leaf"><a href="/universidad/la-uci-de-un-vistazo" title="La UCI de un vistazo">La UCI de un vistazo</a></li>
<li class="leaf"><a href="/universidad/historia" title="Historia">Historia</a></li>
<li class="leaf"><a href="/universidad/mision" title="Misión">Misión</a></li>
<li class="leaf"><a href="/universidad/claustro" title="Claustro de excelencia">Claustro</a></li>
<li class="leaf"><a href="/universidad/estructura" title="Estructura">Estructura</a></li>
<li class="leaf"><a href="/universidad/campus" title="Campus universitario">Campus</a></li>
<li class="leaf"><a href="/universidad/asociaciones" title="Asociaciones">Asociaciones</a></li>
<li class="last leaf"><a href="/universidad/noticias" title="Noticias">Noticias</a></li>
</ul></li>
<li class="expanded"><span title="Estudios" class="nolink">Estudios</span><ul class="menu"><li class="first leaf"><a href="/estudios/aspirantes" title="Aspirantes">Aspirantes</a></li>
<li class="leaf"><a href="/estudios/pregrado" title="Pregrado">Pregrado</a></li>
<li class="leaf"><a href="/estudios/posgrado" title="Posgrado">Posgrado</a></li>
<li class="last leaf"><a href="/estudios/educacion-a-distancia" title="Educación a Distancia">Educación a Distancia</a></li>
</ul></li>
<li class="expanded"><span title="Investigación y Desarrollo" class="nolink">Investigación y Desarrollo</span><ul class="menu"><li class="first leaf"><a href="/investigacion-y-desarrollo/eventos" title="Eventos">Eventos</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/consejo-cientifico" title="Consejo Científico">Consejo Científico</a></li>
<li class="leaf"><a href="/universidad/investigacion-y-desarrollo/lineas-cientificas" title="Líneas científicas">Líneas científicas</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/grupos-de-investigacion" title="Grupos de investigación">Grupos de investigación</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/centros-de-estudio" title="Centros de Estudio">Centros de Estudio</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/centros-de-desarrollo" title="Centros de Desarrollo">Centros de Desarrollo</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/productos" title="Productos">Productos</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/servicios" title="Servicios">Servicios</a></li>
<li class="leaf"><a href="/investigacion-y-desarrollo/editorial" title="Editorial">Editorial</a></li>
<li class="last leaf"><a href="/investigacion-y-desarrollo/publicaciones" title="Publicaciones">Publicaciones</a></li>
</ul></li>
<li class="last expanded"><span class="nolink">Vida Universitaria</span><ul class="menu"><li class="first leaf"><a href="/vida-universitaria/cultura">Cultura</a></li>
<li class="leaf"><a href="/vida-universitaria/deporte">Deporte</a></li>
<li class="leaf"><a href="/vida-universitaria/comunicacion">Comunicación</a></li>
<li class="leaf"><a href="/vida-universitaria/residencia">Residencia</a></li>
<li class="leaf"><a href="/vida-universitaria/bienestar-universitario">Bienestar Universitario</a></li>
<li class="last leaf"><a href="/vida-universitaria/informacion-cientifico-tecnica">Información Científico Técnica</a></li>
</ul></li>
</ul></div>
			<div class="region region-opciones-encabezado">
  <div id="block-locale-language" class="block block-locale" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="language-switcher-locale-url"><li class="es first active"><a href="/" class="language-link active" xml:lang="es">ESP</a></li>
<li class="en last"><a href="/en" class="language-link" xml:lang="en">ENG</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-menu-menu-opciones-de-encabezado" class="block block-menu" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="/directorio/personas" title="Directorio" class="directorio">Directorio</a></li>
<li class="leaf"><a href="/galeria" title="Galería de Imágenes" class="galeria">Galeria</a></li>
<li class="last leaf"><a href="/contactenos" title="Contactos" class="contactos">Contactos</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-menu-menu-enlaces-redes-sociales" class="block block-menu" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="https://www.facebook.com/universidad.uci" title="Síguenos en Facebook" rel="nofollow" class="facebook" target="_blank">Facebook</a></li>
<li class="leaf"><a href="https://www.twitter.com/universidad_uci" title="Síguenos en Twitter" rel="nofollow" class="twitter" target="_blank">Twitter</a></li>
<li class="leaf"><a href="https://www.linkedin.com/school/15097469" title="Síguenos en Linkedin" rel="nofollow" class="linkedin" target="_blank">Linkedin</a></li>
<li class="leaf"><a href="http://www.youtube.com/user/informativouci?feature=BF" title="Síguenos en Youtube" rel="nofollow" class="youtube" target="_blank">Youtube</a></li>
<li class="last leaf"><a href="/rss.xml" title="RSS" class="rss" target="_blank">RSS</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-search-form" class="block block-search" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <form autocomplete="off" action="/" method="post" id="search-block-form" accept-charset="UTF-8"><div><div class="container-inline">
      <h2 class="element-invisible">Formulario de búsqueda</h2>
    <div class="form-item form-type-textfield form-item-search-block-form">
  <label class="element-invisible" for="edit-search-block-form--2">Buscar </label>
 <input title="Escriba lo que quiere buscar." placeholder="Buscar..." type="text" id="edit-search-block-form--2" name="search_block_form" value="" size="15" maxlength="128" class="form-text" />
</div>
<div class="form-actions form-wrapper" id="edit-actions"><input type="submit" id="edit-submit" name="op" value="Buscar" class="form-submit" /></div><input type="hidden" name="form_build_id" value="form-rGoEVGs2sr69BFv1cApsjOerAIKRRgJBp9rxLQeuZcs" />
<input type="hidden" name="form_id" value="search_block_form" />
</div>
</div></form>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>


	
<header id="header">
    <div class="header-content">
        <div class="container hidden-xs no-mobile">
            <div class="row">
                <div class="col-md-3 col-sm-3 col-xs-12 logo-div">
                                            <a class="scrolltop link-logo" href="/#header" title="Inicio" rel="home">
                            <img src="http://www.uci.cu/sites/all/themes/uci/logo.png" alt="Inicio"  width="190px" height="73px"  />
                        </a>
                                    </div>
                <div class="col-md-9 col-sm-9 col-xs-12 hidden-xs text-right header-options">
                    <div class="region region-opciones-encabezado">
  <div class="region region-opciones-encabezado">
  <div id="block-locale-language" class="block block-locale" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="language-switcher-locale-url"><li class="es first active"><a href="/" class="language-link active" xml:lang="es">ESP</a></li>
<li class="en last"><a href="/en" class="language-link" xml:lang="en">ENG</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-menu-menu-opciones-de-encabezado" class="block block-menu" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="/directorio/personas" title="Directorio" class="directorio">Directorio</a></li>
<li class="leaf"><a href="/galeria" title="Galería de Imágenes" class="galeria">Galeria</a></li>
<li class="last leaf"><a href="/contactenos" title="Contactos" class="contactos">Contactos</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-menu-menu-enlaces-redes-sociales" class="block block-menu" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="https://www.facebook.com/universidad.uci" title="Síguenos en Facebook" rel="nofollow" class="facebook" target="_blank">Facebook</a></li>
<li class="leaf"><a href="https://www.twitter.com/universidad_uci" title="Síguenos en Twitter" rel="nofollow" class="twitter" target="_blank">Twitter</a></li>
<li class="leaf"><a href="https://www.linkedin.com/school/15097469" title="Síguenos en Linkedin" rel="nofollow" class="linkedin" target="_blank">Linkedin</a></li>
<li class="leaf"><a href="http://www.youtube.com/user/informativouci?feature=BF" title="Síguenos en Youtube" rel="nofollow" class="youtube" target="_blank">Youtube</a></li>
<li class="last leaf"><a href="/rss.xml" title="RSS" class="rss" target="_blank">RSS</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
<div id="block-search-form" class="block block-search" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <form autocomplete="off" action="/" method="post" id="search-block-form" accept-charset="UTF-8"><div><div class="container-inline">
      <h2 class="element-invisible">Formulario de búsqueda</h2>
    <div class="form-item form-type-textfield form-item-search-block-form">
  <label class="element-invisible" for="edit-search-block-form--2">Buscar </label>
 <input title="Escriba lo que quiere buscar." placeholder="Buscar..." type="text" id="edit-search-block-form--2" name="search_block_form" value="" size="15" maxlength="128" class="form-text" />
</div>
<div class="form-actions form-wrapper" id="edit-actions"><input type="submit" id="edit-submit" name="op" value="Buscar" class="form-submit" /></div><input type="hidden" name="form_build_id" value="form-rGoEVGs2sr69BFv1cApsjOerAIKRRgJBp9rxLQeuZcs" />
<input type="hidden" name="form_id" value="search_block_form" />
</div>
</div></form>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>
 <!-- /.region -->
                </div>
                <div class="col-md-9 col-sm-12 col-xs-12 hidden-xs text-right menu-principal-expandido"><nav><div class="region region-menu-principal">
  <div id="block-superfish-1" class="block block-superfish" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul  id="superfish-1" class="menu sf-menu sf-main-menu sf-horizontal sf-style-none sf-total-items-5 sf-parent-items-4 sf-single-items-1"><li id="menu-465-1" class="active-trail first odd sf-item-1 sf-depth-1 sf-no-children"><a href="/#header" title="Inicio" class="scrolltop sf-depth-1 active">Inicio</a></li><li id="menu-466-1" class="middle even sf-item-2 sf-depth-1 sf-total-children-8 sf-parent-children-0 sf-single-children-8 menuparent"><span title="Sobre la Universidad" class="sf-depth-1 menuparent nolink">Universidad</span><ul><li id="menu-1224-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/universidad/la-uci-de-un-vistazo" title="La UCI de un vistazo" class="sf-depth-2">La UCI de un vistazo</a></li><li id="menu-1226-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/universidad/historia" title="Historia" class="sf-depth-2">Historia</a></li><li id="menu-538-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/universidad/mision" title="Misión" class="sf-depth-2">Misión</a></li><li id="menu-1217-1" class="middle even sf-item-4 sf-depth-2 sf-no-children"><a href="/universidad/claustro" title="Claustro de excelencia" class="sf-depth-2">Claustro</a></li><li id="menu-540-1" class="middle odd sf-item-5 sf-depth-2 sf-no-children"><a href="/universidad/estructura" title="Estructura" class="sf-depth-2">Estructura</a></li><li id="menu-541-1" class="middle even sf-item-6 sf-depth-2 sf-no-children"><a href="/universidad/campus" title="Campus universitario" class="sf-depth-2">Campus</a></li><li id="menu-542-1" class="middle odd sf-item-7 sf-depth-2 sf-no-children"><a href="/universidad/asociaciones" title="Asociaciones" class="sf-depth-2">Asociaciones</a></li><li id="menu-1230-1" class="last even sf-item-8 sf-depth-2 sf-no-children"><a href="/universidad/noticias" title="Noticias" class="sf-depth-2">Noticias</a></li></ul></li><li id="menu-467-1" class="middle odd sf-item-3 sf-depth-1 sf-total-children-4 sf-parent-children-0 sf-single-children-4 menuparent"><span title="Estudios" class="sf-depth-1 menuparent nolink">Estudios</span><ul><li id="menu-551-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/estudios/aspirantes" title="Aspirantes" class="sf-depth-2">Aspirantes</a></li><li id="menu-460-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/estudios/pregrado" title="Pregrado" class="sf-depth-2">Pregrado</a></li><li id="menu-552-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/estudios/posgrado" title="Posgrado" class="sf-depth-2">Posgrado</a></li><li id="menu-666-1" class="last even sf-item-4 sf-depth-2 sf-no-children"><a href="/estudios/educacion-a-distancia" title="Educación a Distancia" class="sf-depth-2">Educación a Distancia</a></li></ul></li><li id="menu-468-1" class="middle even sf-item-4 sf-depth-1 sf-total-children-10 sf-parent-children-0 sf-single-children-10 menuparent"><span title="Investigación y Desarrollo" class="sf-depth-1 menuparent nolink">Investigación y Desarrollo</span><ul><li id="menu-1253-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/eventos" title="Eventos" class="sf-depth-2">Eventos</a></li><li id="menu-1636-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/consejo-cientifico" title="Consejo Científico" class="sf-depth-2">Consejo Científico</a></li><li id="menu-1689-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/universidad/investigacion-y-desarrollo/lineas-cientificas" title="Líneas científicas" class="sf-depth-2">Líneas científicas</a></li><li id="menu-1252-1" class="middle even sf-item-4 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/grupos-de-investigacion" title="Grupos de investigación" class="sf-depth-2">Grupos de investigación</a></li><li id="menu-1637-1" class="middle odd sf-item-5 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/centros-de-estudio" title="Centros de Estudio" class="sf-depth-2">Centros de Estudio</a></li><li id="menu-555-1" class="middle even sf-item-6 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/centros-de-desarrollo" title="Centros de Desarrollo" class="sf-depth-2">Centros de Desarrollo</a></li><li id="menu-1228-1" class="middle odd sf-item-7 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/productos" title="Productos" class="sf-depth-2">Productos</a></li><li id="menu-1106-1" class="middle even sf-item-8 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/servicios" title="Servicios" class="sf-depth-2">Servicios</a></li><li id="menu-580-1" class="middle odd sf-item-9 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/editorial" title="Editorial" class="sf-depth-2">Editorial</a></li><li id="menu-579-1" class="last even sf-item-10 sf-depth-2 sf-no-children"><a href="/investigacion-y-desarrollo/publicaciones" title="Publicaciones" class="sf-depth-2">Publicaciones</a></li></ul></li><li id="menu-469-1" class="last odd sf-item-5 sf-depth-1 sf-total-children-6 sf-parent-children-0 sf-single-children-6 menuparent"><span class="sf-depth-1 menuparent nolink">Vida Universitaria</span><ul><li id="menu-518-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/vida-universitaria/cultura" class="sf-depth-2">Cultura</a></li><li id="menu-535-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/vida-universitaria/deporte" class="sf-depth-2">Deporte</a></li><li id="menu-536-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/vida-universitaria/comunicacion" class="sf-depth-2">Comunicación</a></li><li id="menu-537-1" class="middle even sf-item-4 sf-depth-2 sf-no-children"><a href="/vida-universitaria/residencia" class="sf-depth-2">Residencia</a></li><li id="menu-1638-1" class="middle odd sf-item-5 sf-depth-2 sf-no-children"><a href="/vida-universitaria/bienestar-universitario" class="sf-depth-2">Bienestar Universitario</a></li><li id="menu-1639-1" class="last even sf-item-6 sf-depth-2 sf-no-children"><a href="/vida-universitaria/informacion-cientifico-tecnica" class="sf-depth-2">Información Científico Técnica</a></li></ul></li></ul>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</nav></div>           
            </div> 
        </div>
		<div class="container header-small visible-xs">
			<a href="#" class="dt-mobile-menu-icon floating-btn visible-xs">
						<div class="line"></div>
						<div class="line"></div>
						<div class="line"></div>
			</a>
                                            <a class="scrolltop link-logo" href="/#header" title="Inicio" rel="home">
                            <img src="http://www.uci.cu/sites/all/themes/uci/logo.png" alt="Inicio"  width="190px" height="73px"  />
                        </a>
                    		</div> 
    </div> 
</header>

    <section id="content" class='seccion-conent'>                            
         <!-- /.region -->
    </section>

  
  
  
    <section id="noticias" class='seccion-inicio'>
		<div class="container">
        <div class="row">
            <div class="col-md-12">
        <div class="region region-noticias">
  <div id="block-views-slider-noticia-block" class="block block-views" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-slider-noticia view-id-slider_noticia view-display-id-block view-dom-id-44d4f8e3509d0fdbd6413bf2d65d309c">
        
  
  
      <div class="view-content">
      <div class="noticias_flexslider">    <ul class="slides">          <li class="contextual-links-region">  
          <div class="row">
    <div class="col-md-4 col-texto-noticia">
        <div class="title-block-noticia noticia-right"><h2 class="title-block">Noticias</h2><a href="/universidad/noticias" title="Todas las noticias" class="todas-noticias">Todas las noticias</a></div>
        <div class="line-block-noticia"><div class="line-block-noticia-inner"></div></div>
        <div class="contextual-links-region">
        <div class="titulo-noticia"><a href="/universidad/noticias/la-uci-las-puertas-de-informatica-2018">La UCI a las puertas de Informática 2018</a></div>
        <div class="fecha-noticia">
            <span class="fecha-noticia-dias"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-19T08:15:00-04:00">19/3/2018</span> | </span>
            <span class="fecha-noticia-hora"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-19T08:15:00-04:00">8:15 am</span></span>
        </div>
        <div class="texto-noticia noticia-right"><p>Un espacio para debatir, conocer e intercambiar alrededor de los nuevos avances del mundo de las Tic, resultará Informática 2018</p>
</div>
        <div class="noticias_more_link noticia-right"><a href="/universidad/noticias/la-uci-las-puertas-de-informatica-2018">leer más</a></div>
    </div>
</div>
<div class="col-md-8 col-imagen-noticia"><div class="col-imagen-noticia-content"><img alt="La UCI a las puertas de Informática 2018" title="La UCI a las puertas de Informática 2018" data-image="http://www.uci.cu/sites/default/files/styles/slide_noticia_1001x450/public/imagenes/noticias/stand1-uci.jpg?itok=Lvf70g3s" src=""></div></div>
</div>  </li>
          <li class="contextual-links-region">  
          <div class="row">
    <div class="col-md-4 col-texto-noticia">
        <div class="title-block-noticia noticia-right"><h2 class="title-block">Noticias</h2><a href="/universidad/noticias" title="Todas las noticias" class="todas-noticias">Todas las noticias</a></div>
        <div class="line-block-noticia"><div class="line-block-noticia-inner"></div></div>
        <div class="contextual-links-region">
        <div class="titulo-noticia"><a href="/universidad/noticias/abogan-en-la-uci-por-el-bienestar-universitario">Abogan en la UCI por el bienestar universitario</a></div>
        <div class="fecha-noticia">
            <span class="fecha-noticia-dias"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-17T14:15:00-04:00">17/3/2018</span> | </span>
            <span class="fecha-noticia-hora"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-17T14:15:00-04:00">2:15 pm</span></span>
        </div>
        <div class="texto-noticia noticia-right"><p>El encuentro, que sesiona en la Sala de eventos del docente Camilo Cienfuegos, reúne a más de 40 delegados en representación de 11 instituciones del país</p>
</div>
        <div class="noticias_more_link noticia-right"><a href="/universidad/noticias/abogan-en-la-uci-por-el-bienestar-universitario">leer más</a></div>
    </div>
</div>
<div class="col-md-8 col-imagen-noticia"><div class="col-imagen-noticia-content"><img alt="Abogan en la UCI por el bienestar universitario" title="Abogan en la UCI por el bienestar universitario" data-image="http://www.uci.cu/sites/default/files/styles/slide_noticia_1001x450/public/imagenes/noticias/abogan-en-la-uci-por-el-bienestar-universitario-1.jpg?itok=siG6tmCV" src=""></div></div>
</div>  </li>
          <li class="contextual-links-region">  
          <div class="row">
    <div class="col-md-4 col-texto-noticia">
        <div class="title-block-noticia noticia-right"><h2 class="title-block">Noticias</h2><a href="/universidad/noticias" title="Todas las noticias" class="todas-noticias">Todas las noticias</a></div>
        <div class="line-block-noticia"><div class="line-block-noticia-inner"></div></div>
        <div class="contextual-links-region">
        <div class="titulo-noticia"><a href="/universidad/noticias/informatica-2018-comunicacion-y-aprendizaje">Informática 2018: comunicación y aprendizaje</a></div>
        <div class="fecha-noticia">
            <span class="fecha-noticia-dias"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-16T08:30:00-04:00">16/3/2018</span> | </span>
            <span class="fecha-noticia-hora"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-16T08:30:00-04:00">8:30 am</span></span>
        </div>
        <div class="texto-noticia noticia-right"><p>Informática 2018 será un excelente espacio de socialización, concertación, participación; plataforma interactiva ideal para compartir, comunicar nuestro quehacer y aprender</p>
</div>
        <div class="noticias_more_link noticia-right"><a href="/universidad/noticias/informatica-2018-comunicacion-y-aprendizaje">leer más</a></div>
    </div>
</div>
<div class="col-md-8 col-imagen-noticia"><div class="col-imagen-noticia-content"><img alt="Informática 2018: comunicación y aprendizaje" title="Informática 2018: comunicación y aprendizaje" data-image="http://www.uci.cu/sites/default/files/styles/slide_noticia_1001x450/public/imagenes/noticias/informatica-2018.jpg?itok=M2RDhr1d" src=""></div></div>
</div>  </li>
          <li class="contextual-links-region">  
          <div class="row">
    <div class="col-md-4 col-texto-noticia">
        <div class="title-block-noticia noticia-right"><h2 class="title-block">Noticias</h2><a href="/universidad/noticias" title="Todas las noticias" class="todas-noticias">Todas las noticias</a></div>
        <div class="line-block-noticia"><div class="line-block-noticia-inner"></div></div>
        <div class="contextual-links-region">
        <div class="titulo-noticia"><a href="/universidad/noticias/continua-la-preparacion-en-x-campamento-caribeno-acm-icpc">Continúa la preparación en X Campamento Caribeño ACM-ICPC</a></div>
        <div class="fecha-noticia">
            <span class="fecha-noticia-dias"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-15T16:15:00-04:00">15/3/2018</span> | </span>
            <span class="fecha-noticia-hora"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-15T16:15:00-04:00">4:15 pm</span></span>
        </div>
        <div class="texto-noticia noticia-right"><p>Los 17 equipos participantes del X Campamento Caribeño de Entrenamiento ACM-ICPC amplian su preparación durante cada jornada</p>
</div>
        <div class="noticias_more_link noticia-right"><a href="/universidad/noticias/continua-la-preparacion-en-x-campamento-caribeno-acm-icpc">leer más</a></div>
    </div>
</div>
<div class="col-md-8 col-imagen-noticia"><div class="col-imagen-noticia-content"><img alt="Continúa la preparación en X Campamento Caribeño ACM-ICPC" title="Continúa la preparación en X Campamento Caribeño ACM-ICPC" data-image="http://www.uci.cu/sites/default/files/styles/slide_noticia_1001x450/public/imagenes/noticias/continua-la-preparacion-en-x-campamento-caribeno-acm-icpc-1.jpg?itok=FgYvubx3" src=""></div></div>
</div>  </li>
      </ul></div>    </div>
  
  
  
  
  
  
</div>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
		</div> 
    </div></div> 

    </section>

    <section id="eventos" class='seccion-inicio'>       
        <div class="container">
            <div class="row">       
                <div class="region region-eventos">
  <div id="block-views-eventos-v1-block-1" class="block block-views" data-container-block="">
		  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Eventos</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-eventos-v1 view-id-eventos_v1 view-display-id-block_1 view-dom-id-5fc1daa3cb4b57d57b3759b182bbb4e9">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first col-md-6 col-sm-6 col-xs-12 contextual-links-region col-md-4 col-sm-4 col-xs-12">
      
  <div class="views-field views-field-field-fecha-inicio-1">        <div class="field-content"><span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-02T00:00:00-05:00">02</span> </span>
<span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-02T00:00:00-05:00">Mar</span></span></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="views-field views-field-title"><a href="/investigacion-y-desarrollo/eventos/matricula-en-el-programa-de-doctorado-en-informatica">Matrícula en el programa de Doctorado en Informática</a></div>
<div class="views-field views-field-body"><p>Convocatoria para profesionales del país interesados en desarrollar estudios de doctorado, en el Programa de Doctorado en Informática.</p>
</div>
<div class="views-field-view-node"><a href="/investigacion-y-desarrollo/eventos/matricula-en-el-programa-de-doctorado-en-informatica">leer más</a></div></div>  </div>  </div>
  <div class="views-row views-row-2 views-row-even col-md-6 col-sm-6 col-xs-12 contextual-links-region col-md-4 col-sm-4 col-xs-12">
      
  <div class="views-field views-field-field-fecha-inicio-1">        <div class="field-content"><span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-19T00:00:00-04:00">19</span> </span>
<span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-03-19T00:00:00-04:00">Mar</span></span></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="views-field views-field-title"><a href="/investigacion-y-desarrollo/eventos/informatica-2018">Informática 2018</a></div>
<div class="views-field views-field-body"><p>El Ministerio de Comunicaciones y la Unión de Informáticos de Cuba le invitan a la discusión científico tecnológica y a la exposición de proyectos e iniciativas de las principales temáticas convocadas en cada uno de los eventos que forman parte de la Convención y de la Feria</p>
</div>
<div class="views-field-view-node"><a href="/investigacion-y-desarrollo/eventos/informatica-2018">leer más</a></div></div>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd views-row-last col-md-6 col-sm-6 col-xs-12 contextual-links-region col-md-4 col-sm-4 col-xs-12">
      
  <div class="views-field views-field-field-fecha-inicio-1">        <div class="field-content"><span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-09-24T00:00:00-04:00">24</span> </span>
<span><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-09-24T00:00:00-04:00">Sep</span></span></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="views-field views-field-title"><a href="/investigacion-y-desarrollo/eventos/viii-congreso-iberoamericano-de-ingenieria-de-proyectos">VIII Congreso Iberoamericano de Ingeniería de Proyectos</a></div>
<div class="views-field views-field-body"><p>Este congreso se va a realizar en el marco de la Tercera Conferencia Científica Internacional de la Universidad de las Ciencias Informáticas (UCIENCIA 2018)</p>
</div>
<div class="views-field-view-node"><a href="/investigacion-y-desarrollo/eventos/viii-congreso-iberoamericano-de-ingenieria-de-proyectos">leer más</a></div></div>  </div>  </div>
    </div>
  
  
  
      
<div class="more-link">
  <a href="/investigacion-y-desarrollo/eventos">
    Todos los eventos  </a>
</div>
  
  
  
</div>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
            </div>
        </div>
    </section>

    <section id="estudiar_en_la_uci" class="seccion-inicio">
        <div class="region region-estudiar-en-la-uci">
  <div id="block-estudios-estudios-v1" class="block block-estudios" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="estudiar_uci_bloque col-md-6 col-xs-12 col-sm-12"><div class="contextual-links-region estudiar_uci_item estudiar_uci_item_odd"><div class="estudiar_uci_image"><a href="/estudios/aspirantes"><img src="http://www.uci.cu/sites/default/files/styles/estudiar_en_la_uci/public/imagenes_estudios/applicants.jpg?itok=NTzVopcE"></a></div><div class="estudiar_uci_text"><div class="estudiar_uci_mark"></div><div class="estudiar_uci_title"><a href="/estudios/aspirantes">Aspirantes</a></div><div class="estudiar_uci_descripcion">Nuestros programas académicos de alta calidad en diferentes modalidades, están disponibles para aspirantes nacionales y extranjeros.</div><div class="estudiar_uci_more_link"><a href="/estudios/aspirantes">leer más</a></div></div></div><div class="contextual-links-region estudiar_uci_item estudiar_uci_item_even"><div class="estudiar_uci_image"><a href="/estudios/posgrado"><img src="http://www.uci.cu/sites/default/files/styles/estudiar_en_la_uci/public/imagenes_estudios/post-graduate-studies.jpg?itok=2Fm1WdET"></a></div><div class="estudiar_uci_text"><div class="estudiar_uci_mark"></div><div class="estudiar_uci_title"><a href="/estudios/posgrado">Estudios de posgrado</a></div><div class="estudiar_uci_descripcion">Ofrecemos estudios de posgrado a través de cursos, diplomados, maestrías, doctorados y eventos científicos.</div><div class="estudiar_uci_more_link"><a href="/estudios/posgrado">leer más</a></div></div></div></div><div class="estudiar_uci_bloque col-md-6 col-xs-12 col-sm-12"><div class="contextual-links-region estudiar_uci_item estudiar_uci_item_odd"><div class="estudiar_uci_image"><a href="/estudios/pregrado"><img src="http://www.uci.cu/sites/default/files/styles/estudiar_en_la_uci/public/imagenes_estudios/estudios-de-pregrado_0.jpg?itok=IQN4q2x7"></a></div><div class="estudiar_uci_text"><div class="estudiar_uci_mark"></div><div class="estudiar_uci_title"><a href="/estudios/pregrado">Estudios de pregrado</a></div><div class="estudiar_uci_descripcion">Nos especializamos en la carrera de Ingeniería en Ciencias Informáticas. También impartimos cursos cortos vinculados a las ciencias de la computación.</div><div class="estudiar_uci_more_link"><a href="/estudios/pregrado">leer más</a></div></div></div><div class="contextual-links-region estudiar_uci_item estudiar_uci_item_even"><div class="estudiar_uci_image"><a href="/estudios/educacion-a-distancia"><img src="http://www.uci.cu/sites/default/files/styles/estudiar_en_la_uci/public/imagenes_estudios/distance-education.jpg?itok=LgURSzVN"></a></div><div class="estudiar_uci_text"><div class="estudiar_uci_mark"></div><div class="estudiar_uci_title"><a href="/estudios/educacion-a-distancia">Educación a distancia</a></div><div class="estudiar_uci_descripcion">Implementamos métodos, técnicas y recursos que hacen más efectivo y flexible el proceso de enseñanza-aprendizaje a distancia.</div><div class="estudiar_uci_more_link"><a href="/estudios/educacion-a-distancia">leer más</a></div></div></div></div>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
    </section>

    <section id="productos" class="seccion-inicio">       
        <div class="region region-productos">
  <div id="block-block-11" class="block block-block" data-container-block="">
	        <div class="container">
            <div class="row">
                    	  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Productos</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <p>Desarrollamos soluciones informáticas orientadas a diversos sectores de la economía y los servicios dentro y fuera de Cuba, respaldadas por la certificación internacional <a href="https://sas.cmmiinstitute.com/pars/pars_detail.aspx?a=25323" rel="nofollow" target="_blank" title="Nivel II de CMMI">Nivel II de CMMI</a>. Somos líderes en la migración del país a tecnologías de Software Libre y Código Abierto, a partir del desarrollo de Nova, la distribución cubana de GNU/Linux.</p>
  </div> 
                                </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-block-46" class="block block-block" data-container-block="">
	        <div class="container">
            <div class="row">
                				    <div class="col-md-12 col-sm-12 col-xs-12">
				    	  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <p><a href="http://www.nova.cu" rel="nofollow" class="link-nova-image"  style="background: #fff; padding: 20px; display: block; text-align: center;" target="_blank" title="Nova, distribución cubana de GNU/Linux."><img alt="Nova, distribución cubana de GNU/Linux." class="nova-image" src="/sites/default/files/imagenes/bloques/nova.png" style="max-width: 700px;margin: 0 auto;" title="Nova, distribución cubana de GNU/Linux." /></a></p>
<style type="text/css">@media (max-width: 991px){

    .nova-image {
       max-width: 100% !important;
    }

}

@media (max-width: 633px ){
.estudiar_uci_bloque .estudiar_uci_item_even .estudiar_uci_text .estudiar_uci_mark {
    display: none;
    right: 0;
}
}
#block-views-productos-de-nova-v1-block-2,
#block-views-productos-de-nova-v1-block-3{
    padding-bottom: 50px;
}
.view-productos-de-nova-v1.view-display-id-block_2 .views-row .producto-row-disc,
.view-productos-de-nova-v1.view-display-id-block_3 .views-row .producto-row-disc{
	background: url(../images/sprite.png) no-repeat -5px -1220px;
}

.view-productos-de-nova-v1.view-display-id-block_2 .views-row .producto-row-content,
.view-productos-de-nova-v1.view-display-id-block_3 .views-row .producto-row-content{
    border-bottom: 3px solid #68b445;
    margin-bottom: 20px;
}

.view-productos-de-nova-v1.view-display-id-block_2 .views-row:hover .producto-row-content,
.view-productos-de-nova-v1.view-display-id-block_3 .views-row:hover .producto-row-content{
    border-bottom: 3px solid #68b445;
}

.view-productos-de-nova-v1.view-display-id-block_2 .views-row .producto-row-disc,
.view-productos-de-nova-v1.view-display-id-block_3 .views-row .producto-row-disc{
    width: 80px;
    height: 40px;
    top: -25px;
}

.view-productos-de-nova-v1.view-display-id-block_2 .producto-row-content:hover .producto-row-content-title a,
.view-productos-de-nova-v1.view-display-id-block_3 .producto-row-content:hover .producto-row-content-title a{
    color: #68b445
}
</style>
  </div> 
                    </div>            </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-views-productos-de-nova-v1-block" class="block block-views" data-container-block="">
	        <div class="container">
            <div class="row">
                    	  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-productos-de-nova-v1 view-id-productos_de_nova_v1 view-display-id-block view-dom-id-93c87e83e4819cf1e25864b40159dcfd">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first col-md-3 col-sm-3 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="/investigacion-y-desarrollo/productos/nova/nova-escritorio">Nova ESCRITORIO</a></div>
<div class="producto-row-content-descripcion">Proponemos un ambiente de usuario limpio y libre de distracciones, para aumentar la productividad del usuario</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even col-md-3 col-sm-3 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="/investigacion-y-desarrollo/productos/nova/nova-servidores">Nova SERVIDORES</a></div>
<div class="producto-row-content-descripcion">Dirigida a ordenadores que tienen como función el control de redes, centros de datos, entornos empresariales y de desarrollo</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd col-md-3 col-sm-3 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="/investigacion-y-desarrollo/productos/nova/nova-ligero">Nova Ligero</a></div>
<div class="producto-row-content-descripcion">Distribución optimizada para obtener un mejor rendimiento en equipos con bajas prestaciones re-utilizando el entorno LXDE</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-4 views-row-even views-row-last col-md-3 col-sm-3 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="/investigacion-y-desarrollo/productos/nova/nova-droid">Nova Droid</a></div>
<div class="producto-row-content-descripcion">Compatible con más de 1 millón de aplicaciones disponibles en las diferentes tiendas de aplicaciones de Android</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
    </div>
  
  
  
  
  
  
</div>  </div> 
                                </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-block-12" class="block block-block" data-container-block="">
	        <div class="container">
            <div class="row">
                				    <div class="col-md-12 col-sm-12 col-xs-12">
				    	  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div>
<div>
<p>Las soluciones, implementadas en su mayoría sobre plataformas de software libre o código abierto, se comercializan bajo marcas registradas, de acuerdo a cinco líneas de alto impacto: <span style="font-family:Lato Bold">Salud, Administración Pública, Educación, Empresa-Industria y Telemática</span>.</p>
</div>
</div>
  </div> 
                    </div>            </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-views-productos-block" class="block block-views" data-container-block="">
	        <div class="container">
            <div class="row">
                    	  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-productos view-id-productos view-display-id-block view-dom-id-da6e3d2b647baf3e46aa64f2277e7047">
            <div class="view-header">
      <div id="options" class="filter-menu">
<ul id="filters" class="option-set" data-option-key="filter">
<li  class="xavia-tab"><a class="selected" href="#filter" data-option-value="producto-xavia">xavia</a></li><li  class="xabal-tab"><a class="" href="#filter" data-option-value="producto-xabal">xabal</a></li><li  class="xauce-tab"><a class="" href="#filter" data-option-value="producto-xauce">xauce</a></li><li  class="xedro-tab"><a class="" href="#filter" data-option-value="producto-xedro">xedro</a></li><li  class="xilema-tab"><a class="" href="#filter" data-option-value="producto-xilema">xilema</a></li></ul>
</div> 
    </div>
  
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/platsi-10"><span class="title-marca">xilema</span> PLATSI 1.0</a></div>
<div class="producto-row-content-descripcion">Plataforma de seguridad en las tecnologías de la información</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/auditbd-10"><span class="title-marca">xilema</span> AUDITBD 1.0</a></div>
<div class="producto-row-content-descripcion">Herramienta de auditoría a sistemas gestores de bases de datos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/nova-unificado"><span class="title-marca">xilema</span> NOVA UNIFICADO</a></div>
<div class="producto-row-content-descripcion">Administración y gestión de los servicios telemáticos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-4 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/novadroid"><span class="title-marca">xilema</span> NOVADROID</a></div>
<div class="producto-row-content-descripcion">Sistema operativo Android</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-5 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/clonix"><span class="title-marca">xilema</span> CLONIX</a></div>
<div class="producto-row-content-descripcion">Clonación de sistemas operativos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-6 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/nova-nas"><span class="title-marca">xilema</span> NOVA NAS</a></div>
<div class="producto-row-content-descripcion">Gestión de almacenamiento de datos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-7 views-row-odd producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/gdr"><span class="title-marca">xedro</span> GDR</a></div>
<div class="producto-row-content-descripcion">Generador Dinámico de Reportes</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-8 views-row-even producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/anper"><span class="title-marca">xedro</span> ANPER</a></div>
<div class="producto-row-content-descripcion">Sistema de análisis petrofísico</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-9 views-row-odd producto-xabal col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xabal/audat-10"><span class="title-marca">xabal</span> AUDAT 1.0</a></div>
<div class="producto-row-content-descripcion">Sistema de Auditoría de Datos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-10 views-row-even producto-xabal col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xabal/arkheia-30"><span class="title-marca">xabal</span> ARKHEIA 3.0</a></div>
<div class="producto-row-content-descripcion">Sistema de gestión de archivos históricos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-11 views-row-odd producto-xabal col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xabal/excriba-31"><span class="title-marca">xabal</span> EXCRIBA 3.1</a></div>
<div class="producto-row-content-descripcion">Sistema gestor de documentos administrativos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-12 views-row-even producto-xabal col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xabal/repxos-30"><span class="title-marca">xabal</span> REPXOS 3.0</a></div>
<div class="producto-row-content-descripcion">Repositorio digital</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-13 views-row-odd producto-xabal col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xabal/idbioaccess"><span class="title-marca">xabal</span> IDBIOACCESS</a></div>
<div class="producto-row-content-descripcion">Sistema de control de acceso e identificación</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-14 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/grhs-10"><span class="title-marca">xilema</span> GRHS 1.0</a></div>
<div class="producto-row-content-descripcion">Gestor de Recursos de Hardware y Software</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-15 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/primicia-20"><span class="title-marca">xilema</span> PRIMICIA 2.0</a></div>
<div class="producto-row-content-descripcion">Sistema de televisión corporativa</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-16 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/reko-40"><span class="title-marca">xilema</span> REKO 4.0</a></div>
<div class="producto-row-content-descripcion">Replicador de datos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-17 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/smart-keeper"><span class="title-marca">xilema</span> SMART KEEPER</a></div>
<div class="producto-row-content-descripcion">Pro Suite para el control de acceso a contenidos de Internet</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-18 views-row-even producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/suria-20"><span class="title-marca">xilema</span> SURIA 2.0</a></div>
<div class="producto-row-content-descripcion">Video Vigilancia</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-19 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/orion"><span class="title-marca">xilema</span> ORION</a></div>
<div class="producto-row-content-descripcion">Motor de Búsqueda Personalizado</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-20 views-row-even producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/ris-30"><span class="title-marca">xavia</span>  RIS 3.0</a></div>
<div class="producto-row-content-descripcion">Sistema de Información Radiológica</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-21 views-row-odd producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/pacs-30"><span class="title-marca">xavia</span> PACS 3.0</a></div>
<div class="producto-row-content-descripcion">Sistema para el Almacenamiento, Transmisión y Visualización de Imágenes Médicas</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-22 views-row-even producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/meteorix-20"><span class="title-marca">xavia</span> METEORIX 2.0</a></div>
<div class="producto-row-content-descripcion">Videojuego de rehabilitación visual</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-23 views-row-odd producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/danzo-terapia-10"><span class="title-marca">xavia</span> DANZO TERAPIA 1.0</a></div>
<div class="producto-row-content-descripcion">Entrenador de Funciones Motoras</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-24 views-row-even producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/motorehab-10"><span class="title-marca">xavia</span> MOTOREHAB 1.0</a></div>
<div class="producto-row-content-descripcion">Entrenador de Funciones Motoras</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-25 views-row-odd producto-xavia col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xavia/his-210"><span class="title-marca">xavia</span> HIS 2.1.0</a></div>
<div class="producto-row-content-descripcion">Sistema de Información Hospitalaria</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-26 views-row-even producto-xauce col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xauce/croda-20"><span class="title-marca">xauce</span> CRODA 2.0</a></div>
<div class="producto-row-content-descripcion">Herramienta de Autor</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-27 views-row-odd producto-xauce col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xauce/lvpc-20"><span class="title-marca">xauce</span> LVPC 2.0</a></div>
<div class="producto-row-content-descripcion">Ensamblaje de un Computador</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-28 views-row-even producto-xauce col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xauce/rhoda-22"><span class="title-marca">xauce</span> RHODA 2.2</a></div>
<div class="producto-row-content-descripcion">Repositorio de Objetos de Aprendizaje</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-29 views-row-odd producto-xauce col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xauce/zera-10"><span class="title-marca">xauce</span> ZERA 1.0</a></div>
<div class="producto-row-content-descripcion">Plataforma de Gestión Educativa</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-30 views-row-even producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/arex"><span class="title-marca">xedro</span> AREX</a></div>
<div class="producto-row-content-descripcion">Sistema de medición</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-31 views-row-odd producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/sige-30"><span class="title-marca">xedro</span> SIGE 3.0</a></div>
<div class="producto-row-content-descripcion">Sistema Integrado de Gestión Estadística</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-32 views-row-even producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/sipac-21"><span class="title-marca">xedro</span> SIPAC 2.1</a></div>
<div class="producto-row-content-descripcion">Sistema de planificación de actividades</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-33 views-row-odd producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/sainux"><span class="title-marca">xedro</span> SAINUX</a></div>
<div class="producto-row-content-descripcion">Sistema de Automatización Industrial</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-34 views-row-even producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/maximus-drill-12"><span class="title-marca">xedro</span> MAXIMUS DRILL 1.2</a></div>
<div class="producto-row-content-descripcion">Sistema integral de perforación de pozos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-35 views-row-odd producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/orbita-10"><span class="title-marca">xedro</span> ÓRBITA 1.0</a></div>
<div class="producto-row-content-descripcion">Sistema de control de flota y mantenimiento</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-36 views-row-even producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/gespro-1305"><span class="title-marca">xedro</span> GESPRO 13.05</a></div>
<div class="producto-row-content-descripcion">Suite de Gestión de Proyectos</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-37 views-row-odd producto-xilema col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xilema/genesig-20"><span class="title-marca">xilema</span> GENESIG 2.0</a></div>
<div class="producto-row-content-descripcion">Plataforma soberana de sistemas de información geográfica</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-38 views-row-even views-row-last producto-xedro col-md-3 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing">        <div class="field-content"><div class="producto-row-disc"></div>
<div class="producto-row-content">
<div class="producto-row-content-title"><a href="http://www.uci.cu/investigacion-y-desarrollo/productos/xedro/cedrux"><span class="title-marca">xedro</span> CEDRUX</a></div>
<div class="producto-row-content-descripcion">Sistema de gestión empresarial</div>
</div></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
    </div>
  
  
  
  
      <div class="view-footer">
      <div class="more-link"></div>    </div>
  
  
</div>  </div> 
                                </div>
        </div>
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
    </section>

    <section id="servicios" class="seccion-inicio">          
        <div class="region region-servicios">
  <div id="block-block-1" class="block block-block" data-container-block="">
	        <div class="container">
            <div class="row">
                				    <div class="col-md-12 col-sm-12 col-xs-12">
				    	  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Servicios</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <p>Nuestro catálogo incluye programas académicos de pregrado y postgrado, novedosas prestaciones editoriales a través del sello “Ediciones Futuro Cuba”, así como otros servicios de alta calidad y profesionalidad, disponibles para personas naturales y jurídicas tanto nacionales como extranjeras.</p>
  </div> 
                    </div>            </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-views-tipos-de-servicios-v1-block" class="block block-views" data-container-block="">
	        <div class="container">
            <div class="row">
                    	  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-tipos-de-servicios-v1 view-id-tipos_de_servicios_v1 view-display-id-block view-dom-id-79e6bbfbb6c4af2d496c01dc11b6d1dc">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first col-md-4 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing-1">        <div class="field-content"><div class='servicios-informatico icono-tipo-servicio'></div></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-28">Servicios Informáticos</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><div>
<div>
<p>Orientados a diversos sectores dentro y fuera de Cuba, nuestros servicios informáticos son avalados por la <span style="font-family:Lato Bold">certificación internacional <a href="https://sas.cmmiinstitute.com/pars/pars_detail.aspx?a=25323" rel="nofollow" target="_blank" title="Nivel II de CMMI">Nivel II de CMMI</a></span>, y la experiencia en el trabajo con clientes nacionales y extranjeros.</p>
<p>Las propuestas comerciales abarcan asesoría, consultoría, servicios, productos y soporte, distribuidos en cinco líneas de alto impacto: <span style="font-family:Lato Bold">Salud, Administración Pública, Educación, Empresa-Industria y Telemática</span>.</p>
</div>
</div>
</div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a class="more-link" title='<a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-28">Servicios Informáticos</a>' href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-28">leer más</a></span>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even col-md-4 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing-1">        <div class="field-content"><div class='servicios-academico icono-tipo-servicio'></div></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-27">Servicios Académicos</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p><!--StartFragment--></p><p>Ofrecemos servicios académicos que incluyen programas de alta calidad en diferentes modalidades de estudio y diversos temas como la informática, las ciencias pedagógicas, y la matemática vinculada a la computación.</p>
<p>También impartimos doctorados, maestrías, diplomados, cursos cortos, entrenamientos, así como pasantías y estancias que propician el desarrollo de proyectos investigativos.</p>
</div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a class="more-link" title='<a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-27">Servicios Académicos</a>' href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-27">leer más</a></span>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-3 views-row-odd views-row-last col-md-4 col-sm-4 col-xs-12 contextual-links-region">
      
  <div class="views-field views-field-nothing-1">        <div class="field-content"><div class='servicios-editoriales icono-tipo-servicio'></div></div>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-29">Servicios Editoriales</a></span>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><div>
<div>
<p>A través de nuestro sello editorial <span style="font-family:Lato Bold">“Ediciones Futuro Cuba”</span>, brindamos la posibilidad de aprovechar el escenario que ofrecen las TIC para esta industria, a partir de la difusión de contenido en soportes digitales.</p>
<p>Editamos la <a href="http://rcci.uci.cu/?journal=rcci" rel="nofollow" target="_blank" title="Revista Cubana de Ciencias Informáticas">Revista Cubana de Ciencias Informáticas</a> y la <a href="http://publicaciones.uci.cu/" rel="nofollow" target="_blank" title="Serie Científica de la UCI">Serie Científica de la UCI</a>, con el objetivo de promover el trabajo de profesionales cubanos y extranjeros en las ramas de las Ciencias Informáticas y de la Computación.</p>
</div>
</div>
</div>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><a class="more-link" title='<a href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-29">Servicios Editoriales</a>' href="http://www.uci.cu/investigacion-y-desarrollo/servicios#block-block-29">leer más</a></span>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
    </div>
  
  
  
      
<div class="more-link">
  <a href="/investigacion-y-desarrollo/servicios">
    Todos los servicios  </a>
</div>
  
  
  
</div>  </div> 
                                </div>
        </div>
      
</div> <!-- /.block -->
<div id="block-facebook-blocks-twitter-line-block" class="block block-facebook-blocks" data-container-block="">
	        <div class="container">
            <div class="row">
                				    <div class="col-md-12 col-sm-12 col-xs-12">
				    	  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Tu tweet por la uci</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <a class="twitter-timeline" data-tweet-limit="3" data-chrome="noheader nofooter noborders transparent" data-theme="dark" href="https://twitter.com/universidad_uci">Tweets de la UCI</a><script async src="//platform.twitter.com/widgets.js" charset="utf-8"></script>  </div> 
                    </div>            </div>
        </div>
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
                  
    </section>

<footer id="footer">
    <div class="container footer-cols">
        <div class="row">
            <div class="col-md-4 col-xs-12"><div class="region region-footer-first">
  <div id="block-views-certificados-v1-block" class="block block-views" data-container-block="">
		  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Certificados</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <div class="view view-certificados-v1 view-id-certificados_v1 view-display-id-block view-dom-id-072dd6b957eb1fc10efc03866b0e4501">
        
  
  
      <div class="view-content">
        <div class="views-row views-row-1 views-row-odd views-row-first contextual-links-region">
      
  <div class="views-field views-field-field-imagen">        <div class="field-content"><a href="https://sas.cmmiinstitute.com/pars/pars_detail.aspx?a=25323" title="CMMI" rel="nofollow" target="_blank"><img typeof="foaf:Image" src="http://www.uci.cu/sites/default/files/imagenes_certificados/cmmi-1.png" width="220" height="37" alt="Certificación de la Universidad de las Ciencias Informáticas UCI en el nivel 2 de CMMI" title="Certificación de la Universidad de las Ciencias Informáticas UCI en el nivel 2 de CMMI" /></a></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
  <div class="views-row views-row-2 views-row-even views-row-last contextual-links-region">
      
  <div class="views-field views-field-field-imagen">        <div class="field-content"><a href="http://www.mes.gob.cu/es/indicadores-de-calidad" title="JAN" rel="nofollow" target="_blank"><img typeof="foaf:Image" src="http://www.uci.cu/sites/default/files/imagenes_certificados/jan-1.png" width="220" height="78" alt="JAN" title="JAN" /></a></div>  </div>  
  <div class="views-field views-field-contextual-links">        <span class="field-content"></span>  </div>  </div>
    </div>
  
  
  
  
  
  
</div>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>
            <div class="col-md-4 col-xs-12"><div class="region region-footer-second">
  <div id="block-menu-menu-enlaces-de-interes" class="block block-menu" data-container-block="">
		  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Enlaces de interés</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="http://rcci.uci.cu/?journal=rcci" title="RCCI" rel="nofollow" target="_blank">RCCI</a></li>
<li class="leaf"><a href="http://firefoxmania.uci.cu/" title="Firefoxmanía" rel="nofollow" target="_blank">Firefoxmanía</a></li>
<li class="leaf"><a href="http://reponova.uci.cu" title="NOVA GNU/Linux" rel="nofollow" target="_blank">NOVA GNU/Linux</a></li>
<li class="leaf"><a href="http://humanos.uci.cu/" title="humanOS" rel="nofollow" target="_blank">humanOS</a></li>
<li class="leaf"><a href="http://coj.uci.cu" title="Caribbean Online Judge" rel="nofollow" target="_blank">Caribbean Online Judge</a></li>
<li class="leaf"><a href="http://cmf.uci.cu/" title="Caribbean Mind Forge" rel="nofollow" target="_blank">Caribbean Mind Forge</a></li>
<li class="last leaf"><a href="https://rvirtual.uci.cu/" title="Comunidad Cubana de Videojuegos" rel="nofollow" target="_blank">Comunidad Cubana de Videojuegos</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>
            <div class="col-md-4 col-xs-12"><div class="region region-footer-three">
  <div id="block-simplenews-291" class="block block-simplenews" data-container-block="">
		  
               <div class="title-block-cuadrado"></div>                           
             <div class="title-block-wrapper">                             
    <h2 class="title-block" >Boletín</h2>
    </div> 
          		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    
      <p>¡Manténgase informado de nuestras últimas novedades!</p>
  
      <p><a href="/newsletter/subscriptions">Configurar mis suscripciones</a></p>
  
  
  
    </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>      
        </div> 
    </div>
    <div class="container footer-message">
        <div class="row">
            <div class="col-md-8 col-xs-12"><div class="region region-footer">
  <div id="block-block-3" class="block block-block" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <p>Universidad de las Ciencias Informáticas La Habana, Cuba. Todos los derechos reservados.</p>
  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>
            <div class="col-md-4 col-xs-12 enlaces-pie-message"><div class="region region-footer-links">
  <div id="block-menu-menu-menu-pie-de-p-gina" class="block block-menu" data-container-block="">
		  
    		
    <div class="title-block-arrow-down"></div>
  <div class="content">
    <ul class="menu"><li class="first leaf"><a href="/preguntas-frecuentes">Preguntas Frecuentes</a></li>
<li class="last leaf"><a href="/terminos-y-condiciones-de-uso">Términos y condiciones de uso</a></li>
</ul>  </div> 
      
</div> <!-- /.block -->
</div>
 <!-- /.region -->
</div>
        </div> 
    </div>    
</footer>
    
<a class="back-to-top" title="Ir a arriba" href="#"></a>
<i class="back-to-top-icon glyphicon glyphicon-arrow-up"></i>                <link type="text/css" rel="stylesheet" href="http://www.uci.cu/sites/default/files/css/css_xE-rWrJf-fncB6ztZfd2huxqgxu4WO-qwma6Xer30m4.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uci.cu/sites/default/files/css/css_pQC6VUrzLtR34AOr1y3yKKFdvIRL_FgDQLeroHZqvo0.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uci.cu/sites/default/files/css/css_VOb1ixJXuEfPMZf2HS6-j3PbSvw2iAUSpnBUv03OuZI.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uci.cu/sites/default/files/css/css_SbjLX2-TngNIbpng70en7yqi7bcoQnSTgDcMhC4PZqg.css" media="all" />
	<!-- HTML5 shim and Respond.js for IE8 support of HTML5 elements and media queries -->
            <!-- WARNING: Respond.js doesn't work if you view the page via file:// -->
            <!--[if lt IE 9]>
            <script src="https://oss.maxcdn.com/html5shiv/3.7.3/html5shiv.min.js"></script>
            <script src="https://oss.maxcdn.com/respond/1.4.2/respond.min.js"></script>
	<![endif]-->	
	<script type="text/javascript" src="http://www.uci.cu/sites/all/modules/jquery_update/replace/jquery/1.11/jquery.min.js?v=1.11.2"></script>
<script type="text/javascript" src="http://www.uci.cu/misc/jquery.once.js?v=1.2"></script>
<script type="text/javascript" src="http://www.uci.cu/misc/drupal.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/default/files/languages/es_lyE_ksS6CjFlZpW12NgCPe3lOOVmjZFW-Dn4YGiGEE0.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/colorbox/jquery.colorbox-min.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/modules/colorbox/js/colorbox.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/modules/colorbox/styles/stockholmsyndrome/colorbox_style.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/modules/lightbox2/js/lightbox.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/superfish/jquery.hoverIntent.minified.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/superfish/sfsmallscreen.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/superfish/supposition.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/superfish/superfish.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/libraries/superfish/supersubs.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/modules/superfish/superfish.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/jquery.nicescroll.min.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/bootstrap.min.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/jquery.flexslider.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/uci.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/animaciones.js?p5v3rg"></script>
<script type="text/javascript" src="http://www.uci.cu/sites/all/themes/uci/js/wow.min.js?p5v3rg"></script>
<script type="text/javascript">
<!--//--><![CDATA[//><!--
jQuery.extend(Drupal.settings, {"basePath":"\/","pathPrefix":"","ajaxPageState":{"theme":"uci","theme_token":"OVZbuK_5_4chJLeWitmVs4GWdhMVLywNwc4lkYhH1n8","js":{"sites\/all\/modules\/jquery_update\/replace\/jquery\/1.11\/jquery.min.js":1,"misc\/jquery.once.js":1,"misc\/drupal.js":1,"public:\/\/languages\/es_lyE_ksS6CjFlZpW12NgCPe3lOOVmjZFW-Dn4YGiGEE0.js":1,"sites\/all\/libraries\/colorbox\/jquery.colorbox-min.js":1,"sites\/all\/modules\/colorbox\/js\/colorbox.js":1,"sites\/all\/modules\/colorbox\/styles\/stockholmsyndrome\/colorbox_style.js":1,"sites\/all\/modules\/lightbox2\/js\/lightbox.js":1,"sites\/all\/libraries\/superfish\/jquery.hoverIntent.minified.js":1,"sites\/all\/libraries\/superfish\/sfsmallscreen.js":1,"sites\/all\/libraries\/superfish\/supposition.js":1,"sites\/all\/libraries\/superfish\/superfish.js":1,"sites\/all\/libraries\/superfish\/supersubs.js":1,"sites\/all\/modules\/superfish\/superfish.js":1,"sites\/all\/themes\/uci\/js\/jquery.nicescroll.min.js":1,"sites\/all\/themes\/uci\/js\/bootstrap.min.js":1,"sites\/all\/themes\/uci\/js\/jquery.flexslider.js":1,"sites\/all\/themes\/uci\/js\/uci.js":1,"sites\/all\/themes\/uci\/js\/animaciones.js":1,"sites\/all\/themes\/uci\/js\/wow.min.js":1},"css":{"modules\/system\/system.base.css":1,"modules\/system\/system.menus.css":1,"modules\/system\/system.messages.css":1,"modules\/system\/system.theme.css":1,"sites\/all\/modules\/simplenews\/simplenews.css":1,"sites\/all\/modules\/ldap\/ldap_user\/ldap_user.css":1,"modules\/comment\/comment.css":1,"sites\/all\/modules\/date\/date_api\/date.css":1,"sites\/all\/modules\/date\/date_popup\/themes\/datepicker.1.7.css":1,"sites\/all\/modules\/date\/date_repeat_field\/date_repeat_field.css":1,"modules\/field\/theme\/field.css":1,"modules\/node\/node.css":1,"modules\/search\/search.css":1,"modules\/user\/user.css":1,"sites\/all\/modules\/youtube\/css\/youtube.css":1,"sites\/all\/modules\/views\/css\/views.css":1,"sites\/all\/modules\/ckeditor\/css\/ckeditor.css":1,"sites\/all\/modules\/colorbox\/styles\/stockholmsyndrome\/colorbox_style.css":1,"sites\/all\/modules\/ctools\/css\/ctools.css":1,"sites\/all\/modules\/ldap\/ldap_servers\/ldap_servers.admin.css":1,"sites\/all\/modules\/lightbox2\/css\/lightbox.css":1,"sites\/all\/modules\/validaciones_contenidos\/configuraciones_admin.css":1,"modules\/locale\/locale.css":1,"sites\/all\/libraries\/superfish\/css\/superfish.css":1,"sites\/all\/libraries\/superfish\/css\/superfish-smallscreen.css":1,"sites\/all\/themes\/uci\/css\/animate.css":1,"sites\/all\/themes\/uci\/css\/bootstrap.min.css":1,"sites\/all\/themes\/uci\/css\/style.css":1}},"colorbox":{"opacity":"0.85","current":"{current} de {total}","previous":"\u00ab Prev","next":"Next \u00bb","close":"Close","maxWidth":"98%","maxHeight":"98%","fixed":true,"mobiledetect":true,"mobiledevicewidth":"480px"},"lightbox2":{"rtl":"0","file_path":"\/(\\w\\w\/)public:\/","default_image":"\/sites\/all\/modules\/lightbox2\/images\/brokenimage.jpg","border_size":10,"font_color":"000","box_color":"fff","top_position":"","overlay_opacity":"0.8","overlay_color":"000","disable_close_click":true,"resize_sequence":0,"resize_speed":400,"fade_in_speed":400,"slide_down_speed":600,"use_alt_layout":false,"disable_resize":false,"disable_zoom":false,"force_show_nav":false,"show_caption":true,"loop_items":false,"node_link_text":"View Image Details","node_link_target":false,"image_count":"Image !current of !total","video_count":"Video !current of !total","page_count":"Page !current of !total","lite_press_x_close":"press \u003Ca href=\u0022#\u0022 onclick=\u0022hideLightbox(); return FALSE;\u0022\u003E\u003Ckbd\u003Ex\u003C\/kbd\u003E\u003C\/a\u003E to close","download_link_text":"","enable_login":false,"enable_contact":false,"keys_close":"c x 27","keys_previous":"p 37","keys_next":"n 39","keys_zoom":"z","keys_play_pause":"32","display_image_size":"original","image_node_sizes":"()","trigger_lightbox_classes":"","trigger_lightbox_group_classes":"","trigger_slideshow_classes":"","trigger_lightframe_classes":"","trigger_lightframe_group_classes":"","custom_class_handler":0,"custom_trigger_classes":"","disable_for_gallery_lists":true,"disable_for_acidfree_gallery_lists":true,"enable_acidfree_videos":true,"slideshow_interval":5000,"slideshow_automatic_start":true,"slideshow_automatic_exit":true,"show_play_pause":true,"pause_on_next_click":false,"pause_on_previous_click":true,"loop_slides":false,"iframe_width":600,"iframe_height":400,"iframe_border":1,"enable_video":false,"useragent":"Mozilla\/5.0 (Windows NT 6.1; WOW64; rv:40.0) Gecko\/20100101 Firefox\/40.1"},"urlIsAjaxTrusted":{"\/":true},"superfish":{"1":{"id":"1","sf":{"delay":"300","animation":{"opacity":"show","height":"show"},"speed":"fast"},"plugins":{"smallscreen":{"mode":"window_width","breakpointUnit":"px","expandText":"Desplegar","collapseText":"Plegar","title":"Main menu"},"supposition":true,"supersubs":true}}}});
//--><!]]>
</script>
	
	<script type='text/javascript'>          
           new WOW().init();
    </script>
			<script type='text/javascript'>
				(function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
				(i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
				m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
				})(window,document,'script','https://www.google-analytics.com/analytics.js','ga');
				ga('create', 'UA-96092173-1', 'auto');
				ga('send', 'pageview');
		</script>
		
    </body>
</html>