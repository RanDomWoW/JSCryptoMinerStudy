#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es" dir="ltr" prefix="content: http://purl.org/rss/1.0/modules/content/ dc: http://purl.org/dc/terms/ foaf: http://xmlns.com/foaf/0.1/ og: http://ogp.me/ns# rdfs: http://www.w3.org/2000/01/rdf-schema# sioc: http://rdfs.org/sioc/ns# sioct: http://rdfs.org/sioc/types# skos: http://www.w3.org/2004/02/skos/core# xsd: http://www.w3.org/2001/XMLSchema#">
<head>
  <link rel="profile" href="http://www.w3.org/1999/xhtml/vocab" />
  <meta charset="utf-8">
  <meta name="viewport" content="width=device-width, initial-scale=1.0">
  <meta http-equiv="Content-Type" content="text/html; charset=utf-8" />
<link rel="shortcut icon" href="http://www.uh.cu/sites/default/files/public/icono-t.jpg" type="image/jpeg" />
<meta name="description" content="Somos una universidad de excelencia que busca contribuir al desarrollo de las diferentes esferas de Cuba a través de la formación de profesionales integrales." />
<meta name="abstract" content="Somos una universidad de excelencia que busca contribuir al desarrollo de las diferentes esferas de Cuba a través de la formación de profesionales integrales." />
<meta name="keywords" content="universidad, cuba, estudios, formación, noticias, eventos, investigaciones, intenacionalización, extensión, español, servicios académicos internacionales, pregrado, posgrado, cursos de verano, publicaciones, líneas de investigaciones, proyectos de investigación, facultades, centros, museos, patrimonio, profesores emérito, doctores honoris causa, cooperación internacional, estudiantes internacionales, proyectos comunitarios, movimiento cultural, movimiento deportivo" />
<meta name="generator" content="Drupal 7 (http://drupal.org)" />
<link rel="canonical" href="http://www.uh.cu/" />
<link rel="shortlink" href="http://www.uh.cu/" />
  <title>Universidad de la Habana | Somos una universidad de excelencia que busca contribuir a la formación de profesionales integrales en Cuba.</title>
  <link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css/css_lQaZfjVpwP_oGNqdtWCSpJT1EMqXdMiU84ekLLxQnc4.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css/css_iND_ezW3p8-0VNF8oxS21xuGi8vQIMiXxpLe7N6PigY.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css/css_6v-IgAsjJNGR89jEIEFn3XRmSAVvpW-QryDqTll6__8.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css_injector/css_injector_2.css?p5bykr" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css/css__8X6FMslSyYuTCYDMhnRSOdOdxI5e2a9TPPxzSFlFm4.css" media="all" />
<link type="text/css" rel="stylesheet" href="http://www.uh.cu/sites/default/files/public/css_injector/css_injector_4.css?p5bykr" media="all" />
  <!-- HTML5 element support for IE6-8 -->
  <!--[if lt IE 9]>
    <script src="//html5shiv.googlecode.com/svn/trunk/html5.js"></script>
  <![endif]-->
  <script src="http://www.uh.cu/sites/default/files/public/js/js_Db_2j1ZWrpBz_cEuS8qu_q49p4578FVIE2FmYdItEm0.js"></script>
<script src="http://www.uh.cu/sites/default/files/public/js/js_VmsXuOzPyZfQ-uwuaoxURIli5bT20bsnolVEP9m7wEg.js"></script>
<script src="http://www.uh.cu/sites/default/files/public/js/js__rQx9GxIqWIsVFfgytFwI47onDOWYIRBS-rjTEZdBiw.js"></script>
<script src="http://www.uh.cu/sites/default/files/public/js/js_CS4aK8l_7PFRmXW8TCyJO6Rxq2-KhVQyGs7EPfp2X58.js"></script>
<script>(function(i,s,o,g,r,a,m){i["GoogleAnalyticsObject"]=r;i[r]=i[r]||function(){(i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)})(window,document,"script","https://www.google-analytics.com/analytics.js","ga");ga("create", "UA-114430137-1", {"cookieDomain":"auto"});ga("set", "anonymizeIp", true);ga("send", "pageview");</script>
<script>window.a2a_config=window.a2a_config||{};window.da2a={done:false,html_done:false,script_ready:false,script_load:function(){var a=document.createElement('script'),s=document.getElementsByTagName('script')[0];a.type='text/javascript';a.async=true;a.src='https://static.addtoany.com/menu/page.js';s.parentNode.insertBefore(a,s);da2a.script_load=function(){};},script_onready:function(){da2a.script_ready=true;if(da2a.html_done)da2a.init();},init:function(){for(var i=0,el,target,targets=da2a.targets,length=targets.length;i<length;i++){el=document.getElementById('da2a_'+(i+1));target=targets[i];a2a_config.linkname=target.title;a2a_config.linkurl=target.url;if(el){a2a.init('page',{target:el});el.id='';}da2a.done=true;}da2a.targets=[];}};(function ($){Drupal.behaviors.addToAny = {attach: function (context, settings) {if (context !== document && window.da2a) {if(da2a.script_ready)a2a.init_all('page');da2a.script_load();}}}})(jQuery);a2a_config.callbacks=a2a_config.callbacks||[];a2a_config.callbacks.push({ready:da2a.script_onready});a2a_config.templates=a2a_config.templates||{};</script>
<script src="http://www.uh.cu/sites/default/files/public/js/js_KJdrYSfO7_OAJ2hqRdOX8idAZHHj2azlEwzORRZ4NMM.js"></script>
<script src="http://www.uh.cu/sites/default/files/public/js/js_m7-OgQpeaO7pMZdvXuFDuT6buKPvaSSb4BGzb3laJQw.js"></script>
<script>jQuery.extend(Drupal.settings, {"basePath":"\/","pathPrefix":"","ajaxPageState":{"theme":"bootstrap","theme_token":"_s-faTGA0NN-64qmg9oVNeevA9-VFIQZ_J-EdRxIPGA","jquery_version":"1.10","js":{"0":1,"sites\/all\/themes\/bootstrap\/js\/bootstrap.js":1,"sites\/all\/modules\/jquery_update\/replace\/jquery\/1.10\/jquery.min.js":1,"misc\/jquery.once.js":1,"misc\/drupal.js":1,"sites\/all\/modules\/jquery_update\/replace\/ui\/external\/jquery.cookie.js":1,"sites\/all\/modules\/jquery_update\/replace\/misc\/jquery.form.min.js":1,"misc\/ajax.js":1,"sites\/all\/modules\/jquery_update\/js\/jquery_update.js":1,"sites\/all\/modules\/extlink\/extlink.js":1,"public:\/\/languages\/es_jPKhYtzn0I_AUdTdRgLPfFYdazUxdQ8cArSWvILWa2I.js":1,"sites\/all\/libraries\/bootstrap\/js\/bootstrap.min.js":1,"sites\/all\/libraries\/colorbox\/jquery.colorbox-min.js":1,"sites\/all\/modules\/colorbox\/js\/colorbox.js":1,"sites\/all\/modules\/colorbox\/styles\/default\/colorbox_style.js":1,"sites\/all\/modules\/dhtml_menu\/dhtml_menu.js":1,"sites\/all\/modules\/views_bootstrap\/js\/views-bootstrap-carousel.js":1,"sites\/all\/modules\/views\/js\/base.js":1,"sites\/all\/themes\/bootstrap\/js\/misc\/_progress.js":1,"sites\/all\/modules\/jcarousel\/js\/jquery.jcarousel.min.js":1,"sites\/all\/modules\/jcarousel\/js\/jcarousel.js":1,"sites\/all\/modules\/views\/js\/ajax_view.js":1,"sites\/all\/modules\/google_analytics\/googleanalytics.js":1,"1":1,"2":1,"sites\/all\/libraries\/superfish\/jquery.hoverIntent.minified.js":1,"sites\/all\/libraries\/superfish\/sfsmallscreen.js":1,"sites\/all\/libraries\/superfish\/supposition.js":1,"sites\/all\/libraries\/superfish\/superfish.js":1,"sites\/all\/libraries\/superfish\/supersubs.js":1,"sites\/all\/modules\/superfish\/superfish.js":1,"sites\/all\/themes\/bootstrap\/js\/misc\/ajax.js":1,"sites\/all\/themes\/bootstrap\/js\/modules\/views\/js\/ajax_view.js":1},"css":{"modules\/system\/system.base.css":1,"sites\/all\/modules\/calendar\/css\/calendar_multiday.css":1,"sites\/all\/modules\/date\/date_api\/date.css":1,"sites\/all\/modules\/date\/date_popup\/themes\/datepicker.1.7.css":1,"modules\/field\/theme\/field.css":1,"modules\/node\/node.css":1,"sites\/all\/modules\/youtube\/css\/youtube.css":1,"sites\/all\/modules\/extlink\/extlink.css":1,"sites\/all\/modules\/views\/css\/views.css":1,"sites\/all\/libraries\/bootstrap\/css\/bootstrap.min.css":1,"sites\/all\/libraries\/bootstrap\/css\/bootstrap-theme.min.css":1,"sites\/all\/modules\/colorbox\/styles\/default\/colorbox_style.css":1,"sites\/all\/modules\/ctools\/css\/ctools.css":1,"sites\/all\/modules\/dhtml_menu\/dhtml_menu.css":1,"public:\/\/css\/menu_icons.css":1,"sites\/all\/modules\/panels\/css\/panels.css":1,"sites\/all\/modules\/jcarousel\/skins\/default\/jcarousel-default.css":1,"sites\/all\/modules\/addtoany\/addtoany.css":1,"sites\/all\/libraries\/superfish\/css\/superfish.css":1,"sites\/all\/libraries\/superfish\/css\/superfish-smallscreen.css":1,"sites\/all\/libraries\/superfish\/style\/default.css":1,"public:\/\/css_injector\/css_injector_2.css":1,"public:\/\/css_injector\/css_injector_3.css":1,"public:\/\/css_injector\/css_injector_4.css":1}},"colorbox":{"opacity":"0.85","current":"{current} of {total}","previous":"\u00ab Prev","next":"Next \u00bb","close":"Close","maxWidth":"98%","maxHeight":"98%","fixed":true,"mobiledetect":true,"mobiledevicewidth":"480px"},"dhtmlMenu":{"nav":"hover","animation":{"effects":{"opacity":"opacity","height":0,"width":0},"speed":"500"},"effects":{"siblings":"close-same-tree","children":"none","remember":""},"filter":{"type":"whitelist","list":{"book-toc-16":0,"book-toc-17":0,"book-toc-18":0,"book-toc-19":0,"book-toc-20":0,"book-toc-21":0,"main-menu":0,"management":0,"menu-enlaces-de-inter-s":0,"menu-men-4":0,"menu-otros-enlaces":0,"menu-s-guenos-en":0,"menu-secondary-menu":0,"menu-universidad-de-la-habana":0,"navigation":0,"shortcut-set-1":0,"user-menu":0}}},"jcarousel":{"ajaxPath":"\/jcarousel\/ajax\/views","carousels":{"jcarousel-dom-1":{"view_options":{"view_args":"","view_path":"inicio","view_base_path":null,"view_display_id":"block","view_name":"enlaces_pagina_principal","jcarousel_dom_id":1},"wrap":"circular","skin":"default","visible":5,"scroll":1,"autoPause":1,"start":1,"navigation":"after","selector":".jcarousel-dom-1"}}},"viewsBootstrap":{"carousel":{"1":{"id":1,"name":"Slideshow","attributes":{"interval":10000,"pause":false}}}},"views":{"ajax_path":"\/views\/ajax","ajaxViews":{"views_dom_id:97463e3cf32e355a72bddff8d184d306":{"view_name":"noticias_pagina_principal","view_display_id":"block","view_args":"","view_path":"inicio","view_base_path":"noticias\/recientes","view_dom_id":"97463e3cf32e355a72bddff8d184d306","pager_element":0},"views_dom_id:fd8972fe0c3722f54189ac6c62f9aa60":{"view_name":"calendario","view_display_id":"block_4","view_args":"","view_path":"inicio","view_base_path":"eventos\/proximos","view_dom_id":"fd8972fe0c3722f54189ac6c62f9aa60","pager_element":0},"views_dom_id:5ae94ea46ae5987df093fdc6dcaf3921":{"view_name":"enlaces_pagina_principal","view_display_id":"block","view_args":"","view_path":"inicio","view_base_path":null,"view_dom_id":"5ae94ea46ae5987df093fdc6dcaf3921","pager_element":0}}},"extlink":{"extTarget":"_blank","extClass":"ext","extLabel":"(link is external)","extImgClass":0,"extIconPlacement":0,"extSubdomains":0,"extExclude":"","extInclude":"","extCssExclude":"","extCssExplicit":"","extAlert":0,"extAlertText":"This link will take you to an external web site.","mailtoClass":"mailto","mailtoLabel":"(link sends e-mail)"},"googleanalytics":{"trackOutbound":1,"trackMailto":1,"trackDownload":1,"trackDownloadExtensions":"7z|aac|arc|arj|asf|asx|avi|bin|csv|doc(x|m)?|dot(x|m)?|exe|flv|gif|gz|gzip|hqx|jar|jpe?g|js|mp(2|3|4|e?g)|mov(ie)?|msi|msp|pdf|phps|png|ppt(x|m)?|pot(x|m)?|pps(x|m)?|ppam|sld(x|m)?|thmx|qtm?|ra(m|r)?|sea|sit|tar|tgz|torrent|txt|wav|wma|wmv|wpd|xls(x|m|b)?|xlt(x|m)|xlam|xml|z|zip","trackColorbox":1},"superfish":{"1":{"id":"1","sf":{"animation":{"opacity":"show","height":"show"},"speed":"fast"},"plugins":{"smallscreen":{"mode":"window_width","breakpointUnit":"px","expandText":"Desplegar","collapseText":"Plegar","title":"Men\u00fa principal"},"supposition":true,"supersubs":true}}},"urlIsAjaxTrusted":{"\/":true},"bootstrap":{"anchorsFix":"0","anchorsSmoothScrolling":"0","formHasError":1,"popoverEnabled":1,"popoverOptions":{"animation":1,"html":0,"placement":"right","selector":"","trigger":"click","triggerAutoclose":1,"title":"","content":"","delay":0,"container":"body"},"tooltipEnabled":1,"tooltipOptions":{"animation":1,"html":0,"placement":"auto left","selector":"","trigger":"hover focus","delay":0,"container":"body"}}});</script>
</head>
<body role="document" class="html front not-logged-in no-sidebars page-inicio">
  <div id="skip-link">
    <a href="#main-content" class="element-invisible element-focusable">Pasar al contenido principal</a>
  </div>
    <header id="navbar" role="banner" class="navbar container navbar-default">
    <div class="navegacion">
        <div class="navbar-header">
                            <a class="logo navbar-btn pull-left" href="/"
                   title="Inicio">
                    <img src="http://www.uh.cu/sites/default/files/public/logo-290-aniversario.png" alt="Inicio"/>
                </a>
            
            
                            <button type="button" class="navbar-toggle" data-toggle="collapse" data-target=".navbar-collapse">
                    <span class="sr-only">Toggle navigation</span>
                    <span class="icon-bar"></span>
                    <span class="icon-bar"></span>
                    <span class="icon-bar"></span>
                </button>
                    </div>

                    <div class="navbar-collapse collapse">
                <nav role="navigation">
                                                                  <div class="region region-navigation">
    <section id="block-superfish-1" class="block block-superfish clearfix">

      
  <ul  id="superfish-1" class="menu sf-menu sf-main-menu sf-horizontal sf-style-default sf-total-items-5 sf-parent-items-5 sf-single-items-0"><li id="menu-726-1" class="first odd sf-item-1 sf-depth-1 sf-total-children-7 sf-parent-children-1 sf-single-children-6 menuparent"><a href="/acerca-de-la-uh" class="sf-depth-1 menuparent">Acerca de la UH</a><ul><li id="menu-728-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/acerca-de/historia" class="sf-depth-2">Historia</a></li><li id="menu-730-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/acerca-de-la-uh/mision" class="sf-depth-2">Misión</a></li><li id="menu-732-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/acerca-de/estructura" class="sf-depth-2">Estructura</a></li><li id="menu-734-1" class="middle even sf-item-4 sf-depth-2 sf-total-children-6 sf-parent-children-0 sf-single-children-6 menuparent"><a href="/acerca-de/consejo-universitario" class="sf-depth-2 menuparent">Consejo Universitario</a><ul><li id="menu-750-1" class="first odd sf-item-1 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/equipo-rectoral" class="sf-depth-3">Equipo Rectoral</a></li><li id="menu-752-1" class="middle even sf-item-2 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/directores-de-areas-centrales" class="sf-depth-3">Directores de Áreas Centrales</a></li><li id="menu-754-1" class="middle odd sf-item-3 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/decanos" class="sf-depth-3">Decanos</a></li><li id="menu-756-1" class="middle even sf-item-4 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/directores-de-centros" class="sf-depth-3">Directores de Centros</a></li><li id="menu-758-1" class="middle odd sf-item-5 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/organizaciones-politicas-y-de-masas" class="sf-depth-3">Organizaciones Políticas y de Masas</a></li><li id="menu-760-1" class="last even sf-item-6 sf-depth-3 sf-no-children"><a href="/acerca-de/consejo-universitario/otros-miembros" class="sf-depth-3">Otros Miembros</a></li></ul></li><li id="menu-736-1" class="middle odd sf-item-5 sf-depth-2 sf-no-children"><a href="/acerca-de/patrimonio" class="sf-depth-2">Patrimonio</a></li><li id="menu-738-1" class="middle even sf-item-6 sf-depth-2 sf-no-children"><a href="/acerca-de/profesores-emerito" class="sf-depth-2">Profesores Eméritos</a></li><li id="menu-740-1" class="last odd sf-item-7 sf-depth-2 sf-no-children"><a href="/acerca-de-la-uh/doctores-honoris-causa" class="sf-depth-2">Doctores Honoris Causa</a></li></ul></li><li id="menu-725-1" class="middle even sf-item-2 sf-depth-1 sf-total-children-3 sf-parent-children-2 sf-single-children-1 menuparent"><a href="/estudios" class="sf-depth-1 menuparent">Estudios</a><ul><li id="menu-764-1" class="first odd sf-item-1 sf-depth-2 sf-total-children-2 sf-parent-children-2 sf-single-children-0 menuparent"><a href="/estudiar/pregrado" class="sf-depth-2 menuparent">Pregrado</a><ul><li id="menu-1079-1" class="first odd sf-item-1 sf-depth-3 sf-total-children-2 sf-parent-children-0 sf-single-children-2 menuparent"><a href="/estudios/pregrado/admision-y-matricula" class="sf-depth-3 menuparent">Admisión y Matrícula</a><ul><li id="menu-1084-1" class="first odd sf-item-1 sf-depth-4 sf-no-children"><a href="/estudios/pregrado/admision-y-matricula/postulantes-nacionales" class="sf-depth-4">Postulantes Nacionales</a></li><li id="menu-1124-1" class="last even sf-item-2 sf-depth-4 sf-no-children"><a href="/estudios/pregrado/admision-y-matricula/postulantes-extranjeros" class="sf-depth-4">Postulantes Extranjeros</a></li></ul></li><li id="menu-1080-1" class="last even sf-item-2 sf-depth-3 sf-total-children-3 sf-parent-children-0 sf-single-children-3 menuparent"><a href="/estudios/pregrado/facultades-y-carreras" class="sf-depth-3 menuparent">Facultades y Carreras</a><ul><li id="menu-768-1" class="first odd sf-item-1 sf-depth-4 sf-no-children"><a href="/estudios/pregrado/facultades-y-carreras/ciencias-economicas-y-contables" class="sf-depth-4">Ciencias Económicas y Contables</a></li><li id="menu-770-1" class="middle even sf-item-2 sf-depth-4 sf-no-children"><a href="/estudios/pregrado/facultades-y-carreras/ciencias-naturales-y-exactas" class="sf-depth-4">Ciencias Naturales y Exactas</a></li><li id="menu-772-1" class="last odd sf-item-3 sf-depth-4 sf-no-children"><a href="/estudios/pregrado/facultades-y-carreras/ciencias-sociales-y-humanisticas" class="sf-depth-4">Ciencias Sociales y Humanísticas</a></li></ul></li></ul></li><li id="menu-766-1" class="middle even sf-item-2 sf-depth-2 sf-total-children-6 sf-parent-children-0 sf-single-children-6 menuparent"><a href="/estudios/posgrado" class="sf-depth-2 menuparent">Posgrado</a><ul><li id="menu-1115-1" class="first odd sf-item-1 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/doctorado" class="sf-depth-3">Doctorado</a></li><li id="menu-1116-1" class="middle even sf-item-2 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/maestria" class="sf-depth-3">Maestría</a></li><li id="menu-1122-1" class="middle odd sf-item-3 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/especialidades" class="sf-depth-3">Especialidades</a></li><li id="menu-1117-1" class="middle even sf-item-4 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/cursos" class="sf-depth-3">Cursos</a></li><li id="menu-1121-1" class="middle odd sf-item-5 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/diplomados" class="sf-depth-3">Diplomados</a></li><li id="menu-1119-1" class="last even sf-item-6 sf-depth-3 sf-no-children"><a href="/estudios/posgrado/entrenamientos" class="sf-depth-3">Entrenamientos</a></li></ul></li><li id="menu-774-1" class="last odd sf-item-3 sf-depth-2 sf-no-children"><a href="/estudiar/otros-cursos/cursos-de-verano" class="sf-depth-2">Cursos de Verano</a></li></ul></li><li id="menu-724-1" class="middle odd sf-item-3 sf-depth-1 sf-total-children-8 sf-parent-children-5 sf-single-children-3 menuparent"><a href="/investigacion" class="sf-depth-1 menuparent">Investigación</a><ul><li id="menu-802-1" class="first odd sf-item-1 sf-depth-2 sf-total-children-7 sf-parent-children-0 sf-single-children-7 menuparent"><a href="/investigacion/redes-de-investigacion-universitaria" class="sf-depth-2 menuparent">Redes de Investigación Universitaria</a><ul><li id="menu-804-1" class="first odd sf-item-1 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/red-de-administracion-publica" class="sf-depth-3">Red de administración Pública</a></li><li id="menu-806-1" class="middle even sf-item-2 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/red-de-biotecnologia-y-biomedicina" class="sf-depth-3">Red de Biotecnología y Biomedicina</a></li><li id="menu-808-1" class="middle odd sf-item-3 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/estudios-cooperativos" class="sf-depth-3">Estudios Cooperativos</a></li><li id="menu-810-1" class="middle even sf-item-4 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/fuentes-renovables-de-almacenamiento-de-energia" class="sf-depth-3">Fuentes Renovables de Almacenamiento de Energía</a></li><li id="menu-812-1" class="middle odd sf-item-5 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/medio-ambiente" class="sf-depth-3">Medio Ambiente</a></li><li id="menu-814-1" class="middle even sf-item-6 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/universidad-y-desarrollo-local" class="sf-depth-3">Universidad y Desarrollo Local</a></li><li id="menu-816-1" class="last odd sf-item-7 sf-depth-3 sf-no-children"><a href="/investigacion/redes-de-investigacion-universitarias/materiales-dispositivos-y-medicamentos" class="sf-depth-3">Materiales, dispositivos y medicamentos</a></li></ul></li><li id="menu-818-1" class="middle even sf-item-2 sf-depth-2 sf-total-children-2 sf-parent-children-0 sf-single-children-2 menuparent"><a href="/investigacion/publicaciones" class="sf-depth-2 menuparent">Publicaciones</a><ul><li id="menu-820-1" class="first odd sf-item-1 sf-depth-3 sf-no-children"><a href="/investigacion/publicaciones/articulos-cientificos" class="sf-depth-3">Artículos Científicos</a></li><li id="menu-822-1" class="last even sf-item-2 sf-depth-3 sf-no-children"><a href="/investigacion/publicaciones/libros" class="sf-depth-3">Libros</a></li></ul></li><li id="menu-824-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/investigacion/lineas-de-investigacion" class="sf-depth-2">Líneas de Investigación</a></li><li id="menu-826-1" class="middle even sf-item-4 sf-depth-2 sf-no-children"><a href="/investigacion/proyectos-de-investigacion" class="sf-depth-2">Proyectos de Investigación</a></li><li id="menu-828-1" class="middle odd sf-item-5 sf-depth-2 sf-total-children-1 sf-parent-children-0 sf-single-children-1 menuparent"><a href="/investigacion/premios-la-investigacion" class="sf-depth-2 menuparent">Premios a la Investigación</a><ul><li id="menu-830-1" class="firstandlast odd sf-item-1 sf-depth-3 sf-no-children"><a href="/investigacion/premios-la-investigacion/premios-otorgados-por-la-universidad-de-la-habana" class="sf-depth-3">Universidad de la Habana</a></li></ul></li><li id="menu-832-1" class="middle even sf-item-6 sf-depth-2 sf-total-children-3 sf-parent-children-0 sf-single-children-3 menuparent"><a href="/investigacion/relaciones-con-instituciones-cientificas" class="sf-depth-2 menuparent">Relaciones con Instituciones Científicas</a><ul><li id="menu-840-1" class="first odd sf-item-1 sf-depth-3 sf-no-children"><a href="/investigacion/relaciones-con-instituciones-cientificas/ciencias-naturales-y-exactas" class="sf-depth-3">Ciencias Naturales y Exactas</a></li><li id="menu-842-1" class="middle even sf-item-2 sf-depth-3 sf-no-children"><a href="/investigacion/relaciones-con-instituciones-cientificas/ciencias-sociales-y-humanidades" class="sf-depth-3">Ciencias Sociales y Humanidades</a></li><li id="menu-844-1" class="last odd sf-item-3 sf-depth-3 sf-no-children"><a href="/investigacion/relaciones-con-instituciones-cientificas/ciencias-economicas-y-contables" class="sf-depth-3">Ciencias Económicas</a></li></ul></li><li id="menu-834-1" class="middle odd sf-item-7 sf-depth-2 sf-total-children-1 sf-parent-children-0 sf-single-children-1 menuparent"><a href="/investigacion/trabajo-cientifico-estudiantil" class="sf-depth-2 menuparent">Trabajo Científico Estudiantil</a><ul><li id="menu-846-1" class="firstandlast odd sf-item-1 sf-depth-3 sf-no-children"><a href="/investigacion/trabajo-cientifico-estudiantil/fce-2017" class="sf-depth-3">FCE 2017</a></li></ul></li><li id="menu-848-1" class="last even sf-item-8 sf-depth-2 sf-no-children"><a href="/investigacion/eventos-y-actividades-cientificas/eventos-y-actividades-cientificas" class="sf-depth-2">Eventos y Actividades Científicas</a></li></ul></li><li id="menu-723-1" class="middle even sf-item-4 sf-depth-1 sf-total-children-3 sf-parent-children-0 sf-single-children-3 menuparent"><a href="/internacionalizacion" class="sf-depth-1 menuparent">Internacionalización</a><ul><li id="menu-780-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/internacionalizacion/quienes-somos" class="sf-depth-2">¿Quiénes Somos?</a></li><li id="menu-782-1" class="middle even sf-item-2 sf-depth-2 sf-no-children"><a href="/internacionalizacion/cooperacion-internacional" class="sf-depth-2">Cooperación Internacional</a></li><li id="menu-784-1" class="last odd sf-item-3 sf-depth-2 sf-no-children"><a href="/internacionalizacion/estudiantes-internacionales" class="sf-depth-2">Estudiantes Internacionales</a></li></ul></li><li id="menu-722-1" class="last odd sf-item-5 sf-depth-1 sf-total-children-4 sf-parent-children-1 sf-single-children-3 menuparent"><a href="/extension" class="sf-depth-1 menuparent">Extensión</a><ul><li id="menu-790-1" class="first odd sf-item-1 sf-depth-2 sf-no-children"><a href="/extension/catedras-honorificas" class="sf-depth-2">Cátedras Honoríficas</a></li><li id="menu-792-1" class="middle even sf-item-2 sf-depth-2 sf-total-children-1 sf-parent-children-0 sf-single-children-1 menuparent"><a href="/extension/catedras-honorificas/proyectos-comunitarios" class="sf-depth-2 menuparent">Proyectos Comunitarios</a><ul><li id="menu-794-1" class="firstandlast odd sf-item-1 sf-depth-3 sf-no-children"><a href="/extension/catedras-honorificas/proyectos-comunitarios/proyectos-comunitarios" class="sf-depth-3">Proyectos Comunitarios</a></li></ul></li><li id="menu-796-1" class="middle odd sf-item-3 sf-depth-2 sf-no-children"><a href="/extension/movimiento-cultural" class="sf-depth-2">Movimiento Cultural</a></li><li id="menu-798-1" class="last even sf-item-4 sf-depth-2 sf-no-children"><a href="/extension/movimiento-deportivo" class="sf-depth-2">Movimiento Deportivo</a></li></ul></li></ul>
</section>
<section id="block-menu-menu-secondary-menu" class="block block-menu clearfix">

      
  <ul class="menu nav"><li class="first last leaf"><a href="/contact" title="">Contáctenos</a></li>
</ul>
</section>
<section id="block-search-form" class="block block-search clearfix">

      
  <form class="form-search content-search" action="/" method="post" id="search-block-form" accept-charset="UTF-8"><div><div>
      <h2 class="element-invisible">Formulario de búsqueda</h2>
    <div class="input-group"><input title="Escriba lo que quiere buscar." placeholder="Buscar" class="form-control form-text" type="text" id="edit-search-block-form--2" name="search_block_form" value="" size="15" maxlength="128" /><span class="input-group-btn"><button type="submit" class="btn btn-primary">Buscar</button></span></div><div class="form-actions form-wrapper form-group" id="edit-actions"><button class="element-invisible btn btn-primary form-submit" type="submit" id="edit-submit" name="op" value="Buscar">Buscar</button>
</div><input type="hidden" name="form_build_id" value="form-kdZf5JCuPSrFyKnN_iPa9Im_PaRreLZCJbJkGjZfqDY" />
<input type="hidden" name="form_id" value="search_block_form" />
</div>
</div></form>
</section>
  </div>
                                    </nav>
                <div class="user-menu">
                                            <ul class="menu nav navbar-nav secondary"><li class="first last leaf"><a href="/openid/login" title="">Iniciar sesión</a></li>
</ul>                                    </div>
            </div>
            </div>
</header>

<div class="main-container container">

    <header role="banner" id="page-header">
        
            </header> <!-- /#page-header -->

    <div class="row">

        
        <section class="col-sm-12">
                                    <a id="main-content"></a>
                                        <h1 class="page-header">Bienvenido a Universidad de La Habana</h1>
                                                                                                                  <div class="region region-content">
    <section id="block-system-main" class="block block-system clearfix">

      
  <div class="view view-Slideshow view-id-Slideshow view-display-id-page_1 view-dom-id-ef1967ac16fc3578f1197b5362f67fcc">
        
  
  
      <div class="view-content">
      
<div id="views-bootstrap-carousel-1" class="views-bootstrap-carousel-plugin-style carousel slide"  data-ride="carousel">
      <!-- Carousel indicators -->
    <ol class="carousel-indicators">
              <li data-target="#views-bootstrap-carousel-1" data-slide-to="0" class="active"></li>
              <li data-target="#views-bootstrap-carousel-1" data-slide-to="1" class=""></li>
              <li data-target="#views-bootstrap-carousel-1" data-slide-to="2" class=""></li>
              <li data-target="#views-bootstrap-carousel-1" data-slide-to="3" class=""></li>
              <li data-target="#views-bootstrap-carousel-1" data-slide-to="4" class=""></li>
          </ol>
  
  <!-- Carousel items -->
  <div class="carousel-inner">
          <div class="item active">
        

<div  about="/node/6" typeof="sioc:Item foaf:Document" class="node node-banner node-promoted node-teaser view-mode-teaser">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-bnr-imagen field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/etiquetas/alma-viva"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/banner/public/banner/290-aniv-banner.jpg?itok=16OZeV3b" width="1588" height="550" alt="Aniversario" title="Aniversario" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even">Aniversario</div></div></div><div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded">La Universidad de La Habana celebra su 290 Aniversario</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
      </div>
          <div class="item ">
        

<div  about="/contenido/fidel-entre-los-jovenes" typeof="sioc:Item foaf:Document" class="node node-banner node-promoted node-teaser view-mode-teaser">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-bnr-imagen field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="http://www.uh.cu/galeria/fidel"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/banner/public/banner/p1520775.jpg?itok=vDGgf20J" width="1588" height="550" alt="Fidel entre los jóvenes" title="Fidel entre los jóvenes" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even">Fidel entre los jóvenes</div></div></div><div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded">Ejemplo para cada uno de nosotros</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
      </div>
          <div class="item ">
        

<div  about="/contenido/atlas-de-la-infancia-y-la-adolescencia-en-cuba" typeof="sioc:Item foaf:Document" class="node node-banner node-promoted node-teaser view-mode-teaser">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-bnr-imagen field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="http://www.uh.cu/noticia/atlas-de-la-infancia-y-la-adolescencia-en-cuba"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/banner/public/banner/img_3479dddd.jpg?itok=v5HR_DiO" width="1588" height="550" alt="Atlas de la Infancia y la Adolescencia en Cuba" title="Atlas de la Infancia y la Adolescencia en Cuba" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even">Atlas de la Infancia y la Adolescencia en Cuba</div></div></div><div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded">Un libro de la UNICEF, en colaboración con la Editorial UH</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
      </div>
          <div class="item ">
        

<div  about="/contenido/visita-virtual-la-colina-universitaria" typeof="sioc:Item foaf:Document" class="node node-banner node-promoted node-teaser view-mode-teaser">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-bnr-imagen field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="http://www.uh.cu/visita_virtual_colina_universitaria/bin/tour.html"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/banner/public/banner/img_5737.jpg?itok=AZRzdKeI" width="1588" height="550" alt="Visita Virtual a la Colina Universitaria" title="Visita Virtual a la Colina Universitaria" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even">Visita Virtual a la Colina Universitaria</div></div></div><div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded">Una producción de la Oficina de Asuntos Históricos del Consejo de Estado en coordinación con la FEU</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
      </div>
          <div class="item ">
        

<div  about="/contenido/incuba-0" typeof="sioc:Item foaf:Document" class="node node-banner node-promoted node-teaser view-mode-teaser">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-bnr-imagen field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="http://www.uh.cu/noticia/comienza-la-4ta-ronda-de-incubauhhu-espacio-de-coworking-y-aprendizaje"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/banner/public/banner/incuba.jpg?itok=prqdEDKz" width="1588" height="550" alt="InCuba" title="InCuba" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even">InCuba</div></div></div><div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded">Comienza la 4ta ronda de InCUBA.uhhu: espacio de coworking y aprendizaje</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
      </div>
      </div>

      <!-- Carousel navigation -->
    <a class="carousel-control left" href="#views-bootstrap-carousel-1" data-slide="prev">
      <span class="icon-prev"></span>
    </a>
    <a class="carousel-control right" href="#views-bootstrap-carousel-1" data-slide="next">
      <span class="icon-next"></span>
    </a>
  </div>
    </div>
  
  
  
  
  
  
</div>
</section>
<section id="block-views-estudiar-en-la-uh-block" class="block block-views clearfix">

        <h2 class="block-title">Conoce la UH</h2>
    
  <div class="view view-estudiar-en-la-uh view-id-estudiar_en_la_uh view-display-id-block view-dom-id-0366b4e1017c20485a650cb8d0f58238">
        
  
  
  
      <div class="view-content">
      

<div id="views-bootstrap-grid-1" class="views-bootstrap-grid-plugin-style">
  
          <div class="row">
                  <div class="col col-sm-3">
            

<div  about="/pagina/estudiar-espanol" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-imagenes field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/pagina/estudiar-espanol"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas/public/pages/_mg_5772-2smallsize.jpg?itok=eCMii_Af" width="358" height="296" alt="Estudiar Español" title="Estudiar Español" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/pagina/estudiar-espanol">Estudiar Español</a></h2>
</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/oficinas-y-servicios" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-imagenes field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/pagina/oficinas-y-servicios"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas/public/pages/oficinas-servicios.jpg?itok=_0d4Ojxn" width="358" height="296" alt="Oficinas y Servicios" title="Oficinas y Servicios" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/pagina/oficinas-y-servicios">Oficinas y Servicios</a></h2>
</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/servicios-academicos-internacionales" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-imagenes field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/pagina/servicios-academicos-internacionales"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas/public/pages/estudiantes-estadounidenses-cuba.jpg?itok=gJ4tUuf3" width="358" height="296" alt="Servicios Académicos Internacionales" title="Servicios Académicos Internacionales" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/pagina/servicios-academicos-internacionales">Servicios Académicos Internacionales</a></h2>
</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/como-llegar-la-uh" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-imagenes field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/pagina/como-llegar-la-uh"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas/public/pages/_mg_5678smallsize.jpg?itok=3rHEtK17" width="358" height="296" alt="¿Cómo llegar a la UH?" title="¿Cómo llegar a la UH?" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/pagina/como-llegar-la-uh">¿Cómo llegar a la UH?</a></h2>
</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
              </div>
    
  </div>
    </div>
  
  
  
  
  
</div>
</section>
<section id="block-panels-mini-noticias-y-eventos" class="block block-panels-mini clearfix">

      
  <div class="two-66-33 at-panel panel-display clearfix" id="mini-panel-noticias_y_eventos">
    <div class="region region-two-66-33-first">
    <div class="region-inner clearfix">
      <div class="panel-pane pane-block pane-views-noticias-pagina-principal-block"  >
  
        <h2 class="pane-title">
      Noticias    </h2>
    
  
  <div class="pane-content">
    <div class="view view-noticias-pagina-principal view-id-noticias_pagina_principal view-display-id-block view-dom-id-97463e3cf32e355a72bddff8d184d306">
        
      
<div class="more-link">
  <a href="/noticias/recientes">
    Ver todas  </a>
</div>
  
  
  
      <div class="view-content">
      

<div id="views-bootstrap-grid-2" class="views-bootstrap-grid-plugin-style">
  
          <div class="row">
                  <div class="col col-sm-4">
            

<div  about="/noticia/premios-de-la-ciencia-2017" typeof="sioc:Item foaf:Document" class="node node-article node-promoted node-sticky view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-portada field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/noticia/premios-de-la-ciencia-2017"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas_recortado/public/noticias/_2480319web.jpg?itok=eniDvTDb" width="358" height="215" alt="Premios de la Ciencia 2017" title="Premios de la Ciencia 2017" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/noticia/premios-de-la-ciencia-2017">Premios de la Ciencia 2017</a></h2>
</div></div></div><div class="field field-name-post-date field-type-ds field-label-hidden"><div class="field-items"><div class="field-item even">16 Mar 2018</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-4">
            

<div  about="/noticia/forint-un-proyecto-para-el-intercambio-academico" typeof="sioc:Item foaf:Document" class="node node-article node-promoted view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-portada field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/noticia/forint-un-proyecto-para-el-intercambio-academico"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas_recortado/public/noticias/portada_0.jpg?itok=S_X0Gcpe" width="358" height="215" alt="FORINT, un proyecto para el intercambio académico" title="FORINT, un proyecto para el intercambio académico" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/noticia/forint-un-proyecto-para-el-intercambio-academico">FORINT, un proyecto para el intercambio académico</a></h2>
</div></div></div><div class="field field-name-post-date field-type-ds field-label-hidden"><div class="field-items"><div class="field-item even">15 Mar 2018</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-4">
            

<div  about="/noticia/biomat-2018" typeof="sioc:Item foaf:Document" class="node node-article node-promoted node-sticky view-mode-portada">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-field-portada field-type-image field-label-hidden"><div class="field-items"><div class="field-item even"><a href="/noticia/biomat-2018"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/4_columnas_recortado/public/noticias/img_1780web.jpg?itok=x75DR4Pp" width="358" height="215" alt="BIOMAT 2018" title="BIOMAT 2018" /></a></div></div></div><div class="field field-name-title-field field-type-text field-label-hidden"><div class="field-items"><div class="field-item even"><h2><a href="/noticia/biomat-2018">BIOMAT 2018</a></h2>
</div></div></div><div class="field field-name-post-date field-type-ds field-label-hidden"><div class="field-items"><div class="field-item even">15 Mar 2018</div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
              </div>
    
  </div>
    </div>
  
  
  
  
  
</div>  </div>

  
  </div>
    </div>
  </div>
  <div class="region region-two-66-33-second">
    <div class="region-inner clearfix">
      <div class="panel-pane pane-block pane-views-calendario-block-4"  >
  
        <h2 class="pane-title">
      Eventos    </h2>
    
  
  <div class="pane-content">
    <div class="view view-calendario view-id-calendario view-display-id-block_4 view-dom-id-fd8972fe0c3722f54189ac6c62f9aa60">
        
      
<div class="more-link">
  <a href="/eventos/proximos">
    Ver todos  </a>
</div>
  
  
  
      <div class="view-content">
      

<ul id="views-bootstrap-list-group-1" class="list-group views-bootstrap-list-group-plugin-style">
            <li class="list-group-item">
          
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-02T09:00:00-04:00">02</span> <span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-02T09:00:00-04:00">Abr</span></span>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/evento/puentes-botanicos-2018-congreso-de-la-red-caribena-y-centroamericana-de-jardines-botanicos">Puentes Botánicos 2018 Congreso de la Red Caribeña y Centroamericana de Jardines Botánicos</a></span>  </div>  
  <div class="views-field views-field-field-trns-lugar">        <div class="field-content"><a href="/etiquetas/jardin-botanico-nacional" typeof="skos:Concept" property="rdfs:label skos:prefLabel" datatype="">Jardín Botánico Nacional</a></div>  </div>  
  <div class="views-field views-field-field-date">        <div class="field-content"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-02T09:00:00-04:00">05 Abr 2018</span></div>  </div>      </li>
                <li class="list-group-item">
          
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-nothing">        <span class="field-content"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-03T09:00:00-04:00">03</span> <span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-03T09:00:00-04:00">Abr</span></span>  </div>  
  <div class="views-field views-field-title">        <span class="field-content"><a href="/evento/xxiii-modelo-de-naciones-unidas-de-la-habana-havmun-2018">XXIII Modelo de Naciones Unidas de La Habana (HAVMUN 2018)</a></span>  </div>  
  <div class="views-field views-field-field-trns-lugar">        <div class="field-content"><a href="/etiquetas/universidad-de-la-habana" typeof="skos:Concept" property="rdfs:label skos:prefLabel" datatype="">Universidad de la Habana</a></div>  </div>  
  <div class="views-field views-field-field-date">        <div class="field-content"><span class="date-display-single" property="dc:date" datatype="xsd:dateTime" content="2018-04-03T09:00:00-04:00">06 Abr 2018</span></div>  </div>      </li>
      </ul>
    </div>
  
  
  
  
  
</div>  </div>

  
  </div>
    </div>
  </div>
  </div>

</section>
<section id="block-views-estudiar-en-la-uh-block-1" class="block block-views clearfix">

        <h2 class="block-title">UH en Cifras</h2>
    
  <div class="view view-estudiar-en-la-uh view-id-estudiar_en_la_uh view-display-id-block_1 view-dom-id-f774ee3e37d752c8da75d2bf1244ff2a">
        
  
  
  
      <div class="view-content">
      

<div id="views-bootstrap-grid-3" class="views-bootstrap-grid-plugin-style">
  
          <div class="row">
                  <div class="col col-sm-3">
            

<div  about="/pagina/estudiantes" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-uh_cifras">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><div class="colored red">
<h2 class="big-number">18 894</h2>
<img class="small-img" alt="imagen">
<p class="medium-text">Estudiantes</p>
<p class="small-text"><span class="small-number">14418</span>Pregrado</p>
<br>
<p class="small-text"><span class="small-number">4476</span>Postgrado</p>
</div></div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/postgrado" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-uh_cifras">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><div class="colored blue">
<h2 class="big-number">4 476</h2>
<img class="small-img" alt="imagen">
<p class="medium-text">Estudiantes de postgrado</p>
<p class="small-text"><span class="small-number">832</span>en programas doctorales</p>
<br>
<p class="small-text"><span class="small-number">3 332</span>en programas de maestria</p>
<br>
<p class="small-text"><span class="small-number">312</span>en programas de especialidad</p>
</div></div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/investigaciones" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-uh_cifras">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><div class="colored green">
<h2 class="big-number">402</h2>
<img class="small-img" alt="imagen">
<p class="medium-text">Investigaciones</p>
<p class="small-text"><span class="small-number">402</span>Proyectos de investigación</p>
<br>
<p class="small-text"><span class="small-number">1313</span>Artículos</p>
<br>
<p class="small-text"><span class="small-number">3</span>Patentes</p>
<br>
<p class="small-text"><span class="small-number">13</span>Premios internacionales</p>
<br>
<p class="small-text"><span class="small-number">70</span>Premios nacionales</p>
</div></div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
                  <div class="col col-sm-3">
            

<div  about="/pagina/relaciones-internacionales" typeof="sioc:Item foaf:Document" class="node node-pagina-estatica node-promoted view-mode-uh_cifras">
    <div class="row">
    <div class="col-sm-12 ">
      <div class="field field-name-body field-type-text-with-summary field-label-hidden"><div class="field-items"><div class="field-item even" property="content:encoded"><div class="colored gray">
<h2 class="big-number">1 088</h2>
<img class="small-img" alt="imagen">
<p class="medium-text">Acciones</p>
<p class="small-text">Relaciones Internacionales</p>
<br>
<img class="big-img" alt="imagen">
</div></div></div></div>    </div>
  </div>
</div>


<!-- Needed to activate display suite support on forms -->
          </div>
              </div>
    
  </div>
    </div>
  
  
  
  
  
</div>
</section>
<section id="block-views-enlaces-pagina-principal-block" class="block block-views clearfix">

        <h2 class="block-title">Descubre</h2>
    
  <div class="view view-enlaces-pagina-principal view-id-enlaces_pagina_principal view-display-id-block view-dom-id-5ae94ea46ae5987df093fdc6dcaf3921">
        
  
  
      <div class="view-content">
      <ul class="jcarousel jcarousel-view--enlaces-pagina-principal--block jcarousel-dom-1 jcarousel-skin-default">
      <li class="jcarousel-item-1 odd" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.uh.cu/voces-de-la-uh"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/voces-para-sitio-uh.jpg?itok=v5xp8hk3" width="264" height="231" alt="Voces UH" title="Voces UH" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.uh.cu/voces-de-la-uh">Voces de la UH</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Personalidades relevantes</p>
</div>  </div></li>
      <li class="jcarousel-item-2 even" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="https://incuba.uh.cu/"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/006_designthinking_incuba.jpg?itok=L5rq_oE0" width="264" height="231" alt="InCuba" title="InCuba" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="https://incuba.uh.cu/">InCuba</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Transforma tus ideas a la realidad</p>
</div>  </div></li>
      <li class="jcarousel-item-3 odd" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.uh.cu/visita_virtual_colina_universitaria/bin/tour.html"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/img_6861.jpg?itok=f243Iagk" width="264" height="231" alt="Visita Virtual de la UH" title="Visita Virtual de la UH" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.uh.cu/visita_virtual_colina_universitaria/bin/tour.html">Visita Virtual de la UH</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Caminando por la colina universitaria</p>
</div>  </div></li>
      <li class="jcarousel-item-4 even" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.priais.uh.cu/"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/14373294588095.jpg?itok=ugZaS5hH" width="264" height="231" alt="Biblioteca Central" title="Biblioteca Central" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.priais.uh.cu/">Biblioteca Central</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Consulte nuestro fondo general</p>
</div>  </div></li>
      <li class="jcarousel-item-5 odd" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.uh.cu/investigacion/otri"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/otri.jpg?itok=PaJ9zOWr" width="264" height="231" alt="OTRI" title="OTRI" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.uh.cu/investigacion/otri">OTRI</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content">Transferencias de Investigaciones</div>  </div></li>
      <li class="jcarousel-item-6 even" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.uh.cu/descargas/libro-del-graduado"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/libro_de_graduados.jpg?itok=M-1uX2Dd" width="264" height="231" alt="Libro de Graduados" title="Libro de Graduados" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.uh.cu/descargas/libro-del-graduado">Libro de Graduados</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Libro de graduados de la UH</p>
</div>  </div></li>
      <li class="jcarousel-item-7 odd" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.uh.cu/galerias"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/galerias_2.jpg?itok=6Ei742PG" width="264" height="231" alt="Galerías" title="Galerías" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.uh.cu/galerias">Galerías</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Un recorrido por nuestras memorias</p>
</div>  </div></li>
      <li class="jcarousel-item-8 even" style="display: none;">  
  <div class="views-field views-field-edit-node">        <span class="field-content"></span>  </div>  
  <div class="views-field views-field-field-shrt-imagen">        <div class="field-content"><a href="http://www.scriptorium.uh.cu/xmlui/"><img typeof="foaf:Image" class="img-responsive" src="http://www.uh.cu/sites/default/files/public/styles/5_columnas/public/shortcuts/scriptorium_.jpg?itok=YjVu_DAr" width="264" height="231" alt="Scriptorium" title="Scriptorium" /></a></div>  </div>  
  <div class="views-field views-field-field-enlace">        <div class="field-content"><a href="http://www.scriptorium.uh.cu/xmlui/">Scriptorium</a></div>  </div>  
  <div class="views-field views-field-body">        <div class="field-content"><p>Repositario institucional</p>
</div>  </div></li>
  </ul>
    </div>
  
  
  
  
  
  
</div>
</section>
  </div>
        </section>

        
    </div>
</div>

    <footer class="footer container">
          <div class="region region-footer">
    <section id="block-menu-menu-s-guenos-en" class="block block-menu clearfix">

        <h2 class="block-title">Síguenos en</h2>
    
  <ul class="menu nav"><li class="first leaf"><a href="https://www.facebook.com/universidaduhcuba" title="Facebook" class="menu_icon menu-935">Facebook</a></li>
<li class="leaf"><a href="https://www.twitter.com/UdeLaHabana" title="Twitter" class="menu_icon menu-936">Twitter</a></li>
<li class="last leaf"><a href="https://www.youtube.com/channel/UC9m6ceu60hA1z0sLe5ucFag" title="YouTube" class="menu_icon menu-953">YouTube</a></li>
</ul>
</section>
<section id="block-panels-mini-men-footer" class="block block-panels-mini clearfix">

      
  <div class="four-4x25 at-panel panel-display clearfix" id="mini-panel-men_footer">
  <div class="panel-row row-1 clearfix">
    <div class="region region-four-first">
      <div class="region-inner clearfix">
        <div class="panel-pane pane-block pane-menu-menu-universidad-de-la-habana"  >
  
        <h2 class="pane-title">
      Universidad de la Habana    </h2>
    
  
  <div class="pane-content">
    <ul class="menu nav"><li class="first leaf"><a href="/acerca-de-la-uh" title="">Acerca de la UH</a></li>
<li class="leaf"><a href="/estudios" title="">Estudios</a></li>
<li class="leaf"><a href="/investigacion" title="">Investigación</a></li>
<li class="leaf"><a href="/internacionalizacion" title="">Internacionalización</a></li>
<li class="leaf"><a href="/extension" title="">Extensión</a></li>
<li class="leaf"><a href="/noticias" title="">Noticias</a></li>
<li class="last leaf"><a href="/eventos" title="">Eventos</a></li>
</ul>  </div>

  
  </div>
      </div>
    </div>
    <div class="region region-four-second">
      <div class="region-inner clearfix">
        <div class="panel-pane pane-block pane-menu-menu-otros-enlaces"  >
  
        <h2 class="pane-title">
      Otras Universidades    </h2>
    
  
  <div class="pane-content">
    <ul class="menu nav"><li class="first leaf"><a href="http://cujae.edu.cu/" title="">Universidad Tecnológica de La Habana</a></li>
<li class="leaf"><a href="http://instituciones.sld.cu/elam/" title="">Escuela Latinoamericana de Medicina</a></li>
<li class="leaf"><a href="http://www.unah.edu.cu/" title="">Universidad Agraria de la Habana</a></li>
<li class="leaf"><a href="http://www.uclv.edu.cu/" title="">Universidad Central de Las Villas</a></li>
<li class="leaf"><a href="http://www.reduc.edu.cu/" title="">Universidad de Camagüey</a></li>
<li class="leaf"><a href="https://www.uci.cu/" title="">Universidad de Ciencias Informáticas</a></li>
<li class="leaf"><a href="http://instituciones.sld.cu/ucmh/" title="">Universidad de Ciencias Médicas de La Habana</a></li>
<li class="leaf"><a href="https://www.ucf.edu.cu/" title="">Universidad de Cienfuegos</a></li>
<li class="leaf"><a href="http://www.udg.co.cu/" title="">Universidad de Granma</a></li>
<li class="leaf"><a href="http://www.uho.edu.cu/" title="">Universidad de Holguín</a></li>
<li class="leaf"><a href="http://www.umcc.cu/" title="">Universidad de Matanzas</a></li>
<li class="leaf"><a href="https://www.uo.edu.cu/" title="">Universidad de Oriente</a></li>
<li class="leaf"><a href="http://www.upr.edu.cu/" title="">Universidad de Pinar del Río</a></li>
<li class="leaf"><a href="https://www.ismm.edu.cu/" title="">Instituto Superior Minero Metalúrgico de Moa</a></li>
<li class="leaf"><a href="http://www.isa.cult.cu/" title="">Instituto Superior de Arte</a></li>
<li class="leaf"><a href="http://ua.uart.edu.cu/" title="">Universidad de Artemisa</a></li>
<li class="leaf"><a href="http://www.cug.co.cu/index.php/es/" title="">Universidad de Guantánamo</a></li>
<li class="last leaf"><a href="http://www.ult.edu.cu/" title="">Universidad de Las Tunas</a></li>
</ul>  </div>

  
  </div>
      </div>
    </div>
  </div>
  <div class="panel-row row-2 clearfix">
    <div class="region region-four-third">
      <div class="region-inner clearfix">
        <div class="panel-pane pane-block pane-menu-menu-enlaces-de-inter-s"  >
  
        <h2 class="pane-title">
      Sitios de interés    </h2>
    
  
  <div class="pane-content">
    <ul class="menu nav"><li class="first leaf"><a href="http://catedrasunesco.uh.cu/" title="">Cátedras UNESCO</a></li>
<li class="leaf"><a href="http://www.academiaciencias.cu/es" title="">Academia de Ciencias de Cuba</a></li>
<li class="leaf"><a href="http://www.aduana.co.cu/" title="">Aduana General de la República</a></li>
<li class="leaf"><a href="http://www.fgr.cu/es" title="">Fiscalía General de la República</a></li>
<li class="leaf"><a href="http://www.sld.cu/" title="">Infomed</a></li>
<li class="leaf"><a href="http://www.mes.gob.cu" title="">Ministerio de Educación Superior</a></li>
<li class="leaf"><a href="http://www.redciencia.cu/" title="">Red Cubana de la Ciencia</a></li>
<li class="leaf"><a href="https://antorchae.blogspot.com/" title="">Antorcha Encendida</a></li>
<li class="leaf"><a href="https://www.facebook.com/desdelauh/?hc_ref=ARQkafLoZrhbqzrZj4YZSBp87NksMlCGV4_UiSbm7MMYpXCzh3iPoo273UxPn69iD9w" title="">Desde la UH</a></li>
<li class="leaf"><a href="http://www.feu.cu/" title="">FEU Nacional</a></li>
<li class="leaf"><a href="http://www.citma.gob.cu/" title="">Ministerio de Ciencia, Tecnología y Medio Ambiente</a></li>
<li class="leaf"><a href="http://www.filosofia.cu/" title="">Portal de Filosofía y Pensamiento Cubanos</a></li>
<li class="last leaf"><a href="http://www.project-forint.eu/" title="">Proyecto FORINT</a></li>
</ul>  </div>

  
  </div>
      </div>
    </div>
    <div class="region region-four-fourth">
      <div class="region-inner clearfix">
        <div class="panel-pane pane-block pane-menu-menu-men-4"  >
  
        <h2 class="pane-title">
      Medios de Prensa    </h2>
    
  
  <div class="pane-content">
    <ul class="menu nav"><li class="first leaf"><a href="http://www.acn.cu/" title="">Agencia Cubana de Noticias</a></li>
<li class="leaf"><a href="http://www.cubadebate.cu/" title="">CubaDebate</a></li>
<li class="leaf"><a href="http://www.cubaeduca.cu/" title="">CubaEduca</a></li>
<li class="leaf"><a href="https://www.ecured.cu/EcuRed:Enciclopedia_cubana" title="">EcuRed</a></li>
<li class="leaf"><a href="https://www.gacetaoficial.gob.cu/" title="">Gaceta Oficial de Cuba</a></li>
<li class="leaf"><a href="http://www.granma.cu/" title="">Granma</a></li>
<li class="leaf"><a href="http://www.juventudrebelde.cu/" title="">Juventud Rebelde</a></li>
<li class="leaf"><a href="http://www.juventudtecnica.cu/" title="">Juventud Técnica</a></li>
<li class="leaf"><a href="http://www.prensa-latina.cu/" title="">Prensa Latina</a></li>
<li class="leaf"><a href="http://www.radiocubana.cu/" title="">Radio Cubana</a></li>
<li class="leaf"><a href="http://www.radiosdecuba.com/progreso/" title="">Radio Progreso Cuba</a></li>
<li class="leaf"><a href="http://www.radiorebelde.cu/" title="">Radio Rebelde</a></li>
<li class="leaf"><a href="http://www.radiotaino.cu/" title="">Radio Taíno</a></li>
<li class="leaf"><a href="http://www.almamater.cu/" title="">Revista Alma Mater</a></li>
<li class="leaf"><a href="http://www.trabajadores.cu/" title="">Trabajadores</a></li>
<li class="last leaf"><a href="http://www.tribuna.cu/" title="">Tribuna de La Habana</a></li>
</ul>  </div>

  
  </div>
      </div>
    </div>
  </div>
</div>

</section>
<section id="block-block-1" class="block block-block clearfix">

      
  © 2018 | Universidad de La Habana | Dirección de Comunicación Institucional | Dirección Docente de Informatización
</section>
  </div>
    </footer>
  <script>da2a.targets=[
{title:"Estudiar Espa\u00f1ol",url:"http:\/\/www.uh.cu\/pagina\/estudiar-espanol"},
{title:"Oficinas y Servicios",url:"http:\/\/www.uh.cu\/pagina\/oficinas-y-servicios"},
{title:"Servicios Acad\u00e9micos Internacionales",url:"http:\/\/www.uh.cu\/pagina\/servicios-academicos-internacionales"},
{title:"\u00bfC\u00f3mo llegar a la UH?",url:"http:\/\/www.uh.cu\/pagina\/como-llegar-la-uh"},
{title:"Premios de la Ciencia 2017",url:"http:\/\/www.uh.cu\/noticia\/premios-de-la-ciencia-2017"},
{title:"FORINT, un proyecto para el intercambio acad\u00e9mico",url:"http:\/\/www.uh.cu\/noticia\/forint-un-proyecto-para-el-intercambio-academico"},
{title:"BIOMAT 2018",url:"http:\/\/www.uh.cu\/noticia\/biomat-2018"},
{title:"Estudiantes",url:"http:\/\/www.uh.cu\/pagina\/estudiantes"},
{title:"Postgrado",url:"http:\/\/www.uh.cu\/pagina\/postgrado"},
{title:"Investigaciones",url:"http:\/\/www.uh.cu\/pagina\/investigaciones"},
{title:"Relaciones internacionales",url:"http:\/\/www.uh.cu\/pagina\/relaciones-internacionales"}];
da2a.html_done=true;if(da2a.script_ready&&!da2a.done)da2a.init();da2a.script_load();</script>
<script src="http://www.uh.cu/sites/default/files/public/js/js_wwbWeGFY4LqlDJcM24vTyNxfsF55aMqFTyAsLkutCNE.js"></script>
</body>
</html>