#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html>
    <head lang="es">
        <meta charset="UTF-8">
                            <meta name="DC.title" content="Portal Cuba. Noticias, productos y servicios para Cuba y el mundo">
                    <meta name="DC.subject" content="Cuba, Todo de Cuba, Información sobre Cuba, Información de Cuba, Arte, Ciencia, Tecnología, Deportes, Economía, Negocios, Educación, Eventos, Medios de comunicación, Política y Gobierno, Salud, Turismo, Autores cubanos, Bazar Cuba, Cocina cubana, Consultas médicas, Cuba-NIC, NIC, Cubanic, Cursos en línea, Efemérides, Tiempo, Clima, Pronóstico del tiempo, Libros cubanos, E-books cubanos,  Autores cubanos, Audiovisuales Cubanos, Películas Cubanas, Navegue fácil, Novedades literarias, Páginas amarillas, Sitios cubanos, Cuba.cu, Portal, Portales, Servicios, Servicios de valor agregado, Sitios web, Portal cubano, Portal Cuba, Citmatel, Prensa cubana, Gobierno, Reflexiones de Fidel, Discursos de Raúl Castro, Postales, Bazar de Cuba, Compras para Cuba, Compras para el mundo, Compra de Todo, Compra DTodo, recarga de celulares, activación de líneas para móviles, Noticias, Web, Editorial, Audiolibros, Libros Cuba, Libros Electrónicos, Libro para Escuchar, Cuba Libros">
                    <meta name="DC.publisher" content="Empresa de Tecnologías de la Información y Servicios Telemáticos Avanzados (CITMATEL)">
                    <meta name="DC.keywords" content="Cuba, Todo de Cuba, Información sobre Cuba, Información de Cuba, Arte, Ciencia, Tecnología, Deportes, Economía, Negocios, Educación, Eventos, Medios de comunicación, Política y Gobierno, Salud, Turismo, Autores cubanos, Bazar Cuba, Cocina cubana, Consultas médicas, Cuba-NIC, NIC, Cubanic, Cursos en línea, Efemérides, Tiempo, Clima, Pronóstico del tiempo, Libros cubanos, E-books cubanos,  Autores cubanos, Audiovisuales Cubanos, Películas Cubanas, Navegue fácil, Novedades literarias, Páginas amarillas, Sitios cubanos, Cuba.cu, Portal, Portales, Servicios, Servicios de valor agregado, Sitios web, Portal cubano, Portal Cuba, Citmatel, Prensa cubana, Gobierno, Reflexiones de Fidel, Discursos de Raúl Castro, Postales, Bazar de Cuba, Compras para Cuba, Compras para el mundo, Compra de Todo, Compra DTodo, recarga de celulares, activación de líneas para móviles, Noticias, Web, Editorial, Audiolibros, Libros Cuba, Libros Electrónicos, Libro para Escuchar, Cuba Libros">
                    <meta name="DC.description" content="Noticias de Cuba, estado del tiempo, tasas de  cambio, acceso a cientos de sitios cubanos y promociones especiales son algunas de nuestras propuestas.">
                    <meta name="title" content="Portal Cuba. Noticias, productos y servicios para Cuba y el mundo">
                    <meta name="subject" content="Cuba, Todo de Cuba, Información sobre Cuba, Información de Cuba, Arte, Humanidades, Ciencia, Tecnología, Deportes, Economía, Negocios, Educación, Eventos, Medios de comunicación, Política, Gobierno, Salud y Medicina, Turismo, Autores cubanos, Bazar Cuba, Cocina cubana, Consultas médicas, Cuba-NIC, NIC, Cubanic, Cursos en línea, Efemérides, Tiempo, Clima, Pronóstico del tiempo, Libros cubanos, E-books cubanos,  Autores cubanos, Audiovisuales Cubanos, Películas Cubanas, Navegue fácil, Novedades literarias, Páginas amarillas, Sitios cubanos, Cuba.cu, Portal, Portales, Servicios, Servicios de valor agregado, Sitio web, Sitios web, Portal general, Portal cubano, Portal Cuba, Portales generales, Citmatel, CITMATEL, Ceniai, CENIAI, CENIAInternet, Prensa, Prensa cubana, Gobierno, Política, Postales, Bazar de Cuba, Compras para Cuba, Compras para el mundo, Compra de Todo, Compra DTodo, recarga y activación de móviles, Servicios de valor añadido, Noticias, Web, Editorial, Audiolibros, Libros Cuba, Libros Electrónicos, Libro para Escuchar, Cuba Libros">
                    <meta name="publisher" content="Empresa de Tecnologías de la Información y Servicios Telemáticos Avanzados (CITMATEL)">
                    <meta name="keywords" content="Cuba, Todo de Cuba, Información sobre Cuba, Fidel Castro, Arte, Humanidades, Ciencia, Tecnología, Deportes, Economía, Negocios, Educación, Eventos, Medios de comunicación, Política, Gobierno, Salud y Medicina, Turismo, Autores cubanos, Bazar Cuba, Cocina cubana, Consultas médicas, Cubanic, Cursos en línea, Efemérides, Tiempo, Clima, Pronóstico del tiempo, Libros cubanos, E-books cubanos,  Autores cubanos, Audiovisuales Cubanos, Novedades literarias, Páginas, Portal cubano, Portal Cuba, Citmatel, Ceniai, CENIAI, CENIAInternet, Prensa, Gobierno, Política, Postales, Bazar de Cuba, Compras para Cuba, Compras para el mundo, Compra de Todo, Recarga y activación de móviles, Noticias Cuba, Web, Editorial, Audiolibros, Libros Cuba, Libros Electrónicos, Cuba Libros">
                    <meta name="description" content="Noticias de Cuba, estado del tiempo, tasas de  cambio, acceso a cientos de sitios cubanos y promociones especiales son algunas de nuestras propuestas.">
                    
                <meta http-equiv="X-UA-Compatible" content="IE=edge">
        <!--Para el Responsive-->
        <meta name="viewport" content="width=device-width, initial-scale=1">

        <title>Portal Cuba - Noticias, productos y servicios para Cuba y el mundo </title>
        <link rel="SHORTCUT ICON" href="http://www.cuba.cu/imgs/logo.ico">
        <!-- Bootstrap CSS -->
        <link href="http://www.cuba.cu/assets/bs/css/bootstrap.min.css" rel="stylesheet">
        <!-- Optional theme -->
        <!--<link href="bs/css/bootstrap-theme.css" rel="stylesheet">-->
        <!--MY CSS-->
        <link href="http://www.cuba.cu/assets/css/main.css" rel="stylesheet">
        <!-- HTML5 shim and Respond.js for IE8 support of HTML5 elements and media queries -->
        <!-- WARNING: Respond.js doesn't work if you view the page via file:// -->
        <!--[if lt IE 9]>
        <script src="http://www.cuba.cu/bs/js/html5shiv.min.js"></script>
        <script src="http://www.cuba.cu/bs/js/respond.min.js"></script>
        <![endif]-->
        <link href="http://www.cuba.cu/assets/js/lib/datepiker/css/bootstrap-datepicker3.min.css" rel="stylesheet">
        <script>
            baseURI = "http://www.cuba.cu/";
            date_now = "17/03/2018"
        </script>
        <!-- Piwik -->
        <!--<script type="text/javascript">
            var _paq = _paq || [];
            _paq.push(['trackPageView']);
            _paq.push(['enableLinkTracking']);
            (function() {
                var u="//piwik.ceniai.inf.cu/";
                _paq.push(['setTrackerUrl', u+'piwik.php']);
                _paq.push(['setSiteId', 1]);
                var d=document, g=d.createElement('script'), s=d.getElementsByTagName('script')[0];
                g.type='text/javascript'; g.async=true; g.defer=true; g.src=u+'piwik.js'; s.parentNode.insertBefore(g,s);
            })();
        </script>
        <noscript><p><img src="//piwik.ceniai.inf.cu/piwik.php?idsite=1" style="border:0;" alt=""
                    /></p></noscript>-->
        <!-- End Piwik Code -->
    </head>
    <body>        
        <nav id="pcuba-main-menu" class="navbar navbar-default navbar-fixed-top" role="navigation">
            <div class="container">
                <div class="navbar-header">
                    <a id="logo" class="navbar-brand home-active" href="http://www.cuba.cu/">
                        <img src="http://www.cuba.cu/imgs/logo.png" alt="Portal Cuba"/>
                    </a>
                    <ul class="pcuba-navbar-toggle hidden-lg hidden-md hidden-sm">
                        <li>
                            <button class="btn-toggle" type="button" data-toggle="collapse" data-target=".js-navbar-collapse" title="Men&uacute;">
                                <span class="sr-only">Toggle navigation</span>
                                <span class="menu-toggle"></span>
                            </button>
                        </li>
                        <li>
                            <div class="btn-toggle">
                                <span class="sr-only">Toggle navigation</span>
                                <a href="#" class="searcher-toggle"  data-toggle="dropdown" title="Buscador"></a>
                            </div>
                        </li>
                        <!--<li>+
                            <div class="btn-group btn-toggle idiom-dropdown">
                                <a href="#" class="dropdown-toggle" data-toggle="dropdown" role="button" aria-haspopup="true" aria-expanded="false">ES <span class="fontAwesomeCaret"></span></a>
                                <ul class="dropdown-menu dropdown-menu-right">
                                    <li><a href="#"><span class="style-idiom">English </span><span class="style-abbr">EN</span></a></li>
                                </ul>
                            </div>
                        </li>-->
                    </ul>
                </div>
                <div class="collapse navbar-collapse js-navbar-collapse">
                    <ul class="nav navbar-nav">
                        <li class="pcuba-megamenu">
                            <a id="home-menu" class="text-center rl-single-bottom-align" href="http://www.cuba.cu/">
                                <span class="hidden-xs menu-active"  title="Inicio"></span>
                                <span class="home-word hidden-lg hidden-md hidden-sm menu-active">inicio</span>
                            </a>
                        </li>
                        <li id="category-megamenu" class="dropdown pcuba-megamenu">
                            <a href="#" class="dropdown-toggle " data-toggle="dropdown">categor&iacute;as <span></span></a>
                            <!--
    <ul class="dropdown-menu dropdown-megamenu none-col">
    <ul class="dropdown-menu dropdown-megamenu one-col">
    <ul class="dropdown-menu dropdown-megamenu">
-->    <ul class="dropdown-menu dropdown-megamenu">
        <li class="section-megamenu">
            <ul class="category-list">
                            <li class="category-row" data-category="medio-ambiente">
                    <a href="http://www.cuba.cu/category/medio-ambiente">
                        <span class="category-name">
                            Medio Ambiente                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="politica">
                    <a href="http://www.cuba.cu/category/politica">
                        <span class="category-name">
                            Política                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="ciencia-y-tecnologia">
                    <a href="http://www.cuba.cu/category/ciencia-y-tecnologia">
                        <span class="category-name">
                            Ciencia y Tecnología                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="deporte">
                    <a href="http://www.cuba.cu/category/deporte">
                        <span class="category-name">
                            Deporte                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="turismo">
                    <a href="http://www.cuba.cu/category/turismo">
                        <span class="category-name">
                            Turismo                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="salud">
                    <a href="http://www.cuba.cu/category/salud">
                        <span class="category-name">
                            Salud                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="educacion">
                    <a href="http://www.cuba.cu/category/educacion">
                        <span class="category-name">
                            Educación                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="cultura">
                    <a href="http://www.cuba.cu/category/cultura">
                        <span class="category-name">
                            Cultura                        </span>
                    </a>
                </li>
                            <li class="category-row" data-category="economia">
                    <a href="http://www.cuba.cu/category/economia">
                        <span class="category-name">
                            Economía                        </span>
                    </a>
                </li>
                    </ul>
    </li>
            <li class="section-megamenu summary-megamenu">
            <ul>
                                                                                            <li class="" data-category="medio-ambiente" data-item-count="4">
                <header>en: <span>Medio Ambiente</span></header>
                <ul>
                                                              <li  >
                            <ul class="category-item-list item-list-megamenu">
                                                  <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/medio-ambiente/2018-02-26/evaluaran-conservacion-y-uso-sostenible-de-la-biodiversidad-en-el-caribe/40385">
                                                 Evaluarán conservación y uso sostenible de la biodiversidad en el Caribe                                             </a>
                                         </h3>
                                         <p>
                                                                                          La Habana, 26 feb.- La II Reunión Regional del Comité Directivo del Proyecto GEF-IWEco y la presentación de su Campaña de Comunicación                                          </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                                                                                   <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.cenais.cu/" target="_blank">Centro Nacional de Investigaciones Sismológicas, CENAIS</a></h3>
                                <p class="site-description">
                                    Centro contribuyente a la mitigación del riesgo sísmico en la República de Cuba.                                </p>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                                     <li class="hidden-md" >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.redciencia.cu/cdorigen/arca/protegefam.html" target="_blank">Serie educativa Protege a tu familia del impacto de fenómenos naturales</a></h3>
                                <p class="site-description">
                                    Estos folletos e infografías han sido elaborados como parte de una campaña de educación comunitaria, destinada a enseñar al hombre a protegerse de los eventos naturales de origen geológico y climático.                                 </p>
                            </article>
                        </section>
                    </li>
                                                                                                                                 <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=34" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/abecedario-naturaleza_menu_thumb.jpg" alt=" Abecedario de la naturaleza" title=" Abecedario de la naturaleza"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=128" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/cambio-climatico1_menu_thumb.jpg" alt="Cambio climático, el reto continúa" title="Cambio climático, el reto continúa"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=344" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/Cuba-prevencion-de-desastres-asociados-al-cambio-climatico_menu_thumb.jpg" alt="Cuba: prevención de desastres asociados al cambio climático (Ebook)" title="Cuba: prevención de desastres asociados al cambio climático (Ebook)"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                        </ul>
             </li>
                                                                                            <li class="hidden" data-category="politica" data-item-count="4">
                <header>en: <span>Política</span></header>
                <ul>
                                                                            <li  >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://cubasi.cu/cubasi-noticias-cuba-mundo-ultima-hora/item/66236-machado-ventura-el-rumbo-de-la-revolucion-ya-esta-trazado" target="_blank">Machado Ventura: El rumbo de la Revolución está trazado</a></h3>
                                <p class="site-description">
                                    Discurso de José Ramón Machado Ventura por el Aniversario 64 del asalto a los cuarteles Moncada y Carlos M. de Céspedes.                                </p>
                            </article>
                        </section>
                    </li>
                                                                                                 <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.granma.cu/cuba/2017-06-01/texto-integro-del-discurso-pronunciado-por-raul-en-la-segunda-sesion-extraordinaria-de-la-asamblea-nacional-del-poder-popular" target="_blank">Raúl Castro en Segunda Sesión Extraordinaria de la Asamblea Nacional del Poder Popular</a></h3>
                                <p class="site-description">
                                    Texto íntegro del Discurso pronunciado por Raúl en la Segunda Sesión Extraordinaria de la Asamblea Nacional del Poder Popular                                </p>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                                     <li class="hidden-md" >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.cubadebate.cu/especiales/2017/12/22/raul-en-la-asamblea-nacional-aqui-estamos-y-estaremos-libres-soberanos-e-independientes/#.Wj0JTUbLFnw" target="_blank">Raúl en la Asamblea Nacional: “Aquí estamos y estaremos, libres, soberanos e independientes”</a></h3>
                                <p class="site-description">
                                    Discurso pronunciado por el General de Ejército Raúl Castro Ruz, Primer Secretario del Comité Central del Partido Comunista de Cuba y Presidente de los Consejos de Estado y de Ministros, en la clausura del X Período Ordinario de Sesiones de la VIII Legislatura de la Asamblea Nacional del Poder Popular, en el Palacio de Convenciones, el 21 de diciembre de 2017, “Año 59 de la Revolución”.                                </p>
                            </article>
                        </section>
                    </li>
                                                                                                                                 <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=234" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/fidel-soldado-ideas_menu_thumb.jpg" alt="Ebook Fidel Castro. Soldado de las Ideas" title="Ebook Fidel Castro. Soldado de las Ideas"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=230" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/multimedia-fidel-es-fidel_menu_thumb.jpg" alt="Multimedia Fidel es Fidel" title="Multimedia Fidel es Fidel"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=191" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/revolucion_menu_thumb.jpg" alt="DVD Historia de la Revolución Cubana" title="DVD Historia de la Revolución Cubana"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                        </ul>
             </li>
                                                                                            <li class="hidden" data-category="ciencia-y-tecnologia" data-item-count="5">
                <header>en: <span>Ciencia y Tecnología</span></header>
                <ul>
                                                                            <li  >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="https://supuestosataques.redciencia.cu/" target="_blank">La Ciencia y los Ataques Acústicos</a></h3>
                                <p class="site-description">
                                    Resultados Científicos de la investigación cubana sobre los supuestos ataques acústicos a los diplomáticos estadounidenses                                </p>
                            </article>
                        </section>
                    </li>
                                                                                                 <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.citma.gob.cu/" target="_blank">Ministerio de Ciencia, Tecnología y Medio Ambiente</a></h3>
                                <p class="site-description">
                                    El Ministerio de Ciencia, Tecnología y Medio Ambiente es el Organismo encargado de elaborar, proponer, dirigir y controlar la política del Estado y del Gobierno en materia de Ciencia, Tecnología e Innovación, Medio Ambiente                                </p>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                                     <li class="hidden-md" >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.redciencia.cu/" target="_blank">Red Cubana de la Ciencia</a></h3>
                                <p class="site-description">
                                    Ofrece contenidos y servicios de información sobre la Ciencia en Cuba.                                </p>
                            </article>
                        </section>
                    </li>
                                                                                                                                         <li>
                        <section>
                            <h2>Eventos</h2>
                            <article class="event-item">
                                <div class="wrapper-event-img " >
                                    <img src="http://www.cuba.cu//imgs/event/slider_2_conmarca-1.gif" alt="Convención Internacional de Ciencia, Tecnología e Innovación 2018" title="Convención Internacional de Ciencia, Tecnología e Innovación 2018"/>
                                </div>
                                <h3>
                                    <a href="http://www.convencioncienciacuba.cu/" target="_blank">Convención Internacional de Ciencia, Tecnología e Innovación 2018</a>
                                </h3>
                                <p class="hidden">
                                    El Citma y la Agencia de Energía Nuclear y Tecnologías de Avanzada (Aenta) del Citma, tienen el gusto de invitar a la Convención Internacional de...                                </p>
                            </article>
                        </section>
                    </li>
                                                                </ul>
                        </li>
                                                                                                         </ul>
             </li>
                                                                                            <li class="hidden" data-category="deporte" data-item-count="1">
                <header>en: <span>Deporte</span></header>
                <ul>
                                                                                                            <li  >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=183&subserie=105" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/beisbol-cuban-dvd_menu_thumb.jpg" alt="Momentos del béisbol cubano" title="Momentos del béisbol cubano"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=175&subserie=26" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/ajedrez_menu_thumb.jpg" alt="Aprende a jugar ajedrez" title="Aprende a jugar ajedrez"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=182&subserie=105" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/voli-cuba_menu_thumb.jpg" alt="Confesiones Voleibol cubano" title="Confesiones Voleibol cubano"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                                                                                 </ul>
                     </li>
                                </ul>
             </li>
                                                                                            <li class="hidden" data-category="turismo" data-item-count="3">
                <header>en: <span>Turismo</span></header>
                <ul>
                                                                            <li  >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://reservahoteles.cuba.cu/" target="_blank">Reserva de hoteles en Cuba</a></h3>
                                <p class="site-description">
                                    Usted podrá reservar en diversos hoteles con opciones de precios, numerosos destinos, áreas de la ciudad, el campo o la playa y tiene la posibilidad de escoger el tipo de hotel en que desea hospedarse. Los hoteles se encuentran en las plazas turísticas de La Habana Vieja, Cayo Largo, Varadero, Cayo Coco, Cayo Guillermo, Cayo Santa María, Trinidad, Santiago de Cuba y más....                                </p>
                            </article>
                        </section>
                    </li>
                                                                                                 <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.cuba.cu/reserva/flight/index.php/es" target="_blank">Reservas de vuelos en Cuba</a></h3>
                                <p class="site-description">
                                    Reserva de vuelos económicos a Cuba para usted y toda su familia. Los mejores precios y atención personalizada. Visítenos!                                </p>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                                                                     <li class="hidden-md" >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=224" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/sitios-buceo-cuba1_menu_thumb.jpg" alt="Sitios de buceo" title="Sitios de buceo"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=225" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/302_menu_thumb.jpg" alt=" Audiovisual Recorra Santiago" title=" Audiovisual Recorra Santiago"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=207&subserie=39" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/cuba1_menu_thumb.jpg" alt="Recorra Cuba. La Habana Vieja" title="Recorra Cuba. La Habana Vieja"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                                                                                 </ul>
                     </li>
                                </ul>
             </li>
                                                                                            <li class="hidden" data-category="salud" data-item-count="5">
                <header>en: <span>Salud</span></header>
                <ul>
                                                              <li  >
                            <ul class="category-item-list item-list-megamenu">
                                                  <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/salud/2018-01-02/cuba-logra-su-mas-baja-tasa-de-mortalidad-infantil-40/39486">
                                                 Cuba logra su más baja tasa de mortalidad infantil: 4,0                                             </a>
                                         </h3>
                                         <p>
                                                                                          Si decimos que por décimo año consecutivo Cuba mantiene su tasa de mortalidad infantil por debajo de cinco por cada mil nacidos vivos, ya es razón...                                         </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                                                                             <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/salud/2018-02-02/cuba-expone-en-naciones-unidas-avances-en-la-atencion-al-adulto-mayor-/39980">
                                                 Cuba expone en Naciones Unidas avances en la atención al adulto mayor                                              </a>
                                         </h3>
                                         <p>
                                                                                          Casi el 20% de los cubanos se encuentra en un rango etario mayor de 60 años; mientras que para  el 2025, la población de la tercera edad ascenderá...                                         </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                 </ul>
                         </li>
                                                                                                         <li class="hidden-md" >
                            <ul class="category-item-list item-list-megamenu">
                                                  <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/salud/2018-02-03/un-mundo-sin-cancer-para-2025-nosotros-podemos/40005">
                                                 Un mundo sin cáncer para 2025: Nosotros podemos                                             </a>
                                         </h3>
                                         <p>
                                                                                          La Habana, (PL) Cada año mueren de cáncer 8,8 millones de personas, en su mayoría en los países de ingresos bajos y medianos. Con tales cifras,...                                         </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                                                                             <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/salud/2018-02-19/vuelve-¨salud-para-todos¨/40272">
                                                 Vuelve ¨Salud para Todos¨                                             </a>
                                         </h3>
                                         <p>
                                                                                          Pabexpo acogerá nuevamente la XIV Feria Internacional Salud para Todos del 23 al 27 de abril próximo.                                          </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                 </ul>
                         </li>
                                                                                                                                           </ul>
             </li>
                                                                                            <li class="hidden" data-category="educacion" data-item-count="1">
                <header>en: <span>Educación</span></header>
                <ul>
                                                                                                            <li  >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=77" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/ingles-ninos_menu_thumb.jpg" alt="DVD Holiday English (Inglés para niños)" title="DVD Holiday English (Inglés para niños)"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=149" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/calculo-diferencial_menu_thumb.jpg" alt="Multimedia Cálculo diferencial e integral" title="Multimedia Cálculo diferencial e integral"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=309" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/CDIocusQuimito1_menu_thumb.jpg" alt="Juegos Didácticos Educativos" title="Juegos Didácticos Educativos"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                                                                                 </ul>
                     </li>
                                </ul>
             </li>
                                                                                            <li class="hidden" data-category="cultura" data-item-count="3">
                <header>en: <span>Cultura</span></header>
                <ul>
                                                              <li  >
                            <ul class="category-item-list item-list-megamenu">
                                                  <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/cultura/2018-02-12/se-vendieron-mas-de-350-mil-libros-en-la-fase-habanera-de-la-feria-del-libro/40146">
                                                 Se vendieron más de 350 mil libros en la fase habanera de la Feria del Libro                                             </a>
                                         </h3>
                                         <p>
                                                                                          Los visitantes de la 27 Feria Internacional del Libro (FIL) celebrada en esta capital adquirieron 353 mil ejemplares, reveló hoy el presidente del...                                         </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                                                                                   <li>
                        <section>
                            <h2>Sitios Cubanos</h2>
                            <article class="site-item">
                                <h3><a href="http://www.fac.cu/" target="_blank">Fábrica de Arte Cubano</a></h3>
                                <p class="site-description">
                                    Proyecto artístico impulsado por la necesidad de rescatar, apoyar y promocionar la obra de artistas de todas las ramas del arte.                                </p>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                                                                     <li class="hidden-md" >
                        <ul class="category-item-list item-list-megamenu">
                                        <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=54" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/optica_menu_thumb.jpg" alt="Audiolibro Óptica sin ver" title="Audiolibro Óptica sin ver"/></a>
                                                                                                                                                                                                                                                <a href="http://www.libreriavirtualcuba.com/productos.php?producto=29" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/Poemario_Nicolas_Guillen_menu_thumb.jpg" alt="Poemario Nicolás Guillén" title="Poemario Nicolás Guillén"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                                                                                 </ul>
                     </li>
                                </ul>
             </li>
                                                                                            <li class="hidden" data-category="economia" data-item-count="2">
                <header>en: <span>Economía</span></header>
                <ul>
                                                              <li  >
                            <ul class="category-item-list item-list-megamenu">
                                                  <li>
                                 <section>
                                     <h2>Noticias</h2>
                                     <article class="news-img">
                                         <h3>
                                             <a href="http://www.cuba.cu/economia/2018-02-06/empresarios-del-mundo-apuestan-por-la-zona-especial-de-desarrollo-mariel-fotos-e-infografia/40044">
                                                 Empresarios del mundo apuestan por la Zona Especial de Desarrollo Mariel (Fotos e Infografía)                                             </a>
                                         </h3>
                                         <p>
                                                                                          Con una posición privilegiada al oeste de La Habana, la Zona Especial de Desarrollo Mariel avanza poco a poco en el propósito de ser “referente...                                         </p>
                                         <div class="clearfix"></div>
                                     </article>
                                 </section>
                             </li>
                                                                                                                                                   <li>
                        <section>
                            <h2>Productos</h2>
                            <article class="product-item">
                                <div class="wrapper-product-img">
                                                                                                                                                                                                        <a href="http://www.libreriavirtualcuba.com/productos.php?producto=127" target="_blank"><img src="http://www.cuba.cu/imgs/product/thumbs/economia-ambiental-ebook_menu_thumb.jpg" alt="Ebook  Economía ambiental" title="Ebook  Economía ambiental"/></a>
                                                                                                            </div>
                                <div class="clearfix"></div>
                            </article>
                        </section>
                    </li>
                                            </ul>
                        </li>
                                                                                        </ul>
             </li>
                    </ul>
        </li>
        <li class="clearfix"></li>
    </li>
    <li style="clear: both;"></li>
</ul>
                        </li>
                        <li id="site-megamenu" class="dropdown pcuba-megamenu">
                            <a href="#" class="dropdown-toggle " data-toggle="dropdown">directorio de sitios <span></span></a>
                            <ul class="dropdown-menu dropdown-megamenu
    one-col    ">
    <li class="section-megamenu">
        <ul class="category-list">
                            <li class="category-row" data-category="medio-ambiente" data-category-id="12" ><a href="http://www.cuba.cu/sites/medio-ambiente"><span class="category-name">Medio Ambiente</span><span class="badge">30</span></a></li>
                            <li class="category-row" data-category="politica" data-category-id="6" ><a href="http://www.cuba.cu/sites/politica"><span class="category-name">Política</span><span class="badge">714</span></a></li>
                            <li class="category-row" data-category="ciencia-y-tecnologia" data-category-id="1" ><a href="http://www.cuba.cu/sites/ciencia-y-tecnologia"><span class="category-name">Ciencia y Tecnología</span><span class="badge">99</span></a></li>
                            <li class="category-row" data-category="deporte" data-category-id="3" ><a href="http://www.cuba.cu/sites/deporte"><span class="category-name">Deporte</span><span class="badge">18</span></a></li>
                            <li class="category-row" data-category="turismo" data-category-id="10" ><a href="http://www.cuba.cu/sites/turismo"><span class="category-name">Turismo</span><span class="badge">36</span></a></li>
                            <li class="category-row" data-category="salud" data-category-id="9" ><a href="http://www.cuba.cu/sites/salud"><span class="category-name">Salud</span><span class="badge">42</span></a></li>
                            <li class="category-row" data-category="educacion" data-category-id="4" ><a href="http://www.cuba.cu/sites/educacion"><span class="category-name">Educación</span><span class="badge">28</span></a></li>
                            <li class="category-row" data-category="cultura" data-category-id="2" ><a href="http://www.cuba.cu/sites/cultura"><span class="category-name">Cultura</span><span class="badge">72</span></a></li>
                            <li class="category-row" data-category="medios-de-comunicacion" data-category-id="8" ><a href="http://www.cuba.cu/sites/medios-de-comunicacion"><span class="category-name">Medios de comunicación</span><span class="badge">77</span></a></li>
                            <li class="category-row" data-category="economia" data-category-id="5" ><a href="http://www.cuba.cu/sites/economia"><span class="category-name">Economía</span><span class="badge">50</span></a></li>
                            <li class="category-row" data-category="marco-juridico" data-category-id="7" ><a href="http://www.cuba.cu/sites/marco-juridico"><span class="category-name">Marco jurídico</span><span class="badge">24</span></a></li>
                    </ul>
    </li>
    <li class="section-megamenu summary-megamenu">
        <ul>
                                        <!--
                <br/>
                -->                <li class="show-menu" data-category="medio-ambiente" data-site-count="2">
                    <h2>en: <span>Medio Ambiente</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.redciencia.cu/cdorigen/arca/protegefam.html" target="_blank">Serie educativa Protege a tu familia del impacto de fenómenos naturales</a></h3>
                                            <p class="site-description">
                                                Estos folletos e infografías han sido elaborados como parte de una campaña de educación comunitaria, destinada a enseñar al hombre a protegerse de los eventos naturales de origen geológico y climático.                                             </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cenais.cu/" target="_blank">Centro Nacional de Investigaciones Sismológicas, CENAIS</a></h3>
                                            <p class="site-description">
                                                Centro contribuyente a la mitigación del riesgo sísmico en la República de Cuba.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="politica" data-site-count="3">
                    <h2>en: <span>Política</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.icap.cu/" target="_blank">Instituto Cubano de Amistad con los Pueblos, ICAP</a></h3>
                                            <p class="site-description">
                                                Por la amistad entre los pueblos del mundo.Este sitio es un espacio contra el bloqueo informativo,  que se propone ofrecer a sus lectores la realidad de Cuba y el mundo. Es una publicación comprometida con la verdad, el sentir, las tradiciones y las luchas de los pueblos. Es un granito de arena en el camino hacia la integración de la humanidad.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.parlamentocubano.cu" target="_blank">Asamblea Nacional del Poder Popular</a></h3>
                                            <p class="site-description">
                                                Organo Supremo del poder del Estado en Cuba                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cubadebate.cu/noticias/2017/01/25/raul-castro-nunca-ha-sido-mas-necesario-marchar-por-el-camino-de-la-unidad/#.WIpBVUbLEdc" target="_blank">Raúl Castro: Nunca ha sido más necesario marchar por el camino de la unidad </a></h3>
                                            <p class="site-description">
                                                Discurso pronunciado por el General de Ejército Raúl Castro Ruz, Presidente de los Consejos de Estado y de Ministros de la república de Cuba, en la V Cumbre de la CELAC, en Punta Cana, República Dominicana, el 25 de enero de 2017, “Año 59 de la Revolución”.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="ciencia-y-tecnologia" data-site-count="3">
                    <h2>en: <span>Ciencia y Tecnología</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.redciencia.cu/" target="_blank">Red Cubana de la Ciencia</a></h3>
                                            <p class="site-description">
                                                Ofrece contenidos y servicios de información sobre la Ciencia en Cuba.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.citma.gob.cu/" target="_blank">Ministerio de Ciencia, Tecnología y Medio Ambiente</a></h3>
                                            <p class="site-description">
                                                El Ministerio de Ciencia, Tecnología y Medio Ambiente es el Organismo encargado de elaborar, proponer, dirigir y controlar la política del Estado y del Gobierno en materia de Ciencia, Tecnología e Innovación, Medio Ambiente                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.informaticahabana.cu/" target="_blank">XVII Convención y Feria Internacional Informática 2018</a></h3>
                                            <p class="site-description">
                                                A celebrarse entre los días del 19 al 23 de marzo de 2018, en el Palacio de Convenciones de La Habana, Cuba.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="deporte" data-site-count="2">
                    <h2>en: <span>Deporte</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.jit.cu/" target="_blank">Jit Online</a></h3>
                                            <p class="site-description">
                                                Publicación del Instituto Nacional de Deportes, Educación Física y Recreación INDER.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cubadeportes.cu/" target="_blank">Cubadeportes S.A.</a></h3>
                                            <p class="site-description">
                                                Cubadeportes S.A.es la empresa encargada de coordinar las acciones necesarias para el ofrecimiento de los servicios de asistencia técnica, colaboración e intercambio que pueda prestar el sistema deportivo cubano hacia  cualquier parte del mundo.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="turismo" data-site-count="3">
                    <h2>en: <span>Turismo</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://reservahoteles.cuba.cu/" target="_blank">Reserva de hoteles en Cuba</a></h3>
                                            <p class="site-description">
                                                Usted podrá reservar en diversos hoteles con opciones de precios, numerosos destinos, áreas de la ciudad, el campo o la playa y tiene la posibilidad de escoger el tipo de hotel en que desea hospedarse. Los hoteles se encuentran en las plazas turísticas de La Habana Vieja, Cayo Largo, Varadero, Cayo Coco, Cayo Guillermo, Cayo Santa María, Trinidad, Santiago de Cuba y más....                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cuba.cu/reserva/flight/index.php/es" target="_blank">Reservas de vuelos en Cuba</a></h3>
                                            <p class="site-description">
                                                Reserva de vuelos económicos a Cuba para usted y toda su familia. Los mejores precios y atención personalizada. Visítenos!                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cuba.cu/reserva/car/index.php/es" target="_blank">Reserva de autos en Cuba</a></h3>
                                            <p class="site-description">
                                                Reserva online de autos en Cuba. No pierda la oportunidad de reservar desde la comodidad de su hogar un auto en Cuba y planificar una estancia agradable. Visítenos!                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="salud" data-site-count="2">
                    <h2>en: <span>Salud</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.finlay.sld.cu/" target="_blank">Instituto Finlay. Centro de Investigación-Producción de vacunas</a></h3>
                                            <p class="site-description">
                                                Organización Científica dedicada a la Investigación y producción de vacunas. Es una institución al servicio de la salud, heredera y continuadora de más de cien años de tradición científica. Está considerada en su rama como una de las de mayor prestigio y reconocimiento a nivel internacional.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.smcsalud.cu/smc/es" target="_blank">Servicios Médicos Cubanos</a></h3>
                                            <p class="site-description">
                                                El portal Servicios Médicos Cubanos ofrece información sobre los servicios de salud en Cuba y los principales programas de atención médica, así como los procedimientos para acceder a ellos.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="educacion" data-site-count="2">
                    <h2>en: <span>Educación</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.uh.cu/" target="_blank">Universidad de La Habana</a></h3>
                                            <p class="site-description">
                                                Provee información noticiosa sobre lo que ocurre y es relevante en la  Universidad de La  Habana. Ofrece información sobre los planes y modalidades de estudio en Pregrado y Posgrado de la Universidad de La Habana. Promociona los productos y servicios académicos de la Universidad de La Habana. Brinda información noticiosa a los periodistas e instituciones mediáticas sobre la labor institucional y extensionista de la Universidad de La Habana. Contribuye al desarrollo de la cooperación y las relaciones internacionales de la Universidad de La Habana.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.mes.gob.cu/" target="_blank">Ministerio de Educación Superior, MES</a></h3>
                                            <p class="site-description">
                                                Es el organismo rector de las universidades cubanas. Dirige, orienta, controla y ejecuta la política en materia de enseñanza universitaria, tanto en la preparación de profesionales, como en la educación postgraduada.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="cultura" data-site-count="1">
                    <h2>en: <span>Cultura</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.cubacine.cult.cu/sitios/actores/index.htm" target="_blank">Actores y actrices del Cine cubano</a></h3>
                                            <p class="site-description">
                                                Informacion sobre la Cinematografía latinoamericana y cubana. Sitio de búsqueda y localización de actrices y actores de cine cubano.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="medios-de-comunicacion" data-site-count="2">
                    <h2>en: <span>Medios de comunicación</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.radiocubana.cu" target="_blank">Portal de la Radio Cubana en Internet</a></h3>
                                            <p class="site-description">
                                                Uno de los principales objetivos del Sistema de Radio Cubana es que la programación transmitida por sus emisoras estén al servicio de los conocimientos, la cultura, la recreación y la defensa de la nación y de todos sus ciudadanos. Participa, al igual que el resto de los más importantes medios masivos de comunicación, en el desarrollo de Misiones Informativas en varias regiones del mundo donde los cubanos hacen historia.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="http://www.prensa-latina.cu/" target="_blank">Prensa Latina</a></h3>
                                            <p class="site-description">
                                                Prensa Latina S.A. Agencia Informativa Latinoamericana con sede en La Habana, Cuba, ofrece las últimas informaciones sobre Cuba, latinoamérica y el resto del mundo.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="economia" data-site-count="0">
                    <h2>en: <span>Economía</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                                                                                              </ul>
                        </li>
                                        </ul>
                </li>
                            <!--
                <br/>
                -->                <li class="hidden hidden-menu" data-category="marco-juridico" data-site-count="0">
                    <h2>en: <span>Marco jurídico</span></h2>
                    <ul>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                                                            <li>
                                    <section>
                                        <article class="site-item">
                                            <h3><a href="https://www.gacetaoficial.gob.cu/" target="_blank">Gaceta Oficial de la República de Cuba</a></h3>
                                            <p class="site-description">
                                                Órgano oficial de publicidad normativa, constitucionalmente reconocido. El Ministerio de Justicia es el responsable de su publicación y es el custodio de su archivo central.                                            </p>
                                        </article>
                                    </section>
                                </li>
                                                        </ul>
                        </li>
                                            <li>
                            <ul class="site-list item-list-megamenu">
                                                        </ul>
                        </li>
                                        </ul>
                </li>
                        <div class="clearfix"></div>
        </ul>
        <!--<a href="" class="btn-see-more">VER M�?S</a>-->
    </li>
    <li style="clear: both;"></li>
</ul>
                        </li>
                        <li id="service-megamenu" class="dropdown pcuba-megamenu">
                            <a href="#" class="dropdown-toggle" data-toggle="dropdown">servicios <span></span></a>
                            <ul class="dropdown-menu dropdown-megamenu dropdown-menu-right">
                                <li class="section-megamenu">
    <ul class="category-list">
                    <li class="category-row" data-service-id="24"><a href="http://cocina.cuba.cu/" target="_blank"><span class="category-name">Cocina Cubana</span></a></li>
                    <li class="category-row" data-service-id="23"><a href="http://www.cuba.cu/mipais/" target="_blank"><span class="category-name">Mi País</span></a></li>
                    <li class="category-row" data-service-id="34"><a href="http://tiempo.cuba.cu/" target="_blank"><span class="category-name">El Tiempo</span></a></li>
                    <li class="category-row" data-service-id="30"><a href="http://www.cursosenlineauvc.com/" target="_blank"><span class="category-name">Cursos en línea</span></a></li>
                    <li class="category-row" data-service-id="26"><a href="http://www.redciencia.cu/" target="_blank"><span class="category-name">Red cubana de la Ciencia </span></a></li>
                    <li class="category-row" data-service-id="33"><a href="http://efemerides.cuba.cu/" target="_blank"><span class="category-name">Efemérides</span></a></li>
                    <li class="category-row" data-service-id="35"><a href="http://paginasamarillas.cuba.cu" target="_blank"><span class="category-name">Páginas amarillas</span></a></li>
                    <li class="category-row" data-service-id="28"><a href="http://consultas.cuba.cu" target="_blank"><span class="category-name">Consultas médicas</span></a></li>
                    <li class="category-row" data-service-id="27"><a href="http://www.cubanic.cu/" target="_blank"><span class="category-name">Cubanic</span></a></li>
                    <li class="category-row" data-service-id="36"><a href="http://citmatel.cu/noticias/nuevo-servicio-para-mejorar-el-posicionamiento-en-internet" target="_blank"><span class="category-name">Posicionamiento en Internet</span></a></li>
            </ul>
</li>
<li class="section-megamenu summary-megamenu">
    <ul class="service-list">
                    <li class=" wrapper-abstract-service" data-service-id="23">
                <section class="abstract-service">
                    <h1><span>en: </span>Mi País</h1>
                    <p class="description-service">Información cubana de gran interés histórico-cultural, científica, económica, entre otras temáticas.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://www.cuba.cu/mipais/" target="_blank" >Mi País</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/mipais-logo_menu_thumb.jpg" alt="Servicio Mi País" title="Servicio Mi País"/>
                                                <p class="summary-service">Ciencia, cultura, religión, salud, educación, economía, historia y mucho más sobre Cuba.</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="24">
                <section class="abstract-service">
                    <h1><span>en: </span>Cocina Cubana</h1>
                    <p class="description-service">Interesantes recetas validadas por chef cubanos de gran prestigio internacional.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://cocina.cuba.cu/" target="_blank" >Cocina Cubana</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/cocina-cub5_menu_thumb.png" alt="Cocina cubana" title="Cocina cubana"/>
                                                <p class="summary-service">Si tuviera que preparar un menú bien cubano, ¿qué platos escogería? En este sitio encontrará interesantes ideas de cómo prepararlos.</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="26">
                <section class="abstract-service">
                    <h1><span>en: </span>Red cubana de la Ciencia </h1>
                    <p class="description-service">Noticias y acceso a sitios de la ciencia cubana.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://www.redciencia.cu/" target="_blank" >Red de la Ciencia </a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/logo2015_menu_thumb.jpg" alt="Red de la ciencia" title="Red de la ciencia"/>
                                                <p class="summary-service">Portal de la Red Cubana de la Ciencia</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="27">
                <section class="abstract-service">
                    <h1><span>en: </span>Cubanic</h1>
                    <p class="description-service">Servicio de registro de nombres de dominio bajo .cu</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://www.cubanic.cu/" target="_blank" >Cubanic</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/nic_menu_thumb.jpg" alt="" title=""/>
                                                <p class="summary-service">Nuevas Normas para el otorgamiento de nombres de dominio bajo .cu. Visítenos!!!</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="28">
                <section class="abstract-service">
                    <h1><span>en: </span>Consultas médicas</h1>
                    <p class="description-service">Síntomas, prevención, control y consejos sobre cómo tratar un gran número de enfermedades.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://consultas.cuba.cu" target="_blank" >Consultas médicas</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/consultas2_menu_thumb.jpg" alt="Consultas médicas" title="Consultas médicas"/>
                                                <p class="summary-service">No deje de leer: Disminuir el sedentarismo beneficia el control de la diabetes tipo 2</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="30">
                <section class="abstract-service">
                    <h1><span>en: </span>Cursos en línea</h1>
                    <p class="description-service">Cursos de diversas temáticas certificados por reconocidas instituciones académicas cubanas.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://www.cursosenlineauvc.com/" target="_blank" >Cursos con matrícula abierta</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/banner-cursoenlinea1_menu_thumb.jpg" alt="Cursos en línea" title="Cursos en línea"/>
                                                <p class="summary-service">De forma didáctica y amena, contenidos de interés para tu vida profesional y laboral.</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="33">
                <section class="abstract-service">
                    <h1><span>en: </span>Efemérides</h1>
                    <p class="description-service">Infórmese de los acontecimientos más notables de Cuba y el resto del mundo.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://efemerides.cuba.cu/" target="_blank" >Efemérides</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/efemerides-pcuba2_menu_thumb.jpg" alt="Efemérides" title="Efemérides"/>
                                                <p class="summary-service">Subscríbase a nuestro boletín para recibir en su correo diariamente los acontecimientos más notables de Cuba y el resto del mundo. </p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="34">
                <section class="abstract-service">
                    <h1><span>en: </span>El Tiempo</h1>
                    <p class="description-service">Manténgase informado sobre el estado del tiempo en Cuba.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://tiempo.cuba.cu/" target="_blank" >El Tiempo</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/logo-tiempo_menu_thumb.jpg" alt="El tiempo en Cuba" title="El tiempo en Cuba"/>
                                                <p class="summary-service">Manténgase informado sobre el estado del tiempo en Cuba.</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="35">
                <section class="abstract-service">
                    <h1><span>en: </span>Páginas amarillas</h1>
                    <p class="description-service">Directorio de instituciones cubanas.</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://paginasamarillas.cuba.cu" target="_blank" >Páginas amarillas</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/paginas-amarillas_menu_thumb.png" alt="Páginas amarillas " title="Páginas amarillas "/>
                                                <p class="summary-service">Datos de contacto de las instituciones cubanas destinadas a facilitar al viajero su estancia en nuestro país.</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
                    <li class="hidden wrapper-abstract-service" data-service-id="36">
                <section class="abstract-service">
                    <h1><span>en: </span>Posicionamiento en Internet</h1>
                    <p class="description-service">Posicione la web de su empresa en las primeras posiciones de los buscadores ¡Hazte visible!</p>
                    <article>
                        <h2 class="title-service"><a class="link-service" href="http://citmatel.cu/noticias/nuevo-servicio-para-mejorar-el-posicionamiento-en-internet" target="_blank" >Posicionamiento en Internet</a></h2>
                                                                                                    <img class="img-service" src="http://www.cuba.cu/imgs/service/thumbs/seo-nuevo-servicio1_menu_thumb.jpg" alt="Servicio SEO" title="Servicio SEO"/>
                                                <p class="summary-service">Hazte visible!!!</p>
                    </article>
                    <!--<nav>
                    <a href="" target="_blank" class="btn-see-more">VER MÁS</a>
                </nav>-->
                </section>
            </li>
            </ul>
</li>
<li style="clear: both;"></li>

                            </ul>
                        </li>
                        <!--<li id="news-megamenu" class="dropdown pcuba-megamenu">
                            <a href="#" class="dropdown-toggle " data-toggle="dropdown">noticias <span></span></a>
                                                        </li>-->
                        <li id="shopping-megamenu" class="dropdown mega-dropdown pcuba-megamenu">
                            <a href="#" class="dropdown-toggle" data-toggle="dropdown">compras <span></span></a>
                            <ul class="dropdown-menu dropdown-megamenu dropdown-menu-right">
                                
<li class="section-megamenu">
    <ul class="category-list">
                            <li class="category-row"><a href="http://moviles.compra-dtodo.com/" class="pointer pointed" data-position="0" target="_blank" ><span class="category-name">Móviles CompraDtodo</span></a></li>
                                <li class="category-row"><a href="http://www.compraspacuba.com/" class="" data-position="1" target="_blank" ><span class="category-name">Compraspacuba</span></a></li>
                                <li class="category-row"><a href="http://www.libreriavirtualcuba.com/" class="" data-position="2" target="_blank" ><span class="category-name">Librería virtual</span></a></li>
                                <li class="category-row"><a href="http://www.administresunegocio.cu/" class="" data-position="3" target="_blank" ><span class="category-name">Administre su negocio</span></a></li>
                                <li class="category-row"><a href="http://www.editorialcitmatel.cu/" class="" data-position="4" target="_blank" ><span class="category-name">Editorial Citmatel</span></a></li>
                                <li class="category-row"><a href="http://www.rodasxxi.cu/" class="" data-position="5" target="_blank" ><span class="category-name">Rodas XXI</span></a></li>
                                <li class="category-row"><a href="http://www.libreriavirtualcuba.com/productos.php?producto=91" class="" data-position="6" target="_blank" ><span class="category-name">Tras La Huella</span></a></li>
                        </ul>
</li>
<li class="section-megamenu summary-megamenu">
<ul class="shopping-list">
                                    <li>
            <article id="shopping-0" data-shopping="shopping-0" data-position="0" class="shopping-item shopping-toggled">
                <h1>Móviles CompraDtodo</h1>
                <div id="shopping-0-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                    <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/ofertamarzo-848_menu_thumb.jpg" alt="Recargas de saldo " title="Recargas de saldo ">
                        
                        <h2>Recarga de saldo en Compra Dtodo.com</h2>
                        <nav>
                            <a href="http://moviles.compra-dtodo.com/" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                                                                                                                                                                                                                                                                </div>
        <!-- Left and right controls -->
                        </div>
            </article>
        </li>
                                        <li>
            <article id="shopping-1" data-shopping="shopping-1" data-position="1" class="shopping-item ">
                <h1>Compraspacuba</h1>
                <div id="shopping-1-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

                                <!-- Indicators -->
            <ol class="carousel-indicators">
                                                                            <li data-target="#shopping-1-carousel" data-slide-to="0" class="active"></li>
                                                                                    <li data-target="#shopping-1-carousel" data-slide-to="1" class=""></li>
                                                                                    <li data-target="#shopping-1-carousel" data-slide-to="2" class=""></li>
                                                                                                                                                                                                                                                                                                                                                    </ol>
        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                        <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/mantenerpc_menu_thumb.jpg" alt="Servicio de reparación de PC" title="Servicio de reparación de PC">
                        
                        <h2>Mantenimiento y Reparación de computadoras y accesorios</h2>
                        <nav>
                            <a href="http://www.bazar-virtual.ca/departamento.php?id_dpto=541" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/Excursiones_menu_thumb.jpg" alt="Reserva de excusiones desde el exterior" title="Reserva de excusiones desde el exterior">
                        
                        <h2>Excelentes ofertas para disfrutar en familia.</h2>
                        <nav>
                            <a href="http://www.compraspacuba.com/departamento.php?id_dpto=198" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/1_719_Servicio_de_tarjetas_de_regalos_jpg_menu_thumb.png" alt="Tarjetas de regalo" title="Tarjetas de regalo">
                        
                        <h2>La mejor opción para comprar servicios y productos en toda Cuba.</h2>
                        <nav>
                            <a href="http://www.bazar-virtual.ca/tarjetas.php" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                                                                                                                                                                                                    </div>
        <!-- Left and right controls -->
                    <a class="left carousel-control" href="#shopping-1-carousel" role="button" data-slide="prev">
                    <span class="glyphicon glyphicon-chevron-left" aria-hidden="true"></span>
                    <span class="sr-only">Previous</span>
            </a>
            <a class="right carousel-control" href="#shopping-1-carousel" role="button" data-slide="next">
                    <span class="glyphicon glyphicon-chevron-right" aria-hidden="true"></span>
                    <span class="sr-only">Next</span>
            </a>
                        </div>
            </article>
        </li>
                                                                <li>
            <article id="shopping-2" data-shopping="shopping-2" data-position="2" class="shopping-item ">
                <h1>Librería virtual</h1>
                <div id="shopping-2-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

                                <!-- Indicators -->
            <ol class="carousel-indicators">
                                                                                                                                                                <li data-target="#shopping-2-carousel" data-slide-to="0" class="active"></li>
                                                                                    <li data-target="#shopping-2-carousel" data-slide-to="1" class=""></li>
                                                                                    <li data-target="#shopping-2-carousel" data-slide-to="2" class=""></li>
                                                                                    <li data-target="#shopping-2-carousel" data-slide-to="3" class=""></li>
                                                                                    <li data-target="#shopping-2-carousel" data-slide-to="4" class=""></li>
                                                                                    <li data-target="#shopping-2-carousel" data-slide-to="5" class=""></li>
                                                                                                                                                                            </ol>
        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                                                                                    <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/libreria11_menu_thumb.jpg" alt="Libros cubanos, ebooks, literatura y más. Visítanos!" title="Libros cubanos, ebooks, literatura y más. Visítanos!">
                        
                        <h2>Interesantes propuestas de contenidos cubanos.</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/Audiolibros2018_menu_thumb.jpg" alt="Banner de los audiolibros" title="Banner de los audiolibros">
                        
                        <h2>Una opción más!</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/productos.php?serie=21" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/apk-cuba_menu_thumb.jpg" alt="APK cubanas" title="APK cubanas">
                        
                        <h2>Novedosos contenidos cubanos en su móvil.</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/index.php" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/juegosdidacticos_menu_thumb.jpg" alt="Productos infantiles" title="Productos infantiles">
                        
                        <h2>Aprenda jugando.</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/index.php" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/juegos_menu_thumb.jpg" alt="Aplicaciones para móviles" title="Aplicaciones para móviles">
                        
                        <h2>Un apoyo ameno y divertido para ejercitar los conocimientos matemáticos.</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/productos.php?producto=305" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                <div class="item ">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/Imagen2_menu_thumb.png" alt="Nuevos contenidos Librería Virtual" title="Nuevos contenidos Librería Virtual">
                        
                        <h2>Nuevos contenidos de diversas temáticas y variados formatos y soportes. </h2>
                        <nav>
                            <a href="http://www.editorialcitmatel.cu/" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                                                                            </div>
        <!-- Left and right controls -->
                    <a class="left carousel-control" href="#shopping-2-carousel" role="button" data-slide="prev">
                    <span class="glyphicon glyphicon-chevron-left" aria-hidden="true"></span>
                    <span class="sr-only">Previous</span>
            </a>
            <a class="right carousel-control" href="#shopping-2-carousel" role="button" data-slide="next">
                    <span class="glyphicon glyphicon-chevron-right" aria-hidden="true"></span>
                    <span class="sr-only">Next</span>
            </a>
                        </div>
            </article>
        </li>
                                                                                                    <li>
            <article id="shopping-3" data-shopping="shopping-3" data-position="3" class="shopping-item ">
                <h1>Administre su negocio</h1>
                <div id="shopping-3-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                                                                                                                                                                                                            <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/administre_menu_thumb.png" alt="Administre su negocio" title="Administre su negocio">
                        
                        <h2>¡No pierda tiempo sacando cuentas y optimice su tiempo!</h2>
                        <nav>
                            <a href="http://www.administresunegocio.cu/asn/index" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                                                        </div>
        <!-- Left and right controls -->
                        </div>
            </article>
        </li>
                                        <li>
            <article id="shopping-4" data-shopping="shopping-4" data-position="4" class="shopping-item ">
                <h1>Editorial Citmatel</h1>
                <div id="shopping-4-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                                                                                                                                                                                                                                <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/editorial-citmatel_menu_thumb.jpg" alt="Editorial Citmatel" title="Editorial Citmatel">
                        
                        <h2>Contenidos cubanos de variadas temáticas.</h2>
                        <nav>
                            <a href="http://www.editorialcitmatel.cu" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                                    </div>
        <!-- Left and right controls -->
                        </div>
            </article>
        </li>
                                        <li>
            <article id="shopping-5" data-shopping="shopping-5" data-position="5" class="shopping-item ">
                <h1>Rodas XXI</h1>
                <div id="shopping-5-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                                                                                                                                                                                                                                                    <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/RodasXXI_menu_thumb.png" alt="Rodas XXI" title="Rodas XXI">
                        
                        <h2>Excelentes soluciones económicas que harán su trabajo rápido y viable.</h2>
                        <nav>
                            <a href="http://www.rodasxxi.cu/" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                                                </div>
        <!-- Left and right controls -->
                        </div>
            </article>
        </li>
                                        <li>
            <article id="shopping-6" data-shopping="shopping-6" data-position="6" class="shopping-item ">
                <h1>Tras La Huella</h1>
                <div id="shopping-6-carousel" class="carousel slide pcuba-carousel" data-ride="carousel">

        
                <!-- Wrapper for slides -->
        <div class="carousel-inner" role="listbox">
                                                                                                                                                                                                                                                                                                        <div class="item active">
                                                                                                    <img src="http://www.cuba.cu/imgs/offer/thumbs/TLH_menu_thumb.jpg" alt="Serie Policíaca Cubana" title="Serie Policíaca Cubana">
                        
                        <h2>Serie policíaca basada en hechos reales ocurridos en Cuba.</h2>
                        <nav>
                            <a href="http://www.libreriavirtualcuba.com/productos.php?producto=316" class="btn-see-more" target="_blank">VER MÁS</a>
                        </nav>
                </div>
                                            </div>
        <!-- Left and right controls -->
                        </div>
            </article>
        </li>
                  
    
</ul>
</li>
                            </ul>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm" style="clear: both"></li>
                        <li class="dropdown mega-dropdown hidden-xs menu-searcher">
                            <a href="" class="dropdown-toggle" data-toggle="dropdown" title="Buscador"></a>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm">
                            <p class="navbar-text menu-separator"></p>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm weather-menu">
                            <a href="#">El Tiempo Hoy</a>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm rate-menu">
                            <a href="">Cambio de Moneda</a>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm">
                            <p class="navbar-text menu-separator"></p>
                        </li>
                        <li class="hidden-lg hidden-md hidden-sm">
                            <ul class="external-link">
                                <li><a href="http://www.cuba.cu/feed" title="RSS"></a></li>
                                <li><a href="https://twitter.com/portalcuba" title="Twitter"></a></li>
                                <li><a href="https://www.facebook.com/portalcuba" title="Facebook"></a></li>
                                <div class="clearfix"></div>
                            </ul>
                        </li>
                    </ul>
                    <ul class="nav navbar-nav navbar-right hidden-xs hidden-sm">
                        <!--<li class="dropdown idiom-dropdown">
                            <a href="#" class="dropdown-toggle" data-toggle="dropdown" role="button" aria-haspopup="true" aria-expanded="false">ES <span></span></a>
                            <ul class="dropdown-menu">
                                <li><a href="#"><span class="style-idiom">English </span><span class="style-abbr">EN</span></a></li>
                            </ul>
                        </li>-->
                    </ul>
                </div><!-- /.nav-collapse -->
            </div>
        </nav>
        <header id="pcuba-header" class="container">
            <nav class="row header-nav" role="navigation">
                <div class="col-xs-12 col-md-9 col-sm-8 nav-info">
                    <p class="header-date">
                        <span class="header-day">17</span>
                        <span class="wrapper-month-year">
                            <span class="header-month">MAR.</span>
                            <span class="header-year">2018</span>
                        </span>
                    </p>
                    <p class="pass-time text-right">
                        <span>Actualizado </span><span>sábado <b>17 de marzo</b> de 2018</span>                    </p>
                </div>
                <div class="col-md-3 col-sm-4 hidden-xs nav-social">
                    <a href="mailto:webmaster@cuba.cu" class="nav-msg">
                        <img src="http://www.cuba.cu/imgs/msg.png" alt="message" title="Cont&aacute;ctenos"/>
                    </a>
                    <a href="http://www.cuba.cu/feed"  target="_blank" class="nav-rss fontAwesomeCaret" title="RSS" ></a>
                    <a href="https://twitter.com/portalcuba" target="_blank" class="nav-tw fontAwesomeCaret" title="Twitter"></a>
                    <a href="https://www.facebook.com/portalcuba" target="_blank" class="nav-fb fontAwesomeCaret" title="Facebook"></a>
                </div>
            </nav>
        </header>
        <div title="Subir" class="scrollup" onclick="scrollUp()"><span class="glyphicon glyphicon-menu-up" aria-hidden="true"></span></div><div id="pcuba-searcher" class="container-fluid hidden">
    <div class="row wrapper-btn-close" >
        <div class="col-lg-6 col-lg-offset-3 col-sm-8 col-sm-offset-2">
            <button type="button" class="btn-close"><span class="fontAwesomeCaret"></span></button>
        </div>
    </div>
    <div class="row">
        <div class=" col-lg-6 col-lg-offset-3 col-sm-8 col-sm-offset-2">
            <form method="get" class="searcher-form" action="http://www.cuba.cu/search">
                                <input type="hidden" name="csrf_test_name" value="cd5d6e57444715896bc6a7a1026d94fb" />
                <div class="row">
                    <div class="form-group col-sm-12">
                        <label for="patternInput"></label>
                        <input type="text" class="form-control" id="patternInput" name="patternInput" placeholder="Buscar...">
                    </div>
                </div>
                <div class="row">
                    <div class="dropdown searcher-dropdown col-sm-8">
                        <button class="btn btn-default dropdown-toggle" type="button" id="itemDropdownMenu" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                            <span class="">Todas las secciones</span>
                            <span class="fontAwesomeCaret"></span>
                        </button>
                        <input type="hidden" id="itemSelected" name="itemSelected" value="all"/>
                        <ul class="dropdown-menu dropdown-menu-right" aria-labelledby="itemDropdownMenu">
                            <li class="disabled" ><a href="#" data-item-slug="all">Todas las secciones</a></li>
                            <li><a href="#" data-item-slug="news">Noticias</a></li>
                            <li><a href="#" data-item-slug="site">Sitios</a></li>
                            <li><a href="#" data-item-slug="event" >Eventos</a></li>
                            <li><a href="#" data-item-slug="product">Productos</a></li>
                        </ul>
                    </div>
                    <div class="col-sm-4">
                        <button type="submit" class="btn btn-default searcher-submit fontAwesomeCaret"></button>
                    </div>
                </div>
            </form>
        </div>
    </div>
</div>

<div id="pcuba-weather" class="container-fluid hidden" >
    <div class="row">
        <div class="col-lg-1 col-lg-offset-10">
            <button type="button" class="btn-close"><span class="fontAwesomeCaret"></span></button>
        </div>
    </div>
    <article class="weather-forecast">
        <nav class="location-weather-forecast">
            <form action="">
                <div class="dropdown">
                    <button class="btn btn-default dropdown-toggle" type="button" id="location-dropdownMenu" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                        La Habana                        <span class="fontAwesomeCaret"></span>
                    </button>
                    <ul class="dropdown-menu" aria-labelledby="location-dropdownMenu">
                                                    <li >
                                <a href="#" data-province-id="1">Pinar del Río</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="16">Artemisa</a>
                            </li>
                                                    <li class="disabled">
                                <a href="#" data-province-id="2">La Habana</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="17">Mayabeque</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="4">Matanzas</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="5">Villa Clara</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="6">Cienfuegos</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="7">Sancti Spíritus</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="8">Ciego de Ávila</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="9">Camagüey</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="10">Las Tunas</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="12">Holguín</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="11">Granma</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="13">Santiago de Cuba</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="14">Guantánamo</a>
                            </li>
                                                    <li >
                                <a href="#" data-province-id="15">Isla de la Juventud</a>
                            </li>
                                            </ul>
                </div>
            </form>
        </nav>
        <section class="summary-weather-forecast">
            <p class="sky-weather">Soleado</p>
            <div class="wrapper-icon-temperature">
                <p class="fonticon-weather-forecast"></p>
                <div class="wrapper-temperature">
                    <p id="temperature-max" class="temperature">T. Máx. 27°C</p>
                    <p id="temperature-min" class="temperature">T. Mín. 19°C</p>
                </div>
                <div class="clearfix"></div>
            </div>
            <!--<p class="summary-forecast">
                    </p>-->
        </section>
        <nav >
            <a href="http://tiempo.cuba.cu/" target="_blank" class="see-more-weather">ver pron&oacute;stico completo</a>
        </nav>
    </article>
</div>
<div id="pcuba-rate" class="container-fluid hidden">
    <div class="row">
        <div class="col-lg-1 col-lg-offset-10">
            <button type="button" class="btn-close"><span class="fontAwesomeCaret"></span></button>
        </div>
    </div>
    <article class="rate-change">
        <h1 class="title-rate-change">Cambio de moneda</h1>
        <nav class="coin-type-rate-change">
            <form action="">
                <div class="dropdown">
                    <button class="btn btn-default dropdown-toggle" type="button" id="rate-change-dropdownMenu" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                        1 EUR                        <span class="fontAwesomeCaret"></span>
                    </button>
                    <ul class="dropdown-menu" aria-labelledby="rate-change-dropdownMenu">
                                                    <li class="disabled">
                                <a href="#" data-rate-id="1">1 EUR</a>
                            </li>
                                                    <li >
                                <a href="#" data-rate-id="2">1 GBP</a>
                            </li>
                                                    <li >
                                <a href="#" data-rate-id="3">1 CAD</a>
                            </li>
                                                    <li >
                                <a href="#" data-rate-id="4">1 JPY</a>
                            </li>
                                                    <li >
                                <a href="#" data-rate-id="5">1 MXN</a>
                            </li>
                                                    <li >
                                <a href="#" data-rate-id="6">1 USD</a>
                            </li>
                                            </ul>
                </div>
            </form>
        </nav>
        <section class="result-change">
            <h2 class="title-result-change">Equivale a:</h2>
            <p class="result">
                1.19777 <span>CUC</span>
            </p>
        </section>
        <nav class="nav-bank">
            <a href="http://www.bc.gob.cu/Espanol/tipo_cambio_METROPOLITANO.asp">Tasas de banco metropolitano</a>
        </nav>
    </article>
</div>
<div id="pcuba-home" class="container">
    <div class="row">
        <section id="main-section-home" class="col-md-9 col-sm-8">
            <section class="section-promotional">
                <div id="carousel-promotional" class="carousel slide" data-ride="carousel">
                    <!-- Indicators -->
                    <ol class="carousel-indicators">
                                                    <li data-target="#carousel-promotional" data-slide-to="0" class="active"></li>
                                                    <li data-target="#carousel-promotional" data-slide-to="1" class=""></li>
                                                    <li data-target="#carousel-promotional" data-slide-to="2" class=""></li>
                                                    <li data-target="#carousel-promotional" data-slide-to="3" class=""></li>
                                            </ol>
                    <!-- Wrapper for slides -->
                    <div class="carousel-inner" role="listbox">
                                                    <div class="item active item-promotion">
                                <div class="wrapper-carousel-img">
                                                                            <a href="http://moviles.compra-dtodo.com/" target="_blank">
                                            <img class="img-responsive" src="http://www.cuba.cu/imgs/promotion/ofertamarzo-848.jpg" alt="Recargas de saldo " title="Recargas de saldo ">
                                        </a>
                                    
                                </div>
                                <div class="carousel-promotional-caption hidden">
                                    <div class="col-sm-12 promotion-title"><a href="http://moviles.compra-dtodo.com/" target="_blank">none</a></div>
                                </div>
                            </div>
                                                    <div class="item  item-promotion">
                                <div class="wrapper-carousel-img">
                                                                            <a href="cultura/2018-02-23/ediciones-citmatel-presente-en-ferias-internacionales-del-libro/40359" target="_blank">
                                            <img class="img-responsive" src="http://www.cuba.cu/imgs/promotion/F-pc1.jpg" alt="Banner alegórico a la participación de Citmatel en las Ferias Internacionales del Libro" title="Banner alegórico a la participación de Citmatel en las Ferias Internacionales del Libro">
                                        </a>
                                    
                                </div>
                                <div class="carousel-promotional-caption hidden">
                                    <div class="col-sm-12 promotion-title"><a href="cultura/2018-02-23/ediciones-citmatel-presente-en-ferias-internacionales-del-libro/40359" target="_blank">none</a></div>
                                </div>
                            </div>
                                                    <div class="item  item-promotion">
                                <div class="wrapper-carousel-img">
                                                                            <a href="http://www.informaticahabana.cu/es/eventos/1859/page" target="_blank">
                                            <img class="img-responsive" src="http://www.cuba.cu/imgs/promotion/informatica2018.jpg" alt="XVII Convención de tecnologías y contenidos multimedia y Feria Internacional Informática 2018" title="XVII Convención de tecnologías y contenidos multimedia y Feria Internacional Informática 2018">
                                        </a>
                                    
                                </div>
                                <div class="carousel-promotional-caption hidden">
                                    <div class="col-sm-12 promotion-title"><a href="http://www.informaticahabana.cu/es/eventos/1859/page" target="_blank">none</a></div>
                                </div>
                            </div>
                                                    <div class="item  item-promotion">
                                <div class="wrapper-carousel-img">
                                                                            <a href="http://www.editorialcitmatel.cu/" target="_blank">
                                            <img class="img-responsive" src="http://www.cuba.cu/imgs/promotion/Imagen2.png" alt="Nuevos contenidos Librería Virtual" title="Nuevos contenidos Librería Virtual">
                                        </a>
                                    
                                </div>
                                <div class="carousel-promotional-caption hidden">
                                    <div class="col-sm-12 promotion-title"><a href="http://www.editorialcitmatel.cu/" target="_blank">none</a></div>
                                </div>
                            </div>
                                            </div>

                    <!-- Controls -->
                    <a class="left carousel-control" href="#carousel-promotional" role="button" data-slide="prev">
                        <span class="glyphicon glyphicon-chevron-left" aria-hidden="true"></span>
                        <span class="sr-only">Previous</span>
                    </a>
                    <a class="right carousel-control" href="#carousel-promotional" role="button" data-slide="next">
                        <span class="glyphicon glyphicon-chevron-right" aria-hidden="true"></span>
                        <span class="sr-only">Next</span>
                    </a>
                </div>
            </section>
            <section class="section-newsy row">
                <div class="col-sm-12">
                    <div class="section-news">
                        <div class="row">
    <div class="col-sm-12 col-md-5"><h1>noticias</h1></div>
    <nav class="col-sm-12 col-md-7">
        <div class="dropdown newsy-dropdownMenu">
            <button class="btn btn-default dropdown-toggle" type="button" id="dropdownMenu1" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                TODAS<span class="badge">476</span>                <span class="fontAwesomeCaret"></span>
            </button>
            <ul class="dropdown-menu" aria-labelledby="dropdownMenu1">
                                                                            <li >
                            <a href="" data-category="ciencia-y-tecnologia" data-per-page="16" data-page="1" >Ciencia y Tecnología<span class="badge">47</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="cultura" data-per-page="16" data-page="1" >Cultura<span class="badge">94</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="deporte" data-per-page="16" data-page="1" >Deporte<span class="badge">71</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="educacion" data-per-page="16" data-page="1" >Educación<span class="badge">15</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="economia" data-per-page="16" data-page="1" >Economía<span class="badge">27</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="politica" data-per-page="16" data-page="1" >Política<span class="badge">116</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="marco-juridico" data-per-page="16" data-page="1" >Marco jurídico<span class="badge">1</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="medios-de-comunicacion" data-per-page="16" data-page="1" >Medios de comunicación<span class="badge">2</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="salud" data-per-page="16" data-page="1" >Salud<span class="badge">23</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="turismo" data-per-page="16" data-page="1" >Turismo<span class="badge">22</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="otras" data-per-page="16" data-page="1" >Otras<span class="badge">10</span></a>
                        </li>
                                                                                <li >
                            <a href="" data-category="medio-ambiente" data-per-page="16" data-page="1" >Medio Ambiente<span class="badge">30</span></a>
                        </li>
                                                                                <li class="hidden">
                            <a href="" data-category="religion" data-per-page="16" data-page="1" >Religión</a>
                        </li>
                                                                                <li class="hidden">
                            <a href="" data-category="vii-congreso-del-partido" data-per-page="16" data-page="1" >VII Congreso del Partido</a>
                        </li>
                                                                                <li >
                            <a href="" data-category="sociedad" data-per-page="16" data-page="1" >Sociedad<span class="badge">18</span></a>
                        </li>
                                                </ul>
        </div>
    </nav>
</div>
<div class="news-list row">
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/salud">Salud</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/salud/2018-03-17/cuba-realizara-estudio-de-un-candidato-vacunal-contra-hepatocarcinoma/40698">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/images25_thumb.jpg" title="HeberSaVax" alt="HeberSaVax" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/salud/2018-03-17/cuba-realizara-estudio-de-un-candidato-vacunal-contra-hepatocarcinoma/40698">Cuba realizará estudio de un candidato vacunal contra hepatocarcinoma</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        Un ensayo clínico del candidato vacunal HeberSaVax realizará Cuba con el objetivo de garantizar la supervivencia de los enfermos de cáncer de...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/deporte">Deporte</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/deporte/2018-03-17/yulimar-rojas-sufre-accidente-de-transito-pero-sale-ilesa/40697">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/15-yulimar_thumb.jpg" title="Yulimar Rojas" alt="Yulimar Rojas" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/deporte/2018-03-17/yulimar-rojas-sufre-accidente-de-transito-pero-sale-ilesa/40697">Yulimar Rojas sufre accidente de tránsito, pero sale ilesa</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    La tricampeona mundial de triple salto, Yulimar Rojas, sufrió un accidente de trafico el viernes en Puerto La Cruz, Anzoátegui, cuando se dirigía...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/turismo">Turismo</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/turismo/2018-03-17/cuba-expone-sus-novedades-turisticas-en-salon-mundial-de-paris/40696">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/cubaturismof1_thumb.jpg" title="Cuba expone sus novedades turísticas en Salón Mundial de París" alt="Cuba expone sus novedades turísticas en Salón Mundial de París" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/turismo/2018-03-17/cuba-expone-sus-novedades-turisticas-en-salon-mundial-de-paris/40696">Cuba expone sus novedades turísticas en Salón Mundial de París</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        Numerosas empresas hoteleras y turoperadoras de Cuba presentan hoy sus más novedosas propuestas en el Salón Mundial del Turismo de París, que se...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-17/nominado-largometraje-cubano-a-los-premios-platino-del-cine-2018/40695">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/f0072151_thumb.jpg" title="largometraje cubano Últimos días en La Habana" alt="largometraje cubano Últimos días en La Habana" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-17/nominado-largometraje-cubano-a-los-premios-platino-del-cine-2018/40695">Nominado largometraje cubano a los Premios Platino del Cine 2018</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Últimos días en La Habana, largometraje del reconocido cineasta cubano Fernando Pérez, resultó nominada en siete categorías de los V Premios...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/economia">Economía</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/economia/2018-03-17/favorecen-fondos-chinos-a-importantes-sectores-de-la-economia-cubana/40694">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/0316-china_thumb.jpg" title="Favorecen fondos chinos a importantes sectores de la economía cubana" alt="Favorecen fondos chinos a importantes sectores de la economía cubana" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/economia/2018-03-17/favorecen-fondos-chinos-a-importantes-sectores-de-la-economia-cubana/40694">Favorecen fondos chinos a importantes sectores de la economía cubana</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        Cuba y China rubricaron en esta capital seis instrumentos jurídicos que oficializan la culminación de proyectos ejecutados con fondos chinos, por...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-17/fernando-gonzalez-la-prensa-es-tarea-digna-y-necesaria/40693">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/fernando-gonzalez1_thumb.jpg" title="Fernando González Llort" alt="Fernando González Llort" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-17/fernando-gonzalez-la-prensa-es-tarea-digna-y-necesaria/40693">Fernando González: La prensa es tarea digna y necesaria</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Fernando González Llort, Héroe de la República de Cuba y presidente del Instituto Cubano de Amistad con los Pueblos (ICAP), destacó la labor de...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-17/entregan-premio-nacional-de-musica-en-cuba-a-jose-luis-cortes/40692">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/Jose-L-Cortes-Tosco_thumb.jpg" title="José Luis Cortés" alt="José Luis Cortés" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-17/entregan-premio-nacional-de-musica-en-cuba-a-jose-luis-cortes/40692">Entregan Premio Nacional de Música en Cuba a José Luis Cortés</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        El flautista y pedagogo José Luis Cortés, artísticamente conocido como El Tosco, recibirá hoy el Premio Nacional de Música 2017 de Cuba, en...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-02-23/ediciones-citmatel-presente-en-ferias-internacionales-del-libro/40359">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/F-pc1_thumb.jpg" title="Banner alegórico a la participación de Citmatel en las Ferias Internacionales del Libro" alt="Banner alegórico a la participación de Citmatel en las Ferias Internacionales del Libro" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-02-23/ediciones-citmatel-presente-en-ferias-internacionales-del-libro/40359">Ediciones Citmatel presente en Ferias Internacionales del Libro</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Con un amplio catálogo de productos audiovisuales en CD, DVD, audiolibros, ebooks y aplicaciones para móviles, la Editorial CITMATEL estará...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                        <!--<section class="row section-commercial">-->
                    <div class="col-lg-12 col-carousel">
        <p class="banner-carousel-title">productos</p>
        <p class="banner-carousel-subtitle">Contenidos cubanos de variadas temáticas </p>
        <div class="row">
            <div id="pcuba-banner-carousel" class="carousel slide short-carousel" data-ride="carousel">
                                <div class="banner-carousel-item-list carousel-inner" role="listbox">
                                            <div class="item active">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=27" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia que permitirá, desde el propio hogar, conocer sobre algunos padecimientos crónicos como el asma, la diabetes, el Alzheimer, entre otros. También en #Ebook y #LibroImpreso"><img src="http://www.cuba.cu/imgs/product/thumbs/info-hogar_thumb.jpg" class="img-responsive center-block" title="Multimedia que permitirá, desde el propio hogar, conocer sobre algunos padecimientos crónicos como el asma, la diabetes, el Alzheimer, entre otros. También en #Ebook y #LibroImpreso" alt="Multimedia Información Médica para el hogar" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=286" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Una modesta contribución a la divulgación del papel que han jugado las mujeres en la historia y el desarrollo de la ciencia.#DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/mujer-cientifica_thumb.jpg" class="img-responsive center-block" title="Una modesta contribución a la divulgación del papel que han jugado las mujeres en la historia y el desarrollo de la ciencia.#DescargaenLínea" alt="Ebook Ser Mujer Científica o morir en el intento" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=283" target="_blank" data-toggle="tooltip" data-placement="bottom" title="A través de ocho relatos y un epílogo escritos en un lenguaje ameno y coloquial se aborda el tema de la mujer de esta edad en el contexto de la sociedad contemporánea.#DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/mediosiglo_thumb.jpg" class="img-responsive center-block" title="A través de ocho relatos y un epílogo escritos en un lenguaje ameno y coloquial se aborda el tema de la mujer de esta edad en el contexto de la sociedad contemporánea.#DescargaenLínea" alt="Ebook Las Mediosiglo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=176" target="_blank" data-toggle="tooltip" data-placement="bottom" title="La dimensión humana de dos líderes latinoamericanos y la amistad que los unió en su lucha común por la independencia y la soberanía de América Latina. (También como Descarga en línea)"><img src="http://www.cuba.cu/imgs/product/thumbs/fidelChavez_thumb.jpg" class="img-responsive center-block" title="La dimensión humana de dos líderes latinoamericanos y la amistad que los unió en su lucha común por la independencia y la soberanía de América Latina. (También como Descarga en línea)" alt="Fidel y Chávez" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=366" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Muestra los 14 meridianos principales de acupuntura que más se utilizan en la práctica cotidiana de la Medicina Tradicional China. "><img src="http://www.cuba.cu/imgs/product/thumbs/atlas_thumb.jpg" class="img-responsive center-block" title="Muestra los 14 meridianos principales de acupuntura que más se utilizan en la práctica cotidiana de la Medicina Tradicional China. " alt="Atlas interactivo de acupuntura (Multimedia)    " /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=363" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Interesante juego educativo, que le permitirá aprender y comprender de manera básica la estructura del cuerpo humano. "><img src="http://www.cuba.cu/imgs/product/thumbs/Supersomy_thumb.jpg" class="img-responsive center-block" title="Interesante juego educativo, que le permitirá aprender y comprender de manera básica la estructura del cuerpo humano. " alt="Juego interactivo Supersomy" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=115" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Serie documental que muestra asombrosas imágenes de especies y regiones naturales,  capturadas durante difíciles condiciones."><img src="http://www.cuba.cu/imgs/product/thumbs/naturaleza_thumb.jpg" class="img-responsive center-block" title="Serie documental que muestra asombrosas imágenes de especies y regiones naturales,  capturadas durante difíciles condiciones." alt="Video Naturaleza protegida de Cuba" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=109" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este audiovisual muestra las maravillas y tesoros ecológicos de cuatro parques nacionales declarados Reserva de la Biosfera en Cuba: El Desembarco del Granma en Sierra Maestra, Viñales en Pinar del Río, Alejandro de Humboldt en Holguín y Guantánamo y Caguanes en Sancti Spíritus."><img src="http://www.cuba.cu/imgs/product/thumbs/parques-reserva_thumb.jpg" class="img-responsive center-block" title="Este audiovisual muestra las maravillas y tesoros ecológicos de cuatro parques nacionales declarados Reserva de la Biosfera en Cuba: El Desembarco del Granma en Sierra Maestra, Viñales en Pinar del Río, Alejandro de Humboldt en Holguín y Guantánamo y Caguanes en Sancti Spíritus." alt="Video Parques Reserva de la biosfera" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=249" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Once entrevistas a personalidades del ballet mundial realizadas por la periodista Martha Sánchez. También en #DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/Danza-sin-fronteras_thumb.jpg" class="img-responsive center-block" title="Once entrevistas a personalidades del ballet mundial realizadas por la periodista Martha Sánchez. También en #DescargaenLínea" alt="Ebook Por una Danza sin fronteras" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=368" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Poemario en décimas, escrito con un lenguaje coloquial y una manera de decir característica."><img src="http://www.cuba.cu/imgs/product/thumbs/El-juego-de-la-memoria_thumb.jpg" class="img-responsive center-block" title="Poemario en décimas, escrito con un lenguaje coloquial y una manera de decir característica." alt="Ebook El juego de la memoria" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=365" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Testimonio sobre el enfrentamiento a la contrarrevolución armada en Cuba en los primeros años de la década de 1960."><img src="http://www.cuba.cu/imgs/product/thumbs/Los-pies-en-la-tierra_thumb.jpg" class="img-responsive center-block" title="Testimonio sobre el enfrentamiento a la contrarrevolución armada en Cuba en los primeros años de la década de 1960." alt="Ebook Los pies en la tierra" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=32" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Interesante y atractiva obra que ofrece la oportunidad de adentrarse en el maravilloso mundo de las aves representativas de América Latina y el Caribe. Disponible en Ebook"><img src="http://www.cuba.cu/imgs/product/thumbs/aves_de_america_thumb.jpg" class="img-responsive center-block" title="Interesante y atractiva obra que ofrece la oportunidad de adentrarse en el maravilloso mundo de las aves representativas de América Latina y el Caribe. Disponible en Ebook" alt="Multimedia Aves de América" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=367" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Se exponen las aplicaciones de la biofotonterapia en el tratamiento del dolor, de las afecciones del sistema osteomioarticular, en la cicatrización patológica, en el envejecimiento facial y en las secuelas del acné..."><img src="http://www.cuba.cu/imgs/product/thumbs/biofotonterapia-ebook_thumb.jpg" class="img-responsive center-block" title="Se exponen las aplicaciones de la biofotonterapia en el tratamiento del dolor, de las afecciones del sistema osteomioarticular, en la cicatrización patológica, en el envejecimiento facial y en las secuelas del acné..." alt="Ebook Biofotonterapia en Medicina Natural y Tradicional" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=351" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Si ha padecido de lesiones o dolor en articulaciones, cuello, espalda o tobillos, esta es la oportunidad de conocer una serie de ejercicios para su mejoría. También disponible en CD."><img src="http://www.cuba.cu/imgs/product/thumbs/apk-ejercicios-salud_thumb.jpg" class="img-responsive center-block" title="Si ha padecido de lesiones o dolor en articulaciones, cuello, espalda o tobillos, esta es la oportunidad de conocer una serie de ejercicios para su mejoría. También disponible en CD." alt="APK Ejercicios para la salud" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=348" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Video que recrea el baile del Danzón, considerado el padre de muchos otros ritmos de fama mundial como el Danzonete, el Mambo y el Chachachá."><img src="http://www.cuba.cu/imgs/product/thumbs/danzon_thumb.jpg" class="img-responsive center-block" title="Video que recrea el baile del Danzón, considerado el padre de muchos otros ritmos de fama mundial como el Danzonete, el Mambo y el Chachachá." alt="DVD Yo soy el Danzón" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=364" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Ebook de autoayuda donde se ofrecen soluciones para dejar de beber, única manera de detener los problemas provocados por el alcohol. #DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/Como-eliminar-el-alcoholismo_thumb.jpg" class="img-responsive center-block" title="Ebook de autoayuda donde se ofrecen soluciones para dejar de beber, única manera de detener los problemas provocados por el alcohol. #DescargaenLínea" alt="Ebook Cómo eliminar el alcoholismo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=266" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Documental que muestra una recopilación sobre la situación de los Cinco Héroes cubanos que fueron injustamente condenados en Estados Unidos. #DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/5heroes_thumb.jpg" class="img-responsive center-block" title="Documental que muestra una recopilación sobre la situación de los Cinco Héroes cubanos que fueron injustamente condenados en Estados Unidos. #DescargaenLínea" alt="Los Cinco" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=199" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Documental  que asoma a la vida del sacerdote de las religiones de origen africano Enrique Hernández Armenteros, conocido como Enriquito, el de Guanabacoa. #DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/tata_thumb.jpg" class="img-responsive center-block" title="Documental  que asoma a la vida del sacerdote de las religiones de origen africano Enrique Hernández Armenteros, conocido como Enriquito, el de Guanabacoa. #DescargaenLínea" alt="DVD Soy TATA NGANGA" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=189" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Recorrido virtual por el mayor y más importante centro histórico-cultural dedicado a la vida y obra del Héroe Nacional de Cuba, situado en la Plaza de la Revolución, La Habana. "><img src="http://www.cuba.cu/imgs/product/thumbs/memorial_thumb.jpg" class="img-responsive center-block" title="Recorrido virtual por el mayor y más importante centro histórico-cultural dedicado a la vida y obra del Héroe Nacional de Cuba, situado en la Plaza de la Revolución, La Habana. " alt="Multimedia Memorial José Martí" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=192" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia basada en una reciente investigación histórica del diario de campaña de José Martí. Por vez primera se digitalizan manuscritos originales y numerosas fotografías, que enriquecen el valor documental de la obra. "><img src="http://www.cuba.cu/imgs/product/thumbs/diamante_thumb.jpg" class="img-responsive center-block" title="Multimedia basada en una reciente investigación histórica del diario de campaña de José Martí. Por vez primera se digitalizan manuscritos originales y numerosas fotografías, que enriquecen el valor documental de la obra. " alt="Multimedia Diamante: Diario de campaña de José Martí" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=142" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia que muestra el análisis crítico realizado por investigadores del Centro de Estudios Martianos a la obra de José Martí."><img src="http://www.cuba.cu/imgs/product/thumbs/musicos1_thumb.jpg" class="img-responsive center-block" title="Multimedia que muestra el análisis crítico realizado por investigadores del Centro de Estudios Martianos a la obra de José Martí." alt="DVD Músicos, poetas y pintores" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=23" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Inspirada en el libro La Expedición del Honor, narra las acciones y vicisitudes de la expedición protagonizada por los generales Flor Crombet, Antonio Maceo y José Maceo desde Costa Rica hacia Cuba y su encuentro con las tropas mambisas. "><img src="http://www.cuba.cu/imgs/product/thumbs/duaba_thumb.jpg" class="img-responsive center-block" title="Inspirada en el libro La Expedición del Honor, narra las acciones y vicisitudes de la expedición protagonizada por los generales Flor Crombet, Antonio Maceo y José Maceo desde Costa Rica hacia Cuba y su encuentro con las tropas mambisas. " alt="DVD Duaba" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=205" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Abarca temas como las fases de cultivo, comercialización, marcas y vitolas más representativas, litografías tabacaleras, ritos y usos medicinales primitivos."><img src="http://www.cuba.cu/imgs/product/thumbs/habano_thumb.jpg" class="img-responsive center-block" title="Abarca temas como las fases de cultivo, comercialización, marcas y vitolas más representativas, litografías tabacaleras, ritos y usos medicinales primitivos." alt="CD Habano" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=126&serie=18" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un documental de Ángel Alderete que narra el proceso vivido por los músicos durante la grabación del trabajo discográfico de igual título. "><img src="http://www.cuba.cu/imgs/product/thumbs/leo_thumb.jpg" class="img-responsive center-block" title="Un documental de Ángel Alderete que narra el proceso vivido por los músicos durante la grabación del trabajo discográfico de igual título. " alt="DVD Leo Brouwer" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=51" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un documental dirigido por Roberto Chile que resume la gira realizada por la compañía infantil cubana de teatro “La Colmenita” por ciudades de Estados Unidos."><img src="http://www.cuba.cu/imgs/product/thumbs/colmenita_thumb.jpg" class="img-responsive center-block" title="Un documental dirigido por Roberto Chile que resume la gira realizada por la compañía infantil cubana de teatro “La Colmenita” por ciudades de Estados Unidos." alt="DVD Esencias" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=17" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Guía, paso a paso, con información y consejos prácticos, desde la planificación, concepción, nacimiento y cuidados del bebé hasta los primeros meses de vida. Muestra tres métodos para el cálculo del sexo del bebé."><img src="http://www.cuba.cu/imgs/product/thumbs/17_thumb.jpg" class="img-responsive center-block" title="Guía, paso a paso, con información y consejos prácticos, desde la planificación, concepción, nacimiento y cuidados del bebé hasta los primeros meses de vida. Muestra tres métodos para el cálculo del sexo del bebé." alt="Multimedia El libro de mamá" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=140" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Obra poética de José Martí, llena de ternura y amor filial. Contiene 15 poemas que dedicó a su hijo, nacido en 1878. (Disponible también en: #Audiolibro y #Descargaenlínea)"><img src="http://www.cuba.cu/imgs/product/thumbs/ismaelillo_thumb.jpg" class="img-responsive center-block" title="Obra poética de José Martí, llena de ternura y amor filial. Contiene 15 poemas que dedicó a su hijo, nacido en 1878. (Disponible también en: #Audiolibro y #Descargaenlínea)" alt="Ebook Ismaelillo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=136" target="_blank" data-toggle="tooltip" data-placement="bottom" title="“Giselle” es una pieza emblemática imprescindible en la escena del ballet clásico mundial,  interpretada por la compañía cubana Ballet de Camagüey."><img src="http://www.cuba.cu/imgs/product/thumbs/giselle_thumb.jpg" class="img-responsive center-block" title="“Giselle” es una pieza emblemática imprescindible en la escena del ballet clásico mundial,  interpretada por la compañía cubana Ballet de Camagüey." alt="DVD Giselle" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=352" target="_blank" data-toggle="tooltip" data-placement="bottom" title="¿Padece de dolor en la cervical? El Dr. Miguel Rolando González Corona le dice cómo combatir esta dolencia. (#Descargar en línea) "><img src="http://www.cuba.cu/imgs/product/thumbs/dolor-cuello_thumb.jpg" class="img-responsive center-block" title="¿Padece de dolor en la cervical? El Dr. Miguel Rolando González Corona le dice cómo combatir esta dolencia. (#Descargar en línea) " alt="Ebook Dolor en el cuello" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=353" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Se dice que dos de cada tres adultos van a presentar dolor, a nivel de la columna lumbar, en algún momento de su vida."><img src="http://www.cuba.cu/imgs/product/thumbs/Sacrolumbalgia_thumb.jpg" class="img-responsive center-block" title="Se dice que dos de cada tres adultos van a presentar dolor, a nivel de la columna lumbar, en algún momento de su vida." alt="Ebook Sacrolumbalgia" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=354" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Conozca sobre la hipertensión arterial, así como las indicaciones del autor en las especialidades de Rehabilitación y Fisioterapia. (#ImpresiónbajoDemanda)"><img src="http://www.cuba.cu/imgs/product/thumbs/Hipertension_thumb.jpg" class="img-responsive center-block" title="Conozca sobre la hipertensión arterial, así como las indicaciones del autor en las especialidades de Rehabilitación y Fisioterapia. (#ImpresiónbajoDemanda)" alt="Ebook Hipertensión arterial. Ejercicios dinámicos" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=361" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un instructivo y ameno juego que nos acerca a lugares interesantes, turísticos e históricos de toda Cuba, a nuestra cultura y tradiciones. "><img src="http://www.cuba.cu/imgs/product/thumbs/destino-cuba-apk_thumb.jpg" class="img-responsive center-block" title="Un instructivo y ameno juego que nos acerca a lugares interesantes, turísticos e históricos de toda Cuba, a nuestra cultura y tradiciones. " alt="APK Destino Cuba: una aventura maravillosa" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=191" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Aborda los hechos más relevantes de la Revolución Cubana, desde sus inicios con el triunfo revolucionario el 1ro. de enero de 1959 hasta 1999. Refleja las problemáticas fundamentales de cada periodo histórico, los éxitos obtenidos en diferentes terrenos de la vida social y política del pueblo cubano y realza la figura histórica de Fidel Castro como líder indiscutible del proceso revolucionario."><img src="http://www.cuba.cu/imgs/product/thumbs/revolucion_thumb.jpg" class="img-responsive center-block" title="Aborda los hechos más relevantes de la Revolución Cubana, desde sus inicios con el triunfo revolucionario el 1ro. de enero de 1959 hasta 1999. Refleja las problemáticas fundamentales de cada periodo histórico, los éxitos obtenidos en diferentes terrenos de la vida social y política del pueblo cubano y realza la figura histórica de Fidel Castro como líder indiscutible del proceso revolucionario." alt="DVD Historia de la Revolución Cubana" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=346" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Esta obra es resultado de una investigación histórica sobre las actividades subversivas del gobierno de Estados Unidos. "><img src="http://www.cuba.cu/imgs/product/thumbs/Una-difícil-conquista_thumb.jpg" class="img-responsive center-block" title="Esta obra es resultado de una investigación histórica sobre las actividades subversivas del gobierno de Estados Unidos. " alt="Ebook Una difícil conquista. La alfabetización, la enseñanza y la cultura " /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=135" target="_blank" data-toggle="tooltip" data-placement="bottom" title="El Circo Nacional de Cuba muestra lo mejor de su arte con estilos donde se entremezclan la pureza técnica en la interpretación, el ritmo cubano, la alegría y el colorido del Caribe."><img src="http://www.cuba.cu/imgs/product/thumbs/circuba1_thumb.jpg" class="img-responsive center-block" title="El Circo Nacional de Cuba muestra lo mejor de su arte con estilos donde se entremezclan la pureza técnica en la interpretación, el ritmo cubano, la alegría y el colorido del Caribe." alt="DVD Circuba" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=117" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un excelente recorrido a través de cuatro documentales que abarcan  la evolución de algunos géneros de la música cubana, como Bolero, Chachachá, Mambo y Son."><img src="http://www.cuba.cu/imgs/product/thumbs/ritmos-cubanos_thumb.jpg" class="img-responsive center-block" title="Un excelente recorrido a través de cuatro documentales que abarcan  la evolución de algunos géneros de la música cubana, como Bolero, Chachachá, Mambo y Son." alt="DVD Ritmos Cubanos" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=131" target="_blank" data-toggle="tooltip" data-placement="bottom" title="En este audiovisual podrá aprender los pasos básicos, combinaciones y desplazamientos para bailar rumba. #DescargaenLínea"><img src="http://www.cuba.cu/imgs/product/thumbs/dvd-rumba_thumb.jpg" class="img-responsive center-block" title="En este audiovisual podrá aprender los pasos básicos, combinaciones y desplazamientos para bailar rumba. #DescargaenLínea" alt="DVD Bailando rumba" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=350" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Una detallada mirada al quehacer político y combativo del líder cubano, desde la Cuba neocolonial hasta el derrocamiento de la dictadura batistiana el 1ro de enero de 1959."><img src="http://www.cuba.cu/imgs/product/thumbs/Los-agradecidos-te-acompanan_thumb.jpg" class="img-responsive center-block" title="Una detallada mirada al quehacer político y combativo del líder cubano, desde la Cuba neocolonial hasta el derrocamiento de la dictadura batistiana el 1ro de enero de 1959." alt="Video Los agradecidos te acompañan" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=344" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Libro donde se exponen las características y medidas de prevención ante eventos naturales. Disponible en #Libro."><img src="http://www.cuba.cu/imgs/product/thumbs/Cuba-prevencion-de-desastres-asociados-al-cambio-climatico_thumb.jpg" class="img-responsive center-block" title="Libro donde se exponen las características y medidas de prevención ante eventos naturales. Disponible en #Libro." alt="Cuba: prevención de desastres asociados al cambio climático (Ebook)" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=349" target="_blank" data-toggle="tooltip" data-placement="bottom" title="DVD donde se agrupa una parte de las intervenciones del Comandante en Jefe, Fidel Castro Ruz, en el programa televisivo Mesa Redonda."><img src="http://www.cuba.cu/imgs/product/thumbs/Fidel-en-la-Mesa-Redonda_thumb.jpg" class="img-responsive center-block" title="DVD donde se agrupa una parte de las intervenciones del Comandante en Jefe, Fidel Castro Ruz, en el programa televisivo Mesa Redonda." alt="Fidel" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=132" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Atractivo audiovisual que recoge un compendio de clases prácticas dedicadas al aprendizaje autónomo del baile de origen cubano conocido como salsa. "><img src="http://www.cuba.cu/imgs/product/thumbs/salsa1_thumb.jpg" class="img-responsive center-block" title="Atractivo audiovisual que recoge un compendio de clases prácticas dedicadas al aprendizaje autónomo del baile de origen cubano conocido como salsa. " alt="DVD La salsa se baila así" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=33" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este audiovisual es un acercamiento al género musical el Punto cubano, devenido en las fiestas campesinas. "><img src="http://www.cuba.cu/imgs/product/thumbs/punto-cubano_thumb.jpg" class="img-responsive center-block" title="Este audiovisual es un acercamiento al género musical el Punto cubano, devenido en las fiestas campesinas. " alt="DVD El punto cubano" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=120" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Esta multimedia presenta aspectos curiosos e interesantes del mundo animal en las más variadas temáticas. También disponible como #Ebook y #Libro"><img src="http://www.cuba.cu/imgs/product/thumbs/curiosidades_thumb.jpg" class="img-responsive center-block" title="Esta multimedia presenta aspectos curiosos e interesantes del mundo animal en las más variadas temáticas. También disponible como #Ebook y #Libro" alt="Curiosidades del mundo animal" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=331&subserie=85" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Basada en el Clásico teatral de Abelardo Estorino "La Casa Vieja". Un filme de Lester Hamlet."><img src="http://www.cuba.cu/imgs/product/thumbs/casa-vieja_thumb.jpg" class="img-responsive center-block" title="Basada en el Clásico teatral de Abelardo Estorino "La Casa Vieja". Un filme de Lester Hamlet." alt="DVD Casa Vieja" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=332&serie=6" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Película clásica del cine cubano. Seleccionada entre los 10 Mejores Filmes del cine latinoamericano."><img src="http://www.cuba.cu/imgs/product/thumbs/lucia1_thumb.jpg" class="img-responsive center-block" title="Película clásica del cine cubano. Seleccionada entre los 10 Mejores Filmes del cine latinoamericano." alt="DVD Lucía" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=212" target="_blank" data-toggle="tooltip" data-placement="bottom" title="La relación entre el recurso audiovisual y la pintura, la escultura y el dibujo, se entrelazan en el Proyecto Alas con puntas concebido por el documentalista cubano Roberto Chile."><img src="http://www.cuba.cu/imgs/product/thumbs/trazos-con-arte_thumb.jpg" class="img-responsive center-block" title="La relación entre el recurso audiovisual y la pintura, la escultura y el dibujo, se entrelazan en el Proyecto Alas con puntas concebido por el documentalista cubano Roberto Chile." alt="DVD Trazos con Arte" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=187" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Cronología histórica sobre los hechos terroristas contra Cuba a partir de 1959."><img src="http://www.cuba.cu/imgs/product/thumbs/multimedia-desclasificados_thumb.jpg" class="img-responsive center-block" title="Cronología histórica sobre los hechos terroristas contra Cuba a partir de 1959." alt="Multimedia Documentos desclasificados" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=188" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Testimonios de personalidades cubanas y extranjeras demuestran como la más poderosa potencia del mundo se ha valido de las diferencias políticas para condenar a Cuba a las mayores escaseces económicas, incluso para la adquisición de medicamentos vitales."><img src="http://www.cuba.cu/imgs/product/thumbs/why-bloqueo11_thumb.jpg" class="img-responsive center-block" title="Testimonios de personalidades cubanas y extranjeras demuestran como la más poderosa potencia del mundo se ha valido de las diferencias políticas para condenar a Cuba a las mayores escaseces económicas, incluso para la adquisición de medicamentos vitales." alt="DVD Why? Bloqueo contra la vida" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=268" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Más de medio siglo ha transcurrido desde que el gobierno de Estados Unidos impusiera el bloqueo a Cuba, la más cruel acción desatada contra nación alguna..."><img src="http://www.cuba.cu/imgs/product/thumbs/bloqueo_thumb.jpg" class="img-responsive center-block" title="Más de medio siglo ha transcurrido desde que el gobierno de Estados Unidos impusiera el bloqueo a Cuba, la más cruel acción desatada contra nación alguna..." alt="DVD Bloqueo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=340&serie=5" target="_blank" data-toggle="tooltip" data-placement="bottom" title="¿Cuáles son las causas de la infertilidad?, ¿todas las causas pueden solucionarse con los modernos métodos de reproducción asistida?. También disponible como APK y multimedia."><img src="http://www.cuba.cu/imgs/product/thumbs/Infertilidad1_thumb.jpg" class="img-responsive center-block" title="¿Cuáles son las causas de la infertilidad?, ¿todas las causas pueden solucionarse con los modernos métodos de reproducción asistida?. También disponible como APK y multimedia." alt="Ebook Infertilidad, un problema de dos" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=200" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Dvd que recopila importantes documentales sobre la santería, tradición religiosa extendida en Cuba. Muestra sus diferentes formas, medios de expresividad, celebraciones y deidades a las que se rinde culto."><img src="http://www.cuba.cu/imgs/product/thumbs/cultos1_thumb.jpg" class="img-responsive center-block" title="Dvd que recopila importantes documentales sobre la santería, tradición religiosa extendida en Cuba. Muestra sus diferentes formas, medios de expresividad, celebraciones y deidades a las que se rinde culto." alt="DVD Cultos Afrocubanos" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=179" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Si tuviera que preparar un menú bien cubano ¿qué platos escogería? Más de 500 recetas y secretos de la cocina española y africana. Disponible además como #Ebook, #Descarga en línea."><img src="http://www.cuba.cu/imgs/product/thumbs/cocina_thumb.jpg" class="img-responsive center-block" title="Si tuviera que preparar un menú bien cubano ¿qué platos escogería? Más de 500 recetas y secretos de la cocina española y africana. Disponible además como #Ebook, #Descarga en línea." alt="CD Cocina cubana" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=298" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este libro, a través de sus páginas, recrea las diversas experiencias del autor obtenidas durante las exploraciones realizadas a las cavernas de Cuba. "><img src="http://www.cuba.cu/imgs/product/thumbs/ebook-aventura-en-las-cavernas_thumb.jpg" class="img-responsive center-block" title="Este libro, a través de sus páginas, recrea las diversas experiencias del autor obtenidas durante las exploraciones realizadas a las cavernas de Cuba. " alt="Ebook sobre aventuras en las cavernas cubanas" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=320" target="_blank" data-toggle="tooltip" data-placement="bottom" title="¿Cuáles son las principales propiedades físico-químicas del agua de mar?, ¿cómo se generan el oleaje, las inundaciones costeras y las corrientes marinas?..."><img src="http://www.cuba.cu/imgs/product/thumbs/ebook-meteorologia-marina_thumb.jpg" class="img-responsive center-block" title="¿Cuáles son las principales propiedades físico-químicas del agua de mar?, ¿cómo se generan el oleaje, las inundaciones costeras y las corrientes marinas?..." alt="Ebook sobre Meteorología marina" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=319" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Literatura especializada en el manejo y conservación de las especies salvajes fuera de su hábitat natural."><img src="http://www.cuba.cu/imgs/product/thumbs/manual-manejo-fauna1_thumb.jpg" class="img-responsive center-block" title="Literatura especializada en el manejo y conservación de las especies salvajes fuera de su hábitat natural." alt="Manual de manejo de fauna silvestre en cautiverio (Ebook)" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=309" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Comprende tres juegos didácticos dirigidos a niños que cursan los niveles de enseñanza primaria y media."><img src="http://www.cuba.cu/imgs/product/thumbs/CDIocusQuimito1_thumb.jpg" class="img-responsive center-block" title="Comprende tres juegos didácticos dirigidos a niños que cursan los niveles de enseñanza primaria y media." alt="Juegos Didácticos Educativos" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=303" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Es un diccionario comentado e ilustrado sobre las deidades y leyendas cubanas afrodescendientes."><img src="http://www.cuba.cu/imgs/product/thumbs/carruselpekedeidades_thumb.jpg" class="img-responsive center-block" title="Es un diccionario comentado e ilustrado sobre las deidades y leyendas cubanas afrodescendientes." alt="Ebook Deidades cubanas de origen africano" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=304" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un concierto lleno de sorpresas y emociones, es la propuesta que nos regala en este disco el cantautor cubano Waldo Mendoza. Con un maravilloso espectáculo dedicado especialmente a las madres, Waldo reunió a conocidos intérpretes nacionales e internacionales que deleitaron al público asistente."><img src="http://www.cuba.cu/imgs/product/thumbs/Waldo_thumb.jpg" class="img-responsive center-block" title="Un concierto lleno de sorpresas y emociones, es la propuesta que nos regala en este disco el cantautor cubano Waldo Mendoza. Con un maravilloso espectáculo dedicado especialmente a las madres, Waldo reunió a conocidos intérpretes nacionales e internacionales que deleitaron al público asistente." alt="Waldo Mendoza en concierto: Libre para Amar" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=274" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Ebook que utiliza esquemas propios, diseñados intencionalmente para explicar los elementos fundamentales presentes en la Biotecnología. "><img src="http://www.cuba.cu/imgs/product/thumbs/biotecnologia_thumb.jpg" class="img-responsive center-block" title="Ebook que utiliza esquemas propios, diseñados intencionalmente para explicar los elementos fundamentales presentes en la Biotecnología. " alt="Biotecnología en esquemas y tablas" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=279" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este interesante libro, permitirá adentrarse en el atrayente mundo de las frutas tropicales."><img src="http://www.cuba.cu/imgs/product/thumbs/Frutas_TropicalesD_thumb.jpg" class="img-responsive center-block" title="Este interesante libro, permitirá adentrarse en el atrayente mundo de las frutas tropicales." alt="Frutas tropicales en la cocina cubana" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=230" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Valioso testimonio fotográfico y audiovisual inspirado en la personalidad del líder histórico de la Revolución Cubana, del realizador Roberto Chile. "><img src="http://www.cuba.cu/imgs/product/thumbs/multimedia-fidel-es-fidel_thumb.jpg" class="img-responsive center-block" title="Valioso testimonio fotográfico y audiovisual inspirado en la personalidad del líder histórico de la Revolución Cubana, del realizador Roberto Chile. " alt="Multimedia Fidel es Fidel" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=267" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este material muestra un resumen sobre algunos de los principales hechos criminales cometidos por Estados Unidos contra Cuba."><img src="http://www.cuba.cu/imgs/product/thumbs/mafia_thumb.jpg" class="img-responsive center-block" title="Este material muestra un resumen sobre algunos de los principales hechos criminales cometidos por Estados Unidos contra Cuba." alt="DVD Contrarrevolución y Mafia" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=234" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Bajo el título Fidel Castro, soldado de las ideas. Omar Pérez Salomón pone a disposición del lector una selección de textos extraídos de discursos, informes y entrevistas de Fidel Castro sobre estos contenidos, clasificados por temáticas como la libre expresión, la publicidad comercial, el acceso y difusión de contenidos en Internet."><img src="http://www.cuba.cu/imgs/product/thumbs/fidel-soldado-ideas_thumb.jpg" class="img-responsive center-block" title="Bajo el título Fidel Castro, soldado de las ideas. Omar Pérez Salomón pone a disposición del lector una selección de textos extraídos de discursos, informes y entrevistas de Fidel Castro sobre estos contenidos, clasificados por temáticas como la libre expresión, la publicidad comercial, el acceso y difusión de contenidos en Internet." alt="Ebook Fidel Castro. Soldado de las Ideas" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=263" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Juego diseñado para estudiantes de la enseñanza media que aprenderán a reconocer los elementos químicos, fórmulas y moléculas más comunes."><img src="http://www.cuba.cu/imgs/product/thumbs/quimito_thumb.jpg" class="img-responsive center-block" title="Juego diseñado para estudiantes de la enseñanza media que aprenderán a reconocer los elementos químicos, fórmulas y moléculas más comunes." alt="Aplicación para móviles ¨Quimito¨" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=262" target="_blank" data-toggle="tooltip" data-placement="bottom" title="En Terrorismo en el éter, se expone de manera precisa y contundente, los antecedentes y acontecimientos más importantes de la agresión que en el campo radial y televisivo realiza el gobierno de Estados Unidos y los grupúsculos contrarrevolucionarios de origen cubano radicados en el sur de la Florida, aglutinados y dirigidos por la Agencia Central de Inteligencia (CIA). "><img src="http://www.cuba.cu/imgs/product/thumbs/terrorismo-eter_thumb.jpg" class="img-responsive center-block" title="En Terrorismo en el éter, se expone de manera precisa y contundente, los antecedentes y acontecimientos más importantes de la agresión que en el campo radial y televisivo realiza el gobierno de Estados Unidos y los grupúsculos contrarrevolucionarios de origen cubano radicados en el sur de la Florida, aglutinados y dirigidos por la Agencia Central de Inteligencia (CIA). " alt="Ebook Terrorismo en el Éter" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=53" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Este libro te permitirá conocer el interesante mundo de los anfibios y los beneficios que producen al medio ambiente. "><img src="http://www.cuba.cu/imgs/product/thumbs/anfibios_thumb.jpg" class="img-responsive center-block" title="Este libro te permitirá conocer el interesante mundo de los anfibios y los beneficios que producen al medio ambiente. " alt="Multimedia Anfibios" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=116" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Esta instructiva multimedia basada en el libro de Francisco H. Pérez nos permitirá conocer más sobre el mundo de las abejas, además de profundizar en el desarrollo de la apicultura en Cuba, la producción y exportación de miel."><img src="http://www.cuba.cu/imgs/product/thumbs/abeja-prodigiosa-eb_thumb.jpg" class="img-responsive center-block" title="Esta instructiva multimedia basada en el libro de Francisco H. Pérez nos permitirá conocer más sobre el mundo de las abejas, además de profundizar en el desarrollo de la apicultura en Cuba, la producción y exportación de miel." alt="La abeja prodigiosa (Ebook)" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=127" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Conceptos y aplicaciones prácticas enfocadas hacia los aspectos económicos vinculados a las decisiones ambientales en una empresa."><img src="http://www.cuba.cu/imgs/product/thumbs/economia-ambiental-ebook_thumb.jpg" class="img-responsive center-block" title="Conceptos y aplicaciones prácticas enfocadas hacia los aspectos económicos vinculados a las decisiones ambientales en una empresa." alt="Ebook  Economía ambiental" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=149" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia Este sistema interactivo está dirigido a estudiantes de nivel medio superior y universitario. Los principales temas que se desarrollan son Límite y continuidad, Cálculo diferencial, Cálculo integral y Explorador de Cónicas. Se evalúa constantemente el aprendizaje a través de cientos de ejercicios."><img src="http://www.cuba.cu/imgs/product/thumbs/calculo-diferencial_thumb.jpg" class="img-responsive center-block" title="Multimedia Este sistema interactivo está dirigido a estudiantes de nivel medio superior y universitario. Los principales temas que se desarrollan son Límite y continuidad, Cálculo diferencial, Cálculo integral y Explorador de Cónicas. Se evalúa constantemente el aprendizaje a través de cientos de ejercicios." alt="Multimedia Cálculo diferencial e integral" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=77" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Especialmente diseñado para el aprendizaje del idioma inglés mediante juegos, canciones y otras amenas actividades compartidas entre profesores, payasos y niños."><img src="http://www.cuba.cu/imgs/product/thumbs/ingles-ninos_thumb.jpg" class="img-responsive center-block" title="Especialmente diseñado para el aprendizaje del idioma inglés mediante juegos, canciones y otras amenas actividades compartidas entre profesores, payasos y niños." alt="DVD Holiday English (Inglés para niños)" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=182&subserie=105" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Grandes de todos los tiempos del deporte en Cuba, son las conocidas como Espectaculares Morenas del Caribe, equipo femenino de voleibol que conquistó consecutivamente tres títulos de oro en los Juegos Olímpicos. Testimonio documental estampado con un sello único, desde las historias de las excepcionales deportistas laureadas mundialmente: Regla Torres Herrera y Mireya Luis Hernández, junto a Eugenio George Laffita, considerado el mejor entrenador de voleibol del siglo XX."><img src="http://www.cuba.cu/imgs/product/thumbs/voli-cuba_thumb.jpg" class="img-responsive center-block" title="Grandes de todos los tiempos del deporte en Cuba, son las conocidas como Espectaculares Morenas del Caribe, equipo femenino de voleibol que conquistó consecutivamente tres títulos de oro en los Juegos Olímpicos. Testimonio documental estampado con un sello único, desde las historias de las excepcionales deportistas laureadas mundialmente: Regla Torres Herrera y Mireya Luis Hernández, junto a Eugenio George Laffita, considerado el mejor entrenador de voleibol del siglo XX." alt="Confesiones Voleibol cubano" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=175&subserie=26" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Una multimedia para conocer los elementos básicos del juego, las tres fases de la partida y la historia del ajedrez."><img src="http://www.cuba.cu/imgs/product/thumbs/ajedrez_thumb.jpg" class="img-responsive center-block" title="Una multimedia para conocer los elementos básicos del juego, las tres fases de la partida y la historia del ajedrez." alt="Aprende a jugar ajedrez" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=183&subserie=105" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Audiovisual con las hazañas de algunos de los más destacados peloteros cubanos de las últimas décadas y logros obtenidos por el béisbol cubano."><img src="http://www.cuba.cu/imgs/product/thumbs/beisbol-cuban-dvd_thumb.jpg" class="img-responsive center-block" title="Audiovisual con las hazañas de algunos de los más destacados peloteros cubanos de las últimas décadas y logros obtenidos por el béisbol cubano." alt="Momentos del béisbol cubano" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=125&subserie=24" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Compilación de materiales científicos y educativos que contiene una guía de las unidades litoestratigráficas cubanas."><img src="http://www.cuba.cu/imgs/product/thumbs/geologia_thumb.jpg" class="img-responsive center-block" title="Compilación de materiales científicos y educativos que contiene una guía de las unidades litoestratigráficas cubanas." alt="Compendio de Geología de Cuba y del Caribe" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=172&subserie=14" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Los cuatro cursos que se ofrecen proveen las herramientas necesarias para diseñar, desarrollar y administrar sitios web. Internet. Útiles y herramientas, proporciona una guía para la utilización de los navegadores y algunos servicios disponibles; Dreamweaver CS3, enseña a diseñar páginas web de alta calidad; Administración de sitios facilita las herramientas necesarias para esta labor y Hojas de estilo enseña a utilizar esta tecnología para crear sitios web atractivos y precisos."><img src="http://www.cuba.cu/imgs/product/thumbs/curso-web_thumb.jpg" class="img-responsive center-block" title="Los cuatro cursos que se ofrecen proveen las herramientas necesarias para diseñar, desarrollar y administrar sitios web. Internet. Útiles y herramientas, proporciona una guía para la utilización de los navegadores y algunos servicios disponibles; Dreamweaver CS3, enseña a diseñar páginas web de alta calidad; Administración de sitios facilita las herramientas necesarias para esta labor y Hojas de estilo enseña a utilizar esta tecnología para crear sitios web atractivos y precisos." alt="Cursos básicos prácticos: Desarrollo web" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=231&subserie=75" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Narra la historia de un niño cubano retenido por sus familiares en Miami, luego de ser rescatado en el mar. "><img src="http://www.cuba.cu/imgs/product/thumbs/elian_thumb.jpg" class="img-responsive center-block" title="Narra la historia de un niño cubano retenido por sus familiares en Miami, luego de ser rescatado en el mar. " alt="DVD Elián" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=185&subserie=75" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Obra dedicada al comandante Camilo Cienfuegos Gorriarán, “El Señor de la Vanguardia” y uno de los principales jefes del Ejército Rebelde durante la lucha insurreccional cubana. Siempre con una sonrisa jovial que afirman los testimonios de quienes lo conocieron, se convirtió en su corta trayectoria revolucionaria en un querido líder."><img src="http://www.cuba.cu/imgs/product/thumbs/camilo_thumb.jpg" class="img-responsive center-block" title="Obra dedicada al comandante Camilo Cienfuegos Gorriarán, “El Señor de la Vanguardia” y uno de los principales jefes del Ejército Rebelde durante la lucha insurreccional cubana. Siempre con una sonrisa jovial que afirman los testimonios de quienes lo conocieron, se convirtió en su corta trayectoria revolucionaria en un querido líder." alt="Siempre Camilo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=37&subserie=75" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Fidel Castro es una de las más grandes figuras de nuestros tiempos, así lo afirman tanto amigos como enemigos. Este documental de Estela Bravo ofrece la oportunidad de conocer simplemente al hombre, a través de entrevistas exclusivas a historiadores, figuras públicas, invitados, amigos y familiares. Incluye materiales de los archivos fílmicos de la Revolución Cubana."><img src="http://www.cuba.cu/imgs/product/thumbs/Fidel-la-historia-no-conocida_thumb.jpg" class="img-responsive center-block" title="Fidel Castro es una de las más grandes figuras de nuestros tiempos, así lo afirman tanto amigos como enemigos. Este documental de Estela Bravo ofrece la oportunidad de conocer simplemente al hombre, a través de entrevistas exclusivas a historiadores, figuras públicas, invitados, amigos y familiares. Incluye materiales de los archivos fílmicos de la Revolución Cubana." alt="Fidel. La historia no contada" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=16&subserie=31" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Información útil para sustituir la sal por hierbas aromáticas y especias. Disponible también en Multimedia, Audiolibro y libro impreso."><img src="http://www.cuba.cu/imgs/product/thumbs/comer-sin-sal-quirantes_thumb.jpg" class="img-responsive center-block" title="Información útil para sustituir la sal por hierbas aromáticas y especias. Disponible también en Multimedia, Audiolibro y libro impreso." alt="Comer pero sin sal" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=207&subserie=39" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Un recorrido cultural, histórico y artístico por la Habana Vieja, Patrimonio de la Humanidad."><img src="http://www.cuba.cu/imgs/product/thumbs/cuba1_thumb.jpg" class="img-responsive center-block" title="Un recorrido cultural, histórico y artístico por la Habana Vieja, Patrimonio de la Humanidad." alt="Recorra Cuba. La Habana Vieja" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=218" target="_blank" data-toggle="tooltip" data-placement="bottom" title="En este juego, te encontrarás con Iocus, un pequeño amante de la arqueología."><img src="http://www.cuba.cu/imgs/product/thumbs/iocus1_thumb.jpg" class="img-responsive center-block" title="En este juego, te encontrarás con Iocus, un pequeño amante de la arqueología." alt="Juego infantil IOCUS, la aventura del saber" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=103" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia que muestra la morfología funcional del aparato locomotor, y las estructuras que están involucrados en actividades físicas o ejercicios. "><img src="http://www.cuba.cu/imgs/product/thumbs/anatomia_thumb.jpg" class="img-responsive center-block" title="Multimedia que muestra la morfología funcional del aparato locomotor, y las estructuras que están involucrados en actividades físicas o ejercicios. " alt="Ebook  Anatomía y movimiento" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=225" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Audiovisual que ofrece al espectador un fiel testimonio de los modos de vida y comportamiento de tan pintoresca y dinámica región cubana, Santiago de Cuba."><img src="http://www.cuba.cu/imgs/product/thumbs/302_thumb.jpg" class="img-responsive center-block" title="Audiovisual que ofrece al espectador un fiel testimonio de los modos de vida y comportamiento de tan pintoresca y dinámica región cubana, Santiago de Cuba." alt=" Audiovisual Recorra Santiago" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=128" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Documental cubano que analiza las consecuencias del cambio climático."><img src="http://www.cuba.cu/imgs/product/thumbs/cambio-climatico1_thumb.jpg" class="img-responsive center-block" title="Documental cubano que analiza las consecuencias del cambio climático." alt="Cambio climático, el reto continúa" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=34" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Multimedia que muestra algunos hechos y fenómenos que ocurren en la naturaleza. Disponible en #Ebook y #Libro."><img src="http://www.cuba.cu/imgs/product/thumbs/abecedario-naturaleza_thumb.jpg" class="img-responsive center-block" title="Multimedia que muestra algunos hechos y fenómenos que ocurren en la naturaleza. Disponible en #Ebook y #Libro." alt=" Abecedario de la naturaleza" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=261" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Serie documental acerca de los movimientos sísmicos más relevantes que han tenido lugar en Cuba."><img src="http://www.cuba.cu/imgs/product/thumbs/tierra_thumb.jpg" class="img-responsive center-block" title="Serie documental acerca de los movimientos sísmicos más relevantes que han tenido lugar en Cuba." alt="DVD Cuando tiembla la Tierra" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=171" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Principios de funcionamiento, componentes fundamentales, fallas más comunes. Propone un original método de reparación a bajo voltaje de las fuentes conmutadas. Dirigido a técnicos, profesionales de la electrónica y a todas aquellas personas interesadas en el tema. "><img src="http://www.cuba.cu/imgs/product/thumbs/curso-reparac2_thumb.jpg" class="img-responsive center-block" title="Principios de funcionamiento, componentes fundamentales, fallas más comunes. Propone un original método de reparación a bajo voltaje de las fuentes conmutadas. Dirigido a técnicos, profesionales de la electrónica y a todas aquellas personas interesadas en el tema. " alt="Curso de reparación de fuentes" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=224" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Conozca las reservas ecológicas que atesoran las profundidades marinas en Cuba."><img src="http://www.cuba.cu/imgs/product/thumbs/sitios-buceo-cuba1_thumb.jpg" class="img-responsive center-block" title="Conozca las reservas ecológicas que atesoran las profundidades marinas en Cuba." alt="Sitios de buceo" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=29" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Compilación de poemas de Nicolás Guillén que incluye poesía para niños y adultos.  Disponible también en  Audiolibro, libro impreso y Descarga en línea.
"><img src="http://www.cuba.cu/imgs/product/thumbs/Poemario_Nicolas_Guillen_thumb.jpg" class="img-responsive center-block" title="Compilación de poemas de Nicolás Guillén que incluye poesía para niños y adultos.  Disponible también en  Audiolibro, libro impreso y Descarga en línea.
" alt="Poemario Nicolás Guillén" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=54" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Compendio de poesías dirigido a aquellas personas que no cuentan con el maravilloso don de la visión, y lo que logran cuando los ojos del corazón se proponen llenar el mundo de amor y de esperanza. "><img src="http://www.cuba.cu/imgs/product/thumbs/optica_thumb.jpg" class="img-responsive center-block" title="Compendio de poesías dirigido a aquellas personas que no cuentan con el maravilloso don de la visión, y lo que logran cuando los ojos del corazón se proponen llenar el mundo de amor y de esperanza. " alt="Audiolibro Óptica sin ver" /></a>
                                                            </div>
                        </div>
                                            <div class="item ">
                            <div class="banner-carousel-slide">
                                                                                                                                    <a class="tooltip-product" href="http://www.libreriavirtualcuba.com/productos.php?producto=22" target="_blank" data-toggle="tooltip" data-placement="bottom" title="Agrupa conceptos y técnicas actuales en el tratamiento y la rehabilitación de los pacientes afectados de las rodillas. "><img src="http://www.cuba.cu/imgs/product/thumbs/rodillas11_thumb.jpg" class="img-responsive center-block" title="Agrupa conceptos y técnicas actuales en el tratamiento y la rehabilitación de los pacientes afectados de las rodillas. " alt="Ebook Cuidemos nuestras rodillas" /></a>
                                                            </div>
                        </div>
                                    </div>
                                <a class="pcuba-banner-carousel-control left carousel-control col-sm-2" href="#pcuba-banner-carousel" role="button" data-slide="prev">
                    <span class="glyphicon" aria-hidden="true"></span>
                    <span class="sr-only">Previous</span>
                </a>
                <a class="pcuba-banner-carousel-control right carousel-control col-sm-2" href="#pcuba-banner-carousel" role="button" data-slide="next">
                    <span class="glyphicon" aria-hidden="true"></span>
                    <span class="sr-only">Next</span>
                </a>
            </div>
        </div>
    </div>            <!--</section>-->
                            <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/ciencia-y-tecnologia">Ciencia y Tecnología</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/ciencia-y-tecnologia/2018-03-16/invita-la-uci-a-participar-en-pena-tecnologica-y-hackathon/40691">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/uci-cuba-informatica_thumb.jpg" title=" XIII edición de la Peña Tecnológica Nacional," alt=" XIII edición de la Peña Tecnológica Nacional," />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/ciencia-y-tecnologia/2018-03-16/invita-la-uci-a-participar-en-pena-tecnologica-y-hackathon/40691">Invita la UCI a participar en Peña Tecnológica y Hackathon</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        La XIII edición de la Peña Tecnológica Nacional, espacio que ha contribuido a exponer e impulsar proyectos tecnológicos claves para el desarrollo...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-16/un-dia-con-buena-estrella/40689">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/Celina-Gonzalez_thumb.jpg" title="Celina González." alt="Celina González." />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-16/un-dia-con-buena-estrella/40689">Un día con buena estrella</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Dicen que los niños llegan a este mundo bajo el signo de una estrella para bien o para mal. Parece el que el 16 de marzo un astro favorable a la...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-16/grupo-de-rock-estadounidense-confirma-conciertos-en-cuba-/40690">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/f5b631c83959699cf16f1bc01c0f5409_L_thumb.jpg" title="Banda estadounidense de rock Suicidal Tendencies" alt="Banda estadounidense de rock Suicidal Tendencies" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-16/grupo-de-rock-estadounidense-confirma-conciertos-en-cuba-/40690">Grupo de rock estadounidense confirma conciertos en Cuba </a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        La banda estadounidense de rock Suicidal Tendencies confirmó hoy las fechas y lugares de sendos conciertos que ofrecerá en Cuba en mayo..                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/politica">Política</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/politica/2018-03-16/presidente-de-kenya-inaugura-embajada-de-su-pais-en-cuba-/40687">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/354b5dfbfffb9de98947f9c16313fcf9_L_thumb.jpg" title=" Presidente de Kenya, Uhuru Kenyatta" alt=" Presidente de Kenya, Uhuru Kenyatta" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/politica/2018-03-16/presidente-de-kenya-inaugura-embajada-de-su-pais-en-cuba-/40687">Presidente de Kenya inaugura embajada de su país en Cuba </a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    El presidente de Kenya, Uhuru Kenyatta, inauguró hoy la embajada de su país en La Habana, ceremonia a la cual asistió el canciller cubano...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/cultura">Cultura</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/cultura/2018-03-16/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/40688">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/gentezona-pausini_thumb.jpg" title="Gente de Zona  junto a Laura Pausini " alt="Gente de Zona  junto a Laura Pausini " />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/cultura/2018-03-16/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/40688">Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        La Habana, 16 mar (PL) El grupo cubano Gente de Zona agradeció hoy a sus seguidores por la acogida a la versión del tema Nadie ha dicho, una...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/medio-ambiente">Medio Ambiente</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/medio-ambiente/2018-03-16/por-reducir-riesgos-de-desastres-en-la-isla/40686">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/desastres_naturales_thumb.jpg" title="Inundaciones costeras" alt="Inundaciones costeras" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/medio-ambiente/2018-03-16/por-reducir-riesgos-de-desastres-en-la-isla/40686">Por reducir riesgos de desastres en la isla</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Cuba y la Unión Europea trabajan por asegurar la capacidad de recuperación de las poblaciones afectadas y resarcir los daños provocados por el...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
                                <div class="col-lg-12">
                <div class="row">
                    <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/ciencia-y-tecnologia">Ciencia y Tecnología</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/ciencia-y-tecnologia/2018-03-16/premian-resultados-de-la-investigacion-cientifica-en-cuba/40685">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/miguel-diaz-canel-perfil1_thumb.jpg" title="Miguel Díaz-Canel Bermúdez" alt="Miguel Díaz-Canel Bermúdez" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/ciencia-y-tecnologia/2018-03-16/premian-resultados-de-la-investigacion-cientifica-en-cuba/40685">Premian resultados de la investigación científica en Cuba</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                                                        En una ceremonia efectuada en el paraninfo de la Academia de Ciencias de Cuba (ACC), se reconoció los resultados de la investigación científica en...                                </p>
                            </div>
                        </article>
                    </div>
        
                                        <div class="wrapper-news-art col-md-6 col-sm-12">
                        <article class="news-art news-art-right">
                            <nav>
                                en:
                                <ol class="breadcrumb pcuba-breadcrumb">
                                    <li>noticia</li>
                                    <li class="active"><a href="http://www.cuba.cu/news/politica">Política</a></li>
                                </ol>
                            </nav>
                                                            <div class="wrapper-news-img">
                                    <a href="http://www.cuba.cu/politica/2018-03-16/cuba-sera-sede-del-iii-congreso-internacional-de-investigadores-sobre-juventud/40684">                                                                                                                                                                    <img class="news-img" src="http://www.cuba.cu/imgs/news/thumbs/cuba-congreso-juventud-1_thumb.jpg" title="Logo del III Congreso Internacional de Investigadores sobre la Juventud" alt="Logo del III Congreso Internacional de Investigadores sobre la Juventud" />                                                                            </a>
                                </div>
                                                        <h2 class="news-title"><a href="http://www.cuba.cu/politica/2018-03-16/cuba-sera-sede-del-iii-congreso-internacional-de-investigadores-sobre-juventud/40684">Cuba será sede del III Congreso Internacional de Investigadores sobre Juventud</a></h2>
                            <div class="wrapper-summary">
                                <p class="news-summary">
                                    Del 26 al 30 de marzo tendrá lugar en La Habana el III Congreso Internacional de Investigadores sobre la Juventud, en donde participaran delegados...                                </p>
                            </div>
                        </article>
                    </div>
               </div>
            </div>
        
    </div>

<nav><ul class="pagination pcuba-pagination"><li class="active text-center" data-category="todas" data-per-page="16"><a href="#">1</a></li><li class="text-center" data-category="todas" data-per-page="16"><a href="http://www.cuba.cu/news/todas/pag/2" data-ci-pagination-page="2" rel="start">2</a></li><li title="Siguiente" class="text-center  prev-next" data-category="todas" data-per-page="16" data-page="2"><a href="http://www.cuba.cu/news/todas/pag/2" data-ci-pagination-page="2" rel="next"><span aria-hidden="true" data-page="2"> > </span></a></li><li title="Fin" class="text-center  prev-next" data-category="todas" data-per-page="16" data-page="2"><a href="http://www.cuba.cu/news/todas/pag/30" data-ci-pagination-page="30"><span aria-hidden="true" data-page="2"> >> </span></a></li></ul></nav>
<script src="http://www.cuba.cu/assets/js/main.js"></script>
                    </div>
                </div>
            </section>
            <section class="section-special-banner row">
                <div class="col-sm-12">
            <div class="row">
                        <div class="wrapper-special-banner col-md-6 col-sm-12">
            <article class="special-banner">
                <header>
                    <a href="http://cuba.cu/sites/politica/discursos-del-presidente--raul-castro-ruz" target="_blank"><img class="img-responsive" src="http://www.cuba.cu/imgs/banner/Raul.png" alt="Discursos e intervenciones de Raúl Castro Ruz" title="Discursos e intervenciones de Raúl Castro Ruz"/></a>
                </header>
                <div class="wrapper-site-list">
                    <ul class="site-list">
                        <li>
                            <a href="http://www.cubadebate.cu/especiales/2018/03/05/raul-castro-proclamamos-el-invariable-respaldo-a-la-revolucion-bolivariana/#.Wp6JJ6u21rQ" target="_blank">
                                <span class="bullet-point">-</span><span>Raúl Castro: Proclamamos el invariable respaldo a la Revolución Bolivariana</span>
                            </a>
                        </li>
                <li>
            <a href="http://www.granma.cu/discursos-raul/2018-02-25/tres-mambises-de-nuestros-tiempos-discurso-pronunciado-por-raul-castro-en-la-ceremonia-de-condecoracion-efectuada-en-el-capitolio" target="_blank">
                <span class="bullet-point">-</span><span>Tres mambises de nuestros tiempos: Discurso pronunciado por Raúl Castro en la ceremonia de condecoración efectuada en el Capitolio</span>
            </a>
        </li>
                <li>
            <a href="http://www.cubadebate.cu/especiales/2017/12/22/raul-en-la-asamblea-nacional-aqui-estamos-y-estaremos-libres-soberanos-e-independientes/#.Wj0JTUbLFnw" target="_blank">
                <span class="bullet-point">-</span><span>Raúl en la Asamblea Nacional: “Aquí estamos y estaremos, libres, soberanos e independientes”</span>
            </a>
        </li>
                                    </ul>
                </div>
                <footer>
                    <div class="wrapper-view-more">
                        <a href="http://cuba.cu/sites/politica/discursos-del-presidente--raul-castro-ruz" target="_blank">Ver más...</a>
                    </div>
                </footer>
            </article>
        </div>
                        <div class="wrapper-special-banner col-md-6 col-sm-12">
            <article class="special-banner">
                <header>
                    <a href="http://www.cuba.cu/sites/politica/reflexiones-del-comandante-en-jefe-fidel-castro-ruz" target="_blank"><img class="img-responsive" src="http://www.cuba.cu/imgs/banner/Fidel.png" alt="Reflexiones de Fidel Castro" title="Reflexiones de Fidel Castro"/></a>
                </header>
                <div class="wrapper-site-list">
                    <ul class="site-list">
                        <li>
                            <a href="http://www.cuba.cu/gobierno/reflexiones/reflexiones.html#f091016" target="_blank">
                                <span class="bullet-point">-</span><span>Artículo del compañero Fidel. EL DESTINO INCIERTO DE LA ESPECIE HUMANA. 9 de Octubre de 2016. (Español)</span>
                            </a>
                        </li>
                <li>
            <a href="http://www.cuba.cu/gobierno/reflexiones/reflexiones.html#f120816" target="_blank">
                <span class="bullet-point">-</span><span>Artículo del compañero Fidel. EL CUMPLEAÑOS. 12 de Agosto de 2016. (Español, Francés, Portugués, Italiano, Ruso)</span>
            </a>
        </li>
                <li>
            <a href="http://www.juventudrebelde.cu/cuba/2016-04-19/el-pueblo-cubano-vencera/" target="_blank">
                <span class="bullet-point">-</span><span>Discurso pronunciado por Fidel Castro, en la clausura del VII Congreso del PCC, 19 de abril de 2016</span>
            </a>
        </li>
    
            </ul>
        </div>
        <footer>
            <div class="wrapper-view-more">
                <a href="http://www.cuba.cu/sites/politica/reflexiones-del-comandante-en-jefe-fidel-castro-ruz" target="_blank">Ver más...</a>
            </div>
        </footer>
    </article>
</div>

</div>            </section>
            <!--<section class="row section-commercial">
                            </section>-->
            <section class="row section-tags">
                <div class="col-sm-12">
                    <h2 class="section-tags-title">Tendencias</h2>
                    <nav>
                                                    <a href="http://www.cuba.cu/tag/bloqueo-contra-cuba" class="tag-l">Bloqueo contra Cuba</a>
                                                    <a href="http://www.cuba.cu/tag/relaciones-cuba-estados-unidos" class="tag-l">Relaciones Cuba-Estados Unidos</a>
                                                    <a href="http://www.cuba.cu/tag/fidel-castro" class="tag-l">Fidel Castro</a>
                                                    <a href="http://www.cuba.cu/tag/homenaje-a-fidel-castro" class="tag-l">Homenaje a Fidel Castro</a>
                                                    <a href="http://www.cuba.cu/tag/raul-castro" class="tag-l">Raúl Castro</a>
                                                    <a href="http://www.cuba.cu/tag/estados-unidos" class="tag-s">Estados Unidos</a>
                                                    <a href="http://www.cuba.cu/tag/musica" class="tag-s">Música</a>
                                                    <a href="http://www.cuba.cu/tag/solidaridad" class="tag-s">Solidaridad</a>
                                                    <a href="http://www.cuba.cu/tag/votaciones-de-la-onu" class="tag-s">Votaciones de la ONU</a>
                                                    <a href="http://www.cuba.cu/tag/dia-de-la-ciencia-cubana" class="tag-s">Día de la Ciencia cubana</a>
                                                    <a href="http://www.cuba.cu/tag/-beisbol" class="tag-s"> Béisbol</a>
                                                    <a href="http://www.cuba.cu/tag/cuba" class="tag-xl">Cuba</a>
                                                    <a href="http://www.cuba.cu/tag/feria-del-libro-de-la-habana" class="tag-xl">Feria del Libro de La Habana</a>
                                                    <a href="http://www.cuba.cu/tag/supuestos-ataques-sonicos" class="tag-xl">Supuestos ataques sónicos</a>
                                            </nav>
                </div>
            </section>
        </section>
        <aside id="right-col-home" class="col-md-3 col-sm-4">

            <article class="weather-forecast hidden-xs">
                   <!-- <h1 class="title-weather-forecast">El Tiempo hoy en:</h1>-->
    <nav class="location-weather-forecast">
        <form action="">
            <div class="dropdown">
                <button class="btn btn-default dropdown-toggle" type="button" id="location-dropdownMenu" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                    La Habana                    <span class="fontAwesomeCaret"></span>
                </button>
                <ul class="dropdown-menu" aria-labelledby="location-dropdownMenu">
                                            <li >
                            <a href="#" data-province-id="1">Pinar del Río</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="16">Artemisa</a>
                        </li>
                                            <li class="disabled">
                            <a href="#" data-province-id="2">La Habana</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="17">Mayabeque</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="4">Matanzas</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="5">Villa Clara</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="6">Cienfuegos</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="7">Sancti Spíritus</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="8">Ciego de Ávila</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="9">Camagüey</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="10">Las Tunas</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="12">Holguín</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="11">Granma</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="13">Santiago de Cuba</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="14">Guantánamo</a>
                        </li>
                                            <li >
                            <a href="#" data-province-id="15">Isla de la Juventud</a>
                        </li>
                                    </ul>
            </div>
        </form>
    </nav>
    <section class="summary-weather-forecast">
        <p class="sky-weather">Soleado</p>
        <div class="wrapper-icon-temperature">
            <p class="fonticon-weather-forecast"></p>
            <div class="wrapper-temperature">
                <p id="temperature-max" class="temperature">T. Máx. 27°C</p>
                <p id="temperature-min" class="temperature">T. Mín. 19°C</p>
            </div>
            <div class="clearfix"></div>
        </div>
        <!--<p class="summary-forecast">
                    </p>-->
    </section>
    <nav >
        <a href="http://tiempo.cuba.cu/" target="_blank" class="see-more-weather">ver pron&oacute;stico completo</a>
    </nav>
            </article>
            <article class="rate-change hidden-xs">
                    <h1 class="title-rate-change">Cambio de moneda</h1>
    <nav class="coin-type-rate-change">
        <form action="">
            <div class="dropdown">
                <button class="btn btn-default dropdown-toggle" type="button" id="rate-change-dropdownMenu" data-toggle="dropdown" aria-haspopup="true" aria-expanded="true">
                    1 EUR                    <span class="fontAwesomeCaret"></span>
                </button>
                <ul class="dropdown-menu" aria-labelledby="rate-change-dropdownMenu">
                                            <li class="disabled">
                            <a href="#" data-rate-id="1">1 EUR</a>
                        </li>
                                            <li >
                            <a href="#" data-rate-id="2">1 GBP</a>
                        </li>
                                            <li >
                            <a href="#" data-rate-id="3">1 CAD</a>
                        </li>
                                            <li >
                            <a href="#" data-rate-id="4">1 JPY</a>
                        </li>
                                            <li >
                            <a href="#" data-rate-id="5">1 MXN</a>
                        </li>
                                            <li >
                            <a href="#" data-rate-id="6">1 USD</a>
                        </li>
                                    </ul>
            </div>
        </form>
    </nav>
    <section class="result-change">
        <h2 class="title-result-change">Equivale a:</h2>
        <p class="result">
            1.19777 <span>CUC</span>
        </p>
    </section>
    <nav class="nav-bank">
        <!--<a href="http://www.bc.gob.cu/Espanol/tipo_cambio_METROPOLITANO.asp" target="_blank">Tasas del Banco Metropolitano</a>-->
        <a>Tasas del Banco Metropolitano</a>
    </nav>            </article>


            <section class="info-useful-home">
                <h1 class="title-info-useful-home">
                    información útil
                </h1>
                                    <ul class="info-useful-list">
                                                    <li>
                                <article class="info-useful-item">
                                    <a href="http://www.cuba.cu/infouseful#llamadas-telefonicas" class="info-useful-link"><span class="info-useful-icon"></span> Llamadas telefónicas</a>
                                </article>
                            </li>
                                                    <li>
                                <article class="info-useful-item">
                                    <a href="http://www.cuba.cu/infouseful#informacion-general" class="info-useful-link"><span class="info-useful-icon"></span> Información general</a>
                                </article>
                            </li>
                                                    <li>
                                <article class="info-useful-item">
                                    <a href="http://www.cuba.cu/infouseful#mapas-de-cuba" class="info-useful-link"><span class="info-useful-icon"></span> Mapas de Cuba</a>
                                </article>
                            </li>
                        <!--                        <li>-->
<!--                            <article class="info-useful-item news-archive-link">-->
<!--                                <a href="--><!--" class="info-useful-link"><span class="info-useful-icon"></span> Archivo de Noticias</a>-->
<!--                            </article>-->
<!--                        </li>-->
                    </ul>
                
            </section>
            <section class="banner-aside-home">
                <ul class="banner-aside-home-list">
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://moviles.compra-dtodo.com/" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/ofertamarzo-264.jpg" alt="Oferta de recarga de móviles" title="Oferta de recarga de móviles"/></a></article>
                        </li>
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://www.compraspacuba.com/departamento.php?id_dpto=733" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/compra-dtodo.jpg" alt="CompraDtodo" title="CompraDtodo"/></a></article>
                        </li>
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://www.administresunegocio.cu/asn/index" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/administre-negocio-principal_0_thumb.JPG" alt="Administre su negocio" title="Administre su negocio"/></a></article>
                        </li>
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://reservahoteles.cuba.cu/" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/banners_laterales_Reserv_hoteles.jpg" alt="Reserva de hoteles en Cuba" title="Reserva de hoteles en Cuba"/></a></article>
                        </li>
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://www.libreriavirtualcuba.com" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/libreria.jpg" alt="Librería Virtual" title="Librería Virtual"/></a></article>
                        </li>
                                            <li>
                            <article class="banner-aside-home-item"><a href="http://www.travelnetcuba.it/es/" target="_blank" ><img src="http://www.cuba.cu/imgs/banner/travelnet.jpg" alt="Travelnet" title="Travelnet"/></a></article>
                        </li>
                                    </ul>
            </section>

            <div id="suscription">
                <section class="subscription-home">
    <h1 class="suscription-home-title">
        Suscribirse
    </h1>

        <div class="formSuscription">
        <form id="formSuscription" action="http://www.cuba.cu/suscription/validate" method="post" class="suggestion-form">
            
            <input type="hidden" name="csrf_test_name" value="cd5d6e57444715896bc6a7a1026d94fb" />
            <p>
                Para recibir o anular la suscripci&oacute;n al Bolet&iacute;n del Portal Cuba v&iacute;a correo electr&oacute;nico, con informaci&oacute;n actual sobre nuestro pa&iacute;s, por favor reg&iacute;strese escribiendo su direcci&oacute;n de correo en el recuadro que aparece a continuaci&oacute;n e indique su solicitud.
            </p>
            <div class="form-group ">
                <input required oninvalid="this.setCustomValidity('Llene este campo')" onchange="this.setCustomValidity('')" id="email" name="email" type="text" value="" class="form-control email-box" placeholder="Correo electr&oacute;nico">
                <label class="error" style="display: none;"></label>
            </div>
            <div class="form-group">
                <label><input type="radio" name="optradio" value="1" checked> Suscribirse (recibir correos)</label>
                <label><input type="radio" name="optradio" value="2"> Anular suscripci&oacute;n </label>
            </div>
            <div class="wrapper-catcha-submit">
                <button type="submit" class="btn">ENVIAR</button>
            </div>
        </form>
    </div>
</section>            </div>

            <section>
                <div id="fb-root"></div>
                <script>(function(d, s, id) {
                  var js, fjs = d.getElementsByTagName(s)[0];
                  if (d.getElementById(id)) return;
                  js = d.createElement(s); js.id = id;
                  js.src = "//connect.facebook.net/es_ES/sdk.js#xfbml=1&version=v2.4";
                  fjs.parentNode.insertBefore(js, fjs);
                }(document, 'script', 'facebook-jssdk'));</script>
                        <div class="fb-page" data-href="https://www.facebook.com/portalcuba" data-width="264" data-small-header="false" data-adapt-container-width="true" data-hide-cover="false" data-show-facepile="true" data-show-posts="false"><div class="fb-xfbml-parse-ignore"><blockquote cite="https://www.facebook.com/portalcuba"><a href="https://www.facebook.com/portalcuba">Portal Cuba</a></blockquote></div></div>                
            </section>
            
            <section>
                <a class="twitter-timeline" lang="es" href="https://twitter.com/portalcuba" data-widget-id="644958753377267712">Tweets por el @portalcuba.</a> <script>!function(d,s,id){var js,fjs=d.getElementsByTagName(s)[0],p=/^http:/.test(d.location)?'http':'https';if(!d.getElementById(id)){js=d.createElement(s);js.id=id;js.src=p+"://platform.twitter.com/widgets.js";fjs.parentNode.insertBefore(js,fjs);}}(document,"script","twitter-wjs");</script> 
            </section>
        </aside>
    </div>
</div>    <footer>
<!--            <div class="container">
                <div class="row">
                    <div class="col-sm-12">
                        <a href="https://twitter.com/portalcuba" class="twitter-follow-button" data-show-count="true">Follow @portalcuba</a> <script>!function(d,s,id){var js,fjs=d.getElementsByTagName(s)[0],p=/^http:/.test(d.location)?'http':'https';if(!d.getElementById(id)){js=d.createElement(s);js.id=id;js.src=p+'://platform.twitter.com/widgets.js';fjs.parentNode.insertBefore(js,fjs);}}(document, 'script', 'twitter-wjs');</script>
                    </div>
                    <div class="col-sm-6">
                        <div id="fb-root"></div>
<script>(function(d, s, id) {
  var js, fjs = d.getElementsByTagName(s)[0];
  if (d.getElementById(id)) return;
  js = d.createElement(s); js.id = id;
  js.src = "//connect.facebook.net/es_ES/sdk.js#xfbml=1&version=v2.4";
  fjs.parentNode.insertBefore(js, fjs);
}(document, 'script', 'facebook-jssdk'));</script>
                        <div class="fb-page" data-href="https://www.facebook.com/portalcuba" data-width="264" data-small-header="false" data-adapt-container-width="true" data-hide-cover="false" data-show-facepile="true" data-show-posts="false"><div class="fb-xfbml-parse-ignore"><blockquote cite="https://www.facebook.com/portalcuba"><a href="https://www.facebook.com/portalcuba">Portal Cuba</a></blockquote></div></div>
                    </div>
                    <div class="col-sm-6">
                        <a class="twitter-timeline" href="https://twitter.com/portalcuba" data-widget-id="644958753377267712">Tweets por el @portalcuba.</a> <script>!function(d,s,id){var js,fjs=d.getElementsByTagName(s)[0],p=/^http:/.test(d.location)?'http':'https';if(!d.getElementById(id)){js=d.createElement(s);js.id=id;js.src=p+"://platform.twitter.com/widgets.js";fjs.parentNode.insertBefore(js,fjs);}}(document,"script","twitter-wjs");</script> 
                    </div>
                </div>                
            </div>-->
            <div class="container">
                <div class="row">
                    <div class="col-sm-3">
                        <nav class="info-link">
                            <ul class="internal-link">

                                                                    <li><a title="Quiénes Somos" href="http://www.cuba.cu/quienes-somos">Quiénes Somos</a></li>
                                
                                <li><a href="mailto:webmaster@cuba.cu">contáctenos</a></li>
                                <!--<li><a href="">mapa del sitio</a></li>-->
                                <!--<li><a href="questions">preguntas frecuentes</a></li>
                                <li><a href="privacy_policy">políticas de privacidad</a></li>-->
                            </ul>
                            <ul class="external-link">
                                <li><a target="_blank" href="http://www.cuba.cu/feed" title="RSS"></a></li>
                                <li><a href="https://twitter.com/portalcuba" target="_blank" title="Twitter"></a></li>
                                <li><a href="https://www.facebook.com/portalcuba" target="_blank" title="Facebook"></a></li>
                                <div class="clearfix"></div>
                            </ul>
                        </nav>
                    </div>
                    <div id="comment" class="col-sm-6">
                        <h1>opine o recomiende</h1>

<form id="formComments" action="http://www.cuba.cu/comments/add" method="post" class="suggestion-form">
        
    <input type="hidden" name="csrf_test_name" value="cd5d6e57444715896bc6a7a1026d94fb" />
    <div class="form-group ">
        <input required oninvalid="this.setCustomValidity('Llene este campo')" onchange="this.setCustomValidity('')" id="email" name="email" type="text" value="" class="form-control email-box" placeholder="Correo electrónico">
        <label class="error" style="display: none;"></label>
    </div>
    <div class="form-group ">
        <textarea required oninvalid="this.setCustomValidity('Llene este campo')" onchange="this.setCustomValidity('')" name="suggestion" class="form-control suggestion-box"  placeholder="Comentario u opinión"></textarea>
        <label class="error" style="display: none;"></label>
    </div>
    <p>Escriba los caracteres de la imagen aqu&iacute; <span class="here-box fontAwesomeCaret"> </span></p>
    <div class="wrapper-catcha">
        <div class="wrapper-catcha-img">
            <img id="siimage" src="http://www.cuba.cu/captcha/show" alt="capcha" />
            <p>
                Mostrar otra imagen
                <span class="refresh-box fontAwesomeCaret" onclick="changeCaptcha('http://www.cuba.cu/captcha/show')" onmouseover="" style="cursor:pointer;"></span>
            </p>
        </div>
        <div class="form-group  wrapper-catcha-input">
            <input required oninvalid="this.setCustomValidity('Llene este campo')" onchange="this.setCustomValidity('')" type="text" name="captcha" class="form-control captcha-box" placeholder="">
            <label class="error" style="display: none;"></label>
        </div>
        <div class="wrapper-catcha-submit">
            <button type="submit" class="btn btn-primary submit-box">ENVIAR</button>
        </div>
        <div class="clearfix"></div>
    </div>

</form>

<script type="text/javascript">
    
    function changeCaptcha(url) {
        $('#siimage').attr('src', url +'/' + Math.random());      
    }     


</script>
                    </div>
                    <div class="col-sm-3">
                        <nav class="site-more-see">
                            <h1>lo más visitado</h1>
                            <ul>
                                                                <li><a href="http://www.cenais.cu/" target="_blank">Centro Nacional de Investigaciones Sismológicas, CENAIS</a></li>
                                                                <li><a href="http://www.one.cu/" target="_blank">Oficina Nacional de Estadísticas, ONE</a></li>
                                                            </ul>
                        </nav>
                    </div>
                </div>
                <div class="row"></div>
            </div>
            <div id="copyright" class="container-fluid">
                <div class="row">
                    <div class="col-sm-12">
                        <p>© Copyright 1997 - 2018 <a href="http://www.citmatel.cu" target="_blank">CITMATEL</a> ®. Todos los derechos reservados.</p>
                    </div>
                </div>
            </div>
        </footer>        
        <!--Script-->
        <!--<script src="https://ajax.googleapis.com/ajax/libs/jquery/1.11.1/jquery.min.js"></script>-->
        <!--<script src="http://maxcdn.bootstrapcdn.com/bootstrap/3.2.0/js/bootstrap.min.js"></script>-->
        <!--JQuery v1.11.2-->
        <script src="http://www.cuba.cu/assets/js/lib/jq/jquery-1.11.2.min.js"></script>
        <!--<script src="jq/jquery-2.1.3.js"></script>-->
        <!--Isotope js-->
        <script src="http://www.cuba.cu/assets/js/lib/isotope/isotope.pkgd.min.js"></script>
        <!--Bootstrap js-->
        <script src="http://www.cuba.cu/assets/bs/js/bootstrap.min.js"></script >
        <script src="http://www.cuba.cu/assets/js/main.js"></script>
        <!---Datepiker js-->
        <script src="http://www.cuba.cu/assets/js/lib/datepiker/js/bootstrap-datepicker.min.js"></script>
        <script src="http://www.cuba.cu/assets/js/lib/datepiker/locales/bootstrap-datepicker.es.min.js"></script>
        <script src="http://www.cuba.cu/assets/js/lib/datepiker/locales/bootstrap-datepicker.en.min.js"></script>
        <script src="http://www.cuba.cu/assets/js/jquery.validate.min.js"></script>
        <script src="http://www.cuba.cu/assets/js/additional-methods.min.js"></script>
        
        <script type="text/javascript">
            $.validator.methods.email = function( value, element ) {
                return this.optional( element ) || /^[_a-z0-9-]+(\.[_a-z0-9-]+)*@[a-z0-9-]+(\.[a-z0-9-]+)*(\.[a-z]{2,3})$/.test( value );
            }
            
            jQuery.extend(jQuery.validator.messages, {
                required : 'Campo requerido'
                , email: 'Correo no válido'
            });

            $('#formComments').validate({
                rules: {
                  email: {
                    email: true
                  }
                }
            });

            $('#formSuscription').validate({
                rules: {
                    email: {
                        email: true
                    }
                }
            });
        </script>
        <!-- Go to www.addthis.com/dashboard to customize your tools -->
        <script type="text/javascript" src="//s7.addthis.com/js/300/addthis_widget.js#pubid=ra-570519489936e5a0"></script>
    </body>
</html>