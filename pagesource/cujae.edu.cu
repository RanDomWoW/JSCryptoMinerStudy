#include "hip/hip_runtime.h"
<!DOCTYPE html PUBLIC "-//W3C//DTD XHTML 1.0 Transitional//EN" "http://www.w3.org/TR/xhtml1/DTD/xhtml1-transitional.dtd">
<html prefix="og: http://ogp.me/ns#" itemscope itemtype="http://schema.org/Organization" xmlns="http://www.w3.org/1999/xhtml" xml:lang="es-es" lang="es-es" dir="ltr" >
<head>

<!-- <link type="text/css" href="/cometchat/cometchatcss.php" rel="stylesheet" charset="utf-8">
<script type="text/javascript" src="/cometchat/cometchatjs.php" charset="utf-8"></script> -->
<meta name="alexaVerifyID" content="9_WX5iriAhKViZzv7mJHe_HVycg" />
<meta name="msvalidate.01" content="E38A3E82687D967340D5C4634C746E0A" />
  <meta http-equiv="content-type" content="text/html; charset=utf-8" />
  <meta name="keywords" content="instituto, politecnico, universidad, carreras, tecnicas, ingenieria, ingeniero, arquitecto, quimica, civil, electrica, geofisica, informatica, industrial, mecanica, metalurgica, investigaciones, hidraulica, automatica, biomedica" />
  <meta property="og:url" content="http://cujae.edu.cu/" />
  <meta property="og:title" content="Universidad Tecnológica de La Habana José Antonio Echeverría" />
  <meta property="og:type" content="Article" />
  
  <title>Universidad Tecnológica de La Habana José Antonio Echeverría</title>
  <link href="/templates/lecujae/favicon.ico" rel="shortcut icon" type="image/vnd.microsoft.icon" />
  <link rel="stylesheet" href="/media/system/css/modal.css" type="text/css" />
  <link rel="stylesheet" href="/components/com_k2/css/k2.css" type="text/css" />
  <link rel="stylesheet" href="/plugins/content/jplayer/tmpl/css/style.css" type="text/css" />
  <link rel="stylesheet" href="http://cujae.edu.cu/modules/mod_djimageslider/assets/style.css" type="text/css" />
  <link rel="stylesheet" href="http://cujae.edu.cu/modules/mod_maximenuck/themes/cujae/css/moo_maximenuhck.css" type="text/css" />
  <link rel="stylesheet" href="http://cujae.edu.cu/modules/mod_maximenuck/themes/cujae/css/maximenuhck.php?monid=maximenuck" type="text/css" />
  <link rel="stylesheet" href="http://cujae.edu.cu/modules/mod_maximenuck/assets/maximenuck.css" type="text/css" />
  <link rel="stylesheet" href="http://cujae.edu.cu/modules/mod_socialmedialinks/style.css" type="text/css" />
  <link rel="stylesheet" href="/media/mod_falang/css/template.css" type="text/css" />
  <style type="text/css">

		/* Styles for DJ Image Slider with module id 35 */
		#djslider-loader35 {
			margin: 0 auto;
			position: relative;
			height: 133px; 
			width: 544px;
		}
		#djslider35 {
			margin: 0 auto;
			position: relative;
			height: 133px; 
			width: 544px;
			display: none;
		}
		#slider-container35 {
			position: absolute;
			overflow:hidden;
			left: 0; 
			top: 0;
			height: 133px; 
			width: 544px;			
		}
		#djslider35 ul#slider35 {
			margin: 0 !important;
			padding: 0 !important;
			border: 0 !important;
		}
		#djslider35 ul#slider35 li {
			list-style: none outside !important;
			float: left;
			margin: 0 !important;
			border: 0 !important;
			padding: 0 8px 0px 0 !important;
			position: relative;
			height: 133px;
			width: 84px;
			background: none;
			overflow: hidden;
		}
		#slider35 li img {
			width: 84px;
			height: auto;
			border: 0 !important;
			margin: 0 !important;
		}
		#slider35 li a img, #slider35 li a:hover img {
			border: 0 !important;
		}
		
		/* Slide description area */
		#slider35 .slide-desc {
			position: absolute;
			bottom: 0px;
			left: 0px;
			width: 84px;
		}
		#slider35 .slide-desc-in {
			position: relative;
		}
		#slider35 .slide-desc-bg {
			position:absolute;
			top: 0;
			left: 0;
			width: 100%;
			height: 100%;
		}
		#slider35 .slide-desc-text {
			position: relative;
		}
		#slider35 .slide-desc-text h3 {
			display: block !important;
		}
		
		/* Navigation buttons */
		#navigation35 {
			position: relative;
			top: 58px; 
			margin: 0 -35px;
			text-align: center !important;
		}
		#prev35 {
			cursor: pointer;
			display: block;
			position: absolute;
			left: 0;
			
		}
		#next35 {
			cursor: pointer;
			display: block;
			position: absolute;
			right: 0;
			
		}
		#play35, 
		#pause35 {
			cursor: pointer;
			display: block;
			position: absolute;
			left: 47%;
			top: -99999px;
		}
		#cust-navigation35 {
			position: absolute;
			top: 10px;
			right: 10px;
			z-index: 15;
			display: none;
		}
		
		/* Styles for DJ Image Slider with module id 42 */
		#djslider-loader42 {
			margin: 0 auto;
			position: relative;
			height: 170px; 
			width: 984px;
		}
		#djslider42 {
			margin: 0 auto;
			position: relative;
			height: 170px; 
			width: 984px;
			display: none;
		}
		#slider-container42 {
			position: absolute;
			overflow:hidden;
			left: 0; 
			top: 0;
			height: 170px; 
			width: 984px;			
		}
		#djslider42 ul#slider42 {
			margin: 0 !important;
			padding: 0 !important;
			border: 0 !important;
		}
		#djslider42 ul#slider42 li {
			list-style: none outside !important;
			float: left;
			margin: 0 !important;
			border: 0 !important;
			padding: 0 0px 0px 0 !important;
			position: relative;
			height: 170px;
			width: 984px;
			background: none;
			overflow: hidden;
		}
		#slider42 li img {
			width: 984px;
			height: auto;
			border: 0 !important;
			margin: 0 !important;
		}
		#slider42 li a img, #slider42 li a:hover img {
			border: 0 !important;
		}
		
		/* Slide description area */
		#slider42 .slide-desc {
			position: absolute;
			bottom: 0px;
			left: 0px;
			width: 984px;
		}
		#slider42 .slide-desc-in {
			position: relative;
		}
		#slider42 .slide-desc-bg {
			position:absolute;
			top: 0;
			left: 0;
			width: 100%;
			height: 100%;
		}
		#slider42 .slide-desc-text {
			position: relative;
		}
		#slider42 .slide-desc-text h3 {
			display: block !important;
		}
		
		/* Navigation buttons */
		#navigation42 {
			position: relative;
			top: 70px; 
			margin: 0 5px;
			text-align: center !important;
		}
		#prev42 {
			cursor: pointer;
			display: block;
			position: absolute;
			left: 0;
			
		}
		#next42 {
			cursor: pointer;
			display: block;
			position: absolute;
			right: 0;
			
		}
		#play42, 
		#pause42 {
			cursor: pointer;
			display: block;
			position: absolute;
			left: 47%;
			top: -99999px;
		}
		#cust-navigation42 {
			position: absolute;
			top: 10px;
			right: 10px;
			z-index: 15;
			display: none;
		}
		
  </style>
  <script src="/media/system/js/mootools-core.js" type="text/javascript"></script>
  <script src="/media/system/js/core.js" type="text/javascript"></script>
  <script src="/media/system/js/modal.js" type="text/javascript"></script>
  <script src="/media/k2/assets/js/jquery-1.7.2.min.js" type="text/javascript"></script>
  <script src="/media/k2/assets/js/k2.noconflict.js" type="text/javascript"></script>
  <script src="/components/com_k2/js/k2.js" type="text/javascript"></script>
  <script src="/media/system/js/mootools-more.js" type="text/javascript"></script>
  <script src="/modules/mod_djimageslider/assets/slider.js" type="text/javascript"></script>
  <script type="text/javascript">

		window.addEvent('domready', function() {

			SqueezeBox.initialize({});
			SqueezeBox.assign($$('a.modal'), {
				parse: 'rel'
			});
		});
var K2SitePath = '/';
(function($){ window.addEvent('domready',function(){var Slider35 = new DJImageSlider({id: '35', slider_type: 0, slide_size: 92, visible_slides: 6, show_buttons: 0, show_arrows: 2, preload: 0},{auto: 1, transition: Fx.Transitions.Circ.easeInOut, duration: 300, delay: 3300})}); })(document.id);
<!--
				if (window.attachEvent) window.attachEvent("onload", function() {
				var sfEls = document.getElementById("maximenuck").getElementsByTagName("li");
				for (var i=0; i<sfEls.length; i++) {

					sfEls[i].onmouseover=function() {
						this.className+=" sfhover";
					}

					sfEls[i].onmouseout=function() {
						this.className=this.className.replace(new RegExp(" sfhover\\b"), "");
					}
				}
				});
				//-->
(function($){ window.addEvent('domready',function(){var Slider42 = new DJImageSlider({id: '42', slider_type: 2, slide_size: 984, visible_slides: 1, show_buttons: 0, show_arrows: 1, preload: 0},{auto: 1, transition: Fx.Transitions.Cubic.easeInOut, duration: 3000, delay: 13000})}); })(document.id);
  </script>
  <style type="text/css" media="screen">

#innerTC1 .cloudTC1  {padding:3px; text-decoration: none;line-height:102%; }
#innerTC1 .cloudTC1:link  {color:#056D54; }
#innerTC1 .cloudTC1:visited {color:#999; }
#innerTC1 .cloudTC1:hover {color:#fff;background:#999;text-decoration:none; }
#innerTC1 .cloudTC1:active {color:#fff;background:#999; }#holderTC1{margin-top:5px;margin-bottom:5px;margin-top:5px;overflow:hidden;width:100%;} #holderTC1 table, #holderTC1 tr , #holderTC1 td{padding:0px;margin:0px;border:0px;border-collapse:collapse;}

#holderTC1  p, #holderTC1 h6, #holderTC1 h5, #holderTC1 h4, #holderTC1 h3, #holderTC1 h2, #holderTC1 h1 {font-weight:400;color:#000;text-align:center;}#contentcellTC1 {
padding:3px;}

</style>

<meta itemprop="name" content="Universidad Tecnológica de La Habana José Antonio Echeverría | Universidad Cubana">
<meta itemprop="description" content="Centro de Altos Estudios, rector de las carreras técnicas en Cuba: Ingenierías y Arquitectura">
<meta itemprop="image" content="http://cujae.edu.cu/logo-cujae.jpg">
<link rel="stylesheet" href="/templates/system/css/system.css" type="text/css" />
<link rel="stylesheet" href="/templates/lecujae/css/position.css" type="text/css" media="screen,projection" />
<link rel="stylesheet" href="/templates/lecujae/css/layout.css" type="text/css" media="screen,projection" />
<link rel="stylesheet" href="/templates/lecujae/css/print.css" type="text/css" media="Print" />
<link rel="stylesheet" href="/templates/lecujae/css/template.css" type="text/css" />
<link rel="stylesheet" href="/templates/lecujae/css/personal.css" type="text/css" />
<link rel="stylesheet" href="/templates/lecujae/css/general.css" type="text/css" />
<link rel="stylesheet" href="/templates/lecujae/css/cujae.css" type="text/css" />
<!--[if lte IE 6]>
        <link href="/templates/lecujae/css/ieonly.css" rel="stylesheet" type="text/css" />
                  <![endif]-->
<!--[if IE 7]>
          <link href="/templates/lecujae/css/ie7only.css" rel="stylesheet" type="text/css" />
        <![endif]-->
<script type="text/javascript" src="/templates/lecujae/javascript/hide.js"></script>
<script type="text/javascript">
                        var big ='72%';
                        var small='53%';
                        var altopen='esta abierto';
                        var altclose='esta cerrado';
                        var bildauf='/templates/lecujae/images/plus.png';
                        var bildzu='/templates/lecujae/images/minus.png';
                        var rightopen='Abrir información';
                        var rightclose='Cerrar información';
                </script>
<script type="text/javascript">
  var _gaq = _gaq || [];
  _gaq.push(['_setAccount', 'UA-7016837-1']);
  _gaq.push(['_setDomainName', 'cujae.edu.cu']);
  _gaq.push(['_trackPageview']);
  (function() {
    var ga = document.createElement('script'); ga.type = 'text/javascript'; ga.async = true;
    ga.src = ('https:' == document.location.protocol ? 'https://ssl' : 'http://www') + '.google-analytics.com/ga.js';
    var s = document.getElementsByTagName('script')[0]; s.parentNode.insertBefore(ga, s);
  })();
</script>
<link rel="stylesheet" href="http://cujae.edu.cu/components/com_jbolo/css/chat.css" type="text/css" />
<link rel="stylesheet" href="http://cujae.edu.cu/components/com_jbolo/css/jbolo.css" type="text/css" />

<script type="text/javascript" src="/component/com_jbolo/format,raw/view,js/"> </script>
<script type="text/javascript" src="http://cujae.edu.cu/media/k2/assets/js/jquery-1.7.2.min.js"> </script>
<script type="text/javascript"> $.noConflict(); </script>
<![if !(IE 6)]>
<link href="http://cujae.edu.cu/components/com_jbolo/css/jfb_normal.css" type="text/css" rel="stylesheet"/>
<![endif]>
<!--[if IE 6]>
<link href="http://cujae.edu.cu/components/com_jbolo/css/jfb_ie6.css" type="text/css" rel="stylesheet"/>
<![endif]-->
<!--[if lte IE 7]>
<link href="http://cujae.edu.cu/components/com_jbolo/css/screen_ie.css" type="text/css" rel="stylesheet"/>
<![endif]-->
<script type="text/javascript" src="http://cujae.edu.cu/components/com_jbolo/sound/soundmanager2.js"> </script>
<script type="text/javascript" src="http://cujae.edu.cu/components/com_jbolo/js/fb_chat.js"> </script>
<script type="text/javascript" src="http://cujae.edu.cu/components/com_jbolo/js/fbar.js"> </script></head>
<body>
<div class="back_all">
  <div class="back_img">
    <div class="contenedor">
      <div class="logo"><a href="/"><img src="/templates/lecujae/images/cujae/spacer.gif" width="82" height="97" alt="Logo" /></a></div>
    <div class="cabezal">    
        <div class="derecha">
          <div class="idioma"><div class="mod-languages">

	<ul class="lang-inline">
						 
			<li class="lang-active">            
				<img src="/media/mod_falang/images/es-a.jpg" alt="Español" title="Español" />                Español			</li>
            								            <li>            
			<a href="/en/">
				<img src="/media/mod_falang/images/en.jpg" alt="English" title="English" />				English			</a>
			</li>
            
            				</ul>

</div>
</div>
          <div class="sociales">
            <div class="smile" style="text-align: right "> <noindex><a style="margin:2px;" href="http://www.facebook.com/cujae.oficial" target="_blank" rel="nofollow" ><img src="http://cujae.edu.cu/modules/mod_socialmedialinks/icons/default/size2/facebook.png " alt="Facebook" title="Síguenos en Facebook" /></a></noindex><noindex><a style="margin:2px;" href="https://twitter.com/CujaeRedsocial" target="_blank" rel="nofollow" ><img src="http://cujae.edu.cu/modules/mod_socialmedialinks/icons/default/size2/twitter.png " alt="Twitter" title="Síguenos en Twitter" /></a></noindex><noindex><a style="margin:2px;" href="http://www.youtube.com/user/cujaeoficial" target="_blank" rel="nofollow" ><img src="http://cujae.edu.cu/modules/mod_socialmedialinks/icons/default/size2/youtube.png " alt="Youtube" title="Síguenos en Youtube" /></a></noindex><a style="margin:2px;"  href="/rss/cujae" target="_blank"><img src="http://cujae.edu.cu/modules/mod_socialmedialinks/icons/default/size2/feed.png " alt="Feed" title="RSS Feed" /></a>	</div>
    <div class="clr"></div>

                        <g:plusone size="medium"></g:plusone>
            <script type="text/javascript">
              window.___gcfg = {lang: 'es'};
                (function() {
                  var po = document.createElement('script'); po.type = 'text/javascript'; po.async = true;
                  po.src = 'https://apis.google.com/js/plusone.js';
                  var s = document.getElementsByTagName('script')[0]; s.parentNode.insertBefore(po, s);
                })();
            </script>
                      </div>
          
      <div class="acceso">
            <form action="http://cujae.edu.cu/acceder" method="post" id="login-form" class="cbLoginForm" style="margin:0px;">
&nbsp;
<span><input type="text" name="username" id="mod_login_username" class="inputbox" size="10" style="background-image:url(http://cujae.edu.cu/modules/mod_cblogin/mod_cblogin/username.png); background-repeat: no-repeat; background-position: 0px 0px; padding-left: 30px; min-height: 18px;width:auto; background-color:#fff; font-size: 10px; margin-bottom: 5px; min-height: 14px; padding: 1px 1px 1px 22px;"  alt="Usuario" value="Usuario" onfocus="if (this.value=='Usuario') this.value=''" onblur="if(this.value=='') { this.value='Usuario'; return false; }" title="Usuario" /></span>&nbsp;
&nbsp;<span><input type="password" name="passwd" id="mod_login_password" class="inputbox" size="10" style="background-image:url(http://cujae.edu.cu/modules/mod_cblogin/mod_cblogin/password.png); background-repeat: no-repeat; background-position: 0px 0px; padding-left: 30px; min-height: 18px;width:auto; background-color:#fff; font-size: 10px; margin-bottom: 5px; min-height: 14px; padding: 1px 1px 1px 22px;"  alt="Contraseña" value="paswww" onfocus="if (this.value=='paswww') this.value=''" onblur="if(this.value=='') { this.value='paswww'; return false; }" title="Contraseña" /></span>&nbsp;
<input type="hidden" name="op2" value="login" />
<input type="hidden" name="lang" value="espa" />
<input type="hidden" name="force_session" value="1" />
<input type="hidden" name="return" value="B:aHR0cDovL2N1amFlLmVkdS5jdS8=" />
<input type="hidden" name="message" value="1" />
<input type="hidden" name="loginfrom" value="loginmodule" />
<input type="hidden" name="cbsecuritym3" value="cbm_7f1d4fa6_051a9f5b_b58fb99c0cad98a3a3fe1858f0a44aea" />
<span class="cbLoginButtonSpan"><input type="submit" name="Submit" class="button" value="Acceder" /></span><br />&nbsp;&nbsp;
<span id="mod_login_remembermetext"><label for="mod_login_remember">Recordarme</label></span><span style="vertical-align:bottom;"><input type="checkbox" name="remember" id="mod_login_remember" value="yes" /></span>&nbsp;|&nbsp;<a href="http://cujae.edu.cu/recuperar-clave" class="mod_login">¿Olvido acceso?</a>&nbsp;|&nbsp;<span id="mod_login_noaccount"><a href="http://cujae.edu.cu/registro" class="mod_login">Registrarse</a></span>
</form>
          </div>
        </div>
      </div>
      <div class="slider">
                
	<div id="djslider-loader42" class="djslider-loader">
    <div id="djslider42" class="djslider">
        <div id="slider-container42" class="slider-container">
        	<ul id="slider42">
          		          			<li>
            										<a  href="/investigaciones/publicaciones" target="_self">
													<img src="/images/top/publicaciones.jpg" alt="Nuestras Revistas Científicas" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
									<a href="/investigaciones/publicaciones" target="_self">										Nuestras Revistas Científicas									</a>								</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="http://lv.cujae.edu.cu" target="_blank">
													<img src="/images/top/laboratorios-virtuales.jpg" alt="Laboratorios Virtuales. Simulación de Procesos Tecnológicos" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
									<a href="http://lv.cujae.edu.cu" target="_blank">										Laboratorios Virtuales. Simulación de Procesos Tecnológicos									</a>								</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/posgrado" target="_self">
													<img src="/images/top/posgrado.jpg" alt="Posgrados en la Cujae" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
									<a href="/posgrado" target="_self">										Posgrados en la Cujae									</a>								</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                        	</ul>
        </div>
        <div id="navigation42" class="navigation-container">
        	<img id="prev42" class="prev-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/prev.png" alt="Previous" />
			<img id="next42" class="next-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/next.png" alt="Next" />
			<img id="play42" class="play-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/play.png" alt="Play" />
			<img id="pause42" class="pause-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/pause.png" alt="Pause" />
        </div>
		<div id="cust-navigation42" class="navigation-container-custom">
							<span class="load-button load-button-active"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
			        </div>
    </div>
	</div>
	
	<div style="clear: both"></div>
              </div>
      <div class="navega">
                <!-- debut Maximenu CK, par cedric keiflin sur http://www.joomlack.fr -->
        <div class="maximenuckh" id="maximenuck" style="z-index:10;">
        <div class="maxiroundedleft"></div>
        <div class="maxiroundedcenter">
            <ul class="menu maximenuck" style="">
<li class="maximenuck item101 current active first level1 " style="z-index : 12000;"><a class="maximenuck " href="/"><span class="titreck">Inicio<span class="descck">Portal</span></span></a>
		</li>
<li class="maximenuck item102 parent level1 " style="z-index : 11999;"><span class="separator "><span class="titreck">Quienes somos<span class="descck">Conoce la Cujae</span></span></span>
	<div class="floatck" style="width:360px;"><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first "  style="width:180px;float:left;">
	<ul class="maximenuck2"><li class="maximenuck item104 first level2 " style="z-index : 11998;"><a class="maximenuck " href="/quienes-somos/bienvenida"><span class="titreck">Bienvenida</span></a><li class="maximenuck item105 level2 " style="z-index : 11997;"><a class="maximenuck " href="/quienes-somos/presentacion"><span class="titreck">Presentación</span></a>
		</li>
<li class="maximenuck item152 level2 " style="z-index : 11996;"><a class="maximenuck " href="/quienes-somos/mision-y-vision-de-la-cujae"><span class="titreck">Misión y Visión<span class="descck">de la Cujae</span></span></a>
		</li>
<li class="maximenuck item187 level2 " style="z-index : 11995;"><a class="maximenuck " href="/quienes-somos/estructura-organizativa"><span class="titreck">Estructura organizativa</span></a>
		</li>
<li class="maximenuck item933 level2 " style="z-index : 11994;"><a class="maximenuck " href="/consejo-universitario"><span class="titreck">Consejo Universitario<span class="descck">Nos dirigen...</span></span></a>
		</li>
<li class="maximenuck item153 level2 " style="z-index : 11993;"><a class="maximenuck " href="/quienes-somos/historia"><span class="titreck">Historia</span></a>
		</li>
<li class="maximenuck item946 level2 " style="z-index : 11992;"><a class="maximenuck " href="/quienes-somos/nuestros-simbolos"><span class="titreck">Nuestros Símbolos</span></a>
		</li>
<li class="maximenuck item944 level2 " style="z-index : 11991;"><a class="maximenuck " href="/quienes-somos/himno-de-la-cujae"><span class="titreck">Himno de la Cujae</span></a>
		</li>
<li class="maximenuck item945 level2 " style="z-index : 11990;"><a class="maximenuck " href="/quienes-somos/decanos-y-rectores-de-la-cujae"><span class="titreck">Decanos y Rectores<span class="descck">de la cujae</span></span></a>
		</li>
<li class="maximenuck item708 level2 " style="z-index : 11989;"><a class="maximenuck " href="/quienes-somos/jose-antonio-echeverria"><span class="titreck">José Antonio Echeverría<span class="descck">Resumen biográfico</span></span></a>
		</li>
<li class="maximenuck item154 level2 " style="z-index : 11988;"><a class="maximenuck " href="/quienes-somos/visita-virtual"><span class="titreck">Visita virtual</span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item943 level2 " style="z-index : 11987;"><a class="maximenuck " href="/quienes-somos/honris-causa"><span class="titreck">Honoris Causa</span></a><li class="maximenuck item188 level2 " style="z-index : 11986;"><span class="separator "><span class="titreck">Profesores</span></span>
		</li>
<li class="maximenuck item155 level2 " style="z-index : 11985;"><a class="maximenuck " href="/quienes-somos/de-merito"><span class="titreck">De mérito</span></a>
		</li>
<li class="maximenuck item189 level2 " style="z-index : 11984;"><a class="maximenuck " href="/quienes-somos/consultantes"><span class="titreck">Consultantes</span></a>
		</li>
<li class="maximenuck item192 level2 " style="z-index : 11983;"><a class="maximenuck " href="/quienes-somos/fundadores"><span class="titreck">Fundadores</span></a>
		</li>
<li class="maximenuck item191 level2 " style="z-index : 11982;"><span class="separator "><span class="titreck">Trabajadores</span></span>
		</li>
<li class="maximenuck item158 last level2 " style="z-index : 11981;"><a class="maximenuck " href="/pregrado/claustro"><span class="titreck">Claustro</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item103 parent level1 " style="z-index : 11980;"><span class="separator "><span class="titreck">Estudios<span class="descck">Pregrado y Posgrado</span></span></span>
	<div class="floatck" style="width:540px;"><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first "  style="width:180px;float:left;">
	<ul class="maximenuck2"><li class="maximenuck item106 first level2 " style="z-index : 11979;"><a class="maximenuck " href="/pregrado/bienvenida"><span class="titreck">Pregrado</span></a><li class="maximenuck item119 parent level2 " style="z-index : 11978;"><span class="separator "><span class="titreck">Facultades<span class="descck">Enlace a cada facultad</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item149 first level3 " style="z-index : 11977;"><a class="maximenuck " href="/arquitectura"><span class="titreck">Arquitectura</span></a>
		</li>
<li class="maximenuck item148 level3 " style="z-index : 11976;"><a class="maximenuck " href="/civil"><span class="titreck">Ing. Civil</span></a>
		</li>
<li class="maximenuck item151 level3 " style="z-index : 11975;"><a class="maximenuck " href="/electrica"><span class="titreck">Ing. Eléctrica</span></a>
		</li>
<li class="maximenuck item146 level3 " style="z-index : 11974;"><a class="maximenuck " href="/industrial"><span class="titreck">Ing. Industrial</span></a>
		</li>
<li class="maximenuck item120 level3 " style="z-index : 11973;"><a class="maximenuck " href="/informatica"><span class="titreck">Ing. Informática</span></a>
		</li>
<li class="maximenuck item147 level3 " style="z-index : 11972;"><a class="maximenuck " href="/quimica"><span class="titreck">Ing. Química</span></a>
		</li>
<li class="maximenuck item150 level3 " style="z-index : 11971;"><a class="maximenuck " href="/mecanica"><span class="titreck">Ing. Mecánica</span></a>
		</li>
<li class="maximenuck item975 level3 " style="z-index : 11970;"><a class="maximenuck " href="/automatica-y-biomedica"><span class="titreck">Ing. Automática y Biomédica</span></a>
		</li>
<li class="maximenuck item979 last level3 " style="z-index : 11969;"><a class="maximenuck " href="/telecomunicaciones-y-electronica"><span class="titreck">Ing. en Telecomunicaciones y Electrónica</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item881 level2 " style="z-index : 11968;"><a class="maximenuck " href="/pregrado/visitarnos"><span class="titreck">Desea visitarnos<span class="descck">Visitas dirigidas</span></span></a>
		</li>
<li class="maximenuck item671 level2 " style="z-index : 11967;"><a class="maximenuck " href="/pregrado/admision-para-estudiantes-residentes-en-cuba"><span class="titreck">Nacionales<span class="descck">Admisión Cubanos</span></span></a>
		</li>
<li class="maximenuck item109 parent level2 " style="z-index : 11966;"><span class="separator "><span class="titreck">Extranjeros<span class="descck">Estudiar en la Cujae</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item672 first level3 " style="z-index : 11965;"><a class="maximenuck " href="/pregrado/admision-para-estudiantes-extranjeros"><span class="titreck">Admisión<span class="descck">Extranjeros</span></span></a>
		</li>
<li class="maximenuck item905 level3 " style="z-index : 11964;"><a class="maximenuck " href="/pregrado/cursos-preparatorios-de-nivelacion-e-idioma-espanol"><span class="titreck">Cursos de Nivelación<span class="descck">Preparatorios</span></span></a>
		</li>
<li class="maximenuck item906 level3 " style="z-index : 11963;"><a class="maximenuck " href="/pregrado/carreras-completas-autofinanciadas"><span class="titreck">Carreras completas<span class="descck">Autofinanciadas</span></span></a>
		</li>
<li class="maximenuck item907 level3 " style="z-index : 11962;"><a class="maximenuck " href="/pregrado/estudios-parciales"><span class="titreck">Estudios Parciales</span></a>
		</li>
<li class="maximenuck item908 level3 " style="z-index : 11961;"><a class="maximenuck " href="/pregrado/cursos-y-pasantias-internacionales"><span class="titreck">Cursos y Pasantías<span class="descck">Internacionales</span></span></a>
		</li>
<li class="maximenuck item909 last level3 " style="z-index : 11960;"><a class="maximenuck " href="/pregrado/cursos-de-idioma-espanol-para-extranjeros"><span class="titreck">Cursos de Idioma Español</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item157 level2 " style="z-index : 11959;"><a class="maximenuck " href="/pregrado/teleformacion"><span class="titreck">Teleformación</span></a>
		</li>
<li class="maximenuck item159 level2 " style="z-index : 11958;"><a class="maximenuck " href="/pregrado/universalizacion-de-la-ensenanza"><span class="titreck">Universalización<span class="descck">de la Enseñanza</span></span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item107 level2 " style="z-index : 11957;"><a class="maximenuck " href="/posgrado"><span class="titreck">Posgrado</span></a><li class="maximenuck item514 level2 " style="z-index : 11956;"><a class="maximenuck " href="/"><span class="titreck">Admisión</span></a>
		</li>
<li class="maximenuck item110 level2 " style="z-index : 11955;"><a class="maximenuck " href="/cursos"><span class="titreck">Cursos</span></a>
		</li>
<li class="maximenuck item160 level2 " style="z-index : 11954;"><a class="maximenuck " href="/diplomados"><span class="titreck">Diplomados</span></a>
		</li>
<li class="maximenuck item161 level2 " style="z-index : 11953;"><a class="maximenuck " href="/maestrias"><span class="titreck">Maestrías</span></a>
		</li>
<li class="maximenuck item162 level2 " style="z-index : 11952;"><a class="maximenuck " href="/doctorados"><span class="titreck">Doctorados</span></a>
		</li>
<li class="maximenuck item440 level2 " style="z-index : 11951;"><a class="maximenuck " href="/especialidades"><span class="titreck">Especialidades</span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item624 level2 " style="z-index : 11950;"><span class="separator "><span class="titreck">Otros cursos</span></span><li class="maximenuck item938 last level2 " style="z-index : 11949;"><a class="maximenuck " href="/cursos-de-invierno"><span class="titreck">Cursos de Invierno<span class="descck">programas y más...</span></span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item115 parent level1 " style="z-index : 11948;"><span class="separator "><span class="titreck">Investigaciones<span class="descck">Centros, Publicaciones...</span></span></span>
	<div class="floatck" style="width:360px;"><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first "  style="width:180px;float:left;">
	<ul class="maximenuck2"><li class="maximenuck item163 first level2 " style="z-index : 11947;"><a class="maximenuck " href="/investigaciones/informacion-general"><span class="titreck">Información general</span></a><li class="maximenuck item193 level2 " style="z-index : 11946;"><a class="maximenuck " href="/investigaciones/centros-de-investigacion"><span class="titreck">Centros<span class="descck">de investigación</span></span></a>
		</li>
<li class="maximenuck item347 level2 " style="z-index : 11945;"><a class="maximenuck " href="/investigaciones/publicaciones"><span class="titreck">Publicaciones</span></a>
		</li>
<li class="maximenuck item920 level2 " style="z-index : 11944;"><a class="maximenuck " href="http://redma.cujae.edu.cu"><span class="titreck">REDMA<span class="descck">Portal del Medioambiente</span></span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item194 level2 " style="z-index : 11943;"><a class="maximenuck " href="/investigaciones/proyectos-de-impacto"><span class="titreck">Proyectos de impacto</span></a><li class="maximenuck item956 parent level2 " style="z-index : 11942;"><span class="separator "><span class="titreck">Proyectos de Investigaciones</span></span>
	<div class="floatck" style="width:360px;"><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first "  style="width:180px;float:left;">
	<ul class="maximenuck2"><li class="maximenuck item957 first level3 " style="z-index : 11941;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/tic"><span class="titreck">TIC</span></a><li class="maximenuck item961 level3 " style="z-index : 11940;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/vivienda"><span class="titreck">Vivienda</span></a>
		</li>
<li class="maximenuck item962 level3 " style="z-index : 11939;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/defensa"><span class="titreck">Defensa</span></a>
		</li>
<li class="maximenuck item963 level3 " style="z-index : 11938;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/energia"><span class="titreck">Energía</span></a>
		</li>
<li class="maximenuck item964 level3 " style="z-index : 11937;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/nanotecnologia"><span class="titreck">Nanotecnología</span></a>
		</li>
<li class="maximenuck item965 level3 " style="z-index : 11936;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/medio-ambiente"><span class="titreck">Medio Ambiente</span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item966 level3 " style="z-index : 11935;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/ciencias-de-la-vida"><span class="titreck">Ciencias de la Vida</span></a><li class="maximenuck item967 level3 " style="z-index : 11934;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/educacion"><span class="titreck">Educación</span></a>
		</li>
<li class="maximenuck item968 level3 " style="z-index : 11933;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/alimentos"><span class="titreck">Alimentos</span></a>
		</li>
<li class="maximenuck item969 level3 " style="z-index : 11932;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/industria"><span class="titreck">Industria</span></a>
		</li>
<li class="maximenuck item970 level3 " style="z-index : 11931;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/gestion"><span class="titreck">Gestión</span></a>
		</li>
<li class="maximenuck item971 last level3 " style="z-index : 11930;"><a class="maximenuck " href="/investigaciones/proyectos-de-investigaciones/agua"><span class="titreck">Agua</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item195 level2 " style="z-index : 11929;"><a class="maximenuck " href="/investigaciones/premios-a-investigacion"><span class="titreck">Premios a investigación</span></a>
		</li>
<li class="maximenuck item346 level2 " style="z-index : 11928;"><a class="maximenuck " href="/investigaciones/servicios-cientifico-tecnicos"><span class="titreck">Servicios<span class="descck">Científico-Técnicos</span></span></a>
		</li>
<li class="maximenuck item348 level2 " style="z-index : 11927;"><a class="maximenuck " href="/investigaciones/anir"><span class="titreck">ANIR</span></a>
		</li>
<li class="maximenuck item349 last level2 " style="z-index : 11926;"><a class="maximenuck " href="/investigaciones/forum-de-ciencia-y-tecnica"><span class="titreck">Forum<span class="descck">Ciencia y Técnica</span></span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item504 parent level1 " style="z-index : 11925;"><span class="separator "><span class="titreck">Eventos<span class="descck">Convención...</span></span></span>
	<div class="floatck" style="width:544px;"><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first "  style="width:180px;float:left;">
	<ul class="maximenuck2"><li class="maximenuck item480 first level2 " style="z-index : 11924;"><a class="maximenuck " href="http://ccia.cujae.edu.cu" target="_blank" ><span class="titreck">Convención Científica<span class="descck">de Ingeniería y Arquitectura. CCIA</span></span></a></ul><div class="clr"></div></div><div class="maximenuck2"  style="width:180px;float:left;"><ul class="maximenuck2"><li class="maximenuck item619 level2 " style="z-index : 11923;"><span class="separator "><span class="titreck">Eventos<span class="descck">Pasados</span></span></span><li class="maximenuck item513 level2 " style="z-index : 11922;"><a class="maximenuck " href="http://macdes.cujae.edu.cu" target="_blank" ><span class="titreck">MACDES<span class="descck">Medio Ambiente... Sustentable...</span></span></a>
		</li>
</ul><div class="clr"></div></div><div class="maximenuck2"  style="width:184px;float:left;"><ul class="maximenuck2"><li class="maximenuck item936 level2 " style="z-index : 11921;"><span class="separator "><span class="titreck">Eventos<span class="descck">Externos</span></span></span><li class="maximenuck item947 last level2 " style="z-index : 11920;"><noindex><a class="maximenuck " href="http://www.congresoextension.mes.gob.cu/" rel="nofollow" target="_blank" ><span class="titreck">XIII Congreso Latinoamericano<span class="descck"> de Extensión Universitaria</span></span></a></noindex>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item481 parent level1 " style="z-index : 11919;"><span class="separator "><span class="titreck">Internacional<span class="descck">Relaciones y más</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item768 first level2 " style="z-index : 11918;"><a class="maximenuck " href="/internacionales/convenios"><span class="titreck">Convenios<span class="descck">y Cartas de intención</span></span></a>
		</li>
<li class="maximenuck item910 parent level2 " style="z-index : 11917;"><span class="separator "><span class="titreck">Extranjeros<span class="descck">Estudiar en la Cujae</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item911 first level3 " style="z-index : 11916;"><a class="maximenuck " href="/pregrado/admision-para-estudiantes-extranjeros"><span class="titreck">Admisión<span class="descck">Extranjeros</span></span></a>
		</li>
<li class="maximenuck item912 level3 " style="z-index : 11915;"><a class="maximenuck " href="/pregrado/cursos-preparatorios-de-nivelacion-e-idioma-espanol"><span class="titreck">Cursos de Nivelación<span class="descck">Preparatorios</span></span></a>
		</li>
<li class="maximenuck item913 level3 " style="z-index : 11914;"><a class="maximenuck " href="/pregrado/carreras-completas-autofinanciadas"><span class="titreck">Carreras completas<span class="descck">Autofinanciadas</span></span></a>
		</li>
<li class="maximenuck item914 level3 " style="z-index : 11913;"><a class="maximenuck " href="/pregrado/estudios-parciales"><span class="titreck">Estudios Parciales</span></a>
		</li>
<li class="maximenuck item915 level3 " style="z-index : 11912;"><a class="maximenuck " href="/pregrado/cursos-y-pasantias-internacionales"><span class="titreck">Cursos y Pasantías<span class="descck">Internacionales</span></span></a>
		</li>
<li class="maximenuck item916 last level3 " style="z-index : 11911;"><a class="maximenuck " href="/pregrado/cursos-de-idioma-espanol-para-extranjeros"><span class="titreck">Cursos de Idioma Español</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item939 level2 " style="z-index : 11910;"><a class="maximenuck " href="/cursos-de-invierno"><span class="titreck">Cursos de Invierno<span class="descck">programas y más...</span></span></a>
		</li>
<li class="maximenuck item935 last level2 " style="z-index : 11909;"><a class="maximenuck " href="/internacionales/asociaciones"><span class="titreck">Asociaciones<span class="descck">Internacionales</span></span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item117 parent level1 " style="z-index : 11908;"><span class="separator "><span class="titreck">Vida universitaria<span class="descck">Extensión, Deportes...</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item797 first level2 " style="z-index : 11907;"><a class="maximenuck " href="/noticias"><span class="titreck">Noticias<span class="descck">en la Cujae</span></span></a>
		</li>
<li class="maximenuck item882 level2 " style="z-index : 11906;"><a class="maximenuck " href="/boletin"><span class="titreck">Boletín Cujae<span class="descck">nuestro acontecer</span></span></a>
		</li>
<li class="maximenuck item350 level2 " style="z-index : 11905;"><a class="maximenuck " href="/investigaciones/arte-y-comunicacion"><span class="titreck">Arte y Comunicación</span></a>
		</li>
<li class="maximenuck item351 level2 " style="z-index : 11904;"><a class="maximenuck " href="/vida-universitaria/deportes"><span class="titreck">Deportes</span></a>
		</li>
<li class="maximenuck item352 level2 " style="z-index : 11903;"><a class="maximenuck " href="/vida-universitaria/residencia-estudiantil"><span class="titreck">Residencia Estudiantil</span></a>
		</li>
<li class="maximenuck item353 level2 " style="z-index : 11902;"><a class="maximenuck " href="/vida-universitaria/organizaciones"><span class="titreck">Organizaciones</span></a>
		</li>
<li class="maximenuck item404 last level2 " style="z-index : 11901;"><a class="maximenuck " href="/vida-universitaria/galeria-de-videos"><span class="titreck">Galería de Videos</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li><li class="maximenuck item165 parent level1 " style="z-index : 11900;"><span class="separator "><span class="titreck">Servicios<span class="descck">Comunidad, Correo...</span></span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item533 first level2 " style="z-index : 11899;"><a class="maximenuck " href="/comunidad"><span class="titreck">Comunidad<span class="descck">Encontrar una persona</span></span></a>
		</li>
<li class="maximenuck item358 level2 " style="z-index : 11898;"><a class="maximenuck " href="http://biblioteca.cujae.edu.cu"><span class="titreck">Biblioteca<span class="descck">Medios digitales</span></span></a>
		</li>
<li class="maximenuck item1001 level2 " style="z-index : 11897;"><a class="maximenuck " href="http://tesis.cujae.edu.cu" target="_blank"  title="TESIS-CUJAE es el Repositorio Institucional de Tesis de la Universidad Tecnológica de la Habana José Antonio Echeverría"><span class="titreck">Tesis Cujae <span class="descck"> Tesis de Pregrado y Posgrado</span></span></a>
		</li>
<li class="maximenuck item357 level2 " style="z-index : 11896;"><a class="maximenuck " href="https://correo.cujae.edu.cu"><span class="titreck">Correo<span class="descck">Servidores Cujae</span></span></a>
		</li>
<li class="maximenuck item535 level2 " style="z-index : 11895;"><a class="maximenuck " href="/contactar"><span class="titreck">Contactar<span class="descck">Por info. o ayuda</span></span></a>
		</li>
<li class="maximenuck item940 parent level2 " style="z-index : 11894;"><span class="separator "><span class="titreck">Gestión de Software</span></span>
	<div class="floatck" ><div class="maxidrop-top"><div class="maxidrop-top2"></div></div><div class="maxidrop-main"><div class="maxidrop-main2"><div class="maximenuck2 first " >
	<ul class="maximenuck2"><li class="maximenuck item941 first level3 " style="z-index : 11893;"><a class="maximenuck " href="/servicios/soluciones-informaticas"><span class="titreck">Casa de Software <span class="descck"> Soluciones Informáticas</span></span></a>
		</li>
<li class="maximenuck item942 last level3 " style="z-index : 11892;"><a class="maximenuck " href="/servicios/disertic/direccion-de-servicios-de-las-tecnologias-para-la-informatizacion-y-las-comunicaciones"><span class="titreck">DISERTIC</span></a>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li>
	</ul>
	<div class="clr"></div></div><div class="clr"></div></div></div><div class="maxidrop-bottom"><div class="maxidrop-bottom2"></div></div></div>
	</li>            </ul>
        </div>
        <div class="maxiroundedright"></div>
        <div style="clear:both;"></div>
    </div>
    <div style="clear:both;"></div>
    <!-- fin maximenuCK -->

              </div>
      <div class="over_table">
        <div class="cont_file">
          <div class="cont_cell cell_pri">
            <div id="breadcrumbs">
              
<div class="breadcrumbs">
<span class="showHere">Usted está aquí: </span><span>Inicio</span></div>

            </div>
          </div>
          <div id="busca" class="cont_cell cell_der">
            <div class="buscar">
              
<div id="k2ModuleBox127" class="k2SearchBlock k2LiveSearchBlock">
	<form action="/search" method="get" autocomplete="off" class="k2SearchBlockForm">

		<input type="text" value="Buscar" name="searchword" maxlength="30" size="30" alt="Buscar..." class="inputbox" onblur="if(this.value=='') this.value='Buscar';" onfocus="if(this.value=='Buscar') this.value='';" />

						<input type="image" value="Buscar..." class="button_img" onclick="this.form.searchword.focus();" src="/templates/lecujae/images/buscar.png" />
				
		<input type="hidden" name="categories" value="7,8,11,13,14,15,16,17,18,21,33,34,37,77,86,91,101,104,105,106,107,108,109,110,111,112,119,124,130,140,144,22,36,40,45,50,55,60,65,70,76,92,146,96,103,114,120,122,125,131,136,147,148,149,150,151,152,153,154,157,158,159,44,41,42,43,46,47,48,49,51,52,53,54,56,57,58,59,61,62,63,64,66,67,68,69,71,72,73,74,78,79,80,81,93,113,116,117,118,126,127,128,129,134,135" />
						<input type="hidden" name="format" value="html" />
		<input type="hidden" name="t" value="" />
		<input type="hidden" name="tpl" value="search" />
			</form>

		<div class="k2LiveSearchResults"></div>
	</div>

            </div>
          </div>
        </div>
              </div>
      <div class="cont_table">
        <div class="cont_file">
                    <div id="concolumna" class="cont_cell cell_pri">
            
<div id="system-message-container">
</div>
            
	<div id="djslider-loader35" class="djslider-loader">
    <div id="djslider35" class="djslider">
        <div id="slider-container35" class="slider-container">
        	<ul id="slider35">
          		          			<li>
            										<a  href="/civil/carrera/ingenieria-hidraulica" target="_self">
													<img src="/images/cujae/carreras/hidraulica.jpg" alt="Ingeniería Hidraulica" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Hidraulica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<img src="/images/cujae/carreras/electrica.jpg" alt="Ingeniería Eléctrica" />
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Eléctrica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/mecanica/carrera/ingenieria-mecanica" target="_self">
													<img src="/images/cujae/carreras/mecanica.jpg" alt="Ingeniería Mecánica" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Mecánica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/informatica/carrera/ingenieria-informatica" target="_self">
													<img src="/images/cujae/carreras/informatica.jpg" alt="Ingeniería Informática" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Informática																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<img src="/images/cujae/carreras/biomedica.jpg" alt="Ingeniería Biomédica" />
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Biomédica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/industrial/carrera/ingenieria-industrial" target="_self">
													<img src="/images/cujae/carreras/industrial.jpg" alt="Ingeniería Industrial" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Industrial																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/mecanica/carrera/ingenieria-en-metalurgia-y-materiales" target="_self">
													<img src="/images/cujae/carreras/mecanica.jpg" alt="Ingeniería Metalúrgica" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Metalúrgica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/quimica/carrera/ingenieria-quimica" target="_self">
													<img src="/images/cujae/carreras/quimica.jpg" alt="Ingeniería Química" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Química																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/civil/carrera/ingenieria-civil" target="_self">
													<img src="/images/cujae/carreras/civil.jpg" alt="Ingeniería Civil" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Civil																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<img src="/images/cujae/carreras/automatica.jpg" alt="Ingeniería Automática" />
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Automática																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/civil/carrera/ingenieria-geofisica" target="_self">
													<img src="/images/cujae/carreras/geofisica.jpg" alt="Ingeniería Geofísica" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ingeniería Geofísica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<img src="/images/cujae/carreras/teleco.jpg" alt="Ing. en Tele. y Electrónica" />
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Ing. en Tele. y Electrónica																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                          			<li>
            										<a  href="/arquitectura/carrera/arquitectura" target="_self">
													<img src="/images/cujae/carreras/arquitectura.jpg" alt="Arquitectura" />
													</a>
												
												<!-- Slide description area: START -->
						<div class="slide-desc">
						  <div class="slide-desc-in">	
							<div class="slide-desc-bg"></div>
							<div class="slide-desc-text">
															<div class="slide-title">
																			Arquitectura																	</div>
														
														
														<div style="clear: both"></div>
							</div>
						  </div>
						</div>
						<!-- Slide description area: END -->
												
						
					</li>
                        	</ul>
        </div>
        <div id="navigation35" class="navigation-container">
        	<img id="prev35" class="prev-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/prev.png" alt="Previous" />
			<img id="next35" class="next-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/next.png" alt="Next" />
			<img id="play35" class="play-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/play.png" alt="Play" />
			<img id="pause35" class="pause-button" src="http://cujae.edu.cu/modules/mod_djimageslider/assets/pause.png" alt="Pause" />
        </div>
		<div id="cust-navigation35" class="navigation-container-custom">
							<span class="load-button load-button-active"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
							<span class="load-button"></span>
			        </div>
    </div>
	</div>
	
	<div style="clear: both"></div>
<div id="noticiasp" class="k2ItemsBlock">
      <ul>
        <li class=""> 
      
      <!-- Plugins: BeforeDisplay --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplay --> 
                        <h2>
      <a class="moduleItemTitle" href="/noticias/una-visita-que-prestigia-nuestra-universidad">Una visita que prestigia nuestra Universidad</a>
      </h2>
                  <div class="moduleItemAuthor" style="font-size:12px; padding: 4px 0 6px 0; color:#333;"> Escrito por                <a rel="author" title="Hilda de la Caridad Rebollar Padron" href="/comunidad/hildita">Hilda de la Caridad Rebollar Padron</a>
              <span class="moduleItemDateCreated">&nbsp;|&nbsp;Martes, 20 Marzo 2018 08:03</span>
                            </div>
            
      <!-- Plugins: AfterDisplayTitle --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayTitle --> 
       
      
      <!-- Plugins: BeforeDisplayContent --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplayContent --> 
                  
      <div class="noticiaimg">
	            	<a class="moduleItemImage" href="/noticias/una-visita-que-prestigia-nuestra-universidad" title="Leer más &quot;Una visita que prestigia nuestra Universidad&quot;"> <img src="/media/k2/items/cache/b101fa9bdd0c35b4319fa7cdeb1495ae_M.jpg" alt="Una visita que prestigia nuestra Universidad"/></a>
                        El 19 de marzo de 2018, la Universidad Tecnológica de La Habana, “José Antonio Echeverría”, Cujae, recibió la visita del eminente doctor Hiroshi Ishiguro, destacado profesor de la Universidad de Osaka. El profesor, quien se encuentra de visita en nuestro país desde el 18 del presente mes, decidió realizar un ameno intercambio con profesores, trabajadores y estudiantes del Centro, sobre sus investigaciones en el campo de la robótica y los&#8230;                      		<a class="moduleItemReadMore" href="/noticias/una-visita-que-prestigia-nuestra-universidad"> Leer más... </a>
     	      </div>
                  <div class="clr"></div>
            <div class="clr"></div>
      
      <!-- Plugins: AfterDisplayContent --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayContent --> 
                                                <a class="moduleItemComments" href="/noticias/una-visita-que-prestigia-nuestra-universidad#itemCommentsAnchor"> ¡Deja el primer comentario! </a>
                              <span class="moduleItemHits"> Visto 94 veces </span>
           
      
      <!-- Plugins: AfterDisplay --> 
       
      
      <!-- K2 Plugins: K2AfterDisplay --> 
            <div class="clr"></div>
    </li>
        <li class=""> 
      
      <!-- Plugins: BeforeDisplay --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplay --> 
                        <h2>
      <a class="moduleItemTitle" href="/noticias/entregados-diplomas-acreditativos-de-doctor-honoris-causa-a-dos-eminentes-profesoras-de-la-cujae">Entregados diplomas acreditativos de Doctor Honoris Causa a dos eminentes profesoras de la Cujae</a>
      </h2>
                  <div class="moduleItemAuthor" style="font-size:12px; padding: 4px 0 6px 0; color:#333;"> Escrito por                <a rel="author" title="Hilda de la Caridad Rebollar Padron" href="/comunidad/hildita">Hilda de la Caridad Rebollar Padron</a>
              <span class="moduleItemDateCreated">&nbsp;|&nbsp;Martes, 06 Marzo 2018 09:31</span>
                            </div>
            
      <!-- Plugins: AfterDisplayTitle --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayTitle --> 
       
      
      <!-- Plugins: BeforeDisplayContent --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplayContent --> 
                  
      <div class="noticiaimg">
	            	<a class="moduleItemImage" href="/noticias/entregados-diplomas-acreditativos-de-doctor-honoris-causa-a-dos-eminentes-profesoras-de-la-cujae" title="Leer más &quot;Entregados diplomas acreditativos de Doctor Honoris Causa a dos eminentes profesoras de la Cujae&quot;"> <img src="/media/k2/items/cache/8938fc981672a91f27b49fef3a9aa529_M.jpg" alt="Entregados diplomas acreditativos de Doctor Honoris Causa a dos eminentes profesoras de la Cujae"/></a>
                        La Universidad Tecnológica de La Habana “José Antonio Echeverría”, Cujae en acto solemne efectuado el 2 de marzo de 2018 a las 7 p. m., en el Hotel Parque Central de esta capital, reconoció con la condición de Doctor Honoris Causa a dos prestigiosas profesoras del Centro: las doctoras Gilda M. Vega Cruz y Lourdes M. Zumalacárregui de Cárdenas. La doctora Alicia Alonso Becerra, rectora de la Cujae, hizo entrega&#8230;                      		<a class="moduleItemReadMore" href="/noticias/entregados-diplomas-acreditativos-de-doctor-honoris-causa-a-dos-eminentes-profesoras-de-la-cujae"> Leer más... </a>
     	      </div>
                  <div class="clr"></div>
            <div class="clr"></div>
      
      <!-- Plugins: AfterDisplayContent --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayContent --> 
                                                <a class="moduleItemComments" href="/noticias/entregados-diplomas-acreditativos-de-doctor-honoris-causa-a-dos-eminentes-profesoras-de-la-cujae#itemCommentsAnchor"> ¡Deja el primer comentario! </a>
                              <span class="moduleItemHits"> Visto 298 veces </span>
           
      
      <!-- Plugins: AfterDisplay --> 
       
      
      <!-- K2 Plugins: K2AfterDisplay --> 
            <div class="clr"></div>
    </li>
        <li class=" lastItem"> 
      
      <!-- Plugins: BeforeDisplay --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplay --> 
                        <h2>
      <a class="moduleItemTitle" href="/noticias/la-cujae-universidad-mas-premiada-en-la-iv-olimpiada-de-matematica">La Cujae, Universidad más premiada en la IV Olimpiada de Matemática</a>
      </h2>
                  <div class="moduleItemAuthor" style="font-size:12px; padding: 4px 0 6px 0; color:#333;"> Escrito por                <a rel="author" title="Hilda de la Caridad Rebollar Padron" href="/comunidad/hildita">Hilda de la Caridad Rebollar Padron</a>
              <span class="moduleItemDateCreated">&nbsp;|&nbsp;Jueves, 01 Marzo 2018 10:13</span>
                            </div>
            
      <!-- Plugins: AfterDisplayTitle --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayTitle --> 
       
      
      <!-- Plugins: BeforeDisplayContent --> 
       
      
      <!-- K2 Plugins: K2BeforeDisplayContent --> 
                  
      <div class="noticiaimg">
	            	<a class="moduleItemImage" href="/noticias/la-cujae-universidad-mas-premiada-en-la-iv-olimpiada-de-matematica" title="Leer más &quot;La Cujae, Universidad m&aacute;s premiada en la IV Olimpiada de Matem&aacute;tica&quot;"> <img src="/media/k2/items/cache/633ad04911b9ff78ce6e391fd459409b_M.jpg" alt="La Cujae, Universidad m&aacute;s premiada en la IV Olimpiada de Matem&aacute;tica"/></a>
                        El 9 de febrero de 2018 se realizó la Cuarta Olimpiada de Matemática organizada por el Centro de Estudios de Matemática para las Ciencias Técnicas (CEMAT) de la Universidad Tecnológica de la Habana ¨José Antonio Echavarría¨ y la Sociedad Cubana de Matemática y Computación (SCMC). La sede oficial para la provincia de La Habana fue la Cujae. Participaron los estudiantes de pregrado del curso regular diurno de las carreras de&#8230;                      		<a class="moduleItemReadMore" href="/noticias/la-cujae-universidad-mas-premiada-en-la-iv-olimpiada-de-matematica"> Leer más... </a>
     	      </div>
                  <div class="clr"></div>
            <div class="clr"></div>
      
      <!-- Plugins: AfterDisplayContent --> 
       
      
      <!-- K2 Plugins: K2AfterDisplayContent --> 
                                                <a class="moduleItemComments" href="/noticias/la-cujae-universidad-mas-premiada-en-la-iv-olimpiada-de-matematica#itemCommentsAnchor"> ¡Deja el primer comentario! </a>
                              <span class="moduleItemHits"> Visto 305 veces </span>
           
      
      <!-- Plugins: AfterDisplay --> 
       
      
      <!-- K2 Plugins: K2AfterDisplay --> 
            <div class="clr"></div>
    </li>
        <li class="clearList"></li>
  </ul>
        <div class="k2FeedIcon"> <a href="/rss/noticias-principales/feed" title="Suscribirse a este canal RSS"> <span>Suscribirse a este canal RSS</span> </a>
    <div class="clr"></div>
  </div>
  </div>

            
<!-- Start K2 Category Layout -->
<div id="k2Container" class="itemListView">
	
	
		<!-- Blocks for current category and subcategories -->
	<div class="itemListCategoriesBlock">

		
		
	</div>
	


	</div>
<!-- End K2 Category Layout -->

<!-- JoomlaWorks "K2" (v2.6.5) | Learn more about K2 at http://getk2.org -->


            
          </div>
                    <div id="derecha" class="cont_cell cell_der">
            <div class="moduletable">
 		<div class="module">
			<div>
				<div>
					<div>
											

<div class="custom"  >
	<h3>Video Promocional</h3>
<p>
<!-- JPlayer Plugin (start) -->
<div class="jplayer">
<div class="jplayer-box">
<object type="application/x-shockwave-flash" width="255" height="209" data="http://cujae.edu.cu/plugins/content/jplayer/mediaplayer/player-4-3-132.swf">
	<param name="movie" value="http://cujae.edu.cu/plugins/content/jplayer/mediaplayer/player-4-3-132.swf" />
	<param name="quality" value="high" />
	<param name="wmode" value="transparent" />
	<param name="bgcolor" value="#FFFFFF" />
	<param name="autoplay" value="false" />
	<param name="allowfullscreen" value="true" />
	<param name="allowscriptaccess" value="always" />
	<param name="flashvars" value="file=http://10.8.1.67:8081/AudioVisualesGabinete/Videos_Sitio_Audiovisuales/Promocional Cujae 2012.flv&amp;image=http://10.8.1.67:8081/AudioVisualesGabinete/Videos_Sitio_Audiovisuales/Promocional Cujae 2012.flv.png&amp;autostart=false&amp;fullscreen=true" />
</object>
</div>
<div class="jplayer-text">
</div>
</div>
<!-- JPlayer Plugin (end) -->
</p></div>
					</div>
				</div>
			</div>
		</div>
	</div>
<div class="moduletable">
 <div class="bannergroup">


</div>
</div>
<div class="moduletable">
 <div class="bannergroup">


</div>
</div>
<div class="moduletable">
 <div class="bannergroup">


</div>
</div>

            <div class="moduletable">
 <div id="holderTC1"><div id="innerTC1"><a style="font-weight:400;font-size: 30px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Ingeniería&ordering=&searchphrase=all&Itemid=446&option=com_search" >Ingeniería</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Procesos&ordering=&searchphrase=all&Itemid=446&option=com_search" >Procesos</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Biotecnológicos&ordering=&searchphrase=all&Itemid=446&option=com_search" >Biotecnológicos</a>
<a style="font-weight:400;font-size: 16px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Facultad&ordering=&searchphrase=all&Itemid=446&option=com_search" >Facultad</a>
<a style="font-weight:400;font-size: 15px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Industrial&ordering=&searchphrase=all&Itemid=446&option=com_search" >Industrial</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Información&ordering=&searchphrase=all&Itemid=446&option=com_search" >Información</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=General&ordering=&searchphrase=all&Itemid=446&option=com_search" >General</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Presentación&ordering=&searchphrase=all&Itemid=446&option=com_search" >Presentación</a>
<a style="font-weight:400;font-size: 20px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Misión&ordering=&searchphrase=all&Itemid=446&option=com_search" >Misión</a>
<a style="font-weight:400;font-size: 18px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Visión&ordering=&searchphrase=all&Itemid=446&option=com_search" >Visión</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Claustro&ordering=&searchphrase=all&Itemid=446&option=com_search" >Claustro</a>
<a style="font-weight:400;font-size: 16px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Consejo&ordering=&searchphrase=all&Itemid=446&option=com_search" >Consejo</a>
<a style="font-weight:400;font-size: 16px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Dirección&ordering=&searchphrase=all&Itemid=446&option=com_search" >Dirección</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Directorio&ordering=&searchphrase=all&Itemid=446&option=com_search" >Directorio</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Telefónico&ordering=&searchphrase=all&Itemid=446&option=com_search" >Telefónico</a>
<a style="font-weight:400;font-size: 16px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Investigaciones&ordering=&searchphrase=all&Itemid=446&option=com_search" >Investigaciones</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Automática&ordering=&searchphrase=all&Itemid=446&option=com_search" >Automática</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Eléctrica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Eléctrica</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Civil&ordering=&searchphrase=all&Itemid=446&option=com_search" >Civil</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Hidraulica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Hidraulica</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Geofísica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Geofísica</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Metalúrgica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Metalúrgica</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Mecánica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Mecánica</a>
<a style="font-weight:400;font-size: 16px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Informática&ordering=&searchphrase=all&Itemid=446&option=com_search" >Informática</a>
<a style="font-weight:400;font-size: 15px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Arquitectura&ordering=&searchphrase=all&Itemid=446&option=com_search" >Arquitectura</a>
<a style="font-weight:400;font-size: 14px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Química&ordering=&searchphrase=all&Itemid=446&option=com_search" >Química</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Biomédica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Biomédica</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Telecomunicaciones&ordering=&searchphrase=all&Itemid=446&option=com_search" >Telecomunicaciones</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Electrónica&ordering=&searchphrase=all&Itemid=446&option=com_search" >Electrónica</a>
<a style="font-weight:400;font-size: 12px" class="cloudTC1" href="http://cujae.edu.cu/index.php?searchword=Plan&ordering=&searchphrase=all&Itemid=446&option=com_search" >Plan</a>
</div></div>

</div>
<div class="moduletable">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Usuarios conectados</span></span></span></h3>
 
			<p>Tenemos 1734&#160;visitantes y ningun miembro en Línea</p>

</div>
<div class="moduletable">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Conectados Comunidad</span></span></span></h3>
 Ninguno</div>

            
          </div>
                  </div>
      </div>
    </div>
  </div>
</div>
<div class="back_foot">
  <div class="contenedor_foot"> 
    <!-- FOOTER -->
        <div id="footer-wrap">
      <div class="foot_enlaces">
        <div id="bottom">
          <div class="box1">
            <div class="moduletable_mfooter">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Facultades</span></span></span></h3>
 
<ul class="menu_mfooter">
<li class="item-368"><a href="/mecanica" >Ing. Mecánica</a></li><li class="item-112"><a href="/quimica" >Ing. Química</a></li><li class="item-111"><a href="/informatica" >Ing. Informática</a></li><li class="item-367"><a href="/industrial" >Ing. Industrial</a></li><li class="item-113"><a href="/electrica" >Ing. Eléctrica</a></li><li class="item-366"><a href="/civil" >Ing. Civil</a></li><li class="item-114"><a href="/arquitectura" >Arquitectura</a></li><li class="item-972"><a href="/automatica-y-biomedica" >Ing. Automática y Biomédica</a></li><li class="item-978"><a href="/telecomunicaciones-y-electronica" >Ing. Telecomunicaciones y Electrónica</a></li></ul>
</div>
<div class="moduletable_mfooter">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Enlaces de Interes</span></span></span></h3>
 

<div class="custom_mfooter"  >
	<ul>
<li><a href="/comunidad" title="Comunidad Cujae. Buscar persona">Comunidad</a></li>
<li><a href="http://correo.cujae.edu.cu" title="Servidores de Correo">Correo</a></li>
<li><a href="/noticias" title="Noticias Cujae">Noticias</a></li>
<li><a href="http://biblioteca.cujae.edu.cu" title="Biblioteca. Información dgítal">Biblioteca</a></li>
<li><a href="/vida-universitaria/galeria-de-videos" title="Galeria de Videos">Galeria de Videos</a></li>
</ul></div>
</div>

          </div>
          <div class="box2">
            <div class="moduletable_mfooter">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Carreras</span></span></span></h3>
 
<ul class="menu_mfooter">
<li class="item-182"><a href="/arquitectura/carrera/arquitectura" >Arquitectura</a></li><li class="item-177"><a href="/automatica-y-biomedica/carrera-6/ingenieria-automatica" >Ingeniería Automática</a></li><li class="item-181"><a href="/automatica-y-biomedica/carrera-6/ingenieria-biomedica" >Ingeniería Biomédica</a></li><li class="item-173"><a href="/civil/carrera/ingenieria-civil" >Ingeniería Civil</a></li><li class="item-176"><a href="/electrica/carrera-5-2/ingenieria-electrica" >Ingeniería Eléctrica</a></li><li class="item-174"><a href="/civil/carrera/ingenieria-geofisica" >Ingeniería Geofísica</a></li><li class="item-175"><a href="/civil/carrera/ingenieria-hidraulica" >Ingeniería Hidraulica</a></li><li class="item-183"><a href="/industrial/carrera/ingenieria-industrial" >Ingeniería Industrial</a></li><li class="item-180"><a href="/informatica/carrera/ingenieria-informatica" >Ingeniería Informática</a></li><li class="item-179"><a href="/quimica/carrera/ingenieria-quimica" >Ingeniería Química</a></li><li class="item-184"><a href="/mecanica/carrera/ingenieria-mecanica" >Ingeniería Mecánica</a></li><li class="item-185"><a href="/mecanica/carrera/ingenieria-en-metalurgia-y-materiales" >Ingeniería en Metalúrgía y Materiales</a></li><li class="item-178"><a href="/telecomunicaciones-y-electronica/carrera-6/ingenieria-en-telecomunicaciones-y-electronica" >Ingeniería en Telecomunicaciones y Electrónica</a></li></ul>
</div>

          </div>
          <div class="box3">
            <div class="moduletable_mfooter">
 <h3><span
	class="backh"><span class="backh2"><span class="backh3">Centros de Investigación</span></span></span></h3>
 
<ul class="menu_mfooter">
<li class="item-841"><a href="/crea" >CREA, Centro de Referencia para la Educación de Avanzada</a></li><li class="item-924"><a href="/cecat" >CECAT, Centro de Construcción y Arquitectura Tropical</a></li><li class="item-925"><a href="/ceim" >CEIM, Centro de Estudios de Innovación y Mantenimiento</a></li><li class="item-926"><a href="/cetdir" >CETDIR, Centro de Estudios de Técnicas de Dirección</a></li><li class="item-927"><a href="/ceter" >CETER, Centro de Estudios de Tecnologías Energéticas Renovables</a></li><li class="item-929"><a href="/cih" >CIH, Centro de Investigaciones Hidráulicas</a></li><li class="item-930"><a href="/cime" >CIME, Centro de Investigaciones de Microelectrónica</a></li><li class="item-931"><a href="/cipel" >CIPEL, Centro de Investigaciones y Pruebas Electro-Energéticas</a></li><li class="item-932"><a href="/cipro" >CIPRO, Centro de Estudios de Ingeniería de Procesos</a></li></ul>
</div>

          </div>
      <div class="box4">
            
          </div>
        </div>
      </div>
            <div class="clear"></div>
            <div class="clear"></div>
      
    </div>
        <div id="footer-sub">
      <div class="pie_cujae"> <noindex><a href="https://cmsdude.org" rel="nofollow" target="_blank" >&copy; 1998-2016 <strong>Universidad Tecnológica de La Habana José Antonio Echeverría</strong></a></noindex> </div>
      <div class="pie_menu">
        
        
<ul class="menu_pie_menu">
<li class="item-496 active"><a href="/" >Inicio</a></li><li class="item-497"><a href="/portal/creditos" >Créditos</a></li><li class="item-502"><a href="/contactar" >Contactar</a></li></ul>

      </div>
    </div>
  </div>
</div>
<div class="chat_comunidad">
  <script type=text/javascript>var jfb_theme=0;</script><script type=text/javascript>var slideroptions='4';</script><script type=text/javascript>var currentonlineid=new Array()</script><script type=text/javascript>var currentonlinenames=new Array()</script><script type=text/javascript>var currentonlineunames=new Array()</script>

<!-- Start Activities -->
<div id="jfb_actvty" class="jfb_actvty">
<div class="jfb_actop">
<div class="jfb_actitle">Actividades</div>
<div class="jfb_clact" onClick="handler('jfb_actvty',0);">X</div>
</div>
<div class="jfb_maintest">
<div class="jfb_mainact"> 
{loadposition jbolo_pos1} </div>
 </div>
</div>


<!--Start Chat Area-->
<div id="jfb_chatbx" class="jfb_chatbx">
<div class="jfb_chtop">
<div class="jfb_chtitle">Chat</div>
<div class="jfb_clchat" onClick="handler('jfb_chatbx',0);">X</div>
</div>

<div onclick=showchatdiv(); id="jfb_useroptions">
<span class="jfb_useroptions">
Tu Estado:&nbsp;
</span>
<div style='float:left;' id=inside-ch-box-tl>
Disponible</div>
<span style="float:right;"><img src="http://cujae.edu.cu/components/com_jbolo/img/module/arrow_16x16.png" alt="Click here to change your status" /> </span>
</div>
<div id='ch_box_status'>
	<a style="display:block;" onclick='chat_status(1);'>Disponible</a>
	<a style="display:block;" onclick='chat_status(2);'>Ausente</a>
	<a style="display:block;" onclick='chat_status(0);'>Invisible</a>
	<a style="display:block;" onclick='jfb_show_prompt();'>Ingresa un estado personalizado</a>
</div>	
	
<div class="jfb_mainchat">
Inicia sesión para poder chatear.<!-- LECARO -->
<p style="margin: 2px 5px; padding: 3px 0; font-size: 10px; line-height:12px; border-top:solid 1px #f2f2f2;">Con los usuarios que tiene <strong><a style="font-size:12px;" href="/index.php?option=com_comprofiler&amp;Itemid=527&amp;lang=es&amp;task=manageConnections">V&iacute;nculo</a></strong> en la Comunidad y est&eacute;n conectados podr&aacute; conversar.</p>
<!-- LECARO -->
</div>
</div>

<!--Start Bar-->
<div id="jfb_nav_menu_wrapper">
<div class="jfb_wrapper">
<div class="jfb_barleft"></div>
<div class="jfb_nav_menu">

<div class="jfb_normal" id="jfb_chatactive" onClick="handler('jfb_chatbx',1);">
<div class="jfb_chattext">
Chat  
(<span id="jfb_chatnums">0</span>)
</div>
</div>

<div  style="width:124px !important; float:right !important; display:block !important;">&nbsp;
</div>

<div style="height:20px; width:10px; float:left;"></div>

<div class="jfb_icons">
<span title="Comunidad Cujae">
<a href="/comunidad">
<img src="http://cujae.edu.cu/components/com_jbolo/img/module/gradientgrey/bar-icons/cb16x16.gif" /></a></span> 
</div>

<div class="jfb_icons">
<span title="Actualizar perfil">
<a href="/ver-detalles-usuario">
<img src="http://cujae.edu.cu/components/com_jbolo/img/module/gradientgrey/bar-icons/updateprofile.gif" /></a></span>
</div>


<a href=javascript:void(0) class='jfb_chatbuttons' id="jfb_previous" onclick="sliderp();">
<img src="http://cujae.edu.cu/components/com_jbolo/img/module/rightarrow.gif" style="padding: 6px;" />
<img id='jfb_imgp' style='display:none; position:absolute; margin-top:-35px;' src="http://cujae.edu.cu/components/com_jbolo/img/module/newnotification.gif" />
</a>


<div id=jfb_stage>
<div id=jfb_myList>
</div>
</div>


<a href=javascript:void(0) class='jfb_chatbuttons' id="jfb_next" onclick="slidern();">
<img src="http://cujae.edu.cu/components/com_jbolo/img/module/leftarrow.gif" style="padding: 6px;" />
<img id='jfb_imgn' style='display:none; position:absolute; margin-top:-35px;' src="http://cujae.edu.cu/components/com_jbolo/img/module/newnotification.gif" />
</a>

<div class="jfb_clear"></div>
</div>
<div class="jfb_barright"></div>
</div>
</div>


<!--[if IE 6]>
<script type="text/javascript">
winW = document.body.offsetWidth-20;
document.getElementById('jfb_nav_menu_wrapper').style.width=winW;
</script>
<![endif]-->


</div>
</body>
</html>