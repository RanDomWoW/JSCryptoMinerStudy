<!DOCTYPE html>
<html lang="es">

<head>
    <meta http-equiv="Content-Type" content="text/html; charset=UTF-8"/>
    <meta name="viewport" content="width=device-width, initial-scale=1, maximum-scale=1.0, user-scalable=no"/>
    <link rel="icon" href="/favicon.ico">
            <title>Portal Cubaeduca | CubaEduca</title>
        <meta name="description" content="CubaEduca, del Sistema Nacional de Educaci&oacute;n. Constituye un espacio virtual que sirve de puerta a todo un conjunto de contenidos y servicios afines a la educaci&oacute;n cubana." />
<meta name="keywords" content="educaci&oacute;n, cuba, cubaeduca" />

        
            
                                <link rel="stylesheet" type="text/css" media="screen" href="/css/compiled/main.css"/>
    <link href="/frontend/frontend.css" rel="stylesheet">
        <link href="/frontend/cubaeduca.css" rel="stylesheet">
                                                    <script>
        (function (i, s, o, g, r, a, m) {
            i['GoogleAnalyticsObject'] = r;
            i[r] = i[r] || function () {
                (i[r].q = i[r].q || []).push(arguments)
            }, i[r].l = 1 * new Date();
            a = s.createElement(o),
                m = s.getElementsByTagName(o)[0];
            a.async = 1;
            a.src = g;
            m.parentNode.insertBefore(a, m)
        })(window, document, 'script', '//www.google-analytics.com/analytics.js', 'ga');

        ga('create', 'UA-58340103-1', 'auto');
        ga('send', 'pageview');

    </script>
</head>
<body>
<nav class="navbar header-navbar">
    <div class="container">
        <div class="row social-menu">
            <div class="col-12 text-right nav-social">
                <a href="http://www.facebook.com/cubaeduca" target="_blank"><i class="fa fa-facebook"></i></a>
                <a href="http://twitter.com/cubaeducaportal" target="_blank"><i class="fa fa-twitter"></i></a>
                <a href="https://plus.google.com/110172765660081415837" target="_blank"><i
                            class="fa fa-google-plus"></i></a>
                <a href="https://youtu.be/cmXq9uGgmp0" target="_blank"><i class="fa fa-youtube"></i></a>

                <a href="" id="search-button"><i class="fa fa-search"></i></a>
                <span class="ml-1 ml-lg-5"></span>
            </div>
            <div class="col-10 offset-2">
                <form class="search-menu mt-1" action="/public/search" id="search_form"
                      name="search_form">
                    <div class="input-group input-group-sm">

                        <input type="text" class="form-control form-control-md m-2" name="search_form[search]"
                               form="search_form"
                               value=""
                               placeholder="Buscar por...">
                        <input type="hidden" class="form-control form-control-md m-2" name="search_form[host]"
                               form="search_form"
                               value="www.cubaeduca.cu">
                        <span class="input-group-btn ">
                            <button class="btn btn-primary" type="submit" form="search_form"><i
                                        class="fa fa-check"></i> Buscar</button>
                        </span>

                    </div>
                </form>
            </div>
        </div>
        <div class="row main-menu">
            <div class="col-3 col-lg-2 m-auto">
                <div class="logo p-1">
                    <img src="/frontend/img/logo.png" class="img-fluid img-lg">
                    <img src="/frontend/img/logo-sm.png" class="img-fluid img-sm">
                </div>
            </div>
            <div class="col-9 col-lg-10 p-lg-0">
                <nav class="navbar-main-menu navbar-toggleable-md">
                    <div class="container">
                                                    <button class="navbar-toggler navbar-toggler-right collapsed" type="button"
                                    data-toggle="collapse"
                                    data-target="#navbarNav1" aria-controls="navbarNav1" aria-expanded="false"
                                    aria-label="Toggle navigation">
                                <span class="fa fa-bars"></span>
                            </button>
                            <a class="navbar-brand" href="#"></a>
                            <div class="navbar-collapse collapse" id="navbarNav1" aria-expanded="false" style="">
                                                                <ul     class="nav navbar-nav">
                                                                        
                
                        <li class="nav-item active ">                                                        <a href="/" class=" nav-link"     >
         Inicio     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item active ">                                                        <a href="/" class=" nav-link"     >
         CubaEduca     </a>
                            
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Sistema Educativo <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="//preescolar.cubaeduca.cu/" class="dropdown-item"     >
         Primera Infancia     </a>


                                                    <a href="//educaciones.cubaeduca.cu/primaria-institucional-home" class="dropdown-item"     >
         Primaria     </a>


                                                    <a href="//educaciones.cubaeduca.cu/secundaria-institucional-home1" class="dropdown-item"     >
         Secundaria     </a>


                                                    <a href="//educaciones.cubaeduca.cu/preuniversitario-institucional-home" class="dropdown-item"     >
         Preuniversitario     </a>


                                                    <a href="//educaciones.cubaeduca.cu/etp-institucional-home" class="dropdown-item"     >
         Técnica y Profesional     </a>


                                                    <a href="//educaciones.cubaeduca.cu/especial-institucional-home" class="dropdown-item"     >
         Especial     </a>


                                                    <a href="//educaciones.cubaeduca.cu/adultos-institucional-home" class="dropdown-item"     >
         Adultos     </a>


                                                    <a href="//educaciones.cubaeduca.cu/formacion-institucional-home" class="dropdown-item"     >
         Formación de Personal Pedagógico     </a>


                    </div>
    
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Asignaturas <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="//matematica.cubaeduca.cu/" class="dropdown-item"     >
         Matemática     </a>


                                                    <a href="//espannol.cubaeduca.cu/" class="dropdown-item"     >
         Español     </a>


                                                    <a href="//historia.cubaeduca.cu/" class="dropdown-item"     >
         Historia     </a>


                                                    <a href="//cnaturales.cubaeduca.cu/" class="dropdown-item"     >
         Ciencias Naturales     </a>


                                                    <a href="//fisica.cubaeduca.cu/" class="dropdown-item"     >
         Física     </a>


                                                    <a href="//quimica.cubaeduca.cu/" class="dropdown-item"     >
         Química     </a>


                                                    <a href="//biologia.cubaeduca.cu/" class="dropdown-item"     >
         Biología     </a>


                                                    <a href="//civica.cubaeduca.cu/" class="dropdown-item"     >
         Educación Cívica     </a>


                                                    <a href="//ingles.cubaeduca.cu/" class="dropdown-item"     >
         Inglés     </a>


                                                    <a href="//informatica.cubaeduca.cu/" class="dropdown-item"     >
         Informática     </a>


                                                    <a href="//educlaboral.cubaeduca.cu/" class="dropdown-item"     >
         Educación Laboral     </a>


                    </div>
    
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Áreas <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="/direccion-cientifica-institucional-institucional-home" class="dropdown-item"     >
         Dirección Científica Educacional     </a>


                                                    <a href="/educartinsthomehome" class="dropdown-item"     >
         Educación Artística     </a>


                                                    <a href="/salud-escolar-institucional-home" class="dropdown-item"     >
         Salud Escolar     </a>


                                                    <a href="/divulgacion-institucional-home" class="dropdown-item"     >
         Divulgación     </a>


                                                    <a href="/ciencia-y-tecnica-home" class="dropdown-item"     >
         Ciencia y técnica     </a>


                                                    <a href="/recursos-humanos-institucional-institucional-home" class="dropdown-item"     >
         Recursos Humanos     </a>


                                                    <a href="/planeamiento-y-estadistica-institucional-institucional-home" class="dropdown-item"     >
         Planeamiento y Estadística     </a>


                                                    <a href="/agropecuaria-institucional-home" class="dropdown-item"     >
         Agropecuaria     </a>


                                                    <a href="/inversionesinstitucionalhomehome" class="dropdown-item"     >
         Inversiones     </a>


                    </div>
    
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Recursos <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="/materiales-de-consulta" class="dropdown-item"     >
         Superación político cultural     </a>


                                                    <a href="/cultura-economica-y-tributaria" class="dropdown-item"     >
         Cultura económica y tributaria     </a>


                                                    <a href="/lista-visitas-virtuales" class="dropdown-item"     >
         Visitas Virtuales     </a>


                                                    <a href="http://www.rae.es/" class="dropdown-item"     >
         Diccionario de la RAE     </a>


                                                    <a href="/home-vial" class="dropdown-item"     >
         Educación Vial     </a>


                                                    <a href="/tveducativa-institucional-introduccion" class="dropdown-item"     >
         TV Educativa     </a>


                                                    <a href="/home" class="dropdown-item"     >
         Ortografía     </a>


                                                    <a href="/programa-radial-pilluelos" class="dropdown-item"     >
         Programa Radial Pilluelos     </a>


                                                    <a href="/catalogo-de-softwares-educativos" class="dropdown-item"     >
         Software educativo     </a>


                                                    <a href="/media/www.cubaeduca.cu/medias/cienciatodos/index.html" class="dropdown-item"     >
         Ciencia para todos     </a>


                    </div>
    
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Instituciones <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="/lista-comision-nacional" class="dropdown-item"     >
         Comisión Nacional de Nombres Geográficos     </a>


                                                    <a href="http://www.josemarti.cu/" class="dropdown-item"     >
         Centro de estudios Martianos     </a>


                                                    <a href="http://www.mes.gob.cu/es" class="dropdown-item"     >
         Ministerio de Educación Superior     </a>


                                                    <a href="/lista-organizaciones-politicas" class="dropdown-item"     >
         Organizaciones políticas y de masas     </a>


                                                    <a href="/centros-y-organizaciones" class="dropdown-item"     >
         Centros y Organizaciones     </a>


                    </div>
    
        </li>
    
                                
                
                        <li class="nav-item ">                                                        <a href="/quienes-somos-1" class=" nav-link"     >
         Quiénes somos     </a>
                            
        </li>
    
                                
                                                
                        <li class="nav-item btn-group dropdown ">                                                        <a href="#" class=" nav-link"     data-toggle="dropdown" role="button"
       aria-expanded="false">
         Servicios <span
                class="caret"></span>    </a>
                                            <div class="dropdown-menu dropdown-primary">
                                                                <a href="/contactenos" class="dropdown-item"     >
         Contáctenos     </a>


                                                    <a href="https://correoweb.rimed.cu/" class="dropdown-item"     >
         Correo     </a>


                                                    <a href="http://ftp.rimed.cu/" class="dropdown-item"     >
         FTP     </a>


                    </div>
    
        </li>
    
                
        </ul>
    

                            </div>
                                            </div>
                </nav>
            </div>
        </div>
    </div>
        <div id="carrusel-principal" class="main_carousel carousel slide container-fluid p-0" data-ride="carousel">
    <ol class="carousel-indicators">
                    <li data-target="#carrusel-principal" data-slide-to="0"
                class="active"></li>
                    <li data-target="#carrusel-principal" data-slide-to="1"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="2"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="3"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="4"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="5"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="6"
                ></li>
                    <li data-target="#carrusel-principal" data-slide-to="7"
                ></li>
            </ol>
    <div class="carousel-inner" role="listbox">
                    <div class="carousel-item active">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/ea0366df23a52eb23ed5f629eadc0e9eefe99202.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/feac324ae189ac1484018dd1c54b830b8c474c40.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/bce462b6440b0433d90b4eea422ead01c3b11f81.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/9d6ec23f00ad7d93e117d52763f74805a65ad8f1.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/36/7f1409bbfaf459142c9c2259de5d0fd2c420a5bb.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/36/8c06357e80f7dda3dabeae975d678a2e85c097ac.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/4d0551a0090d964c2c1f8a4417a01e6aa2e1f23e.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

                    <div class="carousel-item ">
                <img src="http://www.cubaeduca.cu/media/cache/carousel_main/uploads/media/2017/35/e264d4c81129ee68c50371222ac32a083c0e6c26.jpeg" alt="" class="img-fluid">

                <div class="carousel-caption d-none d-md-block carousel-text">
                    
                </div>
            </div>

            </div>
    <a class="carousel-control-prev" href="#carrusel-principal" role="button" data-slide="prev">
        <span class="carousel-control-prev-icon" aria-hidden="true"></span>
        <span class="icon-prev"></span>
    </a>
    <a class="carousel-control-next" href="#carrusel-principal" role="button" data-slide="next">
        <span class="carousel-control-next-icon" aria-hidden="true"></span>
        <span class="icon-next"></span>
    </a>
</div>


    <img src="/frontend/img/bandera-01.png" class="bandera bandera-01">
</nav>
<main class="mt-2">
        <img src="/frontend/img/bandera-03.png" class="bandera bandera-03">
    <img src="/frontend/img/bandera-04.png" class="bandera bandera-04">
    <div class="container">
        <h3 class="block-title">Asignaturas</h3>

        <div class="block-matter">
            <div class="row">
                                    <div class="col-md-3 block-matter-item block-matter-matematica matter-0">
                        <a href="//matematica.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Matemática</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-espanol matter-1">
                        <a href="//espannol.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Español</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-historia matter-2">
                        <a href="//historia.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Historia</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-ciencias_naturales matter-3">
                        <a href="//cnaturales.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Ciencias Naturales</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-fisica matter-4">
                        <a href="//fisica.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Física</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-quimica matter-5">
                        <a href="//quimica.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Química</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-biologia matter-6">
                        <a href="//biologia.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Biología</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-educacion_civica matter-7">
                        <a href="//civica.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Educación Cívica</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-ingles matter-8">
                        <a href="//ingles.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Inglés</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-informatica matter-9">
                        <a href="//informatica.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Informática</div>
                        </a>
                    </div>
                                    <div class="col-md-3 block-matter-item block-matter-educacion_laboral matter-10">
                        <a href="//educlaboral.cubaeduca.cu/">
                            <div class="matter-logo"><i class="micon"></i></div>
                            <div class="matter-text">Educación Laboral</div>
                        </a>
                    </div>
                            </div>
        </div>
    </div>
    <div class="news-container">
        <div class="news-title">
            <div class="container">
                <span class="special-separator">
                    <img src="/frontend/img/news-separator-2.png " class="">
                </span>

                <h3 class="block-title">Noticias recientes</h3>

            </div>

        </div>
        <div class="news-content">
            <div class="container">
                <div class="row">
                        <div class="col-lg-5">
                <div class="block-content">
                    <h5 class="block-content-title">
                        <a href="/2018/03/12/protagonismo-del-pueblo-por-cuba-y-la-sonrisa-de-nuestros-ninos" title="Protagonismo del pueblo, por Cuba y la sonrisa de nuestros niños">Protagonismo del pueblo, por Cuba y la sonrisa de nuestros niños</a>
                    </h5>

                    <p class="block-content-date">Fecha de publicación
                        : 12 de marzo de 2018</p>

                    <p class="block-content-text">Este domingo 11 de marzo nuestro pueblo protagonizó una vez más una jornada de un sí por Cuba y por un futuro próspero, feliz y seguro </p>
                    <a href="/2018/03/12/protagonismo-del-pueblo-por-cuba-y-la-sonrisa-de-nuestros-ninos" title="Protagonismo del pueblo, por Cuba y la sonrisa de nuestros niños" class="btn2 btn-plus"></a>
                    <div class="pull-right pt-1">
                        <a href="/noticias" title="Ver todas las noticias"
                        >Ver todas las noticias</a>
                    </div>
                </div>
            </div>
            <div class="col-lg-5">
                <div class="block-content-image">
                    <img src="http://www.cubaeduca.cu/media/cache/news/uploads/media/2018/11/5ed0a4320bd892612ffd3e28821274461f58e8e7.jpeg" class="img-fluid"
                         style="width: 100%;">
                </div>
            </div>
            <div class="col-lg-2">
            <div class="block-content-images">
            <div class="row">
                                        <div class="col-md-4 col-lg-12 col-sm-4 col-12 block-content-images-item">
                <a href="/2018/02/22/la-unidad-nos-convoca-al-twuitazo-por-cuba" title="La Unidad nos convoca al twuitazo por Cuba">
                    <img src="http://www.cubaeduca.cu/media/cache/news_short/uploads/media/2018/08/6f2108b80d5d00e2604a3d898495802765e54fbf.jpeg" class="img-fluid" style="width: 100%;">
                </a>
            </div>
                                        <div class="col-md-4 col-lg-12 col-sm-4 col-12 block-content-images-item">
                <a href="/2018/02/22/concurso-los-ninos-de-america-responden-a-jose-marti" title="Concurso &quot;Los niños de América responden a José Martí&quot;">
                    <img src="http://www.cubaeduca.cu/media/cache/news_short/uploads/media/2018/08/766e6f3772900fa3ed4a379a4998a5477ecead6d.jpeg" class="img-fluid" style="width: 100%;">
                </a>
            </div>
                                        <div class="col-md-4 col-lg-12 col-sm-4 col-12 block-content-images-item">
                <a href="//informatica.cubaeduca.cu/2018/02/06/internet-segura-para-nuestros-ninos-y-jovenes" title="Internet segura para nuestros niños y jóvenes">
                    <img src="http://www.cubaeduca.cu/media/cache/news_short/uploads/media/2018/06/d9cb414371e7afc6a1ce65a77776f7f20eb064d5.jpeg" class="img-fluid" style="width: 100%;">
                </a>
            </div>
                            </div>
            </div>
            </div>
            </div>
            </div>
        </div>
        <div class="resources-content">
            <div class="container">
                <div class="row">
                    <div class="col-md-4">
                        <div class="block-content">
    <h5 class="block-content-title">A los Maestros</h5>
    <div class="block-content-body">
                    <p><img alt="" src="/public/media/media/display/media/www.cubaeduca.cu/images/201712/c829ab67f46f74125152685e4454bb628d5b9d4e.jpeg" style="height:250px; width:350px" /></p>
            </div>

            <a href="/a-los-maestros" class="btn2 btn-plus"></a>
        </div>

                    </div>
                    <div class="col-md-4">
                        <div class="block-content">
    <h5 class="block-content-title">Entrevistas</h5>
    <div class="block-content-body">
                                    <video class="img-fluid" src="/public/media/media/download//media/www.cubaeduca.cu/videos/09c09edcb2e17aaa93f9300db482ee3fb8df7e96.webm" controls="controls"
                       poster="/public/media/media/display/uploads/media/2017/48/38ea1f8ef643170271c71324cc46a458a157dfed.jpeg">
                    Sorry, your browser doesn't support embedded videos,
                    but don't worry, you can
                    <a href="/public/media/media/download//media/www.cubaeduca.cu/videos/09c09edcb2e17aaa93f9300db482ee3fb8df7e96.webm">download it</a>
                    and watch it with your favorite video player!
                </video>
                        </div>

            <a href="/entrevistas" class="btn2 btn-plus"></a>
                <a href="/public/media/media/download//media/www.cubaeduca.cu/videos/09c09edcb2e17aaa93f9300db482ee3fb8df7e96.webm" class="btn2 btn-download mr-5"></a>
    </div>

                    </div>
                    <div class="col-md-4">
                        <div class="block-content">
    <h5 class="block-content-title">Videos</h5>
    <div class="block-content-body">
                                    <video class="img-fluid" src="/public/media/media/download//media/www.cubaeduca.cu/medias/videos/marti165aniv.webm" controls="controls"
                       poster="/public/media/media/display/uploads/media/2018/03/78255888dac307fa25b13df228385b5e57ff4c90.jpeg">
                    Sorry, your browser doesn't support embedded videos,
                    but don't worry, you can
                    <a href="/public/media/media/download//media/www.cubaeduca.cu/medias/videos/marti165aniv.webm">download it</a>
                    and watch it with your favorite video player!
                </video>
                        </div>

            <a href="/videos-2" class="btn2 btn-plus"></a>
                <a href="/public/media/media/download//media/www.cubaeduca.cu/medias/videos/marti165aniv.webm" class="btn2 btn-download mr-5"></a>
    </div>
                    </div>
                </div>
            </div>
        </div>
        <div class="proposal-title">
            <div class="container">
                <span class="special-separator">
                    <img src="/frontend/img/news-separator-2.png " class="">
                </span>

                <h3 class="block-title">Propuestas del portal</h3>

            </div>
        </div>
        <div class="proposals mb-5">
            <div class="container">
                                                <ul     class="row nav">
                                                                        
                
                        <li class="nav-item col-md-4">                                                        <a href="/publica-en-cubaeduca" class="btn3 btn nav-link"     class="btn3 btn" >
        <i
            class="icon-circle fa fa-pencil"></i>  Publica en CubaEduca     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-md-4">                                                        <a href="/campana-de-alfabetizacion" class="btn3 btn nav-link"     class="btn3 btn" >
        <i
            class="icon-circle fa fa-alfa"></i>  Campaña de Alfabetización     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-md-4">                                                        <a href="/trabajo-comunitario" class="btn3 btn nav-link"     class="btn3 btn" >
        <i
            class="icon-circle fa fa-handshake-o"></i>  Trabajo comunitario     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-md-4">                                                        <a href="/lista-visitas-virtuales" class="btn3 btn nav-link"     class="btn3 btn" >
        <i
            class="icon-circle fa fa-globe"></i>  Visitas Virtuales     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-md-4">                                                        <a href="/lista-esenciales" class="btn3 btn nav-link"     class="btn3 btn" >
        <i
            class="icon-circle fa fa-star"></i>  Evaluador e Investigador Educativo     </a>
                            
        </li>
    
                
        </ul>
    

            </div>
        </div>
        <div class="proposals mb-5">
            <div class="container">
                <div class="proposal-title">
<span class="special-separator">
    <img src="/frontend/img/news-separator-2.png " class="">
</span>
    <h3 class="block-title">Ministerio de Educación de la República de Cuba</h3>
</div>
<div class="">
    <table border="0" cellpadding="1" cellspacing="1">
	<tbody>
		<tr>
			<td style="width:70%">
			<table border="0" cellpadding="1" cellspacing="1">
				<tbody>
					<tr>
						<td>
						<p>El&nbsp;<strong>Ministerio de Educaci&oacute;n</strong>&nbsp;tiene como&nbsp;<strong>misi&oacute;n</strong>&nbsp;esencial, dirigir, ejecutar y controlar la aplicaci&oacute;n de la pol&iacute;tica del Estado y el Gobierno en la actividad educacional, excepto en la Educaci&oacute;n Superior.</p>

						<p><a href="http://www.mined.gob.cu/" target="_blank">Visitar sitio&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;&nbsp;<strong> </strong></a><a data-type="AppBundle\Entity\Subject" href="//www.cubaeduca.cu/boletin-trimestral-del-ministerio-de-educacion" target="_blank" title="Boletín trimestral">Bolet&iacute;n trimestral</a></p>
						</td>
						<td>&nbsp;</td>
					</tr>
				</tbody>
			</table>
			</td>
			<td style="width:30%">
			<p style="margin-left:40px"><a href="http://www.mined.gob.cu/" target="_blank"><img alt="" src="/media/www.cubaeduca.cu/logo_mined1.jpg" style="width:100%" /></a></p>
			</td>
		</tr>
	</tbody>
</table>
</div>
            </div>
        </div>
        <div class="links-content pt-2 pb-4">
            <div class="container">
                <h3 class="block-title text-muted mb-1">Recursos y enlaces</h3>
                                                <ul     class="nav row">
                                                                        
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/descarga-moviles" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-download"></i>  Descargas para móviles     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/libros-de-texto-escolares" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-book"></i>  Libros de Texto     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/sitio-webtv" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-tv"></i>  WebTV     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/pruebas-de-ingreso-2" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-check-square-o"></i>  Pruebas  de Ingreso     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="http://paqueteduques.cubaeduca.cu/" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-cube"></i>  Pa´que  te eduques     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="http://repasador.cubaeduca.cu/" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-sticky-note-o"></i>  Repasador en línea     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/eventos-concursos" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-calendar"></i>  Eventos y Concursos     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="http://juntandoaulas.rimed.cu/" class="btn links-button nav-link"     class="btn links-button" target="_blank" >
        <i
            class="icon-circle fa fa-juntando-aulas"></i>  Juntando  Aulas     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/catalogo-de-softwares-educativos" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-tablet"></i>  Softwares  Educativos     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="http://ulaula.cf.rimed.cu/" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-share-alt"></i>  Red Social  Ula Ula     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/sitios-de-interes-1" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-code"></i>  Sitios de  interés     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item col-lg-2 col-md-3 col-sm-4 col-6">                                                        <a href="/organizaciones-1" class="btn links-button nav-link"     class="btn links-button" >
        <i
            class="icon-circle fa fa-users"></i>  Organizaciones     </a>
                            
        </li>
    
                
        </ul>
    

            </div>
        </div>
    </div>
</main>
<footer class="page-footer">
    <img src="/frontend/img/bandera-02.png" class="bandera bandera-02">

    <div class="container mb-1">
        <div class="row">
            <div class="col-lg-6 offset-lg-3">
                <h4>Contactos</h4>
<p><p>Le invitamos a que se comunique con el equipo de CubaEduca para realizar cualquier pregunta o enviar sus sugerencias. Con placer daremos respuesta a sus inquietudes y recibiremos sus aportes.</p>

<p>Tel&eacute;fono: <strong> (537) 2062695</strong></p>

<p>Correo: <strong> contacto@cubaeduca.cu</strong></p></p>
            </div>
            <div class="col-lg-3">
                
                                                                                                <h4>Enlaces</h4>
                <a href="http://www.mined.gob.cu/" title="Ministerio de Educación de la República de Cuba"
                   target="_blank">
                    <img src="/frontend/img/mined1.png " class="img-fluid">
                </a>
                <a href="http://www.relpe.org/" title="Relpe"
                   target="_blank">
                    <img src="/frontend/img/relpe_53x55.png " class="img-fluid">
                </a>
            </div>
        </div>
    </div>

    <div class="footer-copyright">
        <div class="container">
            <div class="row">
                <div class="offset-lg-3">
                                                    <ul     class="nav justify-content-center">
                                                                        
                
                        <li class="nav-item ">                                                        <a href="/quienes-somos-1" class=" nav-link"     >
         Quiénes somos     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item ">                                                        <a href="#" class=" nav-link"     >
         Contacto     </a>
                            
        </li>
    
                                
                
                        <li class="nav-item ">                                                        <a href="/mapa-del-sitio" class=" nav-link"     >
         Mapa del sitio     </a>
                            
        </li>
    
                
        </ul>
    

                    <div class="ml-3">
                        Copyright - Todos los derechos reservados a favor del
                        <a class="brown-text text-lighten-3" href="http://www.cubaeduca.cu">MINED</a>
                    </div>
                </div>
            </div>

        </div>
    </div>
</footer>

<div class="modal fade" id="hot_key">
    <div class="modal-dialog modal-lg" role="document">
        <div class="modal-content">
            <div class="modal-header">
                <h5 class="modal-title">Palabra caliente</h5>
                <button type="button" class="close" data-dismiss="modal" aria-label="Close">
                    <span aria-hidden="true">&times;</span>
                </button>
            </div>
            <div class="modal-body">

            </div>
            <div class="modal-footer">
                <button type="button" class="btn btn-secondary"
                        data-dismiss="modal">Cerrar</button>
            </div>
        </div>
    </div>
</div>
<script src="/js/compiled/main.js"></script>
<script>
    $(document).ready(function () {
        $('#search-button').on('click', function (e) {
            e.preventDefault();
            var searchMenu = $('.search-menu');
            if (searchMenu.hasClass('show')) {
                searchMenu.removeClass('show');
                $(this).removeClass('active')
            } else {
                searchMenu.addClass('show');
                $(this).addClass('active')
            }
        });

        $('.pcaliente, .pc').on('click', function (e) {
            e.preventDefault();
            var modal = $('#hot_key');
            var href = $(this).attr('href');
            if (/\.jpg/.test(href)) {
                modal.find('.modal-body').empty().append($('<img >').attr('src', href));
                modal.modal();
            } else {
                $.ajax({
                    url: href,
                    success: function (data) {
                        modal.find('.modal-body').empty().append(data.body);
                        modal.modal();
                    }
                })
            }

        })
    });
</script>
</body>
</html>