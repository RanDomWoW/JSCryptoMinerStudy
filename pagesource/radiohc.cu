#include "hip/hip_runtime.h"
﻿<!DOCTYPE html>
<html lang="es">
<head>
        <meta charset="UTF-8"/>
  <!--
	<meta name="twitter:title" content="Portada | Radio Habana Cuba | Emisora Internacional"/>
	<meta name="twitter:card" content="summary"/>
	<meta name="twitter:site" content="@RadioHabanaCuba"/>
	<meta name="twitter:creator" content="@RadioHabanaCuba"/>
	<meta name="twitter:description" content="Emisora de Onda Corta que transmite en Español, English, Français, Português, Kreyol, Esperanto y العربية;  informaciones y noticias sobre el acontecer de Cuba y el mundo."/>
 -->
    <!-- metas para responsive -->
    <meta name="viewport" content="width=device-width"/>
    <meta name="viewport" content="height=device-height"/>
    <!-- metas para SEO, se toman los mismos de siempre para no perder posicionamiento en search engines-->
    <meta name="keywords" content="Radio,Habana,Cuba,Prensa,Filatelia,Online,emisora,onda,transmisión,internacional"/>
    <meta name="description" content="Emisora de Onda Corta que transmite en Español, English, Français, Português, Kreyol, Esperanto y العربية; informaciones y noticias sobre el acontecer de Cuba y el mundo."/>
    <meta name="generator" content="Radio Habana Cuba"/>
    <!-- fin de los metas -->
    <!-- Styles for foundation-->
    <!-- <link rel="canonical" href="http://www.radiohc.cu/"/>  -->
    <link rel="stylesheet" href="/themes/v2.0/vendor/foundation5/css/foundation.css">
    <link rel="stylesheet" href="/themes/v2.0/vendor/font-awesome/font-awesome.min.css">
    <link rel="stylesheet" href="/themes/v2.0/css/app.css">
            <script src="/themes/v2.0/vendor/jquery.js"></script>
    <script src="/themes/v2.0/vendor/foundation5/js/vendor/modernizr.js"></script>
    <script src="/themes/v2.0/vendor/foundation5/js/foundation.min.js"></script>
    <script src="/themes/v2.0/js/layout.js"></script>
    <link href="/themes/v2.0/img/favicon.ico" rel="shortcut icon" type="image/x-icon">
    <link rel="stylesheet" type="text/css" href="/themes/v2.0/vendor/slick/slick.css" />
<link rel="stylesheet" type="text/css" href="/themes/v2.0/vendor/slick/slick-theme.css" />
<link rel="stylesheet" type="text/css" href="/themes/v2.0/vendor/mediaelement/css/mediaelementplayer.min.css" />
<script type="text/javascript" src="/themes/v2.0/vendor/slick/slick.min.js"></script>
<script type="text/javascript" src="/themes/v2.0/js/carrusell.js"></script>
<script type="text/javascript" src="/themes/v2.0/vendor/mediaelement/js/mediaelement-and-player.min.js"></script>
<script type="text/javascript" src="/themes/v2.0/vendor/mediaelement/js/audiowidget.js"></script>
<script type="text/javascript" src="/themes/v2.0/js/subscription.js"></script>
<script type="text/javascript">
/*<![CDATA[*/
var isSingle = false;
/*]]>*/
</script>
<title>Portada | Radio Habana Cuba | Emisora Internacional</title>
        <script>
        var baseUrl = '';
    </script>
</head>
<body>
    <div>
                <div class="header_bg row collapse">
                        <div class="page_title large-7 columns">
                            <div class="row collapse">
                    <div class="large-3 columns" id="logo_container">
                        <div >
                            <img src="/themes/v2.0/img/Logo_56_RHC.png" alt="" />                        </div>
                    </div>
                    <div class="large-9 columns" id="title_container">
                        <h1 class="name_rhc">Radio Habana Cuba</h1>
                        <h2 class="name_slogan">Una voz de amistad que recorre el mundo</h2>
                    </div>
                </div>
            </div>
                        <div class="controls large-5 columns" id="header_ctrls">
                        <div id="languages">
                <ul class="inline-list" >
                    <li><a href="/" title="Español"><div class="es"></div></a></li>
                    <li><a href="/en" title="English"><div class="gb"></div></a></li>
                    <li><a href="/pt" title="Português"><div class="pt"></div></a></li>
                    <li><a href="/eo" title="Esperanto"><div class="eo"></div></a></li>
                    <li><a href="/ar" title="العربية"><div class="sa"></div></a></li>
                    <li><a href="/fr" title="Français"><div class="fr"></div></a></li>
					<li><a href="/it" title="Italiano"><div class="it"></div></a></li>
                </ul>
            </div>
            <div id="social">
                <ul class="inline-list">
                    <li class="list-label"><span class="label">Síguenos en</span></li>
                    <li><a href="http://www.facebook.com/RadioHabanaCuba" title="Radio Habana Cuba en Facebook"  target="_blank"><div class="facebook"></div></a></li>
                    <li><a href="http://twitter.com/RadioHabanaCuba" title="Radio Habana Cuba en Twitter"  target="_blank"><div class="twitter"></div></a></li>
					<li><a href="https://www.youtube.com/channel/UCPED0Qc0fWzAvBwhXhJ723Q" title="Radio Habana Cuba en Youtube"  target="_blank"><div class="youtube"></div></a></li>
                    <li><a href=""><div class="googleplus"></div></a></li>
                    <li><a href="/es/statics/rss" title="Canal RSS principal"  target="_blank"><div class="rss"></div></a></li>
                    <li><a href="mailto:radiohc@enet.cu" title="Enviar mensaje"><div class="email"></div></a></li>
                </ul>
              </div>
        </div><h2 class="name_etiqueta">#PorSiempreFidel #LegadoDeFidel</h2>
                    </div>
        <div class="contain-to-grid sticky">
                <nav class="top-bar" data-topbar role="navigation" data-options="sticky_on: large">
                    <ul class="title-area"><li class="name"></li><li class="toggle-topbar menu-icon"><a href="#"><span>Menu</span></a></li></ul><section class="top-bar-section"><ul class="left" id="yw0">
<li><a href="/">Inicio</a></li>
<li class="has-dropdown not-click"><a href="/noticias">Noticias</a>
<ul class="dropdown">
<li><a href="/noticias/nacionales">Nacionales</a></li>
<li><a href="/noticias/internacionales">Internacionales</a></li>
<li><a href="/noticias/cultura">Cultura</a></li>
<li><a href="/noticias/salud">Salud</a></li>
<li><a href="/noticias/economy">Economía/Turismo</a></li>
<li><a href="/noticias/ciencias">Ciencias/Medio Ambiente</a></li>
<li><a href="/noticias/deportes">Deportes</a></li>
</ul>
</li>
<li class="has-dropdown not-click"><a href="/especiales">Especiales</a>
<ul class="dropdown">
<li><a href="/especiales/por-fidel">Por Fidel</a></li>
<li><a href="/especiales/quien-da-luz">Quien da Luz</a></li>
<li><a href="/especiales/elecciones-cuba">Elecciones Cuba</a></li>
<li><a href="/especiales/comentarios">Comentarios</a></li>
<li><a href="/especiales/exclusivas">Especiales</a></li>
<li><a href="/especiales/resumenes">Resumenes</a></li>
<li><a href="/especiales/cubausa">USA-Cuba</a></li>
<li><a href="/especiales/cumbre-americas">Cumbre de las Américas</a></li>
</ul>
</li>
<li class="has-dropdown not-click"><a href="/interesantes">Interesantes</a>
<ul class="dropdown">
<li><a href="/interesantes/caleidoscopio">Caleidoscopio Cubano</a></li>
<li><a href="/interesantes/miscelanea">Miscelánea</a></li>
<li><a href="/interesantes/filatelia">El mundo de la Filatelia</a></li>
<li><a href="/galerias">Galería de Imágenes</a></li>
<li><a href="/interesantes/concursos">Concursos</a></li>
<li><a href="/interesantes/frecuencias">Frecuencia</a></li>
<li><a href="/interesantes/programacion">Programación</a></li>
<li><a href="/correspondencia">Correspondencia</a></li>
<li><a href="/interesantes/estaticas/quienes_somos">¿Quiénes Somos?</a></li>
<li><a href="/interesantes/links">Enlaces Web</a></li>
</ul>
</li>
<li><a href="/audios">Audios bajo demanda</a></li>
<li><a href="/heroes">5 Héroes</a></li>
</ul></section>                    <section class="top-bar-section">

                        <!-- Right Nav Section -->
                        <ul class="right">
                            <li class="divider"></li>
                            <li class="has-form">
                                <div class="row collapse">
                                    <div class="large-8 small-9 columns">
                                        <form class="incrustated" id="search_rhc" action="/es/especiales/search" method="get">                                        <input type="text" placeholder="Búsqueda" name="query">
                                        </form>                                    </div>
                                    <div class="large-4 small-3 columns">
                                        <a href="#" class="info button expand fa fa-search" id="searchlink"></a>
                                    </div>
                                </div>
                            </li>
                        </ul>
                    </section>
                </nav>
            </div>
                <div><div id="main" class="row">
    <div class="large-9 columns" id="content">
        <div class="row">
            <div class="row collapse">
                <div class="highLight"><h6 class="art-title"><span class="intro">Últimas Noticias: </span><a href="/noticias/internacionales/158008-lula-confia-en-que-el-pt-vuelva-a-gobernar-brasil">Lula confía en que el PT vuelva a gobernar Brasil</a></h6></div>            </div>
        </div>
        <div class="row">
            <div class="row collapse section_row">
                <div class="carrusell carrusell_container"><div class="carrusell horizontal_art"><div class="image_article"><img src="/uploads/images/articulos/thumbnails/9113-camapana-lula.jpeg" alt="Imagen de archivo" /></div><div class="content_article"><h5 class="title_article"><a title="Lula confía en que el PT vuelva a gobernar Brasil" href="/noticias/internacionales/158008-lula-confia-en-que-el-pt-vuelva-a-gobernar-brasil">Lula confía en que el PT vuelva a gobernar Brasil</a></h5><div class="excerpt"><p>Brasilia, 20 mar (RHC-PL) El ex presidente Luiz Inácio Lula da Silva manifestó estar convencido de que en las elecciones de octubre próximo el pueblo traerá... <a class="more" href="/noticias/internacionales/158008-lula-confia-en-que-el-pt-vuelva-a-gobernar-brasil">Leer Más</a></p></div></div><div class="clear"></div></div><div class="carrusell horizontal_art"><div class="image_article"><img src="/uploads/images/articulos/thumbnails/9561-correa.jpg" alt="Imagen de archivo" /></div><div class="content_article"><h5 class="title_article"><a title="Rafael Correa: Lenín Moreno es el Temer de Ecuador" href="/noticias/internacionales/158006-rafael-correa-lenin-moreno-es-el-temer-de-ecuador">Rafael Correa: Lenín Moreno es el Temer de Ecuador</a></h5><div class="excerpt"><p>Brasilia, 20 mar (RHC) El expresidente ecuatoriano Rafael Correa ha criticado las recientes posturas y medidas del presidente de Ecuador, Lenin Moreno, después... <a class="more" href="/noticias/internacionales/158006-rafael-correa-lenin-moreno-es-el-temer-de-ecuador">Leer Más</a></p></div></div><div class="clear"></div></div><div class="carrusell horizontal_art"><div class="image_article"><img src="/uploads/images/articulos/thumbnails/2861-protesta-senado-usa.jpg" alt="Foto: Hispantv" /></div><div class="content_article"><h5 class="title_article"><a title="Arrestan a manifestantes en EEUU por pedir ayuda para Puerto Rico" href="/noticias/internacionales/158004-arrestan-a-manifestantes-en-eeuu-por-pedir-ayuda-para-puerto-rico">Arrestan a manifestantes en EEUU por pedir ayuda para Puerto Rico</a></h5><div class="excerpt"><p>Washington, 20 mar (RHC) Varios miembros de la coalición Power4PuertoRico fueron arrestados mientras realizaban una manifestación en el senado de EE.UU., en p... <a class="more" href="/noticias/internacionales/158004-arrestan-a-manifestantes-en-eeuu-por-pedir-ayuda-para-puerto-rico">Leer Más</a></p></div></div><div class="clear"></div></div></div>            </div>
        </div>
        <div class="row" id="blocks">
            <div class="row collapse section_row">
                <div class="large-6 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa cuba"></i></span><a href="/noticias/nacionales">Nacionales</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/nacionales/158001-embajador-de-japon-en-cuba-inaugura-proyectos-agricolas-en-camagüey">Embajador de Japón en Cuba inaugura proyectos agrícolas en Camagüey</a></h5><div class="image_article floated"><img src="/uploads/images/articulos/thumbnails/7465-masuro-watanabe.jpg" alt="" /></div><p class="floated">Camagüey, 20 mar (RHC-ACN) Masaru Watanabe, embajador de Japón en Cuba, inauguró, en la provincia de Camagüey, tres proyectos de producción y conservación de semillas en la agricultura urbana y ... <a class="more" href="/noticias/nacionales/158001-embajador-de-japon-en-cuba-inaugura-proyectos-agricolas-en-camagüey">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/nacionales/158000-afirma-presidente-de-la-asociacion-cubana-de-naciones-unidas-que-nuestra-sociedad-civil-esta-al-servicio-del-pueblo"> * Afirma presidente de la Asociación Cubana de Naciones Unidas que nuestra sociedad civil esta al servicio del pueblo</a></li><li class="font85"><a href="/noticias/nacionales/157999-destacan-labor-del-centro-cubano-de-inmunologia-molecular-proximo-a-cumplir-25-anos-de-creado"> * Destacan labor del Centro cubano de Inmunología Molecular próximo a cumplir 25 años de creado</a></li><li class="font85"><a href="/noticias/nacionales/157991-expuesta-escultura-de-toro-semental-en-la-habana"> * Expuesta escultura de toro semental en La Habana</a></li></ul><div class="section_v font80"><a href="/noticias/nacionales">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
                <div class="large-6 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa elmundo"></i></span><a href="/noticias/internacionales">Internacionales</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/internacionales/158008-lula-confia-en-que-el-pt-vuelva-a-gobernar-brasil">Lula confía en que el PT vuelva a gobernar Brasil</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/9113-camapana-lula.jpeg" alt="Imagen de archivo" /></div><p class="excerpt">Brasilia, 20 mar (RHC-PL) El ex presidente Luiz Inácio Lula da Silva manifestó estar convencido de que en las elecciones de octubre próximo el pueblo traerá... <a class="more" href="/noticias/internacionales/158008-lula-confia-en-que-el-pt-vuelva-a-gobernar-brasil">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/internacionales/158006-rafael-correa-lenin-moreno-es-el-temer-de-ecuador"> * Rafael Correa: Lenín Moreno es el Temer de Ecuador</a></li><li class="font85"><a href="/noticias/internacionales/158004-arrestan-a-manifestantes-en-eeuu-por-pedir-ayuda-para-puerto-rico"> * Arrestan a manifestantes en EEUU por pedir ayuda para Puerto Rico</a></li><li class="font85"><a href="/noticias/internacionales/158002-peru-presentan-pruebas-sobre-compra-de-votos-contra-cese-de-presidente-ppk"> * Perú: Presentan pruebas sobre compra de votos contra cese de presidente PPK</a></li></ul><div class="section_v font80"><a href="/noticias/internacionales">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
            </div>
            <div class="row collapse section_row">
                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa deportes"></i></span><a href="/noticias/deportes">Deportes</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/deportes/157983-caciques-recibe-este-fin-de-semana-a-domadores">Caciques recibe este fin de semana a Domadores</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/5530-maestre.jpg" alt="" /></div><p class="excerpt">La Habana, 20 mar (RHC) Caciques de Venezuela volverá a la acción en la VIII Serie Mundial de Boxeo cuando enfrente este viernes y sábado a los Domadores de ... <a class="more" href="/noticias/deportes/157983-caciques-recibe-este-fin-de-semana-a-domadores">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/deportes/157972-equipo-cubano-de-futbol-topara-con-nicaragua"> * Equipo cubano de fútbol topará con Nicaragua</a></li><li class="font85"><a href="/noticias/deportes/157909-sancti-spiritus-vence-en-campeonato-nacional-de-baloncesto-para-damas"> * Sancti Spíritus vence en Campeonato Nacional de Baloncesto para damas</a></li><li class="font85"><a href="/noticias/deportes/157901-el-ciclismo-de-barranquilla-2018-ya-tiene-a-sus-clasificados"> * El ciclismo de Barranquilla 2018 ya tiene a sus clasificados</a></li></ul><div class="section_v font80"><a href="/noticias/deportes">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa cultura"></i></span><a href="/noticias/cultura">Cultura</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/cultura/158007-cuba-concluye-con-exito-conferencia-internacional-de-los-pueblos">Cuba: Concluye con éxito Conferencia Internacional de los Pueblos</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/1726-conferencia-pueblos.JPG" alt="Foto: Armando Ernesto Contreras Tamayo/ACN" /></div><p class="excerpt">Bayamo, 20 mar (RHC-ACN) La XI Conferencia Internacional de los pueblos y su cultura, dedicada a la música cubana, concluyó de forma exitosa en la ciudad de B... <a class="more" href="/noticias/cultura/158007-cuba-concluye-con-exito-conferencia-internacional-de-los-pueblos">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/cultura/157980-sesiona-desde-hoy-festival-itinerante-de-cine-comunitario-en-granma"> * Sesiona desde hoy Festival Itinerante de Cine Comunitario en Granma</a></li><li class="font85"><a href="/noticias/cultura/157974-en-cuba-festival-havana-world-music"> * En Cuba Festival Havana World Music</a></li><li class="font85"><a href="/noticias/cultura/157973-rendiran-tributo-a-celina-gonzalez-la-reina-del-punto-cubano"> * Rendirán tributo a Celina González, la Reina del Punto Cubano</a></li></ul><div class="section_v font80"><a href="/noticias/cultura">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa exclusiva"></i></span><a href="/especiales/exclusivas">Especiales</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/especiales/exclusivas/157926-corrupcion-detras-del-puente">Corrupción detrás del puente</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/2871-colapso-puente-miami-Florida-Reuters.jpg" alt="Puente colapsado. Foto: Internet" /></div><p class="excerpt">Mucho se ha publicado en la prensa norteamericana   sobre la  desastrosa tragedia ocurrida  en  la ciudad  de Miami,  al desplomarse  un puente peatonal en las ... <a class="more" href="/especiales/exclusivas/157926-corrupcion-detras-del-puente">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/especiales/exclusivas/157857-sergio-gonzalez-el-curita-a-60-anos-de-su-asesinato"> * Sergio González, El Curita: a 60 años de su asesinato</a></li><li class="font85"><a href="/especiales/exclusivas/157819-roberto-morejon-analogico-o-digital-periodista!"> * Roberto Morejón: Analógico o digital? Periodista!</a></li><li class="font85"><a href="/especiales/exclusivas/157803-corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana"> * Corrupción político-empresarial en Miami, muertes, y alianza anticubana</a></li></ul><div class="section_v font80"><a href="/especiales/exclusivas">&lt;&lt;ver más&gt;&gt;</a></div></div></div>
                </div>
            </div>
            <div class="row collapse section_row">
                <div class="large-12 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa comentarios"></i></span><a href="/especiales/comentarios">Comentarios</a></div><div class="section_c"><div class="section_n large-6 columns"><h5 class="title_article"><a href="/especiales/comentarios/157923-exigen-aclarar-asesinato-de-feminista-brasilena">Exigen aclarar asesinato de feminista brasileña</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/5077-Marielle-brasil-asesinato-Cuatro.jpeg" alt="Multitudinaria manifestación en Brasil al grito de ¿quién mató a Marielle Franco? Foto/ Noticias Cuatro." /></div><p class="excerpt">¿Quién mató a Marielle Franco? es el clamor que se extiende por diversos lugares de la geografía brasileña y que pone, una vez más, en serios aprietos al ... <a class="more" href="/especiales/comentarios/157923-exigen-aclarar-asesinato-de-feminista-brasilena">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div><div class="section_n large-6 columns"><h5 class="title_article"><a href="/especiales/comentarios/157920-maestros-puertorriquenos-rechazan-privatizacion-del-sector-educativo">Maestros puertorriqueños rechazan privatización del sector educativo</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/4005-maestros_puerto_rico-protestas-Telesur.jpg" alt="Foto/Telesur" /></div><p class="excerpt">Sin haberse repuesto de los daños ocasionados por el paso del huracán María en septiembre pasado, los puertorriqueños se enfrentan hoy a la decisión de las... <a class="more" href="/especiales/comentarios/157920-maestros-puertorriquenos-rechazan-privatizacion-del-sector-educativo">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/especiales/comentarios/157902-parlamento-que-se-constituira-en-cuba-respondera-a-intereses-de-toda-la-sociedad"> * Parlamento que se constituirá en Cuba responderá a intereses de toda la sociedad</a></li><li class="font85"><a href="/especiales/comentarios/157976-donald-trump-fusilar-a-narcotraficantes"> * Donald Trump: ¿Fusilar a narcotraficantes?</a></li></ul><div class="section_v font80"><a href="/especiales/comentarios">&lt;&lt;ver más&gt;&gt;</a></div></div></div>
                </div>
            </div>
            <div class="row collapse section_row">
                <div class="large-6 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa economia"></i></span><a href="/noticias/economy">Economía/Turismo</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/economy/157965-exporta-grupo-agroforestal-cubano-carbon-vegetal-a-europa">Exporta Grupo agroforestal cubano carbón vegetal a Europa</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/6482-cuba-carbon-vegetal.jpg" alt="El grupo Agroforestal del Ministerio de la Agricultura de Cuba exporta más de 130 mil toneladas anuales de carbón vegetal a Europa.Foto:Archivo." /></div><p class="excerpt">La Habana, 20 mar (RHC) El grupo Agroforestal del Ministerio de la Agricultura de Cuba exporta más de 130 mil toneladas anuales de carbón vegetal a Europa.... <a class="more" href="/noticias/economy/157965-exporta-grupo-agroforestal-cubano-carbon-vegetal-a-europa">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/economy/157904-restringe-estados-unidos-transacciones-que-involucren-al-petro-venezolano"> * Restringe Estados Unidos transacciones que involucren al Petro venezolano</a></li><li class="font85"><a href="/noticias/economy/157894-alto-funcionario-del-ministerio-cubano-de-turismo-cumplimento-agenda-de-trabajo-en-moscu"> * Alto funcionario del ministerio cubano de Turismo cumplimentó agenda de trabajo en Moscú</a></li><li class="font85"><a href="/noticias/economy/157882-develan-estatua-representativa-del-desarrollo-ganadero-de-cuba"> * Develan estatua representativa del desarrollo ganadero de Cuba</a></li></ul><div class="section_v font80"><a href="/noticias/economy">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
                <div class="large-6 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa ciencia"></i></span><a href="/noticias/ciencias">Ciencias/Medio Ambiente</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/ciencias/157988-parques-eolicos-en-reino-unido-aumentan-aporte-de-energia">Parques eólicos en Reino Unido aumentan aporte de energía</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/9206-parque-eolico.jpg" alt="Imagen de archivo" /></div><p class="excerpt">Londres, 20 mar (RHC-PL) Parques eólicos en Reino Unido registran una creciente participación en la generación de electricidad, al cubrir hasta el 37 por cie... <a class="more" href="/noticias/ciencias/157988-parques-eolicos-en-reino-unido-aumentan-aporte-de-energia">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/ciencias/157979-detectan-asteroide-de-otro-sistema-solar"> * Detectan asteroide de otro sistema solar </a></li><li class="font85"><a href="/noticias/ciencias/157975-presenta-academico-japones-en-informatica-habana-robot-humanoide"> * Presenta académico japonés en Informática Habana robot humanoide
</a></li><li class="font85"><a href="/noticias/ciencias/157970-en-cuba-advierten-sobre-perdida-de-especies-de-la-fauna"> * En Cuba advierten sobre pérdida de especies de la fauna

</a></li></ul><div class="section_v font80"><a href="/noticias/ciencias">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
            </div>
            <div class="row collapse section_row">
				                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa salud"></i></span><a href="/noticias/salud">Salud</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/noticias/salud/157978-fidel-castro-artifice-de-la-cardiologia-pediatrica-en-cuba">Fidel Castro, artífice de la cardiología pediátrica en Cuba</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/6170-cardiocentro.jpg" alt="El &#039;William Soler&#039; exhibe un índice de supervivencia superior al 90 por ciento. Foto: Archivo" /></div><p class="excerpt">La Habana, 20 mar (RHC) La doctora Herminia Palenzuela al evocar este 20 de marzo, en Santiago de Cuba, el aporte del líder histórico de la Revolución cubana... <a class="more" href="/noticias/salud/157978-fidel-castro-artifice-de-la-cardiologia-pediatrica-en-cuba">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/noticias/salud/157967-inmunologia-molecular-lider-en-farmacos-contra-el-cancer-en-cuba"> * Inmunología Molecular: líder en fármacos contra el cáncer en Cuba</a></li><li class="font85"><a href="/noticias/salud/157918-investigan-como-apagar-el-cancer-en-el-cuerpo"> * Investigan cómo apagar el cáncer en el cuerpo</a></li><li class="font85"><a href="/noticias/salud/157917-20mrz-dia-mundial-para-el-cuidado-de-la-salud-bucal"> * 20Mrz: Día Mundial para el cuidado de la salud bucal </a></li></ul><div class="section_v font80"><a href="/noticias/salud">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
								                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa calidoscopio"></i></span><a href="/interesantes/caleidoscopio">Caleidoscopio Cubano</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/interesantes/caleidoscopio/157916-cuevas-de-bellamar-un-regalo-de-la-naturaleza-(+-foto)">Cuevas de Bellamar: un regalo de la naturaleza (+ Foto) </a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/7210-cuevas-de-bellamar-internet.jpg" alt="Foto: Internet" /></div><p class="excerpt">Un gran salón como de palacios de cuentos de hadas, aguas mágicas y figuras que asustan o enternecen, según el antojo de la naturaleza, esperan al visitante ... <a class="more" href="/interesantes/caleidoscopio/157916-cuevas-de-bellamar-un-regalo-de-la-naturaleza-(+-foto)">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/interesantes/caleidoscopio/157898-la-superabuela-cubana-que-se-enamoro-de-gardel"> * La superabuela cubana que se enamoró de Gardel</a></li><li class="font85"><a href="/interesantes/caleidoscopio/157859-white-cien-anos"> * White, cien años</a></li><li class="font85"><a href="/interesantes/caleidoscopio/157858-periodismo-y-cultura"> * Periodismo y cultura</a></li></ul><div class="section_v font80"><a href="/interesantes/caleidoscopio">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
								                <div class="large-4 columns bordered">
                    <div class="section_block"><div class="section_h"><span><i class="fa misce"></i></span><a href="/interesantes/miscelanea">Miscelánea</a></div><div class="section_c"><div class="section_n large-12 columns"><h5 class="title_article"><a href="/interesantes/miscelanea/157912-hitler-como-“candidato”-en-la-web-del-tribunal-electoral-de-paraguay">Hitler como “candidato” en la web del Tribunal Electoral de Paraguay</a></h5><div class="image_article"><img src="/uploads/images/articulos/thumbnails/592-adolf-hitler.jpg" alt="Foto: Archivo" /></div><p class="excerpt">La Habana, 20 mar (RHC) Menuda sorpresa se llevaron los paraguayos que buscaron la lista de candidatos para las elecciones generales del 22 de abril en la pági... <a class="more" href="/interesantes/miscelanea/157912-hitler-como-“candidato”-en-la-web-del-tribunal-electoral-de-paraguay">Leer Más</a><span class="comment_q"><i class="fa fa-comment"></i>0</span></p></div></div><div class="section_l column"><ul><li class="font85"><a href="/interesantes/miscelanea/157872-descubren-secreto-escondido-en-dibujo-de-miguel-angel"> *  Descubren secreto escondido en dibujo de Miguel Ángel </a></li><li class="font85"><a href="/interesantes/miscelanea/157856-nace-bebe-en-vuelo-habana-bogota"> * Nace bebé en vuelo Habana-Bogotá</a></li><li class="font85"><a href="/interesantes/miscelanea/157850-china-construye-tunel-de-viento-mas-rapido-del-mundo-para-impulsar-desarrollo-de-avion-espacial"> * China construye túnel de viento más rápido del mundo para impulsar desarrollo de avión espacial</a></li></ul><div class="section_v font80"><a href="/interesantes/miscelanea">&lt;&lt;ver más&gt;&gt;</a></div></div></div>                </div>
				            </div>
        </div>
    </div>
    <div class="large-3 columns" id="modules">
        <div class="module centered">					<a href="http://www.radiohc.cu/assets/streaming.html" target="_blank" onclick="window.open(this.href,this.target,'width=510,height=360,top=150,left=300,toolbar=no,location=no,status=no,menubar=no,scrollbars=no,resizable=NO');return false;"><img src="/themes/v2.0/img/online_es.jpg" alt="" /></a></div>		<div class="section_block module"><div class="section_h large-text-center">Lo más Visto</div><div class="section_l"><ul><li class="font85"><a href="/especiales/exclusivas/157433-biografia-oficial-de-uhuru-muigai-kenyatta-presidente-de-kenya"> * Biografía oficial de Uhuru Muigai Kenyatta, presidente de Kenya</a></li><li class="font85"><a href="/especiales/comentarios/157431-trump-se-rodea-de-halcones"> * Trump se rodea de halcones</a></li><li class="font85"><a href="/especiales/exclusivas/157419-los-retos-del-periodismo-cultural"> * Los retos del periodismo cultural
</a></li><li class="font85"><a href="/especiales/exclusivas/157477-protesta-de-baragua-la-intransigencia-que-salvo-la-revolucion"> * Protesta de Baraguá, la intransigencia que salvó la Revolución</a></li><li class="font85"><a href="/especiales/comentarios/157430-juventud-cubana-+-elecciones-generales-=-futuro"> * Juventud cubana + elecciones generales = futuro</a></li></ul></div></div>        <div class="module audios"><audio id="meaudio" src="http://teveo.icrt.cu/m/p/tc/59/tc59vf.mp3" type="audio/mp3" controls="controls" preload="none"></audio><ul class="playlist"><li class="current" src="http://teveo.icrt.cu/m/p/tc/59/tc59vf.mp3"><span>Cancillería cubana afirma que relaciones Cuba-Estados Unidos experimentaron una regresión en 2017</span></li><li src="http://teveo.icrt.cu/m/p/he/m3/hem368.mp3"><span>Feria Internacional de las Telecomunicaciones abre sus puertas en La Habana </span></li><li src="http://teveo.icrt.cu/m/p/xh/g3/xhg39k.mp3"><span>Inaugurada en La Habana Feria Internacional Agroindustrial Alimentaria</span></li><li src="http://teveo.icrt.cu/m/p/xe/mk/xemkdz.mp3"><span>Maestros puertorriqueños rechazan privatización del sector educativo</span></li><li src="http://teveo.icrt.cu/m/p/vj/ak/vjak9q.mp3"><span>El Caribe, entre las regiones más afectadas por el cambio climático </span></li><li src="http://teveo.icrt.cu/m/p/99/5q/995qsd.mp3"><span>Putin, presidente de Rusia hasta 2024</span></li></ul></div>		<div class="module advertisement"><ul class="no-bullet"><li><a href="https://twitter.com/siempreconcuba"><img src="/uploads/images/advertisement/Tuitazo-Unidos-por-la-paz.png" alt="" /></a></li><li><a href="http://www.radiohc.cu/especiales/por-fidel"><img src="/uploads/images/advertisement/PorSiempreFidel.png" alt="" /></a></li><li><a href="http://www.radiohc.cu/especiales/quien-da-luz"><img src="/uploads/images/advertisement/José-Martí.png" alt="" /></a></li><li><a href="http://www.radiohc.cu/especiales/exclusivas/117514-aniversario-50-de-la-lucha-del-che-y-sus-companeros-en-bolivia"><img src="/uploads/images/advertisement/Che.jpg" alt="" /></a></li><li><a href="http://www.acs-aec.org/index.php?q=es/"><img src="/uploads/images/advertisement/AEC.jpg" alt="" /></a></li><li><a href="https://laradiodelsur.com.ve"><img src="/uploads/images/advertisement/La_Radio_del_Sur_logo.jpg" alt="" /></a></li><li><a href="http://www.pensandoamericas.com"><img src="/uploads/images/advertisement/pensando-americas.jpg" alt="" /></a></li><li><a href="http://www.cuba.cu/gobierno/reflexiones/reflexiones.html"><img src="/uploads/images/advertisement/refexcionesfidel.jpg" alt="" /></a></li><li><a href="http://www.cuba.cu/gobierno/rauldiscursos/index2.html"><img src="/uploads/images/advertisement/discursosraul.jpg" alt="" /></a></li><li><a href="http://mesaredonda.cubadebate.cu/"><img src="/uploads/images/advertisement/mesa-redonda.jpg" alt="" /></a></li><li><a href="http://razonesdecuba.cubadebate.cu/"><img src="/uploads/images/advertisement/razones-de-cuba.jpg" alt="" /></a></li><li><a href="http://www.cheguevara.com/"><img src="/uploads/images/advertisement/cheguevara.jpg" alt="" /></a></li><li><a href="http://www.cubavsbloqueo.cu/"><img src="/uploads/images/advertisement/cubavsbloqueo.jpg" alt="" /></a></li><li><a href="http://www.antiterroristas.cu/"><img src="/uploads/images/advertisement/antiterrorista.jpg" alt="" /></a></li><li><a href="http://www.radiohc.cu/correspondencia"><img src="/uploads/images/advertisement/correspondencia.png" alt="" /></a></li><li><a href="http://www.radiohc.cu/interesantes/concursos"><img src="/uploads/images/advertisement/concursos.png" alt="" /></a></li></ul></div>                <div class="visitcounter section_block module"><div class="section_h large-text-center">Contador de Visitas</div><h3 class="counter">4474727</h3><ul class="data"><li>Máximo: <span class="value-counter">19729</span></li><li>Ayer: <span class="value-counter">3933</span></li><li>Hoy: <span class="value-counter">1047</span></li><li>En Línea: <span class="value-counter">103</span></li><li>Total: <span class="value-counter">4474727</span></li></ul></div>        <div class="subscription section_block module"><div class="section_h large-text-center">Subscribirse</div><div class="section_l hidden" id="suscribed"><p>Gracias por subscribirse a Radio Habana Cuba</p></div><div class="section_l" id="suscribirse"><p>Para Comenzar a recibir noticias por correo ingrese su dirección en el recuadro siguiente</p><input type="text" value="" name="email" id="email" /><div class="controls"><input type="button" class="button" id="send_s" value="send"></div></div></div>            </div>
</div>
</div>
                <div id="footer" class="row footer">
            <div id="up" class="large-1 left columns" >
                <a href="#" ><i class="fa fa-arrow-circle-up fa-2x"></i><br/>
                    <span class="caption_up">subir</span>
                </a>
            </div>
            <div id="secciones" class="large-6 left colums">
                <div class="large-4 columns">
                    <h4 class="header_sections">Noticias</h4>
                    <ul class="menu_list no-bullet">
                                                                                    <li class="font80 text-right">
                                    <a href="/noticias/nacionales">Nacionales</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/internacionales">Internacionales</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/cultura">Cultura</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/salud">Salud</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/economy">Economía/Turismo</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/ciencias">Ciencias/Medio Ambiente</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/noticias/deportes">Deportes</a>
                                </li>
                                                                                                                                                                                </ul>
                </div>
                <div class="large-4 columns">
                    <h4 class="header_sections">Especiales</h4>
                    <ul class="menu_list no-bullet">
                                                                                    <li class="font80 text-right">
                                    <a href="/especiales/por-fidel">Por Fidel</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/especiales/quien-da-luz">Quien da Luz</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/especiales/elecciones-cuba">Elecciones Cuba</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/especiales/comentarios">Comentarios</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/especiales/exclusivas">Especiales</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/especiales/resumenes">Resumenes</a>
                                </li>
                                                                                                                                                                    <li class="font80 text-right">
                                    <a href="/especiales/cubausa">USA-Cuba</a>
                                </li>
                                                                                                                                                                    <li class="font80 text-right">
                                    <a href="/especiales/cumbre-americas">Cumbre de las Américas</a>
                                </li>
                                                                        </ul>
                </div>
                <div class="large-4 columns">
                    <h4 class="header_sections">Interesantes</h4>
                    <ul class="no-bullet">
                                                                                    <li class="font80 text-right">
                                    <a href="/interesantes/caleidoscopio">Caleidoscopio Cubano</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/miscelanea">Miscelánea</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/filatelia">El mundo de la Filatelia</a>
                                </li>
                                                                                                                                                                    <li class="font80 text-right">
                                    <a href="/galerias">Galería de Imágenes</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/concursos">Concursos</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/frecuencias">Frecuencia</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/programacion">Programación</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/correspondencia">Correspondencia</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/estaticas/quienes_somos">¿Quiénes Somos?</a>
                                </li>
                                                                                                                <li class="font80 text-right">
                                    <a href="/interesantes/links">Enlaces Web</a>
                                </li>
                                                                        </ul>
                </div>
            </div>
            <div id="credits" class="large-5 columns">
                <ul class="no-bullet">
                    <li class="text-right"><h5>Director General&nbsp;<small>Tania Hernández Castellanos</small></h5></li>
                    <li class="text-right"><h5>Editor Jefe&nbsp;<small>Pedro Otero Cabañas</small></h5></li>
                    <li class="text-right"><h5>Redactores-Editores&nbsp;<small><a href="/statics/contactenos#nivel2">redactores</a></small></h5></li>
					<li class="text-right"><h5>J'Dep Página Web&nbsp;<small>Maité González Martínez</small></h5></li>
                    <li class="text-right"><h5>Webmaster&nbsp;<small>Reinier Clemente López</small></h5></li>
					<li class="text-right"><h5>Webmaster&nbsp;<small>Adrian Valdés Serrano</small></h5></li>
                </ul>
                <ul class="no-bullet">
                    <li class="text-right"><h4><div class="responsive"></div><small>Dise&ntilde;o "responsive" usando <a href="http://foundation.zurb.com/"> Foundation</a></small></h4></li>
                    <li class="text-right"><h4><img src="/themes/v2.0/img/rhc.png"><small>&nbsp;Radio Habana Cuba &copy; 2015 V&nbsp;1.1.19</small></h4></li>

                </ul>
            </div>
        </div>
        <script>
            $(document).foundation();
        </script>
<!-- Piwik -->
<script>
  var _paq = _paq || [];
  _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
  _paq.push(["setCookieDomain", "*.www.radiohc.cu"]);
  _paq.push(["setDomains", ["*.www.radiohc.icrt.cu","*.radiohc.cu","*.rhc.cu","*.www.radiohc.cu","*.www.rhc.cu"]]);
  _paq.push(['trackPageView']);
  _paq.push(['enableLinkTracking']);
  (function() {
    var u="//piwik.rsa.icrt.cu/";
    _paq.push(['setTrackerUrl', u+'piwik.php']);
    _paq.push(['setSiteId', 12]);
    var d=document, g=d.createElement('script'), s=d.getElementsByTagName('script')[0];
    g.type='text/javascript'; g.async=true; g.defer=true; g.src=u+'piwik.js'; s.parentNode.insertBefore(g,s);
  })();
</script>
<noscript><p><img src="//piwik.rsa.icrt.cu/piwik.php?idsite=12" style="border:0;" alt="" /></p></noscript>
<!-- End Piwik Code -->
</div>
  </body>
</html>