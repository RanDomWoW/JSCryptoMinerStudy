#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="en">
  <head>
    <meta charset="utf-8">
    <meta http-equiv="X-UA-Compatible" content="IE=edge">
    <meta name="viewport" content="width=device-width, initial-scale=1">
    <title>Radio Cadena Agramonte, radio cubana, noticias de Camagüey, Cuba y el Mundo</title>
<meta name="description" content="Radio Cadena Agramonte, radio cubana, noticias de Camagüey, Cuba y el Mundo. Ofrece información del país y el resto del mundo en soportes tales como texto, imagen, audio y video." />
<meta name="keywords" content="radio cadena agramonte, radio cubana, noticias, camaguey, Cuba, el mundo, Fidel Castro Ruz, prensa camagueyana, noticias camagüey, noticias cuba, noticias del mundo, emisora cubana, emisora camaguey, nacionales, internacionales, podcast, audios, agramonte, ignacio agramonte y loynaz, el mayor, nicolas guillen, bloqueo contra cuba, cinco heroes cubanos,  antiterroristas cubanos, entretenimientos, san juan camagueyano, museo agramonte, museo digital, parque agramonte, amalia, foros, radio, ciencia, mundo deportivo, cubanos, reflexiones de Fidel, revolución, blogs, hit parade, 26 de julio" />
<link rel="canonical" href="http://www.cadenagramonte.cu/" />
<meta property="og:type" content="website" />
<meta property="og:title" content="Radio Cadena Agramonte, radio cubana, noticias de Camagüey, Cuba y el Mundo" />
<meta property="og:url" content="http://www.cadenagramonte.cu/" />
<meta property="og:site_name" content="Radio Cadena Agramonte" />
<meta property="og:locale" content="es_ES" />
<meta name="twitter:card" content="summary" />
<meta name="twitter:site" content="@cadenagramonte" />
<meta name="twitter:title" content="Radio Cadena Agramonte, radio cubana, noticias de Camagüey, Cuba y el Mundo" />
<meta name="twitter:description" content="Radio Cadena Agramonte, radio cubana, noticias de Camagüey, Cuba y el Mundo. Ofrece información del país y el resto del mundo en soportes tales como texto, imagen, audio y video." />
    
    <base href="http://www.cadenagramonte.cu/" />
    <script src="http://www.cadenagramonte.cu/js/jquery.min.js"></script>
    <script language="javascript" type="text/javascript" src="http://www.cadenagramonte.cu/js/swfobject.js" ></script>
    <script language="javascript" type="text/javascript" src="http://www.cadenagramonte.cu/js/libreriaAjax.js" ></script>
    
    <link rel="shortcut icon" href="http://www.cadenagramonte.cu/images/favicon.gif" />
    <meta name="google-site-verification" content="s4ci9dr0KXo63YfX-wTYlIPJQeeJoReMjVBlIvlCTGU" />

    <!-- Bootstrap core CSS -->
    <link href="http://www.cadenagramonte.cu/css/bootstrap.css" rel="stylesheet" />

    <!-- Custom styles for this template -->
    <link href="http://www.cadenagramonte.cu/css/starter-template.css" rel="stylesheet" />

    <!-- Just for debugging purposes. Don't actually copy this line! -->
    <!--[if lt IE 9]><script src="../../assets/js/ie8-responsive-file-warning.js"></script><![endif]-->

    <!-- HTML5 shim and Respond.js IE8 support of HTML5 elements and media queries -->
    <!--[if lt IE 9]>
      <script src="https://oss.maxcdn.com/libs/html5shiv/3.7.0/html5shiv.js"></script>
      <script src="https://oss.maxcdn.com/libs/respond.js/1.4.2/respond.min.js"></script>
    <![endif]-->
        
  </head>
<div id="fb-root"></div>
<script>(function(d, s, id) {
  var js, fjs = d.getElementsByTagName(s)[0];
  if (d.getElementById(id)) return;
  js = d.createElement(s); js.id = id;
  js.src = "//connect.facebook.net/es_ES/sdk.js#xfbml=1&version=v2.3&appId=392549639782";
  fjs.parentNode.insertBefore(js, fjs);
}(document, 'script', 'facebook-jssdk'));</script>


  <body>
 
    <div class="container starter-template">
  
  <ul class="nav nav-tabs small">
  <li role="presentation"><a href="phocadownload/cadena_agramonte.mp3" target="_blank"><span class="glyphicon glyphicon-bullhorn" aria-hidden="true"></span> Identificación</a></li>
  <li role="presentation"><a href="articulos/ver/13399:quienes-somos"><span class="glyphicon glyphicon-user" aria-hidden="true"></span> Quiénes somos</a></li>
  <li role="presentation"><a href="mailto:rcadigital@rcagramonte.icrt.cu"><span class="glyphicon glyphicon-envelope" aria-hidden="true"></span> Contáctenos</a></li>
  <li role="presentation"><a href="articulos/ver/365:faqs"><span class="glyphicon glyphicon-question-sign" aria-hidden="true"></span> Faq's</a></li>
  <li role="presentation" class="dropdown">
<a class="dropdown-toggle" data-toggle="dropdown" href="#" role="button" aria-haspopup="true" aria-expanded="false">
<span class="glyphicon glyphicon-link" aria-hidden="true"></span> Blogs<span class="caret"></span></a>
<ul class="dropdown-menu dropdown-menu-right">
<li><a href="https://cubaamiga.blogdiario.com/" target="_blank">CubaAmiga</a></li><li><a href="https://enelfondodelhombre.wordpress.com/" target="_blank">En el Fondo del Hombre</a></li><li><a href="https://mipuntodepenalty.wordpress.com/" target="_blank">Mi punto de penalty</a></li><li><a href="https://madrejovenytrabajadora.wordpress.com/" target="_blank">Madre, joven y trabajadora</a></li><li><a href="https://cronicasdesaludcuba.wordpress.com/" target="_blank">Crónicas de salud</a></li><li><a href="https://debuenatintapp.wordpress.com/" target="_blank">De buena tinta</a></li><li><a href="https://cantaresdepalmas.wordpress.com/" target="_blank">Cantares de palmas</a></li><li><a href="https://camaguebaxcuba.wordpress.com/" target="_blank">Camaguebaxcuba</a></li><li><a href="https://periodistascamagueyanos.wordpress.com/" target="_blank">Blogalidad Camagüey</a></li></ul>
</li>
  <li role="presentation" class="dropdown">
<a class="dropdown-toggle" data-toggle="dropdown" href="#" role="button" aria-haspopup="true" aria-expanded="false">
<span class="glyphicon glyphicon-link" aria-hidden="true"></span> Radio camagüeyana<span class="caret"></span></a>
<ul class="dropdown-menu dropdown-menu-right">
<li><a href="http://radiocamaguey.wordpress.com/" target="_blank">Camagüey</a></li><li><a href="http://www.radiocubitas.icrt.cu/" target="_blank">Cubitas</a></li><li><a href="http://www.radioflorida.icrt.cu/" target="_blank">Florida</a></li><li><a href="http://radioesmeraldadecuba.wordpress.com/" target="_blank">Esmeralda</a></li><li><a href="http://www.radioguaimaro.icrt.cu/" target="_blank">Guáimaro</a></li><li><a href="http://vozbayatabominas.wordpress.com/" target="_blank">La voz del Bayatabo</a></li><li><a href="http://www.radionuevitas.icrt.cu/" target="_blank">Nuevitas</a></li><li><a href="http://www.radiosantacruz.icrt.cu/" target="_blank">Santa Cruz del Sur</a></li><li><a href="http://radiovertientes.wordpress.com/" target="_blank">Vertientes</a></li></ul>
</li>
  
  <li role="presentation" class="dropdown pull-right">
    <a class="dropdown-toggle" data-toggle="dropdown" href="#" role="button" aria-haspopup="true" aria-expanded="false">
      <span class="glyphicon glyphicon-globe" aria-hidden="true"></span> Idiomas <span class="caret"></span>
    </a>
    <ul class="dropdown-menu">
      <li class="text-right"><a href="/english/" target="_blank">English <span class="glyphicon glyphicon-chevron-right" aria-hidden="true"></span></a></li>
      <li class="text-right"><a href="/frances/" target="_blank">Français <span class="glyphicon glyphicon-chevron-right" aria-hidden="true"></span></a></li>
    </ul>
  </li>
</ul>
  
   <div class="row">
   
      <div class="col-sm-8 col-md-8 col-lg-8">
      <a href="./"><img src="images/logo.jpg" class="img-responsive" alt="Radio Cadena Agramonte" /></a>
      </div>

      <div class="col-sm-4 col-md-4">
      
      <div class="row text-center hidden-xs"><br />
        <p class="text-center">
            <a href="http://www.cadenagramonte.cu/articulos/feed/" target="_blank"><img src="images/rss-icon.png" class="img-circle" alt="RSS" title="RSS"/></a>
            <a href="http://www.facebook.com/cadenagramonte" target="_blank"><img src="images/Facebook48.png" class="img-circle" alt="Facebook" title="Facebook" /></a>
            <a href="https://twitter.com/cadenagramonte" target="_blank"><img src="images/Twitter48.png" class="img-circle" alt="Twitter" title="Twitter" /></a>
            <a href="https://www.youtube.com/channel/UClQ6Nku0zw1JtLUZXhQhkOg" target="_blank"><img src="images/Youtube48.png" class="img-circle" alt="Youtube" title="Youtube" /></a>
            <a href="http://www.ivoox.com/escuchar-cadena-agramonte_nq_96918_1.html" target="_blank"><img src="images/ivoox.png" class="img-circle" alt="IVOOX" title="IVOOX" /></a>
            
            </p></div>
            <div class="row text-center"> Jueves, 22 de Marzo de 2018</div>
            <div class="row text-center hidden-xs">
                <p>
                <strong>Audio real:</strong><br />
                    <audio controls="controls"><source src="https://icecast.teveo.icrt.cu/j99xztkT" type="audio/mpeg"></source></audio>
                </p>
            </div>
     
      </div>
              
        </div>
    </div><!-- /.container -->
    
    <div class="container">
    <div class="row">
  <nav class="navbar navbar-default">
  <div class="container-fluid">
    <!-- /Brand and toggle get grouped for better mobile display -->
    <div class="navbar-header">
      <button type="button" class="navbar-toggle collapsed" data-toggle="collapse" data-target="#bs-example-navbar-collapse-1">
        <span class="sr-only">Menú</span>
        <span class="icon-bar"></span>
        <span class="icon-bar"></span>
        <span class="icon-bar"></span>
      </button>
      <a class="navbar-brand visible-xs" href="#">Menú</a>
    </div>
    <div class="collapse navbar-collapse" id="bs-example-navbar-collapse-1">
      <ul class="nav navbar-nav">
    <li class="active"><a href="http://www.cadenagramonte.cu/">Portada</a></li>
<li class="dropdown">
<a href='#' class="dropdown-toggle" data-toggle="dropdown" role="button" aria-expanded="false">Noticias <span class="caret"></span></a>
<ul class="dropdown-menu dropdown-menu-right" role="menu">
<li><a href="http://www.cadenagramonte.cu/articulos/categoria/1:camaguey">Camagüey</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/categoria/2:nacionales">Nacionales</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/categoria/3:internacionales">Internacionales</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/categoria/190:curiosas">Curiosas</a></li>
</ul>
</li>
<li><a href="http://www.cadenagramonte.cu/articulos/seccion/30:opinion">Opinión</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/seccion/2:ambito-cultural">Cultura</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/seccion/3:mundo-deportivo">Deportes</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/seccion/6:ciencia-y-entorno">Ciencia y Entorno</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/archivo">Archivo</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/seccion/41:coberturas">Coberturas</a></li>
<li class="dropdown">
<a href='#' class="dropdown-toggle" data-toggle="dropdown" role="button" aria-expanded="false">Multimedias <span class="caret"></span></a>
<ul class="dropdown-menu dropdown-menu-right" role="menu">
<li><a href="http://www.cadenagramonte.cu/aniversario-60/">Radio Cadena Agramonte: 60 años en el corazón del pueblo</a></li>
<li><a href="http://www.cadenagramonte.cu/phocadownload/multimedia-avellaneda-min.rar">La Avellaneda: grandeza y universalidad</a></li>
<li><a href="http://www.cadenagramonte.cu/phocadownload/multimedia-camaguey-500.rar">Por el orgullo de ser camagüeyanos</a></li>
<li><a href="http://www.cadenagramonte.cu/multimedia-alfabetizacion">Camagüey, Cuba: de la Campaña de Alfabetización al “Yo sí Puedo”</a></li>
<li><a href="http://www.cadenagramonte.cu/animaciones/por-siempre-fidel.swf">Fidel entre nosotros</a></li>
<li><a href="http://www.cadenagramonte.cu/museo">Museo Digital "Ignacio Agramonte"</a></li>
</ul>
</li>
    </ul>
    <form class="navbar-form navbar-right" role="search"  action="articulos/find/" method="post" target="_blank" enctype="multipart/form-data">
        <div class="form-group">
          <input type="text" class="form-control" name="busqueda" placeholder="Buscar" required />
        </div>
      </form>
    </div>
  </div><!-- /.container-fluid -->
</nav>
    </div>
  </div>
  
    <div id="contenedor" class="container" >
    <div class="row">
   <!--- Columna Izquierda --->
    <div class="row">
        <div class="col-sm-5 col-md-5 col-lg-5">
             <div class="jumbotron">
<h2><a href="http://www.cadenagramonte.cu/articulos/ver/77967:sociedad-civil-cubana-defendera-a-los-excluidos-en-cumbre-de-las-americas-declaracion-final">Sociedad civil cubana defenderá a los excluidos en Cumbre de las Américas (+ Declaración Final)</a></h2>
<a href="http://www.cadenagramonte.cu/articulos/ver/77967:sociedad-civil-cubana-defendera-a-los-excluidos-en-cumbre-de-las-americas-declaracion-final"><center><img class="img-thumbnail" alt="Sociedad civil cubana defenderá a los excluidos en Cumbre de las Américas (+ Declaración Final)" title = "Sociedad civil cubana defenderá a los excluidos en Cumbre de las Américas (+ Declaración Final)" src="http://www.prensa-latina.cu/images/2018/marzo/21/foro-civil3.jpg" /></center></a><p>
	La Habana, 22 mar.- La sociedad civil cubana ser&aacute; la voz de los pueblos excluidos de la VIII Cumbre de las Am&eacute;ricas en Per&uacute;, afirm&oacute; en esta capital el H&eacute;roe de la Rep&uacute;blica de Cuba Ram&oacute;n Laba&ntilde;ino Salazar.<h2><a href="http://www.cadenagramonte.cu/articulos/ver/77965:cuba-aboga-por-derecho-humano-de-primer-orden-en-foro-mundial"> Cuba     aboga por  derecho humano de primer orden en Foro Mundial</a></h2>
<a href="http://www.cadenagramonte.cu/articulos/ver/77965:cuba-aboga-por-derecho-humano-de-primer-orden-en-foro-mundial"><center><img class="img-thumbnail" alt=" Cuba     aboga por  derecho humano de primer orden en Foro Mundial" title = " Cuba     aboga por  derecho humano de primer orden en Foro Mundial" src="http://www.prensa-latina.cu/images/2018/marzo/22/Brasil-Foro-Agua.jpg" /></center></a><p>
	Brasilia, 22 mar.- Cuba reiter&oacute; ante el VIII Foro Mundial del Agua que el acceso a este recurso constituye un derecho humano cuyo disfrute por todos los ciudadanos es responsabilidad de los gobiernos, dijo un alto funcionario del sector.<hr /><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77960:diplomatico-cubano-desenmascara-en-peru-a-falsos-representantes-de-sociedad-civil">Diplomático cubano desenmascara en Perú a falsos representantes de sociedad civil</a>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77957:respalda-cuba-en-onu-lucha-mundial-contra-el-racismo">Respalda Cuba en ONU lucha mundial contra el racismo</a>
</ul><p class="text-right"><a href="http://www.cadenagramonte.cu/articulos/en_portada"><strong>Más ...</strong> <span class="glyphicon glyphicon-chevron-right"></span></a></p></div><div class="visible-lg"><h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/multimedia/audios">Audios recientes ... <span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><center><object type="application/x-shockwave-flash" data="http://www.cadenagramonte.cu/player/player_mp3_multi.swf" width="480" height="150">
<param name="movie" value="http://www.cadenagramonte.cu/player/player_mp3_multi.swf" />
<param name="wmode" value="transparent" />
<param name="FlashVars" value="configxml=http://www.cadenagramonte.cu/index.php/multimedia/showConfigAudio/480/150/10" /></object></center>
</div><h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/109:opinion">Opinión<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77953:desde-su-piel-los-versos-del-bien-te-esperan">Desde su piel, los versos del bien te esperan</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77953:desde-su-piel-los-versos-del-bien-te-esperan"><center><img class="img-thumbnail" alt="Desde su piel, los versos del bien te esperan" title = "Desde su piel, los versos del bien te esperan" src="http://www.cadenagramonte.cu/images/stories/poes%C3%ADa.jpg" /></center></a><p>
	<strong>Por Madeleine Sauti&eacute;/ Peri&oacute;dico Granma.</strong><br />
	<br />
	Los poetas esenciales, m&aacute;s que de costumbre, hoy le hablan al planeta.&nbsp; El D&iacute;a Mundial de la Poes&iacute;a, celebrado cada 21 de marzo, por razones asociadas al asomo equinoccial de la primavera, los convida a que levanten su voz, para que nadie en el planeta azul prescinda de la belleza.</div>
</div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/190:curiosas">Curiosas<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77946:anciana-britanica-descubre-su-origen-80-anos-despues-de-haber-sido-abandonada">Anciana británica descubre su origen 80 años después de haber sido abandonada</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77946:anciana-britanica-descubre-su-origen-80-anos-despues-de-haber-sido-abandonada"><center><img class="img-thumbnail" alt="Anciana británica descubre su origen 80 años después de haber sido abandonada" title = "Anciana británica descubre su origen 80 años después de haber sido abandonada" src="http://www.cadenagramonte.cu/images/stories/anciana_britanica.jpg" /></center></a><p>
	Hasta este marzo, Anthea Ring fue una de las ni&ntilde;as abandonadas m&aacute;s famosas de Reino Unido, desde que en 1937 la encontraron atada a una zarza y la Polic&iacute;a brit&aacute;nica comenz&oacute; una b&uacute;squeda nacional para encontrar a sus padres.</div>
</div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/242:fotorreportajes">Fotorreportajes<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/70003:una-plaza-para-el-arte">Una Plaza para el arte</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/70003:una-plaza-para-el-arte"><center><img class="img-thumbnail" alt="Una Plaza para el arte" title = "Una Plaza para el arte" src="http://www.cadenagramonte.cu/images/galerias/bedoya/plaza-del-carmen-bedoya-camaguey-cuba-rakapaga-cadena-agramonte-1.JPG" /></center></a><p>
	La Plaza del Carmen es sin dudas uno de los sitios m&aacute;s visitados en Camag&uuml;ey por lugare&ntilde;os y for&aacute;neos. La historia que la envuelve, la arquitectura y el conjunto escult&oacute;rico a tama&ntilde;o natural levantado por la artista Martha Jim&eacute;nez; hacen de este un lugar encantador, irresistible, obligado.</div>
</div>
   
            
        </div>
        
        
          <!--- Columna Central --->
         <div class="col-sm-4 col-md-4 col-lg-4">
             <h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/1:camaguey">Camagüey<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77961:embajador-japones-en-cuba-visita-comunidad-ganadera-del-camaguey-fotos">Embajador japonés en Cuba visita comunidad ganadera del Camagüey (+ Fotos)</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77961:embajador-japones-en-cuba-visita-comunidad-ganadera-del-camaguey-fotos"><center><img class="img-thumbnail" alt="Embajador japonés en Cuba visita comunidad ganadera del Camagüey (+ Fotos)" title = "Embajador japonés en Cuba visita comunidad ganadera del Camagüey (+ Fotos)" src="http://www.cadenagramonte.cu/images/stories/embajador_lumumba.JPG" /></center></a><p>
	<strong>Por Pedro Paneque Ruiz/ Radio Cadena Agramonte.</strong><br />
	<br />
	Sibanic&uacute;, Camag&uuml;ey, 22 mar.- Durante su segundo d&iacute;a de visita en tierras camag&uuml;eyanas, el embajador de Jap&oacute;n en Cuba, Excelent&iacute;simo Se&ntilde;or Masaru Watanabe, observ&oacute; este mi&eacute;rcoles el funcionamiento del acueducto de la comunidad agropecuaria Patricio Lumumba, en este oriental municipio, que con un proyecto de su pa&iacute;s recuper&oacute; valor de uso.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77962:accidente-de-transito-en-camaguey-deja-29-lesionados">Accidente de tránsito en Camagüey deja 29 lesionados</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77958:intercambian-profesionales-de-la-radio-en-camaguey-sobre-practicas-periodisticas">Intercambian profesionales de la Radio en Camagüey sobre prácticas periodísticas</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77944:abril-llegara-con-reconocimientos-a-centros-estudiantiles-camagueyanos">Abril llegará con reconocimientos a centros estudiantiles camagüeyanos</a></li>
</ul></div></div>
<p>
	<strong>Fidel en Camag&uuml;ey</strong></p>
<div class="carousel slide" data-ride="carousel" id="carousel-fidel-cmg">
	<center>
		<div class="carousel-inner" role="listbox">
			<div class="item active">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-1.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-2.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-3.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-4.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-5.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-7.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-8.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-9.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-10.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-11.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-12.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-13.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-14.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-15.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-16.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-17.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-18.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-19.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-20.jpg" /></div>
			<div class="item">
				<img alt="Fidel in Camagüey" src="http://www.cadenagramonte.cu/images/Fidel-en-camaguey/fidel-en-camaguey-21.jpg" /></div>
		</div>
		<a class="left carousel-control" data-slide="prev" href="#carousel-fidel-cmg" role="button"><span class="sr-only">Previous</span> </a> <a class="right carousel-control" data-slide="next" href="#carousel-fidel-cmg" role="button"> <span class="sr-only">Next</span> </a></center>
</div>
<p style="text-align: center;">
	&nbsp;</p>
<p>
	<a href="http://www.cadenagramonte.cu/aniversario-60/"><img alt="MULTIMEDIA: Radio Cadena Agramonte. 60 Años en el corazón del pueblo" src="http://www.cadenagramonte.cu/images/banners/banner-multimedia-aniversario-60-rca1.jpg" style="width: 100%;" title="MULTIMEDIA: Radio Cadena Agramonte. 60 Años en el corazón del pueblo" /></a></p>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/2:nacionales">Nacionales<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77970:foro-de-la-sociedad-civil-cubana-y-dialogo-hemisferico-en-peru">Foro de la sociedad civil cubana y Diálogo Hemisférico en Perú</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77970:foro-de-la-sociedad-civil-cubana-y-dialogo-hemisferico-en-peru"><center><img class="img-thumbnail" alt="Foro de la sociedad civil cubana y Diálogo Hemisférico en Perú" title = "Foro de la sociedad civil cubana y Diálogo Hemisférico en Perú" src="http://www.cadenagramonte.cu/images/stories/mesa_redonda6.jpg" /></center></a><p>
	La Habana, 22 mar.- La Mesa Redonda brindar&aacute; hoy un amplio resumen de los debates de la sociedad civil cubana en su foro previo a la Cumbre de las Am&eacute;ricas, el cual sesion&oacute; en la sede de la Central de Trabajadores de Cuba, y del desarrollo del Di&aacute;logo Hemisf&eacute;rico que transcurri&oacute; este mismo mi&eacute;rcoles en Per&uacute;.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77963:cuba-expresa-condolencias-por-deceso-de-dirigente-vietnamita">Cuba expresa condolencias por deceso de dirigente vietnamita</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77951:presentan-cartera-de-negocios-en-feria-agroindustrial-cubana">Presentan Cartera de Negocios en Feria Agroindustrial cubana</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77947:parte-final-de-programa-sobre-examenes-de-ingreso-a-la-educacion-superior">Parte final de programa sobre exámenes de ingreso a la Educación Superior</a></li>
</ul></div></div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/seccion/3:mundo-deportivo">Mundo Deportivo<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77968:camaguey-y-esmeralda-cerca-de-discutir-el-titulo-provincial-de-beisbol">Camagüey y Esmeralda cerca de discutir el título provincial de Béisbol</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77968:camaguey-y-esmeralda-cerca-de-discutir-el-titulo-provincial-de-beisbol"><center><img class="img-thumbnail" alt="Camagüey y Esmeralda cerca de discutir el título provincial de Béisbol" title = "Camagüey y Esmeralda cerca de discutir el título provincial de Béisbol" src="http://www.cadenagramonte.cu/images/stories/beisbol4.jpg" /></center></a><p>
	Camag&uuml;ey, 22 mar.- Los equipos de Esmeralda y Camag&uuml;ey se colocaron a un triunfo de acceder a la disputa de la corona de la 42&ordf; Serie Provincial de B&eacute;isbol, de primera categor&iacute;a.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77956:zonal-central-del-boxeo-cubano-en-su-segunda-fecha-en-camaguey">Zonal central del Boxeo cubano en su segunda fecha en Camagüey</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77939:camaguey-vs-cienfuegos-en-lid-clasificatoria-del-futbol-cubano">Camagüey vs. Cienfuegos en lid clasificatoria del Fútbol cubano</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77936:camaguey-lidera-clasificacion-de-la-liga-cubana-de-baloncesto-m">Camagüey lidera clasificación de la Liga cubana de Baloncesto (m)</a></li>
</ul></div></div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/3:internacionales">Internacionales<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77964:rechaza-constituyente-venezolana-sancion-estadounidense-contra-el-petro">Rechaza Constituyente venezolana sanción estadounidense contra el Petro</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77964:rechaza-constituyente-venezolana-sancion-estadounidense-contra-el-petro"><center><img class="img-thumbnail" alt="Rechaza Constituyente venezolana sanción estadounidense contra el Petro" title = "Rechaza Constituyente venezolana sanción estadounidense contra el Petro" src="http://media.cubadebate.cu/wp-content/uploads/2018/02/petro-1-768x463.jpg  " /></center></a><p>
	Caracas, 22 mar.- La Asamblea Nacional Constituyente (ANC) de Venezuela aprob&oacute; un acuerdo para repudiar las sanciones implementadas por el Gobierno de Estados Unidos contra la criptomoneda el Petro.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77959:renuncia-pedro-pablo-kuczysnki-a-la-presidencia-de-peru">Renuncia Pedro Pablo Kuczysnki a la Presidencia de Perú</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77950:parlamento-catalan-podria-elegir-nuevo-presidente-regional">Parlamento catalán podría elegir nuevo presidente regional</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77942:convoca-onu-a-promover-el-respeto-a-la-diversidad-y-la-tolerancia">Convoca ONU a promover el respeto a la diversidad y la tolerancia</a></li>
</ul></div></div>
   
         </div>
        
         <!--- Columna Derecha --->
         <div class="col-sm-3 col-md-3 col-lg-3">
            <h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/162:elecciones-en-cuba">Elecciones en Cuba<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/51257:cuba-en-elecciones-preguntas-y-respuestas">Cuba en elecciones. Preguntas y respuestas</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/51257:cuba-en-elecciones-preguntas-y-respuestas"><center><img class="img-thumbnail" alt="Cuba en elecciones. Preguntas y respuestas" title = "Cuba en elecciones. Preguntas y respuestas" src="http://www.cadenagramonte.cu/images/stories/general-elections-2012-logo1.jpg" /></center></a><p>
	En Cuba, &iquest;qui&eacute;nes tienen derecho a participar como electores; cu&aacute;les son los requisitos que deben reunirse para ejercer el voto; es el voto obligatorio? Sobre las caracter&iacute;sticas del proceso eleccionario en Cuba. Derechos y requisitos te propone conocer Cadena Agramonte.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77768:los-camagueyanos-respaldaron-el-voto-unido-por-cuba-y-el-socialismo">Los camagüeyanos respaldaron el voto unido por Cuba y el Socialismo</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77759:nicaragua-felicita-a-cuba-por-exitosa-jornada-electoral">Nicaragua felicita a Cuba por exitosa jornada electoral</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77765:electos-en-cuba-diputados-al-parlamento-y-delegados-provinciales">Electos en Cuba diputados al Parlamento y delegados provinciales</a></li>
</ul></div></div>
<p>
	<a href="http://www.cadenagramonte.cu/articulos/categoria/232:por-siempre-fidel"><img alt="Hasta siempre Comandante" src="http://www.cadenagramonte.cu/images/banners/banner-hasta-siempre-comandante.jpg" style="width: 100%;" /></a></p>
<p>
	<a href="http://www.cadenagramonte.cu/pdf/revista-camaguey-online/1-febrero-2017.pdf" target="_blank"><img alt="Revista Camagüey Online" src="http://www.cadenagramonte.cu/images/banners/revista-camaguey-online.png" style="width: 100%;" /></a></p>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/seccion/2:ambito-cultural">Ámbito Cultural<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77971:ser-camagueyanos-todos-los-dias-del-ano">Ser camagüeyanos todos los días del año</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77971:ser-camagueyanos-todos-los-dias-del-ano"><center><img class="img-thumbnail" alt="Ser camagüeyanos todos los días del año" title = "Ser camagüeyanos todos los días del año" src="http://www.cadenagramonte.cu/images/stories/fiesta-tinajon-17-8.jpg" /></center></a><p>
	<strong>Por Diosmel Galano Oliver/ Radio Camag&uuml;ey.</strong><br />
	<br />
	&iquest;Qu&eacute; nos identifica como camag&uuml;eyanos? &iquest;Cu&aacute;les son esos rasgos que nos unen al resto de las regiones de esta Isla caribe&ntilde;a y a la vez nos hacen &uacute;nicos?</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77955:recibe-hoy-camaguey-la-fiesta-del-tinajon">Recibe hoy Camagüey la Fiesta del Tinajón  </a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77945:festival-cubano-boleros-de-oro-reverenciara-a-las-feminas">Festival cubano Boleros de Oro reverenciará a las féminas  </a></li>
</ul></div></div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/98:Programación">Programación Radial<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/64474:musica">Música</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/60566:novela-te-quedaras">Novela - Te quedarás</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/58857:novela-la-otra-dolores">Novela - La otra Dolores</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/58856:mundo-deportivo">Mundo Deportivo</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/58855:novela-amores-y-destinos">Novela - Amores y destinos</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/58854:novela-dona-carmelina">Novela - Doña Carmelina</a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/57862:paginas-inmortales">Páginas inmortales</a></li>
</ul></div></div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/96:hit-parade">Hit Parade<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/75426:hit-parade-del-03-11-2017">Hit Parade del 03-11-2017</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/75426:hit-parade-del-03-11-2017"><center><img class="img-thumbnail" alt="Hit Parade del 03-11-2017" title = "Hit Parade del 03-11-2017" src="http://www.cadenagramonte.cu/images/stories/hit-parade-musica-cubana-cuba-rakapaga-camaguey-exitos-musicales-radio-cadena-agramonte.jpg" /></center></a><p>
	Radio Cadena Agramonte les presenta los &eacute;xitos musicales del momento, en Cuba y el mundo.</div>
</div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/seccion/6:ciencia-y-entorno">Ciencia y Entorno<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77966:aplican-sistema-informatico-a-maquinas-de-riego-en-provincias-cubanas">Aplican sistema informático a máquinas de riego en provincias cubanas</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77966:aplican-sistema-informatico-a-maquinas-de-riego-en-provincias-cubanas"><center><img class="img-thumbnail" alt="Aplican sistema informático a máquinas de riego en provincias cubanas" title = "Aplican sistema informático a máquinas de riego en provincias cubanas" src="http://www.cadenagramonte.cu/images/stories/pivote02.png" /></center></a><p>
	La Habana, 22 mar.- La agricultura cubana trabaja en la automatizaci&oacute;n de la irrigaci&oacute;n en la base productiva, en busca de m&aacute;s eficiencia y ahorro en esta atenci&oacute;n, dijo aqu&iacute; una fuente del sector.</div>
<div class="col-sm-12"><ul>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77952:proteger-los-bosques-para-salvar-la-vida">Proteger los bosques para salvar la vida </a></li>
<li><a href="http://www.cadenagramonte.cu/articulos/ver/77949:por-el-presente-y-futuro-celebra-fao-dia-internacional-de-los-bosques">Por el presente y futuro, celebra FAO Día Internacional de los Bosques</a></li>
</ul></div></div>
<h4><span class="label label-default"><a href="http://www.cadenagramonte.cu/articulos/categoria/111:el-tiempo">El tiempo<span class="glyphicon glyphicon-chevron-right"></span></a></span></h4><div class="row">
<div class="col-sm-12">
<h4><a href="http://www.cadenagramonte.cu/articulos/ver/77969:pronostican-lluvias-en-region-oriental-cubana">Pronostican lluvias en región oriental cubana</a></h4><a href="http://www.cadenagramonte.cu/articulos/ver/77969:pronostican-lluvias-en-region-oriental-cubana"><center><img class="img-thumbnail" alt="Pronostican lluvias en región oriental cubana" title = "Pronostican lluvias en región oriental cubana" src="http://www.cadenagramonte.cu/images/stories/lluvias.gif" /></center></a><p>
	La Habana, 22 mar.- Hoy estar&aacute; parcialmente nublado en la regi&oacute;n oriental y se nublar&aacute; en zonas de su costa norte, con algunos chubascos y lluvias, que se extender&aacute;n durante la tarde a localidades del interior y sur de dicha regi&oacute;n. En el resto del pa&iacute;s amanecer&aacute; con poca nubosidad y desde el final de la ma&ntilde;ana estar&aacute; parcialmente nublado, con escasas lluvias.</div>
</div>
        </div>
      
        </div>
       
        <div class="row">
        <div class="col-sm-3 col-md-3 col-lg-3"></div>
        <div class="col-sm-3 col-md-3 col-lg-3"></div>
        <div class="col-sm-3 col-md-3 col-lg-3"></div>
        <div class="col-sm-3 col-md-3 col-lg-3"></div> 
        </div>    </div>
    </div>
        
   <!--- /.Columnas de otros sitios -->
    <div class="container">
    <hr />
    <div class="row">
        <div class="col-sm-4  col-md-4 col-lg-4">
         <div class="panel panel-default">
<div class="panel-heading"><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span><strong> Audios recientes ...</strong></div>
<div class="panel-body">
<ul class="list-unstyled">
<li><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1281:mundo-deportivo-20-marzo-2018">Mundo Deportivo 20-Marzo-2018</a></li>
<li><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1280:mundo-deportivo-19-marzo-2018">Mundo Deportivo 19-Marzo-2018</a></li>
<li><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1279:mundo-deportivo-16-marzo-2018">Mundo Deportivo 16-Marzo-2018</a></li>
<li><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1278:mundo-deportivo-15-marzo-2018">Mundo Deportivo 15-Marzo-2018</a></li>
<li><span class="glyphicon glyphicon-volume-up" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1277:mundo-deportivo-14-marzo-2018">Mundo Deportivo 14-Marzo-2018</a></li>
</ul>
<p class="text-right"><a href="http://www.cadenagramonte.cu/multimedia/audios"><strong>Más ...</strong> <span class="glyphicon glyphicon-chevron-right"></span></a></p></div>
</div>
        </div>
        
        <div class="col-sm-4 col-md-4 col-lg-4">
        <div class="panel panel-default">
<div class="panel-heading"><span class="glyphicon glyphicon-film" aria-hidden="true"></span><strong> Videos recientes ...</strong></div>
<div class="panel-body">
<ul class="list-unstyled">
<li><span class="glyphicon glyphicon-film" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1163:fidel-entre-nosotros">Fidel entre nosotros</a></li>
<li><span class="glyphicon glyphicon-film" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/multimedia/ver/1142:intervencion-de-bruno-rodriguez-en-la-asamblea-general-de-las-naciones-unidas"> Intervención de Bruno Rodríguez en la Asamblea General de las Naciones Unidas</a></li>
</ul>
<p class="text-right"><a href="http://www.cadenagramonte.cu/multimedia/videos"><strong>Más ...</strong> <span class="glyphicon glyphicon-chevron-right"></span></a></p></div>
</div>
<h4>Hoy en la historia</h4>
<ul class="list-unstyled">
<li><span class="glyphicon glyphicon-calendar" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/efemerides/ver/-muere-johann-wolfgang-goethe-alemania">1832 -  Muere Johann Wolfgang Goethe en Alemania</a></li>
<li><span class="glyphicon glyphicon-calendar" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/efemerides/ver/ejecutado-ramon-pinto">1855 -  Ejecutado Ramón Pintó</a></li>
<li><span class="glyphicon glyphicon-calendar" aria-hidden="true"></span> <a href="http://www.cadenagramonte.cu/efemerides/ver/antonio-maceo-ascendido-coronel">1872 -  Antonio Maceo ascendido a coronel</a></li>
</ul>
<p class="text-right"><a href="http://www.cadenagramonte.cu/efemerides"><strong>Más ...</strong> <span class="glyphicon glyphicon-chevron-right"></span></a></p>        </div>
        
    
        <div class="col-sm-2 col-md-2 col-lg-2">
        <h4>Radio camagüeyana</h4>
<ul class="list-unstyled">
<li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://radiocamaguey.wordpress.com/" target="_blank">Camagüey</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://www.radiocubitas.icrt.cu/" target="_blank">Cubitas</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://www.radioflorida.icrt.cu/" target="_blank">Florida</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://radioesmeraldadecuba.wordpress.com/" target="_blank">Esmeralda</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://www.radioguaimaro.icrt.cu/" target="_blank">Guáimaro</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://vozbayatabominas.wordpress.com/" target="_blank">La voz del Bayatabo</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://www.radionuevitas.icrt.cu/" target="_blank">Nuevitas</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://www.radiosantacruz.icrt.cu/" target="_blank">Santa Cruz del Sur</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="http://radiovertientes.wordpress.com/" target="_blank">Vertientes</a></li></ul>        </div>
        
        
        <div class="col-sm-2 col-md-2 col-lg-2"><h4>Blogs</h4>
<ul class="list-unstyled">
<li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://cubaamiga.blogdiario.com/" target="_blank">CubaAmiga</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://enelfondodelhombre.wordpress.com/" target="_blank">En el Fondo del Hombre</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://mipuntodepenalty.wordpress.com/" target="_blank">Mi punto de penalty</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://madrejovenytrabajadora.wordpress.com/" target="_blank">Madre, joven y trabajadora</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://cronicasdesaludcuba.wordpress.com/" target="_blank">Crónicas de salud</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://debuenatintapp.wordpress.com/" target="_blank">De buena tinta</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://cantaresdepalmas.wordpress.com/" target="_blank">Cantares de palmas</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://camaguebaxcuba.wordpress.com/" target="_blank">Camaguebaxcuba</a></li><li><span class="glyphicon glyphicon-link" aria-hidden="true"></span> <a href="https://periodistascamagueyanos.wordpress.com/" target="_blank">Blogalidad Camagüey</a></li></ul></div>
        
        
    </div>
    </div>
    
    <!---PIE DE PAGINA -->
    <div class="container">
    <div class="row">
    <hr />
        <div class="header text-center">
        <h4><small>Radio Cadena Agramonte | Noticias de Camagüey, Cuba y el Mundo | © 2015, Camagüey, Cuba.</small></h4>


        </div>
    </div>
    </div>

    <!-- Bootstrap core JavaScript
    ================================================== -->
    <!-- Placed at the end of the document so the pages load faster -->
    
    <script src="http://www.cadenagramonte.cu/js/bootstrap.min.js"></script>
       <!-- google analytic -->
    <script>
  (function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
  (i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
  m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
  })(window,document,'script','https://www.google-analytics.com/analytics.js','ga');

  ga('create', 'UA-90679536-1', 'auto');
  ga('send', 'pageview');

</script>
    
    <!-- Piwik -->
<script type="text/javascript">
  var _paq = _paq || [];
  _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
  _paq.push(["setCookieDomain", "*.www.cadenagramonte.icrt.cu"]);
  _paq.push(['trackPageView']);
  _paq.push(['enableLinkTracking']);
  (function() {
    var u="//piwik.rsa.icrt.cu/";
    _paq.push(['setTrackerUrl', u+'piwik.php']);
    _paq.push(['setSiteId', 31]);
    var d=document, g=d.createElement('script'), s=d.getElementsByTagName('script')[0];
    g.type='text/javascript'; g.async=true; g.defer=true; g.src=u+'piwik.js'; s.parentNode.insertBefore(g,s);
  })();
</script>
  </body>
</html>