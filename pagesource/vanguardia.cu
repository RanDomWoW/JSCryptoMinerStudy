#include "hip/hip_runtime.h"

<!DOCTYPE html>
<html xmlns="http://www.w3.org/1999/xhtml" xml:lang="es-es" lang="es-es" dir="ltr">


<head>
  <base href="http://www.vanguardia.cu/" />
	<meta http-equiv="content-type" content="text/html; charset=utf-8" />
	<meta name="keywords" content="Vanguardia, villa clara, santa clara, cuba, Che, melaito, humor, opinión, béisbol" />
	<meta name="description" content="Vanguardia, periódico de la provincia de Villa Clara, en el centro de Cuba. Publica noticias e informaciones de interés general sobre Villa Clara, Cuba y el mundo. Desde Santa Clara, la Ciudad del Che, ofrece materiales sobre la vida, obra y vigencia de Ernesto Che Guevara. Cuenta, además, con foros de discusión, galerías de fotos, vídeos y humor gráfico. " />
	<meta name="generator" content="Joomla! - Open Source Content Management" />
	<title>Portada - Vanguardia</title>
	<link href="/?format=feed&amp;type=rss" rel="alternate" type="application/rss+xml" title="RSS 2.0" />
	<link href="/?format=feed&amp;type=atom" rel="alternate" type="application/atom+xml" title="Atom 1.0" />
	<link href="/templates/vanguardia/favicon.ico" rel="shortcut icon" type="image/vnd.microsoft.icon" />
	<link href="/media/vanguardia/plg_content_articlerelated/css/plg_content_articlerelated.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/plg_content_mjformcomment/css/plg_mjformcomment.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/tmpl_vanguardia_vanguardia/css/jui/bootstrap.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/tmpl_vanguardia_vanguardia/css/font-awesome-4.6.3/css/font-awesome.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/tmpl_vanguardia_vanguardia/css/theme/vanguardia/vanguardia.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/mod_vanguardia_gallery/css/mod_vanguardia_gallery.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/mod_vanguardia_news/css/mod_vanguardia_news.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/mod_vanguardia_billboard/css/mod_vanguardia_billboard.min.css" rel="stylesheet" type="text/css" />
	<link href="/media/com_finder/css/finder.css" rel="stylesheet" type="text/css" />
	<link href="/media/vanguardia/mod_vanguardia_weather/css/mod_vanguardia_weather.min.css" rel="stylesheet" type="text/css" />
	<script src="/media/jui/js/jquery.min.js?afda0319961de4c36088cd368de75f42" type="text/javascript"></script>
	<script src="/media/jui/js/jquery-noconflict.js?afda0319961de4c36088cd368de75f42" type="text/javascript"></script>
	<script src="/media/jui/js/jquery-migrate.min.js?afda0319961de4c36088cd368de75f42" type="text/javascript"></script>
	<script src="/media/vanguardia/plg_content_articlerelated/js/plg_content_articlerelated.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/plg_content_mjformcomment/js/plg_mjformcomment.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/tmpl_vanguardia_vanguardia/js/theme/vanguardia/vanguardia.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/tmpl_vanguardia_vanguardia/js/jui/bootstrap.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/tmpl_vanguardia_vanguardia/js/extensions/jquery.lazyload.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/tmpl_vanguardia_vanguardia/js/theme/template.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/mod_vanguardia_gallery/js/mod_vanguardia_gallery.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/mod_vanguardia_news/js/mod_vanguardia_news.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/mod_vanguardia_billboard/js/mod_vanguardia_billboard.min.js" type="text/javascript"></script>
	<script src="/media/jui/js/jquery.autocomplete.min.js" type="text/javascript"></script>
	<script src="/media/vanguardia/mod_vanguardia_weather/js/mod_vanguardia_weather.min.js" type="text/javascript"></script>
	<script type="text/javascript">

jQuery(document).ready(function() {
	var value, searchword = jQuery('.wsearchword');

		// Set the input value if not already set.
		if (!searchword.val())
		{
			searchword.val('Buscar...');
		}

		// Get the current value.
		value = searchword.val();

		// If the current value equals the default value, clear it.
		searchword.on('focus', function()
		{	var el = jQuery(this);
			if (el.val() === 'Buscar...')
			{
				el.val('');
			}
		});

		// If the current value is empty, set the previous value.
		searchword.on('blur', function()
		{	var el = jQuery(this);
			if (!el.val())
			{
				el.val(value);
			}
		});

		jQuery('#mod-finder-searchform').on('submit', function(e){
			e.stopPropagation();
			var advanced = jQuery('#mod-finder-advanced');
			// Disable select boxes with no value selected.
			if ( advanced.length)
			{
				advanced.find('select').each(function(index, el) {
					var el = jQuery(el);
					if(!el.val()){
						el.attr('disabled', 'disabled');
					}
				});
			}
		});
	var suggest = jQuery('.wsearchword').autocomplete({
		serviceUrl: '/component/finder/?task=suggestions.suggest&format=json&tmpl=component',
		paramName: 'q',
		minChars: 1,
		maxHeight: 400,
		width: 300,
		zIndex: 9999,
		deferRequestBy: 500
	});});
	</script>

  <meta name="viewport" content="width=device-width, initial-scale=1">
  <meta name="google-site-verification" content="ujqT6v2yojsLJIMyUimTf8fIR0LyWUyIIJo-X1twlF0" />
</head>

<body class="site_global com_content view-featured no-layout no-task itemid-107">
  
<header>
  <nav class="navbar navbar-top">
    <div class="container">
      <div class="navbar-content">
        <p class="navbar-text navbar-left page-update"><time>Última actualización: Domingo, 18 Marzo 2018, 01:12</time></p>
                  <ul class="nav navbar-nav ">
<li class="item-553"><a href="/sitemap" ><span>Mapa del sitio</span></a></li><li class="item-551"><a href="/galerias" ><span>Galerías </span></a></li><li class="item-534"><a href="/ediciones-impresas" ><span>Ediciones Impresas</span></a></li><li class="item-548"><a href="/quienes-somos" ><span>Quiénes somos</span></a></li><li class="item-549"><a href="/contactenos" ><span>Contáctenos</span></a></li></ul>

              </div>
    </div>
  </nav>

  <div class="container mainLogo">
    <div class="row">
      <div class="col-md-4 logo">
        <a href="/"><img src="http://www.vanguardia.cu/media/vanguardia/tmpl_vanguardia_vanguardia/img/logos/default.svg" alt="Vanguardia" /></a>
      </div>

      <div class="col-md-8 socialPage">
        <a href="/?format=feed&amp;type=rss" class="rss pull-right">
          <i class="fa fa-rss"></i>
        </a>
        <a target="_blank" href="https://www.facebook.com/VanguardiaCuba" class="facebook pull-right">
          <i class="fa fa-facebook-official fa-2x"></i>
        </a>
        <a target="_blank" href="https://twitter.com/VanguardiaCuba" class="twitter pull-right">
          <i class="fa fa-twitter fa-2x"></i>
        </a>
                  <div class="mod-vanguardia-weather predeterminada" id="mod-vanguardia-weather-269">
	
<div class="basic">
  <div class="pull-right temperature">
    <div class="min">17ºC</div>
    <div class="max">24ºC</div>
  </div>

  <img class="pull-right partlycloudy" src="/media/vanguardia/mod_vanguardia_weather/img/partlycloudy.svg" alt="Estado del tiempo en Villa Clara" class="pull-right" />
</div>
</div>

              </div>
    </div>
  </div>

      
<div class="affix-top animated" data-spy="affix" data-offset-top="60">
	<nav class="navbar navbar-vanguardia">
		<div class="container">
			<div class="navbar-header">
				<a class="navbar-brand" href="/">Vanguardia</a>
	    </div>

			<div id="navbar" class="collapse navbar-collapse">
					    		<ul class="nav navbar-nav ">
<li class="item-107 active active"><a href="/" ><span>Portada</span></a></li><li class="item-108"><a href="/villa-clara" ><span>Villa Clara</span></a></li><li class="item-109"><a href="/de-cuba" ><span>De Cuba</span></a></li><li class="item-110"><a href="/del-mundo" ><span>Del Mundo</span></a></li><li class="item-111"><a href="/cultura" ><span>Cultura</span></a></li><li class="item-112 dropdown"><a href="/deporte" ><span>Deporte</span></a></li><li class="item-209"><a href="/opinion-de-periodistas" ><span>Opinión</span></a></li></ul>

									      	
<form id="mod-finder-searchform" action="/busqueda-vanguardia" method="get" class="wsearch navbar-form navbar-right" role="search">
	<div class="form-group has-feedback ">
		<input type="text" name="q" id="mod-finder-searchword" class="wsearchword form-control" value="" /><span class="glyphicon glyphicon-search form-control-feedback" aria-hidden="true"></span>
			</div>
</form>

					    </div><!-- /.nav-collapse -->
		</div>
	</nav><!-- /.navbar -->
</div>
  </header>

  

<div class="container body">
  	<div class="row">
		<main>
<div class="page-featured " itemscope itemtype="http://schema.org/Blog">
      

  <section class="col-md-8 items-leading">
          <article class="leading-0"
        itemprop="blogPost" itemscope itemtype="http://schema.org/BlogPosting">
        
<table class="table">
  <tr>
    <td class="image" rowspan="4">
      
	<figure class="item-image">
							<a href="/villa-clara/10995-gente-agradecida-en-aguada-de-moya">
					<img class="img-responsive lazy"  data-original="http://www.vanguardia.cu/images/materiales/sociedad/2018/03-17-aguada-de-moya-poblado-entrega-viviendas-ivz/vecinos-reconocimiento.jpg" alt="Vecinos de Aguada de Moya, reconocidos por su labor en recuperación de la comunidad."/>
				</a>
				</figure>
    </td>
  </tr>

  <tr>
    <td class="category">
      
	<div class="item-category">
              <a href="/villa-clara"><i class="fa fa-angle-double-right"></i> Villa Clara</a>
      </div>
      
	<div class="item-title">
		<h2 itemprop="name">
							<a href="/villa-clara/10995-gente-agradecida-en-aguada-de-moya" itemprop="url">
					Gente agradecida en Aguada de Moya				</a>
					</h2>
	</div>
    </td>
  </tr>

  <tr>
    <td class="text">
      <p>En el Consejo Popular camajuanense concluyó la recuperación de 88 viviendas arrasadas por el huracán Irma, a cargo de brigadas estatales de varios sectores sociales.</p>
    </td>
  </tr>
</table>
      </article>
            </section>
          


<section class="col-md-4 items-intro">
      <article class="intro-0"
      itemprop="blogPost" itemscope itemtype="http://schema.org/BlogPosting">
      
<div class="panel panel-default">
 	<div class="panel-body">
    
    
	<div class="item-category">
              <a href="/villa-clara"><i class="fa fa-angle-double-right"></i> Villa Clara</a>
      </div>
    
	<div class="item-title">
		<h2 itemprop="name">
							<a href="/villa-clara/10996-cuatro-fallecidos-en-lamentable-accidente-del-transito-en-villa-clara" itemprop="url">
					Cuatro fallecidos en lamentable accidente del tránsito en Villa Clara				</a>
					</h2>
	</div>
    
	<figure class="item-image">
							<a href="/villa-clara/10996-cuatro-fallecidos-en-lamentable-accidente-del-transito-en-villa-clara">
					<img class="img-responsive lazy"  data-original="http://www.vanguardia.cu/images/materiales/defensa/minint/generales/logo-minint-colores-750x470.jpg" alt="Logo del Minint."/>
				</a>
				</figure>
  </div>
</div>
    </article>
      </section>
      </div>
</main>
	</div>
  <div class="row">
		<div class="col-md-8 flujo">
      <!-- Materiales complementarios -->
              <div class="mod-vanguardia-news complementaria " id="mod-vanguardia-news-250">
  
<div class="row basic">
      <div class="col-md-4 ciclo1">
              <div class="category" itemprop="genre">
          <i class="fa fa-angle-double-right"></i> <a href="/deporte">Deporte</a>        </div>
      
              <h3 class="title" itemprop="name">
          <a href="/deporte/10994-camajuani-y-remedios-por-el-titulo-provincial">Camajuaní y Remedios por el título provincial</a>
        </h3>
      
      
              <figure class="images">
          <a href="/deporte/10994-camajuani-y-remedios-por-el-titulo-provincial">
            <img class="img-responsive lazy"
                              data-original="images/materiales/deporte/beisbol/2017/05-05-yanni-delgado-recuperacion/yanni-diaz-portada.jpg" alt="Yanny Delgado, pitcher de Camajuaní." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <div class="introtext"><p>Cuatro equipos terminaron abrazados en la cima de la fase clasificatoria de la Provincial de Béisbol y hubo que acudir al TQB para decidir los ...</div>
          </div>
      <div class="col-md-4 ciclo2">
              <div class="category" itemprop="genre">
          <i class="fa fa-angle-double-right"></i> <a href="/villa-clara">Villa Clara</a>        </div>
      
              <h3 class="title" itemprop="name">
          <a href="/villa-clara/10991-reconocen-avances-en-produccion-de-piezas-para-el-ferrocarril">Reconocen avances en producción de piezas para el Ferrocarril</a>
        </h3>
      
      
              <figure class="images">
          <a href="/villa-clara/10991-reconocen-avances-en-produccion-de-piezas-para-el-ferrocarril">
            <img class="img-responsive lazy"
                              data-original="images/materiales/industria/sideromecanica/2018/03-16-ramiro-valdes-visita-planta-mecanica-fpc/ramiro-valdes-750x470.jpg" alt="Ramiro Valdés aprecia producciones para el ferrocarril en Planta Mecánica de Villa Clara." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <div class="introtext"><p>El Comandante de la Revolución Ramiro Valdés Menéndez catalogó de positiva la confección de piezas de repuesto para los ferrocarriles en Planta Mecánica.</p> </div>
          </div>
      <div class="col-md-4 ciclo3">
              <div class="category" itemprop="genre">
          <i class="fa fa-angle-double-right"></i> <a href="/villa-clara">Villa Clara</a>        </div>
      
              <h3 class="title" itemprop="name">
          <a href="/villa-clara/10990-en-fase-final-reconstruccion-del-puente-elevado-en-santo-domingo">En fase final reconstrucción del puente elevado en Santo Domingo</a>
        </h3>
      
      
              <figure class="images">
          <a href="/villa-clara/10990-en-fase-final-reconstruccion-del-puente-elevado-en-santo-domingo">
            <img class="img-responsive lazy"
                              data-original="images/materiales/construccion/2018/03-16-reconstuccion-puente-elevado-santo-domingo/labores-reparacion.jpg" alt="Puente elevado en Santo Domingo" itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <div class="introtext"><p>A punto de concluir la reparación capital del puente elevado de la Carretera Central, en Santo Domingo.</p> </div>
          </div>
  </div>
</div>

      
      <!-- Materiales de categorias -->
              <section class="categoryTop">
  				
  <div class="panel panel-default ">
    <div class="panel-heading">
              <h4 class="panel-title">
          <a href="/villa-clara">
            Villa Clara          </a>
        </h4>

          </div>

    <div class="mod-vanguardia-news categorias" id="mod-vanguardia-news-252">
      
<div class="row basic">
      <div class=" col-sm-6 col-md-6 ciclo1">
      
              <figure class="images ">
          <a href="/deporte/10993-vamos-por-una-presea-a-barranquilla">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/deporte/softbol/2018/02-27-directora-equipo-nacional-femenino-rbv/luisa-maria-medina-gonzalez-750x470.jpg" alt="Luisa María Medina González, directora del equipo nacional femenino de softbol de Cuba." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/deporte/10993-vamos-por-una-presea-a-barranquilla">«Vamos por una presea a Barranquilla»</a>
        </h4>
      
              <div class=" introtext"><p>Luisa Medina, directora de la selección femenina de softbol, ofreció declaraciones a <strong>Vanguardia</strong> sobre el pasado campeonato nacional y las posibilidades de Cuba en Barranquilla.</p> </div>
          </div>
          <div class=" col-sm-6 col-md-6 ciclo2">
      
              <figure class="images hidden-xs">
          <a href="/villa-clara/10987-remedios-encabeza-la-agricultura-urbana-en-villa-clara">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/agricultura/urbana/generales/organoponico-cuba-foto-jose-m-correa-750x470.jpg" alt="Agricultura urbana en Villa Clara" itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/villa-clara/10987-remedios-encabeza-la-agricultura-urbana-en-villa-clara">Remedios encabeza la agricultura urbana en Villa Clara</a>
        </h4>
      
              <div class=" introtext"><p>La provincia marcha en 5to lugar tras recorrer el Grupo Nacional de Agricultura Urbana, Suburbana y Familiar nueve territorios del país.</p> </div>
          </div>
      </div>
    </div>

  </div>

  			</section>
      
              <section class="categoryMiddle">
  				
  <div class="panel panel-default ">
    <div class="panel-heading">
              <h4 class="panel-title">
          <a href="/deporte">
            Deporte          </a>
        </h4>

          </div>

    <div class="mod-vanguardia-news categorias" id="mod-vanguardia-news-253">
      
<div class="row basic">
      <div class=" col-sm-4 col-md-4 ciclo1">
      
              <figure class="images ">
          <a href="/deporte/10983-aqui-los-13-equipos-de-la-serie-provincial-de-beisbol">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/deporte/beisbol/2016/09-22-estadio/guante-pelota-bate-beisbol-1.gif" alt="Guante, bate y pelota de béisbol." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/deporte/10983-aqui-los-13-equipos-de-la-serie-provincial-de-beisbol">Aquí los 13 equipos de la Serie Provincial de Béisbol</a>
        </h4>
      
              <div class="hidden-sm introtext"><p>Nómina de los 13 equipos que participar en la XIII Serie Provincial del Béisbol de Villa Clara.</p> </div>
          </div>
          <div class=" col-sm-4 col-md-4 ciclo2">
      
              <figure class="images hidden-xs">
          <a href="/deporte/10981-gustavo-rolle-tercer-cubano-al-salon-de-la-fama-de-la-lucha">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/deporte/lucha/2018/03-14-gustavo-rolle-sera-exaltado-salon-fama-mundial/gustavo-rolle.jpg" alt="Gustavo Rollé, profesor cubano de lucha." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/deporte/10981-gustavo-rolle-tercer-cubano-al-salon-de-la-fama-de-la-lucha">Gustavo Rollé, tercer cubano  al Salón de la Fama de la Lucha</a>
        </h4>
      
              <div class="hidden-sm introtext"><p>El destacado profesor será exaltado al Salón de la Fama de la Unión Mundial de Luchas (UWW), en el venidero Campeonato Mundial de Budapest.</p> </div>
          </div>
          <div class=" col-sm-4 col-md-4 ciclo3">
      
              <figure class="images hidden-xs">
          <a href="/deporte/10977-se-encarrilara-el-expreso">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/deporte/futbol/2018/03-13-expreso-de-villa-clara/equipo.jpg" alt="Equipo de fútbol Expreso del Centro, de Villa Clara." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/deporte/10977-se-encarrilara-el-expreso">¿Se encarrilará el Expreso?</a>
        </h4>
      
              <div class="hidden-sm introtext"><p>En su última salida de la primera fase, el Expreso encontró la ruta victoriosa, con triunfo, 2 a1, frente al puntero Ciego de Ávila.</p> </div>
          </div>
      </div>
    </div>

  </div>

  			</section>
      
              <section class="categoryDown">
  				
  <div class="panel panel-default ">
    <div class="panel-heading">
              <h4 class="panel-title">
          <a href="/cultura">
            Cultura          </a>
        </h4>

          </div>

    <div class="mod-vanguardia-news categorias" id="mod-vanguardia-news-254">
      
<div class="row basic">
      <div class=" col-sm-4 col-md-4 ciclo1">
      
              <figure class="images ">
          <a href="/cultura/10989-chivos-y-sapos-se-alistan-para-parrandear">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/cultura/patrimonio/2018/03-16-parrandas-camajuani/parranda-portada.jpg" alt="Parrandas Camajuaní" itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/cultura/10989-chivos-y-sapos-se-alistan-para-parrandear">Chivos y Sapos se alistan para parrandear </a>
        </h4>
      
              <div class="hidden-sm introtext"><p>Las populares parrandas de Camajuaní se desarrollarán el próximo lunes 19 de marzo.</p> </div>
          </div>
          <div class=" col-sm-4 col-md-4 ciclo2">
      
              <figure class="images hidden-xs">
          <a href="/cultura/10985-musica-en-las-venas">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/cultura/musica/2018/03-15-entrevista-bibeity-bastida/bibeity-ninnos.jpg" alt="Bibeity con los niños" itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/cultura/10985-musica-en-las-venas">Música en las venas</a>
        </h4>
      
              <div class="hidden-sm introtext"><p>Bibeity Bastida Ramírez guarda la luz en su voz y en la extrema sensibilidad de sus acordes.</p> </div>
          </div>
          <div class=" col-sm-4 col-md-4 ciclo3">
      
              <figure class="images hidden-xs">
          <a href="/cultura/10970-termino-el-caturla-bienvenidos-al-proximo">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/cultura/musica/2018/03-11-clausura-festival-a-tempo-con-carturla-fdr/coros.JPG" alt="Coros en la clausura del Festival A Tempo con Caturla." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/cultura/10970-termino-el-caturla-bienvenidos-al-proximo">Terminó el «Caturla», bienvenidos al próximo</a>
        </h4>
      
              <div class="hidden-sm introtext"><p>Las maestras Freyda Anido y Beatriz Corona recibieron las distinciones «Hijo Ilustre de la Ciudad» y «Por la Colaboración Cultural», respectivamente.</p> </div>
          </div>
      </div>
    </div>

  </div>

  			</section>
      
      <!-- Materiales de secciones -->
              <section class="sections">
  				
  <div class="panel panel-default ">
    <div class="panel-heading">
      <h4 class="panel-title">
        <a href="/opinion-de-periodistas">
          Opinión        </a>
      </h4>
    </div>

		<div class="mod-vanguardia-news opinion" id="mod-vanguardia-news-259">
			
<div class="row basic">
      <div class="ciclo0 col-xs-12 col-sm-6 col-md-6">
      <div class="row">
                  <div class="col-xs-4 col-sm-12 col-md-4">
                                      <figure class="images">
                <img class="img-responsive lazy"
                                    data-original="images/system/periodistas/idalia-vasquez.jpg" alt="Idalia Vázquez Zerquera" itemprop="thumbnailUrl"
                />
              </figure>
                      </div>
        
                          <div class="col-md-8 hidden-xs hidden-sm">
            <div class="popover right">
              <div class="arrow"></div>
              <h5 class="popover-title title" itemprop="name">
                <a href="/opinion-de-periodistas/10945-guadana-sobre-ruedas">
                  Guadaña sobre ruedas                 </a>
              </h5>
              <div class="popover-content">
                <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
                  <span itemprop="name">Idalia Vázquez Zerquera</span>                </small>
              </div>
            </div>
          </div>
        
                          <div class="info col-sm-12 hidden-xs hidden-md hidden-lg ">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10945-guadana-sobre-ruedas">
                Guadaña sobre ruedas               </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Idalia Vázquez Zerquera</span>            </small>
          </div>
        
                          <div class="col-xs-8 hidden-sm hidden-md hidden-lg">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10945-guadana-sobre-ruedas">
                Guadaña sobre ruedas               </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Idalia Vázquez Zerquera</span>            </small>
          </div>
              </div>
    </div>
              <div class="ciclo1 col-xs-12 col-sm-6 col-md-6">
      <div class="row">
                  <div class="col-xs-4 col-sm-12 col-md-4">
                                      <figure class="images">
                <img class="img-responsive lazy"
                                    data-original="images/system/periodistas/liena-maria.jpg" alt="" itemprop="thumbnailUrl"
                />
              </figure>
                      </div>
        
                          <div class="col-md-8 hidden-xs hidden-sm">
            <div class="popover right">
              <div class="arrow"></div>
              <h5 class="popover-title title" itemprop="name">
                <a href="/opinion-de-periodistas/10918-recetas-y-tarjetones-a-buchito-de-agua-y-paso-de-conga">
                  Recetas y tarjetones: «a buchito de agua y paso de conga»                </a>
              </h5>
              <div class="popover-content">
                <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
                  <span itemprop="name">Liena María Nieves</span>                </small>
              </div>
            </div>
          </div>
        
                          <div class="info col-sm-12 hidden-xs hidden-md hidden-lg ">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10918-recetas-y-tarjetones-a-buchito-de-agua-y-paso-de-conga">
                Recetas y tarjetones: «a buchito de agua y paso de conga»              </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Liena María Nieves</span>            </small>
          </div>
        
                          <div class="col-xs-8 hidden-sm hidden-md hidden-lg">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10918-recetas-y-tarjetones-a-buchito-de-agua-y-paso-de-conga">
                Recetas y tarjetones: «a buchito de agua y paso de conga»              </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Liena María Nieves</span>            </small>
          </div>
              </div>
    </div>
          <div class="clearfix"></div>
              <div class="ciclo2 col-xs-12 col-sm-6 col-md-6">
      <div class="row">
                  <div class="col-xs-4 col-sm-12 col-md-4">
                                      <figure class="images">
                <img class="img-responsive lazy"
                                    data-original="images/system/periodistas/narciso-fernandez.jpg" alt="Narciso Fernández Ramírez" itemprop="thumbnailUrl"
                />
              </figure>
                      </div>
        
                          <div class="col-md-8 hidden-xs hidden-sm">
            <div class="popover right">
              <div class="arrow"></div>
              <h5 class="popover-title title" itemprop="name">
                <a href="/opinion-de-periodistas/10895-acrobacias-de-la-leche">
                  Acrobacias de la leche                 </a>
              </h5>
              <div class="popover-content">
                <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
                  <span itemprop="name">Narciso Fernández Ramírez</span>                </small>
              </div>
            </div>
          </div>
        
                          <div class="info col-sm-12 hidden-xs hidden-md hidden-lg ">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10895-acrobacias-de-la-leche">
                Acrobacias de la leche               </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Narciso Fernández Ramírez</span>            </small>
          </div>
        
                          <div class="col-xs-8 hidden-sm hidden-md hidden-lg">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10895-acrobacias-de-la-leche">
                Acrobacias de la leche               </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Narciso Fernández Ramírez</span>            </small>
          </div>
              </div>
    </div>
              <div class="ciclo3 col-xs-12 col-sm-6 col-md-6">
      <div class="row">
                  <div class="col-xs-4 col-sm-12 col-md-4">
                                      <figure class="images">
                <img class="img-responsive lazy"
                                    data-original="images/system/periodistas/yinet-jimenez-hernandez.jpg" alt="Yinet Jiménez Hernández" itemprop="thumbnailUrl"
                />
              </figure>
                      </div>
        
                          <div class="col-md-8 hidden-xs hidden-sm">
            <div class="popover right">
              <div class="arrow"></div>
              <h5 class="popover-title title" itemprop="name">
                <a href="/opinion-de-periodistas/10888-cerrar-el-ciclo-del-re-ciclaje">
                  Cerrar el ciclo del (re)ciclaje                </a>
              </h5>
              <div class="popover-content">
                <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
                  <span itemprop="name">Yinet Jiménez Hernández</span>                </small>
              </div>
            </div>
          </div>
        
                          <div class="info col-sm-12 hidden-xs hidden-md hidden-lg ">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10888-cerrar-el-ciclo-del-re-ciclaje">
                Cerrar el ciclo del (re)ciclaje              </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Yinet Jiménez Hernández</span>            </small>
          </div>
        
                          <div class="col-xs-8 hidden-sm hidden-md hidden-lg">
            <h5 class="title" itemprop="name">
              <a href="/opinion-de-periodistas/10888-cerrar-el-ciclo-del-re-ciclaje">
                Cerrar el ciclo del (re)ciclaje              </a>
            </h5>

            <small class="createdby" itemprop="author" itemscope itemtype="http://schema.org/Person">
              <span itemprop="name">Yinet Jiménez Hernández</span>            </small>
          </div>
              </div>
    </div>
          </div>
		</div>

  </div>
<div class="panel panel-default melaito visible-xs-block"><div class="panel-body">

<div class="custom melaito visible-xs-block"  >
	<p><img title="Caricatura de Alfredo Martirena elecciones en Cuba." src="/images/materiales/melaito/martirena/2018/caricaturas-elecciones/cuba-vota.jpg" alt="Caricatura de Alfredo Martirena sobre elecciones en Cuba." /></p></div>
</div></div>
  			</section>
      
              <section class="row sections">
                      <div class="col-md-4 sectSexeando">
              
  <div class="panel panel-default ">
    <div class="panel-heading">
              <a href="/sexeando">
          <img src="/images/system/logotipos/sexeando.png" alt="Sección de Vanguardia" />
        </a>

          </div>

    <div class="mod-vanguardia-news categorias" id="mod-vanguardia-news-255">
      
<div class="row basic">
      <div class=" col-sm-12 col-md-12 ciclo1">
      
              <figure class="images ">
          <a href="/sexeando/10750-lo-que-no-sabes-sobre-la-sifilis">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/sexeando/03-11-sifilis/Condon-750x430.jpg" alt="Infecciones de Transmisión Sexual" itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/sexeando/10750-lo-que-no-sabes-sobre-la-sifilis">La bacteria de Cupido</a>
        </h4>
      
              <div class=" introtext"><p>En Cuba, la tasa de sífilis se estableció en 44.6 por cada 100 mil habitantes. Con respecto a la diabetes mellitus, la diferencia es de ...</div>
          </div>
      </div>
    </div>

  </div>

            </div>
  				
                      <div class="col-md-4 sectRazones">
              
  <div class="panel panel-default ">
    <div class="panel-heading">
              <a href="/razones">
          <img src="/images/system/logotipos/razones.png" alt="Sección de Vanguardia" />
        </a>

          </div>

    <div class="mod-vanguardia-news categorias" id="mod-vanguardia-news-257">
      
<div class="row basic">
      <div class=" col-sm-12 col-md-12 ciclo1">
      
              <figure class="images ">
          <a href="/razones/10926-con-la-sombra-convertida-en-luz">
            <img class="img-responsive lazy"
                              data-original="http://www.vanguardia.cu/images/materiales/agricultura/avicultura/2018/03-06-granja-avicola-la-quinta-camajuani-rbv/marilin-guevara-750x470.jpg" alt="Marilín Guevara, trabajadora de la granja avícola de La Quinta, Camajuaní." itemprop="thumbnailUrl"
            />
          </a>
        </figure>
      
              <h4 class="title" itemprop="name">
          <a href="/razones/10926-con-la-sombra-convertida-en-luz">Con la sombra convertida en luz</a>
        </h4>
      
              <div class=" introtext"><p>«Irma» dejó huellas en muchos cubanos, pero un grupo de mujeres de la avicultura en Camajuaní se alzó ante los destrozos y desterró las ruinas.</p> </div>
          </div>
      </div>
    </div>

  </div>

            </div>
  				
                      <div class="col-md-4 sectCartelera">
              
  <div class="panel panel-default">
    <div class="panel-heading">
              <a data-toggle="modal" href="#carteleraModal">
          <img src="/images/system/logotipos/cartelera-cultural.png" alt="Sección de Vanguardia" />
        </a>
          </div>

		<div class="mod-vanguardia-billboard cartelera" id="mod-vanguardia-billboard-277">
			
<div class="row basic">
  
    
    <div class="col-md-12 ">
      <div class="row event">
        <div class="col-xs-4 col-sm-4 col-md-4">
                              <div class="dateNumber">18</div>
          <div class="dateWord">Domingo</div>
        </div>

        <div class="col-xs-8 col-sm-8 col-md-8">
          <div class="eventDetails"><p>Presentación para niños del grupo Teatro La Rosa. Peña infantil Siempre nos une la risa. La Retreta. </p>
</div>
        </div>
      </div>
    </div>
      
    
    <div class="col-md-12 ">
      <div class="row event">
        <div class="col-xs-4 col-sm-4 col-md-4">
                              <div class="dateNumber">19</div>
          <div class="dateWord">Lunes</div>
        </div>

        <div class="col-xs-8 col-sm-8 col-md-8">
          <div class="eventDetails"><p>Parrandas de Camajuaní. Presentación del libro Anestesia de Edelmis Anoceto</p>
</div>
        </div>
      </div>
    </div>
      
  <div class="col-xs-12 col-sm-12 col-md-12">
    <button type="button" class="btn btn-primary btn-block moreEvent" data-toggle="modal" data-target="#carteleraModal">
      Ver más detalles
    </button>
  </div>
</div>
			
<div id="carteleraModal" class="modal fade" tabindex="-1" role="dialog" aria-labelledby="carteleraModalLabel">
  <div class="modal-dialog" role="document">
    <div class="modal-content">
      <div class="modal-header">
        <button type="button" class="close" data-dismiss="modal" aria-label="Close"><span aria-hidden="true">&times;</span></button>
        <img src="/images/system/logotipos/cartelera-cultural.png" alt="Sección de Vanguardia" />
      </div>
      <div class="modal-body">
        <div class="row">
                      <div class="col-md-12">
              <div class="row event odd">
                <div class="col-xs-12 col-sm-2 col-md-2">
                                                      <div class="dateNumber">18</div>
                  <div class="dateWord">Domingo</div>
                </div>

                <div class="col-xs-12 col-sm-10 col-md-10">
                  <div class="eventDetails">
<p>Consejo Provincial de Artes Escénicas: Presentación para niños del grupo Teatro La Rosa con la dirección general de Roxana Pineda. Disco Isla. 10:00 a.m.</p>
<hr />
<p>Peña infantil Siempre nos une la risa. Patio de El Mejunje. 10:00 a.m.</p>
<hr />
<p>Guiñol de Remedios Fidel Galbán presenta la Peña infantil De Caramelo. Cabaret Las Leyendas, Remedios. 10:00 a.m.</p>
<hr />
<p>Club del Danzón. Patio de El Mejunje. 2:00 p.m.</p>
<hr />
<p>Presentación teatral del grupo Mejunje. El Decamerón cubano. Dir. Gral. Ramón Silverio. Sala Margarita Casallas de El Mejunje. 9:00 p.m.</p>
<hr />
<p>Presentación del Quinteto de saxofones. Teatro La Caridad. 9:30 p.m.</p>
<hr />
<p>Espectáculo De Cuba soy con el grupo Baila Cuba, dirección de Yusniel González y la compañía Danza del Alma, dirigida por Ernesto Alejo. Cabaret Cubanacán de Santa Clara. 11:30 p.m.</p>
<hr />
<p>Centro Provincial de Artes Visuales: Peña con el Trío Tres joyas. 6:00 p.m.</p>
<hr />
<p>Museo de Artes Decorativas: Maikel's Quartet promueve la música popular tradicional. Sala de Conciertos.6:00 p.m.</p>
<hr />
<p>Casa de Cultura Juan Marinello: En clave de Sol. Los Fakires promueven y difunden la música tradicional cubana. 4:00 p.m.</p>
<hr />
<p>Parque Leoncio Vidal: La Retreta. La Banda Provincial de Conciertos promueve la música cubana y universal de concierto. Glorieta. 8:00 p.m.</p>
<hr />
<p>El reparador de sueños. La Compañía Alánimo comparte su teatro y más con los niños. 10:00 a.m.</p>
<hr />
<p>Casa de la Ciudad: Formalmente informal. Peña de Vivian Carranza, espacio donde promueve la música popular tradicional e interactúa con su público. 5:00 p.m.</p>
<hr />
<p>Camajuaní: Mañanas de mi campiña. Peña campesina con los poetas aficionados del patio. Área del parque Vidal. 9:00 a.m.</p>
<p>Guateque campesino de Camajuaní. Presentación del grupo Son del Valle, y la presentación de la revista Signos 72 Esto si pa’ Juego, a cargo del escritor Joel Sequeda. Patio de Don Francisco, Museo Municipal Hermanos Vidal Caro. 3:00 p.m.</p>
<p>Tardes Rancheras. Presentación de Los Galanes Aztecas. Área de la plataforma central. 6:00 p.m.</p>
<hr />
<p>Ranchuelo: Sax Cantabille. Descarga del jazz con el saxofonista Pepe Díaz. Galería de Arte. 11:30 a.m.</p>
<hr />
<p>Quemado de Güines: Carrusel de la alegría. Invitado Grupo Collage. Parque de diversiones. 10:00 a.m.</p>
<p>Guateque de mi campiña. Invitada la peña Rocío de la Mañana. Casa de Cultura Luis Jorge León. 2:00 p.m.</p>
<p>Retreta de la Banda de Conciertos Alejandro García Caturla. Parque José Martí. 8:00 p.m.</p></div>
                </div>
              </div>
            </div>
                                  <div class="col-md-12">
              <div class="row event even">
                <div class="col-xs-12 col-sm-2 col-md-2">
                                                      <div class="dateNumber">19</div>
                  <div class="dateWord">Lunes</div>
                </div>

                <div class="col-xs-12 col-sm-10 col-md-10">
                  <div class="eventDetails">
<p>El grupo Alánimo, con dirección general de Carmen Margolles, presenta la obra <em>El patio de Cachita y Antonio</em>, en el poblado de Mata. 10:00 a.m.</p>
<hr />
<p>Centro Cultural El Mejunje: Peña del Bolero Arráncame la Vida. Patio de El Mejunje. 6:00 p.m.<br />Club Extraños en la noche. Patio de El Mejunje. 10:00 p.m.</p>
<hr />
<p><strong>Cine Camilo Cienfuegos:</strong> <em>Abusos sexuales</em><strong>. </strong>Género: Drama.  Hora: 1:00 p.m.<br /><em>Daddy΄s Home 2</em>. Hora: 8:00 p.m.</p>
<hr />
<p><strong>Sala de Video Luis Estévez: </strong><em>La Seducción / The Beguiled/ </em>Estados Unidos/2017/ 91´/Dir. Sofia Coppola /Drama/16 años. Hora: 8:30 p.m.</p>
<hr />
<p><strong>Centro Provincial de Patrimonio Cultural: </strong>Recorrer mis orígenes. Se visita el busto de Leoncio Vidal en el parque que lleva su nombre, recordatorio de su vida y obra. 2:00 p.m.</p>
<hr />
<p><strong>Museo de Artes Decorativas: </strong>Noche de tríos. Con la conducción y dirección de Adonis, el trío invitado nos ofrece su música trovadoresca tradicional. 9:00 p.m.</p>
<hr />
<p><strong>Casa de Cultura Juan Marinello: </strong>Concierto de música Coral con la agrupación Audinos. 9:00 p.m.</p>
<hr />
<p><strong>Cifuentes:</strong> Fantasías Animadas. Invitado el grupo de teatro infantil Alánimo. Casita infantil del poblado de Mata. 10:00 a.m.</p>
<hr />
<p><strong>Santo Domingo: </strong>Fiesta Popular en Rodrigo. Presentación de Honda Retro, a las 4:00 p.m. y de la orquesta La China, 9:00 p.m.</p></div>
                </div>
              </div>
            </div>
                              </div>
      </div>
    </div>
  </div>
</div>
		</div>

  </div>

            </div>
  				  			</section>
          </div>
    <aside class="col-md-4">
              <div class="blockCategoryCenter">
  				

<div class="custom"  >
	<p><a href="/elecciones-por-cuba"><img src="/images/system/banners/elecciones.jpg" alt="" /></a></p></div>
<div class="panel panel-default"><div class="panel-heading"><h4 class="panel-title"> Lo último</h4></div><div class="mod-vanguardia-news ultimo " id="mod-vanguardia-news-251">
	
<table class="table">
	<tbody>
					<tr>
      	<td>
      		<h6>
						<a href="/villa-clara/10995-gente-agradecida-en-aguada-de-moya">Gente agradecida en Aguada de Moya</a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/villa-clara/10996-cuatro-fallecidos-en-lamentable-accidente-del-transito-en-villa-clara">Cuatro fallecidos en lamentable accidente del tránsito en Villa Clara</a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/deporte/10994-camajuani-y-remedios-por-el-titulo-provincial">Camajuaní y Remedios por el título provincial</a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/deporte/10993-vamos-por-una-presea-a-barranquilla">«Vamos por una presea a Barranquilla»</a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/cultura/10989-chivos-y-sapos-se-alistan-para-parrandear">Chivos y Sapos se alistan para parrandear </a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/deporte/10992-este-sabado-se-sabran-los-finalistas-en-el-beisbol-provincial">Este sábado se sabrán los finalistas en el béisbol provincial</a>
					</h6>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/villa-clara/10990-en-fase-final-reconstruccion-del-puente-elevado-en-santo-domingo">En fase final reconstrucción del puente elevado en Santo Domingo</a>
					</h6>
      	</td>
      </tr>
			</tbody>
</table>
</div>
</div>

<div class="custom"  >
	<p><a href="/especial-por-siempre-fidel"><img src="/images/system/banners/por-siempre-fidel.jpg" alt="" /></a></p></div>


<div class="custom"  >
	<p><img src="/images/system/banners/baner-750x150px.jpg" alt="" /></p></div>

  			</div>
                    <div class="blockCenter">
  				

<div class="custom"  >
	<p><a title="Discursos de Raúl" href="/discursos-raul" rel="alternate"><img src="/images/system/banners/discurso-raul.jpg" alt="" /></a></p></div>


<div class="custom"  >
	<p><a title="Reflexiones de Fidel" href="/reflexiones-de-fidel" rel="alternate"><img src="/images/system/banners/reflexiones-fidel.jpg" alt="" /></a></p></div>
<div class="panel panel-default"><div class="panel-heading"><h4 class="panel-title"> Lo más visto</h4></div>
<table class="table mostread">
	<tbody>
					<tr>
      	<td>
      		<h6>
						<a href="/sexeando/10750-lo-que-no-sabes-sobre-la-sifilis">La bacteria de Cupido</a>
					</h6>
					<div class="clearfix"></div>
      		<div class="view pull-left">
      			04 Febrero 2018</div>
      		<div class="hits"><i class="fa fa-eye"></i> 1739</div>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/deporte/10983-aqui-los-13-equipos-de-la-serie-provincial-de-beisbol">Aquí los 13 equipos de la Serie Provincial de Béisbol</a>
					</h6>
					<div class="clearfix"></div>
      		<div class="view pull-left">
      			14 Marzo 2018</div>
      		<div class="hits"><i class="fa fa-eye"></i> 1388</div>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/deporte/10969-manicaragua-se-aseguro-entre-los-cuatro-en-jornada-de-barridas">Manicaragua se aseguró entre los cuatro en jornada de barridas</a>
					</h6>
					<div class="clearfix"></div>
      		<div class="view pull-left">
      			12 Marzo 2018</div>
      		<div class="hits"><i class="fa fa-eye"></i> 1018</div>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/villa-clara/10974-confieren-premio-provincial-de-conservacion-y-restauracion">Confieren premio provincial de Conservación y Restauración </a>
					</h6>
					<div class="clearfix"></div>
      		<div class="view pull-left">
      			13 Marzo 2018</div>
      		<div class="hits"><i class="fa fa-eye"></i> 771</div>
      	</td>
      </tr>
					<tr>
      	<td>
      		<h6>
						<a href="/villa-clara/10990-en-fase-final-reconstruccion-del-puente-elevado-en-santo-domingo">En fase final reconstrucción del puente elevado en Santo Domingo</a>
					</h6>
					<div class="clearfix"></div>
      		<div class="view pull-left">
      			16 Marzo 2018</div>
      		<div class="hits"><i class="fa fa-eye"></i> 751</div>
      	</td>
      </tr>
			</tbody>
</table>
</div>
  			</div>
                    <div class="blockCategoryBottom">
  				<div class="panel panel-default melaito"><div class="panel-heading"><h4 class="panel-title"> Opinión gráfica </h4></div><div class="panel-body">

<div class="custom melaito"  >
	<p><img title="Caricatura de Alfredo Martirena elecciones en Cuba." src="/images/materiales/melaito/martirena/2018/caricaturas-elecciones/cuba-vota.jpg" alt="Caricatura de Alfredo Martirena sobre elecciones en Cuba." /></p></div>
</div></div><div class="panel panel-default"><div class="panel-heading"><h4 class="panel-title"> Recomendamos</h4></div><div class="panel-body">

<div class="mod-vanguardia-news recomendamos " id="mod-vanguardia-news-268">
	
<div class="basic">
  <div class="row">
          <div class="col-xs-12 col-sm-6 col-md-12 content">
        <div class="row">
                      <div class="col-xs-5 col-sm-5 col-md-5">
              
                              <figure class="images">
                  <a href="/villa-clara/10857-fitcuba-2018-descubrir-a-cuba-desde-el-centro">
                    <img class="img-responsive lazy"
                                              data-original="http://www.vanguardia.cu/images/materiales/turismo/2018/02-22-fitcuba-cubanacan/aeropuerto-internacional-abel-santamaria.jpg" alt="Aeropuerto Internacional Abel Santa María" itemprop="thumbnailUrl"
                    />
                  </a>
                </figure>
                          </div>
          
                      <div class="col-xs-7 col-sm-7 col-md-7">
              <h6 class="title">
                <a href="/villa-clara/10857-fitcuba-2018-descubrir-a-cuba-desde-el-centro">FitCuba 2018: descubrir la isla desde el centro</a>
              </h6>
            </div>
          
                  </div>
      </div>
          <div class="col-xs-12 col-sm-6 col-md-12 content">
        <div class="row">
                      <div class="col-xs-5 col-sm-5 col-md-5">
              
                              <figure class="images">
                  <a href="/villa-clara/10840-candonga-que-nace-torcida">
                    <img class="img-responsive lazy"
                                              data-original="http://www.vanguardia.cu/images/materiales/trabajo/cuenta-propia/2018/02-20-candongas-santa-clara-reportaje-smb/candonga-las-flores.jpg" alt="" itemprop="thumbnailUrl"
                    />
                  </a>
                </figure>
                          </div>
          
                      <div class="col-xs-7 col-sm-7 col-md-7">
              <h6 class="title">
                <a href="/villa-clara/10840-candonga-que-nace-torcida">Candonga que nace torcida...</a>
              </h6>
            </div>
          
                  </div>
      </div>
          <div class="col-xs-12 col-sm-6 col-md-12 content">
        <div class="row">
                      <div class="col-xs-5 col-sm-5 col-md-5">
              
                              <figure class="images">
                  <a href="/villa-clara/10820-remedios-vieja-dama-aferrada-al-futuro">
                    <img class="img-responsive lazy"
                                              data-original="http://www.vanguardia.cu/images/materiales/turismo/2018/02-16-remedios/camino-del-principe.jpg" alt="Hotel Camino del Príncipe" itemprop="thumbnailUrl"
                    />
                  </a>
                </figure>
                          </div>
          
                      <div class="col-xs-7 col-sm-7 col-md-7">
              <h6 class="title">
                <a href="/villa-clara/10820-remedios-vieja-dama-aferrada-al-futuro">Remedios, vieja dama aferrada al futuro</a>
              </h6>
            </div>
          
                  </div>
      </div>
      </div>
</div>
</div>

</div></div>
  			</div>
          </aside>
  </div>
  <div class="row">
    <div class="col-md-12 gallery-featured">
          	  <div class="panel panel-default"><div class="panel-heading"><h4 class="panel-title"> Galerías</h4></div><div class="panel-body"><div class="mod-vanguardia-gallery destacadas" id="mod-vanguardia-gallery-266">
	<div class="row">
		<div class="col-xs-12 col-sm-8 col-md-8 galleries">
			
<div class="row mainGallery">
  
        <div class="col-xs-12 col-sm12 col-md-12 main-img">
      <figure>
                  <a href="/galerias/10964-un-voto-por-la-patria">
                    <img class="img-responsive lazy"
                            data-original="images/galerias/2018/marzo/03-11-elecciones/01-elecciones-generales.jpg" itemprop="thumbnailUrl" />
                  </a>
        
                <h4 class="gallery-title">
                      <a href="/galerias/10964-un-voto-por-la-patria">
                        Un voto por la Patria                      </a>
                  </h4>
      </figure>
    </div>

    </div>
			
<div class="row restGallery">
                          <div class="col-xs-12 col-sm-4 col-md-3 ">
        <figure>
                      <a href="/galerias/10965-con-un-voto-de-humor">
                        <img class="img-responsive lazy"
                                data-original="images/materiales/melaito/martirena/2018/caricaturas-elecciones/estatura-electoral-portada.jpg" itemprop="thumbnailUrl" />
                      </a>
                  </figure>
                <h6 class="gallery-title hidden-sm hidden-md">
                      <a href="/galerias/10965-con-un-voto-de-humor">
                        Con un voto de humor                      </a>
                  </h6>
      </div>
                    <div class="col-xs-12 col-sm-4 col-md-3 ">
        <figure>
                      <a href="/elecciones-por-cuba/10949-ya-villa-clara-elige">
                        <img class="img-responsive lazy"
                                data-original="images/galerias/2018/marzo/03-11-santa-clara-en-elecciones-smb/santa-clara-elecciones-2018.jpg" itemprop="thumbnailUrl" />
                      </a>
                  </figure>
                <h6 class="gallery-title hidden-sm hidden-md">
                      <a href="/elecciones-por-cuba/10949-ya-villa-clara-elige">
                        Villa Clara elige #PorCuba                      </a>
                  </h6>
      </div>
                    <div class="col-xs-12 col-sm-4 col-md-3 ">
        <figure>
                      <a href="/villa-clara/10764-luz-de-antorchas-por-marti-por-la-historia">
                        <img class="img-responsive lazy"
                                data-original="images/galerias/2018/febrero/02-06-marcha-antorchas-smb/marcha-avanza.jpg" itemprop="thumbnailUrl" />
                      </a>
                  </figure>
                <h6 class="gallery-title hidden-sm hidden-md">
                      <a href="/villa-clara/10764-luz-de-antorchas-por-marti-por-la-historia">
                        Luz de antorchas por Martí, por la historia                      </a>
                  </h6>
      </div>
                    <div class="col-xs-12 col-sm-4 col-md-3 hidden-sm">
        <figure>
                      <a href="/galerias/10627-festival-nacional-de-la-trova-longina-2018">
                        <img class="img-responsive lazy"
                                data-original="images/galerias/2018/enero/01-tony-avila.jpg" itemprop="thumbnailUrl" />
                      </a>
                  </figure>
                <h6 class="gallery-title hidden-sm hidden-md">
                      <a href="/galerias/10627-festival-nacional-de-la-trova-longina-2018">
                        Festival Nacional de la Trova Longina 2018                       </a>
                  </h6>
      </div>
      </div>
		</div>

		<div class="col-xs-12 col-sm-4 col-md-4 flashazo">
			
<figure class="logoFlash">
  <img class="img-responsive lazy"
    title="¡Hasta siempre, Comandante!"
    data-original="images/system/banners/logo-flashazo-horizontal.png" itemprop="thumbnailUrl" />
</figure>

<figure class="k-effect rotate fileFlash">
  <a href="#" data-toggle="modal" data-target="#flashModal">
    <img class="img-responsive lazy"
      title="¡Hasta siempre, Comandante!"
      data-original="images/system/flachazo/2017/octubre/01-hasta-siempre-comandante.jpg" itemprop="thumbnailUrl" />
    <span class="mask"></span>
  </a>
  <h4 class="titleFlash">
    <a href="#" data-toggle="modal" data-target="#flashModal">
      ¡Hasta siempre, Comandante!    </a>
  </h4>
</figure>

<div class="modal fade" id="flashModal" tabindex="-1" role="dialog">
  <div class="modal-dialog" role="document">
    <div class="modal-content">
      <div class="modal-header">
        <button type="button" class="close" data-dismiss="modal" aria-label="Close"><span aria-hidden="true">&times;</span></button>
      </div>
      <div class="modal-body">
        <figure class="modalFileFlash">
          <img class="img-responsive"
            title="¡Hasta siempre, Comandante!"
            src="/images/system/flachazo/2017/octubre/01-hasta-siempre-comandante.jpg" itemprop="thumbnailUrl" />
        </figure>

        <div class="modalInfo">
          <h4 class="modalTitleFlash">¡Hasta siempre, Comandante!</h4>
          <div class="author">Fotógrafo: Ramón Barreras Valdés</div>
          <p class="descriptionFlash">Hoy, cuando Villa Clara, en nombre de Cuba, honra al Guerrillero Heroico Ernesto Che Guevara, también en la Plaza, en las calles y en el corazón de este pueblo, sigue presente nuestro Comandante en Jefe.</p>
        </div>
      </div>
    </div>
  </div>
</div>
		</div>
	</div>
</div>
</div></div>
          </div>
  </div>
</div>

  
<footer>
  <div class="container">
    <div class="row menusFooter">
      <div class="panel panel-default">
        <div class="panel-heading">
          <h5 class="panel-title">Publicaciones nacionales</h5>
        </div>

        <ul class="list-group">
          <li class="list-group-item">
            <a href="http://www.acn.cu/" target="_blank">ACN</a> <span>|</span>
            <a href="http://www.cubadebate.cu/" target="_blank">Cubadebate</a> <span>|</span>
            <a href="http://www.cubahora.cu" target="_blank">Cubahora</a> <span>|</span>
            <a href="http://www.granma.cu/" target="_blank">Granma</a> <span>|</span>
            <a href="http://www.trabajadores.cu" target="_blank">Trabajadores</a> <span>|</span>
            <a href="http://www.juventudrebelde.cu/" target="_blank">Juventud Rebelde</a> <span>|</span>
            <a href="http://www.prensa-latina.cu/" target="_blank">Prensa Latina</a>
          </li>
        </ul>
      </div>

      <div class="panel panel-default">
        <div class="panel-heading">
          <h5 class="panel-title">Periódicos provinciales</h5>
        </div>

        <ul class="list-group">
          <li class="list-group-item">
            <a href="http://www.escambray.cu/" target="_blank">Escambray</a> <span>|</span>
            <a href="http://www.5septiembre.cu/" target="_blank">Cinco de Septiembre</a> <span>|</span>
            <a href="http://www.invasor.cu/" target="_blank">Invasor</a> <span>|</span>
            <a href="http://www.giron.cu/" target="_blank">Girón</a> <span>|</span>
            <a href="http://www.adelante.cu/" target="_blank">Adelante</a> <span>|</span>
            <a href="http://www.ahora.cu/" target="_blank">Ahora</a> <span>|</span>
            <a href="http://www.guerrillero.cu/" target="_blank">Guerrillero</a> <span>|</span>
            <a href="http://www.lademajagua.cu/" target="_blank">La Demajagua</a> <span>|</span>
            <a href="http://www.periodico26.cu/" target="_blank">Periódico 26</a> <span>|</span>
            <a href="http://www.sierramaestra.cu/" target="_blank">Sierra Maestra</a> <span>|</span>
            <a href="http://www.venceremos.cu/" target="_blank">Venceremos</a> <span>|</span>
            <a href="http://www.tribuna.cu/" target="_blank">Tribuna de La Habana</a> <span>|</span>
            <a href="http://diariomayabeque.cu/" target="_blank">Mayabeque</a> <span>|</span>
            <a href="http://www.artemisadiario.cu/" target="_blank">El Artemiseño</a> <span>|</span>
            <a href="http://www.periodicovictoria.cu/" target="_blank">Victoria</a>
          </li>
        </ul>
      </div>
    </div>

    <div class="copyright">
      <p>Copyright &copy; 2018 Vanguardia. Todos los derechos reservados.</p>
      <p>Se autoriza la reproducción parcial o total de trabajos de nuestro sitio, siempre que se acredite la fuente.</p>
    </div>
  </div>
</footer>

        <script type="text/javascript">
   var _paq = _paq || [];
   _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
   _paq.push(["setCookieDomain", "*.www.vanguardia.cu"]);
   _paq.push(["setDomains", ["*.www.vanguardia.cu"]]);
   _paq.push(['trackPageView']);
   _paq.push(['enableLinkTracking']);
   (function() {
     var u=(("https:" == document.location.protocol) ? "https" : "http") 
+ "://piwikstats.cip.cu/";
     _paq.push(['setTrackerUrl', u+'piwik.php']);
     _paq.push(['setSiteId', 113]);
     var d=document, g=d.createElement('script'),
s=d.getElementsByTagName('script')[0]; g.type='text/javascript';
     g.defer=true; g.async=true; g.src=u+'piwik.js';
s.parentNode.insertBefore(g,s);
   })();
</script>
<noscript><p><img src="http://piwikstats.cip.cu/piwik.php?idsite=113" 
style="border:0;" alt="" /></p></noscript>  </body>

</html>