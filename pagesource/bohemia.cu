#include "hip/hip_runtime.h"
﻿

<!DOCTYPE html PUBLIC "-//W3C//DTD XHTML 1.0 Transitional//EN"
    "http://www.w3.org/TR/xhtml1/DTD/xhtml1-transitional.dtd">

<html xmlns="http://www.w3.org/1999/xhtml" lang="es-ES">

<head profile="http://gmpg.org/xfn/11">
	
	<meta http-equiv="Content-Type" content="text/html; charset=UTF-8" />
    
    <meta name="viewport" content="width=device-width" />
	
	
	<title>
		Revista Bohemia | Revista cubana de actualidad general fundada en 1908	</title>
	
	<link rel="shortcut icon" href="/favicon.ico" type="image/x-icon" />
    
	<link rel="stylesheet" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/style.css" type="text/css" /> <!-- the main structure and main page elements style --> 
    
    <link rel="stylesheet" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/js/js.css" type="text/css" media="screen" /> <!-- styles for the various jquery plugins -->
    <!--[if IE 7]>
            <link rel="stylesheet" type="text/css" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/css/ie7.css" />
    <![endif]-->
    
    <!--[if IE 8]>
            <link rel="stylesheet" type="text/css" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/css/ie8.css" />
    <![endif]-->
    
    <!--[if gt IE 8]>
            <link rel="stylesheet" type="text/css" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/css/ie9.css" />
    <![endif]-->
    
    <link rel="stylesheet" href="http://bohemia.cu/wp-content/themes/Made-Theme/made/custom.css" type="text/css" /> <!-- custom css for users to edit instead of built-in stylesheets -->
    
        
        	<style type="text/css">	
			#top-menu ul li.current_page_ancestor a, 
			#top-menu ul li.current_page_parent a {background:none;color: #DCE6EE;}
			#top-menu ul li a:hover, #top-menu ul li:hover a, #top-menu ul li.over a {background: #FFF;color: #000;}
    		.cat-menu ul li.current_page_ancestor a, 
			.cat-menu ul li.current_page_parent a {background:none;}
			.cat-menu ul li a:hover, .cat-menu ul li:hover a, .cat-menu ul li.over a {background: url(http://bohemia.cu/wp-content/themes/Made-Theme/made/images/cat-menu-highlight.png) repeat-x 0px 0px;}
		</style>
        
        
    	<style type="text/css">		
			body { background-image:url(http://bohemia.cu/wp-content/themes/Made-Theme/made/images/backgrounds/bg-none.png); background-position:center top;
							background-repeat:no-repeat;
						}
			#page-highlight {display:none;} /*only looks good with light bg, so just don't show it*/
			#top-menu-shadow {background:none;} /*still want the height of the element, just don't show it*/ }		
		</style>    
    
	<!-- color scheme -->
	<style type="text/css">
        #logo-bar-wrapper {background-color:#FFFFFF;}
		        <!--#dontmiss-header {color:##C32C0D;}-->
		#dontmiss-header {color:#746657; !important}
    </style>
    
        
    	
	<link rel="pingback" href="http://bohemia.cu/xmlrpc.php" />

	    
        
	<style type='text/css'>Default CSS Comment</style>
<!-- All in One SEO Pack 2.4.4.1 by Michael Torbert of Semper Fi Web Design[-1,-1] -->
<meta name="description"  content="Fundada en 1908, la revista aborda la actualidad de Cuba y el Mundo, en temas como política, deportes, cultura, salud, historia, opinión y entretenimientos" />

<meta name="keywords"  content="Cuba, Fidel Castro, Raúl Castro, Revista Bohemia, Noticias de Cuba, Noticias del mundo, Noticias de Deportes, Noticias de Cultura, Opinión, Historia de Cuba, Recetas de Cocina, crucigramas, húmor, entretenimiento, La Habana, actualidad" />
<meta name="google-site-verification" content="&quot;7M30BORWduXBexrH9n0Q-wTpcc2WSc8p92TtCir8oaY&quot;" />
<script type="application/ld+json">
        {
          "@context": "http://schema.org",
          "@type": "WebSite",		  "name": "Bohemia",		  "url": "http://bohemia.cu/"
        }
</script>
<link rel='next' href='http://bohemia.cu/page/2/' />

<link rel="canonical" href="http://bohemia.cu/" />
			<script type="text/javascript" >
				window.ga=window.ga||function(){(ga.q=ga.q||[]).push(arguments)};ga.l=+new Date;
				ga('create', 'UA-8586404-1', { 'cookieDomain': 'www.bohemia.cu' } );
				// Plugins
				ga('require', 'linkid', 'linkid.js');ga('require', 'outboundLinkTracker');
				ga('send', 'pageview');
			</script>
			<script async src="https://www.google-analytics.com/analytics.js"></script>
			<script async src="https://cdnjs.cloudflare.com/ajax/libs/autotrack/2.4.0/autotrack.js"></script>
<!-- /all in one seo pack -->
<link rel='dns-prefetch' href='//s.w.org' />
<link rel="alternate" type="application/rss+xml" title="Revista Bohemia &raquo; Feed" href="http://bohemia.cu/feed/" />
<link rel="alternate" type="application/rss+xml" title="Revista Bohemia &raquo; RSS de los comentarios" href="http://bohemia.cu/comments/feed/" />
		<script type="text/javascript">
			window._wpemojiSettings = {"baseUrl":"https:\/\/s.w.org\/images\/core\/emoji\/2.4\/72x72\/","ext":".png","svgUrl":"https:\/\/s.w.org\/images\/core\/emoji\/2.4\/svg\/","svgExt":".svg","source":{"concatemoji":"http:\/\/bohemia.cu\/wp-includes\/js\/wp-emoji-release.min.js?ver=4.9.4"}};
			!function(a,b,c){function d(a,b){var c=String.fromCharCode;l.clearRect(0,0,k.width,k.height),l.fillText(c.apply(this,a),0,0);var d=k.toDataURL();l.clearRect(0,0,k.width,k.height),l.fillText(c.apply(this,b),0,0);var e=k.toDataURL();return d===e}function e(a){var b;if(!l||!l.fillText)return!1;switch(l.textBaseline="top",l.font="600 32px Arial",a){case"flag":return!(b=d([55356,56826,55356,56819],[55356,56826,8203,55356,56819]))&&(b=d([55356,57332,56128,56423,56128,56418,56128,56421,56128,56430,56128,56423,56128,56447],[55356,57332,8203,56128,56423,8203,56128,56418,8203,56128,56421,8203,56128,56430,8203,56128,56423,8203,56128,56447]),!b);case"emoji":return b=d([55357,56692,8205,9792,65039],[55357,56692,8203,9792,65039]),!b}return!1}function f(a){var c=b.createElement("script");c.src=a,c.defer=c.type="text/javascript",b.getElementsByTagName("head")[0].appendChild(c)}var g,h,i,j,k=b.createElement("canvas"),l=k.getContext&&k.getContext("2d");for(j=Array("flag","emoji"),c.supports={everything:!0,everythingExceptFlag:!0},i=0;i<j.length;i++)c.supports[j[i]]=e(j[i]),c.supports.everything=c.supports.everything&&c.supports[j[i]],"flag"!==j[i]&&(c.supports.everythingExceptFlag=c.supports.everythingExceptFlag&&c.supports[j[i]]);c.supports.everythingExceptFlag=c.supports.everythingExceptFlag&&!c.supports.flag,c.DOMReady=!1,c.readyCallback=function(){c.DOMReady=!0},c.supports.everything||(h=function(){c.readyCallback()},b.addEventListener?(b.addEventListener("DOMContentLoaded",h,!1),a.addEventListener("load",h,!1)):(a.attachEvent("onload",h),b.attachEvent("onreadystatechange",function(){"complete"===b.readyState&&c.readyCallback()})),g=c.source||{},g.concatemoji?f(g.concatemoji):g.wpemoji&&g.twemoji&&(f(g.twemoji),f(g.wpemoji)))}(window,document,window._wpemojiSettings);
		</script>
		<style type="text/css">
img.wp-smiley,
img.emoji {
	display: inline !important;
	border: none !important;
	box-shadow: none !important;
	height: 1em !important;
	width: 1em !important;
	margin: 0 .07em !important;
	vertical-align: -0.1em !important;
	background: none !important;
	padding: 0 !important;
}
</style>
<link rel='stylesheet' id='srp-layout-stylesheet-css'  href='http://bohemia.cu/wp-content/plugins/special-recent-posts/css/layout.css?ver=4.9.4' type='text/css' media='all' />
<link rel='stylesheet' id='wordpress-popular-posts-css-css'  href='http://bohemia.cu/wp-content/plugins/wordpress-popular-posts/public/css/wpp.css?ver=4.0.13' type='text/css' media='all' />
<script type='text/javascript' src='http://bohemia.cu/wp-includes/js/jquery/jquery.js?ver=1.12.4'></script>
<script type='text/javascript' src='http://bohemia.cu/wp-includes/js/jquery/jquery-migrate.min.js?ver=1.4.1'></script>
<style type="text/css">
	#category-posts-2-internal .cat-post-item img {max-width: initial; max-height: initial; margin: initial;}
#category-posts-2-internal .cat-post-author {margin-bottom: 0;}
#category-posts-2-internal .cat-post-thumbnail {margin: 5px 10px 5px 0;}
#category-posts-2-internal .cat-post-item:before {content: ""; clear: both;}
#category-posts-2-internal .cat-post-title {font-size: 15px;}
#category-posts-2-internal .cat-post-current .cat-post-title {font-weight: bold; text-transform: uppercase;}
#category-posts-2-internal .cat-post-date {font-size: 14px; line-height: 18px; font-style: italic; margin-bottom: 5px;}
#category-posts-2-internal .cat-post-comment-num {font-size: 14px; line-height: 18px;}
#category-posts-2-internal .cat-post-item {border-bottom: 1px solid #ccc;	list-style: none; list-style-type: none; margin: 3px 0;	padding: 3px 0;}
#category-posts-2-internal .cat-post-item:last-child {border-bottom: none;}
#category-posts-2-internal .cat-post-item:after {content: ""; display: table;	clear: both;}
#category-posts-2-internal p {margin:5px 0 0 0}
#category-posts-2-internal li > div {margin:5px 0 0 0; clear:both;}
#category-posts-2-internal .dashicons {vertical-align:middle;}
#category-posts-2-internal .cat-post-crop {overflow: hidden; display:inline-block}
#category-posts-2-internal .cat-post-item img {margin: initial;}
#category-posts-2-internal .cat-post-thumbnail {float:left;}
</style>
		
	
	<script type="text/javascript" >
		function muestraOculta() {
			document.getElementById('reglas').style.display = "none";
			document.getElementById('darian').style.display = "block";
}

  
    </script>
	
	
</head>

<body class="home blog">

	
	<div id="top-menu-wrapper"> <!-- begin top menu -->
    	
        <div class="ribbon-shadow-left">&nbsp;</div>
    
    	<div id="top-menu">
            
            <div class="container  mid">
            
				<div class="menu"><ul id="menu-lista-de-paginas" class="menu"><li id="menu-item-556" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-556"><a href="http://bohemia.cu/quienes-somos/">¿Quiénes somos?</a></li>
<li id="menu-item-555" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-555"><a href="http://bohemia.cu/conctactenos/">Conctáctenos</a></li>
<li id="menu-item-7590" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-7590"><a href="http://bohemia.cu/blogs/">Blogs</a></li>
</ul></div>                
                &nbsp;
                
                <select id="select-menu-top-menu"><option>Page Navigation</option><option   value="http://bohemia.cu/quienes-somos/">¿Quiénes somos?</option><option   value="http://bohemia.cu/conctactenos/">Conctáctenos</option><option   value="http://bohemia.cu/blogs/">Blogs</option></select>                
            </div>
            
            
            
                        
                <div id="search">
                
                    <div class="wrapper">
                    
                        <div class="inner">
                
                            <!-- SEARCH -->  
                            <form method="get" id="searchformtop" action="http://bohemia.cu/">                             
                                <input type="text" value="buscar" onfocus="if (this.value == 'buscar') {this.value = '';}" onblur="if (this.value == '') {this.value = 'buscar';}" name="s" id="s" />          
                            </form>                       
                            
                        </div>
                        
                    </div>
                
                </div>
                
                        
            <br class="clearer" />
        
        </div>
        
        <div class="ribbon-shadow-right">&nbsp;</div>
    
    </div>
	
	<div id="page-wrapper"> <!-- everything below the top menu should be inside the page wrapper div -->
    
    	<div id="logo-bar-wrapper">  <!--begin the main header logo area-->

            <div id="logo-bar">
            
                <div id="logo-wrapper">
                
                    <div id="logo"><!--logo and section header area-->
            
                                                    <a href="http://bohemia.cu/">
                                <img id="site-logo" alt="Revista Bohemia" src="http://bohemia.cu/wp-content/uploads/2016/12/logo-bohemia.jpg" />
                                <img id="site-logo-iphone" alt="Revista Bohemia" src="http://bohemia.cu/wp-content/uploads/2016/12/logo-bohemia-moviles.jpg" />
                                <img id="site-logo-ipad" alt="Revista Bohemia" src="http://bohemia.cu/wp-content/uploads/2016/12/logo-bohemia-moviles.jpg" />
                            </a>
                                                																	
						
					</div>
						
						
						
                    </div>
                    
					
									
                    <br class="clearer" />
										                    
                    <div class="subtitle">Revista cubana de actualidad general fundada en 1908</div>
                    
                </div>  
                
                                    <div id="ad-header">  <!--header ad--> 
                                
                    </div>
                				
					<!-- el manejo de redes sociales-->

                 <!-- social widget by default -->
								<div id="top-widget">
							
									
									
										<div class="top-social">
									
											<a href="/feed/" class="rss" target="_blank" >&nbsp;</a>
											
											<a href="https://www.facebook.com/bohemiadigital1908/" class="facebook" target="_blank" >&nbsp;</a>
											
											<a href="http://twitter.com/bohemiaonline" class="twitter" target="_blank" >&nbsp;</a>
											
											<a href="https:/plus.google.com/108247922473719013486" class="google" target="_blank" >&nbsp;</a>
											
											<a href="https:/pinterest.com/bohemiacuba" class="pinterest" target="_blank" >&nbsp;</a>
											
											<a href="https://www.youtube.com/user/Bohemia1908" class="youtube" target="_blank" >&nbsp;</a>
										
										</div>

                                    <br class="clearer" />


                                    <div id="fecha_actual">

                                        La Habana 17 de Marzo de 2018<br>                                    </div>

								</div>

            	
            
				




				
				
				
				
                <br class="clearer" />
                
			
				
				
		
				
            
            <div id="logo-bar-shadow">&nbsp;</div>
            
        </div> <!--end the logo area -->
            
        <div id="cat-menu" class="cat-menu">
        
        	<a class="home-link" href="http://bohemia.cu">&nbsp;</a>
    
            <ul id="menu-lista-de-categorias" class="menu"><li id="menu-item-357" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-357"><a href="http://bohemia.cu/category/nacionales/">Nacionales</a></li>
<li id="menu-item-359" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-359"><a href="http://bohemia.cu/category/mundo/">Mundo</a></li>
<li id="menu-item-356" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-356"><a href="http://bohemia.cu/category/cultura/">Cultura</a></li>
<li id="menu-item-360" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-360"><a href="http://bohemia.cu/category/deportes/">Deportes</a></li>
<li id="menu-item-358" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-358"><a href="http://bohemia.cu/category/ciencia/">Ciencia</a></li>
<li id="menu-item-362" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-362"><a href="http://bohemia.cu/category/en-cuba/">En Cuba</a></li>
<li id="menu-item-361" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-361"><a href="http://bohemia.cu/category/opinion/">Opinión</a></li>
<li id="menu-item-363" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-363"><a href="http://bohemia.cu/category/historia/">Historia</a></li>
<li id="menu-item-368" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-368"><a href="http://bohemia.cu/category/salud/">Salud</a></li>
<li id="menu-item-45701" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-45701"><a href="http://bohemia.cu/category/multimedia/">Multimedia</a></li>
</ul> 
            
            <select id="select-menu-main-menu"><option>Category Navigation</option><option   value="http://bohemia.cu/category/nacionales/">Nacionales</option><option   value="http://bohemia.cu/category/mundo/">Mundo</option><option   value="http://bohemia.cu/category/cultura/">Cultura</option><option   value="http://bohemia.cu/category/deportes/">Deportes</option><option   value="http://bohemia.cu/category/ciencia/">Ciencia</option><option   value="http://bohemia.cu/category/en-cuba/">En Cuba</option><option   value="http://bohemia.cu/category/opinion/">Opinión</option><option   value="http://bohemia.cu/category/historia/">Historia</option><option   value="http://bohemia.cu/category/salud/">Salud</option><option   value="http://bohemia.cu/category/multimedia/">Multimedia</option></select>                            
            
                            
                    <div id="random-article">
                
                        <a title="Random Article" href="http://bohemia.cu/deportes/2018/01/alazanes-por-alcanzar-su-segundo-titulo-en-el-beisbol-cubano/"><img alt="Random Article" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/random-article.png" width="22" height="22" /></a>
                    
                    </div>
                    
                 
                
                    
        </div> 
        
        <br class="clearer hide-responsive-small" />
        
                
                
            
    
    <div id="dontmiss-bar">
    
    	<div class="ribbon-shadow-left">&nbsp;</div>
    
    	<div id="dontmiss-header">Lo último</div>
        
        <div id="dontmiss-arrow">&nbsp;</div>
            
        <div class="dontmiss" id="dontmiss">    
                    
				
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/">                                
                            Ramiro Valdés aboga en Villa Clara por mayor vínculo entre industria y universidad                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/">                                
                            Inauguran Mercado Mayorista de Alimentos en la capital                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/">                                
                            Fondos de China benefician sectores de la economía cubana                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/">                                
                            Juan Miguel Echevarría ratifica su calidad con registro de 8,40 metros                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="Nominada “Últimos días en La Habana” a los Premios Platino 2018." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/">                                
                            Nominada “Últimos días en La Habana” a los Premios Platino 2018                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/nacionales/2018/03/premiara-calidad-de-postgrados-cubanos-organismo-internacional/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/calidad-de-posgrados-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="Premiará calidad de postgrados cubanos organismo internacional." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/calidad-de-posgrados-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/calidad-de-posgrados-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/calidad-de-posgrados-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/calidad-de-posgrados-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/nacionales/2018/03/premiara-calidad-de-postgrados-cubanos-organismo-internacional/">                                
                            Premiará calidad de postgrados cubanos organismo internacional                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/">                                
                            Inaugura presidente de Kenya embajada de su país en La Habana                                
                        </a> 
                    </div>                  
                    
                </div>
                
            	
        
                <div class="panel">                    
                    
                    <div class="image">
                    	<a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/insomnio-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/insomnio-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/insomnio-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/insomnio-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/insomnio-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a> 
                    </div>
                    <div class="title">                           
                        <a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/">                                
                            ¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?                                
                        </a> 
                    </div>                  
                    
                </div>
                
                    
        </div>
        
        <div id="dontmiss-email" class="signup">
        
        	            
            	<h3>Suscribirse a Bohemia Digital</h3>
                
               <form id="feedburner_subscribe" class="subscribe" action="https://feedburner.google.com/fb/a/mailverify" method="post" target="popupwindow" onsubmit="window.open('https://feedburner.google.com/fb/a/mailverify?uri=RevistaBohemia', 'popupwindow', 'scrollbars=yes,width=550,height=520');return true">
					<div class="email-wrapper">
                        <input type="text" name="email"/>
                        <input type="hidden" value="RevistaBohemia" name="uri"/><input type="hidden" name="loc" value="es_ES"/>
                        <input type="submit" value="Subscribe" /><p>Delivered by <a href="https://feedburner.google.com" target="_blank">FeedBurner</a></p>
                    </div>			
					<input type="image" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/signup.png" class="btn" title="Usted recibir&aacute diariamente un email con nuestro nuevo contenido." onclick="document.feedburner_subscribe.submit();" />
				</form>
                
                <br class="clearer" />
            
                    
        </div>
        
        <br class="clearer" />
    
    </div> <!-- end don't miss posts -->
    

        
                
        <div id="main-wrapper">
        
        <div id="main-wrapper-dark"> <!-- this is only used for the dark skin since it already uses an image for the background texture and light does not -->
        
			            
                <div class="full-width-ad">  
                
                                        
                </div>
            
                        
                        
                        
                <div class="full-width-ad">  
                
                                        
                </div>
            
            

<div class="main-content-left">

	    
    <br class="clearer" />
    
                    
        <div class="left-ad">  
        
                        
        </div>
    
        
        
                    
        <div class="left-ad">  
        
                        
        </div>
    
        
        
                    
        <div class="left-ad">  
        
                        
        </div>
    
        
        
                    
        <div class="left-ad">  
        
                        
        </div>
    
        
        
    	
<!--recuperar  epigrafe---------------------------------------------->





<div class="categorypanels-wrapper">

    <div class="ribbon-shadow-left">&nbsp;</div>
    
    <div class="section-wrapper"> <!-- spotlight section header -->
    
        <div class="section">
        
                    
        </div>        
    
    </div>
    
    <div class="ribbon-shadow-right">&nbsp;</div>   
    
    <div class="section-arrow">&nbsp;</div>
    
    <div class="categorypanels">
    
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Nacionales 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/">Ramiro Valdés aboga en Villa Clara por mayor vínculo entre industria y universidad</a></h2>
                                        
                                        <div class="excerpt">Santa Clara, 16 mar (ACN) El Comandante de la Revolución Ramiro Valdés Menéndez, miembro del Buró Político del Partido y vicepresidente de los Consejos de Estado y de Ministros, destacó hoy la importancia del vínculo de ...</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/">Inauguran Mercado Mayorista de Alimentos en la capital</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/">Fondos de China benefician sectores de la economía cubana</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Mundo 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/embajada-kenya-cuba-1-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/">Inaugura presidente de Kenya embajada de su país en La Habana</a></h2>
                                        
                                        <div class="excerpt">El presidente de Kenya, Uhuru Kenyatta, inauguró hoy la embajada de su país en La Habana, ceremonia a la cual asistió el canciller cubano, Bruno Rodríguez.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/mundo/2018/03/inaugura-presidente-de-kenya-embajada-de-su-pais-en-la-habana/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/mundo/2018/03/denuncia-lavrov-campana-difamatoria-sin-precedentes-contra-rusia/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2016/03/serguei-lavrov-1-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2016/03/serguei-lavrov-1-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2016/03/serguei-lavrov-1-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2016/03/serguei-lavrov-1-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2016/03/serguei-lavrov-1-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/mundo/2018/03/denuncia-lavrov-campana-difamatoria-sin-precedentes-contra-rusia/">Denuncia Lavrov campaña difamatoria sin precedentes contra Rusia</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/mundo/2018/03/denuncia-lavrov-campana-difamatoria-sin-precedentes-contra-rusia/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/mundo/2018/03/peru-parlamento-da-hoy-nuevo-paso-hacia-destitucion-de-presidente/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/peru-parlamento-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Parlamento peruano / PL" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/peru-parlamento-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/peru-parlamento-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/peru-parlamento-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/peru-parlamento-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/mundo/2018/03/peru-parlamento-da-hoy-nuevo-paso-hacia-destitucion-de-presidente/">Perú: Parlamento da hoy nuevo paso hacia destitución de presidente</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/mundo/2018/03/peru-parlamento-da-hoy-nuevo-paso-hacia-destitucion-de-presidente/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Deportes 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/">Juan Miguel Echevarría ratifica su calidad con registro de 8,40 metros</a></h2>
                                        
                                        <div class="excerpt">El saltador de longitud cubano Juan Miguel Echevarría demostró hoy, con un salto de 8,40 metros, que no fue casualidad su título en el pasado Mundial de Atletismo bajo techo.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/deportes/2018/03/ultimo-ser-o-no-ser/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/baloncestoLSBCAP-PRI-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Son muy peligrosas las habaneras que juegan por el perímetro, por causa de su oportuna puntería y respetable eficacia en los contrataques." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/baloncestoLSBCAP-PRI-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/baloncestoLSBCAP-PRI-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/baloncestoLSBCAP-PRI-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/baloncestoLSBCAP-PRI-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/deportes/2018/03/ultimo-ser-o-no-ser/">Último ser o no ser</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/deportes/2018/03/ultimo-ser-o-no-ser/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/deportes/2018/03/campeon-a-las-cuatro-decadas/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/ciclismo-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="El presidente del Inder, Antonio Becali, entrega el trofeo de campeón al espirituano Joel Solenzal. (CALIXTO N. LLANES/JIT)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ciclismo-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/ciclismo-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ciclismo-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ciclismo-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/deportes/2018/03/campeon-a-las-cuatro-decadas/">Campeón a las cuatro décadas</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/deportes/2018/03/campeon-a-las-cuatro-decadas/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Cultura 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Nominada “Últimos días en La Habana” a los Premios Platino 2018." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/">Nominada “Últimos días en La Habana” a los Premios Platino 2018</a></h2>
                                        
                                        <div class="excerpt">Por SAYLIN HERNÁNDEZ TORRES Últimos días en La Habana, largometraje del reconocido cineasta cubano Fernando Pérez, resultó nominada en siete categorías de los V Premios Platino del Cine Iberoamericano, según la agencia i...</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/cultura/2018/03/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/gentezona-pausini-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/gentezona-pausini-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/gentezona-pausini-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/gentezona-pausini-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/gentezona-pausini-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/cultura/2018/03/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/">Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/cultura/2018/03/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/cultura/2018/03/cartelera-de-la-tv-cubana-302/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/cultura/2018/03/cartelera-de-la-tv-cubana-302/">Cartelera de la TV Cubana</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/cultura/2018/03/cartelera-de-la-tv-cubana-302/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Salud 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/insomnio-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/">¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?</a></h2>
                                        
                                        <div class="excerpt">Dormir poco o dormir mal, así como reducir las horas de sueño necesario, puede alterar el sistema hormonal y el metabolismo, favoreciendo el desarrollo de diabetes, obesidad y enfermedades cardiovasculares</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/salud/2018/03/obesidad-complica-tratamiento-de-cancer-de-mama/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/obesidad-cancer-mama-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/obesidad-cancer-mama-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/obesidad-cancer-mama-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/obesidad-cancer-mama-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/obesidad-cancer-mama-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/salud/2018/03/obesidad-complica-tratamiento-de-cancer-de-mama/">Obesidad complica tratamiento de cáncer de mama</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/salud/2018/03/obesidad-complica-tratamiento-de-cancer-de-mama/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/salud/2018/03/expertos-de-mas-de-80-paises-en-convencion-internacional-de-salud/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/salud-en-cuba-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Expertos de más de 80 países en Convención Internacional de Salud." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/salud-en-cuba-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/salud-en-cuba-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/salud-en-cuba-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/salud-en-cuba-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/salud/2018/03/expertos-de-mas-de-80-paises-en-convencion-internacional-de-salud/">Expertos de más de 80 países en Convención Internacional de Salud</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/salud/2018/03/expertos-de-mas-de-80-paises-en-convencion-internacional-de-salud/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Ciencia 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/ciencia/2018/03/son-realmente-peligrosas-las-aranas/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/arana-errante-brasilena-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Araña Errante Brasileña. Foto: animalesextremos.com" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/ciencia/2018/03/son-realmente-peligrosas-las-aranas/">¿Son realmente peligrosas las arañas?</a></h2>
                                        
                                        <div class="excerpt"> Las arañas causan fobia y se las percibe como un riesgo para la salud humana, aunque de las más de 47.000 especies conocidas solo 175 podrían ser peligrosas</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/ciencia/2018/03/son-realmente-peligrosas-las-aranas/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/ciencia/2018/03/son-realmente-peligrosas-las-aranas/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/ciencia/2018/03/stephen-hawking-el-vencedor-el-genio-que-estara-siempre/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/stephen-genio-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Stephen wauking, el genio. (Foto: muyinteresante.com.mx" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/stephen-genio-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/stephen-genio-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/stephen-genio-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/stephen-genio-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/ciencia/2018/03/stephen-hawking-el-vencedor-el-genio-que-estara-siempre/">Stephen Hawking, el vencedor, el genio que estará siempre</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/ciencia/2018/03/stephen-hawking-el-vencedor-el-genio-que-estara-siempre/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/ciencia/2018/03/murio-stephen-hawking-autor-de-la-teoria-del-big-bang/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2017/11/Stephen-Hawking-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2017/11/Stephen-Hawking-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2017/11/Stephen-Hawking-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2017/11/Stephen-Hawking-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2017/11/Stephen-Hawking-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/ciencia/2018/03/murio-stephen-hawking-autor-de-la-teoria-del-big-bang/">Murió Stephen Hawking, autor de la teoría del big bang</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/ciencia/2018/03/murio-stephen-hawking-autor-de-la-teoria-del-big-bang/#comments">2 comentarios</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Tecnología 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-300x170.jpg 300w, http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-230x131.jpg 230w, http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-190x108.jpg 190w, http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-180x103.jpg 180w, http://bohemia.cu/wp-content/uploads/2017/02/television-digital-cuba-350x200.jpg 350w" sizes="(max-width: 300px) 100vw, 300px" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/">Comenzará en julio el apagón parcial de la TV analógica (+ video)</a></h2>
                                        
                                        <div class="excerpt">Especialistas cubanos coordinan hoy los últimos detalles para iniciar en julio próximo el paso hacia la televisión digital con el denominado apagón parcial de la señal analógica del canal Educativo.
</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/tecnologia/2018/03/restauran-sensacion-de-movimiento-en-pacientes-amputados/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/restaurar-sensacion-mov-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Descubren la forma de restaurar la sensación de movimiento en pacientes amputados. (Foto: ABC.es)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/restaurar-sensacion-mov-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/restaurar-sensacion-mov-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/restaurar-sensacion-mov-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/restaurar-sensacion-mov-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/tecnologia/2018/03/restauran-sensacion-de-movimiento-en-pacientes-amputados/">Restauran sensación de movimiento en pacientes amputados</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/tecnologia/2018/03/restauran-sensacion-de-movimiento-en-pacientes-amputados/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/tecnologia/2018/03/despiden-a-un-robot-chef-y-sabes-por-que/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/robot-flippy-cocina-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/robot-flippy-cocina-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/robot-flippy-cocina-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/robot-flippy-cocina-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/robot-flippy-cocina-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/tecnologia/2018/03/despiden-a-un-robot-chef-y-sabes-por-que/">&#8216;Despiden&#8217; a un robot-chef y ¿sabes por qué?</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/tecnologia/2018/03/despiden-a-un-robot-chef-y-sabes-por-que/#comments">1 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Medio ambiente 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/medio-ambiente/2018/03/acidificacion-del-mar-amenaza-a-barreras-de-coral/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/acidificacion-barrera-coral-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="La acidificación del mar amenaza una tercera parte de las barreras de coral | La Prensa" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/acidificacion-barrera-coral-300x170.jpg 300w, http://bohemia.cu/wp-content/uploads/2018/03/acidificacion-barrera-coral-180x101.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/medio-ambiente/2018/03/acidificacion-del-mar-amenaza-a-barreras-de-coral/">Acidificación del mar amenaza a barreras de coral</a></h2>
                                        
                                        <div class="excerpt">Un estudio científico ha determinado que  el incremento de la acidificación del océano podría destruir una tercera parte de las barreras de coral y comprometer su futuro</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/medio-ambiente/2018/03/acidificacion-del-mar-amenaza-a-barreras-de-coral/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/medio-ambiente/2018/03/acidificacion-del-mar-amenaza-a-barreras-de-coral/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/medio-ambiente/2018/03/vinculan-olas-de-frio-con-las-altas-temperaturas-en-el-artico/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/nieve-LLUVIAS-EEUU-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Tormenta de nieve y lluvia en EEUU/PL" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/nieve-LLUVIAS-EEUU-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/nieve-LLUVIAS-EEUU-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/nieve-LLUVIAS-EEUU-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/nieve-LLUVIAS-EEUU-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/medio-ambiente/2018/03/vinculan-olas-de-frio-con-las-altas-temperaturas-en-el-artico/">Vinculan olas de frío con las altas temperaturas en el Ártico</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/medio-ambiente/2018/03/vinculan-olas-de-frio-con-las-altas-temperaturas-en-el-artico/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/medio-ambiente/2018/03/hallan-mineral-nunca-visto-dentro-de-un-diamante/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/diamante-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Diamante, Foto RT" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/diamante-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/diamante-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/diamante-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/diamante-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/medio-ambiente/2018/03/hallan-mineral-nunca-visto-dentro-de-un-diamante/">Hallan mineral nunca visto dentro de un diamante</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/medio-ambiente/2018/03/hallan-mineral-nunca-visto-dentro-de-un-diamante/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Historia 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/historia/2018/03/cuando-la-capitulacion-devino-tregua/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/maceo-baragua-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="José Martí calificó a la Protesta entre los más glorioso de la historia de Cuba. (Autor no identificado)" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                        <h5 id='epigrafe' > CUBA 1878 </h5>                                                               
                                        <h2><a href="http://bohemia.cu/historia/2018/03/cuando-la-capitulacion-devino-tregua/">Cuando la capitulación devino tregua</a></h2>
                                        
                                        <div class="excerpt">Con su gesto, Maceo trastornaba de un golpe la Historia de Cuba y a la vez, salvaba el prestigio de los Hombres del 68</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/historia/2018/03/cuando-la-capitulacion-devino-tregua/#comments">1 comentario</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/historia/2018/03/cuando-la-capitulacion-devino-tregua/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/historia/2018/03/marca-indeleble-en-el-calendario-de-la-historia/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/13marzo-radioreloj-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Asalto a Rafio Reloj. José Antonio Echerría/ radioreloj.cu" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/13marzo-radioreloj-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/13marzo-radioreloj-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/13marzo-radioreloj-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/13marzo-radioreloj-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/historia/2018/03/marca-indeleble-en-el-calendario-de-la-historia/">Marca indeleble en el calendario de la Historia</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/historia/2018/03/marca-indeleble-en-el-calendario-de-la-historia/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/historia/2018/03/espiritu-vivo-de-la-revolucion-cubana/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/raulcastro-segundofrente-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Raúl Castro. Segundo Frente Oriental Frank País/ Foto: CMKC" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/raulcastro-segundofrente-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/raulcastro-segundofrente-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/raulcastro-segundofrente-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/raulcastro-segundofrente-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/historia/2018/03/espiritu-vivo-de-la-revolucion-cubana/">Espíritu vivo de la Revolución cubana</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/historia/2018/03/espiritu-vivo-de-la-revolucion-cubana/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Gazapos 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/gazapos/2018/03/cada-culpa-sobre-el-culpable/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/hitler-mandela-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Hitler, inició la Segunda Guerra Mundial y Mandela estuvo sometido a prisión por los racistas sudafricanos." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/gazapos/2018/03/cada-culpa-sobre-el-culpable/">Cada culpa sobre el culpable</a></h2>
                                        
                                        <div class="excerpt">Hace un tiempo el lector Francisco Antonio Rovira Tray, de Las Tunas, me informó  que un periódico había incurrido en lo que clasifiqué como un gazapo histórico, pero el mismo medio vuelve a equivocarse...</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/gazapos/2018/03/cada-culpa-sobre-el-culpable/#comments">2 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/gazapos/2018/03/cada-culpa-sobre-el-culpable/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/gazapos/2018/03/boludos-y-otros/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/boludos-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Boludos/ Taringa" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/boludos-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/boludos-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/boludos-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/boludos-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/gazapos/2018/03/boludos-y-otros/">¡Boludos! y otros&#8230;</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/gazapos/2018/03/boludos-y-otros/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/gazapos/2018/02/la-cama-es-para/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/cama-amor-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/cama-amor-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/cama-amor-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/cama-amor-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/cama-amor-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/gazapos/2018/02/la-cama-es-para/">La cama es para&#8230;</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/gazapos/2018/02/la-cama-es-para/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            ¿Sabías? 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/bebes-razonamientos-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Pues sí…, los bebés pueden razonar antes de empezar a hablar." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/">Pues sí…, los bebés pueden razonar antes de empezar a hablar</a></h2>
                                        
                                        <div class="excerpt">Es el resultado de un estudio que pretendía averiguar en qué momento del desarrollo se ponen en marcha los mecanismos que permiten el razonamiento y su relación con el lenguaje.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/sabias/2018/03/saliva-genetica-empatia-y-autismo/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/genetica-y-empatia-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Genética explica en mucho la empatía que sentimos con otras personas. (Foto: ojo.pe)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/genetica-y-empatia-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/genetica-y-empatia-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/genetica-y-empatia-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/genetica-y-empatia-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/sabias/2018/03/saliva-genetica-empatia-y-autismo/">Saliva, genética, empatía y autismo</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/sabias/2018/03/saliva-genetica-empatia-y-autismo/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/sabias/2018/03/onu-finlandeses-los-mas-felices-y-estadounidenses-mas-descontentos/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/felicidad-finlad-eeuu-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/felicidad-finlad-eeuu-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/felicidad-finlad-eeuu-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/felicidad-finlad-eeuu-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/felicidad-finlad-eeuu-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/sabias/2018/03/onu-finlandeses-los-mas-felices-y-estadounidenses-mas-descontentos/">ONU: finlandeses los más felices y estadounidenses más descontentos</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/sabias/2018/03/onu-finlandeses-los-mas-felices-y-estadounidenses-mas-descontentos/#comments">1 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Extraño 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/extrano/2018/03/atender-mas-al-smartphone-que-a-la-persona-amada/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/prioriza-mas-smarthphone-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Prioriza más el smartphone que la persona amada/ Foto: b17.ru" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/extrano/2018/03/atender-mas-al-smartphone-que-a-la-persona-amada/">¿Atender más al smartphone que a la persona amada?</a></h2>
                                        
                                        <div class="excerpt">Parece increible, pero sì:  a 10 años de acompañar a sus usuarios y  facilitarles la inmediatez en la comunicación, el smarthphone también está enajenando la vida de las personas, familias y amigos</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/extrano/2018/03/atender-mas-al-smartphone-que-a-la-persona-amada/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/extrano/2018/03/atender-mas-al-smartphone-que-a-la-persona-amada/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/extrano/2018/03/bombones-de-oro-y-el-mundo-al-reves/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/bombón-oro-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Un bombón portugués elaborado con oro es el más caro del mundo y vale 9 mil 560 dólares. (Foto: El Siglo)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/bombón-oro-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/bombón-oro-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/bombón-oro-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/bombón-oro-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/extrano/2018/03/bombones-de-oro-y-el-mundo-al-reves/">Bombones de oro&#8230; y el mundo al revés</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/extrano/2018/03/bombones-de-oro-y-el-mundo-al-reves/#comments">2 comentarios</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/extrano/2018/03/cuidado-cancer-por-ver-television/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/ver-tv-cancer-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Pasar mucho tiempo viendo la tv provoca avance en el cáncer de colon, según científicos. ( Foto: laverdadnoticias.com)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ver-tv-cancer-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/ver-tv-cancer-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ver-tv-cancer-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ver-tv-cancer-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/extrano/2018/03/cuidado-cancer-por-ver-television/">Cuidado: ¡Cáncer por ver televisión!</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/extrano/2018/03/cuidado-cancer-por-ver-television/#comments">1 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Le contesta Bohemia 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/le-contesta-bohemia/2018/03/cerebro-agradecido/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/cerebro-ensaladas-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Cerebro agradecido." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/le-contesta-bohemia/2018/03/cerebro-agradecido/">Cerebro agradecido</a></h2>
                                        
                                        <div class="excerpt">El envejecimiento es inevitable y con él llega un acortamiento de los telómeros que son regiones de ADN, estabilizadoras de la estructura de los cromosomas en las células.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/le-contesta-bohemia/2018/03/cerebro-agradecido/#comments">1 comentario</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/le-contesta-bohemia/2018/03/cerebro-agradecido/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/le-contesta-bohemia/2018/03/con-fragancia/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/echar-perfumes-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Con fragancia." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/echar-perfumes-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/echar-perfumes-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/echar-perfumes-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/echar-perfumes-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/le-contesta-bohemia/2018/03/con-fragancia/">Con fragancia</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/le-contesta-bohemia/2018/03/con-fragancia/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/le-contesta-bohemia/2018/02/gripe-masculina/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/influenza-hombres-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/influenza-hombres-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/influenza-hombres-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/influenza-hombres-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/influenza-hombres-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/le-contesta-bohemia/2018/02/gripe-masculina/">¡Gripe masculina!</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/le-contesta-bohemia/2018/02/gripe-masculina/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Mi veterinario 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/mi-veterinario/2018/03/practicar-la-eutanasia-piadosa/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/perro-mantasnuestra3-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Perro enfermo/ Eroski Consumer" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/mi-veterinario/2018/03/practicar-la-eutanasia-piadosa/">¿Practicar la eutanasia piadosa?</a></h2>
                                        
                                        <div class="excerpt">Daniel damovi@nauta.cu. CIENFUEGOS: La propuesta de eutanasia piadosa nunca vendrá de su veterinario, que luchará sin descanso por salvarla</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/mi-veterinario/2018/03/practicar-la-eutanasia-piadosa/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/mi-veterinario/2018/03/practicar-la-eutanasia-piadosa/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/mi-veterinario/2018/03/el-mejor-amigo-del-hombre-y-del-arte/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/perros-ratreadores-arte-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="(Perros rastreadores de arte/RT)" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/perros-ratreadores-arte-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/perros-ratreadores-arte-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/perros-ratreadores-arte-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/perros-ratreadores-arte-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/mi-veterinario/2018/03/el-mejor-amigo-del-hombre-y-del-arte/">El mejor amigo del hombre.. ¿y del arte?</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/mi-veterinario/2018/03/el-mejor-amigo-del-hombre-y-del-arte/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/mi-veterinario/2018/02/por-que-los-perros-se-suicidan-en-el-puente-de-overtoun-bridge/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/puente-perros-suicidios-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/puente-perros-suicidios-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/puente-perros-suicidios-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/puente-perros-suicidios-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/puente-perros-suicidios-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/mi-veterinario/2018/02/por-que-los-perros-se-suicidan-en-el-puente-de-overtoun-bridge/">¿Por qué los perros se suicidan en el puente de Overtoun Bridge?</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/mi-veterinario/2018/02/por-que-los-perros-se-suicidan-en-el-puente-de-overtoun-bridge/#comments">3 comentarios</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Consejos 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/consejos/2018/03/como-el-ruido-afecta-a-nuestra-salud/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/ruidos-danos-salud-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Cómo el ruido afecta a nuestra salud." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/consejos/2018/03/como-el-ruido-afecta-a-nuestra-salud/">Cómo el ruido afecta a nuestra salud</a></h2>
                                        
                                        <div class="excerpt">Las bocinas de los carros, la música alta, las alarmas, las construcciones, los timbres, los silbidos…, forman parte del ambiente de las ciudades y lo peor de todo es que nos estamos acostumbrando a ello.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/consejos/2018/03/como-el-ruido-afecta-a-nuestra-salud/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/consejos/2018/03/como-el-ruido-afecta-a-nuestra-salud/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/consejos/2018/03/es-bueno-que-los-ninos-se-ensucien/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/nino-sucio-juego-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Es bueno que los niños se ensucien." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/nino-sucio-juego-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/nino-sucio-juego-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/nino-sucio-juego-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/nino-sucio-juego-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/consejos/2018/03/es-bueno-que-los-ninos-se-ensucien/">Es bueno que los niños se ensucien</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/consejos/2018/03/es-bueno-que-los-ninos-se-ensucien/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/consejos/2018/02/los-ninos-y-las-tareas-domesticas/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/organizar-la-ropa-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Los niños y las tareas domésticas." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/organizar-la-ropa-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/organizar-la-ropa-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/organizar-la-ropa-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/organizar-la-ropa-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/consejos/2018/02/los-ninos-y-las-tareas-domesticas/">Los niños y las tareas domésticas</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/consejos/2018/02/los-ninos-y-las-tareas-domesticas/#comments">2 comentarios</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Cocina 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/cocina/2018/03/picadillo/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/picadillo-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="Picadillo." title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/cocina/2018/03/picadillo/">Picadillo</a></h2>
                                        
                                        <div class="excerpt">Las carnes molidas condimentadas y con su salsita, “para mojar el arroz”, nos gustan a casi todos los cubanos. Igual sucede con la carne con papas o con el bistec en cazuela que tanto deleite proporcionan al paladar.</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/cocina/2018/03/picadillo/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/cocina/2018/03/picadillo/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/cocina/2018/03/natilla-criolla-con-vainilla/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/natilla-vainilla-cubana-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Natilla criolla con vainilla." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/natilla-vainilla-cubana-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/natilla-vainilla-cubana-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/natilla-vainilla-cubana-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/natilla-vainilla-cubana-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/cocina/2018/03/natilla-criolla-con-vainilla/">Natilla criolla con vainilla</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/cocina/2018/03/natilla-criolla-con-vainilla/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/cocina/2018/03/pollo-hawaiano/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/pollo-hawaiano-1-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="Pollo hawaiano." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/pollo-hawaiano-1-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/pollo-hawaiano-1-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/pollo-hawaiano-1-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/pollo-hawaiano-1-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/cocina/2018/03/pollo-hawaiano/">Pollo hawaiano</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/cocina/2018/03/pollo-hawaiano/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
                        
                <div class="categorypanel">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Crucigramas 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-6-del-2018/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/crucigrama-6-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-6-del-2018/">Crucigrama edición 6 del 2018</a></h2>
                                        
                                        <div class="excerpt">Clic para resolverlo on line &nbsp; Descargar PDF Crucigrama 6/2018    &nbsp; Respuestas crucigrama 6 del 2018</div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-6-del-2018/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-6-del-2018/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-5-del-2018/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/crucigrama5-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/crucigrama5-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/crucigrama5-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/crucigrama5-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/crucigrama5-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-5-del-2018/">Crucigrama edición 5 del 2018</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/crucigramas/2018/03/crucigrama-edicion-5-del-2018/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/crucigramas/2018/02/crucigrama-edicion-4-del-2018/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/02/crucigrama4-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/02/crucigrama4-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/02/crucigrama4-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/02/crucigrama4-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/02/crucigrama4-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/crucigramas/2018/02/crucigrama-edicion-4-del-2018/">Crucigrama edición 4 del 2018</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/crucigramas/2018/02/crucigrama-edicion-4-del-2018/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                        
                        
                        
                <div class="categorypanel right">
                
                                                
                                                            
                                <div class="vertical"> 
                                
                                	<div class="category"> 
                    
                                        <div class="ribbon-shadow-left">&nbsp;</div>
                                    
                                         
                                        
                                        <div class="catname">
                                                   
                                            Humor 
                                            
                                        </div> 
                                        
                                        <div class="category-arrow">&nbsp;</div> 
                                                 
                                    </div>
                        
                                    <a class="darken" href="http://bohemia.cu/humor/2018/03/humor-edicion-6-del-2018/"><img width="300" height="170" src="http://bohemia.cu/wp-content/uploads/2018/03/humor1-1-300x170.jpg" class="attachment-spotlight size-spotlight wp-post-image" alt="" title="" /></a>       
                                    
                                    <div class="inner">

                                        
                                        
                                                                                                       
                                        <h2><a href="http://bohemia.cu/humor/2018/03/humor-edicion-6-del-2018/">Humor edición 6 del 2018</a></h2>
                                        
                                        <div class="excerpt"></div>

										<div class="more-bar">
                                            
                                             
                                            
                                                                                            
                                                <div class="comments">
                                                
                                                    <a href="http://bohemia.cu/humor/2018/03/humor-edicion-6-del-2018/#respond">0 comentarios</a>                                                
                                                </div>
                                                
                                                                                        
                                            <div class="more"><a href="http://bohemia.cu/humor/2018/03/humor-edicion-6-del-2018/">M&aacute;s</a></div>
                                        
                                        </div>
                                        <br class="clearer" />
                                        
                                    </div>
                                
                                </div>
                                
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-3/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/humor3-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/humor3-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/humor3-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/humor3-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/humor3-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-3/">Humor edición 5 del 2018</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-3/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                    
                                                        
                            	<div class="post-panel-wrapper">
                            
									                                
                                    <div class="post-panel"> 
                            
                                        <div class="post-thumbnail">
                                        
                                            <a class="darken small" href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-4/"><img width="70" height="70" src="http://bohemia.cu/wp-content/uploads/2018/03/humor4-70x70.jpg" class="attachment-widget-thumbnail size-widget-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/humor4-70x70.jpg 70w, http://bohemia.cu/wp-content/uploads/2018/03/humor4-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/humor4-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/humor4-40x40.jpg 40w" sizes="(max-width: 70px) 100vw, 70px" /></a>
                                        
                                        </div>
                                           
                                        <div class="post-info">  
                                                              
                                            <h3><a href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-4/">Humor edición 5 del 2018</a></h3>
                                            
                                            <div class="post-meta">
                                            
                                                                                                
                                                                                                
                                                    <div class="comments">
                                                    
                                                        <a href="http://bohemia.cu/humor/2018/03/humor-edicion-5-del-2018-4/#respond">0 comentario</a>                                                    
                                                    </div>
                                                    
                                                                                                
                                                <div class="clearer"></div>
                                            
                                            </div>   
                                        
                                        </div>                            
             
                                        <div class="clearer"></div>
                                    
                                    </div>
                                    
                                </div>
                            
                               
                            
                                                
                     
                    
                </div> <!-- end categorypanels section -->
                        
                        
                                            
                <div class="clearer non-responsive"></div>
        
                        
                                                
                <div class="clear-responsive">&nbsp;</div>
        
                        
            
    </div>
    
</div>

<br class="clearer" />

        
        
                    
        <div class="left-ad">  
        
                        
        </div>
    
        
        
        
    <div id="homepage-widgets">

	    
    	
    
    
</div>
</div>


	<div class="sidebar">

		<div class="widget-wrapper"><div class="widget"><h1 class="widget-title srp-widget-title"><a class="srp-widget-title-link" href="http://bohemia.cu/category/ediciones-impresas/" title="Ediciones impresas">Ediciones impresas</a></h1><!-- BEGIN Special Recent Posts FREE Edition v2.0.4 --><div class="srp-widget-container"><div id="wdg_specialrecentpostsfree-37-srp-singlepost-1" class="srp-widget-singlepost"><div class="srp-post-content-container"><div class="srp-thumbnail-box"><a class="srp-post-thumbnail-link" href="http://bohemia.cu/ediciones-impresas/2018/03/portada-edicion-no-6-del-2018/" title="Portada edici&oacute;n No.6 del 2018"><img src="http://bohemia.cu/wp-content/plugins/special-recent-posts/cache/srpthumb-p106265-240x300-no.jpg" class="srp-post-thumbnail" alt="Portada edici&oacute;n No.6 del 2018" /></a></div><div class="srp-content-box"><h3 class="srp-post-title"><a class="srp-post-title-link" href="http://bohemia.cu/ediciones-impresas/2018/03/portada-edicion-no-6-del-2018/" title="Portada edici&oacute;n No.6 del 2018">Portada edici&oacute;n No.6 del 2018</a></h3></div></div></div></div><!-- END Special Recent Posts FREE Edition v2.0.4 --></div></div><div class="widget-wrapper"><div class="widget">        	
        <div id="tabbed-posts" class="complex-list small">
            <ul class="tabnav">
				<li><a href="#tabs-recent">Recientes</a></li>				
								
				
                
                <li><a href="#tabs-comments">Comentarios</a></li>                            </ul>
            <br class="clearer" />
            <div class="tabdiv-wrapper">
        
        		
				
                
                            
                    <div id="tabs-recent" class="tabdiv">
                        <ul>
                                                        
								<li>
                                    
                                    <div class="floatleft">
                                
                                    	<a href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/" class="thumbnail darken small" title="Ramiro Valdés aboga en Villa Clara por mayor vínculo entre industria y universidad"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ramiro-planta-mecanica-vc-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a>				 
                                    </div>
                                    
                                    <div class="floatleft">
                                    
                                    	<a class="post-title wide" href="http://bohemia.cu/nacionales/2018/03/ramiro-valdes-aboga-en-villa-clara-por-mayor-vinculo-entre-industria-y-universidad/" title="Ramiro Valdés aboga en Villa Clara por mayor vínculo entre industria y universidad">Ramiro Valdés aboga en Villa Clara por mayor vínculo entre industria y universidad</a>                      
                                        
                                                                            
										                                    
                                    </div>
                                    
                                    <br class="clearer" />
									
								</li>
                                                            
								<li>
                                    
                                    <div class="floatleft">
                                
                                    	<a href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/" class="thumbnail darken small" title="Inauguran Mercado Mayorista de Alimentos en la capital"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/mercabal-ofertas-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a>				 
                                    </div>
                                    
                                    <div class="floatleft">
                                    
                                    	<a class="post-title wide" href="http://bohemia.cu/nacionales/2018/03/inauguran-mercado-mayorista-de-alimentos-en-la-capital/" title="Inauguran Mercado Mayorista de Alimentos en la capital">Inauguran Mercado Mayorista de Alimentos en la capital</a>                      
                                        
                                                                            
										                                    
                                    </div>
                                    
                                    <br class="clearer" />
									
								</li>
                                                            
								<li>
                                    
                                    <div class="floatleft">
                                
                                    	<a href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/" class="thumbnail darken small" title="Fondos de China benefician sectores de la economía cubana"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/actas-cuba-china-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a>				 
                                    </div>
                                    
                                    <div class="floatleft">
                                    
                                    	<a class="post-title wide" href="http://bohemia.cu/nacionales/2018/03/fondos-de-china-benefician-sectores-de-la-economia-cubana/" title="Fondos de China benefician sectores de la economía cubana">Fondos de China benefician sectores de la economía cubana</a>                      
                                        
                                                                            
										                                    
                                    </div>
                                    
                                    <br class="clearer" />
									
								</li>
                                                            
								<li>
                                    
                                    <div class="floatleft">
                                
                                    	<a href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/" class="thumbnail darken small" title="Juan Miguel Echevarría ratifica su calidad con registro de 8,40 metros"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="" title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/Juan-Miguel-Echevarria-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a>				 
                                    </div>
                                    
                                    <div class="floatleft">
                                    
                                    	<a class="post-title wide" href="http://bohemia.cu/deportes/2018/03/juan-miguel-echevarria-ratifica-su-calidad-con-registro-de-840-metros/" title="Juan Miguel Echevarría ratifica su calidad con registro de 8,40 metros">Juan Miguel Echevarría ratifica su calidad con registro de 8,40 metros</a>                      
                                        
                                                                            
										                                    
                                    </div>
                                    
                                    <br class="clearer" />
									
								</li>
                                                            
								<li>
                                    
                                    <div class="floatleft">
                                
                                    	<a href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/" class="thumbnail darken small" title="Nominada “Últimos días en La Habana” a los Premios Platino 2018"><img width="40" height="40" src="http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-40x40.jpg" class="attachment-footer-thumbnail size-footer-thumbnail wp-post-image" alt="Nominada “Últimos días en La Habana” a los Premios Platino 2018." title="" srcset="http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-40x40.jpg 40w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-150x150.jpg 150w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-160x160.jpg 160w, http://bohemia.cu/wp-content/uploads/2018/03/ultimos-dias-en-habana-70x70.jpg 70w" sizes="(max-width: 40px) 100vw, 40px" /></a>				 
                                    </div>
                                    
                                    <div class="floatleft">
                                    
                                    	<a class="post-title wide" href="http://bohemia.cu/cultura/2018/03/nominada-ultimos-dias-en-la-habana-a-los-premios-platino-2018/" title="Nominada “Últimos días en La Habana” a los Premios Platino 2018">Nominada “Últimos días en La Habana” a los Premios Platino 2018</a>                      
                                        
                                                                            
										                                    
                                    </div>
                                    
                                    <br class="clearer" />
									
								</li>
                             
                                                        <li class="last">&nbsp;</li>  
                        </ul>
                    </div>
                    
                				
				
								
				
                
                            
                    <div id="tabs-comments" class="tabdiv">
                        <ul>
                                                    <li><a class="first" href="http://bohemia.cu/nacionales/2018/03/flor-del-canaveral/#comment-35813">"Más de ellas es lo que necesitamos en nuestra fuerza de trabajo felicidades."<span> -&nbsp;ZacK639</span></a></li>
                                                    <li><a href="http://bohemia.cu/mundo/2018/03/a-unidad-mundial-a-favor-de-la-paz-llaman-desde-cuba/#comment-35806">"Saludos de paz y bendiciones desde México. Muchas gracias amigos, compañeros y hermanas cubanas por ayuda..."<span> -&nbsp;Fernando Acosta Riveros</span></a></li>
                                                    <li><a href="http://bohemia.cu/nacionales/2018/03/flor-del-canaveral/#comment-35804">"Felicidades a esta Mujer. Es verdad que somos fuertes"<span> -&nbsp;BDPZ</span></a></li>
                                                    <li><a href="http://bohemia.cu/ciencia/2018/03/murio-stephen-hawking-autor-de-la-teoria-del-big-bang/#comment-35803">"Carlos, que en la era de la información andemos tan desinformados, qué contradictorio, no?. Eso mismo dij..."<span> -&nbsp;Makusa</span></a></li>
                                                    <li><a href="http://bohemia.cu/gazapos/2018/03/cada-culpa-sobre-el-culpable/#comment-35802">"Qué está pasando con esos periodistas? Equivocarse es de humanos..., pero rectificar es de sabios, más c..."<span> -&nbsp;Makusa</span></a></li>
                                                <li class="last">&nbsp;</li>  
                        </ul>
                    </div> 
                    
                                
                            
            </div>
                                     
        </div>
        
        </div></div>        
    
<!-- WordPress Popular Posts Plugin [W] [daily] [views] [regular] -->

<div class="widget-wrapper"><div class="widget">
<div class="section-wrapper"><div class="section">Lo más consultado hoy</div></div>
<ul class="wpp-list wpp-list-with-thumbnails">
<li>
<a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/" title="¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?" target="_self"><img src="http://bohemia.cu/wp-content/uploads/wordpress-popular-posts/107162-featured-75x75.jpg" width="75" height="75" alt="¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?" class="wpp-thumbnail wpp_cached_thumb wpp_featured" /></a>
<a href="http://bohemia.cu/salud/2018/03/sabes-que-los-malos-habitos-del-sueno-pueden-provocar-enfermedades/" title="¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?" class="wpp-post-title" target="_self">¿Sabes que los malos hábitos del sueño pueden provocar enfermedades?</a>
</li>
<li>
<a href="http://bohemia.cu/cultura/2018/03/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/" title="Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)" target="_self"><img src="http://bohemia.cu/wp-content/uploads/wordpress-popular-posts/107144-featured-75x75.jpg" width="75" height="75" alt="Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)" class="wpp-thumbnail wpp_cached_thumb wpp_featured" /></a>
<a href="http://bohemia.cu/cultura/2018/03/gente-de-zona-agradece-a-sus-seguidores-exito-junto-a-laura-pausini-video/" title="Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)" class="wpp-post-title" target="_self">Gente de Zona agradece a sus seguidores éxito junto a Laura Pausini (+Video)</a>
</li>
<li>
<a href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/" title="Pues sí…, los bebés pueden razonar antes de empezar a hablar" target="_self"><img src="http://bohemia.cu/wp-content/uploads/wordpress-popular-posts/107150-featured-75x75.jpg" width="75" height="75" alt="Pues sí…, los bebés pueden razonar antes de empezar a hablar" class="wpp-thumbnail wpp_cached_thumb wpp_featured" /></a>
<a href="http://bohemia.cu/sabias/2018/03/pues-si-los-bebes-pueden-razonar-antes-de-empezar-a-hablar/" title="Pues sí…, los bebés pueden razonar antes de empezar a hablar" class="wpp-post-title" target="_self">Pues sí…, los bebés pueden razonar antes de empezar a hablar</a>
</li>
<li>
<a href="http://bohemia.cu/cocina/2018/03/picadillo/" title="Picadillo" target="_self"><img src="http://bohemia.cu/wp-content/uploads/wordpress-popular-posts/107176-featured-75x75.jpg" width="75" height="75" alt="Picadillo" class="wpp-thumbnail wpp_cached_thumb wpp_featured" /></a>
<a href="http://bohemia.cu/cocina/2018/03/picadillo/" title="Picadillo" class="wpp-post-title" target="_self">Picadillo</a>
</li>
<li>
<a href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/" title="Comenzará en julio el apagón parcial de la TV analógica (+ video)" target="_self"><img src="http://bohemia.cu/wp-content/uploads/wordpress-popular-posts/107142-featured-75x75.jpg" width="75" height="75" alt="Comenzará en julio el apagón parcial de la TV analógica (+ video)" class="wpp-thumbnail wpp_cached_thumb wpp_featured" /></a>
<a href="http://bohemia.cu/tecnologia/2018/03/comenzara-en-julio-el-apagon-parcial-de-la-tv-analogica/" title="Comenzará en julio el apagón parcial de la TV analógica (+ video)" class="wpp-post-title" target="_self">Comenzará en julio el apagón parcial de la TV analógica (+ video)</a>
</li>
</ul>

</div></div>
<div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Especiales de Bohemia</div></div><!-- BEGIN Special Recent Posts FREE Edition v2.0.4 --><div class="srp-widget-container"><div id="wdg_specialrecentpostsfree-36-srp-singlepost-1" class="srp-widget-singlepost"><div class="srp-post-content-container"><div class="srp-thumbnail-box"><a class="srp-post-thumbnail-link" href="http://bohemia.cu/en-cuba/2018/02/peligro-que-nos-afecta-a-todos/" title="CORRUPCI&Oacute;N: Peligro que nos afecta a todos"><img src="http://bohemia.cu/wp-content/plugins/special-recent-posts/cache/srpthumb-p104592-240x350-no.jpg" class="srp-post-thumbnail" alt="CORRUPCI&Oacute;N: Peligro que nos afecta a todos" /></a></div><div class="srp-content-box"><h3 class="srp-post-title"><a class="srp-post-title-link" href="http://bohemia.cu/en-cuba/2018/02/peligro-que-nos-afecta-a-todos/" title="CORRUPCI&Oacute;N: Peligro que nos afecta a todos">CORRUPCI&Oacute;N: Peligro que nos afecta a todos</a></h3></div></div></div></div><!-- END Special Recent Posts FREE Edition v2.0.4 --></div></div><div class="widget-wrapper"><div class="widget"><h2 class="widget-title srp-widget-title"><a class="srp-widget-title-link" href="http://bohemia.cu/category/opinion/" title="Opini&oacute;n">Opini&oacute;n</a></h2><!-- BEGIN Special Recent Posts FREE Edition v2.0.4 --><div class="srp-widget-container"><div id="wdg_specialrecentpostsfree-34-srp-singlepost-1" class="srp-widget-singlepost"><div class="srp-post-content-container"><div class="srp-thumbnail-box"><a class="srp-post-thumbnail-link" href="http://bohemia.cu/opinion/2018/03/una-fuerza-de-tarea-mas-para-la-subversion-en-cuba/" title="Una fuerza de tarea m&aacute;s para la subversi&oacute;n en Cuba"><img src="http://bohemia.cu/wp-content/plugins/special-recent-posts/cache/srpthumb-p106714-150x150-no.jpg" class="srp-post-thumbnail" alt="Una fuerza de tarea m&aacute;s para la subversi&oacute;n en Cuba" /></a></div><div class="srp-content-box"><h3 class="srp-post-title"><a class="srp-post-title-link" href="http://bohemia.cu/opinion/2018/03/una-fuerza-de-tarea-mas-para-la-subversion-en-cuba/" title="Una fuerza de tarea m&aacute;s para la subversi&oacute;n en Cuba">Una fuerza de tarea m&aacute;s para la subversi&oacute;n en Cuba</a></h3></div></div></div></div><!-- END Special Recent Posts FREE Edition v2.0.4 --></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Colaboración médica cubana beneficia a pobladores haitianos</div></div><iframe width='280' height='185' src='//www.youtube.com/embed/HDcamyTfpwc?autoplay=0&loop=0&rel=0' frameborder='0' allowfullscreen></iframe></div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/tag/elecciones" target="_self" class="widget_sp_image-image-link"><img width="300" height="200" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones.jpg 300w, http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones-163x109.jpg 163w, http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones-203x135.jpg 203w, http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones-162x108.jpg 162w, http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones-105x70.jpg 105w, http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones-180x120.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" src="http://bohemia.cu/wp-content/uploads/2018/03/logo-elecciones.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section"><a href="http://bohemia.cu/category/galerias/">Galerías</a></div></div><ul id="category-posts-2-internal" class="category-posts-internal">
<li class='cat-post-item'><div><a class="cat-post-thumbnail cat-post-none" href="http://bohemia.cu/galerias/2018/03/mujeres-orgullosas-de-insistir-en-tocar-el-horizonte/" title="MUJERES: orgullosas de insistir en tocar el horizonte"><span><img width="265" height="176" src="http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales.jpg" class="attachment-265x230 size-265x230 wp-post-image" alt="Orgullosas de insistir en tocar el horizonte." srcset="http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales.jpg 400w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-300x200.jpg 300w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-163x109.jpg 163w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-203x135.jpg 203w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-162x108.jpg 162w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-105x70.jpg 105w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-180x120.jpg 180w, http://bohemia.cu/wp-content/uploads/2018/03/activas-sociales-350x233.jpg 350w" sizes="(max-width: 265px) 100vw, 265px" /></span></a>
<a class="cat-post-title" href="http://bohemia.cu/galerias/2018/03/mujeres-orgullosas-de-insistir-en-tocar-el-horizonte/" rel="bookmark">MUJERES: orgullosas de insistir en tocar el horizonte</a></div></li></ul>
</div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/tag/cartelera-tv" target="_self" class="widget_sp_image-image-link"><img width="320" height="200" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1.jpg 320w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-300x188.jpg 300w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-216x135.jpg 216w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-173x108.jpg 173w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-180x113.jpg 180w, http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1-110x70.jpg 110w" sizes="(max-width: 320px) 100vw, 320px" src="http://bohemia.cu/wp-content/uploads/2016/07/canales-cartelera-1.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">El tiempo</div></div>			<div class="textwidget"><p><!-- www.TuTiempo.net - Ancho:183px - Alto:50px --></p>
<div id="TT_vCnAbhtBdQMcMcQK7AVDjzjzztnKarM">Pronóstico de Tutiempo.net</div>
<p><script type="text/javascript" src="http://www.tutiempo.net/widget/eltiempo_vCnAbhtBdQMcMcQK7AVDjzjzztnKarM"></script></p>
</div>
		</div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Fidel: Hasta la victoria siempre</div></div><a href="http://bohemia.cu/tag/fidelcastro" target="_self" class="widget_sp_image-image-link" title="Fidel: Hasta la victoria siempre"><img width="299" height="168" alt="Fidel: Hasta la victoria siempre" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/11/banner-lateral.jpg 299w, http://bohemia.cu/wp-content/uploads/2016/11/banner-lateral-230x129.jpg 230w, http://bohemia.cu/wp-content/uploads/2016/11/banner-lateral-190x108.jpg 190w, http://bohemia.cu/wp-content/uploads/2016/11/banner-lateral-180x101.jpg 180w" sizes="(max-width: 299px) 100vw, 299px" src="http://bohemia.cu/wp-content/uploads/2016/11/banner-lateral.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/tag/che-guevara" target="_self" class="widget_sp_image-image-link"><img width="300" height="200" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2017/08/logo9.jpg 300w, http://bohemia.cu/wp-content/uploads/2017/08/logo9-163x109.jpg 163w, http://bohemia.cu/wp-content/uploads/2017/08/logo9-203x135.jpg 203w, http://bohemia.cu/wp-content/uploads/2017/08/logo9-162x108.jpg 162w, http://bohemia.cu/wp-content/uploads/2017/08/logo9-105x70.jpg 105w, http://bohemia.cu/wp-content/uploads/2017/08/logo9-180x120.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" src="http://bohemia.cu/wp-content/uploads/2017/08/logo9.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/old/dictadura-batista/" target="_blank" class="widget_sp_image-image-link"><img width="300" height="200" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura.jpg 300w, http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura-163x109.jpg 163w, http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura-203x135.jpg 203w, http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura-162x108.jpg 162w, http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura-105x70.jpg 105w, http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura-180x120.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" src="http://bohemia.cu/wp-content/uploads/2017/03/logo-crimenes-dictadura.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/tag/Fidel90/" target="_self" class="widget_sp_image-image-link"><img width="336" height="242" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia.jpg 336w, http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia-300x216.jpg 300w, http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia-187x135.jpg 187w, http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia-150x108.jpg 150w, http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia-180x130.jpg 180w" sizes="(max-width: 336px) 100vw, 336px" src="http://bohemia.cu/wp-content/uploads/2016/06/Fidel-en-Bohemia.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">cartera oportunidades 2017-2018 (pdf)</div></div><a href="http://bohemia.cu/wp-content/uploads/2017/11/Cartera-de-Oportunidades-de-Inversin-Extranjera-2017-2018.pdf" target="_blank" class="widget_sp_image-image-link" title="cartera oportunidades 2017-2018 (pdf)"><img width="299" height="168" alt="cartera oportunidades 2017-2018 (pdf)" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/11/cartera-oportunidades.jpg 299w, http://bohemia.cu/wp-content/uploads/2016/11/cartera-oportunidades-230x129.jpg 230w, http://bohemia.cu/wp-content/uploads/2016/11/cartera-oportunidades-190x108.jpg 190w, http://bohemia.cu/wp-content/uploads/2016/11/cartera-oportunidades-180x101.jpg 180w" sizes="(max-width: 299px) 100vw, 299px" src="http://bohemia.cu/wp-content/uploads/2016/11/cartera-oportunidades.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Cuba vs Bloqueo</div></div><a href="http://bohemia.cu/tag/bloqueo/" target="_self" class="widget_sp_image-image-link" title="Cuba vs Bloqueo"><img width="300" height="173" alt="Cuba vs Bloqueo" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/10/banner-chico.jpg 300w, http://bohemia.cu/wp-content/uploads/2016/10/banner-chico-230x133.jpg 230w, http://bohemia.cu/wp-content/uploads/2016/10/banner-chico-187x108.jpg 187w, http://bohemia.cu/wp-content/uploads/2016/10/banner-chico-180x104.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" src="http://bohemia.cu/wp-content/uploads/2016/10/banner-chico.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Cobertura especial</div></div><a href="http://bohemia.cu/tag/congreso-pcc/" target="_self" class="widget_sp_image-image-link" title="Cobertura especial"><img width="300" height="150" alt="Cobertura especial" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/05/logo-congreso-pcc-600x300-300x170-1.jpg 300w, http://bohemia.cu/wp-content/uploads/2016/05/logo-congreso-pcc-600x300-300x170-1-230x115.jpg 230w, http://bohemia.cu/wp-content/uploads/2016/05/logo-congreso-pcc-600x300-300x170-1-190x95.jpg 190w, http://bohemia.cu/wp-content/uploads/2016/05/logo-congreso-pcc-600x300-300x170-1-180x90.jpg 180w" sizes="(max-width: 300px) 100vw, 300px" src="http://bohemia.cu/wp-content/uploads/2016/05/logo-congreso-pcc-600x300-300x170-1.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget"><a href="http://bohemia.cu/wp-content/uploads/2016/05/Conceptualizacion-Modelo-Economico-Social-Cubano-Desarrollo-Socialista.pdf" target="_blank" class="widget_sp_image-image-link"><img width="297" height="87" class="attachment-full" style="max-width: 100%;" srcset="http://bohemia.cu/wp-content/uploads/2016/06/documento-vll-congreso-pcc-pdf-1.jpg 297w, http://bohemia.cu/wp-content/uploads/2016/06/documento-vll-congreso-pcc-pdf-1-230x67.jpg 230w, http://bohemia.cu/wp-content/uploads/2016/06/documento-vll-congreso-pcc-pdf-1-190x56.jpg 190w, http://bohemia.cu/wp-content/uploads/2016/06/documento-vll-congreso-pcc-pdf-1-180x53.jpg 180w" sizes="(max-width: 297px) 100vw, 297px" src="http://bohemia.cu/wp-content/uploads/2016/06/documento-vll-congreso-pcc-pdf-1.jpg" /></a></div></div><div class="widget-wrapper"><div class="widget">
						<!-- 1.0.0 on WP4.9.4-->
						<a href="http://bohemia.cu/category/reflexiones-de-fidel-castro/" target="_self" >				<img src="http://bohemia.cu/wp-content/uploads/2015/12/reflexiones-fidel.jpg" alt="" title="" class="banner-image" width="100%" /></a>			<!-- /Ads Image Banner Widget Plugin -->
			<p id="text_description"> </p>
			</div></div><div class="widget-wrapper"><div class="widget">
						<!-- 1.0.0 on WP4.9.4-->
						<a href="http://bohemia.cu/category/discursos-de-raul/" target="_self" >				<img src="http://bohemia.cu/wp-content/uploads/2016/01/discursos-raul.jpg" alt="" title="" class="banner-image" width="100%" /></a>			<!-- /Ads Image Banner Widget Plugin -->
			<p id="text_description">  </p>
			</div></div><div class="widget-wrapper"><div class="widget">
						<!-- 1.0.0 on WP4.9.4-->
						<a href="http://bohemia.cu/category/cubamemoria/" target="_self" >				<img src="http://bohemia.cu/wp-content/uploads/2015/12/cuba-memoria-O.jpg" alt="" title="" class="banner-image" width="100%" /></a>			<!-- /Ads Image Banner Widget Plugin -->
			<p id="text_description"> </p>
			</div></div><div class="widget-wrapper"><div class="widget"><div class="section-wrapper"><div class="section">Más categorías</div></div><div class="menu-mas-categorias-container"><ul id="menu-mas-categorias" class="menu"><li id="menu-item-53560" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53560"><a href="http://bohemia.cu/category/sabias/">¿Sabías?</a></li>
<li id="menu-item-53569" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53569"><a href="http://bohemia.cu/category/extrano/">Extraño</a></li>
<li id="menu-item-53592" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53592"><a href="http://bohemia.cu/category/bohemia-vieja/">Bohemia Vieja</a></li>
<li id="menu-item-83795" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-83795"><a href="http://bohemia.cu/category/galerias/">Galerías</a></li>
<li id="menu-item-53573" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53573"><a href="http://bohemia.cu/category/grafiopinion/">Grafiopinión</a></li>
<li id="menu-item-53572" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53572"><a href="http://bohemia.cu/category/gazapos/">Gazapos</a></li>
<li id="menu-item-53579" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53579"><a href="http://bohemia.cu/category/tecnologia/">Tecnología</a></li>
<li id="menu-item-53575" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53575"><a href="http://bohemia.cu/category/le-contesta-bohemia/">Le contesta Bohemia</a></li>
<li id="menu-item-53564" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53564"><a href="http://bohemia.cu/category/consejos/">Consejos</a></li>
<li id="menu-item-53570" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53570"><a href="http://bohemia.cu/category/filatelia/">Filatelia</a></li>
<li id="menu-item-53577" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53577"><a href="http://bohemia.cu/category/mi-veterinario/">Mi veterinario</a></li>
<li id="menu-item-53568" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53568"><a href="http://bohemia.cu/category/ediciones-impresas/">Ediciones impresas</a></li>
<li id="menu-item-53563" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53563"><a href="http://bohemia.cu/category/cocina/">Cocina</a></li>
<li id="menu-item-53565" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53565"><a href="http://bohemia.cu/category/crucigramas/">Crucigramas</a></li>
<li id="menu-item-53576" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-53576"><a href="http://bohemia.cu/category/medio-ambiente/">Medio ambiente</a></li>
</ul></div></div></div><div class="widget-wrapper"><div class="widget">			<div class="section-wrapper"><div class="section">                Suscribirse a Bohemia Digital<span class="feedburner">&nbsp;</span>
            </div></div>                
        <div class="signup">

			Ingrese su dirección de correo electrónico                        
                          
					   <form id="feedburner_subscribe" class="subscribe" action="https://feedburner.google.com/fb/a/mailverify" method="post" target="popupwindow" onsubmit="window.open('https://feedburner.google.com/fb/a/mailverify?uri=RevistaBohemia', 'popupwindow', 'scrollbars=yes,width=550,height=520');return true">
							<div class="email-wrapper">
								<input type="text" name="email"/>
								<input type="hidden" value="RevistaBohemia" name="uri"/><input type="hidden" name="loc" value="es_ES"/>
								<input type="submit" value="Subscribe" /><p>Delivered by <a href="https://feedburner.google.com" target="_blank">FeedBurner</a></p>
							</div>			
							<input type="image" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/signup.png" class="btn" title="Usted recibir&aacute diariamente un email con nuestro nuevo contenido." onclick="document.feedburner_subscribe.submit();" />
						</form>
			<!--
			<form id="feedburner_subscribe" class="subscribe" action="http://feedburner.google.com/fb/a/mailverify" method="post" target="popupwindow" onsubmit="window.open('http://feedburner.google.com/fb/a/mailverify?uri=http://feeds.feedburner.com/RevistaBohemia', 'popupwindow', 'scrollbars=yes,width=550,height=520');return true">
                
                <div class="email-wrapper">
                    <input type="text" name="email"/>
                    <input type="hidden" value="http://feeds.feedburner.com/RevistaBohemia" name="uri"/>
                    <input type="hidden" name="loc" value="en_US"/>
                </div>
                    
                <input type="image" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/signup.png" class="btn" title="You will receive a daily email with new content from our website." onclick="document.feedburner_subscribe.submit();" />
                       	
			</form>
			
			-->
            
            <br class="clearer" />
            
        </div>
        
		</div></div>	
    </div>	
    

<br class="clearer" />

<!--
<div class="hide-pagination">
	</div>
-->


		            
            <div class="full-width-ad">  
            
                                
            </div>
            
            <br class="clearer" />
        
                
    </div><!--end main wrapper dark-->
    
    </div><!--end main white content wrapper -->
    
    <div id="footer-wrapper"> <!--begin footer wrapper -->
        
        <div id="footer">
        
        	        
                <div class="footer-menu">
            
                    <a class="home-link" href="http://bohemia.cu">&nbsp;</a>
            
                    <ul><li class="page_item page-item-95"><a href="http://bohemia.cu/quienes-somos/">¿Quiénes somos?</a></li>
<li class="page_item page-item-7577"><a href="http://bohemia.cu/blogs/">Blogs</a></li>
<li class="page_item page-item-601"><a href="http://bohemia.cu/comerciales/">Comerciales</a></li>
<li class="page_item page-item-336"><a href="http://bohemia.cu/conctactenos/">Conctáctenos</a></li>
<li class="page_item page-item-596"><a href="http://bohemia.cu/consejo-editorial/">Consejo Editorial</a></li>
<li class="page_item page-item-599"><a href="http://bohemia.cu/directorio-de-prensa/">Directorio</a></li>
<li class="page_item page-item-606"><a href="http://bohemia.cu/enlaces-de-interes/">Enlaces de interés</a></li>
</ul> 
                
                </div> 
                
                 <br class="clearer" />
                
                    
            <div class="inner">
        
                <div class="panel">
                
                    <div class="widget"><h1 class="widget-title srp-widget-title"><a class="srp-widget-title-link" href="http://bohemia.cu/category/ediciones-impresas/" title="Ediciones impresas">Ediciones impresas</a></h1><!-- BEGIN Special Recent Posts FREE Edition v2.0.4 --><div class="srp-widget-container"><div id="wdg_specialrecentpostsfree-19-srp-singlepost-1" class="srp-widget-singlepost"><div class="srp-post-content-container"><div class="srp-thumbnail-box"><a class="srp-post-thumbnail-link" href="http://bohemia.cu/ediciones-impresas/2018/03/portada-edicion-no-6-del-2018/" title="Portada edici&oacute;n No.6 del 2018"><img src="http://bohemia.cu/wp-content/plugins/special-recent-posts/cache/srpthumb-p106265-150x200-no.jpg" class="srp-post-thumbnail" alt="Portada edici&oacute;n No.6 del 2018" /></a></div></div></div></div><!-- END Special Recent Posts FREE Edition v2.0.4 --></div>                
                </div>
                
                <div class="panel">
                
                    <div class="widget"><h2>Nosotros</h2>		<ul>
			<li class="page_item page-item-95"><a href="http://bohemia.cu/quienes-somos/">¿Quiénes somos?</a></li>
<li class="page_item page-item-596"><a href="http://bohemia.cu/consejo-editorial/">Consejo Editorial</a></li>
		</ul>
		</div><div class="widget">			<h2>                Suscríbete vía Email<span class="feedburner">&nbsp;</span>
            </h2>                
        <div class="signup">

			Entre su dirección de correo                         
                          
					   <form id="feedburner_subscribe" class="subscribe" action="https://feedburner.google.com/fb/a/mailverify" method="post" target="popupwindow" onsubmit="window.open('https://feedburner.google.com/fb/a/mailverify?uri=RevistaBohemia', 'popupwindow', 'scrollbars=yes,width=550,height=520');return true">
							<div class="email-wrapper">
								<input type="text" name="email"/>
								<input type="hidden" value="RevistaBohemia" name="uri"/><input type="hidden" name="loc" value="es_ES"/>
								<input type="submit" value="Subscribe" /><p>Delivered by <a href="https://feedburner.google.com" target="_blank">FeedBurner</a></p>
							</div>			
							<input type="image" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/signup.png" class="btn" title="Usted recibir&aacute diariamente un email con nuestro nuevo contenido." onclick="document.feedburner_subscribe.submit();" />
						</form>
			<!--
			<form id="feedburner_subscribe" class="subscribe" action="http://feedburner.google.com/fb/a/mailverify" method="post" target="popupwindow" onsubmit="window.open('http://feedburner.google.com/fb/a/mailverify?uri=http://feeds.feedburner.com/RevistaBohemia', 'popupwindow', 'scrollbars=yes,width=550,height=520');return true">
                
                <div class="email-wrapper">
                    <input type="text" name="email"/>
                    <input type="hidden" value="http://feeds.feedburner.com/RevistaBohemia" name="uri"/>
                    <input type="hidden" name="loc" value="en_US"/>
                </div>
                    
                <input type="image" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/images/signup.png" class="btn" title="You will receive a daily email with new content from our website." onclick="document.feedburner_subscribe.submit();" />
                       	
			</form>
			
			-->
            
            <br class="clearer" />
            
        </div>
        
		</div>                
                </div>
                
                <div class="panel">
                
                    <div class="widget"><h2>Prensa cubana</h2>		<ul>
			<li class="page_item page-item-599"><a href="http://bohemia.cu/directorio-de-prensa/">Directorio</a></li>
		</ul>
		</div>                
                </div>
                
                <div class="panel right">
                
                    <div class="widget"><h2>Enlaces de interés</h2>		<ul>
			<li class="page_item page-item-606"><a href="http://bohemia.cu/enlaces-de-interes/">Enlaces de interés</a></li>
		</ul>
		</div>                        
                
                </div>
                
                <br class="clearer" />
                
            </div>
            
            <div class="copyright">
            
            	<div class="ribbon-shadow-left">&nbsp;</div>
            
                <div class="floatleft">
            
                    Copyright &copy; 2018 Revista Bohemia,&nbsp;All Rights Reserved.                    
                </div>
                
                <div class="floatright">
                
                    <div class="floatleft">
                    
                        			<div class="textwidget"><p>Bohemia. FUNDADA EN 1908. Revista Ilustrada de actualidad general.  Director: José Fernández Vega. Subdirectoras: Liset García Rodríguez y Bárbara Avendaño Pérez. Editor Jefe: Esteban Ramírez Alonso. Equipo Web:  Irene Izquierdo, Víctor Manuel González, Maria Victoria Bahler Vargas, Aleida Cabrera López,  y Orlando I. Romero Fernández<br />
Redacción: Avenida Independencia y San Pedro. Plaza. La Habana.  Cuba. CP: 10691 Teléfonos: 7881-2353 y 7881-1464  E-mail: digital@bohemia.cu</p>
</div>
		 
                        
                    </div>
                    
                </div>
                
                <br class="clearer" />
                
                <div class="ribbon-shadow-right">&nbsp;</div>
            
            </div>
            
        </div>
    
    </div> <!--end footer wrapper-->

	<script type='text/javascript' src='http://bohemia.cu/wp-includes/js/wp-embed.min.js?ver=4.9.4'></script>
	
	  
    
    <script type="text/javascript" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/js/plugins.js"></script> <!-- jquery plugin js -->
    
    <!-- need to setup review category tabs here since we don't know how many review types there are -->
    <script type="text/javascript">
		jQuery.noConflict(); 
		
		//DOCUMENT.READY
		jQuery(document).ready(function() { 
			//loop through each post type and setup a jquery tabs object
								jQuery('#tabbed-Película-reviews > ul').tabs({ fx: { opacity: 'toggle', duration: 150 } });		
			 
			
						
				//colorbox
				jQuery('.review .article-image a').colorbox({transition:'fade', speed:250});
				jQuery('.single-post .content .article-image a').colorbox({transition:'fade', speed:250});
				jQuery('.colorbox').colorbox({transition:'fade', speed:250});
				jQuery('.colorboxiframe').colorbox({transition:'fade', speed:250, iframe:true, innerWidth:640, innerHeight:390});
				jQuery(".page-content a[href$='.jpg'],a[href$='.png'],a[href$='.gif']").colorbox(); 
				jQuery('.page-content .gallery a').colorbox({  rel:'gallery' });
										
			 
			//initialize smooth div scroll on Don't Miss slider
			jQuery("#dontmiss").smoothDivScroll({ 
				autoScrollingMode: "always", 
				autoScrollingDirection: "endlessloopright", 
				autoScrollingStep: 1, 
				autoScrollingInterval: 50 
			});
		
			// Logo parade event handlers
			jQuery("#dontmiss").bind("mouseover", function() {
				jQuery(this).smoothDivScroll("stopAutoScrolling");
			}).bind("mouseout", function() {
				jQuery(this).smoothDivScroll("startAutoScrolling");
			});
			
			/* uitotop scroller:
			var defaults = {
	  			containerID: 'toTop', // fading element id
				containerHoverID: 'toTopHover', // fading element hover id
				scrollSpeed: 1200,
				easingType: 'linear' 
	 		};
			*/
			
			jQuery().UItoTop({ easingType: 'easeOutExpo' });	
	
		});
	
		//the reason they are here instead of in custom.js is because they contain php variables which can't
		//be applied in a .js file. Also, make sure these come before the darken function.
		
		//WINDOW.LOAD
		jQuery(window).load(function() {
			//spotlight slider	
			jQuery(function() {
				jQuery(".main-content-left #spotlight-slider, .main-content-left #spotlight-slider-responsive").jCarouselLite({		
					auto: 30000,
					easing: "easeInOutExpo",
					speed: 1100,
					visible: 2			
				});	
			});
			jQuery(function() {
				jQuery(".main-content #spotlight-slider, .main-content #spotlight-slider-responsive").jCarouselLite({		
					auto: 30000,
					easing: "easeInOutExpo",
					speed: 1100,
					visible: 3			
				});	
			});		
			//featured slider			
			jQuery('#featured').nivoSlider({				
				effect: 'random', // Specify sets like: 'fold,fade,sliceDown'
				slices: 10, // For slice animations
				boxCols: 6, // For box animations
				boxRows: 3, // For box animations
				animSpeed: 200, // Slide transition speed
				pauseTime: 30000, // How long each slide will show
				startSlide: 0, // Set starting Slide (0 index)
				directionNav: true, // Next and Prev navigation
				directionNavHide: false, // Only show on hover
				controlNav: false, // 1,2,3... navigation
				controlNavThumbs: false, // Use thumbnails for Control Nav
				pauseOnHover: true, // Stop animation while hovering
				manualAdvance: false, // Force manual transitions
				prevText: 'Prev', // Prev directionNav text
				nextText: 'Next', // Next directionNav text
				beforeChange: function(){}, // Triggers before a slide transition
				afterChange: function(){}, // Triggers after a slide transition
				slideshowEnd: function(){}, // Triggers after all slides have been shown
				lastSlide: function(){}, // Triggers when last slide is shown
				afterLoad: function(){} // Triggers when slider has loaded							 
			});	
					
		});		
    </script>
    
    <!-- make sure this js file is called after image sliders are setup or else the mosaic and darken effects won't work on hidden image elements-->
    <script type="text/javascript" src="http://bohemia.cu/wp-content/themes/Made-Theme/made/js/custom.js"></script> <!-- made js -->   
    


<!-- código añadido por ecoestadistica.com 


<a href="http://www.ecoestadistica.com" target="_blank" id="608627148"><img src="http://www.ecoestadistica.com/images/eco.gif" alt="ecoestadistica.com" border="0"></a>
<script type="text/javascript" language="JavaScript">
document.write('<script language="Javascript" src="http://www.ecoestadistica.com/Control.php?rp_id=608627148&rp_hrf='+document.getElementById(608627148).href+'&rp_loc='+escape(document.location)+'&rp_ref='+escape(document.referrer)+'&rp_scr='+screen.width+'x'+screen.height+'"><\/script>');
</script>


 fin del código añadido por ecoestadistica.com -->
 
 <!--NUEVO CODIGO DE ECOSISTEMA-->
 
<!-- código añadido por ecoestadistica.com 
<a href="http://www.ecoestadistica.com" target="_blank" id="398587906"><img src="http://www.ecoestadistica.com/images/eco.gif" alt="ecoestadistica.com" border="0"></a>
<script type="text/javascript" language="JavaScript">
document.write('<script language="Javascript" src="http://www.ecoestadistica.com/Control.php?rp_id=398587906&rp_hrf='+document.getElementById(398587906).href+'&rp_loc='+escape(document.location)+'&rp_ref='+escape(document.referrer)+'&rp_scr='+screen.width+'x'+screen.height+'"><\/script>');
</script>
<!-- fin del código añadido por ecoestadistica.com -->


	
</div>

<!-- Piwik -->
<script type="text/javascript">
  var _paq = _paq || [];
  _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
  _paq.push(["setCookieDomain", "*.bohemia.cu"]);
  _paq.push(["setDomains", ["*.bohemia.cu"]]);
  _paq.push(['trackPageView']);
  _paq.push(['enableLinkTracking']);
  (function() {
    var u="//piwikstats.cip.cu/";
    _paq.push(['setTrackerUrl', u+'piwik.php']);
    _paq.push(['setSiteId', 142]);
    var d=document, g=d.createElement('script'), s=d.getElementsByTagName('script')[0];
    g.type='text/javascript'; g.async=true; g.defer=true; g.src=u+'piwik.js'; s.parentNode.insertBefore(g,s);
  })();
</script>
<noscript><p><img src="//piwikstats.cip.cu/piwik.php?idsite=142" style="border:0;" alt="" /></p></noscript>
<!-- End Piwik Code -->

</body>

</html>