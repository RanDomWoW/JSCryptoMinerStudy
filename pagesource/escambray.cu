#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es-ES" >
<head>
<meta charset="UTF-8" />
<meta http-equiv="X-UA-Compatible" content="IE=edge;chrome=1">
<meta http-equiv="Content-Type" content="text / html; charset = utf-8"/>
<meta name="format-detection" content="address=no,email=no,telephone=no">
<meta name="title" content=" " />
<meta name="lang" content="es" />
<meta name="description"  content="" />
<meta name="date" scheme="W3CDTF" content="2018-03-07:11:02pm" />
<meta name="DC.date.issued" scheme="W3CDTF" content="2018-03-07" />
<meta name="DC.date" scheme="W3CDTF" content="2018-03-07" />
<meta name="organization" content="Ediciones Escambray ISSN 9664-1277" />
<meta name="locality" content="Sancti Spíritus, Cuba" />
<meta name="subject" content="Acontecer de la provincia de Sancti Spíritus, Cuba" />
<title>Escambray. Periódico de Sancti Spíritus. Cuba &#8211; Escambray. Periódico de la provincia de Sancti Spíritus, en el centro de Cuba, con noticias de Cuba, el mundo y Sancti Spíritus</title>
<meta property="og:title" content="Escambray. Periódico de Sancti Spíritus. Cuba"/>
<meta property="og:type" content="website"/>
<meta property="og:description" content="Escambray. Periódico de la provincia de Sancti Spíritus, en el centro de Cuba, con noticias de Cuba,"/>
<meta property="og:url" content="http://www.escambray.cu/"/>
<meta property="og:site_name" content="Escambray. Periódico de Sancti Spíritus. Cuba"/>
<link rel='dns-prefetch' href='//fonts.googleapis.com' />
<link rel='dns-prefetch' href='//s.w.org' />
<link rel='stylesheet' id='wp-polls-css'  href='http://www.escambray.cu/wp-content/plugins/wp-polls/polls-css.css' type='text/css' media='all' />
<style id='wp-polls-inline-css' type='text/css'>
.wp-polls .pollbar {
	margin: 1px;
	font-size: 6px;
	line-height: 8px;
	height: 8px;
	background-image: url('http://www.escambray.cu/wp-content/plugins/wp-polls/images/default/pollbg.gif');
	border: 1px solid #c8c8c8;
}

</style>
<link rel='stylesheet' id='tie-style-css'  href='http://www.escambray.cu/wp-content/themes/escambray/style.css' type='text/css' media='all' />
<link rel='stylesheet' id='tie-ilightbox-skin-css'  href='http://www.escambray.cu/wp-content/themes/escambray/css/ilightbox/dark-skin/skin.css' type='text/css' media='all' />
<link rel='stylesheet' id='Droid+Sans-css'  href='http://fonts.googleapis.com/css?family=Droid+Sans%3Aregular%2C700' type='text/css' media='all' />
<script type='text/javascript' src='http://www.escambray.cu/wp-includes/js/jquery/jquery.js'></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-includes/js/jquery/jquery-migrate.min.js' defer onload=''></script>
<link rel="EditURI" type="application/rsd+xml" title="RSD" href="http://www.escambray.cu/xmlrpc.php?rsd" />
<link rel="canonical" href="http://www.escambray.cu/" />
<link rel='shortlink' href='http://www.escambray.cu/' />

<!-- Protected by WP-SpamFree :: JS BEGIN -->
<script type="text/javascript" src="http://www.escambray.cu/wp-content/plugins/wp-spamfree/js/wpsfv2-js.php"></script> 
<!-- Protected by WP-SpamFree :: JS END -->

<!-- <meta name="NextGEN" version="2.2.50" /> -->
<link rel="shortcut icon" href="http://www.escambray.cu/wp-content/themes/escambray/favicon.ico" title="Favicon" />
<!--[if IE]>
<script type="text/javascript">jQuery(document).ready(function (){ jQuery(".menu-item").has("ul").children("a").attr("aria-haspopup", "true");});</script>
<![endif]-->
<!--[if lt IE 9]>
<script src="http://www.escambray.cu/wp-content/themes/escambray/js/html5.js"></script>
<script src="http://www.escambray.cu/wp-content/themes/escambray/js/selectivizr-min.js"></script>
<![endif]-->
<!--[if IE 9]>
<link rel="stylesheet" type="text/css" media="all" href="http://www.escambray.cu/wp-content/themes/escambray/css/ie9.css" />
<![endif]-->
<!--[if IE 8]>
<link rel="stylesheet" type="text/css" media="all" href="http://www.escambray.cu/wp-content/themes/escambray/css/ie8.css" />
<![endif]-->
<!--[if IE 7]>
<link rel="stylesheet" type="text/css" media="all" href="http://www.escambray.cu/wp-content/themes/escambray/css/ie7.css" />
<![endif]-->


<meta name="viewport" content="width=device-width, initial-scale=1.0" />
<link rel="apple-touch-icon-precomposed" href="http://www.escambray.cu/wp-content/themes/escambray/favicon.ico" />



<style type="text/css" media="screen">


</style>

<meta itemprop="name" content="Escambray"> 
<meta itemprop="description" content=""> 
<meta itemprop="image" content="" />
<meta property="article:modified_time" content="" />
<meta property="article:publisher" content="https://www.facebook.com/pages/Escambray/" />
<meta property="article:published_time" content="2018-03-07" />
<meta property="og:image" content="" />
<meta property="og:image:width" content="520">
<meta property="og:image:height" content="277">
<meta property="og:updated_time" content="2018-03-07:11:02" />
<meta property="fb:app_id" 		content="205692256170"/>
<meta property="og:locale" content="es_ES">
<meta name="twitter:card" content="summary"> 
<meta name="twitter:creator" content="@escambraycu"/>
<meta name="twitter:site" content="@escambraycu"/>
<meta name="twitter:url" value="http://www.escambray.cu/" />
<meta name="twitter:title" value="Portada centro" />
<meta name="twitter:description" value="" />
<meta name="twitter:image" value="" />
<meta name="twitter:image:src" content=""> 
<link rel="alternate" href="http://www.escambray.cu/" hreflang="es" />
<link rel="shortcut icon" href="http://www.escambray.cu/wp-content/uploads/2014/08/favicon.png" />
<link rel="icon" href="http://www.escambray.cu/wp-content/uploads/2014/08/favicon.png" sizes="32x32"/>
<link rel='stylesheet' id='nextgen_widgets_style-css'  href='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/widget/static/widgets.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='nextgen_basic_thumbnails_style-css'  href='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_basic_gallery/static/thumbnails/nextgen_basic_thumbnails.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='ngg_trigger_buttons-css'  href='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_gallery_display/static/trigger_buttons.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='dashicons-css'  href='http://www.escambray.cu/wp-includes/css/dashicons.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='thickbox-css'  href='http://www.escambray.cu/wp-includes/js/thickbox/thickbox.css' type='text/css' media='all' />
<link rel='stylesheet' id='fontawesome-css'  href='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_gallery_display/static/fontawesome/font-awesome.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='nextgen_pagination_style-css'  href='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_pagination/static/style.min.css' type='text/css' media='all' />
<link rel='stylesheet' id='nggallery-css'  href='http://www.escambray.cu/wp-content/ngg_styles/nggallery.css' type='text/css' media='all' />
</head>
<body id="top" class="home page-template page-template-template-home-center page-template-template-home-center-php page page-id-166962">
<div class="wrapper-outer">

	<aside id="slide-out">
			<div class="search-mobile">
			<form method="get" id="searchform-mobile" action="http://www.escambray.cu/">
				<button class="search-button" type="submit" value=""><i class="fa fa-search"></i></button>
				<input type="text" id="s-mobile" name="s" title="" value="" onfocus="if (this.value == 'Search') {this.value = '';}" onblur="if (this.value == '') {this.value = '';}"  />
			</form>
		</div>
				<div class="social-icons">
		<a class="ttip-none" title="Rss" href="http://www.escambray.cu/feed/" target="_blank"><i class="fa fa-rss"></i></a><a class="ttip-none" title="Facebook" href="http://www.facebook.com/pages/Escambray/205692256170" target="_blank"><i class="fa fa-facebook"></i></a><a class="ttip-none" title="Twitter" href="http://www.twitter.com/escambraycu" target="_blank"><i class="fa fa-twitter"></i></a><a class="ttip-none" title="Flickr" href="https://www.flickr.com/photos/144830829@N04/" target="_blank"><i class="tieicon-flickr"></i></a>
			</div>

			<div id="mobile-menu" ></div>
	</aside>

		<div id="wrapper" class="boxed-all">
		<div class="inner-wrapper">

		<header id="theme-header" class="theme-header center-logo">
						<div id="top-nav" class="top-nav">
				<div class="container">

							<span class="today-date">
				<script language="" languaje="JavaScript" async="async">
var mydate=new Date()
var year=mydate.getYear()
if (year < 1000)
year+=1900
var day=mydate.getDay()
var month=mydate.getMonth()
var daym=mydate.getDate()
if (daym<10)
daym="0"+daym
var dayarray=new Array("Domingo","Lunes","Martes","Miércoles","Jueves","Viernes","Sábado")
var montharray=new Array("Enero","Febrero","Marzo","Abril","Mayo","Junio","Julio","Agosto","Septiembre","Octubre","Noviembre","Diciembre")
document.write("<p>"+dayarray[day]+"-"+montharray[month]+" "+daym+" "+year+"</p>")
 </script></span>
				<div class="top-menu"><ul id="menu-top" class="menu"><li id="menu-item-164508" class="menu-item menu-item-type-custom menu-item-object-custom menu-item-164508"><a href="http://en.escambray.cu/">English</a></li>
<li id="menu-item-166952" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-166952"><a href="http://www.escambray.cu/directorio-de-prensa-cubana/">Directorio de prensa</a></li>
<li id="menu-item-166953" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-166953"><a href="http://www.escambray.cu/edicion-impresa/">Edición impresa</a></li>
<li id="menu-item-166954" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-166954"><a href="http://www.escambray.cu/efemerides/">Efémerides de Cuba</a></li>
<li id="menu-item-167519" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-167519"><a href="http://www.escambray.cu/el-tiempo/">El tiempo</a></li>
</ul></div>
						<div class="search-block">
						<form method="get" id="searchform-header" action="http://www.escambray.cu/">
							<button class="search-button" type="submit" value=""><i class="fa fa-search"></i></button>
							<input class="search-live" type="text" id="s-header" name="s" title="" value="" onfocus="if (this.value == 'Search') {this.value = '';}" onblur="if (this.value == '') {this.value = '';}"  />
						</form>
					</div>
			<div class="social-icons">
		<a class="ttip-none" title="Rss" href="http://www.escambray.cu/feed/" target="_blank"><i class="fa fa-rss"></i></a><a class="ttip-none" title="Facebook" href="http://www.facebook.com/pages/Escambray/205692256170" target="_blank"><i class="fa fa-facebook"></i></a><a class="ttip-none" title="Twitter" href="http://www.twitter.com/escambraycu" target="_blank"><i class="fa fa-twitter"></i></a><a class="ttip-none" title="Flickr" href="https://www.flickr.com/photos/144830829@N04/" target="_blank"><i class="tieicon-flickr"></i></a>
			</div>

	
	
				</div><!-- .container /-->
			</div><!-- .top-menu /-->
			
		<div class="header-content">

					<a id="slide-out-open" class="slide-out-open" href="#"><span></span></a>
		
			<div class="logo" style=" margin-top:7px; margin-bottom:1px;">
			<h1>								<a title="Escambray. Periódico de Sancti Spíritus. Cuba" href="http://www.escambray.cu/">
					<img src="http://www.escambray.cu/wp-content/themes/escambray/images/logo.png" alt="Escambray. Periódico de Sancti Spíritus. Cuba"  /><strong>Escambray. Periódico de Sancti Spíritus. Cuba Escambray. Periódico de la provincia de Sancti Spíritus, en el centro de Cuba, con noticias de Cuba, el mundo y Sancti Spíritus</strong>
				</a>
			</h1>			</div>
	
	<div class="headerleft">
	 <h1> Periódico de Sancti Spíritus</h1>
	</div>
<div class="clear"></div>
</div>
													<nav id="main-nav" class="fixed-enabled">
				<div class="container">

				
					<div class="main-menu"><ul id="menu-principal" class="menu"><li id="menu-item-166957" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166957"><a href="http://www.escambray.cu/category/ss/">Sancti Spíritus</a></li>
<li id="menu-item-166956" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166956"><a href="http://www.escambray.cu/category/cuba/">Cuba</a></li>
<li id="menu-item-166958" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166958"><a href="http://www.escambray.cu/category/mundo/">Internacionales</a></li>
<li id="menu-item-166959" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166959"><a href="http://www.escambray.cu/category/deporte/">Deporte</a></li>
<li id="menu-item-166960" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166960"><a href="http://www.escambray.cu/category/cultura/">Cultura</a></li>
<li id="menu-item-166961" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-166961"><a href="http://www.escambray.cu/category/ct/">Ciencia y tecnología</a></li>
<li id="menu-item-167520" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-167520"><a href="http://www.escambray.cu/especiales/">Especiales</a></li>
</ul></div>					

				</div>
			</nav>
					</header>

	

	<div id="main-content" class="container"><section class="bloque_actualidad_destacadas" data-vr-zone="Apertura">
<div id="titular_content-center">
     <h2><a href="http://www.escambray.cu/2018/sancti-spiritus-estrena-linea-para-el-beneficio-del-frijol/">Sancti Spíritus estrena línea para el beneficio del frijol</a></h2>
									<a href="http://www.escambray.cu/2018/sancti-spiritus-estrena-linea-para-el-beneficio-del-frijol/"><img src="http://www.escambray.cu/wp-content/uploads/2018/03/secadero-iguara-1-foto-pepe.jpg" /></a>
												<p class="articulo-entradilla" itemprop="description">Entra en funcionamiento una parte de la planta que se construye en Yaguajay, primera de su tipo en Cuba y que más adelante incorporará el proceso de secado de granos </p> 
								<ul><li>
										<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2016/sancti-spiritus-en-26-produccion-de-frijol-sustituye-importaciones-fotos/">
							<img src="http://www.escambray.cu/wp-content/uploads/2016/07/frijoles-en-banao-foto-elisdany-lopez.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
															
						<h3><a href="http://www.escambray.cu/2016/sancti-spiritus-en-26-produccion-de-frijol-sustituye-importaciones-fotos/" rel="bookmark">Producción de frijol sustituye importaciones (+Fotos)</a></h3>
				</li></ul>
								
<div style="clear:both;"></div>                                  
</div>
<article>
<div class="homepagecenterL">
	<div class="articulos_apertura">
	<article>
	<div class="articulo_interior">
												<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Cayo-Largo-un-destino-turístico-al-sur-de-Cuba..jpg" /> 	
						  <h2><a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/">Cubana VIP en Cayo Largo del Sur</a></h2>			  
						<p class="articulo-entradilla" itemprop="description">Viajar en condición de turista hasta Cayo Largo del Sur no siempre representa un ejercicio placentero, aunque para ello hayas pagado en tiempo y forma 162 CUC </p> 
												<div style="clear:both;"></div>   
	</div>
	</article>
	</div>
</div>	
<div class="homepagecenterR">
	<div class="articulos_apertura">
	<article>
	<div class="articulo_interior">
												  <h2><a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/">Sancti Spíritus: Más de 3 000 plazas para la Educación Superior</a></h2>
						  	<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Estudiantes-8-1.jpg" /> 						<p>En la modalidad de Curso Regular Diurno predominan las carreras del área de las Ciencias Médicas, mientras en el Curso por Encuentros se contabilizan diferentes especialidades<a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/">&raquo;</a></p>
												<div style="clear:both;"></div>   
	</div>
	</article>
	</div>
</div>
</section>
<div class="clear"></div>
</article>
	<section id="bloque_actualidad_cierre">
<article>
<div id="articulos_cierre">
 <div id="articulos_interior">
 <ul id="articulos_interior">
<li>
						<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">
							<img src="http://www.escambray.cu/wp-content/uploads/2018/03/turismo-cuba2.jpg" /> 
							</a>
						</div>
											

<h2><a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/" rel="bookmark" title="Novedades turísticas cubanas en Salón Mundial de París">Novedades turísticas cubanas en Salón Mundial de París</a></h2>
</li></ul>
</div>
 <div id="articulos_interior">
 <ul id="articulos_interior">
<li>
						<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">
							<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Moscu-Londres.jpg" /> 
							</a>
						</div>
											

<h2><a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/" rel="bookmark" title="Rusia declara persona non grata a 23 diplomáticos británicos">Rusia declara persona non grata a 23 diplomáticos británicos</a></h2>
</li></ul>
</div>
 <div id="articulos_interior">
 <ul id="articulos_interior">
<li>
						<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">
							<img src="http://www.escambray.cu/wp-content/uploads/2018/03/trabajo-de-los-instructores-de-arte-en-sancti-spiritus-foto-vicente-brito.jpg" /> 
							</a>
						</div>
											

<h2><a href="http://www.escambray.cu/2018/por-una-mejor-brigada/" rel="bookmark" title="Por una mejor brigada">Por una mejor brigada</a></h2>
</li></ul>
</div>
 <div id="articulos_interior">
 <ul id="articulos_interior">
<li>
						<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/">
							<img src="http://www.escambray.cu/wp-content/uploads/2018/03/baloncesto-femenino-espirituanas-vs-capitalinas.jpg" /> 
							</a>
						</div>
											

<h2><a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/" rel="bookmark" title="Espirituanas dan primero en final de Liga Superior de Baloncesto">Espirituanas dan primero en final de Liga Superior de Baloncesto</a></h2>
</li></ul>
</div>
 </div>
</article>
	<div class="clear"></div>
</section>
<section>
<article>
<header class="bloque-cabecera"><h3 class="cabecera-titulo">Sugerencias</h3></header>
<div id="home_SC">
	 <div id="contenido_SC">
 <ul id="contenido_SC">
<li>
<a href="http://www.escambray.cu/2018/las-corduras-del-psiquiatrico-fotos/"><img src="http://www.escambray.cu/wp-content/uploads/2018/03/el-hospital-siquiatrico-deviene-institucion-docente.jpg" /> </a>
<h3><a href="http://www.escambray.cu/2018/las-corduras-del-psiquiatrico-fotos/" rel="bookmark" title="Las corduras del Psiquiátrico (+fotos)">Las corduras del Psiquiátrico (+fotos)</a></h3>
</li></ul>
</div>
 <div id="contenido_SC">
 <ul id="contenido_SC">
<li>
<a href="http://www.escambray.cu/2018/sancti-spiritus-inventarios-parados/"><img src="http://www.escambray.cu/wp-content/uploads/2018/03/control-sobre-los-inventarios.jpg" /> </a>
<h3><a href="http://www.escambray.cu/2018/sancti-spiritus-inventarios-parados/" rel="bookmark" title="Sancti Spíritus: Inventarios parados…">Sancti Spíritus: Inventarios parados…</a></h3>
</li></ul>
</div>
 <div id="contenido_SC">
 <ul id="contenido_SC">
<li>
<a href="http://www.escambray.cu/2018/escambray-con-nuevo-diseno-web-mejorar-la-forma-perfeccionar-el-contenido/"><img src="http://www.escambray.cu/wp-content/uploads/2018/03/Portada.jpg" /> </a>
<h3><a href="http://www.escambray.cu/2018/escambray-con-nuevo-diseno-web-mejorar-la-forma-perfeccionar-el-contenido/" rel="bookmark" title="Escambray con nuevo diseño web: mejorar la forma, perfeccionar el contenido">Escambray con nuevo diseño web: mejorar la forma, perfeccionar el contenido</a></h3>
</li></ul>
</div>
 </div>
</article>
<div class="clear"></div>
</section>
<section>
<article>
<header class="bloque-cabecera"><h3 class="cabecera-titulo">OPINIÓN</h3></header>
<div id="home_SC">
 <div id="contenido_O">
 <ul id="contenido_O">
<li>
<div class="author-avatar">
<a href="http://www.escambray.cu/author/carmen-rodriguez/" title=""> <img src="http://www.escambray.cu/wp-content/uploads/2018/03/Carmen-rodriguez.png" alt="" /></a>
</div>
<h2><a href="http://www.escambray.cu/2018/las-elecciones-del-pueblo/" rel="bookmark" title="Las elecciones del pueblo">Las elecciones del pueblo</a></h2>
<h5><a href="http://www.escambray.cu/author/carmen-rodriguez/" title="Entradas de Carmen Rodríguez" rel="author">Carmen Rodríguez</a></h5>
</li></ul>
</div>
 <div id="contenido_O">
 <ul id="contenido_O">
<li>
<div class="author-avatar">
<a href="http://www.escambray.cu/author/enrique-ojito/" title=""> <img src="http://www.escambray.cu/wp-content/uploads/2015/08/Ojito.png" alt="" /></a>
</div>
<h2><a href="http://www.escambray.cu/2018/sancti-spiritus-inventarios-parados/" rel="bookmark" title="Sancti Spíritus: Inventarios parados…">Sancti Spíritus: Inventarios parados…</a></h2>
<h5><a href="http://www.escambray.cu/author/enrique-ojito/" title="Entradas de Enrique Ojito" rel="author">Enrique Ojito</a></h5>
</li></ul>
</div>
 <div id="contenido_O">
 <ul id="contenido_O">
<li>
<div class="author-avatar">
<a href="http://www.escambray.cu/author/delia-proenza/" title=""> <img src="http://www.escambray.cu/wp-content/uploads/2015/08/Delia.png" alt="" /></a>
</div>
<h2><a href="http://www.escambray.cu/2018/sancti-spiritus-por-bosques-sin-ilegalidades/" rel="bookmark" title="Sancti Spíritus: Por bosques sin ilegalidades">Sancti Spíritus: Por bosques sin ilegalidades</a></h2>
<h5><a href="http://www.escambray.cu/author/delia-proenza/" title="Entradas de Delia Proenza" rel="author">Delia Proenza</a></h5>
</li></ul>
</div>
 </div>
	</article>
</section>
			<div class="content">
		<article class="post-listing post post-166962 page type-page status-publish " id="the-post">
					
		<section class="cat-box list-box tie-cat-3">
			<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/ss/">Sancti Spíritus</a></h2>
			</div>
			<div class="cat-box-content">
			
								<ul>
													<li class="first-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Cayo-Largo-un-destino-turístico-al-sur-de-Cuba..jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
											
						<h2 class="post-box-title"><a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/" rel="bookmark">Cubana VIP en Cayo Largo del Sur</a></h2>
							<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">17/03/2018</a></span> 
	
</p>
	
						<div class="entry">
								<p>Viajar en condición de turista hasta Cayo Largo del Sur no siempre representa un ejercicio placentero, aunque para ello hayas pagado en tiempo y forma 162 CUC<a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/"> &raquo;</a></p>
							</div>							
						</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Estudiantes-8-1.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/" rel="bookmark">Sancti Spíritus: Más de 3 000 plazas para la Educación Superior</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">17/03/2018</a></span> 
	
</p>
					</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/sancti-spiritus-estrena-linea-para-el-beneficio-del-frijol/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/secadero-iguara-1-foto-pepe.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/sancti-spiritus-estrena-linea-para-el-beneficio-del-frijol/" rel="bookmark">Sancti Spíritus estrena línea para el beneficio del frijol</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">17/03/2018</a></span> 
	
</p>
					</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/ponderan-en-sancti-spiritus-esfuerzos-para-la-recuperacion-de-cuba-tras-irma/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Virginia-André-coordinadora-de-la-UE-para-Cuba-y-el-Caribe-foto-luis-francisco-jacomino.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/ponderan-en-sancti-spiritus-esfuerzos-para-la-recuperacion-de-cuba-tras-irma/" rel="bookmark">Ponderan en Sancti Spíritus esfuerzos para la recuperación de Cuba tras Irma</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">16/03/2018</a></span> 
	
</p>
					</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/debate-entre-los-miembros-de-la-mayor-organizacion-de-masas-del-pais/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/debates-con-cederistas-espirituanos.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/debate-entre-los-miembros-de-la-mayor-organizacion-de-masas-del-pais/" rel="bookmark">Debate entre los miembros de la mayor organización de masas del país</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">16/03/2018</a></span> 
	
</p>
					</li>
													</ul>
				<div class="clear"></div>

								</div>
		</section>

				<section class="cat-box scroll-box tie-cat-4">
		
					<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/cuba/">Cuba</a></h2>
			</div>
				
			<div class="cat-box-content">
								<div id="slideshow4" class="group_items-box">
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/turismo-cuba2.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/" rel="bookmark">Novedades turísticas cubanas en Salón Mundial de París</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/inauguran-en-cuba-embajada-de-kenya-fotos/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Inauguran-en-Cuba-embajada-de-Kenya-foto-pl.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/inauguran-en-cuba-embajada-de-kenya-fotos/" rel="bookmark">Inauguran en Cuba embajada de Kenya (+fotos)</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/presidente-cubano-agradece-solidaridad-de-kenya-con-la-isla-caribena/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Recibió-Raúl-al-presidente-de-Kenya-Uhuru-Kenyatta-foto-acn.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/presidente-cubano-agradece-solidaridad-de-kenya-con-la-isla-caribena/" rel="bookmark">Presidente cubano agradece solidaridad de Kenya con la isla</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/cuba-presenta-multiproducto-turistico-en-salon-mundial-de-paris/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/cuba-fra-turismo2.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/cuba-presenta-multiproducto-turistico-en-salon-mundial-de-paris/" rel="bookmark">Cuba presenta multiproducto turístico en Salón Mundial de París</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/en-baragua-se-salvo-el-honor-de-cuba/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/protesta-de-baragua-foto-archivo-de-granma.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/en-baragua-se-salvo-el-honor-de-cuba/" rel="bookmark">En Baraguá se salvó el honor de Cuba</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/en-cuba-presidente-de-kenya/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Kenya-Uhuru-Kenyatta.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/en-cuba-presidente-de-kenya/" rel="bookmark">En Cuba Presidente de Kenya</a></h3>
						<p class="post-meta">
																				</p>
					</div>
								<div class="clear"></div>
				</div>
				<div id="nav4" class="scroll-nav"></div>
								</div><!-- .cat-box-content /-->
		</section>
		<div class="clear"></div>
<script type="text/javascript">
	jQuery(document).ready(function() {
		var vids = jQuery("#slideshow4 .scroll-item");
		for(var i = 0; i < vids.length; i+=3) {
		  vids.slice(i, i+3).wrapAll('<div class="group_items"></div>');
		}
		jQuery(function() {
			jQuery('#slideshow4').cycle({
				fx:     'scrollHorz',
				timeout: 5000,
				pager:  '#nav4',
				slideExpr: '.group_items',
				speed: 500,
				slideResize: false,
				pause: true
			});
		});
  });
</script>
		<section class="cat-box scroll-box tie-cat-5">
		
					<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/mundo/">Internacionales</a></h2>
			</div>
				
			<div class="cat-box-content">
								<div id="slideshow5" class="group_items-box">
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/Moscu-Londres.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/" rel="bookmark">Rusia declara persona non grata a 23 diplomáticos británicos</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/asciende-a-seis-los-muertos-por-colapso-de-puente-en-miami-fotos/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/puente-peatonal-colapsa-en-miami-foto-ap.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/asciende-a-seis-los-muertos-por-colapso-de-puente-en-miami-fotos/" rel="bookmark">Ascienden a seis los muertos por colapso de puente en Miami (+fotos)</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/despidos-y-renuncias-de-funcionarios-en-ee-uu-desorden-en-la-casa-blanca/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/despidos-de-la-casa-blanca-en-la-era-de-trump.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/despidos-y-renuncias-de-funcionarios-en-ee-uu-desorden-en-la-casa-blanca/" rel="bookmark">Despidos y renuncias de funcionarios: ¿Desorden en la Casa Blanca?</a></h3>
						<p class="post-meta">
																				</p>
					</div>
								<div class="clear"></div>
				</div>
				<div id="nav5" class="scroll-nav"></div>
								</div><!-- .cat-box-content /-->
		</section>
		<div class="clear"></div>
<script type="text/javascript">
	jQuery(document).ready(function() {
		var vids = jQuery("#slideshow5 .scroll-item");
		for(var i = 0; i < vids.length; i+=3) {
		  vids.slice(i, i+3).wrapAll('<div class="group_items"></div>');
		}
		jQuery(function() {
			jQuery('#slideshow5').cycle({
				fx:     'scrollHorz',
				timeout: 5000,
				pager:  '#nav5',
				slideExpr: '.group_items',
				speed: 500,
				slideResize: false,
				pause: true
			});
		});
  });
</script>
			<section class="cat-box wide-box tie-cat-6">
			<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/deporte/">Deporte</a></h2>
			</div>
			<div class="cat-box-content">
			
								<ul>
													<li class="first-news">
						<div class="inner-content">
													<div class="post-thumbnail">
								<a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/">
								<img src="http://www.escambray.cu/wp-content/uploads/2018/03/baloncesto-femenino-espirituanas-vs-capitalinas.jpg" />  <span class="fa overlay-icon"></span>
								</a>
							</div>
												<h2 class="post-box-title"><a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/" rel="bookmark">Espirituanas dan primero en final de Liga Superior de Baloncesto</a></h2>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">17/03/2018</a></span> 
	
</p>
					
							<div class="entry">
								<p>Este sábado a las dos de la tarde la sala Yara acogerá el segundo partido de esta final<a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/"> &raquo;</a></p>
							</div>
						</div>
					</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/serie-provincial-de-beisbol-trinidad-empata-en-la-final-vs-sancti-spiritus/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Luis-Daniel-Serrano-1-Large.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/serie-provincial-de-beisbol-trinidad-empata-en-la-final-vs-sancti-spiritus/" rel="bookmark">Serie Provincial de Béisbol: Trinidad empata en la final vs Sancti Spíritus</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">16/03/2018</a></span> 
	
</p>
					
					</li>
																		<li class="other-news">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/trinidad-defendera-su-titulo-otra-vez-en-final-de-la-pelota-provincial/">
			<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/serie-provincial-de-beisbol-trinidad-vs-sancti-spiritus.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/trinidad-defendera-su-titulo-otra-vez-en-final-de-la-pelota-provincial/" rel="bookmark">Trinidad defenderá su título otra vez en final de la pelota provincial</a></h3>
						<p class="post-meta"> 
	
<a href="http://www.escambray.cu/2018/03/">14/03/2018</a></span> 
	
</p>
					
					</li>
													</ul>
				<div class="clear"></div>

								</div><!-- .cat-box-content /-->
		</section><!-- Wide Box -->

				<section class="cat-box scroll-box tie-cat-7">
		
					<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/cultura/">Cultura</a></h2>
			</div>
				
			<div class="cat-box-content">
								<div id="slideshow7" class="group_items-box">
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/trabajo-de-los-instructores-de-arte-en-sancti-spiritus-foto-vicente-brito.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/por-una-mejor-brigada/" rel="bookmark">Por una mejor brigada</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/presentan-en-guayos-cabaiguan-libro-sobre-elcire-perez/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/libro-sobre-elcire-perez.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/presentan-en-guayos-cabaiguan-libro-sobre-elcire-perez/" rel="bookmark">Presentan en Guayos, Cabaiguán, libro sobre Elcire Pérez</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/decimas-bajo-la-ceiba-fotos/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/1-6-pena-campesina-de-guayos-foto-luaris-henriquez.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/decimas-bajo-la-ceiba-fotos/" rel="bookmark">Décimas bajo la ceiba (+fotos)</a></h3>
						<p class="post-meta">
																				</p>
					</div>
								<div class="clear"></div>
				</div>
				<div id="nav7" class="scroll-nav"></div>
								</div><!-- .cat-box-content /-->
		</section>
		<div class="clear"></div>
<script type="text/javascript">
	jQuery(document).ready(function() {
		var vids = jQuery("#slideshow7 .scroll-item");
		for(var i = 0; i < vids.length; i+=3) {
		  vids.slice(i, i+3).wrapAll('<div class="group_items"></div>');
		}
		jQuery(function() {
			jQuery('#slideshow7').cycle({
				fx:     'scrollHorz',
				timeout: 5000,
				pager:  '#nav7',
				slideExpr: '.group_items',
				speed: 500,
				slideResize: false,
				pause: true
			});
		});
  });
</script>
		<section class="cat-box scroll-box tie-cat-8">
		
					<div class="cat-box-title">
				<h2><a href="http://www.escambray.cu/category/ct/">Ciencia y tecnología</a></h2>
			</div>
				
			<div class="cat-box-content">
								<div id="slideshow8" class="group_items-box">
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/luto-en-el-mundo-de-la-ciencia-por-muerte-de-stephen-hawking/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/stephen-hawking.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/luto-en-el-mundo-de-la-ciencia-por-muerte-de-stephen-hawking/" rel="bookmark">Luto en el mundo de la ciencia por muerte de Stephen Hawking</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/crece-la-conexion-desde-la-casa/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/nauta-hogar-1.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/crece-la-conexion-desde-la-casa/" rel="bookmark">Crece la conexión desde la casa</a></h3>
						<p class="post-meta">
																				</p>
					</div>
									<div class="scroll-item">
								<div class="post-thumbnail">
			<a href="http://www.escambray.cu/2018/vindicacion-de-la-cenicienta-electrica-infografia/">
			<img src="http://www.escambray.cu/wp-content/uploads/2018/03/subestacion-de-tuinucu-foto-vicente-brito.jpg" />  <span class="fa overlay-icon"></span>
			</a>
		</div>
									
						<h3 class="post-box-title"><a href="http://www.escambray.cu/2018/vindicacion-de-la-cenicienta-electrica-infografia/" rel="bookmark">Vindicación de la «cenicienta eléctrica» (+Infografía)</a></h3>
						<p class="post-meta">
																				</p>
					</div>
								<div class="clear"></div>
				</div>
				<div id="nav8" class="scroll-nav"></div>
								</div><!-- .cat-box-content /-->
		</section>
		<div class="clear"></div>
<script type="text/javascript">
	jQuery(document).ready(function() {
		var vids = jQuery("#slideshow8 .scroll-item");
		for(var i = 0; i < vids.length; i+=3) {
		  vids.slice(i, i+3).wrapAll('<div class="group_items"></div>');
		}
		jQuery(function() {
			jQuery('#slideshow8').cycle({
				fx:     'scrollHorz',
				timeout: 5000,
				pager:  '#nav8',
				slideExpr: '.group_items',
				speed: 500,
				slideResize: false,
				pause: true
			});
		});
  });
</script>
		</article>
			</div>
<aside id="sidebar">
	<div class="theiaStickySidebar">
<div id="polls-widget-3" class="widget widget_polls-widget"><div class="widget-top"><h4>Encuesta</h4></div>
						<div class="widget-container"><div id="polls-50" class="wp-polls">
	<form id="polls_form_50" class="wp-polls-form" action="/index.php" method="post">
		<p style="display: none;"><input type="hidden" id="poll_50_nonce" name="wp-polls-nonce" value="47b2cd1994" /></p>
		<p style="display: none;"><input type="hidden" name="poll_id" value="50" /></p>
		<p style="display: none;"><input type="hidden" id="poll_multiple_ans_50" name="poll_multiple_ans_50" value="6" /></p>
		<p style="text-align: left"><strong>¿Qué le gusta más del nuevo diseño de Escambray?</strong></p><div id="polls-50-ans" class="wp-polls-ans"><ul class="wp-polls-ul">
		<li><input type="checkbox" id="poll-answer-259" name="poll_50" value="259" /> <label for="poll-answer-259">Adaptabilidad a todo tipo de dispositivos.</label></li>
		<li><input type="checkbox" id="poll-answer-260" name="poll_50" value="260" /> <label for="poll-answer-260">Despliegue de las imágenes.</label></li>
		<li><input type="checkbox" id="poll-answer-261" name="poll_50" value="261" /> <label for="poll-answer-261">Mayor accesibilidad a las redes sociales.</label></li>
		<li><input type="checkbox" id="poll-answer-262" name="poll_50" value="262" /> <label for="poll-answer-262">Facilidades para la lectura en formato web.</label></li>
		<li><input type="checkbox" id="poll-answer-263" name="poll_50" value="263" /> <label for="poll-answer-263">No tengo criterio al respecto.</label></li>
		<li><input type="checkbox" id="poll-answer-264" name="poll_50" value="264" /> <label for="poll-answer-264">Otros.</label></li>
		</ul><p style="text-align: center"><input type="button" name="vote" value="   Vote   " class="Buttons" onclick="poll_vote(50);" /></p><p style="text-align: center"><a href="#ViewPollResults" onclick="poll_result(50); return false;" title="View Results Of This Poll">View Results</a></p></div>
	</form>
</div>
<div id="polls-50-loading" class="wp-polls-loading"><img src="http://www.escambray.cu/wp-content/plugins/wp-polls/images/loading.gif" width="16" height="16" alt="Cargando ..." title="Cargando ..." class="wp-polls-image" />&nbsp;Cargando ...</div>
</div></div><!-- .widget /--><div id="text-html-widget-2" class="widget text-html"><div class="widget-top"><h4> </h4></div>
						<div class="widget-container"><div ><a href="http://www.escambray.cu/category/cartas/"><img class="image " src="http://www.escambray.cu/wp-content/uploads/2014/08/logo-carta.jpg" alt="" width="300" height="56" /></a>
				</div><div class="clear"></div></div></div><!-- .widget /--><div id="authors-posts-widget-2" class="widget authors-posts"><div class="widget-top"><h4> 		</h4></div>
						<div class="widget-container">				<ul>
							<li>
						<h3><a href="http://www.escambray.cu/2018/el-anexo-no-desaparece/">El anexo no desaparece</a></h3>
		</li>
				<li>
						<h3><a href="http://www.escambray.cu/2018/por-que-no-se-pare-en-fomento/">¿Por qué no se pare en Fomento?</a></h3>
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /--><div id="ngg-images-5" class="widget ngg_images"><div class="hslice" id="ngg-webslice"><div class="entry-title widget-top"><h4>Mujeres que tejen historias</h4></div>
						<div class="widget-container"><div class="ngg-widget entry-content">
            <a href="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/mujeres-1-foto-xiomara-alsina.jpg"
           title="La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Xiomara Alsina/ Escambray)"
           data-image-id='3142'
           class='thickbox' rel='76780ffb9449b33f6f14086e6a43eb2a'           ><img title="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Xiomara Alsina/ Escambray)"
                 alt="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Xiomara Alsina/ Escambray)"
                 src="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/thumbs/thumbs_mujeres-1-foto-xiomara-alsina.jpg"
                 width="86"
                 height="59"
            /></a>
            <a href="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/mujeres-5.jpg"
           title="La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
           data-image-id='3146'
           class='thickbox' rel='76780ffb9449b33f6f14086e6a43eb2a'           ><img title="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
                 alt="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
                 src="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/thumbs/thumbs_mujeres-5.jpg"
                 width="86"
                 height="59"
            /></a>
            <a href="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/mujeres-9.jpg"
           title="La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
           data-image-id='3150'
           class='thickbox' rel='76780ffb9449b33f6f14086e6a43eb2a'           ><img title="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
                 alt="Mujeres que tejen historias. La mujer espirituana, protagonista del desarrollo económico y social del territorio. (Foto: Vicente Brito/ Escambray)"
                 src="http://www.escambray.cu/wp-content/gallery/espirituanas-tejen-historias/thumbs/thumbs_mujeres-9.jpg"
                 width="86"
                 height="59"
            /></a>
    </div>

</div></div></div><!-- .widget /--><div id="posts-list-widget-5" class="widget posts-list"><div class="widget-top"><h4>Lo último		</h4></div>
						<div class="widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/turismo-cuba2.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">Novedades turísticas cubanas en Salón Mundial de París</a></h3>
			 <span class="tie-date">17 marzo, 2018</span>		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Moscu-Londres.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">Rusia declara persona non grata a 23 diplomáticos británicos</a></h3>
			 <span class="tie-date">17 marzo, 2018</span>		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/trabajo-de-los-instructores-de-arte-en-sancti-spiritus-foto-vicente-brito.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">Por una mejor brigada</a></h3>
			 <span class="tie-date">17 marzo, 2018</span>		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /--><div id="text-33" class="widget widget_text"><div class="widget-top"><h4> </h4></div>
						<div class="widget-container">			<div class="textwidget"><iframe src="https://www.facebook.com/plugins/page.php?href=https%3A%2F%2Fwww.facebook.com%2Fperiodicosanctispiritus%2F&tabs&width=340&height=214&small_header=false&adapt_container_width=false&hide_cover=false&show_facepile=true&appId" style="border:none;overflow:hidden" scrolling="no" frameborder="0" allowTransparency="true"></iframe></div>
		</div></div><!-- .widget /--><div id="social-2" class="widget social-icons-widget"><div class="widget-top"><h4> </h4></div>
						<div class="widget-container">		<div class="social-icons social-colored">
		<a class="ttip-none" title="Rss" href="http://www.escambray.cu/feed/" ><i class="fa fa-rss"></i></a><a class="ttip-none" title="Facebook" href="http://www.facebook.com/pages/Escambray/205692256170" ><i class="fa fa-facebook"></i></a><a class="ttip-none" title="Twitter" href="http://www.twitter.com/escambraycu" ><i class="fa fa-twitter"></i></a><a class="ttip-none" title="Flickr" href="https://www.flickr.com/photos/144830829@N04/" ><i class="tieicon-flickr"></i></a>
			</div>

</div></div><!-- .widget /-->	</div>
</aside><div class="clear"></div>
<div id="home-multimedia">
<div id="first">
	<ul><ul"><h4 class="home-multimedia-title">Escambray en Youtube</h4><iframe width="854" height="480" src="https://www.youtube.com/embed/WGwzB_6owmE" frameborder="0" allow="autoplay; encrypted-media" allowfullscreen></iframe></ul></ul>
</div>
<div id="second">
	<ul><li id="media_image-2" class="widget widget_media_image"><h4 class="home-multimedia-title">Graficarte</h4><img class="image " src="http://www.escambray.cu/wp-content/uploads/2012/09/ELECCIONES-ok-170x250.jpg" alt="" width="170" height="250" /></li>
</ul>
</div>
<div id="third">
	<ul><li id="custom_html-3" class="widget_text widget widget_custom_html"><h4 class="home-multimedia-title">Edición impresa</h4><div class="textwidget custom-html-widget"><a href="http://www.escambray.cu/edicion-impresa/"  title="Edición impresa" alt="Edición impresa" ><img src="http://www.escambray.cu/wp-content/uploads/2018/03/escambray-17-marzo.jpg" width="200" class="image aligncenter" /></a>
</div></li>
</ul>
</div>
<div style="clear:both;"></div>
</div><div style="clear:both;"></div>  
<footer id="theme-footer">
	<div id="footer-widget-area" class="footer-3c">
			<div id="footer-first" class="footer-widgets-box">
			<div id="categort-posts-widget-3" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Sancti Spíritus		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Cayo-Largo-un-destino-turístico-al-sur-de-Cuba..jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/cubana-vip-en-cayo-largo-del-sur/">Cubana VIP en Cayo Largo del Sur</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Estudiantes-8-1.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/sancti-spiritus-mas-de-3-000-plazas-para-la-educacion-superior/">Sancti Spíritus: Más de 3 000 plazas para la Educación Superior</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
				<div id="footer-second" class="footer-widgets-box">
			<div id="categort-posts-widget-4" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Cuba		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/turismo-cuba2.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/novedades-turisticas-cubanas-en-salon-mundial-de-paris/">Novedades turísticas cubanas en Salón Mundial de París</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/inauguran-en-cuba-embajada-de-kenya-fotos/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Inauguran-en-Cuba-embajada-de-Kenya-foto-pl.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/inauguran-en-cuba-embajada-de-kenya-fotos/">Inauguran en Cuba embajada de Kenya (+fotos)</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
				<div id="footer-third" class="footer-widgets-box">
			<div id="categort-posts-widget-5" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Internacionales		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Moscu-Londres.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/rusia-declara-persona-non-grata-a-23-diplomaticos-britanicos/">Rusia declara persona non grata a 23 diplomáticos británicos</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/asciende-a-seis-los-muertos-por-colapso-de-puente-en-miami-fotos/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/puente-peatonal-colapsa-en-miami-foto-ap.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/asciende-a-seis-los-muertos-por-colapso-de-puente-en-miami-fotos/">Ascienden a seis los muertos por colapso de puente en Miami (+fotos)</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
	
		</div>
	<div class="clear"></div>
</footer>
<footer id="theme-footer">
	<div id="footer-widget-area" class="footer-3c">
			<div id="footer-first" class="footer-widgets-box">
			<div id="categort-posts-widget-6" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Deporte		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/baloncesto-femenino-espirituanas-vs-capitalinas.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/espirituanas-dan-primero-en-final-de-liga-superior-de-baloncesto/">Espirituanas dan primero en final de Liga Superior de Baloncesto</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/serie-provincial-de-beisbol-trinidad-empata-en-la-final-vs-sancti-spiritus/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/Luis-Daniel-Serrano-1-Large.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/serie-provincial-de-beisbol-trinidad-empata-en-la-final-vs-sancti-spiritus/">Serie Provincial de Béisbol: Trinidad empata en la final vs Sancti Spíritus</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
	

			<div id="footer-second" class="footer-widgets-box">
			<div id="categort-posts-widget-7" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Cultura		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/trabajo-de-los-instructores-de-arte-en-sancti-spiritus-foto-vicente-brito.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/por-una-mejor-brigada/">Por una mejor brigada</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/presentan-en-guayos-cabaiguan-libro-sobre-elcire-perez/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/libro-sobre-elcire-perez.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/presentan-en-guayos-cabaiguan-libro-sobre-elcire-perez/">Presentan en Guayos, Cabaiguán, libro sobre Elcire Pérez</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
	
			<div id="footer-third" class="footer-widgets-box">
			<div id="categort-posts-widget-8" class="footer-widget categort-posts"><div class="footer-widget-top"><h4>Ciencia y tecnología		</h4></div>
						<div class="footer-widget-container">				<ul>
							<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/luto-en-el-mundo-de-la-ciencia-por-muerte-de-stephen-hawking/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/stephen-hawking.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/luto-en-el-mundo-de-la-ciencia-por-muerte-de-stephen-hawking/">Luto en el mundo de la ciencia por muerte de Stephen Hawking</a></h3>
			
		</li>
				<li >
									<div class="post-thumbnail">
							<a href="http://www.escambray.cu/2018/crece-la-conexion-desde-la-casa/">
							<img class="attachment-tie-small size-tie-small wp-post-image tie-appear" src="http://www.escambray.cu/wp-content/uploads/2018/03/nauta-hogar-1.jpg" />  <span class="fa overlay-icon"></span>
							</a>
						</div>
						<h3><a href="http://www.escambray.cu/2018/crece-la-conexion-desde-la-casa/">Crece la conexión desde la casa</a></h3>
			
		</li>
						</ul>
		<div class="clear"></div>
	</div></div><!-- .widget /-->		</div>
		
	</div>
	<div class="clear"></div>
</footer>


	
<div class="clear"></div>
	<div class="clear"></div>
</div>	
<div class="clear"></div>
<div class="footer-bottom">
<div class="container">
		<div class="social-icons">
		<a class="ttip-none" title="Rss" href="http://www.escambray.cu/feed/" target="_blank"><i class="fa fa-rss"></i></a><a class="ttip-none" title="Facebook" href="http://www.facebook.com/pages/Escambray/205692256170" target="_blank"><i class="fa fa-facebook"></i></a><a class="ttip-none" title="Twitter" href="http://www.twitter.com/escambraycu" target="_blank"><i class="fa fa-twitter"></i></a><a class="ttip-none" title="Flickr" href="https://www.flickr.com/photos/144830829@N04/" target="_blank"><i class="tieicon-flickr"></i></a>
			</div>

<img border="0" src="http://www.escambray.cu/wp-content/uploads/2014/08/logo-escambray.jpg" width="200">    Fundado el 4 de enero de 1979. Copyright © 2018 / ISSN 9664-1277	
		<div class="clear"></div>
</div>
</div>
</div>
</div>
</div>
	<div id="topcontrol" class="fa fa-angle-up" title="Scroll To Top"></div>
<!-- ngg_resource_manager_marker --><script type='text/javascript'>
/* <![CDATA[ */
var pollsL10n = {"ajax_url":"http:\/\/www.escambray.cu\/wp-admin\/admin-ajax.php","text_wait":"Tu \u00faltima petici\u00f3n est\u00e1 a\u00fan proces\u00e1ndose. Por favor, espera un momento \u2026","text_valid":"Por favor, elige una respuesta v\u00e1lida para la encuesta.","text_multiple":"N\u00famero m\u00e1ximo de opciones permitidas:","show_loading":"1","show_fading":"1"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/wp-polls/polls-js.js' defer onload=''></script>
<script type='text/javascript'>
/* <![CDATA[ */
var tie = {"mobile_menu_active":"true","mobile_menu_top":"true","lightbox_all":"true","lightbox_gallery":"true","lightbox_skin":"dark","lightbox_thumb":"vertical","lightbox_arrows":"","sticky_sidebar":"1","is_singular":"1","SmothScroll":"","reading_indicator":"","lang_no_results":"No Results","lang_results_found":"Results Found"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/themes/escambray/js/tie-scripts.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/themes/escambray/js/ilightbox.packed.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-includes/js/wp-embed.min.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/themes/escambray/js/jquery.cycle.all.js' defer onload=''></script>
<script type='text/javascript'>
/* <![CDATA[ */
var photocrati_ajax = {"url":"http:\/\/www.escambray.cu\/?photocrati_ajax=1","wp_home_url":"http:\/\/www.escambray.cu","wp_site_url":"http:\/\/www.escambray.cu","wp_root_url":"http:\/\/www.escambray.cu","wp_plugins_url":"http:\/\/www.escambray.cu\/wp-content\/plugins","wp_content_url":"http:\/\/www.escambray.cu\/wp-content","wp_includes_url":"http:\/\/www.escambray.cu\/wp-includes\/","ngg_param_slug":"nggallery"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/ajax/static/ajax.min.js' defer onload=''></script>
<script type='text/javascript'>
/* <![CDATA[ */

var galleries = {};
galleries.gallery_76780ffb9449b33f6f14086e6a43eb2a = {"id":null,"source":"random_images","container_ids":["346"],"gallery_ids":[],"album_ids":[],"tag_ids":[],"display_type":"photocrati-nextgen_basic_thumbnails","exclusions":[],"order_by":"sortorder","order_direction":"ASC","image_ids":[],"entity_ids":[],"tagcloud":false,"returns":"included","slug":"widget-ngg-images-5","sortorder":[],"display_settings":{"override_thumbnail_settings":"0","thumbnail_width":"240","thumbnail_height":"160","thumbnail_crop":"1","images_per_page":3,"number_of_columns":"3","ajax_pagination":"1","show_all_in_lightbox":false,"use_imagebrowser_effect":false,"show_slideshow_link":false,"slideshow_link_text":"[Mostrar presentaci\u00f3n de diapositivas]","template":"\/var\/www\/escambray\/Esp\/wp-content\/plugins\/nextgen-gallery\/products\/photocrati_nextgen\/modules\/widget\/templates\/display_gallery.php","use_lightbox_effect":true,"display_no_images_error":1,"disable_pagination":true,"thumbnail_quality":"100","thumbnail_watermark":0,"ngg_triggers_display":"never","maximum_entity_count":3,"image_type":"thumb","show_thumbnail_link":false,"image_width":86,"image_height":59,"widget_setting_title":"Mujeres que tejen historias","widget_setting_before_widget":"<div id=\"ngg-images-5\" class=\"widget ngg_images\"><div class=\"hslice\" id=\"ngg-webslice\">","widget_setting_before_title":"<div class=\"entry-title widget-top\"><h4>","widget_setting_after_widget":"<\/div><\/div><\/div><!-- .widget \/-->","widget_setting_after_title":"<\/h4><\/div>\n\t\t\t\t\t\t<div class=\"widget-container\">","widget_setting_width":86,"widget_setting_height":59,"widget_setting_show_setting":"thumbnail","widget_setting_widget_id":"ngg-images-5"},"excluded_container_ids":[],"maximum_entity_count":3,"__defaults_set":true,"variation":1,"_errors":[],"ID":"76780ffb9449b33f6f14086e6a43eb2a"};
galleries.gallery_76780ffb9449b33f6f14086e6a43eb2a.wordpress_page_root = "http:\/\/www.escambray.cu\/";
var nextgen_lightbox_settings = {"static_path":"http:\/\/www.escambray.cu\/wp-content\/plugins\/nextgen-gallery\/products\/photocrati_nextgen\/modules\/lightbox\/static","context":"nextgen_images"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_gallery_display/static/common.min.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_basic_gallery/static/thumbnails/nextgen_basic_thumbnails.min.js' defer onload=''></script>
<script type='text/javascript'>
/* <![CDATA[ */

var nextgen_thickbox_i18n = {"next":"Next &gt;","prev":"&lt; Prev","image":"Image","of":"De","close":"Close","noiframes":"This feature requires inline frames. You have iframes disabled or your browser does not support them."};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/lightbox/static/lightbox_context.min.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/lightbox/static/thickbox/nextgen_thickbox_init.min.js' defer onload=''></script>
<script type='text/javascript'>
/* <![CDATA[ */
var thickboxL10n = {"next":"Siguiente >","prev":"< Anterior","image":"Imagen","of":"de","close":"Cerrar","noiframes":"Esta funci\u00f3n requiere de frames insertados. Tienes los iframes desactivados o tu navegador no los soporta.","loadingAnimation":"http:\/\/www.escambray.cu\/wp-includes\/js\/thickbox\/loadingAnimation.gif"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.escambray.cu/wp-includes/js/thickbox/thickbox.js' defer onload=''></script>
<script type='text/javascript' src='http://www.escambray.cu/wp-content/plugins/nextgen-gallery/products/photocrati_nextgen/modules/nextgen_basic_gallery/static/thumbnails/ajax_pagination.min.js' defer onload=''></script>
</body>
</html>